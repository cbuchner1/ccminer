/*
 * X17 algorithm built on cbuchner1's original X11
 * 
 */

extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"


#include "miner.h"
}

// aus cpu-miner.c
extern int device_map[8];

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void quark_blake512_cpu_init(int thr_id, int threads);
extern void quark_blake512_cpu_setBlock_80(void *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void quark_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_bmw512_cpu_init(int thr_id, int threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, int threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
//extern void quark_doublegroestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, int threads);
extern void quark_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_keccak512_cpu_init(int thr_id, int threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, int threads);
extern void quark_jh512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_luffa512_cpu_init(int thr_id, int threads);
extern void x11_luffa512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_cubehash512_cpu_init(int thr_id, int threads);
extern void x11_cubehash512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_shavite512_cpu_init(int thr_id, int threads);
extern void x11_shavite512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_simd512_cpu_init(int thr_id, int threads);
extern void x11_simd512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, int threads);
extern void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_hamsi512_cpu_init(int thr_id, int threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, int threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_shabal512_cpu_init(int thr_id, int threads);
extern void x13_shabal512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void whirlpool512_cpu_init(int thr_id, int threads, int flag);
extern void whirlpool512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void sha512_cpu_init(int thr_id, int threads);
extern void sha512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void haval256_cpu_init(int thr_id, int threads);
extern void haval256_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);



extern void quark_check_cpu_init(int thr_id, int threads);
extern void quark_check_cpu_setTarget(const void *ptarget);
extern uint32_t quark_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

extern void quark_compactTest_cpu_init(int thr_id, int threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *inpHashes, 
											uint32_t *d_noncesTrue, size_t *nrmTrue, uint32_t *d_noncesFalse, size_t *nrmFalse,
											int order);

// X13 Hashfunktion
inline void x17hash(void *state, const void *input)
{
    // blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11-hamsi12-fugue13-shabal14-whirlpool15

    sph_blake512_context ctx_blake;
    sph_bmw512_context ctx_bmw;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    sph_luffa512_context ctx_luffa;
    sph_cubehash512_context ctx_cubehash;
    sph_shavite512_context ctx_shavite;
    sph_simd512_context ctx_simd;
    sph_echo512_context ctx_echo;
    sph_hamsi512_context ctx_hamsi;
    sph_fugue512_context ctx_fugue;
	sph_shabal512_context  ctx_shabal;
    sph_whirlpool_context  ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;

    uint32_t hash[16];

    sph_blake512_init(&ctx_blake);
    // ZBLAKE;
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);

    sph_bmw512_init(&ctx_bmw);
    // ZBMW;
    sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
    sph_bmw512_close(&ctx_bmw, (void*) hash);

    sph_groestl512_init(&ctx_groestl);
    // ZGROESTL;
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_skein512_init(&ctx_skein);
    // ZSKEIN;
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    sph_jh512_init(&ctx_jh);
    // ZJH;
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    // ZKECCAK;
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_luffa512_init(&ctx_luffa);
    // ZLUFFA;
    sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
    sph_luffa512_close (&ctx_luffa, (void*) hash);

    sph_cubehash512_init(&ctx_cubehash);
    // ZCUBEHASH;
    sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
    sph_cubehash512_close(&ctx_cubehash, (void*) hash);

    sph_shavite512_init(&ctx_shavite);
    // ZSHAVITE;
    sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
    sph_shavite512_close(&ctx_shavite, (void*) hash);

    sph_simd512_init(&ctx_simd);
    // ZSIMD
    sph_simd512 (&ctx_simd, (const void*) hash, 64);
    sph_simd512_close(&ctx_simd, (void*) hash);

    sph_echo512_init(&ctx_echo);
    // ZECHO
    sph_echo512 (&ctx_echo, (const void*) hash, 64);
    sph_echo512_close(&ctx_echo, (void*) hash); 

    sph_hamsi512_init(&ctx_hamsi);
    sph_hamsi512 (&ctx_hamsi, (const void*) hash, 64);
    sph_hamsi512_close(&ctx_hamsi, (void*) hash); 

    sph_fugue512_init(&ctx_fugue);
    sph_fugue512 (&ctx_fugue, (const void*) hash, 64);
    sph_fugue512_close(&ctx_fugue, (void*) hash); 

	sph_shabal512_init(&ctx_shabal);
    sph_shabal512 (&ctx_shabal, (const void*) hash, 64);
    sph_shabal512_close(&ctx_shabal, (void*) hash); 

    sph_whirlpool_init(&ctx_whirlpool);
    sph_whirlpool (&ctx_whirlpool, (const void*) hash, 64);
    sph_whirlpool_close(&ctx_whirlpool, (void*) hash); 

	
	sph_sha512_init(&ctx_sha512);
    sph_sha512(&ctx_sha512,(const void*) hash, 64);
    sph_sha512_close(&ctx_sha512,(void*)  hash);
	

    sph_haval256_5_init(&ctx_haval);
    sph_haval256_5(&ctx_haval,(const void*) hash, 64);
    sph_haval256_5_close(&ctx_haval,(void*)  hash);

    memcpy(state, hash, 32);
}


extern bool opt_benchmark;

extern "C" int scanhash_x17(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*256*8;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		quark_blake512_cpu_init(thr_id, throughput);

		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x13_shabal512_cpu_init(thr_id, throughput);
		whirlpool512_cpu_init(thr_id, throughput,0);

		sha512_cpu_init(thr_id, throughput);

		haval256_cpu_init(thr_id, throughput);

		quark_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	//unsigned char echobefore[64], echoafter[64];

    uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	quark_blake512_cpu_setBlock_80((void*)endiandata);
	quark_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

        // erstes Blake512 Hash mit CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r BMW512
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Groestl512
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Skein512
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r JH512
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Keccak512
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Luffa512
		x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Cubehash512
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Shavite512
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r SIMD512
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r ECHO512
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

        x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x13_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		
		whirlpool512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		      sha512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		
		haval256_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
	
		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = quark_check_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			x17hash(vhash64, endiandata);

			if( (vhash64[7]<=Htarg) && fulltest(vhash64, ptarget) ) {
                
                pdata[19] = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
