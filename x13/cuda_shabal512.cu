/*
 * Quick and dirty addition of Shabal-512 for X15
 * 
 * Built on cbuchner1's implementation, actual hashing code
 * heavily based on phm's sgminer
 *
 */

/*
 * Shabal-512 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  phm
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdint.h>
#include <memory.h>


#define USE_SHARED 1

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define sM    16

#define C32   SPH_C32
#define T32   SPH_T32

#define O1   13
#define O2    9
#define O3    6


#if __CUDA_ARCH__ < 350 
    // Kepler (Compute 3.0)
    #define SPH_ROTL32(x, n) SPH_T32(((x) << (n)) | ((x) >> (32 - (n))))
#else
    // Kepler (Compute 3.5)
    #define SPH_ROTL32(x, n) __funnelshift_l( (x), (x), (n) )
#endif


// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__constant__ uint32_t C_512[16]; 
static const uint32_t C_init_512[] = {
	C32(0xD9BF68D1), C32(0x58BAD750), C32(0x56028CB2), C32(0x8134F359),
	C32(0xB5D469D8), C32(0x941A8CC2), C32(0x418B2A6E), C32(0x04052780),
	C32(0x7F07D787), C32(0x5194358F), C32(0x3C60D665), C32(0xBE97D79A),
	C32(0x950C3434), C32(0xAED9A06D), C32(0x2537DC8D), C32(0x7CDB5969)
};
__constant__ uint32_t A_512[16];
static const uint32_t A_init_512[] = {
	C32(0x20728DFD), C32(0x46C0BD53), C32(0xE782B699), C32(0x55304632),
	C32(0x71B4EF90), C32(0x0EA9E82C), C32(0xDBB930F1), C32(0xFAD06B8B),
	C32(0xBE0CAE40), C32(0x8BD14410), C32(0x76D2ADAC), C32(0x28ACAB7F)
};
__constant__ uint32_t B_512[16];
static const uint32_t B_init_512[] = {
	C32(0xC1099CB7), C32(0x07B385F3), C32(0xE7442C26), C32(0xCC8AD640),
	C32(0xEB6F56C7), C32(0x1EA81AA9), C32(0x73B9D314), C32(0x1DE85D08),
	C32(0x48910A5A), C32(0x893B22DB), C32(0xC5A0DF44), C32(0xBBC4324E),
	C32(0x72D2F240), C32(0x75941D99), C32(0x6D8BDE82), C32(0xA1A7502B)
};

#define INPUT_BLOCK_ADD    { \
		B0 = T32(B0 + M0); \
		B1 = T32(B1 + M1); \
		B2 = T32(B2 + M2); \
		B3 = T32(B3 + M3); \
		B4 = T32(B4 + M4); \
		B5 = T32(B5 + M5); \
		B6 = T32(B6 + M6); \
		B7 = T32(B7 + M7); \
		B8 = T32(B8 + M8); \
		B9 = T32(B9 + M9); \
		BA = T32(BA + MA); \
		BB = T32(BB + MB); \
		BC = T32(BC + MC); \
		BD = T32(BD + MD); \
		BE = T32(BE + ME); \
		BF = T32(BF + MF); \
	} 

#define INPUT_BLOCK_SUB    { \
		C0 = T32(C0 - M0); \
		C1 = T32(C1 - M1); \
		C2 = T32(C2 - M2); \
		C3 = T32(C3 - M3); \
		C4 = T32(C4 - M4); \
		C5 = T32(C5 - M5); \
		C6 = T32(C6 - M6); \
		C7 = T32(C7 - M7); \
		C8 = T32(C8 - M8); \
		C9 = T32(C9 - M9); \
		CA = T32(CA - MA); \
		CB = T32(CB - MB); \
		CC = T32(CC - MC); \
		CD = T32(CD - MD); \
		CE = T32(CE - ME); \
		CF = T32(CF - MF); \
	} 

#define XOR_W    { \
		A00 ^= Wlow; \
		A01 ^= Whigh; \
	} 

#define SWAP(v1, v2)    { \
		uint32_t tmp = (v1); \
		(v1) = (v2); \
		(v2) = tmp; \
	} 

#define SWAP_BC    { \
		SWAP(B0, C0); \
		SWAP(B1, C1); \
		SWAP(B2, C2); \
		SWAP(B3, C3); \
		SWAP(B4, C4); \
		SWAP(B5, C5); \
		SWAP(B6, C6); \
		SWAP(B7, C7); \
		SWAP(B8, C8); \
		SWAP(B9, C9); \
		SWAP(BA, CA); \
		SWAP(BB, CB); \
		SWAP(BC, CC); \
		SWAP(BD, CD); \
		SWAP(BE, CE); \
		SWAP(BF, CF); \
	} 

#define PERM_ELT(xa0, xa1, xb0, xb1, xb2, xb3, xc, xm)    { \
		xa0 = T32((xa0 \
			^ (((xa1 << 15) | (xa1 >> 17)) * 5U) \
			^ xc) * 3U) \
			^ xb1 ^ (xb2 & ~xb3) ^ xm; \
		xb0 = T32(~(((xb0 << 1) | (xb0 >> 31)) ^ xa0)); \
	} 

#define PERM_STEP_0    { \
		PERM_ELT(A00, A0B, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A01, A00, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A02, A01, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A03, A02, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A04, A03, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A05, A04, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A06, A05, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A07, A06, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A08, A07, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A09, A08, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A0A, A09, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A0B, A0A, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A00, A0B, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A01, A00, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A02, A01, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A03, A02, BF, BC, B8, B5, C9, MF); \
	} 

#define PERM_STEP_1   { \
		PERM_ELT(A04, A03, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A05, A04, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A06, A05, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A07, A06, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A08, A07, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A09, A08, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A0A, A09, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A0B, A0A, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A00, A0B, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A01, A00, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A02, A01, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A03, A02, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A04, A03, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A05, A04, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A06, A05, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A07, A06, BF, BC, B8, B5, C9, MF); \
	} 

#define PERM_STEP_2   { \
		PERM_ELT(A08, A07, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A09, A08, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A0A, A09, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A0B, A0A, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A00, A0B, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A01, A00, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A02, A01, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A03, A02, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A04, A03, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A05, A04, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A06, A05, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A07, A06, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A08, A07, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A09, A08, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A0A, A09, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A0B, A0A, BF, BC, B8, B5, C9, MF); \
	} 

#define APPLY_P    { \
		B0 = T32(B0 << 17) | (B0 >> 15); \
		B1 = T32(B1 << 17) | (B1 >> 15); \
		B2 = T32(B2 << 17) | (B2 >> 15); \
		B3 = T32(B3 << 17) | (B3 >> 15); \
		B4 = T32(B4 << 17) | (B4 >> 15); \
		B5 = T32(B5 << 17) | (B5 >> 15); \
		B6 = T32(B6 << 17) | (B6 >> 15); \
		B7 = T32(B7 << 17) | (B7 >> 15); \
		B8 = T32(B8 << 17) | (B8 >> 15); \
		B9 = T32(B9 << 17) | (B9 >> 15); \
		BA = T32(BA << 17) | (BA >> 15); \
		BB = T32(BB << 17) | (BB >> 15); \
		BC = T32(BC << 17) | (BC >> 15); \
		BD = T32(BD << 17) | (BD >> 15); \
		BE = T32(BE << 17) | (BE >> 15); \
		BF = T32(BF << 17) | (BF >> 15); \
		PERM_STEP_0; \
		PERM_STEP_1; \
		PERM_STEP_2; \
		A0B = T32(A0B + C6); \
		A0A = T32(A0A + C5); \
		A09 = T32(A09 + C4); \
		A08 = T32(A08 + C3); \
		A07 = T32(A07 + C2); \
		A06 = T32(A06 + C1); \
		A05 = T32(A05 + C0); \
		A04 = T32(A04 + CF); \
		A03 = T32(A03 + CE); \
		A02 = T32(A02 + CD); \
		A01 = T32(A01 + CC); \
		A00 = T32(A00 + CB); \
		A0B = T32(A0B + CA); \
		A0A = T32(A0A + C9); \
		A09 = T32(A09 + C8); \
		A08 = T32(A08 + C7); \
		A07 = T32(A07 + C6); \
		A06 = T32(A06 + C5); \
		A05 = T32(A05 + C4); \
		A04 = T32(A04 + C3); \
		A03 = T32(A03 + C2); \
		A02 = T32(A02 + C1); \
		A01 = T32(A01 + C0); \
		A00 = T32(A00 + CF); \
		A0B = T32(A0B + CE); \
		A0A = T32(A0A + CD); \
		A09 = T32(A09 + CC); \
		A08 = T32(A08 + CB); \
		A07 = T32(A07 + CA); \
		A06 = T32(A06 + C9); \
		A05 = T32(A05 + C8); \
		A04 = T32(A04 + C7); \
		A03 = T32(A03 + C6); \
		A02 = T32(A02 + C5); \
		A01 = T32(A01 + C4); \
		A00 = T32(A00 + C3); \
	} 

#define INCR_W   { \
		if ((Wlow = T32(Wlow + 1)) == 0) \
			Whigh = T32(Whigh + 1); \
	} 

                                        
__global__ void x13_shabal512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;


        uint32_t *inpHash = (uint32_t*)&g_hash[8 * hashPosition];
		
			
union {
uint8_t h1[64];
uint32_t h4[16];
uint64_t h8[8];
} hash;  

		
        
	    #pragma unroll 16
		for (int i=0;i<16;i++) {
			hash.h4[i]= inpHash[i];}
		
///////// input big /////////////////////        
		uint32_t A00 = A_512[0], A01 = A_512[1], A02 = A_512[2], A03 = A_512[3], A04 = A_512[4], A05 = A_512[5], A06 = A_512[6], A07 = A_512[7],
	    A08 = A_512[8], A09 = A_512[9], A0A = A_512[10], A0B = A_512[11];
    uint32_t B0 = B_512[0], B1 = B_512[1], B2 = B_512[2], B3 = B_512[3], B4 = B_512[4], B5 = B_512[5], B6 = B_512[6], B7 = B_512[7],
	    B8 = B_512[8], B9 = B_512[9], BA = B_512[10], BB = B_512[11], BC = B_512[12], BD = B_512[13], BE = B_512[14], BF = B_512[15];
    uint32_t C0 = C_512[0], C1 = C_512[1], C2 = C_512[2], C3 = C_512[3], C4 = C_512[4], C5 = C_512[5], C6 = C_512[6], C7 = C_512[7],
	    C8 = C_512[8], C9 = C_512[9], CA = C_512[10], CB = C_512[11], CC = C_512[12], CD = C_512[13], CE = C_512[14], CF = C_512[15];
    uint32_t M0, M1, M2, M3, M4, M5, M6, M7, M8, M9, MA, MB, MC, MD, ME, MF;
    uint32_t Wlow = 1, Whigh = 0;

    M0 = hash.h4[0];
    M1 = hash.h4[1];
    M2 = hash.h4[2];
    M3 = hash.h4[3];
    M4 = hash.h4[4];
    M5 = hash.h4[5];
    M6 = hash.h4[6];
    M7 = hash.h4[7];
    M8 = hash.h4[8];
    M9 = hash.h4[9];
    MA = hash.h4[10];
    MB = hash.h4[11];
    MC = hash.h4[12];
    MD = hash.h4[13];
    ME = hash.h4[14];
    MF = hash.h4[15];

    INPUT_BLOCK_ADD;
    XOR_W;
    APPLY_P;
    INPUT_BLOCK_SUB;
    SWAP_BC;
    INCR_W;

    M0 = 0x80;
    M1 = M2 = M3 = M4 = M5 = M6 = M7 = M8 = M9 = MA = MB = MC = MD = ME = MF = 0;

    INPUT_BLOCK_ADD;
    XOR_W;
    APPLY_P;

 #pragma unroll 3
 for (unsigned i = 0; i < 3; i ++) {
	SWAP_BC;
	XOR_W;
	APPLY_P;
    }

    hash.h4[0] = B0;
    hash.h4[1] = B1;
    hash.h4[2] = B2;
    hash.h4[3] = B3;
    hash.h4[4] = B4;
    hash.h4[5] = B5;
    hash.h4[6] = B6;
    hash.h4[7] = B7;
    hash.h4[8] = B8;
    hash.h4[9] = B9;
    hash.h4[10] = BA;
    hash.h4[11] = BB;
    hash.h4[12] = BC;
    hash.h4[13] = BD;
    hash.h4[14] = BE;
    hash.h4[15] = BF;
      
      #pragma unroll 16
      for (int u = 0; u < 16; u ++) 
            inpHash[u] = hash.h4[u];    
 }
}


void x13_shabal512_cpu_init(int thr_id, int threads)
{
    
	hipMemcpyToSymbol(HIP_SYMBOL(A_512),A_init_512,sizeof(A_init_512),0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(B_512),B_init_512,sizeof(B_init_512),0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(C_512),C_init_512,sizeof(C_init_512),0, hipMemcpyHostToDevice);
}


__host__ void x13_shabal512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{

	const int threadsperblock = 256; // Alignment mit mixtab Gr�sse. NICHT �NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	x13_shabal512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	MyStreamSynchronize(NULL, order, thr_id);
}
