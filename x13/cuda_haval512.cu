#include "hip/hip_runtime.h"
/*
 * Haval-512
 * 
 * Built on cbuchner1's implementation, actual hashing code
 * heavily based on phm's sgminer
 *
 */

/*
 * Haval-512 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <memory.h>


#define USE_SHARED 1

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#include "cuda_helper.h"




// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__constant__ uint32_t c_PaddedMessage80[32];
static __constant__ uint32_t initVector[8];

static const uint32_t c_initVector[8] = {
	SPH_C32(0x243F6A88),
	SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E),
	SPH_C32(0x03707344),
	SPH_C32(0xA4093822),
	SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98),
	SPH_C32(0xEC4E6C89)
};

#define PASS1(n, in)   { \
   STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], SPH_C32(0x00000000)); \
   STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 1], SPH_C32(0x00000000)); \
   STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[ 2], SPH_C32(0x00000000)); \
   STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], SPH_C32(0x00000000)); \
   STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], SPH_C32(0x00000000)); \
   STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[ 5], SPH_C32(0x00000000)); \
   STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[ 6], SPH_C32(0x00000000)); \
   STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[ 7], SPH_C32(0x00000000)); \
 \
   STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 8], SPH_C32(0x00000000)); \
   STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], SPH_C32(0x00000000)); \
   STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[10], SPH_C32(0x00000000)); \
   STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[11], SPH_C32(0x00000000)); \
   STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[12], SPH_C32(0x00000000)); \
   STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[13], SPH_C32(0x00000000)); \
   STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[14], SPH_C32(0x00000000)); \
   STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[15], SPH_C32(0x00000000)); \
 \
   STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[16], SPH_C32(0x00000000)); \
   STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[17], SPH_C32(0x00000000)); \
   STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[18], SPH_C32(0x00000000)); \
   STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[19], SPH_C32(0x00000000)); \
   STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[20], SPH_C32(0x00000000)); \
   STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[21], SPH_C32(0x00000000)); \
   STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[22], SPH_C32(0x00000000)); \
   STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[23], SPH_C32(0x00000000)); \
 \
   STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[24], SPH_C32(0x00000000)); \
   STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[25], SPH_C32(0x00000000)); \
   STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[26], SPH_C32(0x00000000)); \
   STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[27], SPH_C32(0x00000000)); \
   STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[28], SPH_C32(0x00000000)); \
   STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[29], SPH_C32(0x00000000)); \
   STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[30], SPH_C32(0x00000000)); \
   STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[31], SPH_C32(0x00000000)); \
	} 

#define PASS2(n, in)    { \
   STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], SPH_C32(0x452821E6)); \
   STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[14], SPH_C32(0x38D01377)); \
   STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[26], SPH_C32(0xBE5466CF)); \
   STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[18], SPH_C32(0x34E90C6C)); \
   STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[11], SPH_C32(0xC0AC29B7)); \
   STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[28], SPH_C32(0xC97C50DD)); \
   STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 7], SPH_C32(0x3F84D5B5)); \
   STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[16], SPH_C32(0xB5470917)); \
 \
   STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], SPH_C32(0x9216D5D9)); \
   STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[23], SPH_C32(0x8979FB1B)); \
   STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[20], SPH_C32(0xD1310BA6)); \
   STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[22], SPH_C32(0x98DFB5AC)); \
   STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], SPH_C32(0x2FFD72DB)); \
   STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[10], SPH_C32(0xD01ADFB7)); \
   STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 4], SPH_C32(0xB8E1AFED)); \
   STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 8], SPH_C32(0x6A267E96)); \
 \
   STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[30], SPH_C32(0xBA7C9045)); \
   STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], SPH_C32(0xF12C7F99)); \
   STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[21], SPH_C32(0x24A19947)); \
   STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[ 9], SPH_C32(0xB3916CF7)); \
   STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[17], SPH_C32(0x0801F2E2)); \
   STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[24], SPH_C32(0x858EFC16)); \
   STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[29], SPH_C32(0x636920D8)); \
   STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 6], SPH_C32(0x71574E69)); \
 \
   STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[19], SPH_C32(0xA458FEA3)); \
   STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[12], SPH_C32(0xF4933D7E)); \
   STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[15], SPH_C32(0x0D95748F)); \
   STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[13], SPH_C32(0x728EB658)); \
   STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], SPH_C32(0x718BCD58)); \
   STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[25], SPH_C32(0x82154AEE)); \
   STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[31], SPH_C32(0x7B54A41D)); \
   STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[27], SPH_C32(0xC25A59B5)); \
	} 

#define PASS3(n, in)    { \
   STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[19], SPH_C32(0x9C30D539)); \
   STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], SPH_C32(0x2AF26013)); \
   STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 4], SPH_C32(0xC5D1B023)); \
   STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[20], SPH_C32(0x286085F0)); \
   STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[28], SPH_C32(0xCA417918)); \
   STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[17], SPH_C32(0xB8DB38EF)); \
   STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 8], SPH_C32(0x8E79DCB0)); \
   STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[22], SPH_C32(0x603A180E)); \
 \
   STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[29], SPH_C32(0x6C9E0E8B)); \
   STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[14], SPH_C32(0xB01E8A3E)); \
   STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[25], SPH_C32(0xD71577C1)); \
   STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[12], SPH_C32(0xBD314B27)); \
   STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[24], SPH_C32(0x78AF2FDA)); \
   STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[30], SPH_C32(0x55605C60)); \
   STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[16], SPH_C32(0xE65525F3)); \
   STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[26], SPH_C32(0xAA55AB94)); \
 \
   STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[31], SPH_C32(0x57489862)); \
   STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[15], SPH_C32(0x63E81440)); \
   STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 7], SPH_C32(0x55CA396A)); \
   STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], SPH_C32(0x2AAB10B6)); \
   STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], SPH_C32(0xB4CC5C34)); \
   STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[ 0], SPH_C32(0x1141E8CE)); \
   STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[18], SPH_C32(0xA15486AF)); \
   STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[27], SPH_C32(0x7C72E993)); \
 \
   STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[13], SPH_C32(0xB3EE1411)); \
   STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], SPH_C32(0x636FBC2A)); \
   STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[21], SPH_C32(0x2BA9C55D)); \
   STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[10], SPH_C32(0x741831F6)); \
   STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[23], SPH_C32(0xCE5C3E16)); \
   STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[11], SPH_C32(0x9B87931E)); \
   STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 5], SPH_C32(0xAFD6BA33)); \
   STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[ 2], SPH_C32(0x6C24CF5C)); \
	} 

#define PASS4(n, in)  { \
   STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[24], SPH_C32(0x7A325381)); \
   STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 4], SPH_C32(0x28958677)); \
   STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 0], SPH_C32(0x3B8F4898)); \
   STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[14], SPH_C32(0x6B4BB9AF)); \
   STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], SPH_C32(0xC4BFE81B)); \
   STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[ 7], SPH_C32(0x66282193)); \
   STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[28], SPH_C32(0x61D809CC)); \
   STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[23], SPH_C32(0xFB21A991)); \
 \
   STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[26], SPH_C32(0x487CAC60)); \
   STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], SPH_C32(0x5DEC8032)); \
   STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[30], SPH_C32(0xEF845D5D)); \
   STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[20], SPH_C32(0xE98575B1)); \
   STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[18], SPH_C32(0xDC262302)); \
   STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[25], SPH_C32(0xEB651B88)); \
   STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[19], SPH_C32(0x23893E81)); \
   STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 3], SPH_C32(0xD396ACC5)); \
 \
   STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[22], SPH_C32(0x0F6D6FF3)); \
   STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[11], SPH_C32(0x83F44239)); \
   STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[31], SPH_C32(0x2E0B4482)); \
   STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[21], SPH_C32(0xA4842004)); \
   STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 8], SPH_C32(0x69C8F04A)); \
   STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[27], SPH_C32(0x9E1F9B5E)); \
   STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[12], SPH_C32(0x21C66842)); \
   STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 9], SPH_C32(0xF6E96C9A)); \
 \
   STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[ 1], SPH_C32(0x670C9C61)); \
   STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[29], SPH_C32(0xABD388F0)); \
   STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 5], SPH_C32(0x6A51A0D2)); \
   STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[15], SPH_C32(0xD8542F68)); \
   STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[17], SPH_C32(0x960FA728)); \
   STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[10], SPH_C32(0xAB5133A3)); \
   STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[16], SPH_C32(0x6EEF0B6C)); \
   STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[13], SPH_C32(0x137A3BE4)); \
	}

#define PASS5(n, in)    { \
   STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[27], SPH_C32(0xBA3BF050)); \
   STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], SPH_C32(0x7EFB2A98)); \
   STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[21], SPH_C32(0xA1F1651D)); \
   STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[26], SPH_C32(0x39AF0176)); \
   STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[17], SPH_C32(0x66CA593E)); \
   STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[11], SPH_C32(0x82430E88)); \
   STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[20], SPH_C32(0x8CEE8619)); \
   STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[29], SPH_C32(0x456F9FB4)); \
 \
   STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[19], SPH_C32(0x7D84A5C3)); \
   STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 0], SPH_C32(0x3B8B5EBE)); \
   STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[12], SPH_C32(0xE06F75D8)); \
   STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[ 7], SPH_C32(0x85C12073)); \
   STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[13], SPH_C32(0x401A449F)); \
   STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 8], SPH_C32(0x56C16AA6)); \
   STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[31], SPH_C32(0x4ED3AA62)); \
   STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[10], SPH_C32(0x363F7706)); \
 \
   STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], SPH_C32(0x1BFEDF72)); \
   STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], SPH_C32(0x429B023D)); \
   STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[14], SPH_C32(0x37D0D724)); \
   STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[30], SPH_C32(0xD00A1248)); \
   STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[18], SPH_C32(0xDB0FEAD3)); \
   STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 6], SPH_C32(0x49F1C09B)); \
   STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[28], SPH_C32(0x075372C9)); \
   STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[24], SPH_C32(0x80991B7B)); \
 \
   STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 2], SPH_C32(0x25D479D8)); \
   STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[23], SPH_C32(0xF6E8DEF7)); \
   STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[16], SPH_C32(0xE3FE501A)); \
   STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[22], SPH_C32(0xB6794C3B)); \
   STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], SPH_C32(0x976CE0BD)); \
   STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 1], SPH_C32(0x04C006BA)); \
   STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[25], SPH_C32(0xC1A94FB6)); \
   STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[15], SPH_C32(0x409F60C4)); \
	} 

#define F1(x6, x5, x4, x3, x2, x1, x0) \
	(((x1) & ((x0) ^ (x4))) ^ ((x2) & (x5)) ^ ((x3) & (x6)) ^ (x0))


#define F2(x6, x5, x4, x3, x2, x1, x0) \
	(((x2) & (((x1) & ~(x3)) ^ ((x4) & (x5)) ^ (x6) ^ (x0))) \
	^ ((x4) & ((x1) ^ (x5))) ^ ((x3 & (x5)) ^ (x0)))


#define F3(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ (x6) ^ (x0))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ (x0))


#define F4(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ ((x4) | (x6)) ^ (x5))) \
	^ ((x4) & ((~(x2) & (x5)) ^ (x1) ^ (x6) ^ (x0))) \
	^ ((x2) & (x6)) ^ (x0))

#define F5(x6, x5, x4, x3, x2, x1, x0) \
	(((x0) & ~(((x1) & (x2) & (x3)) ^ (x5))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ ((x3) & (x6)))

#define FP5_1(x6, x5, x4, x3, x2, x1, x0) \
	F1(x3, x4, x1, x0, x5, x2, x6)
#define FP5_2(x6, x5, x4, x3, x2, x1, x0) \
	F2(x6, x2, x1, x0, x3, x4, x5)
#define FP5_3(x6, x5, x4, x3, x2, x1, x0) \
	F3(x2, x6, x0, x4, x3, x1, x5)
#define FP5_4(x6, x5, x4, x3, x2, x1, x0) \
	F4(x1, x5, x3, x2, x0, x4, x6)
#define FP5_5(x6, x5, x4, x3, x2, x1, x0) \
	F5(x2, x5, x0, x6, x4, x3, x1)


#define STEP(n, p, x7, x6, x5, x4, x3, x2, x1, x0, w, c)   { \
		uint32_t t = FP ## n ## _ ## p(x6, x5, x4, x3, x2, x1, x0); \
		(x7) = SPH_T32(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w) + (c)); \
	} 

__global__ void m7_haval256_gpu_hash_120(int threads, uint32_t startNounce, uint64_t *outputHash)
{
	

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        
		uint32_t nounce = startNounce + thread;
			
union {
uint32_t h4[16];
uint64_t h8[8];
} hash;  

		
	uint32_t u0, u1, u2, u3, u4, u5, u6, u7; 
	uint32_t s0,s1,s2,s3,s4,s5,s6,s7;
	uint32_t buf[32];
	s0 = initVector[0];
	s1 = initVector[1];
	s2 = initVector[2];
	s3 = initVector[3];
    s4 = initVector[4];
	s5 = initVector[5];
	s6 = initVector[6];
	s7 = initVector[7];

		u0 = s0; 
		u1 = s1; 
		u2 = s2; 
		u3 = s3; 
		u4 = s4; 
		u5 = s5; 
		u6 = s6; 
		u7 = s7; 		
///////// input big /////////////////////        
#pragma unroll 29
		for (int i=0;i<29;i++) {
			buf[i]=c_PaddedMessage80[i];} 
			buf[29]=nounce;
			buf[30]=c_PaddedMessage80[30]+0x00010000;  //need to fix that
			buf[31]=0;
			
			PASS1(5, buf); 
		    PASS2(5, buf); 
		    PASS3(5, buf); 
		    PASS4(5, buf); 
		    PASS5(5, buf); 
		   

		    s0 = sph_t32(s0 + u0); 
		    s1 = sph_t32(s1 + u1); 
		    s2 = sph_t32(s2 + u2); 
		    s3 = sph_t32(s3 + u3); 
		    s4 = sph_t32(s4 + u4); 
		    s5 = sph_t32(s5 + u5); 
		    s6 = sph_t32(s6 + u6); 
		    s7 = sph_t32(s7 + u7); 
		    u0 = s0; 
		    u1 = s1; 
		    u2 = s2; 
		    u3 = s3; 
		    u4 = s4; 
	     	u5 = s5; 
	 	    u6 = s6; 
	    	u7 = s7; 

            
/////////////////////
#pragma unroll 32
		for (int i=0;i<32;i++) {buf[i]=0;}
		    
			buf[29]=0x40290000;
			buf[30]=0x000003d0;

			

			PASS1(5, buf); 
		    PASS2(5, buf); 
		    PASS3(5, buf); 
		    PASS4(5, buf); 
		    PASS5(5, buf); 
		   
			
		    s0 = sph_t32(s0 + u0); 
		    s1 = sph_t32(s1 + u1); 
		    s2 = sph_t32(s2 + u2); 
		    s3 = sph_t32(s3 + u3); 
		    s4 = sph_t32(s4 + u4); 
		    s5 = sph_t32(s5 + u5); 
		    s6 = sph_t32(s6 + u6); 
		    s7 = sph_t32(s7 + u7); 
////////////////////
	        hash.h4[0]=s0;
            hash.h4[1]=s1;
	        hash.h4[2]=s2;
            hash.h4[3]=s3;
	        hash.h4[4]=s4;
            hash.h4[5]=s5;  
	        hash.h4[6]=s6;
            hash.h4[7]=s7;

#pragma unroll 4
for (int i=0;i<4;i++) {outputHash[i*threads+thread]=hash.h8[i];} 
 } // threads
}

__global__ void haval256_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;


        uint32_t *inpHash = (uint32_t*)&g_hash[8 * hashPosition];
		
			
union {
uint8_t h1[64];
uint32_t h4[16];
uint64_t h8[8];
} hash;  

		
	uint32_t u0, u1, u2, u3, u4, u5, u6, u7; 
	uint32_t s0,s1,s2,s3,s4,s5,s6,s7;
	uint32_t buf[32];
	s0 = initVector[0];
	s1 = initVector[1];
	s2 = initVector[2];
	s3 = initVector[3];
    s4 = initVector[4];
	s5 = initVector[5];
	s6 = initVector[6];
	s7 = initVector[7];

		u0 = s0; 
		u1 = s1; 
		u2 = s2; 
		u3 = s3; 
		u4 = s4; 
		u5 = s5; 
		u6 = s6; 
		u7 = s7; 
	
        
	    #pragma unroll 16
		for (int i=0;i<16;i++) {
			hash.h4[i]= inpHash[i];}
		
///////// input big /////////////////////        
#pragma unroll 32
		for (int i=0;i<32;i++) {
			if (i<16) {buf[i]=hash.h4[i];} else {
				       buf[i]=0;}}
		    buf[16]=0x00000001;
			buf[29]=0x40290000;
			buf[30]=0x00000200;
			
			PASS1(5, buf); 
		    PASS2(5, buf); 
		    PASS3(5, buf); 
		    PASS4(5, buf); 
		    PASS5(5, buf); 
		   

		    s0 = sph_t32(s0 + u0); 
		    s1 = sph_t32(s1 + u1); 
		    s2 = sph_t32(s2 + u2); 
		    s3 = sph_t32(s3 + u3); 
		    s4 = sph_t32(s4 + u4); 
		    s5 = sph_t32(s5 + u5); 
		    s6 = sph_t32(s6 + u6); 
		    s7 = sph_t32(s7 + u7); 

	        hash.h4[0]=s0;
            hash.h4[1]=s1;
	        hash.h4[2]=s2;
            hash.h4[3]=s3;
	        hash.h4[4]=s4;
            hash.h4[5]=s5;
	        hash.h4[6]=s6;
            hash.h4[7]=s7;

      #pragma unroll 16
      for (int u = 0; u < 16; u ++) 
            inpHash[u] = hash.h4[u];    
 } // threads
}


void haval256_cpu_init(int thr_id, int threads)
{
    
	
	hipMemcpyToSymbol(HIP_SYMBOL(initVector),c_initVector,sizeof(c_initVector),0, hipMemcpyHostToDevice);
	
}

__host__ void haval256_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{

	const int threadsperblock = 256; // Alignment mit mixtab Gr�sse. NICHT �NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	haval256_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void haval256_setBlock_120(void *pdata)
{
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 122);
	memset(PaddedMessage+122, 0, 6);
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 32*sizeof(uint32_t), 0, hipMemcpyHostToDevice);

}

__host__ void m7_haval256_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{

	const int threadsperblock = 256; // Alignment mit mixtob Gr�sse. NICHT �NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);
//	dim3 grid(1);
//	dim3 block(1);
	size_t shared_size = 0;
	
	m7_haval256_gpu_hash_120<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

