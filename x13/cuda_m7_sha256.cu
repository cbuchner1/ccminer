#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h> 
#include <stdint.h>
#include <memory.h>

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))
#define ROTR    SPH_ROTR32
#include "cuda_helper.h"
#define host_swab32(x)        ( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

 __constant__ uint32_t c_PaddedMessage80[32]; // padded message (80 bytes + padding)
__constant__ uint64_t pTarget[4];
__constant__ uint32_t pbuf[8];
uint32_t *d_mnounce[8];
uint32_t *d_MNonce[8];


static __constant__ uint32_t H256[8];
static __constant__ uint32_t K[64];
// muss expandiert werden
__constant__ uint32_t sha256_gpu_blockHeader[16]; // 2x512 Bit Message
__constant__ uint32_t sha256_gpu_register[8];


static const uint32_t cpu_H256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85), SPH_C32(0x3C6EF372),
	SPH_C32(0xA54FF53A), SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};
static const uint32_t cpu_K[64] = {
	SPH_C32(0x428A2F98), SPH_C32(0x71374491),
	SPH_C32(0xB5C0FBCF), SPH_C32(0xE9B5DBA5),
	SPH_C32(0x3956C25B), SPH_C32(0x59F111F1),
	SPH_C32(0x923F82A4), SPH_C32(0xAB1C5ED5),
	SPH_C32(0xD807AA98), SPH_C32(0x12835B01),
	SPH_C32(0x243185BE), SPH_C32(0x550C7DC3),
	SPH_C32(0x72BE5D74), SPH_C32(0x80DEB1FE),
	SPH_C32(0x9BDC06A7), SPH_C32(0xC19BF174),
	SPH_C32(0xE49B69C1), SPH_C32(0xEFBE4786),
	SPH_C32(0x0FC19DC6), SPH_C32(0x240CA1CC),
	SPH_C32(0x2DE92C6F), SPH_C32(0x4A7484AA),
	SPH_C32(0x5CB0A9DC), SPH_C32(0x76F988DA),
	SPH_C32(0x983E5152), SPH_C32(0xA831C66D),
	SPH_C32(0xB00327C8), SPH_C32(0xBF597FC7),
	SPH_C32(0xC6E00BF3), SPH_C32(0xD5A79147),
	SPH_C32(0x06CA6351), SPH_C32(0x14292967),
	SPH_C32(0x27B70A85), SPH_C32(0x2E1B2138),
	SPH_C32(0x4D2C6DFC), SPH_C32(0x53380D13),
	SPH_C32(0x650A7354), SPH_C32(0x766A0ABB),
	SPH_C32(0x81C2C92E), SPH_C32(0x92722C85),
	SPH_C32(0xA2BFE8A1), SPH_C32(0xA81A664B),
	SPH_C32(0xC24B8B70), SPH_C32(0xC76C51A3),
	SPH_C32(0xD192E819), SPH_C32(0xD6990624),
	SPH_C32(0xF40E3585), SPH_C32(0x106AA070),
	SPH_C32(0x19A4C116), SPH_C32(0x1E376C08),
	SPH_C32(0x2748774C), SPH_C32(0x34B0BCB5),
	SPH_C32(0x391C0CB3), SPH_C32(0x4ED8AA4A),
	SPH_C32(0x5B9CCA4F), SPH_C32(0x682E6FF3),
	SPH_C32(0x748F82EE), SPH_C32(0x78A5636F),
	SPH_C32(0x84C87814), SPH_C32(0x8CC70208),
	SPH_C32(0x90BEFFFA), SPH_C32(0xA4506CEB),
	SPH_C32(0xBEF9A3F7), SPH_C32(0xC67178F2)
};


static __device__ __forceinline__ uint32_t bsg2_0(uint32_t x)
{
	uint32_t r1 = SPH_ROTR32(x,2);
	uint32_t r2 = SPH_ROTR32(x,13);
	uint32_t r3 = SPH_ROTR32(x,22);
	return xor3b(r1,r2,r3); 
}
static __device__ __forceinline__ uint32_t bsg2_1(uint32_t x)
{
	uint32_t r1 = SPH_ROTR32(x,6);
	uint32_t r2 = SPH_ROTR32(x,11);
	uint32_t r3 = SPH_ROTR32(x,25);
	return xor3b(r1,r2,r3);
}
static __device__ __forceinline__ uint32_t ssg2_0(uint32_t x)
{
	uint64_t r1 = SPH_ROTR32(x,7);
	uint64_t r2 = SPH_ROTR32(x,18);
	uint64_t r3 = shr_t32(x,3);
	return xor3b(r1,r2,r3);
}
static __device__ __forceinline__ uint32_t ssg2_1(uint32_t x)
{
	uint64_t r1 = SPH_ROTR32(x,17);
	uint64_t r2 = SPH_ROTR32(x,19);
	uint64_t r3 = shr_t32(x,10);
	return xor3b(r1,r2,r3);
}

static __device__ __forceinline__ void sha2_step1(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t in,const uint32_t Kshared)
{
uint32_t t1,t2;
uint32_t vxandx = xandx(e, f, g);
uint32_t bsg21 =bsg2_1(e);
uint32_t bsg20 =bsg2_0(a);
uint32_t andorv =andor32(a,b,c);

t1 = h + bsg21 + vxandx + Kshared + in; 
t2 = bsg20 + andorv; 
d = d + t1; 
h = t1 + t2; 
}

static __forceinline__ void sha2_step1_host(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t in,const uint32_t Kshared)
{



uint32_t t1,t2;
uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
uint32_t bsg21 =ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
uint32_t bsg20 =ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
uint32_t andorv =((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

t1 = h + bsg21 + vxandx + Kshared + in; 
t2 = bsg20 + andorv; 
d = d + t1; 
h = t1 + t2; 
}

static __device__ __forceinline__ void sha2_step2(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t* in,uint32_t pc,const uint32_t Kshared)
{
uint32_t t1,t2;

int pcidx1 = (pc-2) & 0xF;
int pcidx2 = (pc-7) & 0xF;
int pcidx3 = (pc-15) & 0xF;
uint32_t inx0 = in[pc];
uint32_t inx1 = in[pcidx1];
uint32_t inx2 = in[pcidx2];
uint32_t inx3 = in[pcidx3];


uint32_t ssg21 = ssg2_1(inx1);
uint32_t ssg20 = ssg2_0(inx3);
uint32_t vxandx = xandx(e, f, g);
uint32_t bsg21 =bsg2_1(e);
uint32_t bsg20 =bsg2_0(a);
uint32_t andorv =andor32(a,b,c);

in[pc] = ssg21+inx2+ssg20+inx0;

t1 = h + bsg21 + vxandx + Kshared + in[pc]; 
t2 = bsg20 + andorv; 
d =  d + t1; 
h = t1 + t2; 

}

static __forceinline__ void sha2_step2_host(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t* in,uint32_t pc,const uint32_t Kshared)
{
uint32_t t1,t2;

int pcidx1 = (pc-2) & 0xF;
int pcidx2 = (pc-7) & 0xF;
int pcidx3 = (pc-15) & 0xF;
uint32_t inx0 = in[pc];
uint32_t inx1 = in[pcidx1];
uint32_t inx2 = in[pcidx2];
uint32_t inx3 = in[pcidx3];


uint32_t ssg21 = ROTR(inx1, 17) ^ ROTR(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
uint32_t ssg20 = ROTR(inx3, 7) ^ ROTR(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
uint32_t bsg21 =ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
uint32_t bsg20 =ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
uint32_t andorv =((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

in[pc] = ssg21+inx2+ssg20+inx0;

t1 = h + bsg21 + vxandx + Kshared + in[pc]; 
t2 = bsg20 + andorv; 
d =  d + t1; 
h = t1 + t2; 

}


static __device__ __forceinline__ void sha2_round_body(uint32_t* in, uint32_t* r,const uint32_t* Kshared)
{
		
		
		uint32_t a=r[0];
        uint32_t b=r[1];
        uint32_t c=r[2];
        uint32_t d=r[3];
        uint32_t e=r[4];
        uint32_t f=r[5];
        uint32_t g=r[6];
        uint32_t h=r[7];
			
		sha2_step1(a,b,c,d,e,f,g,h,in[0],Kshared[0]);
		sha2_step1(h,a,b,c,d,e,f,g,in[1],Kshared[1]);
		sha2_step1(g,h,a,b,c,d,e,f,in[2],Kshared[2]);
		sha2_step1(f,g,h,a,b,c,d,e,in[3],Kshared[3]);
		sha2_step1(e,f,g,h,a,b,c,d,in[4],Kshared[4]);
		sha2_step1(d,e,f,g,h,a,b,c,in[5],Kshared[5]);
		sha2_step1(c,d,e,f,g,h,a,b,in[6],Kshared[6]);
		sha2_step1(b,c,d,e,f,g,h,a,in[7],Kshared[7]);
		sha2_step1(a,b,c,d,e,f,g,h,in[8],Kshared[8]);
		sha2_step1(h,a,b,c,d,e,f,g,in[9],Kshared[9]);
		sha2_step1(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
		sha2_step1(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
		sha2_step1(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
		sha2_step1(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
		sha2_step1(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
		sha2_step1(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

#pragma unroll 3
		for (int i=0;i<3;i++) {

		sha2_step2(a,b,c,d,e,f,g,h,in,0,Kshared[16+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,1,Kshared[17+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,2,Kshared[18+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,3,Kshared[19+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,4,Kshared[20+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,5,Kshared[21+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,6,Kshared[22+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,7,Kshared[23+16*i]);
		sha2_step2(a,b,c,d,e,f,g,h,in,8,Kshared[24+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,9,Kshared[25+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);

		}
		
		

		 r[0] = r[0] + a;
		 r[1] = r[1] + b;
		 r[2] = r[2] + c;
		 r[3] = r[3] + d;
		 r[4] = r[4] + e;
		 r[5] = r[5] + f;
		 r[6] = r[6] + g;
		 r[7] = r[7] + h;
}

static __forceinline__ void sha2_round_body_host(uint32_t* in, uint32_t* r,const uint32_t* Kshared)
{
		
		
		uint32_t a=r[0];
        uint32_t b=r[1];
        uint32_t c=r[2];
        uint32_t d=r[3];
        uint32_t e=r[4];
        uint32_t f=r[5];
        uint32_t g=r[6];
        uint32_t h=r[7];
			
		sha2_step1_host(a,b,c,d,e,f,g,h,in[0],Kshared[0]);
		sha2_step1_host(h,a,b,c,d,e,f,g,in[1],Kshared[1]);
		sha2_step1_host(g,h,a,b,c,d,e,f,in[2],Kshared[2]);
		sha2_step1_host(f,g,h,a,b,c,d,e,in[3],Kshared[3]);
		sha2_step1_host(e,f,g,h,a,b,c,d,in[4],Kshared[4]);
		sha2_step1_host(d,e,f,g,h,a,b,c,in[5],Kshared[5]);
		sha2_step1_host(c,d,e,f,g,h,a,b,in[6],Kshared[6]);
		sha2_step1_host(b,c,d,e,f,g,h,a,in[7],Kshared[7]);
		sha2_step1_host(a,b,c,d,e,f,g,h,in[8],Kshared[8]);
		sha2_step1_host(h,a,b,c,d,e,f,g,in[9],Kshared[9]);
		sha2_step1_host(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
		sha2_step1_host(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
		sha2_step1_host(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
		sha2_step1_host(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
		sha2_step1_host(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
		sha2_step1_host(b,c,d,e,f,g,h,a,in[15],Kshared[15]);


		for (int i=0;i<3;i++) {

		sha2_step2_host(a,b,c,d,e,f,g,h,in,0,Kshared[16+16*i]);
		sha2_step2_host(h,a,b,c,d,e,f,g,in,1,Kshared[17+16*i]);
		sha2_step2_host(g,h,a,b,c,d,e,f,in,2,Kshared[18+16*i]);
		sha2_step2_host(f,g,h,a,b,c,d,e,in,3,Kshared[19+16*i]);
		sha2_step2_host(e,f,g,h,a,b,c,d,in,4,Kshared[20+16*i]);
		sha2_step2_host(d,e,f,g,h,a,b,c,in,5,Kshared[21+16*i]);
		sha2_step2_host(c,d,e,f,g,h,a,b,in,6,Kshared[22+16*i]);
		sha2_step2_host(b,c,d,e,f,g,h,a,in,7,Kshared[23+16*i]);
		sha2_step2_host(a,b,c,d,e,f,g,h,in,8,Kshared[24+16*i]);
		sha2_step2_host(h,a,b,c,d,e,f,g,in,9,Kshared[25+16*i]);
		sha2_step2_host(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha2_step2_host(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha2_step2_host(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha2_step2_host(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha2_step2_host(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha2_step2_host(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);

		}

		 r[0] = r[0] + a;
		 r[1] = r[1] + b;
		 r[2] = r[2] + c;
		 r[3] = r[3] + d;
		 r[4] = r[4] + e;
		 r[5] = r[5] + f;
		 r[6] = r[6] + g;
		 r[7] = r[7] + h;
}


__global__ void __launch_bounds__(512,1) m7_sha256_gpu_hash_120(int threads, uint32_t startNounce, uint64_t *outputHash)
{

   
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {

		uint32_t nounce = startNounce +  thread ; // original implementation

        uint32_t buf[8];
		uint32_t in2[16]={0};
		uint32_t in3[16]={0};

        #pragma unroll 13
		for (int i=0;i<13;i++) {in2[i]= cuda_swab32(c_PaddedMessage80[i+16]);}
		in2[13]=cuda_swab32(nounce);
		in2[14]=cuda_swab32(c_PaddedMessage80[30]);

		                        in3[15]=0x3d0;
          
        #pragma unroll 8
		for (int i=0;i<8;i++) {buf[i]= pbuf[i];}    

                    sha2_round_body(in2,buf,K);
					sha2_round_body(in3,buf,K);

#pragma unroll 4
for (int i=0;i<4;i++) {outputHash[i*threads+thread]=cuda_swab32ll(((uint64_t*)buf)[i]);}


//////////////////////////////////////////////////////////////////////////////////////////////////	  
	} // threads

}


__global__ void  m7_sha256_gpu_hash_300(int threads, uint32_t startNounce, uint64_t *g_hash1, uint64_t *g_nonceVector, uint32_t *resNounce)
{
/*	
	__shared__ uint32_t Kshared[64];
	if (threadIdx.x < 64) {
		Kshared[threadIdx.x]=K[threadIdx.x];
	}
	__syncthreads();
*/
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {

        
     
		
union {
uint8_t h1[304];
uint32_t h4[76];
uint64_t h8[38];
} hash;  


        uint32_t in[16],buf[8];

		
		#pragma unroll 8
		for (int i=0;i<8;i++) {((uint64_t*)in)[i]= cuda_swab32ll(g_hash1[threads*i+thread]);}
        #pragma unroll 8
		for (int i=0;i<8;i++) {buf[i] = H256[i];}    

		sha2_round_body(in,buf,K);

		#pragma unroll 8
		for (int i=0;i<8;i++) {((uint64_t*)in)[i]= cuda_swab32ll(g_hash1[threads*(i+8)+thread]);}
		sha2_round_body(in,buf,K);

		#pragma unroll 8
		for (int i=0;i<8;i++) {((uint64_t*)in)[i]= cuda_swab32ll(g_hash1[threads*(i+16)+thread]);}
		sha2_round_body(in,buf,K);

		#pragma unroll 8
		for (int i=0;i<8;i++) {((uint64_t*)in)[i]= cuda_swab32ll(g_hash1[threads*(i+24)+thread]);}
		sha2_round_body(in,buf,K);

		#pragma unroll 5
		for (int i=0;i<5;i++) {((uint64_t*)in)[i]= cuda_swab32ll(g_hash1[threads*(i+32)+thread]);}
		((uint64_t*)in)[5]= g_hash1[threads*(5+32)+thread];
		in[11]=0;
		in[12]=0;
		in[13]=0;
		in[14]=0;


                   in[15]=0x968;
				   
                   int it=0;				
				   do {
                   in[15]-=8;
				   it++;
				   }  while (((uint8_t*)in)[44-it]==0);
				   ((uint8_t*)in)[44-it+1]=0x80;
		
           ((uint64_t*)in)[5]= cuda_swab32ll(((uint64_t*)in)[5]);

				   sha2_round_body(in,buf,K);

uint32_t nounce = startNounce +thread;
		bool rc = true;


    if (cuda_swab32ll(((uint64_t*)buf)[3]) > pTarget[3]) {rc = false;} 
//// only needed for solo mining, commenting it out will probably increased rejected block (no big deal actually)
	/*
	else if (cuda_swab32ll(((uint64_t*)buf)[3]) == pTarget[3]) {  // in case ptarget=buf=0
		          if (cuda_swab32ll(((uint64_t*)buf)[2]) > pTarget[2]) {rc = false;} 
	         else if (cuda_swab32ll(((uint64_t*)buf)[2]) == pTarget[2]) {
				         if (cuda_swab32ll(((uint64_t*)buf)[1]) > pTarget[1]) {rc = false;} 
	                     else if (cuda_swab32ll(((uint64_t*)buf)[1]) == pTarget[1]) {
				                  if (cuda_swab32ll(((uint64_t*)buf)[0]) > pTarget[0]) {rc = false;} 
								  else if (cuda_swab32ll(((uint64_t*)buf)[0]) == pTarget[0]) {rc = true;}
						 }}}
      */      
	
	

		if(rc == true)
		{
			if(resNounce[0] > nounce)
				resNounce[0] = nounce;

		}


////
	} // threads
}



__host__ void m7_sha256_cpu_init(int thr_id, int threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL(	H256),cpu_H256,sizeof(cpu_H256),0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(	K),cpu_K,sizeof(cpu_K),0, hipMemcpyHostToDevice );
	hipMalloc(&d_MNonce[thr_id], sizeof(uint32_t)); 
	hipHostMalloc(&d_mnounce[thr_id], 1*sizeof(uint32_t));
}


__host__  uint32_t m7_sha256_cpu_hash_300(int thr_id, int threads, uint32_t startNounce, uint64_t *d_nonceVector,uint64_t *d_hash, int order)
{
	
	uint32_t result = 0xffffffff;
	hipMemset(d_MNonce[thr_id], 0xff, sizeof(uint32_t));
	//const int threadsperblock = 384; // Alignment mit mixtob Gr�sse. NICHT �NDERN
	const int threadsperblock = 512;
	
	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;
	
	
	m7_sha256_gpu_hash_300<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector, d_MNonce[thr_id]);
	hipMemcpy(d_mnounce[thr_id], d_MNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	MyStreamSynchronize(NULL, order, thr_id);
	result = *d_mnounce[thr_id];
	return result;
}


__host__ void m7_sha256_cpu_hash_120(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{

	const int threadsperblock = 512; // Alignment mit mixtob Gr�sse. NICHT �NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock); 
//	dim3 grid(1);
//	dim3 block(1);
	size_t shared_size = 0;
	
	m7_sha256_gpu_hash_120<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void m7_sha256_setBlock_120(void *pdata,const void *ptarget)  //not useful
{
	unsigned char PaddedMessage[128];
	uint8_t ending =0x80;
	memcpy(PaddedMessage, pdata, 122);
	memset(PaddedMessage+122,ending,1); 
	memset(PaddedMessage+123, 0, 5); //useless
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL( pTarget), ptarget, 4*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	/// do first loop here... ///
    
	uint32_t * alt_data = (uint32_t*) PaddedMessage; 
	uint32_t in[16],buf[8];
	for (int i=0;i<16;i++) {in[i]= host_swab32(alt_data[i]);}
	for (int i=0;i<8;i++) {buf[i]= cpu_H256[i];}     
			                sha2_round_body_host(in,buf,cpu_K);
    hipMemcpyToSymbol(HIP_SYMBOL( pbuf), buf, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}
