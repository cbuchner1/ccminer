/*
 * Goalcoin
 * 
 */

extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"

#include "miner.h"
}

// aus cpu-miner.c
extern int device_map[8];

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void quark_blake512_cpu_init(int thr_id, int threads);
extern void quark_blake512_cpu_setBlock_80(void *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void quark_blake512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_bmw512_cpu_init(int thr_id, int threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, int threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
//extern void quark_doublegroestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, int threads);
extern void quark_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_keccak512_cpu_init(int thr_id, int threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, int threads);
extern void quark_jh512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_luffa512_cpu_init(int thr_id, int threads);
extern void x11_luffa512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_cubehash512_cpu_init(int thr_id, int threads);
extern void x11_cubehash512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_shavite512_cpu_init(int thr_id, int threads);
extern void x11_shavite512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_simd512_cpu_init(int thr_id, int threads);
extern void x11_simd512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, int threads);
extern void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_hamsi512_cpu_init(int thr_id, int threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, int threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_shabal512_cpu_init(int thr_id, int threads);
extern void x13_shabal512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void whirlpool512_cpu_init(int thr_id, int threads,int flag);
extern void whirlpool512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern uint32_t whirlpool512_cpu_finalhash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);



// goalcoin hash function
inline void goalhash(void *state, const void *input)
{
    // blake-groestl-jh-keccak-skein-whirlpool

    sph_blake512_context ctx_blake;
    
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    sph_whirlpool_context  ctx_whirlpool;

    uint32_t hash[16];

    sph_blake512_init(&ctx_blake);
    // ZBLAKE;
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);

    

    sph_groestl512_init(&ctx_groestl);
    // ZGROESTL;
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    // ZJH;
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    // ZKECCAK;
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    // ZSKEIN;
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    sph_whirlpool_init(&ctx_whirlpool);
    sph_whirlpool (&ctx_whirlpool, (const void*) hash, 64);
    sph_whirlpool_close(&ctx_whirlpool, (void*) hash); 


    memcpy(state, hash, 32);
}


extern bool opt_benchmark;

extern "C" int scanhash_goal(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*256*8;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);		
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		whirlpool512_cpu_init(thr_id, throughput,0);


		init[thr_id] = true;
	}

	//unsigned char echobefore[64], echoafter[64];

    uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	quark_blake512_cpu_setBlock_80((void*)endiandata);
	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		int order = 0;

        // erstes Blake512 Hash mit CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Groestl512
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r JH512
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Keccak512
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
         // das ist der unbedingte Branch f�r Skein512
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);		
			
		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			goalhash(vhash64, endiandata);

			if( (vhash64[7]<=Htarg) && fulltest(vhash64, ptarget) ) {
                
                pdata[19] = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
