/*
 * whirlpool routine for new algorithm
 * 
 */

extern "C"
{
#include "sph/sph_whirlpool.h"

#include "miner.h"
}

// aus cpu-miner.c
extern int device_map[8];

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void whirlpool512_cpu_init(int thr_id, int threads, int flag);
extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void whirlpool512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern uint32_t whirlpool512_cpu_finalhash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_check_cpu_init(int thr_id, int threads);
extern void quark_check_cpu_setTarget(const void *ptarget);

extern uint32_t quark_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

// fresh Hashfunktion
inline void wh_hash(void *state, const void *input)
{
    // shavite-simd-shavite-simd-echo

    
    sph_whirlpool_context ctx_whirlpool;
    
	
    uint32_t hash[16];

	// shavite 1
    sph_whirlpool1_init(&ctx_whirlpool);
    sph_whirlpool1 (&ctx_whirlpool, input, 80);
    sph_whirlpool1_close(&ctx_whirlpool, (void*) hash);
	

	sph_whirlpool1_init(&ctx_whirlpool);
    sph_whirlpool1 (&ctx_whirlpool, (const void*) hash, 64);
    sph_whirlpool1_close(&ctx_whirlpool, (void*) hash); 
	
	sph_whirlpool1_init(&ctx_whirlpool);
    sph_whirlpool1 (&ctx_whirlpool, (const void*) hash, 64);
    sph_whirlpool1_close(&ctx_whirlpool, (void*) hash); 

	sph_whirlpool1_init(&ctx_whirlpool);
    sph_whirlpool1 (&ctx_whirlpool, (const void*) hash, 64);
    sph_whirlpool1_close(&ctx_whirlpool, (void*) hash); 

    memcpy(state, hash, 32);
}


extern bool opt_benchmark;

extern "C" int scanhash_wh(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*256*8*4;
	
	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		whirlpool512_cpu_init(thr_id, throughput,1);
		
//		quark_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	//unsigned char echobefore[64], echoafter[64];

    uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);	}
	whirlpool512_setBlock_80((void*)endiandata, ptarget);
//	quark_check_cpu_setTarget(ptarget);
	do {
		int order = 0;
		 
		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);		  		
		whirlpool512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);		
		whirlpool512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);		
		uint32_t foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);

			wh_hash(vhash64, endiandata);

			if( (vhash64[7]<=Htarg) && fulltest(vhash64, ptarget) ) {
                
                pdata[19] = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU! vhash64 %08x and htarg %08x", thr_id, foundNonce,vhash64[7],Htarg);
			}
		}
		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
