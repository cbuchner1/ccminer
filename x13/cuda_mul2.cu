#include "hip/hip_runtime.h"
/*
 * sha256 djm34, catia
 * 
 */

/*
 * sha-256 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */

#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>


#include "cuda_helper.h"

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);



typedef struct t4_t{
	uint64_t high,low;
} t4_t;

__device__ __forceinline__ 
ulonglong2 umul64wide (unsigned long long int a, 
                       unsigned long long int b)
{
    ulonglong2 res;
    asm ("{\n\t"
         ".reg .u32 r0, r1, r2, r3, alo, ahi, blo, bhi;\n\t"
         "mov.b64         {alo,ahi}, %2;   \n\t"
         "mov.b64         {blo,bhi}, %3;   \n\t"
         "mul.lo.u32      r0, alo, blo;    \n\t"
         "mul.hi.u32      r1, alo, blo;    \n\t"
         "mad.lo.cc.u32   r1, alo, bhi, r1;\n\t"
         "madc.hi.u32     r2, alo, bhi,  0;\n\t"
         "mad.lo.cc.u32   r1, ahi, blo, r1;\n\t"
         "madc.hi.cc.u32  r2, ahi, blo, r2;\n\t"
         "madc.hi.u32     r3, ahi, bhi,  0;\n\t"
         "mad.lo.cc.u32   r2, ahi, bhi, r2;\n\t"
         "addc.u32        r3, r3,  0;      \n\t"
         "mov.b64         %0, {r0,r1};     \n\t"  
         "mov.b64         %1, {r2,r3};     \n\t"
         "}"
         : "=l"(res.x), "=l"(res.y)
         : "l"(a), "l"(b));
    return res;
}

#define umul_ppmm(h,l,m,n) \
{ \
	ulonglong2 foom = umul64wide(m,n); \
	h = foom.y; \
	l = foom.x; \
}


__device__ __forceinline__ void umul_ppmmT4(t4_t *h, t4_t *l, t4_t m, t4_t n)
{
    asm ("{\n\t"
         ".reg .u32 o0, o1, o2, o3, o4;    \n\t"
         ".reg .u32 o5, o6, o7, i8, i9;    \n\t"
         ".reg .u32 i10, i11, i12, i13;    \n\t"
         ".reg .u32 i14, i15, i16, i17;    \n\t"
         ".reg .u32 i18, i19, i20, i21;    \n\t"
         ".reg .u32 i22, i23;              \n\t"
         "mov.b64         { i8, i9}, %4;   \n\t"
         "mov.b64         {i10,i11}, %5;   \n\t"
         "mov.b64         {i12,i13}, %6;   \n\t"
         "mov.b64         {i14,i15}, %7;   \n\t"
         "mov.b64         {i16,i17}, %8;   \n\t"
         "mov.b64         {i18,i19}, %9;   \n\t"
         "mov.b64         {i20,i21},%10;   \n\t"
         "mov.b64         {i22,i23},%11;   \n\t"
         "mul.lo.u32      o0,  i8, i16;    \n\t"
         "mul.hi.u32      o1,  i8, i16;    \n\t"
         "mad.lo.cc.u32   o1,  i8, i17, o1;\n\t"
         "madc.hi.u32     o2,  i8, i17,  0;\n\t"
         "mad.lo.cc.u32   o1,  i9, i16, o1;\n\t"
         "madc.hi.cc.u32  o2,  i9, i16, o2;\n\t"
         "madc.hi.u32     o3,  i8, i18,  0;\n\t"
         "mad.lo.cc.u32   o2,  i8, i18, o2;\n\t"
         "madc.hi.cc.u32  o3,  i9, i17, o3;\n\t"
         "madc.hi.u32     o4,  i8, i19,  0;\n\t"
         "mad.lo.cc.u32   o2,  i9, i17, o2;\n\t"
         "madc.hi.cc.u32  o3, i10, i16, o3;\n\t"
         "madc.hi.cc.u32  o4,  i9, i18, o4;\n\t"
         "addc.u32        o5,   0,   0;\n\t"
         "mad.lo.cc.u32   o2, i10, i16, o2;\n\t"
	 "madc.lo.cc.u32  o3,  i8, i19, o3;\n\t"
         "madc.hi.cc.u32  o4, i10, i17, o4;\n\t"
         "madc.hi.cc.u32  o5,  i9, i19, o5;\n\t"
         "addc.u32        o6,   0,   0;\n\t"
         "mad.lo.cc.u32   o3,  i9, i18, o3;\n\t"
         "madc.hi.cc.u32  o4, i11, i16, o4;\n\t"
         "madc.hi.cc.u32  o5, i10, i18, o5;\n\t"
         "addc.u32        o6,   0,  o6;\n\t"
         "mad.lo.cc.u32   o3, i10, i17, o3;\n\t"
         "addc.u32        o4,   0,  o4;\n\t"
         "mad.hi.cc.u32   o5, i11, i17, o5;\n\t"
         "madc.hi.cc.u32  o6, i10, i19, o6;\n\t"
         "addc.u32        o7,   0,   0;\n\t"
         "mad.lo.cc.u32   o3, i11, i16, o3;\n\t"
         "madc.lo.cc.u32  o4,  i9, i19, o4;\n\t"
         "addc.u32        o5,   0,  o5;\n\t"
         "mad.hi.cc.u32   o6, i11, i18, o6;\n\t"
         "addc.u32        o7,   0,  o7;\n\t"
         "mad.lo.cc.u32   o4, i10, i18, o4;\n\t"
         "addc.u32        o5,   0,  o5;\n\t"
         "mad.hi.u32      o7, i11, i19, o7;\n\t"
         "mad.lo.cc.u32   o4, i11, i17, o4;\n\t"
         "addc.u32        o5,   0,  o5;\n\t"
         "mad.lo.cc.u32   o5, i10, i19, o5;\n\t"
         "addc.u32        o6,   0,  o6;\n\t"
         "mad.lo.cc.u32   o5, i11, i18, o5;\n\t"
         "addc.u32        o6,   0,  o6;\n\t"
         "mad.lo.cc.u32   o6, i11, i19, o6;\n\t"
         "addc.u32        o7,   0,  o7;\n\t"
         "mov.b64         %0, {o0,o1};     \n\t"
         "mov.b64         %1, {o2,o3};     \n\t"
         "mov.b64         %2, {o4,o5};     \n\t"
         "mov.b64         %3, {o6,o7};     \n\t"
         "}"
         : "=l"(l->low), "=l"(l->high), "=l"(h->low), "=l"(h->high)
         : "l"(m.low), "l"(m.high), "l"(0ULL), "l"(0ULL),
           "l"(n.low), "l"(n.high), "l"(0ULL), "l"(0ULL));
}

#if 0
__device__ __forceinline__ void umul_ppmmT4(t4_t *h, t4_t *l, t4_t m, t4_t n){
	uint64_t th,tl;
	uint32_t c,c2;
	umul_ppmm(l->high,l->low,m.low,n.low);

	umul_ppmm(th,tl,m.high,n.low);
	l->high += tl;
	c = (l->high < tl);
	h->low = th + c;
	c = (h->low < c);
	h->high = c;

	//Second word
	umul_ppmm(th,tl,m.low,n.high);
	l->high += tl;
	c = l->high < tl;
	h->low += th;
	c2 = h->low < th;
	h->low += c;
	c2 += h->low < c;
	h->high += c2;

	umul_ppmm(th,tl,m.high,n.high);
	h->low += tl;
	c = h->low < tl;
	h->high += th + c;
}
#endif


__device__ __forceinline__ t4_t T4(uint32_t thread, uint32_t threads, uint32_t idx, uint64_t *g){
	t4_t ret;
	ret.high = g[(idx*2 + 1)*threads + thread];
	ret.low = g[(idx*2)*threads + thread];

	

	return ret;
}

__device__ __forceinline__ void T4_store(uint32_t thread, uint32_t threads, uint32_t idx, uint64_t *g, t4_t val){
	g[(idx*2 + 1)*threads + thread]=val.high;
	g[(idx*2)*threads + thread]=val.low;

	

}

__device__ __forceinline__ void T4_set(t4_t *d, uint64_t v){
	d->high = 0;
	d->low = v;
}

__device__ __forceinline__ t4_t T4_add(t4_t a, t4_t b){
	t4_t ret;
	uint32_t c=0;
	ret.low = a.low + b.low;
	if(ret.low < a.low)
	    c=1;
	ret.high = a.high + b.high + c;
	return ret;
}

__device__ __forceinline__ t4_t T4_add(uint64_t a, t4_t b){
	t4_t ret;
	uint32_t c=0;
	ret.low = a + b.low;
	if(ret.low < a)
	    c=1;
	ret.high = b.high + c;
	return ret;
}


__device__ __forceinline__ uint32_t T4_lt(t4_t a, t4_t b){
	if(a.high < b.high)
		return 1;
	if(a.high == b.high && a.low < b.low)
		return 1;
	return 0;
}

__device__ __forceinline__ uint32_t T4_gt(t4_t a, uint64_t b){
	if(a.high)
		return 1;
	if(a.low > b)
		return 1;
	return 0;
}


__device__ void mulScalarT4(uint32_t thread, uint32_t threads, uint32_t len, uint64_t* g_p, uint64_t* g_v, t4_t sml, uint32_t *size){
  t4_t ul, cl, hpl, lpl;
  uint32_t i;
  T4_set(&cl,0);
  for(i=0; i < len; i++) {
      ul = T4(thread,threads,i,g_v);
      umul_ppmmT4 (&hpl, &lpl, ul, sml);

      lpl = T4_add(lpl,cl);
      cl = T4_add(T4_lt(lpl,cl),hpl);

      T4_store(thread,threads,i,g_p,lpl);
    }

    T4_store(thread,threads,len,g_p,cl);
    *size = len + T4_gt(cl,0);
}


__device__ void mulScalar(uint32_t thread, uint32_t threads, uint32_t len, uint64_t* g_p, uint64_t* g_v, uint64_t sml, uint32_t *size){
  uint64_t ul, cl, hpl, lpl;
  uint32_t i;
  cl = 0;
  for(i=0; i < len; i++) {
      ul = g_v[i*threads + thread];
      umul_ppmm (hpl, lpl, ul, sml);

      lpl += cl;
      cl = (lpl < cl) + hpl;

      g_p[i*threads + thread] = lpl;
    }

    g_p[len*threads + thread] = cl;
    *size = len + (cl != 0);
}

uint64_t __device__ addmul_1g (uint32_t thread, uint32_t threads, uint64_t *sum, uint32_t sofst, uint64_t *x, uint64_t xsz, uint64_t a){
	uint64_t carry=0;
	uint32_t i;
	uint64_t ul,lpl,hpl,rl;

	for(i=0; i < xsz; i++){
		
      		ul = x[i*threads + thread];
      		umul_ppmm (hpl, lpl, ul, a);

      		lpl += carry;
      		carry = (lpl < carry) + hpl;

      		rl = sum[(i+sofst) * threads + thread];
      		lpl = rl + lpl;
      		carry += lpl < rl;
      		sum[(i+sofst)*threads + thread] = lpl;
    	}

  	return carry;
}

t4_t __device__ addmul_1gT4 (uint32_t thread, uint32_t threads, uint64_t *sum, uint32_t sofst, uint64_t *x, uint64_t xsz, t4_t a){
	t4_t carry;
	uint32_t i;
	t4_t ul,lpl,hpl,rl;
	T4_set(&carry,0);
	for(i=0; i < xsz; i++){
		
      		ul = T4(thread,threads,i,x);
      		umul_ppmmT4 (&hpl, &lpl, ul, a);

      		lpl = T4_add(lpl,carry);
      		carry = T4_add(T4_lt(lpl,carry), hpl);

      		rl = T4(thread,threads,i+sofst,sum);
      		lpl = T4_add(rl,lpl);
      		carry = T4_add(T4_lt(lpl,rl),carry);
      		T4_store(thread,threads,i+sofst,sum,lpl);
    	}

  	return carry;
}



__global__ void gpu_mul(int threads, uint32_t ulegs, uint32_t vlegs, uint64_t *g_u, uint64_t *g_v, uint64_t *g_p)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
	if(ulegs < vlegs){
		uint64_t t1=ulegs;
		ulegs = vlegs;
		vlegs = t1;

		uint64_t *t2 = g_u;
		g_u = g_v;
		g_v = t2;
	}

	uint32_t vofst=1,rofst=1,psize=0;
	mulScalar(thread,threads,ulegs,g_p,g_u,g_v[thread],&psize);   

#if 1

  	while (vofst < vlegs) {

	    	g_p[(psize+0)*threads+thread] = 0;

            	g_p[(ulegs+rofst)*threads + thread] = addmul_1g (thread, threads, g_p ,rofst , g_u, ulegs,  g_v[vofst*threads+thread]);

	    	vofst++; rofst++;
	    	psize++;
        }




#endif
    }
}

__global__ void  gpu_mulT4(int threads, uint32_t ulegs, uint32_t vlegs, uint64_t *g_u, uint64_t *g_v, uint64_t *g_p)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {

	if(ulegs < vlegs){  
		uint64_t t1=ulegs;
		ulegs = vlegs;   
		vlegs = t1;

		uint64_t *t2 = g_u;
		g_u = g_v;
		g_v = t2;
	}

	ulegs >>= 1; vlegs >>= 1;

	

	uint32_t vofst=1,rofst=1,psize=0;
	mulScalarT4(thread,threads,ulegs,g_p,g_u,T4(thread,threads,0,g_v),&psize);

#if 1
	t4_t zero;
	T4_set(&zero,0);
	

#pragma unroll
	    for (vofst=1;vofst<vlegs;vofst++) {  
	    	T4_store(thread,threads,psize,g_p,zero);

            	T4_store(thread,threads,ulegs+rofst,g_p,addmul_1gT4 (thread, threads, g_p ,rofst , g_u, ulegs,T4(thread,threads,vofst,g_v)));
			rofst++;
	    	psize++;
        }


#endif
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__host__ void cpu_mul(int thr_id, int threads, uint32_t alegs, uint32_t blegs, uint64_t *g_a, uint64_t *g_b, uint64_t *g_p,int order)
{

	const int threadsperblock = 512; // Alignment mit mixtab Gr\F6sse. NICHT \C4NDERN

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size =0;
  	gpu_mul<<<grid, block, shared_size>>>(threads, alegs, blegs, g_a, g_b, g_p) ;

}

__host__ void cpu_mulT4(int thr_id, int threads, uint32_t alegs, uint32_t blegs, uint64_t *g_a, uint64_t *g_b, uint64_t *g_p, int order)
{

	const int threadsperblock = 256; 

	dim3 grid(2*(threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size =0;
  	
	gpu_mulT4<<<grid, block, shared_size>>>(threads, blegs, alegs, g_b, g_a, g_p) ;
}

__host__ void mul_init(){

}
