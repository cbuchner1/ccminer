#include "hip/hip_runtime.h"
/*
 * Quick Hamsi-512 for X13
 * by tsiv - 2014
 */

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"

typedef unsigned char BitSequence;

static __constant__ uint32_t d_alpha_n[32];
static __constant__ uint32_t d_alpha_f[32];
static __constant__ uint32_t d_T512[64][16];

static const uint32_t alpha_n[] = {
	SPH_C32(0xff00f0f0), SPH_C32(0xccccaaaa), SPH_C32(0xf0f0cccc),
	SPH_C32(0xff00aaaa), SPH_C32(0xccccaaaa), SPH_C32(0xf0f0ff00),
	SPH_C32(0xaaaacccc), SPH_C32(0xf0f0ff00), SPH_C32(0xf0f0cccc),
	SPH_C32(0xaaaaff00), SPH_C32(0xccccff00), SPH_C32(0xaaaaf0f0),
	SPH_C32(0xaaaaf0f0), SPH_C32(0xff00cccc), SPH_C32(0xccccf0f0),
	SPH_C32(0xff00aaaa), SPH_C32(0xccccaaaa), SPH_C32(0xff00f0f0),
	SPH_C32(0xff00aaaa), SPH_C32(0xf0f0cccc), SPH_C32(0xf0f0ff00),
	SPH_C32(0xccccaaaa), SPH_C32(0xf0f0ff00), SPH_C32(0xaaaacccc),
	SPH_C32(0xaaaaff00), SPH_C32(0xf0f0cccc), SPH_C32(0xaaaaf0f0),
	SPH_C32(0xccccff00), SPH_C32(0xff00cccc), SPH_C32(0xaaaaf0f0),
	SPH_C32(0xff00aaaa), SPH_C32(0xccccf0f0)
};

static const uint32_t alpha_f[] = {
	SPH_C32(0xcaf9639c), SPH_C32(0x0ff0f9c0), SPH_C32(0x639c0ff0),
	SPH_C32(0xcaf9f9c0), SPH_C32(0x0ff0f9c0), SPH_C32(0x639ccaf9),
	SPH_C32(0xf9c00ff0), SPH_C32(0x639ccaf9), SPH_C32(0x639c0ff0),
	SPH_C32(0xf9c0caf9), SPH_C32(0x0ff0caf9), SPH_C32(0xf9c0639c),
	SPH_C32(0xf9c0639c), SPH_C32(0xcaf90ff0), SPH_C32(0x0ff0639c),
	SPH_C32(0xcaf9f9c0), SPH_C32(0x0ff0f9c0), SPH_C32(0xcaf9639c),
	SPH_C32(0xcaf9f9c0), SPH_C32(0x639c0ff0), SPH_C32(0x639ccaf9),
	SPH_C32(0x0ff0f9c0), SPH_C32(0x639ccaf9), SPH_C32(0xf9c00ff0),
	SPH_C32(0xf9c0caf9), SPH_C32(0x639c0ff0), SPH_C32(0xf9c0639c),
	SPH_C32(0x0ff0caf9), SPH_C32(0xcaf90ff0), SPH_C32(0xf9c0639c),
	SPH_C32(0xcaf9f9c0), SPH_C32(0x0ff0639c)
};

#define hamsi_s00   m0
#define hamsi_s01   m1
#define hamsi_s02   c0
#define hamsi_s03   c1
#define hamsi_s04   m2
#define hamsi_s05   m3
#define hamsi_s06   c2
#define hamsi_s07   c3
#define hamsi_s08   c4
#define hamsi_s09   c5
#define hamsi_s0A   m4
#define hamsi_s0B   m5
#define hamsi_s0C   c6
#define hamsi_s0D   c7
#define hamsi_s0E   m6
#define hamsi_s0F   m7
#define hamsi_s10   m8
#define hamsi_s11   m9
#define hamsi_s12   c8
#define hamsi_s13   c9
#define hamsi_s14   mA
#define hamsi_s15   mB
#define hamsi_s16   cA
#define hamsi_s17   cB
#define hamsi_s18   cC
#define hamsi_s19   cD
#define hamsi_s1A   mC
#define hamsi_s1B   mD
#define hamsi_s1C   cE
#define hamsi_s1D   cF
#define hamsi_s1E   mE
#define hamsi_s1F   mF

#define SBOX(a, b, c, d) { \
		uint32_t t; \
		t = (a); \
		(a) &= (c); \
		(a) ^= (d); \
		(c) ^= (b); \
		(c) ^= (a); \
		(d) |= t; \
		(d) ^= (b); \
		t ^= (c); \
		(b) = (d); \
		(d) |= t; \
		(d) ^= (a); \
		(a) &= (b); \
		t ^= (a); \
		(b) ^= (d); \
		(b) ^= t; \
		(a) = (c); \
		(c) = (b); \
		(b) = (d); \
		(d) = SPH_T32(~t); \
	}

#define HAMSI_L(a, b, c, d) { \
		(a) = ROTL32(a, 13); \
		(c) = ROTL32(c, 3); \
		(b) ^= (a) ^ (c); \
		(d) ^= (c) ^ ((a) << 3); \
		(b) = ROTL32(b, 1); \
		(d) = ROTL32(d, 7); \
		(a) ^= (b) ^ (d); \
		(c) ^= (d) ^ ((b) << 7); \
		(a) = ROTL32(a, 5); \
		(c) = ROTL32(c, 22); \
	}

#define ROUND_BIG(rc, alpha) { \
		hamsi_s00 ^= alpha[0x00]; \
		hamsi_s08 ^= alpha[0x08]; \
		hamsi_s10 ^= alpha[0x10]; \
		hamsi_s18 ^= alpha[0x18]; \
		hamsi_s01 ^= alpha[0x01] ^ (uint32_t)(rc); \
		hamsi_s09 ^= alpha[0x09]; \
		hamsi_s11 ^= alpha[0x11]; \
		hamsi_s19 ^= alpha[0x19]; \
		hamsi_s02 ^= alpha[0x02]; \
		hamsi_s0A ^= alpha[0x0A]; \
		hamsi_s12 ^= alpha[0x12]; \
		hamsi_s1A ^= alpha[0x1A]; \
		hamsi_s03 ^= alpha[0x03]; \
		hamsi_s0B ^= alpha[0x0B]; \
		hamsi_s13 ^= alpha[0x13]; \
		hamsi_s1B ^= alpha[0x1B]; \
		hamsi_s04 ^= alpha[0x04]; \
		hamsi_s0C ^= alpha[0x0C]; \
		hamsi_s14 ^= alpha[0x14]; \
		hamsi_s1C ^= alpha[0x1C]; \
		hamsi_s05 ^= alpha[0x05]; \
		hamsi_s0D ^= alpha[0x0D]; \
		hamsi_s15 ^= alpha[0x15]; \
		hamsi_s1D ^= alpha[0x1D]; \
		hamsi_s06 ^= alpha[0x06]; \
		hamsi_s0E ^= alpha[0x0E]; \
		hamsi_s16 ^= alpha[0x16]; \
		hamsi_s1E ^= alpha[0x1E]; \
		hamsi_s07 ^= alpha[0x07]; \
		hamsi_s0F ^= alpha[0x0F]; \
		hamsi_s17 ^= alpha[0x17]; \
		hamsi_s1F ^= alpha[0x1F]; \
		SBOX(hamsi_s00, hamsi_s08, hamsi_s10, hamsi_s18); \
		SBOX(hamsi_s01, hamsi_s09, hamsi_s11, hamsi_s19); \
		SBOX(hamsi_s02, hamsi_s0A, hamsi_s12, hamsi_s1A); \
		SBOX(hamsi_s03, hamsi_s0B, hamsi_s13, hamsi_s1B); \
		SBOX(hamsi_s04, hamsi_s0C, hamsi_s14, hamsi_s1C); \
		SBOX(hamsi_s05, hamsi_s0D, hamsi_s15, hamsi_s1D); \
		SBOX(hamsi_s06, hamsi_s0E, hamsi_s16, hamsi_s1E); \
		SBOX(hamsi_s07, hamsi_s0F, hamsi_s17, hamsi_s1F); \
		HAMSI_L(hamsi_s00, hamsi_s09, hamsi_s12, hamsi_s1B); \
		HAMSI_L(hamsi_s01, hamsi_s0A, hamsi_s13, hamsi_s1C); \
		HAMSI_L(hamsi_s02, hamsi_s0B, hamsi_s14, hamsi_s1D); \
		HAMSI_L(hamsi_s03, hamsi_s0C, hamsi_s15, hamsi_s1E); \
		HAMSI_L(hamsi_s04, hamsi_s0D, hamsi_s16, hamsi_s1F); \
		HAMSI_L(hamsi_s05, hamsi_s0E, hamsi_s17, hamsi_s18); \
		HAMSI_L(hamsi_s06, hamsi_s0F, hamsi_s10, hamsi_s19); \
		HAMSI_L(hamsi_s07, hamsi_s08, hamsi_s11, hamsi_s1A); \
		HAMSI_L(hamsi_s00, hamsi_s02, hamsi_s05, hamsi_s07); \
		HAMSI_L(hamsi_s10, hamsi_s13, hamsi_s15, hamsi_s16); \
		HAMSI_L(hamsi_s09, hamsi_s0B, hamsi_s0C, hamsi_s0E); \
		HAMSI_L(hamsi_s19, hamsi_s1A, hamsi_s1C, hamsi_s1F); \
	}


#define P_BIG  { \
		for( int r = 0; r < 6; r++ ) \
			ROUND_BIG(r, d_alpha_n); \
	}

#define PF_BIG { \
		for( int r = 0; r < 12; r++ ) \
			ROUND_BIG(r, d_alpha_f); \
	}

#define T_BIG  { \
		/* order is important */ \
		cF = (h[0xF] ^= hamsi_s17); \
		cE = (h[0xE] ^= hamsi_s16); \
		cD = (h[0xD] ^= hamsi_s15); \
		cC = (h[0xC] ^= hamsi_s14); \
		cB = (h[0xB] ^= hamsi_s13); \
		cA = (h[0xA] ^= hamsi_s12); \
		c9 = (h[0x9] ^= hamsi_s11); \
		c8 = (h[0x8] ^= hamsi_s10); \
		c7 = (h[0x7] ^= hamsi_s07); \
		c6 = (h[0x6] ^= hamsi_s06); \
		c5 = (h[0x5] ^= hamsi_s05); \
		c4 = (h[0x4] ^= hamsi_s04); \
		c3 = (h[0x3] ^= hamsi_s03); \
		c2 = (h[0x2] ^= hamsi_s02); \
		c1 = (h[0x1] ^= hamsi_s01); \
		c0 = (h[0x0] ^= hamsi_s00); \
	}


static const uint32_t T512[64][16] = {
	{ SPH_C32(0xef0b0270), SPH_C32(0x3afd0000), SPH_C32(0x5dae0000),
	  SPH_C32(0x69490000), SPH_C32(0x9b0f3c06), SPH_C32(0x4405b5f9),
	  SPH_C32(0x66140a51), SPH_C32(0x924f5d0a), SPH_C32(0xc96b0030),
	  SPH_C32(0xe7250000), SPH_C32(0x2f840000), SPH_C32(0x264f0000),
	  SPH_C32(0x08695bf9), SPH_C32(0x6dfcf137), SPH_C32(0x509f6984),
	  SPH_C32(0x9e69af68) },
	{ SPH_C32(0xc96b0030), SPH_C32(0xe7250000), SPH_C32(0x2f840000),
	  SPH_C32(0x264f0000), SPH_C32(0x08695bf9), SPH_C32(0x6dfcf137),
	  SPH_C32(0x509f6984), SPH_C32(0x9e69af68), SPH_C32(0x26600240),
	  SPH_C32(0xddd80000), SPH_C32(0x722a0000), SPH_C32(0x4f060000),
	  SPH_C32(0x936667ff), SPH_C32(0x29f944ce), SPH_C32(0x368b63d5),
	  SPH_C32(0x0c26f262) },
	{ SPH_C32(0x145a3c00), SPH_C32(0xb9e90000), SPH_C32(0x61270000),
	  SPH_C32(0xf1610000), SPH_C32(0xce613d6c), SPH_C32(0xb0493d78),
	  SPH_C32(0x47a96720), SPH_C32(0xe18e24c5), SPH_C32(0x23671400),
	  SPH_C32(0xc8b90000), SPH_C32(0xf4c70000), SPH_C32(0xfb750000),
	  SPH_C32(0x73cd2465), SPH_C32(0xf8a6a549), SPH_C32(0x02c40a3f),
	  SPH_C32(0xdc24e61f) },
	{ SPH_C32(0x23671400), SPH_C32(0xc8b90000), SPH_C32(0xf4c70000),
	  SPH_C32(0xfb750000), SPH_C32(0x73cd2465), SPH_C32(0xf8a6a549),
	  SPH_C32(0x02c40a3f), SPH_C32(0xdc24e61f), SPH_C32(0x373d2800),
	  SPH_C32(0x71500000), SPH_C32(0x95e00000), SPH_C32(0x0a140000),
	  SPH_C32(0xbdac1909), SPH_C32(0x48ef9831), SPH_C32(0x456d6d1f),
	  SPH_C32(0x3daac2da) },
	{ SPH_C32(0x54285c00), SPH_C32(0xeaed0000), SPH_C32(0xc5d60000),
	  SPH_C32(0xa1c50000), SPH_C32(0xb3a26770), SPH_C32(0x94a5c4e1),
	  SPH_C32(0x6bb0419d), SPH_C32(0x551b3782), SPH_C32(0x9cbb1800),
	  SPH_C32(0xb0d30000), SPH_C32(0x92510000), SPH_C32(0xed930000),
	  SPH_C32(0x593a4345), SPH_C32(0xe114d5f4), SPH_C32(0x430633da),
	  SPH_C32(0x78cace29) },
	{ SPH_C32(0x9cbb1800), SPH_C32(0xb0d30000), SPH_C32(0x92510000),
	  SPH_C32(0xed930000), SPH_C32(0x593a4345), SPH_C32(0xe114d5f4),
	  SPH_C32(0x430633da), SPH_C32(0x78cace29), SPH_C32(0xc8934400),
	  SPH_C32(0x5a3e0000), SPH_C32(0x57870000), SPH_C32(0x4c560000),
	  SPH_C32(0xea982435), SPH_C32(0x75b11115), SPH_C32(0x28b67247),
	  SPH_C32(0x2dd1f9ab) },
	{ SPH_C32(0x29449c00), SPH_C32(0x64e70000), SPH_C32(0xf24b0000),
	  SPH_C32(0xc2f30000), SPH_C32(0x0ede4e8f), SPH_C32(0x56c23745),
	  SPH_C32(0xf3e04259), SPH_C32(0x8d0d9ec4), SPH_C32(0x466d0c00),
	  SPH_C32(0x08620000), SPH_C32(0xdd5d0000), SPH_C32(0xbadd0000),
	  SPH_C32(0x6a927942), SPH_C32(0x441f2b93), SPH_C32(0x218ace6f),
	  SPH_C32(0xbf2c0be2) },
	{ SPH_C32(0x466d0c00), SPH_C32(0x08620000), SPH_C32(0xdd5d0000),
	  SPH_C32(0xbadd0000), SPH_C32(0x6a927942), SPH_C32(0x441f2b93),
	  SPH_C32(0x218ace6f), SPH_C32(0xbf2c0be2), SPH_C32(0x6f299000),
	  SPH_C32(0x6c850000), SPH_C32(0x2f160000), SPH_C32(0x782e0000),
	  SPH_C32(0x644c37cd), SPH_C32(0x12dd1cd6), SPH_C32(0xd26a8c36),
	  SPH_C32(0x32219526) },
	{ SPH_C32(0xf6800005), SPH_C32(0x3443c000), SPH_C32(0x24070000),
	  SPH_C32(0x8f3d0000), SPH_C32(0x21373bfb), SPH_C32(0x0ab8d5ae),
	  SPH_C32(0xcdc58b19), SPH_C32(0xd795ba31), SPH_C32(0xa67f0001),
	  SPH_C32(0x71378000), SPH_C32(0x19fc0000), SPH_C32(0x96db0000),
	  SPH_C32(0x3a8b6dfd), SPH_C32(0xebcaaef3), SPH_C32(0x2c6d478f),
	  SPH_C32(0xac8e6c88) },
	{ SPH_C32(0xa67f0001), SPH_C32(0x71378000), SPH_C32(0x19fc0000),
	  SPH_C32(0x96db0000), SPH_C32(0x3a8b6dfd), SPH_C32(0xebcaaef3),
	  SPH_C32(0x2c6d478f), SPH_C32(0xac8e6c88), SPH_C32(0x50ff0004),
	  SPH_C32(0x45744000), SPH_C32(0x3dfb0000), SPH_C32(0x19e60000),
	  SPH_C32(0x1bbc5606), SPH_C32(0xe1727b5d), SPH_C32(0xe1a8cc96),
	  SPH_C32(0x7b1bd6b9) },
	{ SPH_C32(0xf7750009), SPH_C32(0xcf3cc000), SPH_C32(0xc3d60000),
	  SPH_C32(0x04920000), SPH_C32(0x029519a9), SPH_C32(0xf8e836ba),
	  SPH_C32(0x7a87f14e), SPH_C32(0x9e16981a), SPH_C32(0xd46a0000),
	  SPH_C32(0x8dc8c000), SPH_C32(0xa5af0000), SPH_C32(0x4a290000),
	  SPH_C32(0xfc4e427a), SPH_C32(0xc9b4866c), SPH_C32(0x98369604),
	  SPH_C32(0xf746c320) },
	{ SPH_C32(0xd46a0000), SPH_C32(0x8dc8c000), SPH_C32(0xa5af0000),
	  SPH_C32(0x4a290000), SPH_C32(0xfc4e427a), SPH_C32(0xc9b4866c),
	  SPH_C32(0x98369604), SPH_C32(0xf746c320), SPH_C32(0x231f0009),
	  SPH_C32(0x42f40000), SPH_C32(0x66790000), SPH_C32(0x4ebb0000),
	  SPH_C32(0xfedb5bd3), SPH_C32(0x315cb0d6), SPH_C32(0xe2b1674a),
	  SPH_C32(0x69505b3a) },
	{ SPH_C32(0x774400f0), SPH_C32(0xf15a0000), SPH_C32(0xf5b20000),
	  SPH_C32(0x34140000), SPH_C32(0x89377e8c), SPH_C32(0x5a8bec25),
	  SPH_C32(0x0bc3cd1e), SPH_C32(0xcf3775cb), SPH_C32(0xf46c0050),
	  SPH_C32(0x96180000), SPH_C32(0x14a50000), SPH_C32(0x031f0000),
	  SPH_C32(0x42947eb8), SPH_C32(0x66bf7e19), SPH_C32(0x9ca470d2),
	  SPH_C32(0x8a341574) },
	{ SPH_C32(0xf46c0050), SPH_C32(0x96180000), SPH_C32(0x14a50000),
	  SPH_C32(0x031f0000), SPH_C32(0x42947eb8), SPH_C32(0x66bf7e19),
	  SPH_C32(0x9ca470d2), SPH_C32(0x8a341574), SPH_C32(0x832800a0),
	  SPH_C32(0x67420000), SPH_C32(0xe1170000), SPH_C32(0x370b0000),
	  SPH_C32(0xcba30034), SPH_C32(0x3c34923c), SPH_C32(0x9767bdcc),
	  SPH_C32(0x450360bf) },
	{ SPH_C32(0xe8870170), SPH_C32(0x9d720000), SPH_C32(0x12db0000),
	  SPH_C32(0xd4220000), SPH_C32(0xf2886b27), SPH_C32(0xa921e543),
	  SPH_C32(0x4ef8b518), SPH_C32(0x618813b1), SPH_C32(0xb4370060),
	  SPH_C32(0x0c4c0000), SPH_C32(0x56c20000), SPH_C32(0x5cae0000),
	  SPH_C32(0x94541f3f), SPH_C32(0x3b3ef825), SPH_C32(0x1b365f3d),
	  SPH_C32(0xf3d45758) },
	{ SPH_C32(0xb4370060), SPH_C32(0x0c4c0000), SPH_C32(0x56c20000),
	  SPH_C32(0x5cae0000), SPH_C32(0x94541f3f), SPH_C32(0x3b3ef825),
	  SPH_C32(0x1b365f3d), SPH_C32(0xf3d45758), SPH_C32(0x5cb00110),
	  SPH_C32(0x913e0000), SPH_C32(0x44190000), SPH_C32(0x888c0000),
	  SPH_C32(0x66dc7418), SPH_C32(0x921f1d66), SPH_C32(0x55ceea25),
	  SPH_C32(0x925c44e9) },
	{ SPH_C32(0x0c720000), SPH_C32(0x49e50f00), SPH_C32(0x42790000),
	  SPH_C32(0x5cea0000), SPH_C32(0x33aa301a), SPH_C32(0x15822514),
	  SPH_C32(0x95a34b7b), SPH_C32(0xb44b0090), SPH_C32(0xfe220000),
	  SPH_C32(0xa7580500), SPH_C32(0x25d10000), SPH_C32(0xf7600000),
	  SPH_C32(0x893178da), SPH_C32(0x1fd4f860), SPH_C32(0x4ed0a315),
	  SPH_C32(0xa123ff9f) },
	{ SPH_C32(0xfe220000), SPH_C32(0xa7580500), SPH_C32(0x25d10000),
	  SPH_C32(0xf7600000), SPH_C32(0x893178da), SPH_C32(0x1fd4f860),
	  SPH_C32(0x4ed0a315), SPH_C32(0xa123ff9f), SPH_C32(0xf2500000),
	  SPH_C32(0xeebd0a00), SPH_C32(0x67a80000), SPH_C32(0xab8a0000),
	  SPH_C32(0xba9b48c0), SPH_C32(0x0a56dd74), SPH_C32(0xdb73e86e),
	  SPH_C32(0x1568ff0f) },
	{ SPH_C32(0x45180000), SPH_C32(0xa5b51700), SPH_C32(0xf96a0000),
	  SPH_C32(0x3b480000), SPH_C32(0x1ecc142c), SPH_C32(0x231395d6),
	  SPH_C32(0x16bca6b0), SPH_C32(0xdf33f4df), SPH_C32(0xb83d0000),
	  SPH_C32(0x16710600), SPH_C32(0x379a0000), SPH_C32(0xf5b10000),
	  SPH_C32(0x228161ac), SPH_C32(0xae48f145), SPH_C32(0x66241616),
	  SPH_C32(0xc5c1eb3e) },
	{ SPH_C32(0xb83d0000), SPH_C32(0x16710600), SPH_C32(0x379a0000),
	  SPH_C32(0xf5b10000), SPH_C32(0x228161ac), SPH_C32(0xae48f145),
	  SPH_C32(0x66241616), SPH_C32(0xc5c1eb3e), SPH_C32(0xfd250000),
	  SPH_C32(0xb3c41100), SPH_C32(0xcef00000), SPH_C32(0xcef90000),
	  SPH_C32(0x3c4d7580), SPH_C32(0x8d5b6493), SPH_C32(0x7098b0a6),
	  SPH_C32(0x1af21fe1) },
	{ SPH_C32(0x75a40000), SPH_C32(0xc28b2700), SPH_C32(0x94a40000),
	  SPH_C32(0x90f50000), SPH_C32(0xfb7857e0), SPH_C32(0x49ce0bae),
	  SPH_C32(0x1767c483), SPH_C32(0xaedf667e), SPH_C32(0xd1660000),
	  SPH_C32(0x1bbc0300), SPH_C32(0x9eec0000), SPH_C32(0xf6940000),
	  SPH_C32(0x03024527), SPH_C32(0xcf70fcf2), SPH_C32(0xb4431b17),
	  SPH_C32(0x857f3c2b) },
	{ SPH_C32(0xd1660000), SPH_C32(0x1bbc0300), SPH_C32(0x9eec0000),
	  SPH_C32(0xf6940000), SPH_C32(0x03024527), SPH_C32(0xcf70fcf2),
	  SPH_C32(0xb4431b17), SPH_C32(0x857f3c2b), SPH_C32(0xa4c20000),
	  SPH_C32(0xd9372400), SPH_C32(0x0a480000), SPH_C32(0x66610000),
	  SPH_C32(0xf87a12c7), SPH_C32(0x86bef75c), SPH_C32(0xa324df94),
	  SPH_C32(0x2ba05a55) },
	{ SPH_C32(0x75c90003), SPH_C32(0x0e10c000), SPH_C32(0xd1200000),
	  SPH_C32(0xbaea0000), SPH_C32(0x8bc42f3e), SPH_C32(0x8758b757),
	  SPH_C32(0xbb28761d), SPH_C32(0x00b72e2b), SPH_C32(0xeecf0001),
	  SPH_C32(0x6f564000), SPH_C32(0xf33e0000), SPH_C32(0xa79e0000),
	  SPH_C32(0xbdb57219), SPH_C32(0xb711ebc5), SPH_C32(0x4a3b40ba),
	  SPH_C32(0xfeabf254) },
	{ SPH_C32(0xeecf0001), SPH_C32(0x6f564000), SPH_C32(0xf33e0000),
	  SPH_C32(0xa79e0000), SPH_C32(0xbdb57219), SPH_C32(0xb711ebc5),
	  SPH_C32(0x4a3b40ba), SPH_C32(0xfeabf254), SPH_C32(0x9b060002),
	  SPH_C32(0x61468000), SPH_C32(0x221e0000), SPH_C32(0x1d740000),
	  SPH_C32(0x36715d27), SPH_C32(0x30495c92), SPH_C32(0xf11336a7),
	  SPH_C32(0xfe1cdc7f) },
	{ SPH_C32(0x86790000), SPH_C32(0x3f390002), SPH_C32(0xe19ae000),
	  SPH_C32(0x98560000), SPH_C32(0x9565670e), SPH_C32(0x4e88c8ea),
	  SPH_C32(0xd3dd4944), SPH_C32(0x161ddab9), SPH_C32(0x30b70000),
	  SPH_C32(0xe5d00000), SPH_C32(0xf4f46000), SPH_C32(0x42c40000),
	  SPH_C32(0x63b83d6a), SPH_C32(0x78ba9460), SPH_C32(0x21afa1ea),
	  SPH_C32(0xb0a51834) },
	{ SPH_C32(0x30b70000), SPH_C32(0xe5d00000), SPH_C32(0xf4f46000),
	  SPH_C32(0x42c40000), SPH_C32(0x63b83d6a), SPH_C32(0x78ba9460),
	  SPH_C32(0x21afa1ea), SPH_C32(0xb0a51834), SPH_C32(0xb6ce0000),
	  SPH_C32(0xdae90002), SPH_C32(0x156e8000), SPH_C32(0xda920000),
	  SPH_C32(0xf6dd5a64), SPH_C32(0x36325c8a), SPH_C32(0xf272e8ae),
	  SPH_C32(0xa6b8c28d) },
	{ SPH_C32(0x14190000), SPH_C32(0x23ca003c), SPH_C32(0x50df0000),
	  SPH_C32(0x44b60000), SPH_C32(0x1b6c67b0), SPH_C32(0x3cf3ac75),
	  SPH_C32(0x61e610b0), SPH_C32(0xdbcadb80), SPH_C32(0xe3430000),
	  SPH_C32(0x3a4e0014), SPH_C32(0xf2c60000), SPH_C32(0xaa4e0000),
	  SPH_C32(0xdb1e42a6), SPH_C32(0x256bbe15), SPH_C32(0x123db156),
	  SPH_C32(0x3a4e99d7) },
	{ SPH_C32(0xe3430000), SPH_C32(0x3a4e0014), SPH_C32(0xf2c60000),
	  SPH_C32(0xaa4e0000), SPH_C32(0xdb1e42a6), SPH_C32(0x256bbe15),
	  SPH_C32(0x123db156), SPH_C32(0x3a4e99d7), SPH_C32(0xf75a0000),
	  SPH_C32(0x19840028), SPH_C32(0xa2190000), SPH_C32(0xeef80000),
	  SPH_C32(0xc0722516), SPH_C32(0x19981260), SPH_C32(0x73dba1e6),
	  SPH_C32(0xe1844257) },
	{ SPH_C32(0x54500000), SPH_C32(0x0671005c), SPH_C32(0x25ae0000),
	  SPH_C32(0x6a1e0000), SPH_C32(0x2ea54edf), SPH_C32(0x664e8512),
	  SPH_C32(0xbfba18c3), SPH_C32(0x7e715d17), SPH_C32(0xbc8d0000),
	  SPH_C32(0xfc3b0018), SPH_C32(0x19830000), SPH_C32(0xd10b0000),
	  SPH_C32(0xae1878c4), SPH_C32(0x42a69856), SPH_C32(0x0012da37),
	  SPH_C32(0x2c3b504e) },
	{ SPH_C32(0xbc8d0000), SPH_C32(0xfc3b0018), SPH_C32(0x19830000),
	  SPH_C32(0xd10b0000), SPH_C32(0xae1878c4), SPH_C32(0x42a69856),
	  SPH_C32(0x0012da37), SPH_C32(0x2c3b504e), SPH_C32(0xe8dd0000),
	  SPH_C32(0xfa4a0044), SPH_C32(0x3c2d0000), SPH_C32(0xbb150000),
	  SPH_C32(0x80bd361b), SPH_C32(0x24e81d44), SPH_C32(0xbfa8c2f4),
	  SPH_C32(0x524a0d59) },
	{ SPH_C32(0x69510000), SPH_C32(0xd4e1009c), SPH_C32(0xc3230000),
	  SPH_C32(0xac2f0000), SPH_C32(0xe4950bae), SPH_C32(0xcea415dc),
	  SPH_C32(0x87ec287c), SPH_C32(0xbce1a3ce), SPH_C32(0xc6730000),
	  SPH_C32(0xaf8d000c), SPH_C32(0xa4c10000), SPH_C32(0x218d0000),
	  SPH_C32(0x23111587), SPH_C32(0x7913512f), SPH_C32(0x1d28ac88),
	  SPH_C32(0x378dd173) },
	{ SPH_C32(0xc6730000), SPH_C32(0xaf8d000c), SPH_C32(0xa4c10000),
	  SPH_C32(0x218d0000), SPH_C32(0x23111587), SPH_C32(0x7913512f),
	  SPH_C32(0x1d28ac88), SPH_C32(0x378dd173), SPH_C32(0xaf220000),
	  SPH_C32(0x7b6c0090), SPH_C32(0x67e20000), SPH_C32(0x8da20000),
	  SPH_C32(0xc7841e29), SPH_C32(0xb7b744f3), SPH_C32(0x9ac484f4),
	  SPH_C32(0x8b6c72bd) },
	{ SPH_C32(0xcc140000), SPH_C32(0xa5630000), SPH_C32(0x5ab90780),
	  SPH_C32(0x3b500000), SPH_C32(0x4bd013ff), SPH_C32(0x879b3418),
	  SPH_C32(0x694348c1), SPH_C32(0xca5a87fe), SPH_C32(0x819e0000),
	  SPH_C32(0xec570000), SPH_C32(0x66320280), SPH_C32(0x95f30000),
	  SPH_C32(0x5da92802), SPH_C32(0x48f43cbc), SPH_C32(0xe65aa22d),
	  SPH_C32(0x8e67b7fa) },
	{ SPH_C32(0x819e0000), SPH_C32(0xec570000), SPH_C32(0x66320280),
	  SPH_C32(0x95f30000), SPH_C32(0x5da92802), SPH_C32(0x48f43cbc),
	  SPH_C32(0xe65aa22d), SPH_C32(0x8e67b7fa), SPH_C32(0x4d8a0000),
	  SPH_C32(0x49340000), SPH_C32(0x3c8b0500), SPH_C32(0xaea30000),
	  SPH_C32(0x16793bfd), SPH_C32(0xcf6f08a4), SPH_C32(0x8f19eaec),
	  SPH_C32(0x443d3004) },
	{ SPH_C32(0x78230000), SPH_C32(0x12fc0000), SPH_C32(0xa93a0b80),
	  SPH_C32(0x90a50000), SPH_C32(0x713e2879), SPH_C32(0x7ee98924),
	  SPH_C32(0xf08ca062), SPH_C32(0x636f8bab), SPH_C32(0x02af0000),
	  SPH_C32(0xb7280000), SPH_C32(0xba1c0300), SPH_C32(0x56980000),
	  SPH_C32(0xba8d45d3), SPH_C32(0x8048c667), SPH_C32(0xa95c149a),
	  SPH_C32(0xf4f6ea7b) },
	{ SPH_C32(0x02af0000), SPH_C32(0xb7280000), SPH_C32(0xba1c0300),
	  SPH_C32(0x56980000), SPH_C32(0xba8d45d3), SPH_C32(0x8048c667),
	  SPH_C32(0xa95c149a), SPH_C32(0xf4f6ea7b), SPH_C32(0x7a8c0000),
	  SPH_C32(0xa5d40000), SPH_C32(0x13260880), SPH_C32(0xc63d0000),
	  SPH_C32(0xcbb36daa), SPH_C32(0xfea14f43), SPH_C32(0x59d0b4f8),
	  SPH_C32(0x979961d0) },
	{ SPH_C32(0xac480000), SPH_C32(0x1ba60000), SPH_C32(0x45fb1380),
	  SPH_C32(0x03430000), SPH_C32(0x5a85316a), SPH_C32(0x1fb250b6),
	  SPH_C32(0xfe72c7fe), SPH_C32(0x91e478f6), SPH_C32(0x1e4e0000),
	  SPH_C32(0xdecf0000), SPH_C32(0x6df80180), SPH_C32(0x77240000),
	  SPH_C32(0xec47079e), SPH_C32(0xf4a0694e), SPH_C32(0xcda31812),
	  SPH_C32(0x98aa496e) },
	{ SPH_C32(0x1e4e0000), SPH_C32(0xdecf0000), SPH_C32(0x6df80180),
	  SPH_C32(0x77240000), SPH_C32(0xec47079e), SPH_C32(0xf4a0694e),
	  SPH_C32(0xcda31812), SPH_C32(0x98aa496e), SPH_C32(0xb2060000),
	  SPH_C32(0xc5690000), SPH_C32(0x28031200), SPH_C32(0x74670000),
	  SPH_C32(0xb6c236f4), SPH_C32(0xeb1239f8), SPH_C32(0x33d1dfec),
	  SPH_C32(0x094e3198) },
	{ SPH_C32(0xaec30000), SPH_C32(0x9c4f0001), SPH_C32(0x79d1e000),
	  SPH_C32(0x2c150000), SPH_C32(0x45cc75b3), SPH_C32(0x6650b736),
	  SPH_C32(0xab92f78f), SPH_C32(0xa312567b), SPH_C32(0xdb250000),
	  SPH_C32(0x09290000), SPH_C32(0x49aac000), SPH_C32(0x81e10000),
	  SPH_C32(0xcafe6b59), SPH_C32(0x42793431), SPH_C32(0x43566b76),
	  SPH_C32(0xe86cba2e) },
	{ SPH_C32(0xdb250000), SPH_C32(0x09290000), SPH_C32(0x49aac000),
	  SPH_C32(0x81e10000), SPH_C32(0xcafe6b59), SPH_C32(0x42793431),
	  SPH_C32(0x43566b76), SPH_C32(0xe86cba2e), SPH_C32(0x75e60000),
	  SPH_C32(0x95660001), SPH_C32(0x307b2000), SPH_C32(0xadf40000),
	  SPH_C32(0x8f321eea), SPH_C32(0x24298307), SPH_C32(0xe8c49cf9),
	  SPH_C32(0x4b7eec55) },
	{ SPH_C32(0x58430000), SPH_C32(0x807e0000), SPH_C32(0x78330001),
	  SPH_C32(0xc66b3800), SPH_C32(0xe7375cdc), SPH_C32(0x79ad3fdd),
	  SPH_C32(0xac73fe6f), SPH_C32(0x3a4479b1), SPH_C32(0x1d5a0000),
	  SPH_C32(0x2b720000), SPH_C32(0x488d0000), SPH_C32(0xaf611800),
	  SPH_C32(0x25cb2ec5), SPH_C32(0xc879bfd0), SPH_C32(0x81a20429),
	  SPH_C32(0x1e7536a6) },
	{ SPH_C32(0x1d5a0000), SPH_C32(0x2b720000), SPH_C32(0x488d0000),
	  SPH_C32(0xaf611800), SPH_C32(0x25cb2ec5), SPH_C32(0xc879bfd0),
	  SPH_C32(0x81a20429), SPH_C32(0x1e7536a6), SPH_C32(0x45190000),
	  SPH_C32(0xab0c0000), SPH_C32(0x30be0001), SPH_C32(0x690a2000),
	  SPH_C32(0xc2fc7219), SPH_C32(0xb1d4800d), SPH_C32(0x2dd1fa46),
	  SPH_C32(0x24314f17) },
	{ SPH_C32(0xa53b0000), SPH_C32(0x14260000), SPH_C32(0x4e30001e),
	  SPH_C32(0x7cae0000), SPH_C32(0x8f9e0dd5), SPH_C32(0x78dfaa3d),
	  SPH_C32(0xf73168d8), SPH_C32(0x0b1b4946), SPH_C32(0x07ed0000),
	  SPH_C32(0xb2500000), SPH_C32(0x8774000a), SPH_C32(0x970d0000),
	  SPH_C32(0x437223ae), SPH_C32(0x48c76ea4), SPH_C32(0xf4786222),
	  SPH_C32(0x9075b1ce) },
	{ SPH_C32(0x07ed0000), SPH_C32(0xb2500000), SPH_C32(0x8774000a),
	  SPH_C32(0x970d0000), SPH_C32(0x437223ae), SPH_C32(0x48c76ea4),
	  SPH_C32(0xf4786222), SPH_C32(0x9075b1ce), SPH_C32(0xa2d60000),
	  SPH_C32(0xa6760000), SPH_C32(0xc9440014), SPH_C32(0xeba30000),
	  SPH_C32(0xccec2e7b), SPH_C32(0x3018c499), SPH_C32(0x03490afa),
	  SPH_C32(0x9b6ef888) },
	{ SPH_C32(0x88980000), SPH_C32(0x1f940000), SPH_C32(0x7fcf002e),
	  SPH_C32(0xfb4e0000), SPH_C32(0xf158079a), SPH_C32(0x61ae9167),
	  SPH_C32(0xa895706c), SPH_C32(0xe6107494), SPH_C32(0x0bc20000),
	  SPH_C32(0xdb630000), SPH_C32(0x7e88000c), SPH_C32(0x15860000),
	  SPH_C32(0x91fd48f3), SPH_C32(0x7581bb43), SPH_C32(0xf460449e),
	  SPH_C32(0xd8b61463) },
	{ SPH_C32(0x0bc20000), SPH_C32(0xdb630000), SPH_C32(0x7e88000c),
	  SPH_C32(0x15860000), SPH_C32(0x91fd48f3), SPH_C32(0x7581bb43),
	  SPH_C32(0xf460449e), SPH_C32(0xd8b61463), SPH_C32(0x835a0000),
	  SPH_C32(0xc4f70000), SPH_C32(0x01470022), SPH_C32(0xeec80000),
	  SPH_C32(0x60a54f69), SPH_C32(0x142f2a24), SPH_C32(0x5cf534f2),
	  SPH_C32(0x3ea660f7) },
	{ SPH_C32(0x52500000), SPH_C32(0x29540000), SPH_C32(0x6a61004e),
	  SPH_C32(0xf0ff0000), SPH_C32(0x9a317eec), SPH_C32(0x452341ce),
	  SPH_C32(0xcf568fe5), SPH_C32(0x5303130f), SPH_C32(0x538d0000),
	  SPH_C32(0xa9fc0000), SPH_C32(0x9ef70006), SPH_C32(0x56ff0000),
	  SPH_C32(0x0ae4004e), SPH_C32(0x92c5cdf9), SPH_C32(0xa9444018),
	  SPH_C32(0x7f975691) },
	{ SPH_C32(0x538d0000), SPH_C32(0xa9fc0000), SPH_C32(0x9ef70006),
	  SPH_C32(0x56ff0000), SPH_C32(0x0ae4004e), SPH_C32(0x92c5cdf9),
	  SPH_C32(0xa9444018), SPH_C32(0x7f975691), SPH_C32(0x01dd0000),
	  SPH_C32(0x80a80000), SPH_C32(0xf4960048), SPH_C32(0xa6000000),
	  SPH_C32(0x90d57ea2), SPH_C32(0xd7e68c37), SPH_C32(0x6612cffd),
	  SPH_C32(0x2c94459e) },
	{ SPH_C32(0xe6280000), SPH_C32(0x4c4b0000), SPH_C32(0xa8550000),
	  SPH_C32(0xd3d002e0), SPH_C32(0xd86130b8), SPH_C32(0x98a7b0da),
	  SPH_C32(0x289506b4), SPH_C32(0xd75a4897), SPH_C32(0xf0c50000),
	  SPH_C32(0x59230000), SPH_C32(0x45820000), SPH_C32(0xe18d00c0),
	  SPH_C32(0x3b6d0631), SPH_C32(0xc2ed5699), SPH_C32(0xcbe0fe1c),
	  SPH_C32(0x56a7b19f) },
	{ SPH_C32(0xf0c50000), SPH_C32(0x59230000), SPH_C32(0x45820000),
	  SPH_C32(0xe18d00c0), SPH_C32(0x3b6d0631), SPH_C32(0xc2ed5699),
	  SPH_C32(0xcbe0fe1c), SPH_C32(0x56a7b19f), SPH_C32(0x16ed0000),
	  SPH_C32(0x15680000), SPH_C32(0xedd70000), SPH_C32(0x325d0220),
	  SPH_C32(0xe30c3689), SPH_C32(0x5a4ae643), SPH_C32(0xe375f8a8),
	  SPH_C32(0x81fdf908) },
	{ SPH_C32(0xb4310000), SPH_C32(0x77330000), SPH_C32(0xb15d0000),
	  SPH_C32(0x7fd004e0), SPH_C32(0x78a26138), SPH_C32(0xd116c35d),
	  SPH_C32(0xd256d489), SPH_C32(0x4e6f74de), SPH_C32(0xe3060000),
	  SPH_C32(0xbdc10000), SPH_C32(0x87130000), SPH_C32(0xbff20060),
	  SPH_C32(0x2eba0a1a), SPH_C32(0x8db53751), SPH_C32(0x73c5ab06),
	  SPH_C32(0x5bd61539) },
	{ SPH_C32(0xe3060000), SPH_C32(0xbdc10000), SPH_C32(0x87130000),
	  SPH_C32(0xbff20060), SPH_C32(0x2eba0a1a), SPH_C32(0x8db53751),
	  SPH_C32(0x73c5ab06), SPH_C32(0x5bd61539), SPH_C32(0x57370000),
	  SPH_C32(0xcaf20000), SPH_C32(0x364e0000), SPH_C32(0xc0220480),
	  SPH_C32(0x56186b22), SPH_C32(0x5ca3f40c), SPH_C32(0xa1937f8f),
	  SPH_C32(0x15b961e7) },
	{ SPH_C32(0x02f20000), SPH_C32(0xa2810000), SPH_C32(0x873f0000),
	  SPH_C32(0xe36c7800), SPH_C32(0x1e1d74ef), SPH_C32(0x073d2bd6),
	  SPH_C32(0xc4c23237), SPH_C32(0x7f32259e), SPH_C32(0xbadd0000),
	  SPH_C32(0x13ad0000), SPH_C32(0xb7e70000), SPH_C32(0xf7282800),
	  SPH_C32(0xdf45144d), SPH_C32(0x361ac33a), SPH_C32(0xea5a8d14),
	  SPH_C32(0x2a2c18f0) },
	{ SPH_C32(0xbadd0000), SPH_C32(0x13ad0000), SPH_C32(0xb7e70000),
	  SPH_C32(0xf7282800), SPH_C32(0xdf45144d), SPH_C32(0x361ac33a),
	  SPH_C32(0xea5a8d14), SPH_C32(0x2a2c18f0), SPH_C32(0xb82f0000),
	  SPH_C32(0xb12c0000), SPH_C32(0x30d80000), SPH_C32(0x14445000),
	  SPH_C32(0xc15860a2), SPH_C32(0x3127e8ec), SPH_C32(0x2e98bf23),
	  SPH_C32(0x551e3d6e) },
	{ SPH_C32(0x1e6c0000), SPH_C32(0xc4420000), SPH_C32(0x8a2e0000),
	  SPH_C32(0xbcb6b800), SPH_C32(0x2c4413b6), SPH_C32(0x8bfdd3da),
	  SPH_C32(0x6a0c1bc8), SPH_C32(0xb99dc2eb), SPH_C32(0x92560000),
	  SPH_C32(0x1eda0000), SPH_C32(0xea510000), SPH_C32(0xe8b13000),
	  SPH_C32(0xa93556a5), SPH_C32(0xebfb6199), SPH_C32(0xb15c2254),
	  SPH_C32(0x33c5244f) },
	{ SPH_C32(0x92560000), SPH_C32(0x1eda0000), SPH_C32(0xea510000),
	  SPH_C32(0xe8b13000), SPH_C32(0xa93556a5), SPH_C32(0xebfb6199),
	  SPH_C32(0xb15c2254), SPH_C32(0x33c5244f), SPH_C32(0x8c3a0000),
	  SPH_C32(0xda980000), SPH_C32(0x607f0000), SPH_C32(0x54078800),
	  SPH_C32(0x85714513), SPH_C32(0x6006b243), SPH_C32(0xdb50399c),
	  SPH_C32(0x8a58e6a4) },
	{ SPH_C32(0x033d0000), SPH_C32(0x08b30000), SPH_C32(0xf33a0000),
	  SPH_C32(0x3ac20007), SPH_C32(0x51298a50), SPH_C32(0x6b6e661f),
	  SPH_C32(0x0ea5cfe3), SPH_C32(0xe6da7ffe), SPH_C32(0xa8da0000),
	  SPH_C32(0x96be0000), SPH_C32(0x5c1d0000), SPH_C32(0x07da0002),
	  SPH_C32(0x7d669583), SPH_C32(0x1f98708a), SPH_C32(0xbb668808),
	  SPH_C32(0xda878000) },
	{ SPH_C32(0xa8da0000), SPH_C32(0x96be0000), SPH_C32(0x5c1d0000),
	  SPH_C32(0x07da0002), SPH_C32(0x7d669583), SPH_C32(0x1f98708a),
	  SPH_C32(0xbb668808), SPH_C32(0xda878000), SPH_C32(0xabe70000),
	  SPH_C32(0x9e0d0000), SPH_C32(0xaf270000), SPH_C32(0x3d180005),
	  SPH_C32(0x2c4f1fd3), SPH_C32(0x74f61695), SPH_C32(0xb5c347eb),
	  SPH_C32(0x3c5dfffe) },
	{ SPH_C32(0x01930000), SPH_C32(0xe7820000), SPH_C32(0xedfb0000),
	  SPH_C32(0xcf0c000b), SPH_C32(0x8dd08d58), SPH_C32(0xbca3b42e),
	  SPH_C32(0x063661e1), SPH_C32(0x536f9e7b), SPH_C32(0x92280000),
	  SPH_C32(0xdc850000), SPH_C32(0x57fa0000), SPH_C32(0x56dc0003),
	  SPH_C32(0xbae92316), SPH_C32(0x5aefa30c), SPH_C32(0x90cef752),
	  SPH_C32(0x7b1675d7) },
	{ SPH_C32(0x92280000), SPH_C32(0xdc850000), SPH_C32(0x57fa0000),
	  SPH_C32(0x56dc0003), SPH_C32(0xbae92316), SPH_C32(0x5aefa30c),
	  SPH_C32(0x90cef752), SPH_C32(0x7b1675d7), SPH_C32(0x93bb0000),
	  SPH_C32(0x3b070000), SPH_C32(0xba010000), SPH_C32(0x99d00008),
	  SPH_C32(0x3739ae4e), SPH_C32(0xe64c1722), SPH_C32(0x96f896b3),
	  SPH_C32(0x2879ebac) },
	{ SPH_C32(0x5fa80000), SPH_C32(0x56030000), SPH_C32(0x43ae0000),
	  SPH_C32(0x64f30013), SPH_C32(0x257e86bf), SPH_C32(0x1311944e),
	  SPH_C32(0x541e95bf), SPH_C32(0x8ea4db69), SPH_C32(0x00440000),
	  SPH_C32(0x7f480000), SPH_C32(0xda7c0000), SPH_C32(0x2a230001),
	  SPH_C32(0x3badc9cc), SPH_C32(0xa9b69c87), SPH_C32(0x030a9e60),
	  SPH_C32(0xbe0a679e) },
	{ SPH_C32(0x00440000), SPH_C32(0x7f480000), SPH_C32(0xda7c0000),
	  SPH_C32(0x2a230001), SPH_C32(0x3badc9cc), SPH_C32(0xa9b69c87),
	  SPH_C32(0x030a9e60), SPH_C32(0xbe0a679e), SPH_C32(0x5fec0000),
	  SPH_C32(0x294b0000), SPH_C32(0x99d20000), SPH_C32(0x4ed00012),
	  SPH_C32(0x1ed34f73), SPH_C32(0xbaa708c9), SPH_C32(0x57140bdf),
	  SPH_C32(0x30aebcf7) },
	{ SPH_C32(0xee930000), SPH_C32(0xd6070000), SPH_C32(0x92c10000),
	  SPH_C32(0x2b9801e0), SPH_C32(0x9451287c), SPH_C32(0x3b6cfb57),
	  SPH_C32(0x45312374), SPH_C32(0x201f6a64), SPH_C32(0x7b280000),
	  SPH_C32(0x57420000), SPH_C32(0xa9e50000), SPH_C32(0x634300a0),
	  SPH_C32(0x9edb442f), SPH_C32(0x6d9995bb), SPH_C32(0x27f83b03),
	  SPH_C32(0xc7ff60f0) },
	{ SPH_C32(0x7b280000), SPH_C32(0x57420000), SPH_C32(0xa9e50000),
	  SPH_C32(0x634300a0), SPH_C32(0x9edb442f), SPH_C32(0x6d9995bb),
	  SPH_C32(0x27f83b03), SPH_C32(0xc7ff60f0), SPH_C32(0x95bb0000),
	  SPH_C32(0x81450000), SPH_C32(0x3b240000), SPH_C32(0x48db0140),
	  SPH_C32(0x0a8a6c53), SPH_C32(0x56f56eec), SPH_C32(0x62c91877),
	  SPH_C32(0xe7e00a94) }
};

__global__
void x13_hamsi512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *Hash = (uint32_t*)&g_hash[hashPosition<<3];
		unsigned char *h1 = (unsigned char *)Hash;

		uint32_t c0 = SPH_C32(0x73746565), c1 = SPH_C32(0x6c706172), c2 = SPH_C32(0x6b204172), c3 = SPH_C32(0x656e6265);
		uint32_t c4 = SPH_C32(0x72672031), c5 = SPH_C32(0x302c2062), c6 = SPH_C32(0x75732032), c7 = SPH_C32(0x3434362c);
		uint32_t c8 = SPH_C32(0x20422d33), c9 = SPH_C32(0x30303120), cA = SPH_C32(0x4c657576), cB = SPH_C32(0x656e2d48);
		uint32_t cC = SPH_C32(0x65766572), cD = SPH_C32(0x6c65652c), cE = SPH_C32(0x2042656c), cF = SPH_C32(0x6769756d);
		uint32_t m0, m1, m2, m3, m4, m5, m6, m7, m8, m9, mA, mB, mC, mD, mE, mF;
		uint32_t h[16] = { c0, c1, c2, c3, c4, c5, c6, c7, c8, c9, cA, cB, cC, cD, cE, cF };
		uint32_t *tp, db, dm;

		for(int i = 0; i < 64; i += 8) {

			m0 = 0; m1 = 0; m2 = 0; m3 = 0; m4 = 0; m5 = 0; m6 = 0; m7 = 0;
			m8 = 0; m9 = 0; mA = 0; mB = 0; mC = 0; mD = 0; mE = 0; mF = 0;
			tp = &d_T512[0][0];

			#pragma unroll 2
			for (int u = 0; u < 8; u ++) {
				db = h1[i+u];
				#pragma unroll 2
				for (int v = 0; v < 8; v ++, db >>= 1) {
					dm = -(uint32_t)(db & 1);
					m0 ^= dm & *(tp+ 0); m1 ^= dm & *(tp+ 1);
					m2 ^= dm & *(tp+ 2); m3 ^= dm & *(tp+ 3);
					m4 ^= dm & *(tp+ 4); m5 ^= dm & *(tp+ 5);
					m6 ^= dm & *(tp+ 6); m7 ^= dm & *(tp+ 7);
					m8 ^= dm & *(tp+ 8); m9 ^= dm & *(tp+ 9);
					mA ^= dm & *(tp+10); mB ^= dm & *(tp+11);
					mC ^= dm & *(tp+12); mD ^= dm & *(tp+13);
					mE ^= dm & *(tp+14); mF ^= dm & *(tp+15);
					tp += 16;
				}
			}

			for( int r = 0; r < 6; r += 2 ) {
				ROUND_BIG(r, d_alpha_n);
				ROUND_BIG(r+1, d_alpha_n);
			}
			T_BIG;
		}

		tp = &d_T512[0][0] + 112;

		m0 = *(tp+ 0); m1 = *(tp+ 1);
		m2 = *(tp+ 2); m3 = *(tp+ 3);
		m4 = *(tp+ 4); m5 = *(tp+ 5);
		m6 = *(tp+ 6); m7 = *(tp+ 7);
		m8 = *(tp+ 8); m9 = *(tp+ 9);
		mA = *(tp+10); mB = *(tp+11);
		mC = *(tp+12); mD = *(tp+13);
		mE = *(tp+14); mF = *(tp+15);

		for( int r = 0; r < 6; r += 2 ) {
			ROUND_BIG(r, d_alpha_n);
			ROUND_BIG(r+1, d_alpha_n);
		}
		T_BIG;

		tp = &d_T512[0][0] + 784;

		m0 = *(tp+ 0); m1 = *(tp+ 1);
		m2 = *(tp+ 2); m3 = *(tp+ 3);
		m4 = *(tp+ 4); m5 = *(tp+ 5);
		m6 = *(tp+ 6); m7 = *(tp+ 7);
		m8 = *(tp+ 8); m9 = *(tp+ 9);
		mA = *(tp+10); mB = *(tp+11);
		mC = *(tp+12); mD = *(tp+13);
		mE = *(tp+14); mF = *(tp+15);

		for( int r = 0; r < 12; r += 2 ) {
			ROUND_BIG(r, d_alpha_f);
			ROUND_BIG(r+1, d_alpha_f);
		}
		T_BIG;

		#pragma unroll 16
		for (int i = 0; i < 16; i++)
			Hash[i] = cuda_swab32(h[i]);
	}
}

__host__
void x13_hamsi512_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_alpha_n), alpha_n, sizeof(uint32_t)*32, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_alpha_f), alpha_f, sizeof(uint32_t)*32, 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_T512), T512, sizeof(uint32_t)*64*16, 0, hipMemcpyHostToDevice));
}

__host__
void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x13_hamsi512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	//MyStreamSynchronize(NULL, order, thr_id);
}
