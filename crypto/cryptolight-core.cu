#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>

#include "cryptolight.h"
#define LONG_SHL_IDX 18
#define LONG_LOOPS32 0x40000

#include "cn_aes.cuh"

#define MUL_SUM_XOR_DST(a,c,dst) { \
	uint64_t hi, lo = cuda_mul128(((uint64_t *)a)[0], ((uint64_t *)dst)[0], &hi) + ((uint64_t *)c)[1]; \
	hi += ((uint64_t *)c)[0]; \
	((uint64_t *)c)[0] = ((uint64_t *)dst)[0] ^ hi; \
	((uint64_t *)c)[1] = ((uint64_t *)dst)[1] ^ lo; \
	((uint64_t *)dst)[0] = hi; \
	((uint64_t *)dst)[1] = lo; }

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
	*product_hi = __umul64hi(multiplier, multiplicand);
	return(multiplier * multiplicand);
}

__global__
void cryptolight_core_gpu_phase1(int threads, uint32_t * long_state, uint32_t * ctx_state, uint32_t * ctx_key1)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		const int oft = thread * 50 + sub + 16; // not aligned 16!
		const int long_oft = (thread << LONG_SHL_IDX) + sub;
		uint32_t __align__(16) key[40];
		uint32_t __align__(16) text[4];

		// copy 160 bytes
		#pragma unroll
		for (int i = 0; i < 40; i += 4)
			AS_UINT4(&key[i]) = AS_UINT4(ctx_key1 + thread * 40 + i);

		AS_UINT2(&text[0]) = AS_UINT2(&ctx_state[oft]);
		AS_UINT2(&text[2]) = AS_UINT2(&ctx_state[oft + 2]);

		__syncthreads();
		for(int i = 0; i < LONG_LOOPS32; i += 32) {
			cn_aes_pseudo_round_mut(sharedMemory, text, key);
			AS_UINT4(&long_state[long_oft + i]) = AS_UINT4(text);
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__global__
void cryptolight_old_gpu_phase2(const int threads, const int bfactor, const int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	__syncthreads();

#if 0 && __CUDA_ARCH__ >= 300

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	const int sub = threadIdx.x & 3;

	if(thread < threads)
	{
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		uint32_t * __restrict__ long_state = &d_long_state[thread << LONG_SHL_IDX];
		uint32_t * __restrict__ ctx_a = d_ctx_a + thread * 4;
		uint32_t * __restrict__ ctx_b = d_ctx_b + thread * 4;
		uint32_t a, b, c, x[4];
		uint32_t t1[4], t2[4], res;
		uint64_t reshi, reslo;
		int j;

		a = ctx_a[sub];
		b = ctx_b[sub];

		#pragma unroll 8
		for(int i = start; i < end; ++i)
		{
			//j = ((uint32_t *)a)[0] & 0xFFFF0;
			j = (__shfl((int)a, 0, 4) & E2I_MASK1) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			c = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0xFFFF0]);
			j = (__shfl((int)c, 0, 4) & E2I_MASK1) >> 2;
			#pragma unroll
			for(int k = 0; k < 2; k++)
				t1[k] = __shfl((int)c, k, 4);
			#pragma unroll
			for(int k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;

			//j = ((uint32_t *)a)[0] & 0xFFFF0;
			j = (__shfl((int)a, 0, 4) & E2I_MASK1) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			b = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(b, c, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0xFFFF0]);
			j = (__shfl((int)b, 0, 4) & E2I_MASK1) >> 2;

			#pragma unroll
			for(int k = 0; k < 2; k++)
				t1[k] = __shfl((int)b, k, 4);

			#pragma unroll
			for(int k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;
		}

		if(bfactor > 0)
		{
			ctx_a[sub] = a;
			ctx_b[sub] = b;
		}
	}

#else // __CUDA_ARCH__ < 300

	const int thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		const int longptr = thread << LONG_SHL_IDX;
		uint32_t * long_state = &d_long_state[longptr];

		uint64_t * ctx_a = (uint64_t*)(&d_ctx_a[thread * 4]);
		uint64_t * ctx_b = (uint64_t*)(&d_ctx_b[thread * 4]);
		uint4 A = AS_UINT4(ctx_a);
		uint4 B = AS_UINT4(ctx_b);
		uint32_t* a = (uint32_t*)&A;
		uint32_t* b = (uint32_t*)&B;

		for (int i = start; i < end; i++) // end = 262144
		{
			uint32_t c[4];
			uint32_t j = (a[0] >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			XOR_BLOCKS_DST(c, b, &long_state[j]);
			MUL_SUM_XOR_DST(c, a, &long_state[(c[0] >> 2) & E2I_MASK2]);

			j = (a[0] >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			XOR_BLOCKS_DST(b, c, &long_state[j]);
			MUL_SUM_XOR_DST(b, a, &long_state[(b[0] >> 2) & E2I_MASK2]);
		}

		if (bfactor > 0) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
#endif // __CUDA_ARCH__ >= 300
}

__device__ __forceinline__ void store_variant1(uint32_t* long_state)
{
	uint4* Z = (uint4*) long_state;
	const uint32_t tmp = (Z->z >> 24); // __byte_perm(src, 0, 0x7773);
	const uint32_t index = (((tmp >> 3) & 6u) | (tmp & 1u)) << 1;
	Z->z = (Z->z & 0x00ffffffu) | ((tmp ^ ((0x75310u >> index) & 0x30u)) << 24);
}

#define MUL_SUM_XOR_DST_1(a,c,dst,tweak) { \
        uint64_t hi, lo = cuda_mul128(((uint64_t *)a)[0], ((uint64_t *)dst)[0], &hi) + ((uint64_t *)c)[1]; \
        hi += ((uint64_t *)c)[0]; \
        ((uint64_t *)c)[0] = ((uint64_t *)dst)[0] ^ hi; \
        ((uint64_t *)c)[1] = ((uint64_t *)dst)[1] ^ lo; \
        ((uint64_t *)dst)[0] = hi; \
        ((uint64_t *)dst)[1] = lo ^ tweak; }

__global__
void cryptolight_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint32_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint64_t * __restrict__ d_tweak)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;
		const uint32_t longptr = thread << LONG_SHL_IDX;
		uint32_t * long_state = &d_long_state[longptr];
		uint64_t tweak = d_tweak[thread];

		void * ctx_a = (void*)(&d_ctx_a[thread << 2]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);
		uint32_t* a = (uint32_t*)&A;
		uint32_t* b = (uint32_t*)&B;

		for (int i = start; i < end; i++)
		{
			uint32_t c[4];
			uint32_t j = (A.x >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			XOR_BLOCKS_DST(c, b, &long_state[j]);
			store_variant1(&long_state[j]);
			MUL_SUM_XOR_DST_1(c, a, &long_state[(c[0] >> 2) & E2I_MASK2], tweak);

			j = (A.x >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			XOR_BLOCKS_DST(b, c, &long_state[j]);
			store_variant1(&long_state[j]);
			MUL_SUM_XOR_DST_1(b, a, &long_state[(b[0] >> 2) & E2I_MASK2], tweak);
		}
		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

__global__
void cryptolight_core_gpu_phase3(int threads, const uint32_t * long_state, uint32_t * ctx_state, uint32_t * ctx_key2)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		const int long_oft = (thread << LONG_SHL_IDX) + sub;
		const int oft = thread * 50 + sub + 16;
		uint32_t __align__(16) key[40];
		uint32_t __align__(16) text[4];

		#pragma unroll
		for (int i = 0; i < 40; i += 4)
			AS_UINT4(&key[i]) = AS_UINT4(ctx_key2 + thread * 40 + i);

		AS_UINT2(&text[0]) = AS_UINT2(&ctx_state[oft + 0]);
		AS_UINT2(&text[2]) = AS_UINT2(&ctx_state[oft + 2]);

		__syncthreads();
		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			#pragma unroll
			for(int j = 0; j < 4; j++)
				text[j] ^= long_state[long_oft + i + j];

			cn_aes_pseudo_round_mut(sharedMemory, text, key);
		}

		AS_UINT2(&ctx_state[oft + 0]) = AS_UINT2(&text[0]);
		AS_UINT2(&ctx_state[oft + 2]) = AS_UINT2(&text[2]);
	}
}

extern int device_bfactor[MAX_GPUS];

__host__
void cryptolight_core_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, uint32_t *d_ctx_state,
	uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2, int variant, uint64_t *d_ctx_tweak)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	const int bfactor = device_bfactor[thr_id];
	const int bsleep = bfactor ? 100 : 0;

	int i, partcount = 1 << bfactor;
	int dev_id = device_map[thr_id];

	cryptolight_core_gpu_phase1 <<<grid, block8 >>>(blocks*threads, d_long_state, d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	if(partcount > 1) usleep(bsleep);

	for(i = 0; i < partcount; i++)
	{
		dim3 b = device_sm[dev_id] >= 300 ? block4 : block;
		if (variant == 0)
			cryptolight_old_gpu_phase2 <<<grid, b>>> (blocks*threads, bfactor, i, d_long_state, d_ctx_a, d_ctx_b);
		else
			cryptolight_gpu_phase2 <<<grid, b>>> (blocks*threads, bfactor, i, d_long_state, d_ctx_a, d_ctx_b, d_ctx_tweak);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		if(partcount > 1) usleep(bsleep);
	}

	cryptolight_core_gpu_phase3 <<<grid, block8 >>>(blocks*threads, d_long_state, d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}
