#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>

#include "cryptolight.h"
#define LONG_SHL_IDX 18
#define LONG_LOOPS32 0x40000

#include "cn_aes.cuh"

#define MUL_SUM_XOR_DST(a,c,dst) { \
	uint64_t hi, lo = cuda_mul128(((uint64_t *)a)[0], ((uint64_t *)dst)[0], &hi) + ((uint64_t *)c)[1]; \
	hi += ((uint64_t *)c)[0]; \
	((uint64_t *)c)[0] = ((uint64_t *)dst)[0] ^ hi; \
	((uint64_t *)c)[1] = ((uint64_t *)dst)[1] ^ lo; \
	((uint64_t *)dst)[0] = hi; \
	((uint64_t *)dst)[1] = lo; }

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
	*product_hi = __umul64hi(multiplier, multiplicand);
	return(multiplier * multiplicand);
}

__global__
void cryptolight_core_gpu_phase1(int threads, uint32_t * long_state, uint32_t * ctx_state, uint32_t * ctx_key1)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		const int oft = thread * 52 + sub + 16; // not aligned 16!
		const int long_oft = (thread << LONG_SHL_IDX) + sub;
		uint32_t __align__(16) key[40];
		uint32_t __align__(16) text[4];

		// copy 160 bytes
		#pragma unroll
		for (int i = 0; i < 40; i += 4)
			AS_UINT4(&key[i]) = AS_UINT4(ctx_key1 + thread * 40 + i);

		AS_UINT2(&text[0]) = AS_UINT2(&ctx_state[oft]);
		AS_UINT2(&text[2]) = AS_UINT2(&ctx_state[oft + 2]);

		__syncthreads();
		for(int i = 0; i < LONG_LOOPS32; i += 32) {
			cn_aes_pseudo_round_mut(sharedMemory, text, key);
			AS_UINT4(&long_state[long_oft + i]) = AS_UINT4(text);
		}
	}
}

__global__
void cryptolight_core_gpu_phase2(const int threads, const int bfactor, const int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	__syncthreads();

#if 0 && __CUDA_ARCH__ >= 300

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	const int sub = threadIdx.x & 3;

	if(thread < threads)
	{
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		uint32_t * __restrict__ long_state = &d_long_state[thread << LONG_SHL_IDX];
		uint32_t * __restrict__ ctx_a = d_ctx_a + thread * 4;
		uint32_t * __restrict__ ctx_b = d_ctx_b + thread * 4;
		uint32_t a, b, c, x[4];
		uint32_t t1[4], t2[4], res;
		uint64_t reshi, reslo;
		int j;

		a = ctx_a[sub];
		b = ctx_b[sub];

		#pragma unroll 8
		for(int i = start; i < end; ++i)
		{
			//j = ((uint32_t *)a)[0] & 0xFFFF0;
			j = (__shfl((int)a, 0, 4) & E2I_MASK1) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			c = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0xFFFF0]);
			j = (__shfl((int)c, 0, 4) & E2I_MASK1) >> 2;
			#pragma unroll
			for(int k = 0; k < 2; k++)
				t1[k] = __shfl((int)c, k, 4);
			#pragma unroll
			for(int k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;

			//j = ((uint32_t *)a)[0] & 0xFFFF0;
			j = (__shfl((int)a, 0, 4) & E2I_MASK1) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			b = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(b, c, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0xFFFF0]);
			j = (__shfl((int)b, 0, 4) & E2I_MASK1) >> 2;

			#pragma unroll
			for(int k = 0; k < 2; k++)
				t1[k] = __shfl((int)b, k, 4);

			#pragma unroll
			for(int k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;
		}

		if(bfactor > 0)
		{
			ctx_a[sub] = a;
			ctx_b[sub] = b;
		}
	}

#else // __CUDA_ARCH__ < 300

	const int thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		const int longptr = thread << LONG_SHL_IDX;
		uint32_t * long_state = &d_long_state[longptr];

		uint64_t * ctx_a = (uint64_t*)(&d_ctx_a[thread * 4]);
		uint64_t * ctx_b = (uint64_t*)(&d_ctx_b[thread * 4]);
		uint4 A = AS_UINT4(ctx_a);
		uint4 B = AS_UINT4(ctx_b);
		uint32_t* a = (uint32_t*)&A;
		uint32_t* b = (uint32_t*)&B;

		for (int i = start; i < end; i++) // end = 262144
		{
			uint32_t c[4];
			uint32_t j = (a[0] >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			XOR_BLOCKS_DST(c, b, &long_state[j]);
			MUL_SUM_XOR_DST(c, a, &long_state[(c[0] >> 2) & E2I_MASK2]);

			j = (a[0] >> 2) & E2I_MASK2;
			cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			XOR_BLOCKS_DST(b, c, &long_state[j]);
			MUL_SUM_XOR_DST(b, a, &long_state[(b[0] >> 2) & E2I_MASK2]);
		}

		if (bfactor > 0) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
#endif // __CUDA_ARCH__ >= 300
}

__global__
void cryptolight_core_gpu_phase3(int threads, const uint32_t * long_state, uint32_t * ctx_state, uint32_t * ctx_key2)
{
	__shared__ uint32_t __align__(16) sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		const int long_oft = (thread << LONG_SHL_IDX) + sub;
		const int oft = thread * 52 + sub + 16;
		uint32_t __align__(16) key[40];
		uint32_t __align__(16) text[4];

		#pragma unroll
		for (int i = 0; i < 40; i += 4)
			AS_UINT4(&key[i]) = AS_UINT4(ctx_key2 + thread * 40 + i);

		AS_UINT2(&text[0]) = AS_UINT2(&ctx_state[oft + 0]);
		AS_UINT2(&text[2]) = AS_UINT2(&ctx_state[oft + 2]);

		__syncthreads();
		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			#pragma unroll
			for(int j = 0; j < 4; j++)
				text[j] ^= long_state[long_oft + i + j];

			cn_aes_pseudo_round_mut(sharedMemory, text, key);
		}

		AS_UINT2(&ctx_state[oft + 0]) = AS_UINT2(&text[0]);
		AS_UINT2(&ctx_state[oft + 2]) = AS_UINT2(&text[2]);
	}
}

extern int device_bfactor[MAX_GPUS];

__host__
void cryptolight_core_cpu_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, uint64_t *d_ctx_state,
	uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	const int bfactor = device_bfactor[thr_id];
	const int bsleep = bfactor ? 100 : 0;

	int i, partcount = 1 << bfactor;
	int dev_id = device_map[thr_id];

	cryptolight_core_gpu_phase1 <<<grid, block8 >>>(blocks*threads, d_long_state, (uint32_t*)d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	if(partcount > 1) usleep(bsleep);

	for(i = 0; i < partcount; i++)
	{
		cryptolight_core_gpu_phase2 <<<grid, (device_sm[dev_id] >= 300 ? block4 : block)>>>(blocks*threads, bfactor, i, d_long_state, d_ctx_a, d_ctx_b);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		if(partcount > 1) usleep(bsleep);
	}

	cryptolight_core_gpu_phase3 <<<grid, block8 >>>(blocks*threads, d_long_state, (uint32_t*)d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}
