
#include "cryptolight.h"

extern char *device_config[MAX_GPUS]; // -l 32x16

static __thread uint32_t cn_blocks  = 32;
static __thread uint32_t cn_threads = 16;

static uint32_t *d_long_state[MAX_GPUS];
static uint64_t *d_ctx_state[MAX_GPUS];
static uint32_t *d_ctx_key1[MAX_GPUS];
static uint32_t *d_ctx_key2[MAX_GPUS];
static uint32_t *d_ctx_text[MAX_GPUS];
static uint32_t *d_ctx_a[MAX_GPUS];
static uint32_t *d_ctx_b[MAX_GPUS];

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_cryptolight(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	int res = 0;
	uint32_t throughput = 0;

	uint32_t *ptarget = work->target;
	uint8_t *pdata = (uint8_t*) work->data;
	uint32_t *nonceptr = (uint32_t*) (&pdata[39]);
	const uint32_t first_nonce = *nonceptr;
	uint32_t nonce = first_nonce;

	if(opt_benchmark) {
		ptarget[7] = 0x00ff;
	}

	if(!init[thr_id])
	{
		if (device_config[thr_id]) {
			sscanf(device_config[thr_id], "%ux%u", &cn_blocks, &cn_threads);
			throughput = cuda_default_throughput(thr_id, cn_blocks*cn_threads);
			gpulog(LOG_INFO, thr_id, "Using %u x %u kernel launch config, %u threads",
				cn_blocks, cn_threads, throughput);
		} else {
			throughput = cuda_default_throughput(thr_id, cn_blocks*cn_threads);
			if (throughput != cn_blocks*cn_threads && cn_threads) {
				cn_blocks = throughput / cn_threads;
				throughput = cn_threads * cn_blocks;
			}
			gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u threads (%ux%u)",
				throughput2intensity(throughput), throughput, cn_blocks, cn_threads);
		}

		if(sizeof(size_t) == 4 && throughput > UINT32_MAX / MEMORY) {
			gpulog(LOG_ERR, thr_id, "THE 32bit VERSION CAN'T ALLOCATE MORE THAN 4GB OF MEMORY!");
			gpulog(LOG_ERR, thr_id, "PLEASE REDUCE THE NUMBER OF THREADS OR BLOCKS");
			exit(1);
		}

		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		const size_t alloc = MEMORY * throughput;
		cryptonight_extra_cpu_init(thr_id, throughput);

		hipMalloc(&d_long_state[thr_id], alloc);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_state[thr_id], 26 * sizeof(uint64_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_key1[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_key2[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_text[thr_id], 32 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_a[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_b[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);

		init[thr_id] = true;
	}

	throughput = cn_blocks*cn_threads;

	do
	{
		const uint32_t Htarg = ptarget[7];
		uint32_t resNonces[2] = { UINT32_MAX, UINT32_MAX };

		cryptonight_extra_cpu_setData(thr_id, pdata, ptarget);
		cryptonight_extra_cpu_prepare(thr_id, throughput, nonce, d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id]);
		cryptolight_core_cpu_hash(thr_id, cn_blocks, cn_threads, d_long_state[thr_id], d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id]);
		cryptonight_extra_cpu_final(thr_id, throughput, nonce, resNonces, d_ctx_state[thr_id]);

		*hashes_done = nonce - first_nonce + throughput;

		if(resNonces[0] != UINT32_MAX)
		{
			uint32_t vhash[8];
			uint32_t tempdata[19];
			uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
			memcpy(tempdata, pdata, 76);
			*tempnonceptr = resNonces[0];
			cryptolight_hash(vhash, tempdata, 76);
			if(vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				res = 1;
				work->nonces[0] = resNonces[0];
				work_set_target_ratio(work, vhash);
				// second nonce
				if(resNonces[1] != UINT32_MAX)
				{
					*tempnonceptr = resNonces[1];
					cryptolight_hash(vhash, tempdata, 76);
					if(vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
						res++;
						work->nonces[1] = resNonces[1];
					} else if (vhash[7] > Htarg) {
						gpu_increment_reject(thr_id);
					}
				}
				goto done;
			} else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for nonce %08x does not validate on CPU!", resNonces[0]);
			}
		}

		if ((uint64_t) throughput + nonce >= max_nonce - 127) {
			nonce = max_nonce;
			break;
		}

		nonce += throughput;
		gpulog(LOG_DEBUG, thr_id, "nonce %08x", nonce);

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + nonce);

done:
	gpulog(LOG_DEBUG, thr_id, "nonce %08x exit", nonce);
	work->valid_nonces = res;
	*nonceptr = nonce;
	return res;
}

void free_cryptolight(int thr_id)
{
	if (!init[thr_id])
		return;

	hipFree(d_long_state[thr_id]);
	hipFree(d_ctx_state[thr_id]);
	hipFree(d_ctx_key1[thr_id]);
	hipFree(d_ctx_key2[thr_id]);
	hipFree(d_ctx_text[thr_id]);
	hipFree(d_ctx_a[thr_id]);
	hipFree(d_ctx_b[thr_id]);

	cryptonight_extra_cpu_free(thr_id);

	hipDeviceSynchronize();

	init[thr_id] = false;
}
