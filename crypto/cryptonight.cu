
#include "cryptonight.h"

extern char *device_config[MAX_GPUS]; // -l 32x16

static __thread uint32_t cn_blocks;
static __thread uint32_t cn_threads;

// used for gpu intensity on algo init
static __thread bool gpu_init_shown = false;
#define gpulog_init(p,thr,fmt, ...) if (!gpu_init_shown) \
	gpulog(p, thr, fmt, ##__VA_ARGS__)

static uint64_t *d_long_state[MAX_GPUS];
static uint32_t *d_ctx_state[MAX_GPUS];
static uint32_t *d_ctx_key1[MAX_GPUS];
static uint32_t *d_ctx_key2[MAX_GPUS];
static uint32_t *d_ctx_text[MAX_GPUS];
static uint64_t *d_ctx_tweak[MAX_GPUS];
static uint32_t *d_ctx_a[MAX_GPUS];
static uint32_t *d_ctx_b[MAX_GPUS];

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_cryptonight(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, int variant)
{
	int res = 0;
	uint32_t throughput = 0;

	uint32_t *ptarget = work->target;
	uint8_t *pdata = (uint8_t*) work->data;
	uint32_t *nonceptr = (uint32_t*) (&pdata[39]);
	const uint32_t first_nonce = *nonceptr;
	uint32_t nonce = first_nonce;
	int dev_id = device_map[thr_id];

	if(opt_benchmark) {
		ptarget[7] = 0x00ff;
	}

	if(!init[thr_id])
	{
		int mem = cuda_available_memory(thr_id);
		int mul = device_sm[dev_id] >= 300 ? 4 : 1; // see cryptonight-core.cu
		cn_threads = device_sm[dev_id] >= 600 ? 16 : 8; // real TPB is x4 on SM3+
		cn_blocks = device_mpcount[dev_id] * 4;
		if (cn_blocks*cn_threads*2.2 > mem) cn_blocks = device_mpcount[dev_id] * 2;

		if (!opt_quiet)
			gpulog_init(LOG_INFO, thr_id, "%s, %d MB available, %hd SMX", device_name[dev_id],
				mem, device_mpcount[dev_id]);

		if (!device_config[thr_id] && strcmp(device_name[dev_id], "TITAN V") == 0) {
			device_config[thr_id] = strdup("80x24");
		}

		if (device_config[thr_id]) {
			int res = sscanf(device_config[thr_id], "%ux%u", &cn_blocks, &cn_threads);
			throughput = cuda_default_throughput(thr_id, cn_blocks*cn_threads);
			gpulog_init(LOG_INFO, thr_id, "Using %ux%u(x%d) kernel launch config, %u threads",
				cn_blocks, cn_threads, mul, throughput);
		} else {
			throughput = cuda_default_throughput(thr_id, cn_blocks*cn_threads);
			if (throughput != cn_blocks*cn_threads && cn_threads) {
				cn_blocks = throughput / cn_threads;
				throughput = cn_threads * cn_blocks;
			}
			gpulog_init(LOG_INFO, thr_id, "%u threads (%g) with %u blocks",// of %ux%d",
				throughput, throughput2intensity(throughput), cn_blocks);//, cn_threads, mul);
		}

		if(sizeof(size_t) == 4 && throughput > UINT32_MAX / MEMORY) {
			gpulog(LOG_ERR, thr_id, "THE 32bit VERSION CAN'T ALLOCATE MORE THAN 4GB OF MEMORY!");
			gpulog(LOG_ERR, thr_id, "PLEASE REDUCE THE NUMBER OF THREADS OR BLOCKS");
			exit(1);
		}

		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		const size_t alloc = MEMORY * throughput;
		cryptonight_extra_init(thr_id);

		hipMalloc(&d_long_state[thr_id], alloc);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_state[thr_id], 50 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_key1[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_key2[thr_id], 40 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_text[thr_id], 32 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_a[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_b[thr_id], 4 * sizeof(uint32_t) * throughput);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		hipMalloc(&d_ctx_tweak[thr_id], sizeof(uint64_t) * throughput);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);

		gpu_init_shown = true;
		init[thr_id] = true;
	}

	throughput = cn_blocks*cn_threads;

	do
	{
		const uint32_t Htarg = ptarget[7];
		uint32_t resNonces[2] = { UINT32_MAX, UINT32_MAX };

		cryptonight_extra_setData(thr_id, pdata, ptarget);
		cryptonight_extra_prepare(thr_id, throughput, nonce, d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id], variant, d_ctx_tweak[thr_id]);
		cryptonight_core_cuda(thr_id, cn_blocks, cn_threads, d_long_state[thr_id], d_ctx_state[thr_id], d_ctx_a[thr_id], d_ctx_b[thr_id], d_ctx_key1[thr_id], d_ctx_key2[thr_id], variant, d_ctx_tweak[thr_id]);
		cryptonight_extra_final(thr_id, throughput, nonce, resNonces, d_ctx_state[thr_id]);

		*hashes_done = nonce - first_nonce + throughput;

		if(resNonces[0] != UINT32_MAX)
		{
			uint32_t vhash[8];
			uint32_t tempdata[19];
			uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
			memcpy(tempdata, pdata, 76);
			*tempnonceptr = resNonces[0];
			cryptonight_hash_variant(vhash, tempdata, 76, variant);
			if(vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				res = 1;
				work->nonces[0] = resNonces[0];
				work_set_target_ratio(work, vhash);
				// second nonce
				if(resNonces[1] != UINT32_MAX)
				{
					*tempnonceptr = resNonces[1];
					cryptonight_hash_variant(vhash, tempdata, 76, variant);
					if(vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
						res++;
						work->nonces[1] = resNonces[1];
					} else {
						gpu_increment_reject(thr_id);
					}
				}
				goto done;
			} else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for nonce %08x does not validate on CPU!", resNonces[0]);
			}
		}

		if ((uint64_t) throughput + nonce >= max_nonce - 127) {
			nonce = max_nonce;
			break;
		}

		nonce += throughput;
		gpulog(LOG_DEBUG, thr_id, "nonce %08x", nonce);

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + nonce);

done:
	gpulog(LOG_DEBUG, thr_id, "nonce %08x exit", nonce);
	work->valid_nonces = res;
	*nonceptr = nonce;
	return res;
}

void free_cryptonight(int thr_id)
{
	if (!init[thr_id])
		return;

	hipFree(d_long_state[thr_id]);
	hipFree(d_ctx_state[thr_id]);
	hipFree(d_ctx_key1[thr_id]);
	hipFree(d_ctx_key2[thr_id]);
	hipFree(d_ctx_text[thr_id]);
	hipFree(d_ctx_tweak[thr_id]);
	hipFree(d_ctx_a[thr_id]);
	hipFree(d_ctx_b[thr_id]);

	cryptonight_extra_free(thr_id);

	hipDeviceSynchronize();

	init[thr_id] = false;
}
