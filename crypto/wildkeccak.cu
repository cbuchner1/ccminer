#include "hip/hip_runtime.h"
// WildKeccak CUDA Kernel, Code based on Linux Wolf0 bbr-miner implementation from 2014
// Adapted to ccminer 2.0 - tpruvot 2016-2017
//
// NOTE FOR SP: this ccminer version is licensed under GPLv3 Licence

extern "C" {
#include <errno.h>
#include <stdio.h>
#include <unistd.h>
}

#include <miner.h>
#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h> // todo

#include "wildkeccak.h"

extern char *device_config[MAX_GPUS]; // -l
extern uint64_t* pscratchpad_buff;

static uint64_t*    d_input[MAX_GPUS];
static uint32_t*    d_retnonce[MAX_GPUS];
static ulonglong4*  d_scratchpad[MAX_GPUS];

static uint64_t*    h_scratchpad[MAX_GPUS] = { 0 };
static hipStream_t bufpad_stream[MAX_GPUS] = { 0 };
static hipStream_t kernel_stream[MAX_GPUS] = { 0 };

uint64_t scratchpad_size = 0;

uint32_t WK_CUDABlocks   = 64;
uint32_t WK_CUDAThreads  = 256;

#define st0 	vst0.x
#define st1 	vst0.y
#define st2 	vst0.z
#define st3 	vst0.w

#define st4 	vst4.x
#define st5 	vst4.y
#define st6 	vst4.z
#define st7 	vst4.w

#define st8 	vst8.x
#define st9 	vst8.y
#define st10	vst8.z
#define st11	vst8.w

#define st12	vst12.x
#define st13	vst12.y
#define st14	vst12.z
#define st15	vst12.w

#define st16	vst16.x
#define st17	vst16.y
#define st18	vst16.z
#define st19	vst16.w

#define st20	vst20.x
#define st21	vst20.y
#define st22	vst20.z
#define st23	vst20.w

#if __CUDA_ARCH__ >= 320

__device__ __forceinline__ uint64_t cuda_rotl641(const uint64_t value)
{
	uint2 result;
	asm("shf.l.wrap.b32 %0, %1, %2, 1U;" : "=r"(result.x)
		: "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))));
	asm("shf.l.wrap.b32 %0, %1, %2, 1U;" : "=r"(result.y)
		: "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))));
	return __double_as_longlong(__hiloint2double(result.y, result.x));
}

#else
__noinline__ __device__ uint64_t cuda_rotl641(const uint64_t x) { return((x << 1) | (x >> 63)); }
#endif

__noinline__ __device__ uint64_t bitselect(const uint64_t a, const uint64_t b, const uint64_t c) { return(a ^ (c & (b ^ a))); }

#define ROTL641(x) (cuda_rotl641(x))

#define RND() \
	bc[0] = st0 ^ st5 ^ st10 * st15 * st20 ^ ROTL641(st2 ^ st7 ^ st12 * st17 * st22); \
	bc[1] = st1 ^ st6 ^ st11 * st16 * st21 ^ ROTL641(st3 ^ st8 ^ st13 * st18 * st23); \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL641(st4 ^ st9 ^ st14 * st19 * st24); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL641(st0 ^ st5 ^ st10 * st15 * st20); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL641(st1 ^ st6 ^ st11 * st16 * st21); \
	tmp1 = st1 ^ bc[0]; \
	\
	st0  ^= bc[4]; \
	st1  = ROTL64(st6  ^ bc[0], 44); \
	st6  = ROTL64(st9  ^ bc[3], 20); \
	st9  = ROTL64(st22 ^ bc[1], 61); \
	st22 = ROTL64(st14 ^ bc[3], 39); \
	st14 = ROTL64(st20 ^ bc[4], 18); \
	st20 = ROTL64(st2  ^ bc[1], 62); \
	st2  = ROTL64(st12 ^ bc[1], 43); \
	st12 = ROTL64(st13 ^ bc[2], 25); \
	st13 = ROTL64(st19 ^ bc[3], 8); \
	st19 = ROTL64(st23 ^ bc[2], 56); \
	st23 = ROTL64(st15 ^ bc[4], 41); \
	st15 = ROTL64(st4  ^ bc[3], 27); \
	st4  = ROTL64(st24 ^ bc[3], 14); \
	st24 = ROTL64(st21 ^ bc[0], 2); \
	st21 = ROTL64(st8  ^ bc[2], 55); \
	st8  = ROTL64(st16 ^ bc[0], 45); \
	st16 = ROTL64(st5  ^ bc[4], 36); \
	st5  = ROTL64(st3  ^ bc[2], 28); \
	st3  = ROTL64(st18 ^ bc[2], 21); \
	st18 = ROTL64(st17 ^ bc[1], 15); \
	st17 = ROTL64(st11 ^ bc[0], 10); \
	st11 = ROTL64(st7  ^ bc[1], 6); \
	st7  = ROTL64(st10 ^ bc[4], 3); \
	st10 = ROTL641(tmp1); \
	\
	tmp1 = st0; tmp2 = st1; st0 = bitselect(st0 ^ st2, st0, st1); st1 = bitselect(st1 ^ st3, st1, st2); \
	 st2 = bitselect(st2 ^ st4, st2, st3); st3 = bitselect(st3 ^ tmp1, st3, st4); st4 = bitselect(st4 ^ tmp2, st4, tmp1); \
	tmp1 = st5; tmp2 = st6; st5 = bitselect(st5 ^ st7, st5, st6); st6 = bitselect(st6 ^ st8, st6, st7); \
	 st7 = bitselect(st7 ^ st9, st7, st8); st8 = bitselect(st8 ^ tmp1, st8, st9); st9 = bitselect(st9 ^ tmp2, st9, tmp1); \
	tmp1 = st10; tmp2 = st11; st10 = bitselect(st10 ^ st12, st10, st11); st11 = bitselect(st11 ^ st13, st11, st12); \
	st12 = bitselect(st12 ^ st14, st12, st13); st13 = bitselect(st13 ^ tmp1, st13, st14); st14 = bitselect(st14 ^ tmp2, st14, tmp1); \
	tmp1 = st15; tmp2 = st16; st15 = bitselect(st15 ^ st17, st15, st16); st16 = bitselect(st16 ^ st18, st16, st17); \
	st17 = bitselect(st17 ^ st19, st17, st18); st18 = bitselect(st18 ^ tmp1, st18, st19); st19 = bitselect(st19 ^ tmp2, st19, tmp1); \
	tmp1 = st20; tmp2 = st21; st20 = bitselect(st20 ^ st22, st20, st21); st21 = bitselect(st21 ^ st23, st21, st22); \
	st22 = bitselect(st22 ^ st24, st22, st23); st23 = bitselect(st23 ^ tmp1, st23, st24); st24 = bitselect(st24 ^ tmp2, st24, tmp1); \
	st0 ^= 1;

#define LASTRND1() \
	bc[0] = st0 ^ st5 ^ st10 * st15 * st20 ^ ROTL64(st2 ^ st7 ^ st12 * st17 * st22, 1); \
	bc[1] = st1 ^ st6 ^ st11 * st16 * st21 ^ ROTL64(st3 ^ st8 ^ st13 * st18 * st23, 1); \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL64(st4 ^ st9 ^ st14 * st19 * st24, 1); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL64(st0 ^ st5 ^ st10 * st15 * st20, 1); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL64(st1 ^ st6 ^ st11 * st16 * st21, 1); \
	\
	st0 ^= bc[4]; \
	st1 = ROTL64(st6 ^ bc[0], 44); \
	st2 = ROTL64(st12 ^ bc[1], 43); \
	st4 = ROTL64(st24 ^ bc[3], 14); \
	st3 = ROTL64(st18 ^ bc[2], 21); \
	\
	tmp1 = st0; st0 = bitselect(st0 ^ st2, st0, st1); st1 = bitselect(st1 ^ st3, st1, st2); st2 = bitselect(st2 ^ st4, st2, st3); st3 = bitselect(st3 ^ tmp1, st3, st4); \
	st0 ^= 1;

#define LASTRND2() \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL64(st4 ^ st9 ^ st14 * st19 * st24, 1); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL64(st0 ^ st5 ^ st10 * st15 * st20, 1); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL64(st1 ^ st6 ^ st11 * st16 * st21, 1); \
	\
	st0 ^= bc[4]; \
	st4 = ROTL64(st24 ^ bc[3], 14); \
	st3 = ROTL64(st18 ^ bc[2], 21); \
	st3 = bitselect(st3 ^ st0, st3, st4);

__device__ ulonglong4 operator^(const ulonglong4 &a, const ulonglong4 &b)
{
	return(make_ulonglong4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w));
}

#define MIX(vst) vst = vst ^ scratchpad[vst.x % scr_size] ^ scratchpad[vst.y % scr_size] ^ scratchpad[vst.z % scr_size] ^ scratchpad[vst.w % scr_size];

#define MIX_ALL MIX(vst0); MIX(vst4); MIX(vst8); MIX(vst12); MIX(vst16); MIX(vst20);

__global__
void wk(uint32_t* __restrict__ retnonce, const uint64_t* __restrict__ input, const ulonglong4* __restrict__ scratchpad,
	const uint32_t scr_size, const uint32_t target, uint64_t startNonce)
{
	ulonglong4 vst0, vst4, vst8, vst12, vst16, vst20;
	uint64_t bc[5];
	uint64_t st24, tmp1, tmp2;

	const uint64_t nonce = startNonce + (blockDim.x * blockIdx.x) + threadIdx.x;
	vst0  = make_ulonglong4((nonce << 8) + (input[0] & 0xFF), input[1] & 0xFFFFFFFFFFFFFF00ULL, input[2], input[3]);
	vst4  = make_ulonglong4(input[4], input[5], input[6], input[7]);
	vst8  = make_ulonglong4(input[8], input[9], (input[10] & 0xFF) | 0x100, 0);
	vst12 = make_ulonglong4(0, 0, 0, 0);
	vst16 = make_ulonglong4(0x8000000000000000ULL, 0, 0, 0);
	vst20 = make_ulonglong4(0, 0, 0, 0);
	st24  = 0;

	RND();
	MIX_ALL;

	for(int i = 0; i < 22; i++) {
		RND();
		MIX_ALL;
	}

	LASTRND1();

	vst4  = make_ulonglong4(1, 0, 0, 0);
	vst8  = make_ulonglong4(0, 0, 0, 0);
	vst12 = make_ulonglong4(0, 0, 0, 0);
	vst16 = make_ulonglong4(0x8000000000000000ULL, 0, 0, 0);
	vst20 = make_ulonglong4(0, 0, 0, 0);
	st24  = 0;

	RND();
	MIX_ALL;

	#pragma unroll
	for(int i = 0; i < 22; i++) {
		RND();
		MIX_ALL;
	}

	LASTRND2();

	if((st3 >> 32) <= target) {
		retnonce[0] = (uint32_t) nonce;
		retnonce[1] = retnonce[0];
	}
}

__host__
void wildkeccak_kernel(const int thr_id, const uint32_t threads, const uint32_t startNounce, const uint2 target, uint32_t *resNonces)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_retnonce[thr_id], 0xff, 2 * sizeof(uint32_t), kernel_stream[thr_id]));

	const uint32_t threadsperblock = WK_CUDAThreads;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	wk <<<grid, block, 0, kernel_stream[thr_id]>>> (d_retnonce[thr_id], d_input[thr_id], d_scratchpad[thr_id],
		(uint32_t)(scratchpad_size >> 2), target.y, startNounce);

	hipMemcpyAsync(resNonces, d_retnonce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost, kernel_stream[thr_id]);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_wildkeccak(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *ptarget = work->target;
	uint32_t throughput = 0;
	uint64_t n, nonce, first;
	uint8_t *pdata = (uint8_t*) work->data;
	memcpy(&first, &pdata[1], 8);
	n = nonce = first;

	if (!scratchpad_size || !h_scratchpad[thr_id]) {
		if (h_scratchpad[thr_id])
			applog(LOG_ERR, "Scratchpad size is not set!");
		work->data[0] = 0; // invalidate
		sleep(1);
		return -EBUSY;
	}

	if (!init[thr_id]) {

		if (device_config[thr_id]) {
			sscanf(device_config[thr_id], "%ux%u", &WK_CUDABlocks, &WK_CUDAThreads);
			gpulog(LOG_INFO, thr_id, "Using %u x %u kernel launch config, %u threads",
				WK_CUDABlocks, WK_CUDAThreads, throughput);
		} else {
			throughput = cuda_default_throughput(thr_id, WK_CUDABlocks*WK_CUDAThreads);
			gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		}

		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_SAFE_CALL(hipMalloc(&d_input[thr_id], 88));
		CUDA_SAFE_CALL(hipMalloc(&d_retnonce[thr_id], 2*sizeof(uint32_t)));

		int status = (int) hipMalloc(&d_scratchpad[thr_id], WILD_KECCAK_SCRATCHPAD_BUFFSIZE);
		if (status != hipSuccess) {
			gpulog(LOG_ERR, thr_id, "Unable to allocate device memory, %u MB, err %d",
				(uint32_t) (WILD_KECCAK_SCRATCHPAD_BUFFSIZE/(1024*1024)), status);
			exit(-ENOMEM);
		}

		hipStreamCreate(&bufpad_stream[thr_id]);
		hipStreamCreate(&kernel_stream[thr_id]);

		CUDA_SAFE_CALL(hipMemcpyAsync(d_scratchpad[thr_id], h_scratchpad[thr_id], scratchpad_size << 3, hipMemcpyHostToDevice, bufpad_stream[thr_id]));

		init[thr_id] = true;
	}

	throughput = WK_CUDABlocks * WK_CUDAThreads;

	hipMemcpy(d_input[thr_id], pdata, 88, hipMemcpyHostToDevice);
//	hipMemset(d_retnonce[thr_id], 0xFF, 2*sizeof(uint32_t));

	if (h_scratchpad[thr_id]) {
		hipStreamSynchronize(bufpad_stream[thr_id]);
	}

	do {
//		const uint32_t blocks = WK_CUDABlocks, threads = WK_CUDAThreads;
//		const dim3 block(blocks);
//		const dim3 thread(threads);
		uint32_t h_retnonce[2] = { UINT32_MAX, UINT32_MAX };
		uint2 target = make_uint2(ptarget[6], ptarget[7]);

		wildkeccak_kernel(thr_id, throughput, (uint32_t) nonce, target, h_retnonce);
		/*
		wk <<<block, thread, 0, kernel_stream[thr_id]>>> (d_retnonce[thr_id], d_input[thr_id], d_scratchpad[thr_id],
			(uint32_t)(scratchpad_size >> 2), nonce, ptarget[7]);
		*/

		*hashes_done = (unsigned long) (n - first + throughput);

		hipStreamSynchronize(kernel_stream[thr_id]);
		if(h_retnonce[0] != UINT32_MAX) {
			uint8_t _ALIGN(64) cpuhash[32];
			uint32_t* vhash = (uint32_t*) cpuhash;
			uint64_t nonce64;
			memcpy(&pdata[1], &h_retnonce[0], sizeof(uint32_t));
			memcpy(&nonce64, &pdata[1], 8);
			wildkeccak_hash(cpuhash, pdata, pscratchpad_buff, scratchpad_size);
			if (!cpuhash[31] && vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work_set_target_ratio(work, vhash);
				//applog_hex(pdata,   84);
				//applog_hex(cpuhash, 32);
				//applog_hex(ptarget, 32);
				memcpy(work->nonces, &nonce64, 8);
				if (n + throughput > max_nonce) {
					*hashes_done = (unsigned long) (max_nonce - first);
				}
				work->valid_nonces = 1;
				return 1;
			} else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for nonce %08x does not validate on CPU!", h_retnonce[0]);
			}
		}

		if (n + throughput >= max_nonce) {
			n = max_nonce;
			break;
		}

		n += throughput;
		nonce += throughput;

	} while(!work_restart[thr_id].restart);

	*hashes_done = (unsigned long) (n - first + 1);
	return 0;
}

void wildkeccak_scratchpad_need_update(uint64_t* pscratchpad_buff)
{
	for(int i = 0; i < opt_n_threads; i++) {
		h_scratchpad[i] = pscratchpad_buff;
		if (init[i]) {
			gpulog(LOG_DEBUG, i, "Starting scratchpad update...");
			hipMemcpyAsync(d_scratchpad[i], h_scratchpad[i], scratchpad_size << 3, hipMemcpyHostToDevice, bufpad_stream[i]);
			work_restart[i].restart = true;
		}
	}
}

void free_wildkeccak(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_scratchpad[thr_id]);
	hipFree(d_input[thr_id]);
	hipFree(d_retnonce[thr_id]);

	hipStreamDestroy(bufpad_stream[thr_id]);
	hipStreamDestroy(kernel_stream[thr_id]);

	hipDeviceSynchronize();

	init[thr_id] = false;
}
