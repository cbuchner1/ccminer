#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#ifndef _WIN32
#include <unistd.h>
#endif

#include <miner.h>
#include <cuda_helper.h>
#include "cryptonight.h"

typedef uint8_t BitSequence;
typedef uint64_t DataLength;

static uint32_t *d_input[MAX_GPUS] = { 0 };
static uint32_t *d_target[MAX_GPUS];
static uint32_t *d_result[MAX_GPUS];

#include "cn_keccak.cuh"
#include "cn_blake.cuh"
#include "cn_groestl.cuh"
#include "cn_jh.cuh"
#include "cn_skein.cuh"

__constant__ uint8_t d_sub_byte[16][16] = {
	{0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76},
	{0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0},
	{0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15},
	{0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75},
	{0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84},
	{0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf},
	{0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8},
	{0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2},
	{0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73},
	{0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb},
	{0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79},
	{0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08},
	{0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a},
	{0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e},
	{0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf},
	{0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16}
};

__device__ __forceinline__
void cryptonight_aes_set_key(uint32_t * __restrict__ key, const uint32_t * __restrict__ data)
{
	const uint32_t aes_gf[] = {
		0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36
	};

	MEMSET4(key, 0, 40);
	MEMCPY4(key, data, 8);

	#pragma unroll
	for(int i = 8; i < 40; i++)
	{
		uint8_t temp[4];
		*(uint32_t *)temp = key[i - 1];

		if(i % 8 == 0) {
			*(uint32_t *)temp = ROTR32(*(uint32_t *)temp, 8);
			for(int j = 0; j < 4; j++)
				temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];
			*(uint32_t *)temp ^= aes_gf[i / 8 - 1];
		}
		else if(i % 8 == 4) {
			#pragma unroll
			for(int j = 0; j < 4; j++)
				temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];
		}

		key[i] = key[(i - 8)] ^ *(uint32_t *)temp;
	}
}

__global__
void cryptonight_extra_gpu_prepare(const uint32_t threads, uint32_t * __restrict__ d_input, uint32_t startNonce,
	uint64_t * d_ctx_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint32_t * __restrict__ d_ctx_key1, uint32_t * __restrict__ d_ctx_key2)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if(thread < threads)
	{
		uint32_t ctx_state[50];
		uint32_t ctx_a[4];
		uint32_t ctx_b[4];
		uint32_t ctx_key1[40];
		uint32_t ctx_key2[40];
		uint32_t input[19];

		MEMCPY4(input, d_input, 19);
		*((uint32_t *)(((char *)input) + 39)) = startNonce + thread;

		cn_keccak((uint8_t *)input, (uint8_t *)ctx_state);
		cryptonight_aes_set_key(ctx_key1, ctx_state);
		cryptonight_aes_set_key(ctx_key2, ctx_state + 8);
		XOR_BLOCKS_DST(ctx_state, ctx_state + 8, ctx_a);
		XOR_BLOCKS_DST(ctx_state + 4, ctx_state + 12, ctx_b);

		MEMCPY8(&d_ctx_state[thread * 26], ctx_state, 25);
		MEMCPY4(d_ctx_a + thread * 4, ctx_a, 4);
		MEMCPY4(d_ctx_b + thread * 4, ctx_b, 4);
		MEMCPY4(d_ctx_key1 + thread * 40, ctx_key1, 40);
		MEMCPY4(d_ctx_key2 + thread * 40, ctx_key2, 40);
	}
}

__global__
void cryptonight_extra_gpu_keccak(uint32_t threads, uint32_t * d_ctx_state)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if(thread < threads)
	{
		uint64_t* ctx_state = (uint64_t*) (&d_ctx_state[thread * 52U]);
		uint64_t state[25];
		#pragma unroll
		for(int i = 0; i < 25; i++)
			state[i] = ctx_state[i];

		cn_keccakf2(state);

		// to reduce the final kernel stack frame, cut algos in 2 kernels
		// ps: these 2 final kernels are not important for the overall xmr hashrate (< 1%)
		switch (((uint8_t*)state)[0] & 0x03)
		{
			case 0: {
				uint32_t hash[8];
				cn_blake((uint8_t*)state, 200, (uint8_t*)hash);
				((uint32_t*)ctx_state)[0] = 0;
				((uint32_t*)ctx_state)[6] = hash[6];
				((uint32_t*)ctx_state)[7] = hash[7];
				break;
			}
			case 1: {
				uint32_t hash[8];
				cn_groestl((BitSequence*)state, 200, (BitSequence*)hash);
				((uint32_t*)ctx_state)[0] = 0;
				((uint32_t*)ctx_state)[6] = hash[6];
				((uint32_t*)ctx_state)[7] = hash[7];
				break;
			}
			default: {
				#pragma unroll
				for(int i = 0; i < 25; i++)
					ctx_state[i] = state[i];
			}
		}
	}
}

__global__
void cryptonight_extra_gpu_final(uint32_t threads, const uint32_t startNonce, uint64_t * __restrict__ d_ctx_state,
	const uint32_t* d_target, uint32_t * resNonces)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if(thread < threads)
	{
		uint64_t* const state = &d_ctx_state[thread * 26U];

		uint32_t hash[8];
		switch(((uint8_t *)state)[0] & 0x03)
		{
			case 0: {
				uint32_t* h32 = (uint32_t*)state;
				hash[6] = h32[6];
				hash[7] = h32[7];
				break;
			}
			case 2: {
				cn_jh256((uint8_t*)state, 200, hash);
				break;
			}
			case 3: {
				cn_skein((uint8_t*)state, 200, hash);
				break;
			}
		}

		if(hash[7] <= d_target[1] && hash[6] <= d_target[0])
		{
			const uint32_t nonce = startNonce + thread;
			uint32_t tmp = atomicExch(resNonces, nonce);
			if(tmp != UINT32_MAX)
				resNonces[1] = tmp;
		}
	}
}

__host__
void cryptonight_extra_cpu_setData(int thr_id, const void *data, const void *ptarget)
{
	uint32_t *pTargetIn = (uint32_t*) ptarget;
	hipMemcpy(d_input[thr_id], data, 19 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_target[thr_id], &pTargetIn[6], 2*sizeof(uint32_t), hipMemcpyHostToDevice);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}

__host__
void cryptonight_extra_cpu_init(int thr_id, uint32_t threads)
{
	hipMalloc(&d_input[thr_id], 19 * sizeof(uint32_t));
	hipMalloc(&d_target[thr_id], 2*sizeof(uint32_t));
	hipMalloc(&d_result[thr_id], 2*sizeof(uint32_t));
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}

__host__
void cryptonight_extra_cpu_prepare(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_ctx_state, uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2)
{
	uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cryptonight_extra_gpu_prepare <<<grid, block >>> (threads, d_input[thr_id], startNonce, d_ctx_state, d_ctx_a, d_ctx_b, d_ctx_key1, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}

__host__
void cryptonight_extra_cpu_final(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resnonce, uint64_t *d_ctx_state)
{
	uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	hipMemset(d_result[thr_id], 0xFF, 2*sizeof(uint32_t));
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	cryptonight_extra_gpu_keccak <<<grid, block >>> (threads, (uint32_t*)d_ctx_state);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	cryptonight_extra_gpu_final <<<grid, block >>> (threads, startNonce, d_ctx_state, d_target[thr_id], d_result[thr_id]);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	hipMemcpy(resnonce, d_result[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}

__host__
void cryptonight_extra_cpu_free(int thr_id)
{
	if (d_input[thr_id]) {
		hipFree(d_input[thr_id]);
		hipFree(d_target[thr_id]);
		hipFree(d_result[thr_id]);
		d_input[thr_id] = NULL;
	}
}