#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#ifndef _WIN32
#include <unistd.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if CUDA_VERSION >= 9000 && __CUDA_ARCH__ >= 300
#undef __shfl
#define __shfl(var, srcLane, width) __shfl_sync(0xFFFFFFFFu, var, srcLane, width)
#endif

#include "cryptonight.h"

#define LONG_SHL32 19 // 1<<19 (uint32_t* index)
#define LONG_SHL64 18 // 1<<18 (uint64_t* index)
#define LONG_LOOPS32 0x80000U

#include "cn_aes.cuh"

__global__
void cryptonight_gpu_phase1(const uint32_t threads, uint32_t * __restrict__ d_long_state,
	uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1)
{
	__shared__ uint32_t sharedMemory[1024];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	if(thread < threads)
	{
		cn_aes_gpu_init(sharedMemory);
		__syncthreads();

		const uint32_t sub = (threadIdx.x & 0x7U) << 2;
		uint32_t *longstate = &d_long_state[(thread << LONG_SHL32) + sub];
		uint32_t __align__(8) key[40];
		MEMCPY8(key, &ctx_key1[thread * 40U], 20);
		uint32_t __align__(8) text[4];
		MEMCPY8(text, &ctx_state[thread * 50U + sub + 16U], 2);

		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			cn_aes_pseudo_round_mut(sharedMemory, text, key);
			MEMCPY8(&longstate[i], text, 2);
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__device__ __forceinline__ ulonglong2 cuda_mul128(const uint64_t multiplier, const uint64_t multiplicand)
{
	ulonglong2 product;
	product.x = __umul64hi(multiplier, multiplicand);
	product.y = multiplier * multiplicand;
	return product;
}

static __forceinline__ __device__ void operator += (ulonglong2 &a, const ulonglong2 b) {
	a.x += b.x; a.y += b.y;
}

static __forceinline__ __device__ ulonglong2 operator ^ (const ulonglong2 &a, const ulonglong2 &b) {
	return make_ulonglong2(a.x ^ b.x, a.y ^ b.y);
}

__device__ __forceinline__ void MUL_SUM_XOR_DST_0(const uint64_t m, uint4 &a, void* far_dst)
{
	ulonglong2 d = AS_UL2(far_dst);
	ulonglong2 p = cuda_mul128(m, d.x);
	p += AS_UL2(&a);
	AS_UL2(&a) = p ^ d;
	AS_UL2(far_dst) = p;
}

__global__
#if __CUDA_ARCH__ >= 500
//__launch_bounds__(128,12) /* force 40 regs to allow -l ...x32 */
#endif
void cryptonight_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint64_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;

		void * ctx_a = (void*)(&d_ctx_a[thread << 2U]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2U]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		uint64_t * long_state = &d_long_state[thread << LONG_SHL64];
		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;

			uint32_t j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			AS_UINT4(&long_state[j]) = C ^ B; // st.global.u32.v4
			MUL_SUM_XOR_DST_0((AS_UL2(&C)).x, A, &long_state[(C.x & E2I_MASK) >> 3]);

			j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			AS_UINT4(&long_state[j]) = C ^ B;
			MUL_SUM_XOR_DST_0((AS_UL2(&B)).x, A, &long_state[(B.x & E2I_MASK) >> 3]);
		}

		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__device__ __forceinline__ void store_variant1(uint64_t* long_state, uint4 Z)
{
	const uint32_t tmp = (Z.z >> 24); // __byte_perm(src, 0, 0x7773);
	const uint32_t index = (((tmp >> 3) & 6u) | (tmp & 1u)) << 1;
	Z.z = (Z.z & 0x00ffffffu) | ((tmp ^ ((0x75310u >> index) & 0x30u)) << 24);
	AS_UINT4(long_state) = Z;
}

__device__ __forceinline__ void store_variant2(uint64_t* long_state, uint4 Z)
{
	const uint32_t tmp = (Z.z >> 24); // __byte_perm(src, 0, 0x7773);
	const uint32_t index = (((tmp >> 4) & 6u) | (tmp & 1u)) << 1;
	Z.z = (Z.z & 0x00ffffffu) | ((tmp ^ ((0x75312u >> index) & 0x30u)) << 24);
	AS_UINT4(long_state) = Z;
}

__device__ __forceinline__ void MUL_SUM_XOR_DST_1(const uint64_t m, uint4 &a, void* far_dst, uint64_t tweak)
{
	ulonglong2 d = AS_UL2(far_dst);
	ulonglong2 p = cuda_mul128(m, d.x);
	p += AS_UL2(&a);
	AS_UL2(&a) = p ^ d;
	p.y = p.y ^ tweak;
	AS_UL2(far_dst) = p;
}

__global__
void monero_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint64_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint64_t * __restrict__ d_tweak)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;
		uint64_t tweak = d_tweak[thread];

		void * ctx_a = (void*)(&d_ctx_a[thread << 2]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		uint64_t * long_state = &d_long_state[thread << LONG_SHL64];
		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;
			uint32_t j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			store_variant1(&long_state[j], C ^ B); // st.global
			MUL_SUM_XOR_DST_1((AS_UL2(&C)).x, A, &long_state[(C.x & E2I_MASK) >> 3], tweak);

			j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			store_variant1(&long_state[j], C ^ B);
			MUL_SUM_XOR_DST_1((AS_UL2(&B)).x, A, &long_state[(B.x & E2I_MASK) >> 3], tweak);
		}
		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__global__
void stellite_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint64_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint64_t * __restrict__ d_tweak)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;
		uint64_t tweak = d_tweak[thread];

		void * ctx_a = (void*)(&d_ctx_a[thread << 2]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		uint64_t * long_state = &d_long_state[thread << LONG_SHL64];
		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;
			uint32_t j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			store_variant2(&long_state[j], C ^ B); // st.global
			MUL_SUM_XOR_DST_1((AS_UL2(&C)).x, A, &long_state[(C.x & E2I_MASK) >> 3], tweak);

			j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			store_variant2(&long_state[j], C ^ B);
			MUL_SUM_XOR_DST_1((AS_UL2(&B)).x, A, &long_state[(B.x & E2I_MASK) >> 3], tweak);
		}
		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__global__
void cryptonight_gpu_phase3(const uint32_t threads, const uint32_t * __restrict__ d_long_state,
	uint32_t * __restrict__ d_ctx_state, const uint32_t * __restrict__ d_ctx_key2)
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;

	if(thread < threads)
	{
		const int sub = (threadIdx.x & 7) << 2;
		const uint32_t *longstate = &d_long_state[(thread << LONG_SHL32) + sub];
		uint32_t key[40], text[4];
		MEMCPY8(key, d_ctx_key2 + thread * 40, 20);
		MEMCPY8(text, d_ctx_state + thread * 50 + sub + 16, 2);

		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			#pragma unroll
			for(int j = 0; j < 4; ++j)
				text[j] ^= longstate[i + j];

			cn_aes_pseudo_round_mut(sharedMemory, text, key);
		}

		MEMCPY8(d_ctx_state + thread * 50 + sub + 16, text, 2);
	}
}

// --------------------------------------------------------------------------------------------------------------

extern int device_bfactor[MAX_GPUS];

__host__
void cryptonight_core_cuda(int thr_id, uint32_t blocks, uint32_t threads, uint64_t *d_long_state, uint32_t *d_ctx_state,
	uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2, int variant, uint64_t *d_ctx_tweak)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	const uint16_t bfactor = (uint16_t) device_bfactor[thr_id];
	const uint32_t partcount = 1U << bfactor;
	const uint32_t throughput = (uint32_t) (blocks*threads);

	const int bsleep = bfactor ? 100 : 0;
	const int dev_id = device_map[thr_id];

	cryptonight_gpu_phase1 <<<grid, block8>>> (throughput, (uint32_t*) d_long_state, d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	if(partcount > 1) usleep(bsleep);

	for (uint32_t i = 0; i < partcount; i++)
	{
		dim3 b = device_sm[dev_id] >= 300 ? block4 : block;
		if (variant == 0)
			cryptonight_gpu_phase2 <<<grid, b>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b);
		else if (variant == 1 || cryptonight_fork == 8)
			monero_gpu_phase2 <<<grid, b>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b, d_ctx_tweak);
		else if (variant == 2 && cryptonight_fork == 3)
			stellite_gpu_phase2 <<<grid, b>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b, d_ctx_tweak);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		if(partcount > 1) usleep(bsleep);
	}
	//hipDeviceSynchronize();
	//exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	cryptonight_gpu_phase3 <<<grid, block8>>> (throughput, (uint32_t*) d_long_state, d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}
