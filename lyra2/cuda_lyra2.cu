#include "hip/hip_runtime.h"
/*
 * lyra2 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   djm34
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>


extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];

#include "cuda_helper.h"


static __constant__ uint2 blake2b_IV[8] =
{
	{ 0xf3bcc908, 0x6a09e667  }, 
	{ 0x84caa73b, 0xbb67ae85  },
	{ 0xfe94f82b, 0x3c6ef372  },
	{ 0x5f1d36f1, 0xa54ff53a  },
	{ 0xade682d1, 0x510e527f  },
	{ 0x2b3e6c1f, 0x9b05688c  },
	{ 0xfb41bd6b, 0x1f83d9ab  },
	{ 0x137e2179, 0x5be0cd19  }
};

#define reduceDuplexRowSetup(rowIn, rowInOut, rowOut) \
  { \
	for (int i = 0; i < 8; i++) \
			{ \
\
		for (int j = 0; j < 12; j++) {state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];} \
		round_lyra_v35(state); \
		for (int j = 0; j < 12; j++) {Matrix[j + 84 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];} \
\
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
			} \
 \
  } 

#define reduceDuplexRow(rowIn, rowInOut, rowOut) \
  { \
	 for (int i = 0; i < 8; i++) \
	 	 	 	 { \
		 for (int j = 0; j < 12; j++) \
			 state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
 \
		 round_lyra_v35(state); \
		 for (int j = 0; j < 12; j++) {Matrix[j + 12 * i][rowOut] ^= state[j];} \
\
		 Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		 Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		 Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		 Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		 Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		 Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		 Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		 Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		 Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		 Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		 Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		 Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
	 	 	 	 } \
 \
  } 
#define absorbblock(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
  } 

//// compute 30 version 
#define reduceDuplexRowSetup_v30(rowIn, rowInOut, rowOut) \
  { \
	for (int i = 0; i < 8; i++) \
				{ \
\
		for (int j = 0; j < 12; j++) {state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];} \
		round_lyra_v30(state); \
		for (int j = 0; j < 12; j++) {Matrix[j + 84 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];} \
\
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
				} \
 \
  } 

#define reduceDuplexRow_v30(rowIn, rowInOut, rowOut) \
  { \
	 for (int i = 0; i < 8; i++) \
	 	 	 	 	 { \
		 for (int j = 0; j < 12; j++) \
			 state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
 \
		 round_lyra_v30(state); \
		 for (int j = 0; j < 12; j++) {Matrix[j + 12 * i][rowOut] ^= state[j];} \
\
		 Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		 Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		 Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		 Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		 Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		 Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		 Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		 Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		 Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		 Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		 Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		 Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
	 	 	 	 	 } \
 \
  } 
#define absorbblock_v30(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
  } 




 static __device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
 {
	 a += b; d ^= a; d = ROR2(d, 32);
	 c += d; b ^= c; b = ROR2(b, 24);
	 a += b; d ^= a; d = ROR2(d, 16);
	 c += d; b ^= c; b = ROR2(b, 63);
 }


 static __device__ __forceinline__ void Gfunc_v30(uint64_t & a, uint64_t &b, uint64_t &c, uint64_t &d)
 {
	 a += b; d ^= a; d = ROTR64(d, 32);
	 c += d; b ^= c; b = ROTR64(b, 24);
	 a += b; d ^= a; d = ROTR64(d, 16);
	 c += d; b ^= c; b = ROTR64(b, 63);
 }

 
static __device__ __forceinline__ void round_lyra_v35(uint2 *s) 
{
	Gfunc_v35(s[0], s[4], s[8],  s[12]);
	Gfunc_v35(s[1], s[5], s[9],  s[13]);
	Gfunc_v35(s[2], s[6], s[10], s[14]);
	Gfunc_v35(s[3], s[7], s[11], s[15]);
	Gfunc_v35(s[0], s[5], s[10], s[15]);
	Gfunc_v35(s[1], s[6], s[11], s[12]);
	Gfunc_v35(s[2], s[7], s[8],  s[13]);
	Gfunc_v35(s[3], s[4], s[9],  s[14]);
}

static __device__ __forceinline__ void round_lyra_v30(uint64_t *s)
{
	Gfunc_v30(s[0], s[4], s[8], s[12]);
	Gfunc_v30(s[1], s[5], s[9], s[13]);
	Gfunc_v30(s[2], s[6], s[10], s[14]);
	Gfunc_v30(s[3], s[7], s[11], s[15]);
	Gfunc_v30(s[0], s[5], s[10], s[15]);
	Gfunc_v30(s[1], s[6], s[11], s[12]);
	Gfunc_v30(s[2], s[7], s[8], s[13]);
	Gfunc_v30(s[3], s[4], s[9], s[14]);
}



__global__ void __launch_bounds__(160, 1) lyra2_gpu_hash_32_v30(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t state[16];
#pragma unroll
		for (int i = 0; i<4; i++) { state[i] = outputHash[threads*i + thread]; } //password
#pragma unroll
		for (int i = 0; i<4; i++) { state[i + 4] = state[i]; } //salt 
#pragma unroll
		for (int i = 0; i<8; i++) { state[i + 8] = devectorize(blake2b_IV[i]); }

		//     blake2blyra x2 
#pragma unroll 24
		for (int i = 0; i<24; i++) { round_lyra_v30(state); } //because 12 is not enough

		uint64_t Matrix[96][8]; // not cool
		/// reducedSqueezeRow0
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
int idx = 84-12*i;
#pragma unroll 12
			for (int j = 0; j<12; j++) { Matrix[j + idx][0] = state[j]; }
			round_lyra_v30(state);
		}

		/// reducedSqueezeRow1
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
int idx0= 12*i;
int idx1= 84-idx0; 
#pragma unroll 12
			for (int j = 0; j<12; j++) { state[j] ^= Matrix[j + idx0][0]; }
			round_lyra_v30(state);
#pragma unroll 12  
			for (int j = 0; j<12; j++) { Matrix[j + idx1][1] = Matrix[j + idx0][0] ^ state[j]; }
		}


		reduceDuplexRowSetup_v30(1, 0, 2);
		reduceDuplexRowSetup_v30(2, 1, 3);
		reduceDuplexRowSetup_v30(3, 0, 4);
		reduceDuplexRowSetup_v30(4, 3, 5);
		reduceDuplexRowSetup_v30(5, 2, 6);
		reduceDuplexRowSetup_v30(6, 1, 7);



		uint64_t rowa;
		rowa = state[0] & 7;
		reduceDuplexRow_v30(7, rowa, 0);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(0, rowa, 3);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(3, rowa, 6);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(6, rowa, 1);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(1, rowa, 4);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(4, rowa, 7);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(7, rowa, 2);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(2, rowa, 5);

		absorbblock_v30(rowa);


#pragma unroll
		for (int i = 0; i<4; i++) {
			outputHash[threads*i + thread] = state[i];
		} //password


	} //thread
}


__global__ void __launch_bounds__(160, 1) lyra2_gpu_hash_32(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 state[16];
#pragma unroll
		for (int i = 0; i<4; i++) { LOHI(state[i].x, state[i].y, outputHash[threads*i + thread]); } //password
#pragma unroll
		for (int i = 0; i<4; i++) { state[i + 4] = state[i]; } //salt 
#pragma unroll
		for (int i = 0; i<8; i++) { state[i + 8] = blake2b_IV[i]; }

		//     blake2blyra x2 
#pragma unroll 24
		for (int i = 0; i<24; i++) { round_lyra_v35(state); } //because 12 is not enough

		uint2 Matrix[96][8]; // not cool

		/// reducedSqueezeRow0
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { Matrix[j + 84 - 12 * i][0] = state[j]; }
			round_lyra_v35(state);
		}

		/// reducedSqueezeRow1
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { state[j] ^= Matrix[j + 12 * i][0]; }
			round_lyra_v35(state);
#pragma unroll 12  
			for (int j = 0; j<12; j++) { Matrix[j + 84 - 12 * i][1] = Matrix[j + 12 * i][0] ^ state[j]; }
		}

		reduceDuplexRowSetup(1, 0, 2);
		reduceDuplexRowSetup(2, 1, 3);
		reduceDuplexRowSetup(3, 0, 4);
		reduceDuplexRowSetup(4, 3, 5);
		reduceDuplexRowSetup(5, 2, 6);
		reduceDuplexRowSetup(6, 1, 7);



		uint32_t rowa;
		rowa = state[0].x & 7;
		reduceDuplexRow(7, rowa, 0);
		rowa = state[0].x & 7;
		reduceDuplexRow(0, rowa, 3);
		rowa = state[0].x & 7;
		reduceDuplexRow(3, rowa, 6);
		rowa = state[0].x & 7;
		reduceDuplexRow(6, rowa, 1);
		rowa = state[0].x & 7;
		reduceDuplexRow(1, rowa, 4);
		rowa = state[0].x & 7;
		reduceDuplexRow(4, rowa, 7);
		rowa = state[0].x & 7;
		reduceDuplexRow(7, rowa, 2);
		rowa = state[0].x & 7;
		reduceDuplexRow(2, rowa, 5);

		absorbblock(rowa);


#pragma unroll
		for (int i = 0; i<4; i++) {
			outputHash[threads*i + thread] = devectorize(state[i]);
		} //password


	} //thread
}

   
void lyra2_cpu_init(int thr_id, int threads)
{
//not used    	
} 


__host__ void lyra2_cpu_hash_32(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	
	const int threadsperblock = 160;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;
	
	if (compute_version[thr_id]>=35) {
	lyra2_gpu_hash_32 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	}
	else {  // kernel for compute30 card
	lyra2_gpu_hash_32_v30 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	}
    
	MyStreamSynchronize(NULL, order, thr_id);

}

