#include "hip/hip_runtime.h"
/**
 * Lyra2 (v1) cuda implementation based on djm34 work - SM 5/5.2
 * tpruvot@github 2015
 */

#include <stdio.h>
#include <memory.h>

#define TPB50 16
#define TPB52 8

#include "cuda_lyra2_sm2.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#if !defined(__CUDA_ARCH__) ||  __CUDA_ARCH__ >= 500

#include "cuda_vector_uint2x4.h"

#define memshift 3

#define Ncol 8
#define NcolMask 0x7

__device__ uint2x4* DMatrix;

static __device__ __forceinline__
void Gfunc(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d ^= a; d = SWAPUINT2(d);
	c += d; b ^= c; b = ROR2(b, 24);
	a += b; d ^= a; d = ROR2(d, 16);
	c += d; b ^= c; b = ROR2(b, 63);
}

static __device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc(s[0].w, s[1].w, s[2].w, s[3].w);
	Gfunc(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc(s[0].w, s[1].x, s[2].y, s[3].z);
}

static __device__ __forceinline__
void reduceDuplex(uint2x4 state[4], uint32_t thread)
{
	uint2x4 state1[3];

	const uint32_t ps1 = (256 * thread);
	const uint32_t ps2 = (memshift * 7 + memshift * 8 + 256 * thread);

	#pragma unroll 4
	for (int i = 0; i < 8; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 - i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];
	}
}

static __device__ __forceinline__
void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2x4 state[4], uint32_t thread)
{
	uint2x4 state1[3], state2[3];

	const uint32_t ps1 = (             memshift*8 * rowIn    + 256 * thread);
	const uint32_t ps2 = (             memshift*8 * rowInOut + 256 * thread);
	const uint32_t ps3 = (memshift*7 + memshift*8 * rowOut   + 256 * thread);

	#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		for (int j = 0; j < 3; j++)
			state1[j]= __ldg4(&(DMatrix + s1)[j]);
		for (int j = 0; j < 3; j++)
			state2[j]= __ldg4(&(DMatrix + s2)[j]);
		for (int j = 0; j < 3; j++) {
			uint2x4 tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}

		round_lyra(state);

		for (int j = 0; j < 3; j++) {
			const uint32_t s3 = ps3 - i*memshift;
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

		((uint2*)state2)[0] ^= ((uint2*)state)[11];

		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];

		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}

static __device__ __forceinline__
void reduceDuplexRowt(const int rowIn, const int rowInOut, const int rowOut, uint2x4* state, const uint32_t thread)
{
	const uint32_t ps1 = (memshift * 8 * rowIn    + 256 * thread);
	const uint32_t ps2 = (memshift * 8 * rowInOut + 256 * thread);
	const uint32_t ps3 = (memshift * 8 * rowOut   + 256 * thread);

	#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		uint2x4 state1[3], state2[3];

		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;

		for (int j = 0; j < 3; j++) {
			state1[j] = __ldg4(&(DMatrix + s1)[j]);
			state2[j] = __ldg4(&(DMatrix + s2)[j]);
		}

		#pragma unroll
		for (int j = 0; j < 3; j++) {
			state1[j] += state2[j];
			state[j]  ^= state1[j];
		}

		round_lyra(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];

		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

		if (rowInOut == rowOut) {
			for (int j = 0; j < 3; j++) {
				state2[j] ^= state[j];
				(DMatrix + s2)[j]=state2[j];
			}
		} else {
			const uint32_t s3 = ps3 + i*memshift;
			for (int j = 0; j < 3; j++) {
				(DMatrix + s2)[j] = state2[j];
				(DMatrix + s3)[j] ^= state[j];
			}
		}
	}
}

#if __CUDA_ARCH__ == 500
__global__ __launch_bounds__(TPB50, 1)
#else
__global__ __launch_bounds__(TPB52, 2)
#endif
void lyra2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	const uint2x4 blake2b_IV[2] = {
		{{ 0xf3bcc908, 0x6a09e667 }, { 0x84caa73b, 0xbb67ae85 }, { 0xfe94f82b, 0x3c6ef372 }, { 0x5f1d36f1, 0xa54ff53a }},
		{{ 0xade682d1, 0x510e527f }, { 0x2b3e6c1f, 0x9b05688c }, { 0xfb41bd6b, 0x1f83d9ab }, { 0x137e2179, 0x5be0cd19 }}
	};

	if (thread < threads)
	{
		uint2x4 state[4];

		((uint2*)state)[0] = __ldg(&g_hash[thread]);
		((uint2*)state)[1] = __ldg(&g_hash[thread + threads]);
		((uint2*)state)[2] = __ldg(&g_hash[thread + threads*2]);
		((uint2*)state)[3] = __ldg(&g_hash[thread + threads*3]);

		state[1] = state[0];
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<24; i++)
			round_lyra(state); //because 12 is not enough

		const uint32_t ps1 = (memshift * 7  + 256 * thread);
		for (int i = 0; i < 8; i++)
		{
			const uint32_t s1 = ps1 - memshift * i;
			for (int j = 0; j < 3; j++)
				(DMatrix + s1)[j] = (state)[j];
			round_lyra(state);
		}

		reduceDuplex(state, thread);

		reduceDuplexRowSetup(1, 0, 2, state,  thread);
		reduceDuplexRowSetup(2, 1, 3, state,  thread);
		reduceDuplexRowSetup(3, 0, 4, state,  thread);
		reduceDuplexRowSetup(4, 3, 5, state,  thread);
		reduceDuplexRowSetup(5, 2, 6, state,  thread);
		reduceDuplexRowSetup(6, 1, 7, state,  thread);

		uint32_t rowa = state[0].x.x & 7;
		reduceDuplexRowt(7, rowa, 0, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(0, rowa, 3, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(3, rowa, 6, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(6, rowa, 1, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(1, rowa, 4, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(4, rowa, 7, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(7, rowa, 2, state, thread);
		rowa = state[0].x.x & 7;
		reduceDuplexRowt(2, rowa, 5, state, thread);

		const int32_t shift = (memshift * 8 * rowa + 256 * thread);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);

		g_hash[thread]             = ((uint2*)state)[0];
		g_hash[thread + threads]   = ((uint2*)state)[1];
		g_hash[thread + threads*2] = ((uint2*)state)[2];
		g_hash[thread + threads*3] = ((uint2*)state)[3];
	}
}
#else
/* for unsupported SM arch */
__device__ void* DMatrix;
__global__ void lyra2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
#endif

__host__
void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t* d_matrix)
{
	cuda_get_arch(thr_id);
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];
	uint32_t tpb = TPB52;
	if (device_sm[dev_id] == 500) tpb = TPB50;
	if (device_sm[dev_id] == 350) tpb = TPB30; // to enhance (or not)
	if (device_sm[dev_id] <= 300) tpb = TPB30;

	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	if (device_sm[dev_id] >= 500)
		lyra2_gpu_hash_32 <<< grid, block >>> (threads, startNounce, (uint2*)d_hash);
	else
		lyra2_gpu_hash_32_sm2 <<< grid, block >>> (threads, startNounce, d_hash);

}
