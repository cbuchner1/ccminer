#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_skein.h"
#include "sph/sph_groestl.h"
#include "lyra2/Lyra2.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
//extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);

//extern void keccak256_sm3_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
//extern void keccak256_sm3_init(int thr_id, uint32_t threads);
//extern void keccak256_sm3_free(int thr_id);

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint64_t *d_outputHash, bool gtx750ti);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_cpu_free(int thr_id);
extern void groestl256_setTarget(const void *ptarget);
extern uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern uint32_t groestl256_getSecNonce(int thr_id, int num);


extern "C" void allium_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_cubehash256_context  ctx_cube;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;

extern "C" int scanhash_allium(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	static __thread bool gtx750ti;
	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);

		if (strstr(props.name, "750 Ti")) gtx750ti = true;
		else gtx750ti = false;

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);
		//keccak256_sm3_init(thr_id, throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);

		//cuda_get_arch(thr_id);
		if (device_sm[dev_id] >= 500)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;

		//blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		//keccak256_sm3_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, d_hash[thr_id], gtx750ti);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, d_hash[thr_id], gtx750ti);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			allium_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = groestl256_getSecNonce(thr_id, 1);
				if (work->nonces[1] != UINT32_MAX) {
					be32enc(&endiandata[19], work->nonces[1]);
					allium_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_allium(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	//keccak256_sm3_free(thr_id);
	groestl256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
