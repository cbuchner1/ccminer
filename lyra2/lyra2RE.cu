#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint64_t* d_hash[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_setTarget(const void *ptarget);
extern uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern uint32_t groestl256_getSecNonce(int thr_id, int num);

extern "C" void lyra2_hash(void *state, const void *input)
{
	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 18 : 17;
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << intensity); // 18=256*256*4;
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], throughput * 64));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		*hashes_done = pdata[19] - first_nonce + throughput;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		foundNonce = groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];

			be32enc(&endiandata[19], foundNonce);
			lyra2_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = groestl256_getSecNonce(thr_id, 1);
				if (secNonce != UINT32_MAX)
				{
					be32enc(&endiandata[19], secNonce);
					lyra2_hash(vhash64, endiandata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						if (opt_debug)
							applog(LOG_BLUE, "GPU #%d: found second nonce %08x", device_map[thr_id], secNonce);
						pdata[21] = secNonce;
						res++;
					}
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	return 0;
}
