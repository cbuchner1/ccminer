#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_cpu_free(int thr_id);
extern void groestl256_setTarget(const void *ptarget);
extern uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern uint32_t groestl256_getSecNonce(int thr_id, int num);

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("lyra %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

extern "C" void lyra2re_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 17 : 16;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		CUDA_LOG_ERROR();

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);

		// DMatrix
		hipMalloc(&d_matrix[thr_id], (size_t)16 * 8 * 8 * sizeof(uint64_t) * throughput);
		lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE("S")

		*hashes_done = pdata[19] - first_nonce + throughput;

		foundNonce = groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];

			be32enc(&endiandata[19], foundNonce);
			lyra2re_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = groestl256_getSecNonce(thr_id, 1);
				work_set_target_ratio(work, vhash64);
				if (secNonce != UINT32_MAX)
				{
					be32enc(&endiandata[19], secNonce);
					lyra2re_hash(vhash64, endiandata);
					if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
						if (opt_debug)
							gpulog(LOG_BLUE, thr_id, "found second nonce %08x", secNonce);
						if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio)
							work_set_target_ratio(work, vhash64);
						pdata[21] = secNonce;
						res++;
					}
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	keccak256_cpu_free(thr_id);
	groestl256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
