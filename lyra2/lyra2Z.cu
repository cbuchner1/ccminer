#include "hip/hip_runtime.h"
extern "C" {
#include <sph/sph_blake.h>
#include "Lyra2Z.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void lyra2Z_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2Z_cpu_init_sm2(int thr_id, uint32_t threads);
extern void lyra2Z_cpu_free(int thr_id);
extern uint32_t lyra2Z_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, bool gtx750ti);

extern void lyra2Z_setTarget(const void *ptarget);
extern uint32_t lyra2Z_getSecNonce(int thr_id, int num);

extern "C" void lyra2Z_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hashA[8], hashB[8];
	sph_blake256_context ctx_blake;

	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	LYRA2Z(hashB, 32, hashA, 32, hashA, 32, 8, 8, 8);

	memcpy(state, hashB, 32);
}

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static __thread bool gtx750ti = false;

extern "C" int scanhash_lyra2Z(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t _ALIGN(64) endiandata[20];
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
		if (device_sm[dev_id] <= 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);
		gtx750ti = (strstr(props.name, "750 Ti") != NULL);

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);

		if (device_sm[dev_id] >= 350)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2Z_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}
		else
			lyra2Z_cpu_init_sm2(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	lyra2Z_setTarget(ptarget);

	do {
		int order = 0;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = lyra2Z_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], gtx750ti);

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			lyra2Z_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = lyra2Z_getSecNonce(thr_id, 1);
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1;
				if (work->nonces[1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2Z_hash(vhash, endiandata);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)	gpulog(LOG_WARNING, thr_id,
					"result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0];
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2Z(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	if (device_sm[dev_id] >= 350)
		hipFree(d_matrix[thr_id]);
	lyra2Z_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
