#include "hip/hip_runtime.h"
/**
 * Lyra2 (v2) CUDA Implementation
 *
 * Based on djm34/VTC sources and incredible 2x boost by Nanashi Meiyo-Meijin (May 2016)
 */
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_lyra2v2_sm3.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#define TPB 32

#if __CUDA_ARCH__ >= 500

#include "cuda_lyra2_vectors.h"

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DMatrix;

__device__ __forceinline__ uint2 LD4S(const int index)
{
	extern __shared__ uint2 shared_mem[];
	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

__device__ __forceinline__ void ST4S(const int index, const uint2 data)
{
	extern __shared__ uint2 shared_mem[];
	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

__device__ __forceinline__ uint2 shuffle2(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d ^= a; d = SWAPUINT2(d);
	c += d; b ^= c; b = ROR2(b, 24);
	a += b; d ^= a; d = ROR2(d, 16);
	c += d; b ^= c; b = ROR2(b, 63);
}

__device__ __forceinline__
void round_lyra_v5(uint2x4 s[4])
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}

__device__ __forceinline__
void round_lyra_v5(uint2 s[4])
{
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 1, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 3, 4);
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 3, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 1, 4);
}

__device__ __forceinline__
void reduceDuplexRowSetup2(uint2 state[4])
{
	uint2 state1[Ncol][3], state0[Ncol][3], state2[3];
	int i, j;

	#pragma unroll
	for (int i = 0; i < Ncol; i++)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] = state[j];
		round_lyra_v5(state);
	}

	//#pragma unroll 4
	for (i = 0; i < Ncol; i++)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state0[i][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] = state0[i][j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] ^= state[j];
	}

	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s0 = memshift * Ncol * 0 + i * memshift;
		const uint32_t s2 = memshift * Ncol * 2 + memshift * (Ncol - 1) - i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[i][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = state1[i][j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] ^= state[j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		} else {
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s0 + j, state0[i][j]);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[i][j] = state2[j];

	}

	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = memshift * Ncol * 1 + i*memshift;
		const uint32_t s3 = memshift * Ncol * 3 + memshift * (Ncol - 1) - i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[Ncol - i - 1][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] ^= state[j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s3 + j, state0[Ncol - i - 1][j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		} else  {
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s1 + j, state1[i][j]);
	}
}

__device__
void reduceDuplexRowt2(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4])
{
	uint2 state1[3], state2[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;
	const uint32_t ps3 = memshift * Ncol * rowOut;

	for (int i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 + i*memshift;

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = LD4S(s1 + j);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra_v5(state);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		} else {
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
	}
}

__device__
void reduceDuplexRowt2x4(const int rowInOut, uint2 state[4])
{
	const int rowIn = 2;
	const int rowOut = 3;

	int i, j;
	uint2 last[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;

	#pragma unroll
	for (int j = 0; j < 3; j++)
		last[j] = LD4S(ps2 + j);

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= LD4S(ps1 + j) + last[j];

	round_lyra_v5(state);

	uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
	uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
	uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

	if (threadIdx.x == 0) {
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	} else {
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == rowOut)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (i = 1; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + LD4S(s2 + j);

		round_lyra_v5(state);
	}

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	const uint2x4 blake2b_IV[2] = {
		0xf3bcc908UL, 0x6a09e667UL, 0x84caa73bUL, 0xbb67ae85UL,
		0xfe94f82bUL, 0x3c6ef372UL, 0x5f1d36f1UL, 0xa54ff53aUL,
		0xade682d1UL, 0x510e527fUL, 0x2b3e6c1fUL, 0x9b05688cUL,
		0xfb41bd6bUL, 0x1f83d9abUL, 0x137e2179UL, 0x5be0cd19UL
	};

	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000001UL, 0x00000000UL,
		0x00000004UL, 0x00000000UL, 0x00000004UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};

	uint2x4 state[4];

	if (thread < threads)
	{
		state[0].x = state[1].x = __ldg(&inputHash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&inputHash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&inputHash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&inputHash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		DMatrix[blockDim.x * gridDim.x * 0 + thread] = state[0];
		DMatrix[blockDim.x * gridDim.x * 1 + thread] = state[1];
		DMatrix[blockDim.x * gridDim.x * 2 + thread] = state[2];
		DMatrix[blockDim.x * gridDim.x * 3 + thread] = state[3];
	}
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_2(uint32_t threads)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = ((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[1] = ((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplexRowSetup2(state);

		uint32_t rowa;
		int prev = 3;

		for (int i = 0; i < 3; i++)
		{
			rowa = __shfl(state[0].x, 0, 4) & 3;
			reduceDuplexRowt2(prev, rowa, i, state);
			prev = i;
		}

		rowa = __shfl(state[0].x, 0, 4) & 3;
		reduceDuplexRowt2x4(rowa, state);

		((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[0];
		((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint2x4 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 0 + thread]);
		state[1] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 1 + thread]);
		state[2] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 2 + thread]);
		state[3] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 3 + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra_v5(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;
	}
}

#else
#include "cuda_helper.h"
#if __CUDA_ARCH__ < 200
__device__ void* DMatrix;
#endif
__global__ void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash) {}
__global__ void lyra2v2_gpu_hash_32_2(uint32_t threads) {}
__global__ void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash) {}
#endif


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	cuda_get_arch(thr_id);
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	if (device_sm[dev_id] >= 500) {

		const uint32_t tpb = TPB;

		dim3 grid2((threads + tpb - 1) / tpb);
		dim3 block2(tpb);
		dim3 grid4((threads * 4 + tpb - 1) / tpb);
		dim3 block4(4, tpb / 4);

		lyra2v2_gpu_hash_32_1 <<< grid2, block2 >>> (threads, (uint2*)g_hash);
		lyra2v2_gpu_hash_32_2 <<< grid4, block4, 48 * sizeof(uint2) * tpb >>> (threads);
		lyra2v2_gpu_hash_32_3 <<< grid2, block2 >>> (threads, (uint2*)g_hash);

	} else {

		uint32_t tpb = 16;
		if (cuda_arch[dev_id] >= 350) tpb = TPB35;
		else if (cuda_arch[dev_id] >= 300) tpb = TPB30;
		else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

		dim3 grid((threads + tpb - 1) / tpb);
		dim3 block(tpb);
		lyra2v2_gpu_hash_32_v3 <<< grid, block >>> (threads, startNounce, (uint2*)g_hash);

	}
}
