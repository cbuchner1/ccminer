#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#define TPB52 32
#define TPB50 32
#define TPB30 32
#define TPB20 32

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 520
#endif

#include "cuda_lyra2_vectors.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
__device__ void __threadfence_block();
#if __CUDA_ARCH__ >= 300
__device__ uint32_t __shfl(uint32_t a, uint32_t b, uint32_t c);
#endif
#endif

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DState;

__device__ __forceinline__ uint2 LD4S(const int index)
{
	extern __shared__ uint2 shared_mem[];

	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

__device__ __forceinline__ void ST4S(const int index, const uint2 data)
{
	extern __shared__ uint2 shared_mem[];

	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

__device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; uint2 tmp = d; d.y = a.x ^ tmp.x; d.x = a.y ^ tmp.y;
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl(a, b, c);
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t *_ptr = (uint32_t*)shared_mem;

	__threadfence_block();
	uint32_t buf = _ptr[thread];

	_ptr[thread] = a;
	__threadfence_block();
	uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	_ptr[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a;
	__threadfence_block();
	uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a1;
	__threadfence_block();
	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a2;
	__threadfence_block();
	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a3;
	__threadfence_block();
	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;
	__threadfence_block();
}

#endif


__device__ __forceinline__ void round_lyra(uint2 s[4])
{
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
}

__device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}


__device__ __forceinline__ void reduceDuplexRowSetupV2(uint2 state[4])
{
	int i, j;
	uint2 state1[Ncol][3], state0[Ncol][3], state2[3];

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] = state[j];
		round_lyra(state);
	}

	//#pragma unroll 4
	for (i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state0[i][j];

		round_lyra(state);

#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] = state0[i][j];

#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] ^= state[j];
	}

	uint32_t s0 = 0;
	uint32_t s2 = 33;
	for (i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[i][j];

		round_lyra(state);

#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = state1[i][j];

#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] ^= state[j];

#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		}
		else
		{
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s0 + j, state0[i][j]);

#pragma unroll
		for (j = 0; j < 3; j++)
			state0[i][j] = state2[j];

		s0 += memshift;
		s2 -= memshift;
	}

	s2 += 24;
	for (i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[Ncol - i - 1][j];

		round_lyra(state);

#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] ^= state[j];
#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s2 + j, state0[Ncol - i - 1][j]);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		}
		else
		{
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s0 + j, state1[i][j]);

		s0 += memshift;
		s2 -= memshift;
	}
}

__device__ void reduceDuplexRowtV2(uint2 state[4])
{
	uint32_t rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	uint2 state2[3], state1[3], last[3];
	uint32_t s1 = 36;
	uint32_t s2 = 12 * rowInOut;
	uint32_t s3 = 0;

	for (int i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + state2[j];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			ST4S(s2 + j, state2[j]);
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
		}

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}
	s1 = 0;
	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	s2 = 12 * rowInOut;

	for (int i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + state2[j];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			ST4S(s2 + j, state2[j]);
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
		}

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	s2 = 12 * rowInOut;

	for (int i = 0; i < Ncol; i++)
	{
#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + state2[j];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			ST4S(s2 + j, state2[j]);
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
		}

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	s2 = 12 * rowInOut;

#pragma unroll
	for (int j = 0; j < 3; j++)
		last[j] = LD4S(s2 + j);

#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= LD4S(s1 + j) + last[j];

	round_lyra(state);

	//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	}
	else
	{
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == 3)
	{
#pragma unroll 
		for (int j = 0; j < 3; j++)
			last[j] ^= state[j];
	}
	s1 += memshift;
	s2 += memshift;

	for (int i = 1; i < Ncol; i++)
	{
#pragma unroll 
		for (int j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + LD4S(s2 + j);

		round_lyra(state);

		s1 += memshift;
		s2 += memshift;
	}

#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__constant__ uint28 blake2b_IV[2] = {
	0xf3bcc908lu, 0x6a09e667lu,
	0x84caa73blu, 0xbb67ae85lu,
	0xfe94f82blu, 0x3c6ef372lu,
	0x5f1d36f1lu, 0xa54ff53alu,
	0xade682d1lu, 0x510e527flu,
	0x2b3e6c1flu, 0x9b05688clu,
	0xfb41bd6blu, 0x1f83d9ablu,
	0x137e2179lu, 0x5be0cd19lu
};

__constant__ uint28 Mask[2] = {
	0x00000020lu, 0x00000000lu,
	0x00000020lu, 0x00000000lu,
	0x00000020lu, 0x00000000lu,
	0x00000001lu, 0x00000000lu,
	0x00000004lu, 0x00000000lu,
	0x00000004lu, 0x00000000lu,
	0x00000080lu, 0x00000000lu,
	0x00000000lu, 0x01000000lu
};

__global__ __launch_bounds__(64, 1)
void lyra2v2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0].x = state[1].x = __ldg(&outputHash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&outputHash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&outputHash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&outputHash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

#pragma unroll 2
		for (int i = 0; i<12; i++)
			round_lyra(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

#pragma unroll 2
		for (int i = 0; i<12; i++)
			round_lyra(state);

		DState[blockDim.x * gridDim.x * 0 + blockDim.x * blockIdx.x + threadIdx.x] = state[0];
		DState[blockDim.x * gridDim.x * 1 + blockDim.x * blockIdx.x + threadIdx.x] = state[1];
		DState[blockDim.x * gridDim.x * 2 + blockDim.x * blockIdx.x + threadIdx.x] = state[2];
		DState[blockDim.x * gridDim.x * 3 + blockDim.x * blockIdx.x + threadIdx.x] = state[3];

	} //thread
}

#if __CUDA_ARCH__ < 300
__global__ __launch_bounds__(TPB20, 1)
#elif __CUDA_ARCH__ < 500
__global__ __launch_bounds__(TPB30, 1)
#elif __CUDA_ARCH__ == 500
__global__ __launch_bounds__(TPB50, 1)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void lyra2v2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = ((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[1] = ((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DState)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplexRowSetupV2(state);

		reduceDuplexRowtV2(state);

		((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[0];
		((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DState)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	} //thread
}

__global__ __launch_bounds__(64, 1)
void lyra2v2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&DState[blockDim.x * gridDim.x * 0 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[1] = __ldg4(&DState[blockDim.x * gridDim.x * 1 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[2] = __ldg4(&DState[blockDim.x * gridDim.x * 2 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[3] = __ldg4(&DState[blockDim.x * gridDim.x * 3 + blockDim.x * blockIdx.x + threadIdx.x]);

#pragma unroll 2
		for (int i = 0; i < 12; i++)
			round_lyra(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;

	} //thread
}

__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	int dev_id = device_map[thr_id % MAX_GPUS];
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DState), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (cuda_arch[dev_id] > 500) tpb = TPB52;
	else if (cuda_arch[dev_id] == 500) tpb = TPB50;
	else if (cuda_arch[dev_id] >= 300) tpb = TPB30;
	else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	if (cuda_arch[dev_id] < 500)
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(lyra2v2_gpu_hash_32_2), hipFuncCachePreferShared);

	lyra2v2_gpu_hash_32_1 << <grid2, block2 >> > (threads, startNounce, (uint2*)g_hash);

	lyra2v2_gpu_hash_32_2 << <grid1, block1, 48 * sizeof(uint2) * tpb >> > (threads, startNounce, g_hash);

	lyra2v2_gpu_hash_32_3 << <grid2, block2 >> > (threads, startNounce, (uint2*)g_hash);
	//MyStreamSynchronize(NULL, order, thr_id);
}
