#include "hip/hip_runtime.h"
/**
 * Lyra2 (v1) cuda implementation based on djm34 work
 * tpruvot@github 2015, Nanashi 08/2016 (from 1.8-r2)
 * Lyra2Z implentation for Zcoin based on all the previous
 * djm34 2017
 **/

#include <stdio.h>
#include <memory.h>

#define TPB52 32
#define TPB30 160
#define TPB20 160

#include "cuda_lyra2Z_sm5.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
__device__ uint32_t __shfl(uint32_t a, uint32_t b, uint32_t c);
#define atomicMin()
#define __CUDA_ARCH__ 520
#endif

static uint32_t *h_GNonces[16]; // this need to get fixed as the rest of that routine
static uint32_t *d_GNonces[16];

#define reduceDuplexRow(rowIn, rowInOut, rowOut) { \
	for (int i = 0; i < 8; i++) { \
		for (int j = 0; j < 12; j++) \
			state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
		round_lyra_sm2(state); \
		for (int j = 0; j < 12; j++) \
			Matrix[j + 12 * i][rowOut] ^= state[j]; \
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10+ 12 * i][rowInOut] ^= state[9]; \
		Matrix[11+ 12 * i][rowInOut] ^= state[10]; \
	} \
  }

#define absorbblock(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
  }

__device__ __forceinline__
static void round_lyra_sm2(uint2 *s)
{
	Gfunc(s[0], s[4], s[8], s[12]);
	Gfunc(s[1], s[5], s[9], s[13]);
	Gfunc(s[2], s[6], s[10], s[14]);
	Gfunc(s[3], s[7], s[11], s[15]);
	Gfunc(s[0], s[5], s[10], s[15]);
	Gfunc(s[1], s[6], s[11], s[12]);
	Gfunc(s[2], s[7], s[8], s[13]);
	Gfunc(s[3], s[4], s[9], s[14]);
}

__device__ __forceinline__
void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2 state[16], uint2 Matrix[96][8])
{
#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < 8; i++)
	{
		#pragma unroll
		for (int j = 0; j < 12; j++)
			state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];

		round_lyra_sm2(state);

		#pragma unroll
		for (int j = 0; j < 12; j++)
			Matrix[j + 84 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];

		Matrix[0 +  12 * i][rowInOut] ^= state[11];
		Matrix[1 +  12 * i][rowInOut] ^= state[0];
		Matrix[2 +  12 * i][rowInOut] ^= state[1];
		Matrix[3 +  12 * i][rowInOut] ^= state[2];
		Matrix[4 +  12 * i][rowInOut] ^= state[3];
		Matrix[5 +  12 * i][rowInOut] ^= state[4];
		Matrix[6 +  12 * i][rowInOut] ^= state[5];
		Matrix[7 +  12 * i][rowInOut] ^= state[6];
		Matrix[8 +  12 * i][rowInOut] ^= state[7];
		Matrix[9 +  12 * i][rowInOut] ^= state[8];
		Matrix[10 + 12 * i][rowInOut] ^= state[9];
		Matrix[11 + 12 * i][rowInOut] ^= state[10];
	}
}

#if __CUDA_ARCH__ < 350

__constant__ static uint2 blake2b_IV_sm2[8] = {
	{ 0xf3bcc908, 0x6a09e667 }, { 0x84caa73b, 0xbb67ae85 },
	{ 0xfe94f82b, 0x3c6ef372 }, { 0x5f1d36f1, 0xa54ff53a },
	{ 0xade682d1, 0x510e527f }, { 0x2b3e6c1f, 0x9b05688c },
	{ 0xfb41bd6b, 0x1f83d9ab }, { 0x137e2179, 0x5be0cd19 }
};

__global__ __launch_bounds__(TPB30, 1)
void lyra2Z_gpu_hash_32_sm2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint2 Mask[8] = {
		{ 0x00000020, 0x00000000 },{ 0x00000020, 0x00000000 },
		{ 0x00000020, 0x00000000 },{ 0x00000008, 0x00000000 },
		{ 0x00000008, 0x00000000 },{ 0x00000008, 0x00000000 },
		{ 0x00000080, 0x00000000 },{ 0x00000000, 0x01000000 }
	};
	if (thread < threads)
	{
		uint2 state[16];

		#pragma unroll
		for (int i = 0; i<4; i++) {
			LOHI(state[i].x, state[i].y, g_hash[threads*i + thread]);
		} //password

		#pragma unroll
		for (int i = 0; i<4; i++) {
			state[i + 4] = state[i];
		} //salt

		#pragma unroll
		for (int i = 0; i<8; i++) {
			state[i + 8] = blake2b_IV_sm2[i];
		}

		// blake2blyra x2
		//#pragma unroll 24
		for (int i = 0; i<12; i++) {
			round_lyra_sm2(state);
		}

		for (int i = 0; i<8; i++)
			state[i] ^= Mask[i];

		for (int i = 0; i<12; i++) {
			round_lyra_sm2(state);
		}

		uint2 Matrix[96][8]; // not cool

		// reducedSqueezeRow0
		#pragma unroll 8
		for (int i = 0; i < 8; i++)
		{
			#pragma unroll 12
			for (int j = 0; j<12; j++) {
				Matrix[j + 84 - 12 * i][0] = state[j];
			}
			round_lyra_sm2(state);
		}

		// reducedSqueezeRow1
		#pragma unroll 8
		for (int i = 0; i < 8; i++)
		{
			#pragma unroll 12
			for (int j = 0; j<12; j++) {
				state[j] ^= Matrix[j + 12 * i][0];
			}
			round_lyra_sm2(state);
			#pragma unroll 12
			for (int j = 0; j<12; j++) {
				Matrix[j + 84 - 12 * i][1] = Matrix[j + 12 * i][0] ^ state[j];
			}
		}

		reduceDuplexRowSetup(1, 0, 2, state, Matrix);
		reduceDuplexRowSetup(2, 1, 3, state, Matrix);
		reduceDuplexRowSetup(3, 0, 4, state, Matrix);
		reduceDuplexRowSetup(4, 3, 5, state, Matrix);
		reduceDuplexRowSetup(5, 2, 6, state, Matrix);
		reduceDuplexRowSetup(6, 1, 7, state, Matrix);

		uint32_t rowa;
		uint32_t prev = 7;
		uint32_t iterator = 0;
		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}
		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}
		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}
		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}
		for (uint32_t i = 0; i<8; i++) {
			rowa = state[0].x & 7;
			reduceDuplexRow(prev, rowa, iterator);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		absorbblock(rowa);
		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
	} //thread
}
#else
__global__ void lyra2Z_gpu_hash_32_sm2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resNonces) {}
#endif

#if __CUDA_ARCH__ > 500

#include "cuda_lyra2_vectors.h"
//#include "cuda_vector_uint2x4.h"

#define Nrow 8
#define Ncol 8
#define memshift 3

#define BUF_COUNT 0

__device__ uint2 *DMatrix;

__device__ __forceinline__
void LD4S(uint2 res[3], const int row, const int col, const int thread, const int threads)
{
#if BUF_COUNT != 8
	extern __shared__ uint2 shared_mem[];
	const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;
#endif
#if BUF_COUNT != 0
	const int d0 = (memshift *(Ncol * row + col) * threads + thread)*blockDim.x + threadIdx.x;
#endif

#if BUF_COUNT == 8
	#pragma unroll
	for (int j = 0; j < 3; j++)
		res[j] = *(DMatrix + d0 + j * threads * blockDim.x);
#elif BUF_COUNT == 0
	#pragma unroll
	for (int j = 0; j < 3; j++)
		res[j] = shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
#else
	if (row < BUF_COUNT) {
		#pragma unroll
		for (int j = 0; j < 3; j++)
			res[j] = *(DMatrix + d0 + j * threads * blockDim.x);
	} else {
		#pragma unroll
		for (int j = 0; j < 3; j++)
			res[j] = shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
	}
#endif
}

__device__ __forceinline__
void ST4S(const int row, const int col, const uint2 data[3], const int thread, const int threads)
{
#if BUF_COUNT != 8
	extern __shared__ uint2 shared_mem[];
	const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;
#endif
#if BUF_COUNT != 0
	const int d0 = (memshift *(Ncol * row + col) * threads + thread)*blockDim.x + threadIdx.x;
#endif

#if BUF_COUNT == 8
	#pragma unroll
	for (int j = 0; j < 3; j++)
		*(DMatrix + d0 + j * threads * blockDim.x) = data[j];

#elif BUF_COUNT == 0
	#pragma unroll
	for (int j = 0; j < 3; j++)
		shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data[j];
#else
	if (row < BUF_COUNT) {
	#pragma unroll
		for (int j = 0; j < 3; j++)
			*(DMatrix + d0 + j * threads * blockDim.x) = data[j];
	} else {
	#pragma unroll
		for (int j = 0; j < 3; j++)
			shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data[j];
	}
#endif
}

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl(a, b, c);
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__
void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t *_ptr = (uint32_t*)shared_mem;

	__threadfence_block();
	uint32_t buf = _ptr[thread];

	_ptr[thread] = a;
	__threadfence_block();
	uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	_ptr[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a;
	__threadfence_block();
	uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a1;
	__threadfence_block();
	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a2;
	__threadfence_block();
	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a3;
	__threadfence_block();
	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;
	__threadfence_block();
}
#endif

__device__ __forceinline__ void round_lyra(uint2 s[4])
{
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
}

static __device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc(s[0].w, s[1].w, s[2].w, s[3].w);
	Gfunc(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc(s[0].w, s[1].x, s[2].y, s[3].z);
}

static __device__ __forceinline__
void reduceDuplex(uint2 state[4], uint32_t thread, const uint32_t threads)
{
	uint2 state1[3];

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < Nrow; i++)
	{
		ST4S(0, Ncol - i - 1, state, thread, threads);

		round_lyra(state);
	}

	#pragma unroll 4
	for (int i = 0; i < Nrow; i++)
	{
		LD4S(state1, 0, i, thread, threads);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];
		ST4S(1, Ncol - i - 1, state1, thread, threads);
	}
}

static __device__ __forceinline__
void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], uint32_t thread, const uint32_t threads)
{
	uint2 state1[3], state2[3];

	#pragma unroll 1
	for (int i = 0; i < Nrow; i++)
	{
		LD4S(state1, rowIn, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		ST4S(rowOut, Ncol - i - 1, state1, thread, threads);

		//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		} else {
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(rowInOut, i, state2, thread, threads);
	}
}

static __device__ __forceinline__
void reduceDuplexRowt(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], const uint32_t thread, const uint32_t threads)
{
	for (int i = 0; i < Nrow; i++)
	{
		uint2 state1[3], state2[3];

		LD4S(state1, rowIn, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(rowInOut, i, state2, thread, threads);

		LD4S(state1, rowOut, i, thread, threads);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		ST4S(rowOut, i, state1, thread, threads);
	}
}

#if 0
static __device__ __forceinline__
void reduceDuplexRowt_8(const int rowInOut, uint2* state, const uint32_t thread, const uint32_t threads)
{
	uint2 state1[3], state2[3], last[3];

	LD4S(state1, 2, 0, thread, threads);
	LD4S(last, rowInOut, 0, thread, threads);

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= state1[j] + last[j];

	round_lyra(state);

	//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	} else {
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == 5)
	{
		#pragma unroll
		for (int j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (int i = 1; i < Nrow; i++)
	{
		LD4S(state1, 2, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);
	}

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}
#endif

static __device__ __forceinline__
void reduceDuplexRowt_8_v2(const int rowIn, const int rowOut, const int rowInOut, uint2* state, const uint32_t thread, const uint32_t threads)
{
	uint2 state1[3], state2[3], last[3];

	LD4S(state1, rowIn, 0, thread, threads);
	LD4S(last, rowInOut, 0, thread, threads);

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= state1[j] + last[j];

	round_lyra(state);

	//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	}
	else {
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == rowOut) {
		#pragma unroll
		for (int j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (int i = 1; i < Nrow; i++)
	{
		LD4S(state1, rowIn, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);
	}

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__global__
__launch_bounds__(64, 1)
void lyra2Z_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000008UL, 0x00000000UL,
		0x00000008UL, 0x00000000UL, 0x00000008UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};
	const uint2x4 blake2b_IV[2] = {
		0xf3bcc908lu, 0x6a09e667lu,
		0x84caa73blu, 0xbb67ae85lu,
		0xfe94f82blu, 0x3c6ef372lu,
		0x5f1d36f1lu, 0xa54ff53alu,
		0xade682d1lu, 0x510e527flu,
		0x2b3e6c1flu, 0x9b05688clu,
		0xfb41bd6blu, 0x1f83d9ablu,
		0x137e2179lu, 0x5be0cd19lu
	};
	if (thread < threads)
	{
		uint2x4 state[4];

		state[0].x = state[1].x = __ldg(&g_hash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&g_hash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&g_hash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&g_hash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<12; i++)
			round_lyra(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for (int i = 0; i<12; i++)
			round_lyra(state); //because 12 is not enough

		((uint2x4*)DMatrix)[threads * 0 + thread] = state[0];
		((uint2x4*)DMatrix)[threads * 1 + thread] = state[1];
		((uint2x4*)DMatrix)[threads * 2 + thread] = state[2];
		((uint2x4*)DMatrix)[threads * 3 + thread] = state[3];
	}
}

__global__
__launch_bounds__(TPB52, 1)
void lyra2Z_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = __ldg(&DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x]);
		state[1] = __ldg(&DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x]);
		state[2] = __ldg(&DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x]);
		state[3] = __ldg(&DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x]);

		reduceDuplex(state, thread, threads);
		reduceDuplexRowSetup(1, 0, 2, state, thread, threads);
		reduceDuplexRowSetup(2, 1, 3, state, thread, threads);
		reduceDuplexRowSetup(3, 0, 4, state, thread, threads);
		reduceDuplexRowSetup(4, 3, 5, state, thread, threads);
		reduceDuplexRowSetup(5, 2, 6, state, thread, threads);
		reduceDuplexRowSetup(6, 1, 7, state, thread, threads);

		uint32_t rowa; // = WarpShuffle(state[0].x, 0, 4) & 7;
		uint32_t prev = 7;
		uint32_t iterator = 0;

	//for (uint32_t j=0;j<4;j++) {

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

		for (uint32_t i = 0; i<8; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 3) & 7;
		}

		for (uint32_t i = 0; i<7; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 7;
			reduceDuplexRowt(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 7;
		}

	//}
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt_8_v2(prev,iterator,rowa, state, thread, threads);

		DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x] = state[0];
		DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x] = state[1];
		DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x] = state[2];
		DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__
__launch_bounds__(64, 1)
void lyra2Z_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *resNonces)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&((uint2x4*)DMatrix)[threads * 0 + thread]);
		state[1] = __ldg4(&((uint2x4*)DMatrix)[threads * 1 + thread]);
		state[2] = __ldg4(&((uint2x4*)DMatrix)[threads * 2 + thread]);
		state[3] = __ldg4(&((uint2x4*)DMatrix)[threads * 3 + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);

		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
/*
		g_hash[thread + threads * 0] = state[0].x;
		g_hash[thread + threads * 1] = state[0].y;
		g_hash[thread + threads * 2] = state[0].z;
		g_hash[thread + threads * 3] = state[0].w;
*/
	}
}
#else
#if __CUDA_ARCH__ < 350
__device__ void* DMatrix;
#endif
__global__ void lyra2Z_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
__global__ void lyra2Z_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash) {}
__global__ void lyra2Z_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *resNonces) {}
#endif

__host__
void lyra2Z_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_GNonces[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2 * sizeof(uint32_t));
}

__host__
void lyra2Z_cpu_init_sm2(int thr_id, uint32_t threads)
{
	// just assign the device pointer allocated in main loop
	hipMalloc(&d_GNonces[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2 * sizeof(uint32_t));
}

__host__
void lyra2Z_cpu_free(int thr_id)
{
	hipFree(d_GNonces[thr_id]);
	hipHostFree(h_GNonces[thr_id]);
}

__host__
uint32_t lyra2Z_getSecNonce(int thr_id, int num)
{
	uint32_t results[2];
	memset(results, 0xFF, sizeof(results));
	hipMemcpy(results, d_GNonces[thr_id], sizeof(results), hipMemcpyDeviceToHost);
	if (results[1] == results[0])
		return UINT32_MAX;
	return results[num];
}

__host__
void lyra2Z_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);
}

__host__
uint32_t lyra2Z_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, bool gtx750ti)
{
	uint32_t result = UINT32_MAX;
	hipMemset(d_GNonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (device_sm[dev_id] == 500)
		tpb = TPB50;
	if (device_sm[dev_id] == 200)
		tpb = TPB20;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	dim3 grid3((threads + tpb - 1) / tpb);
	dim3 block3(tpb);

	if (device_sm[dev_id] >= 520)
	{
		lyra2Z_gpu_hash_32_1 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2Z_gpu_hash_32_2 <<< grid1, block1, 24 * (8 - 0) * sizeof(uint2) * tpb >>> (threads, startNounce, d_hash);

		lyra2Z_gpu_hash_32_3 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash, d_GNonces[thr_id]);
	}
	else if (device_sm[dev_id] == 500 || device_sm[dev_id] == 350)
	{
		size_t shared_mem = 0;

		if (gtx750ti)
			shared_mem = 8192;
		else
			shared_mem = 6144;

		lyra2Z_gpu_hash_32_1_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2Z_gpu_hash_32_2_sm5 <<< grid1, block1, shared_mem >>> (threads, startNounce, (uint2*)d_hash);

		lyra2Z_gpu_hash_32_3_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash, d_GNonces[thr_id]);
	}
	else
		lyra2Z_gpu_hash_32_sm2 <<< grid3, block3 >>> (threads, startNounce, d_hash, d_GNonces[thr_id]);

	// get first found nonce
	hipMemcpy(h_GNonces[thr_id], d_GNonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	result = *h_GNonces[thr_id];

	return result;
}
