#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint64_t *d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
//extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);

//extern void keccak256_sm3_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
//extern void keccak256_sm3_init(int thr_id, uint32_t threads);
//extern void keccak256_sm3_free(int thr_id);

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);
extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* d_matrix);

extern void bmw256_setTarget(const void *ptarget);
extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_free(int thr_id);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces);

void lyra2v2_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);

	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_lyra2v2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] < 500) ? 18 : is_windows() ? 19 : 20;
	if (strstr(device_name[dev_id], "GTX 10")) intensity = 20;
	uint32_t throughput = cuda_default_throughput(dev_id, 1UL << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		size_t matrix_sz = 16 * sizeof(uint64_t) * 4 * 3;
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);
		//keccak256_sm3_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		cuda_get_arch(thr_id); // cuda_arch[] also used in cubehash256

		// SM 3 implentation requires a bit more memory
		if (device_sm[dev_id] < 500 || cuda_arch[dev_id] < 500)
			matrix_sz = 16 * sizeof(uint64_t) * 4 * 4;
			
		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
		lyra2v2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		api_set_throughput(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	bmw256_setTarget(ptarget);

	do {
		int order = 0;

		//blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		//keccak256_sm3_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);

		memset(work->nonces, 0, sizeof(work->nonces));
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], work->nonces);

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			lyra2v2_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2v2_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && !abort_flag);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_lyra2v2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	bmw256_cpu_free(thr_id);
	//keccak256_sm3_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
