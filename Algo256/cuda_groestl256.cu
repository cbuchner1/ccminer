#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>



extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];
uint32_t *d_gnounce[8];
uint32_t *d_GNonce[8];
__constant__ uint32_t pTarget[8];

#include "cuda_helper.h"


////////////////////////////////////////////////////////////////////////

#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define C32e(x)     ((SPH_C32(x) >> 24) \
                    | ((SPH_C32(x) >>  8) & SPH_C32(0x0000FF00)) \
                    | ((SPH_C32(x) <<  8) & SPH_C32(0x00FF0000)) \
                    | ((SPH_C32(x) << 24) & SPH_C32(0xFF000000)))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)
#define MAXWELL_OR_FERMI 1
#if MAXWELL_OR_FERMI
#define USE_SHARED 1
// Maxwell and Fermi cards get the best speed with SHARED access it seems.
#if USE_SHARED
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
#else
#define T0up(x) tex1Dfetch(t0up2, x)
#define T0dn(x) tex1Dfetch(t0dn2, x)
#define T1up(x) tex1Dfetch(t1up2, x)
#define T1dn(x) tex1Dfetch(t1dn2, x)
#define T2up(x) tex1Dfetch(t2up2, x)
#define T2dn(x) tex1Dfetch(t2dn2, x)
#define T3up(x) tex1Dfetch(t3up2, x)
#define T3dn(x) tex1Dfetch(t3dn2, x)
#endif
#else
#define USE_SHARED 1
// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) tex1Dfetch(t0dn2, x)
#define T1up(x) tex1Dfetch(t1up2, x)
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) tex1Dfetch(t2up2, x)
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) tex1Dfetch(t3dn2, x)
#endif

texture<unsigned int, 1, hipReadModeElementType> t0up2;
texture<unsigned int, 1, hipReadModeElementType> t0dn2;
texture<unsigned int, 1, hipReadModeElementType> t1up2;
texture<unsigned int, 1, hipReadModeElementType> t1dn2;
texture<unsigned int, 1, hipReadModeElementType> t2up2;
texture<unsigned int, 1, hipReadModeElementType> t2dn2;
texture<unsigned int, 1, hipReadModeElementType> t3up2;
texture<unsigned int, 1, hipReadModeElementType> t3dn2;


#define RSTT(d0, d1, a, b0, b1, b2, b3, b4, b5, b6, b7)   do { \
		t[d0] = T0up(B32_0(a[b0])) \
			^ T1up(B32_1(a[b1])) \
			^ T2up(B32_2(a[b2])) \
			^ T3up(B32_3(a[b3])) \
			^ T0dn(B32_0(a[b4])) \
			^ T1dn(B32_1(a[b5])) \
			^ T2dn(B32_2(a[b6])) \
			^ T3dn(B32_3(a[b7])); \
		t[d1] = T0dn(B32_0(a[b0])) \
			^ T1dn(B32_1(a[b1])) \
			^ T2dn(B32_2(a[b2])) \
			^ T3dn(B32_3(a[b3])) \
			^ T0up(B32_0(a[b4])) \
			^ T1up(B32_1(a[b5])) \
			^ T2up(B32_2(a[b6])) \
			^ T3up(B32_3(a[b7])); \
	} while (0)


extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];


__device__ __forceinline__ void groestl256_perm_P(uint32_t *a, uint32_t *mixtabs)
{
	
        uint32_t t[16];
	#pragma unroll 
	for (int r = 0; r<10; r++)
	{

		    a[0x0] ^= PC32up(0x00, r); 
			a[0x2] ^= PC32up(0x10, r); 
			a[0x4] ^= PC32up(0x20, r); 
			a[0x6] ^= PC32up(0x30, r); 
			a[0x8] ^= PC32up(0x40, r);
			a[0xA] ^= PC32up(0x50, r); 
			a[0xC] ^= PC32up(0x60, r); 
			a[0xE] ^= PC32up(0x70, r); 
			RSTT(0x0, 0x1, a, 0x0, 0x2, 0x4, 0x6, 0x9, 0xB, 0xD, 0xF); 
			RSTT(0x2, 0x3, a, 0x2, 0x4, 0x6, 0x8, 0xB, 0xD, 0xF, 0x1); 
			RSTT(0x4, 0x5, a, 0x4, 0x6, 0x8, 0xA, 0xD, 0xF, 0x1, 0x3); 
			RSTT(0x6, 0x7, a, 0x6, 0x8, 0xA, 0xC, 0xF, 0x1, 0x3, 0x5); 
			RSTT(0x8, 0x9, a, 0x8, 0xA, 0xC, 0xE, 0x1, 0x3, 0x5, 0x7); 
			RSTT(0xA, 0xB, a, 0xA, 0xC, 0xE, 0x0, 0x3, 0x5, 0x7, 0x9); 
			RSTT(0xC, 0xD, a, 0xC, 0xE, 0x0, 0x2, 0x5, 0x7, 0x9, 0xB); 
			RSTT(0xE, 0xF, a, 0xE, 0x0, 0x2, 0x4, 0x7, 0x9, 0xB, 0xD); 

#pragma unroll 
		for (int k = 0; k<16; k++)
			a[k] = t[k];

	}
}

__device__ __forceinline__ void groestl256_perm_Pf( uint32_t *a, uint32_t *mixtabs)
{

	uint32_t t[16];
#pragma unroll 
	for (int r = 0; r<9; r++)
	{
		a[0x0] ^= PC32up(0x00, r);
		a[0x2] ^= PC32up(0x10, r);
		a[0x4] ^= PC32up(0x20, r);
		a[0x6] ^= PC32up(0x30, r);
		a[0x8] ^= PC32up(0x40, r);
		a[0xA] ^= PC32up(0x50, r);
		a[0xC] ^= PC32up(0x60, r);
		a[0xE] ^= PC32up(0x70, r);
		RSTT(0x0, 0x1, a, 0x0, 0x2, 0x4, 0x6, 0x9, 0xB, 0xD, 0xF);
		RSTT(0x2, 0x3, a, 0x2, 0x4, 0x6, 0x8, 0xB, 0xD, 0xF, 0x1);
		RSTT(0x4, 0x5, a, 0x4, 0x6, 0x8, 0xA, 0xD, 0xF, 0x1, 0x3);
		RSTT(0x6, 0x7, a, 0x6, 0x8, 0xA, 0xC, 0xF, 0x1, 0x3, 0x5);
		RSTT(0x8, 0x9, a, 0x8, 0xA, 0xC, 0xE, 0x1, 0x3, 0x5, 0x7);
		RSTT(0xA, 0xB, a, 0xA, 0xC, 0xE, 0x0, 0x3, 0x5, 0x7, 0x9);
		RSTT(0xC, 0xD, a, 0xC, 0xE, 0x0, 0x2, 0x5, 0x7, 0x9, 0xB);
		RSTT(0xE, 0xF, a, 0xE, 0x0, 0x2, 0x4, 0x7, 0x9, 0xB, 0xD);

#pragma unroll 
		for (int k = 0; k<16; k++)
			a[k] = t[k];

	}
	a[0x0] ^= 0x09;
	a[0x2] ^= 0x19;
	a[0x4] ^= 0x29;
	a[0xE] ^= 0x79;
	RSTT(0xE, 0xF, a, 0xE, 0x0, 0x2, 0x4, 0x7, 0x9, 0xB, 0xD);
	    a[14] = t[14];
		a[15] = t[15];

}

__device__ __forceinline__ void groestl256_perm_Q(uint32_t *a, uint32_t *mixtabs)
{
		uint32_t t[16];
	#pragma unroll 
	for (int r = 0; r<10; r++)
	{
		    a[0x0] = ~a[0x0];
			a[0x1] ^= QC32dn(0x00, r); 
			a[0x2] = ~a[0x2];
			a[0x3] ^= QC32dn(0x10, r); 
			a[0x4] = ~a[0x4];
			a[0x5] ^= QC32dn(0x20, r); 
			a[0x6] = ~a[0x6];
			a[0x7] ^= QC32dn(0x30, r); 
			a[0x8] = ~a[0x8];
			a[0x9] ^= QC32dn(0x40, r); 
			a[0xA] = ~a[0xA];
			a[0xB] ^= QC32dn(0x50, r); 
			a[0xC] = ~a[0xC];
			a[0xD] ^= QC32dn(0x60, r); 
			a[0xE] = ~a[0xE];
			a[0xF] ^= QC32dn(0x70, r); 
			RSTT(0x0, 0x1, a, 0x2, 0x6, 0xA, 0xE, 0x1, 0x5, 0x9, 0xD); 
			RSTT(0x2, 0x3, a, 0x4, 0x8, 0xC, 0x0, 0x3, 0x7, 0xB, 0xF); 
			RSTT(0x4, 0x5, a, 0x6, 0xA, 0xE, 0x2, 0x5, 0x9, 0xD, 0x1); 
			RSTT(0x6, 0x7, a, 0x8, 0xC, 0x0, 0x4, 0x7, 0xB, 0xF, 0x3); 
			RSTT(0x8, 0x9, a, 0xA, 0xE, 0x2, 0x6, 0x9, 0xD, 0x1, 0x5); 
			RSTT(0xA, 0xB, a, 0xC, 0x0, 0x4, 0x8, 0xB, 0xF, 0x3, 0x7); 
			RSTT(0xC, 0xD, a, 0xE, 0x2, 0x6, 0xA, 0xD, 0x1, 0x5, 0x9); 
			RSTT(0xE, 0xF, a, 0x0, 0x4, 0x8, 0xC, 0xF, 0x3, 0x7, 0xB); 

#pragma unroll 
		for (int k = 0; k<16; k++)
			a[k] = t[k];


	}
}

__global__ __launch_bounds__(256,1) void groestl256_gpu_hash32(int threads, uint32_t startNounce, uint64_t *outputHash, uint32_t *nonceVector)
{
	
#if USE_SHARED
  extern __shared__ uint32_t mixtabs[];

	if (threadIdx.x < 256)
	{
		*((uint32_t*)mixtabs + (threadIdx.x)) = tex1Dfetch(t0up2, threadIdx.x);
		*((uint32_t*)mixtabs + (256 + threadIdx.x)) = tex1Dfetch(t0dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (512 + threadIdx.x)) = tex1Dfetch(t1up2, threadIdx.x);
		*((uint32_t*)mixtabs + (768 + threadIdx.x)) = tex1Dfetch(t1dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (1024 + threadIdx.x)) = tex1Dfetch(t2up2, threadIdx.x);
		*((uint32_t*)mixtabs + (1280 + threadIdx.x)) = tex1Dfetch(t2dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (1536 + threadIdx.x)) = tex1Dfetch(t3up2, threadIdx.x);
		*((uint32_t*)mixtabs + (1792 + threadIdx.x)) = tex1Dfetch(t3dn2, threadIdx.x);
	}

	__syncthreads();
#endif


	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// GROESTL
		uint32_t message[16];
		uint32_t state[16];

		uint32_t nonce = startNounce +thread;

#pragma unroll 
		for (int k = 0; k<4; k++) LOHI(message[2*k],message[2*k+1],outputHash[k*threads+thread]);
#pragma unroll 
		for (int k = 9; k<15; k++)
			message[k] = 0;

		message[8] = 0x80;
		message[15] = 0x01000000;

#pragma unroll 16
		for (int u = 0; u<16; u++) state[u] = message[u];
		state[15] ^= 0x10000;

		// Perm
		
#if USE_SHARED
		
		groestl256_perm_P(state, mixtabs);		
		state[15] ^= 0x10000;		
		groestl256_perm_Q(message, mixtabs);
		
#else
		groestl256_perm_P(state, NULL);
		state[15] ^= 0x10000;
		groestl256_perm_Q(message, NULL);
#endif
#pragma unroll 16
		for (int u = 0; u<16; u++) state[u] ^= message[u];
		message[14] = state[14];
		message[15] = state[15];

#if USE_SHARED
		groestl256_perm_Pf(state, mixtabs);
#else
		groestl256_perm_Pf(state, NULL);
#endif
state[14] ^= message[14];
state[15] ^= message[15];

		if (((uint64_t*)state)[7] <= ((uint64_t*)pTarget)[3]) { nonceVector[0] = nonce; }
}
}


#define texDef(texname, texmem, texsource, texsize) \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); } \





   
void groestl256_cpu_init(int thr_id, int threads)
{
    
	// Texturen mit obigem Makro initialisieren
	texDef(t0up2, d_T0up, T0up_cpu, sizeof(uint32_t) * 256);
	texDef(t0dn2, d_T0dn, T0dn_cpu, sizeof(uint32_t) * 256);
	texDef(t1up2, d_T1up, T1up_cpu, sizeof(uint32_t) * 256);
	texDef(t1dn2, d_T1dn, T1dn_cpu, sizeof(uint32_t) * 256);
	texDef(t2up2, d_T2up, T2up_cpu, sizeof(uint32_t) * 256);
	texDef(t2dn2, d_T2dn, T2dn_cpu, sizeof(uint32_t) * 256);
	texDef(t3up2, d_T3up, T3up_cpu, sizeof(uint32_t) * 256);
	texDef(t3dn2, d_T3dn, T3dn_cpu, sizeof(uint32_t) * 256);

	hipMalloc(&d_GNonce[thr_id], sizeof(uint32_t)); 
	hipHostMalloc(&d_gnounce[thr_id], 1*sizeof(uint32_t));
} 


__host__ uint32_t groestl256_cpu_hash_32(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_GNonce[thr_id], 0xff, sizeof(uint32_t));



	const int threadsperblock = 256;  


	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

#if USE_SHARED
	size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
	size_t shared_size = 0;
#endif	
	groestl256_gpu_hash32<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash, d_GNonce[thr_id]);

	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(d_gnounce[thr_id], d_GNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();
	result = *d_gnounce[thr_id];

	return result;
}


__host__ void groestl256_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}