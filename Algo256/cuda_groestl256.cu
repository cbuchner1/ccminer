#include "hip/hip_runtime.h"
#include <memory.h>

#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#include "cuda_helper.h"

static uint32_t *h_GNonces[MAX_GPUS];
static uint32_t *d_GNonces[MAX_GPUS];
static unsigned int* d_textures[MAX_GPUS][8];

__constant__ uint32_t pTarget[8];

#define C32e(x) \
	  ((SPH_C32(x) >> 24) \
	| ((SPH_C32(x) >>  8) & SPH_C32(0x0000FF00)) \
	| ((SPH_C32(x) <<  8) & SPH_C32(0x00FF0000)) \
	| ((SPH_C32(x) << 24) & SPH_C32(0xFF000000)))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)

#define MAXWELL_OR_FERMI 1
#if MAXWELL_OR_FERMI
	#define USE_SHARED 1
	// Maxwell and Fermi cards get the best speed with SHARED access it seems.
	#if USE_SHARED
	#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
	#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
	#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
	#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
	#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
	#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
	#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
	#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
	#else
	#define T0up(x) tex1Dfetch(t0up2, x)
	#define T0dn(x) tex1Dfetch(t0dn2, x)
	#define T1up(x) tex1Dfetch(t1up2, x)
	#define T1dn(x) tex1Dfetch(t1dn2, x)
	#define T2up(x) tex1Dfetch(t2up2, x)
	#define T2dn(x) tex1Dfetch(t2dn2, x)
	#define T3up(x) tex1Dfetch(t3up2, x)
	#define T3dn(x) tex1Dfetch(t3dn2, x)
	#endif
#else
	#define USE_SHARED 1
	// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
	#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
	#define T0dn(x) tex1Dfetch(t0dn2, x)
	#define T1up(x) tex1Dfetch(t1up2, x)
	#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
	#define T2up(x) tex1Dfetch(t2up2, x)
	#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
	#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
	#define T3dn(x) tex1Dfetch(t3dn2, x)
#endif

static texture<unsigned int, 1, hipReadModeElementType> t0up2;
static texture<unsigned int, 1, hipReadModeElementType> t0dn2;
static texture<unsigned int, 1, hipReadModeElementType> t1up2;
static texture<unsigned int, 1, hipReadModeElementType> t1dn2;
static texture<unsigned int, 1, hipReadModeElementType> t2up2;
static texture<unsigned int, 1, hipReadModeElementType> t2dn2;
static texture<unsigned int, 1, hipReadModeElementType> t3up2;
static texture<unsigned int, 1, hipReadModeElementType> t3dn2;

#define RSTT(d0, d1, a, b0, b1, b2, b3, b4, b5, b6, b7) do { \
	t[d0] = T0up(B32_0(a[b0])) \
		^ T1up(B32_1(a[b1])) \
		^ T2up(B32_2(a[b2])) \
		^ T3up(B32_3(a[b3])) \
		^ T0dn(B32_0(a[b4])) \
		^ T1dn(B32_1(a[b5])) \
		^ T2dn(B32_2(a[b6])) \
		^ T3dn(B32_3(a[b7])); \
	t[d1] = T0dn(B32_0(a[b0])) \
		^ T1dn(B32_1(a[b1])) \
		^ T2dn(B32_2(a[b2])) \
		^ T3dn(B32_3(a[b3])) \
		^ T0up(B32_0(a[b4])) \
		^ T1up(B32_1(a[b5])) \
		^ T2up(B32_2(a[b6])) \
		^ T3up(B32_3(a[b7])); \
	} while (0)


extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

__device__ __forceinline__
void groestl256_perm_P(uint32_t thread,uint32_t *a, char *mixtabs)
{
	#pragma unroll 10
	for (int r = 0; r<10; r++)
	{
		uint32_t t[16];

		a[0x0] ^= PC32up(0x00, r);
		a[0x2] ^= PC32up(0x10, r);
		a[0x4] ^= PC32up(0x20, r);
		a[0x6] ^= PC32up(0x30, r);
		a[0x8] ^= PC32up(0x40, r);
		a[0xA] ^= PC32up(0x50, r);
		a[0xC] ^= PC32up(0x60, r);
		a[0xE] ^= PC32up(0x70, r);
		RSTT(0x0, 0x1, a, 0x0, 0x2, 0x4, 0x6, 0x9, 0xB, 0xD, 0xF);
		RSTT(0x2, 0x3, a, 0x2, 0x4, 0x6, 0x8, 0xB, 0xD, 0xF, 0x1);
		RSTT(0x4, 0x5, a, 0x4, 0x6, 0x8, 0xA, 0xD, 0xF, 0x1, 0x3);
		RSTT(0x6, 0x7, a, 0x6, 0x8, 0xA, 0xC, 0xF, 0x1, 0x3, 0x5);
		RSTT(0x8, 0x9, a, 0x8, 0xA, 0xC, 0xE, 0x1, 0x3, 0x5, 0x7);
		RSTT(0xA, 0xB, a, 0xA, 0xC, 0xE, 0x0, 0x3, 0x5, 0x7, 0x9);
		RSTT(0xC, 0xD, a, 0xC, 0xE, 0x0, 0x2, 0x5, 0x7, 0x9, 0xB);
		RSTT(0xE, 0xF, a, 0xE, 0x0, 0x2, 0x4, 0x7, 0x9, 0xB, 0xD);

		#pragma unroll 16
		for (int k = 0; k<16; k++)
			a[k] = t[k];
	}
}

__device__ __forceinline__
void groestl256_perm_Q(uint32_t thread, uint32_t *a, char *mixtabs)
{
	#pragma unroll
	for (int r = 0; r<10; r++)
	{
		uint32_t t[16];

		a[0x0] ^= QC32up(0x00, r);
		a[0x1] ^= QC32dn(0x00, r);
		a[0x2] ^= QC32up(0x10, r);
		a[0x3] ^= QC32dn(0x10, r);
		a[0x4] ^= QC32up(0x20, r);
		a[0x5] ^= QC32dn(0x20, r);
		a[0x6] ^= QC32up(0x30, r);
		a[0x7] ^= QC32dn(0x30, r);
		a[0x8] ^= QC32up(0x40, r);
		a[0x9] ^= QC32dn(0x40, r);
		a[0xA] ^= QC32up(0x50, r);
		a[0xB] ^= QC32dn(0x50, r);
		a[0xC] ^= QC32up(0x60, r);
		a[0xD] ^= QC32dn(0x60, r);
		a[0xE] ^= QC32up(0x70, r);
		a[0xF] ^= QC32dn(0x70, r);
		RSTT(0x0, 0x1, a, 0x2, 0x6, 0xA, 0xE, 0x1, 0x5, 0x9, 0xD);
		RSTT(0x2, 0x3, a, 0x4, 0x8, 0xC, 0x0, 0x3, 0x7, 0xB, 0xF);
		RSTT(0x4, 0x5, a, 0x6, 0xA, 0xE, 0x2, 0x5, 0x9, 0xD, 0x1);
		RSTT(0x6, 0x7, a, 0x8, 0xC, 0x0, 0x4, 0x7, 0xB, 0xF, 0x3);
		RSTT(0x8, 0x9, a, 0xA, 0xE, 0x2, 0x6, 0x9, 0xD, 0x1, 0x5);
		RSTT(0xA, 0xB, a, 0xC, 0x0, 0x4, 0x8, 0xB, 0xF, 0x3, 0x7);
		RSTT(0xC, 0xD, a, 0xE, 0x2, 0x6, 0xA, 0xD, 0x1, 0x5, 0x9);
		RSTT(0xE, 0xF, a, 0x0, 0x4, 0x8, 0xC, 0xF, 0x3, 0x7, 0xB);

		#pragma unroll
		for (int k = 0; k<16; k++)
			a[k] = t[k];
	}
}

__global__ __launch_bounds__(256,1)
void groestl256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash, uint32_t *resNonces)
{
#if USE_SHARED
	extern __shared__ char mixtabs[];

	if (threadIdx.x < 256) {
		*((uint32_t*)mixtabs + (threadIdx.x)) = tex1Dfetch(t0up2, threadIdx.x);
		*((uint32_t*)mixtabs + (256 + threadIdx.x)) = tex1Dfetch(t0dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (512 + threadIdx.x)) = tex1Dfetch(t1up2, threadIdx.x);
		*((uint32_t*)mixtabs + (768 + threadIdx.x)) = tex1Dfetch(t1dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (1024 + threadIdx.x)) = tex1Dfetch(t2up2, threadIdx.x);
		*((uint32_t*)mixtabs + (1280 + threadIdx.x)) = tex1Dfetch(t2dn2, threadIdx.x);
		*((uint32_t*)mixtabs + (1536 + threadIdx.x)) = tex1Dfetch(t3up2, threadIdx.x);
		*((uint32_t*)mixtabs + (1792 + threadIdx.x)) = tex1Dfetch(t3dn2, threadIdx.x);
	}

	__syncthreads();
#endif

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// GROESTL
		uint32_t message[16];
		uint32_t state[16];

		#pragma unroll
		for (int k = 0; k<4; k++)
			LOHI(message[2*k], message[2*k+1], outputHash[k*threads+thread]);

		#pragma unroll
		for (int k = 9; k<15; k++)
			message[k] = 0;

		message[8] = 0x80;
		message[15] = 0x01000000;

		#pragma unroll 16
		for (int u = 0; u<16; u++)
			state[u] = message[u];

		state[15] ^= 0x10000;

		// Perm

#if USE_SHARED
		groestl256_perm_P(thread, state, mixtabs);
		state[15] ^= 0x10000;
		groestl256_perm_Q(thread, message, mixtabs);
#else
		groestl256_perm_P(thread, state, NULL);
		state[15] ^= 0x10000;
		groestl256_perm_P(thread, message, NULL);
#endif
		#pragma unroll 16
		for (int u = 0; u<16; u++) state[u] ^= message[u];
		#pragma unroll 16
		for (int u = 0; u<16; u++) message[u] = state[u];
#if USE_SHARED
		groestl256_perm_P(thread, message, mixtabs);
#else
		groestl256_perm_P(thread, message, NULL);
#endif
		state[14] ^= message[14];
		state[15] ^= message[15];

		uint32_t nonce = startNounce + thread;
		if (state[15] <= pTarget[7]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
	}
}

#define texDef(id, texname, texmem, texsource, texsize) { \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	d_textures[thr_id][id] = texmem; \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); \
	} \
}

__host__
void groestl256_cpu_init(int thr_id, uint32_t threads)
{
	// Texturen mit obigem Makro initialisieren
	texDef(0, t0up2, d_T0up, T0up_cpu, sizeof(uint32_t) * 256);
	texDef(1, t0dn2, d_T0dn, T0dn_cpu, sizeof(uint32_t) * 256);
	texDef(2, t1up2, d_T1up, T1up_cpu, sizeof(uint32_t) * 256);
	texDef(3, t1dn2, d_T1dn, T1dn_cpu, sizeof(uint32_t) * 256);
	texDef(4, t2up2, d_T2up, T2up_cpu, sizeof(uint32_t) * 256);
	texDef(5, t2dn2, d_T2dn, T2dn_cpu, sizeof(uint32_t) * 256);
	texDef(6, t3up2, d_T3up, T3up_cpu, sizeof(uint32_t) * 256);
	texDef(7, t3dn2, d_T3dn, T3dn_cpu, sizeof(uint32_t) * 256);

	hipMalloc(&d_GNonces[thr_id], 2*sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2*sizeof(uint32_t));
}

__host__
void groestl256_cpu_free(int thr_id)
{
	for (int i=0; i<8; i++)
		hipFree(d_textures[thr_id][i]);

	hipFree(d_GNonces[thr_id]);
	hipHostFree(h_GNonces[thr_id]);
}

__host__
uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	uint32_t result = UINT32_MAX;
	hipMemset(d_GNonces[thr_id], 0xff, 2*sizeof(uint32_t));
	const uint32_t threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

#if USE_SHARED
	size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
	size_t shared_size = 0;
#endif
	groestl256_gpu_hash_32<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash, d_GNonces[thr_id]);

	MyStreamSynchronize(NULL, order, thr_id);

	// get first found nonce
	hipMemcpy(h_GNonces[thr_id], d_GNonces[thr_id], 1*sizeof(uint32_t), hipMemcpyDeviceToHost);
	result = *h_GNonces[thr_id];

	return result;
}

__host__
uint32_t groestl256_getSecNonce(int thr_id, int num)
{
	uint32_t results[2];
	memset(results, 0xFF, sizeof(results));
	hipMemcpy(results, d_GNonces[thr_id], sizeof(results), hipMemcpyDeviceToHost);
	if (results[1] == results[0])
		return UINT32_MAX;
	return results[num];
}

__host__
void groestl256_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);
}
