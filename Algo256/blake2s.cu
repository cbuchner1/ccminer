#include "hip/hip_runtime.h"
/**
 * Based on the SPH implementation of blake2s
 * Provos Alexis - 2016
 */

#include "miner.h"

#include <string.h>
#include <stdint.h>

#include "sph/blake2s.h"
#include "sph/sph_types.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

#include "cuda_helper.h"

#ifdef __CUDA_ARCH__

__device__ __forceinline__
uint32_t ROR8(const uint32_t a) {
	return __byte_perm(a, 0, 0x0321);
}

__device__ __forceinline__
uint32_t ROL16(const uint32_t a) {
	return __byte_perm(a, 0, 0x1032);
}

#else
#define ROR8(u)  (u >> 8)
#define ROL16(u) (u << 16)
#endif

__device__ __forceinline__
uint32_t xor3x(uint32_t a, uint32_t b, uint32_t c)
{
	uint32_t result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b),"r"(c)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

static const uint32_t blake2s_IV[8] = {
	0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
	0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL
};

static const uint8_t blake2s_sigma[10][16] = {
	{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
	{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 },
	{  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 },
	{  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 },
	{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 },
	{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 },
	{  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 },
	{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 },
};

#define G(r,i,a,b,c,d) \
	do { \
		a = a + b + m[blake2s_sigma[r][2*i+0]]; \
		d = SPH_ROTR32(d ^ a, 16); \
		c = c + d; \
		b = SPH_ROTR32(b ^ c, 12); \
		a = a + b + m[blake2s_sigma[r][2*i+1]]; \
		d = SPH_ROTR32(d ^ a, 8); \
		c = c + d; \
		b = SPH_ROTR32(b ^ c, 7); \
	} while(0)
#define ROUND(r)  \
	do { \
		G(r,0,v[0],v[4],v[ 8],v[12]); \
		G(r,1,v[1],v[5],v[ 9],v[13]); \
		G(r,2,v[2],v[6],v[10],v[14]); \
		G(r,3,v[3],v[7],v[11],v[15]); \
		G(r,4,v[0],v[5],v[10],v[15]); \
		G(r,5,v[1],v[6],v[11],v[12]); \
		G(r,6,v[2],v[7],v[ 8],v[13]); \
		G(r,7,v[3],v[4],v[ 9],v[14]); \
	} while(0)

extern "C" void blake2s_hash(void *output, const void *input)
{
	uint32_t m[16];
	uint32_t v[16];
	uint32_t h[8];

	uint32_t *in = (uint32_t*)input;
//	COMPRESS
	for(int i = 0; i < 16; ++i )
		m[i] = in[i];

	h[0] = 0x01010020 ^ blake2s_IV[0];
	h[1] = blake2s_IV[1];
	h[2] = blake2s_IV[2];
	h[3] = blake2s_IV[3];
	h[4] = blake2s_IV[4];
	h[5] = blake2s_IV[5];
	h[6] = blake2s_IV[6];
	h[7] = blake2s_IV[7];

	for(int i = 0; i < 8; ++i )
		v[i] = h[i];

	v[ 8] = blake2s_IV[0];		v[ 9] = blake2s_IV[1];
	v[10] = blake2s_IV[2];		v[11] = blake2s_IV[3];
	v[12] = 64 ^ blake2s_IV[4];	v[13] = blake2s_IV[5];
	v[14] = blake2s_IV[6];		v[15] = blake2s_IV[7];

	ROUND( 0 ); ROUND( 1 );
	ROUND( 2 ); ROUND( 3 );
	ROUND( 4 ); ROUND( 5 );
	ROUND( 6 ); ROUND( 7 );
	ROUND( 8 ); ROUND( 9 );

	for(size_t i = 0; i < 8; ++i)
		h[i] ^= v[i] ^ v[i + 8];

//	COMPRESS
	m[0] = in[16]; m[1] = in[17];
	m[2] = in[18]; m[3] = in[19];
	for(size_t i = 4; i < 16; ++i)
		m[i] = 0;

	for(size_t i = 0; i < 8; ++i)
		v[i] = h[i];

	v[ 8] = blake2s_IV[0];		v[ 9] = blake2s_IV[1];
	v[10] = blake2s_IV[2];		v[11] = blake2s_IV[3];
	v[12] = 0x50 ^ blake2s_IV[4];	v[13] = blake2s_IV[5];
	v[14] = ~blake2s_IV[6];		v[15] = blake2s_IV[7];

	ROUND( 0 ); ROUND( 1 );
	ROUND( 2 ); ROUND( 3 );
	ROUND( 4 ); ROUND( 5 );
	ROUND( 6 ); ROUND( 7 );
	ROUND( 8 ); ROUND( 9 );

	for(size_t i = 0; i < 8; ++i)
		h[i] ^= v[i] ^ v[i + 8];

	memcpy(output, h, 32);
}

#define TPB 1024
#define NPT 256
#define maxResults 16
#define NBN 1

__constant__ uint32_t _ALIGN(32) midstate[20];

static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

#define GS4(a,b,c,d,e,f,a1,b1,c1,d1,e1,f1,a2,b2,c2,d2,e2,f2,a3,b3,c3,d3,e3,f3){ \
	a += b + e;		a1+= b1 + e1;	 	a2+= b2 + e2;		a3+= b3 + e3; \
	d  = ROL16( d ^ a);	d1 = ROL16(d1 ^ a1);	d2 = ROL16(d2 ^ a2);	d3 = ROL16(d3 ^ a3); \
	c +=d; 			c1+=d1;			c2+=d2;			c3+=d3;\
	b  = ROTR32(b ^ c, 12); b1 = ROTR32(b1^c1, 12);	b2 = ROTR32(b2^c2, 12);	b3 = ROTR32(b3^c3, 12); \
	a += b + f;		a1+= b1 + f1;		a2+= b2 + f2;		a3+= b3 + f3; \
	d  = ROR8(d ^ a);	d1 = ROR8(d1^a1);	d2 = ROR8(d2^a2);	d3 = ROR8(d3^a3); \
	c  += d;		c1 += d1;		c2 += d2;		c3 += d3;\
	b  = ROTR32(b ^ c, 7);	b1 = ROTR32(b1^c1, 7);	b2 = ROTR32(b2^c2, 7);	b3 = ROTR32(b3^c3, 7); \
}

__global__ __launch_bounds__(TPB,1)
void blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t ptarget7)
{
	const uint32_t step = gridDim.x * blockDim.x;

	uint32_t m[ 3];
	uint32_t v[16];

	m[0] = midstate[16];
	m[1] = midstate[17];
	m[2] = midstate[18];

	const uint32_t h7 = midstate[19];

	for(uint32_t thread   = blockDim.x * blockIdx.x + threadIdx.x ; thread <threads; thread+=step){
		#pragma unroll
		for(int i=0;i<16;i++){
			v[ i] = midstate[ i];
		}

		uint32_t nonce = cuda_swab32(startNonce + thread);
//		Round( 0 );
		v[ 1] += nonce;
		v[13] = ROR8(v[13] ^ v[ 1]);
		v[ 9] += v[13];
		v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);

		v[ 1]+= v[ 6];
		v[ 0]+= v[ 5];

		v[12] = ROL16(v[12] ^ v[ 1]);
		v[13] = ROL16(v[13] ^ v[ 2]);
		v[15] = ROL16(v[15] ^ v[ 0]);

		v[11]+= v[12];				v[ 8]+= v[13];				v[ 9]+= v[14];				v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 12);	v[ 7] = ROTR32(v[ 7] ^ v[ 8], 12);	v[ 4] = ROTR32(v[ 4] ^ v[ 9], 12);	v[ 5] = ROTR32(v[ 5] ^ v[10], 12);
		v[ 1]+= v[ 6];				v[ 2]+= v[ 7];				v[ 3]+= v[ 4];				v[ 0]+= v[ 5];
		v[12] = ROR8(v[12] ^ v[ 1]);		v[13] = ROR8(v[13] ^ v[ 2]);		v[14] = ROR8(v[14] ^ v[ 3]);		v[15] = ROR8(v[15] ^ v[ 0]);
		v[11]+= v[12]; 				v[ 8]+= v[13];				v[ 9]+= v[14];				v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 7);	v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);	v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);	v[ 5] = ROTR32(v[ 5] ^ v[10], 7);

		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 1],0,	v[ 1],v[ 6],v[11],v[12],m[ 0],m[ 2],	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],0,nonce);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,m[ 0],	v[ 2],v[ 6],v[10],v[14],0,m[ 2],	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,0,	v[ 1],v[ 6],v[11],v[12],nonce,0,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 1],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],nonce,m[ 1],	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 2],0,	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 0],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,m[ 0],	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],m[ 2],0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 1],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],nonce,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],m[ 2],0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],m[ 0],0,	v[ 3],v[ 7],v[11],v[15],0,nonce);
		GS4(v[ 0],v[ 5],v[10],v[15],0,0,	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],m[ 1],0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],m[ 1],0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 0],0,	v[ 1],v[ 6],v[11],v[12],0,nonce,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 2],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,m[ 1],	v[ 3],v[ 7],v[11],v[15],nonce,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 0],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],m[ 2],0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,nonce,	v[ 3],v[ 7],v[11],v[15],m[ 0],0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 2],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],m[ 1],0,	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,m[ 2],	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],m[ 1],0);

//		GS(9,4,v[ 0],v[ 5],v[10],v[15]);
		v[ 0] += v[ 5];
		v[ 2] += v[ 7] + nonce;
		v[15] = ROL16(v[15] ^ v[ 0]);
		v[13] = ROL16(v[13] ^ v[ 2]);
		v[10] += v[15];
		v[ 8] += v[13];
		v[ 5] = ROTR32(v[ 5] ^ v[10], 12);
		v[ 7] = ROTR32(v[ 7] ^ v[ 8], 12);
		v[ 0] += v[ 5];
		v[ 2] += v[ 7];
		v[15] = ROR8(v[15] ^ v[ 0]);
		v[13] = ROR8(v[13] ^ v[ 2]);

		v[ 8] += v[13];
		v[ 7] = ROTR32(v[ 7] ^ v[ 8], 7);

		if (xor3x(h7,v[7],v[15]) <= ptarget7){
			uint32_t pos = atomicInc(&resNonce[0],0xffffffff)+1;
			if(pos < maxResults)
				resNonce[pos] = nonce;
			return;
		}
	}
}

__global__ __launch_bounds__(TPB,1)
void blake2s_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce)
{
	const uint32_t step = gridDim.x * blockDim.x;

	uint32_t m[ 3];
	uint32_t v[16];

	m[0] = midstate[16];
	m[1] = midstate[17];
	m[2] = midstate[18];

	const uint32_t h7 = midstate[19];

	for(uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x ; thread <threads; thread+=step)
	{
		#pragma unroll
		for(int i=0;i<16;i++){
			v[ i] = midstate[ i];
		}

		uint32_t nonce = cuda_swab32(startNonce+thread);

//		Round( 0 );
		v[ 1] += nonce;
		v[13] = ROR8(v[13] ^ v[ 1]);
		v[ 9] += v[13];
		v[ 5] = ROTR32(v[ 5] ^ v[ 9], 7);

		v[ 1]+= v[ 6];
		v[ 0]+= v[ 5];

		v[13] = ROL16(v[13] ^ v[ 2]);		v[12] = ROL16(v[12] ^ v[ 1]);		v[15] = ROL16(v[15] ^ v[ 0]);

		v[ 8]+= v[13];				v[11]+= v[12];				v[ 9]+= v[14];				v[10]+= v[15];
		v[ 7] = ROTR32(v[ 7] ^ v[ 8], 12);	v[ 6] = ROTR32(v[ 6] ^ v[11], 12);	v[ 4] = ROTR32(v[ 4] ^ v[ 9], 12);	v[ 5] = ROTR32(v[ 5] ^ v[10], 12);
		v[ 2]+= v[ 7];				v[ 1]+= v[ 6];				v[ 3]+= v[ 4];				v[ 0]+= v[ 5];
		v[13] = ROR8(v[13] ^ v[ 2]);		v[12] = ROR8(v[12] ^ v[ 1]);		v[14] = ROR8(v[14] ^ v[ 3]);		v[15] = ROR8(v[15] ^ v[ 0]);
		v[ 8]+= v[13];				v[11]+= v[12];				v[ 9]+= v[14];				v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 7);	v[ 7] = ROTR32(v[ 7] ^ v[8], 7);	v[ 4] = ROTR32(v[ 4] ^ v[ 9], 7);	v[ 5] = ROTR32(v[ 5] ^ v[10], 7);

		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 1],0,	v[ 1],v[ 6],v[11],v[12],m[ 0],m[ 2],	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],0,nonce);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,m[ 0],	v[ 2],v[ 6],v[10],v[14],0,m[ 2],	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,0,	v[ 1],v[ 6],v[11],v[12],nonce,0,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 1],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],nonce,m[ 1],	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 2],0,	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 0],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,m[ 0],	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],m[ 2],0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 1],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],nonce,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],m[ 2],0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],m[ 0],0,	v[ 3],v[ 7],v[11],v[15],0,nonce);
		GS4(v[ 0],v[ 5],v[10],v[15],0,0,	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],m[ 1],0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],m[ 1],0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],0,0);
		GS4(v[ 0],v[ 5],v[10],v[15],m[ 0],0,	v[ 1],v[ 6],v[11],v[12],0,nonce,	v[ 2],v[ 7],v[ 8],v[13],0,m[ 2],	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,m[ 1],	v[ 3],v[ 7],v[11],v[15],nonce,0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 0],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],0,0,	v[ 3],v[ 4],v[ 9],v[14],m[ 2],0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,0,	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,nonce,	v[ 3],v[ 7],v[11],v[15],m[ 0],0);
		GS4(v[ 0],v[ 5],v[10],v[15],0,m[ 2],	v[ 1],v[ 6],v[11],v[12],0,0,	v[ 2],v[ 7],v[ 8],v[13],m[ 1],0,	v[ 3],v[ 4],v[ 9],v[14],0,0);
		GS4(v[ 0],v[ 4],v[ 8],v[12],0,m[ 2],	v[ 1],v[ 5],v[ 9],v[13],0,0,	v[ 2],v[ 6],v[10],v[14],0,0,	v[ 3],v[ 7],v[11],v[15],m[ 1],0);

		v[ 0] += v[ 5];
		v[ 2] += v[ 7] + nonce;
		v[15] = ROL16(v[15] ^ v[ 0]);
		v[13] = ROL16(v[13] ^ v[ 2]);
		v[10] += v[15];
		v[ 8] += v[13];
		v[ 5] = ROTR32(v[ 5] ^ v[10], 12);
		v[ 7] = ROTR32(v[ 7] ^ v[ 8], 12);
		v[ 0] += v[ 5];
		v[ 2] += v[ 7];
		v[15] = ROTR32(v[15] ^ v[ 0],1);
		v[13] = ROR8(v[13] ^ v[ 2]);

		v[ 8] += v[13];

		if(xor3x(v[ 7],h7,v[ 8])==v[15]){
			uint32_t pos = atomicInc(&resNonce[0],0xffffffff)+1;
			if(pos < maxResults)
				resNonce[pos]=nonce;
			return;
		}
	}
}

static void blake2s_setBlock(const uint32_t* input,const uint32_t ptarget7)
{
	uint32_t _ALIGN(64) m[16];
	uint32_t _ALIGN(64) v[16];
	uint32_t _ALIGN(64) h[21];

//	COMPRESS
	for(int i = 0; i < 16; ++i )
		m[i] = input[i];

	h[0] = 0x01010020 ^ blake2s_IV[0];
	h[1] = blake2s_IV[1];
	h[2] = blake2s_IV[2]; h[3] = blake2s_IV[3];
	h[4] = blake2s_IV[4]; h[5] = blake2s_IV[5];
	h[6] = blake2s_IV[6]; h[7] = blake2s_IV[7];

	for(int i = 0; i < 8; ++i )
		v[i] = h[i];

	v[ 8] = blake2s_IV[0];		v[ 9] = blake2s_IV[1];
	v[10] = blake2s_IV[2];		v[11] = blake2s_IV[3];
	v[12] = 64 ^ blake2s_IV[4];	v[13] = blake2s_IV[5];
	v[14] = blake2s_IV[6];		v[15] = blake2s_IV[7];

	ROUND( 0 ); ROUND( 1 );
	ROUND( 2 ); ROUND( 3 );
	ROUND( 4 ); ROUND( 5 );
	ROUND( 6 ); ROUND( 7 );
	ROUND( 8 ); ROUND( 9 );

	for(int i = 0; i < 8; ++i )
		h[i] ^= v[i] ^ v[i + 8];

	h[16] = input[16];
	h[17] = input[17];
	h[18] = input[18];

	h[ 8] = 0x6A09E667; h[ 9] = 0xBB67AE85;
	h[10] = 0x3C6EF372; h[11] = 0xA54FF53A;
	h[12] = 0x510E522F; h[13] = 0x9B05688C;
	h[14] =~0x1F83D9AB; h[15] = 0x5BE0CD19;

	h[ 0]+= h[ 4] + h[16];
	h[12] = SPH_ROTR32(h[12] ^ h[ 0],16);
	h[ 8]+= h[12];
	h[ 4] = SPH_ROTR32(h[ 4] ^ h[ 8],12);
	h[ 0]+= h[ 4] + h[17];
	h[12] = SPH_ROTR32(h[12] ^ h[ 0],8);
	h[ 8]+= h[12];
	h[ 4] = SPH_ROTR32(h[ 4] ^ h[ 8],7);

	h[ 1]+= h[ 5] + h[18];
	h[13] = SPH_ROTR32(h[13] ^ h[ 1], 16);
	h[ 9]+= h[13];
	h[ 5] = ROTR32(h[ 5] ^ h[ 9], 12);

	h[ 2]+= h[ 6];
	h[14] = SPH_ROTR32(h[14] ^ h[ 2],16);
	h[10]+= h[14];
	h[ 6] = SPH_ROTR32(h[ 6] ^ h[10], 12);
	h[ 2]+= h[ 6];
	h[14] = SPH_ROTR32(h[14] ^ h[ 2],8);
	h[10]+= h[14];
	h[ 6] = SPH_ROTR32(h[ 6] ^ h[10], 7);

	h[19] = h[7]; //constant h[7] for nonce check

	h[ 3]+= h[ 7];
	h[15] = SPH_ROTR32(h[15] ^ h[ 3],16);
	h[11]+= h[15];
	h[ 7] = SPH_ROTR32(h[ 7] ^ h[11], 12);
	h[ 3]+= h[ 7];
	h[15] = SPH_ROTR32(h[15] ^ h[ 3],8);
	h[11]+= h[15];
	h[ 7] = SPH_ROTR32(h[ 7] ^ h[11], 7);

	h[ 1]+= h[ 5];
	h[ 3]+= h[ 4];
	h[14] = SPH_ROTR32(h[14] ^ h[ 3],16);

	h[ 2]+= h[ 7];
	if(ptarget7==0){
		h[19] = SPH_ROTL32(h[19],7); //align the rotation with v[7] v[15];
	}
	hipMemcpyToSymbol(HIP_SYMBOL(midstate), h, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake2s(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *resNonces;

	const uint32_t first_nonce = pdata[19];

	const int dev_id = device_map[thr_id];
	int rc = 0;
	int intensity = is_windows() ? 25 : 28;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	const dim3 grid((throughput + (NPT*TPB)-1)/(NPT*TPB));
	const dim3 block(TPB);

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], maxResults * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], maxResults * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}
	resNonces = h_resNonce[thr_id];

	for (int i=0; i < 19; i++) {
		be32enc(&endiandata[i], pdata[i]);
	}
	blake2s_setBlock(endiandata,ptarget[7]);

	hipMemset(d_resNonce[thr_id], 0x00, maxResults*sizeof(uint32_t));

	do {
		if(ptarget[7]) {
			blake2s_gpu_hash_nonce<<<grid, block>>>(throughput,pdata[19],d_resNonce[thr_id],ptarget[7]);
		} else {
			blake2s_gpu_hash_nonce<<<grid, block>>>(throughput,pdata[19],d_resNonce[thr_id]);
		}
		hipMemcpy(resNonces, d_resNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

		if(resNonces[0])
		{
			hipMemcpy(resNonces, d_resNonce[thr_id], maxResults*sizeof(uint32_t), hipMemcpyDeviceToHost);
			hipMemset(d_resNonce[thr_id], 0x00, sizeof(uint32_t));

			if(resNonces[0] >= maxResults) {
				gpulog(LOG_WARNING, thr_id, "candidates flood: %u", resNonces[0]);
				resNonces[0] = maxResults-1;
			}

			uint32_t vhashcpu[8];
			uint32_t nonce = sph_bswap32(resNonces[1]);
			be32enc(&endiandata[19], nonce);
			blake2s_hash(vhashcpu, endiandata);

			*hashes_done = pdata[19] - first_nonce + throughput;

			if(vhashcpu[6] <= ptarget[6] && fulltest(vhashcpu, ptarget))
			{
				work_set_target_ratio(work, vhashcpu);
				work->nonces[0] = nonce;
				rc = work->valid_nonces = 1;

				// search for 2nd best nonce
				for(uint32_t j=2; j <= resNonces[0]; j++)
				{
					nonce = sph_bswap32(resNonces[j]);
					be32enc(&endiandata[19], nonce);
					blake2s_hash(vhashcpu, endiandata);
					if(vhashcpu[6] <= ptarget[6] && fulltest(vhashcpu, ptarget))
					{
						gpulog(LOG_DEBUG, thr_id, "Multiple nonces: 1/%08x - %u/%08x", work->nonces[0], j, nonce);

						work->nonces[1] = nonce;
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio[0]) {
							work->shareratio[1] = work->shareratio[0];
							work->sharediff[1] = work->sharediff[0];
							xchg(work->nonces[1], work->nonces[0]);
							work_set_target_ratio(work, vhashcpu);
						} else if (work->valid_nonces == 1) {
							bn_set_target_ratio(work, vhashcpu, 1);
						}

						work->valid_nonces++;
						rc = 2;
						break;
					}
				}
				pdata[19] = max(work->nonces[0], work->nonces[1]); // next scan start
				return rc;
			} else if (vhashcpu[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", resNonces[0]);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && (uint64_t)max_nonce > (uint64_t)throughput + pdata[19]);

	*hashes_done = pdata[19] - first_nonce;

	return rc;
}

// cleanup
extern "C" void free_blake2s(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

