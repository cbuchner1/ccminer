#include "hip/hip_runtime.h"
/**
 * Blake2-S 256 CUDA implementation
 * @author tpruvot@github March 2016
 */
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <memory.h>

#include "miner.h"

extern "C" {
#define NATIVE_LITTLE_ENDIAN
#include <sph/blake2s.h>
}

//#define GPU_MIDSTATE
#define MIDLEN 76
#define A 64

static __thread blake2s_state ALIGN(A) s_midstate;
static __thread blake2s_state ALIGN(A) s_ctx;

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

#ifndef GPU_MIDSTATE
__constant__ uint2 d_data[10];
#else
__constant__ blake2s_state ALIGN(8) d_state[1];
#endif

/* 16 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* threads per block */
#define TPB 512

/* max count of found nonces in one call */
#define NBN 2
#if NBN > 1
static uint32_t extra_results[NBN] = { UINT32_MAX };
#endif

extern "C" void blake2s_hash(void *output, const void *input)
{
	uint8_t _ALIGN(A) hash[BLAKE2S_OUTBYTES];
	blake2s_state blake2_ctx;

	blake2s_init(&blake2_ctx, BLAKE2S_OUTBYTES);
	blake2s_update(&blake2_ctx, (uint8_t*) input, 80);
	blake2s_final(&blake2_ctx, hash, BLAKE2S_OUTBYTES);

	memcpy(output, hash, 32);
}

__host__
inline void blake2s_hash_end(uint32_t *output, const uint32_t *input)
{
	s_ctx.buflen = MIDLEN;
	memcpy(&s_ctx, &s_midstate, 32 + 16 + MIDLEN);
	blake2s_update(&s_ctx, (uint8_t*) &input[MIDLEN/4], 80-MIDLEN);
	blake2s_final(&s_ctx, (uint8_t*) output, BLAKE2S_OUTBYTES);
}

__host__
void blake2s_setBlock(uint32_t *penddata, blake2s_state *pstate)
{
#ifndef GPU_MIDSTATE
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), penddata, 80, 0, hipMemcpyHostToDevice));
#else
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_state), pstate, sizeof(blake2s_state), 0, hipMemcpyHostToDevice));
#endif
}

__device__ __forceinline__
uint64_t gpu_load64(void *src) {
	return *(uint64_t*)(src);
}

__device__ __forceinline__
void gpu_store32(void *dst, uint32_t dw) {
	*(uint32_t*)(dst) = dw;
}

__device__ __forceinline__
void gpu_store64(void *dst, uint64_t lw) {
	*(uint64_t*)(dst) = lw;
}

__device__ __forceinline__
void gpu_blake2s_set_lastnode(blake2s_state *S) {
	S->f[1] = ~0U;
}

__device__ __forceinline__
void gpu_blake2s_clear_lastnode(blake2s_state *S) {
	S->f[1] = 0U;
}

__device__ __forceinline__
void gpu_blake2s_increment_counter(blake2s_state *S, const uint32_t inc)
{
	S->t[0] += inc;
	S->t[1] += ( S->t[0] < inc );
}

__device__ __forceinline__
void gpu_blake2s_set_lastblock(blake2s_state *S)
{
	if (S->last_node) gpu_blake2s_set_lastnode(S);
	S->f[0] = ~0U;
}

__device__
void gpu_blake2s_compress(blake2s_state *S, const uint32_t *block)
{
	uint32_t m[16];
	uint32_t v[16];

	const uint32_t blake2s_IV[8] = {
		0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
		0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL
	};

	const uint8_t blake2s_sigma[10][16] = {
		{  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
		{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },
		{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
		{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 },
		{  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 },
		{  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 },
		{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 },
		{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 },
		{  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 },
		{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 },
	};

	#pragma unroll
	for(int i = 0; i < 16; i++)
		m[i] = block[i];

	#pragma unroll
	for(int i = 0; i < 8; i++)
		v[i] = S->h[i];

	v[ 8] = blake2s_IV[0];
	v[ 9] = blake2s_IV[1];
	v[10] = blake2s_IV[2];
	v[11] = blake2s_IV[3];
	v[12] = S->t[0] ^ blake2s_IV[4];
	v[13] = S->t[1] ^ blake2s_IV[5];
	v[14] = S->f[0] ^ blake2s_IV[6];
	v[15] = S->f[1] ^ blake2s_IV[7];

	#define G(r,i,a,b,c,d) { \
		a += b + m[blake2s_sigma[r][2*i+0]]; \
		d = __byte_perm(d ^ a, 0, 0x1032); /* d = ROTR32(d ^ a, 16); */ \
		c = c + d; \
		b = ROTR32(b ^ c, 12); \
		a += b + m[blake2s_sigma[r][2*i+1]]; \
		d = __byte_perm(d ^ a, 0, 0x0321); /* ROTR32(d ^ a, 8); */ \
		c = c + d; \
		b = ROTR32(b ^ c, 7); \
	}

	#define ROUND(r) { \
		G(r,0,v[ 0],v[ 4],v[ 8],v[12]); \
		G(r,1,v[ 1],v[ 5],v[ 9],v[13]); \
		G(r,2,v[ 2],v[ 6],v[10],v[14]); \
		G(r,3,v[ 3],v[ 7],v[11],v[15]); \
		G(r,4,v[ 0],v[ 5],v[10],v[15]); \
		G(r,5,v[ 1],v[ 6],v[11],v[12]); \
		G(r,6,v[ 2],v[ 7],v[ 8],v[13]); \
		G(r,7,v[ 3],v[ 4],v[ 9],v[14]); \
	}

	ROUND( 0 );
	ROUND( 1 );
	ROUND( 2 );
	ROUND( 3 );
	ROUND( 4 );
	ROUND( 5 );
	ROUND( 6 );
	ROUND( 7 );
	ROUND( 8 );
	ROUND( 9 );

	#pragma unroll
	for(int i = 0; i < 8; i++)
		S->h[i] = S->h[i] ^ v[i] ^ v[i + 8];

	#undef G
	#undef ROUND
}

#if 0
/* unused but kept as reference */
__device__ __forceinline__
void gpu_blake2s_update(blake2s_state *S, const uint8_t *in, uint64_t inlen)
{
	while(inlen > 0)
	{
		const int left = S->buflen;
		size_t fill = 2 * BLAKE2S_BLOCKBYTES - left;
		if(inlen > fill)
		{
			memcpy(S->buf + left, in, fill); // Fill buffer
			S->buflen += fill;

			gpu_blake2s_increment_counter(S, BLAKE2S_BLOCKBYTES);
			gpu_blake2s_compress(S, (uint32_t*) S->buf); // Compress
			memcpy(S->buf, S->buf + BLAKE2S_BLOCKBYTES, BLAKE2S_BLOCKBYTES); // Shift buffer left
			S->buflen -= BLAKE2S_BLOCKBYTES;
			in += fill;
			inlen -= fill;
		}
		else // inlen <= fill
		{
			memcpy(S->buf + left, in, (size_t) inlen);
			S->buflen += (size_t) inlen; // Be lazy, do not compress
			in += inlen;
			inlen -= inlen;
		}
	}
}
#endif

#ifndef GPU_MIDSTATE
__device__ __forceinline__
void gpu_blake2s_fill_data(blake2s_state *S, const uint32_t nonce)
{
	uint2 *b2 = (uint2*) S->buf;
	#pragma unroll
	for (int i=0; i < 9; i++)
		b2[i] = d_data[i];
	b2[9].x = d_data[9].x;
	b2[9].y = nonce;
	S->buflen = 80;
}
#endif

__device__ __forceinline__
void gpu_blake2s_update_nonce(blake2s_state *S, const uint32_t nonce)
{
	gpu_store32(&S->buf[76], nonce);
	S->buflen = 80;
}

__device__ __forceinline__
uint2 gpu_blake2s_final(blake2s_state *S)
{
	//if (S->buflen > BLAKE2S_BLOCKBYTES)
	{
		gpu_blake2s_increment_counter(S, BLAKE2S_BLOCKBYTES);
		gpu_blake2s_compress(S, (uint32_t*) S->buf);
		S->buflen -= BLAKE2S_BLOCKBYTES;
		//memcpy(S->buf, S->buf + BLAKE2S_BLOCKBYTES, S->buflen);
	}

	gpu_blake2s_increment_counter(S, (uint32_t)S->buflen);
	gpu_blake2s_set_lastblock(S);
	//memset(&S->buf[S->buflen], 0, 2 * BLAKE2S_BLOCKBYTES - S->buflen); /* Padding */
	gpu_blake2s_compress(S, (uint32_t*) (S->buf + BLAKE2S_BLOCKBYTES));

	//#pragma unroll
	//for (int i = 0; i < 8; i++)
	//	out[i] = S->h[i];
	return make_uint2(S->h[6], S->h[7]);
}

/* init2 xors IV with input parameter block */
__device__ __forceinline__
void gpu_blake2s_init_param(blake2s_state *S, const blake2s_param *P)
{
	//blake2s_IV
	S->h[0] = 0x6A09E667UL;
	S->h[1] = 0xBB67AE85UL;
	S->h[2] = 0x3C6EF372UL;
	S->h[3] = 0xA54FF53AUL;
	S->h[4] = 0x510E527FUL;
	S->h[5] = 0x9B05688CUL;
	S->h[6] = 0x1F83D9ABUL;
	S->h[7] = 0x5BE0CD19UL;

	S->t[0] = 0; S->t[1] = 0;
	S->f[0] = 0; S->f[1] = 0;
	S->last_node = 0;

	S->buflen = 0;

	#pragma unroll
	for (int i = 8; i < sizeof(S->buf)/8; i++)
		gpu_store64(S->buf + (8*i), 0);

	uint64_t *p = (uint64_t*) P;

	/* IV XOR ParamBlock */
	#pragma unroll
	for (int i = 0; i < 4; i++)
		S->h[i] ^= gpu_load64(&p[i]);
}

// Sequential blake2s initialization
__device__ __forceinline__
void gpu_blake2s_init(blake2s_state *S, const uint8_t outlen)
{
	blake2s_param P[1];

	// if (!outlen || outlen > BLAKE2S_OUTBYTES) return;

	P->digest_length = outlen;
	P->key_length    = 0;
	P->fanout        = 1;
	P->depth         = 1;

	P->leaf_length = 0;
	gpu_store64(P->node_offset, 0);
	//P->node_depth    = 0;
	//P->inner_length  = 0;

	gpu_store64(&P->salt, 0);
	gpu_store64(&P->personal, 0);

	gpu_blake2s_init_param(S, P);
}

__device__ __forceinline__
void gpu_copystate(blake2s_state *dst, blake2s_state *src)
{
	uint64_t* d64 = (uint64_t*) dst;
	uint64_t* s64 = (uint64_t*) src;
	#pragma unroll
	for (int i=0; i < (32 + 16 + 2 * BLAKE2S_BLOCKBYTES)/8; i++)
		gpu_store64(&d64[i], s64[i]);
	dst->buflen = src->buflen;
	dst->last_node = src->last_node;
}

__global__
void blake2s_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint2 target2, const int swap)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t nonce = swap ? cuda_swab32(startNonce + thread) : startNonce + thread;
	blake2s_state ALIGN(8) blake2_ctx;

#ifndef GPU_MIDSTATE
	gpu_blake2s_init(&blake2_ctx, BLAKE2S_OUTBYTES);
	//gpu_blake2s_update(&blake2_ctx, (uint8_t*) d_data, 76);
	gpu_blake2s_fill_data(&blake2_ctx, nonce);
#else
	gpu_copystate(&blake2_ctx, &d_state[0]);
	gpu_blake2s_update_nonce(&blake2_ctx, nonce);
#endif

	uint2 h2 = gpu_blake2s_final(&blake2_ctx);
	if (h2.y <= target2.y && h2.x <= target2.x) {
#if NBN == 2
		if (resNonce[0] != UINT32_MAX)
			resNonce[1] = nonce;
		else
			resNonce[0] = nonce;
#else
		resNonce[0] = nonce;
#endif
	}
}

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

__host__
uint32_t blake2s_hash_cuda(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint2 target2, const int swap)
{
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake2s_gpu_hash <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id], target2, swap);
	hipDeviceSynchronize();

	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = swab32_if(h_resNonce[thr_id][0], swap);
#if NBN > 1
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = swab32_if(h_resNonce[thr_id][n+1], swap);
#endif
	}
	return result;
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake2s(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const int swap = 1; // to toggle nonce endian

	const uint32_t first_nonce = pdata[19];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 28 : 25;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) {
		ptarget[6] = swab32(0xFFFF0);
		ptarget[7] = 0;
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	for (int i=0; i < 19; i++) {
		be32enc(&endiandata[i], pdata[i]);
	}

	// midstate
	memset(s_midstate.buf, 0, sizeof(s_midstate.buf));
	blake2s_init(&s_midstate, BLAKE2S_OUTBYTES);
	blake2s_update(&s_midstate, (uint8_t*) endiandata, MIDLEN);
	memcpy(&s_ctx, &s_midstate, sizeof(blake2s_state));

	blake2s_setBlock(endiandata, &s_midstate);

	const uint2 target = make_uint2(ptarget[6], ptarget[7]);

	do {
		uint32_t foundNonce = blake2s_hash_cuda(thr_id, throughput, pdata[19], target, swap);

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(A) vhashcpu[8];

			//blake2s_hash(vhashcpu, endiandata);
			endiandata[19] = swab32_if(foundNonce, swap);
			blake2s_hash_end(vhashcpu, endiandata);

			if (vhashcpu[7] <= target.y && fulltest(vhashcpu, ptarget)) {
				work_set_target_ratio(work, vhashcpu);
				pdata[19] = work->nonces[0] = swab32_if(foundNonce, !swap);
#if NBN > 1
				if (extra_results[0] != UINT32_MAX) {
					endiandata[19] = swab32_if(extra_results[0], swap);
					blake2s_hash_end(vhashcpu, endiandata);
					if (vhashcpu[7] <= target.y && fulltest(vhashcpu, ptarget)) {
						work->nonces[1] = swab32_if(extra_results[0], !swap);
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio) {
							work_set_target_ratio(work, vhashcpu);
							xchg(work->nonces[1], pdata[19]);
						}
						return 2;
					}
					extra_results[0] = UINT32_MAX;
				}
#endif
				return 1;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + pdata[19]);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_blake2s(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
