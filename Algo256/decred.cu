#include "hip/hip_runtime.h"
/**
 * Blake-256 Decred 180-Bytes input Cuda Kernel
 *
 * Tanguy Pruvot, Alexis Provos - Feb/Sep 2016
 */

#include <stdint.h>
#include <memory.h>
#include <miner.h>

extern "C" {
#include <sph/sph_blake.h>
}

/* threads per block */
#define TPB 640

/* max count of found nonces in one call (like sgminer) */
#define MAX_RESULTS 4

/* hash by cpu with blake 256 */
extern "C" void decred_hash(void *output, const void *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}

#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#define atomicInc(p, max) (*p)++
#endif

__constant__ uint32_t _ALIGN(16) c_h[2];
__constant__ uint32_t _ALIGN(16) c_data[32];
__constant__ uint32_t _ALIGN(16) c_xors[215];

/* Buffers of candidate nonce(s) */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

#define ROR8(a)  __byte_perm(a, 0, 0x0321)
#define ROL16(a) __byte_perm(a, 0, 0x1032)

/* macro bodies */
#define pxorGS(a,b,c,d) { \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = ROL16(v[d] ^ v[a]); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a]+= c_xors[i++] + v[b]; \
	v[d] = ROR8(v[d] ^ v[a]); \
	v[c]+= v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}

#define pxorGS2(a,b,c,d, a1,b1,c1,d1) {\
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxory1GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxory0GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= (c_xors[i++]^nonce) + v[ b];    v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxorx1GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= (c_xors[i++]^nonce) + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define pxorx0GS2(a,b,c,d, a1,b1,c1,d1) { \
	v[ a]+= (c_xors[i++]^nonce) + v[ b];    v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROL16(v[ d] ^ v[ a]);           v[d1] = ROL16(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 12);      v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[ a]+= c_xors[i++] + v[ b];            v[a1]+= c_xors[i++] + v[b1]; \
	v[ d] = ROR8(v[ d] ^ v[ a]);            v[d1] = ROR8(v[d1] ^ v[a1]); \
	v[ c]+= v[ d];                          v[c1]+= v[d1]; \
	v[ b] = ROTR32(v[ b] ^ v[ c], 7);       v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

__global__ __launch_bounds__(TPB,1)
void decred_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint32_t highTarget)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		uint32_t v[16];
		#pragma unroll
		for(int i=0; i<16; i+=4) {
			*(uint4*)&v[i] = *(uint4*)&c_data[i];
		}

		const uint32_t nonce = startNonce + thread;
		v[ 1]+= (nonce ^ 0x13198A2E);
		v[13] = ROR8(v[13] ^ v[1]);
		v[ 9]+= v[13];
		v[ 5] = ROTR32(v[5] ^ v[9], 7);

		int i = 0;
		v[ 1]+= c_xors[i++];// + v[ 6];
		v[ 0]+= v[5];
		v[12] = ROL16(v[12] ^ v[ 1]);         v[15] = ROL16(v[15] ^ v[ 0]);
		v[11]+= v[12];                        v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 12);    v[ 5] = ROTR32(v[5] ^ v[10], 12);
		v[ 1]+= c_xors[i++] + v[ 6];          v[ 0]+= c_xors[i++] + v[ 5];
		v[12] = ROR8(v[12] ^ v[ 1]);          v[15] = ROR8(v[15] ^ v[ 0]);
		v[11]+= v[12];                        v[10]+= v[15];
		v[ 6] = ROTR32(v[ 6] ^ v[11], 7);     v[ 5] = ROTR32(v[ 5] ^ v[10], 7);

		pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxory1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorx1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorx1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorx1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxory1GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxory1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorx1GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxory0GS2( 2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorx0GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxory1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxory1GS2( 2, 7, 8, 13, 3, 4, 9, 14);
		pxorGS2(   0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorx1GS2( 0, 5, 10, 15, 1, 6, 11, 12); pxorGS2(   2, 7, 8, 13, 3, 4, 9, 14);
		pxorx1GS2( 0, 4, 8, 12, 1, 5, 9, 13); pxorGS2(   2, 6, 10, 14, 3, 7, 11, 15); pxorGS2(   0, 5, 10, 15, 1, 6, 11, 12); pxorGS(    2, 7, 8, 13);

		if ((c_h[1]^v[15]) == v[7]) {
			v[ 3] += c_xors[i++] + v[4];
			v[14] = ROL16(v[14] ^ v[3]);
			v[ 9] += v[14];
			v[ 4] = ROTR32(v[4] ^ v[9], 12);
			v[ 3] += c_xors[i++] + v[4];
			v[14] = ROR8(v[14] ^ v[3]);
			if(cuda_swab32((c_h[0]^v[6]^v[14])) <= highTarget) {
				uint32_t pos = atomicInc(&resNonce[0], UINT32_MAX)+1;
				resNonce[pos] = nonce;
				return;
			}
		}
	}
}

__host__
void decred_cpu_setBlock_52(const uint32_t *input)
{
/*
	Precompute everything possible and pass it on constant memory
*/
	const uint32_t z[16] = {
		0x243F6A88U, 0x85A308D3U, 0x13198A2EU, 0x03707344U,
		0xA4093822U, 0x299F31D0U, 0x082EFA98U, 0xEC4E6C89U,
		0x452821E6U, 0x38D01377U, 0xBE5466CFU, 0x34E90C6CU,
		0xC0AC29B7U, 0xC97C50DDU, 0x3F84D5B5U, 0xB5470917U
	};

	int i=0;
	uint32_t _ALIGN(64) preXOR[215];
	uint32_t _ALIGN(64)   data[16];
	uint32_t _ALIGN(64)      m[16];
	uint32_t _ALIGN(64)      h[ 2];

	sph_blake256_context ctx;
	sph_blake256_set_rounds(14);
	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 128);

	data[ 0] = ctx.H[0];
	data[ 1] = ctx.H[1];
	data[ 2] = ctx.H[2];
	data[ 3] = ctx.H[3];
	data[ 4] = ctx.H[4];
	data[ 5] = ctx.H[5];
	data[ 8] = ctx.H[6];

	data[12] = swab32(input[35]);
	data[13] = ctx.H[7];

	// pre swab32
	m[ 0] = swab32(input[32]);	m[ 1] = swab32(input[33]);
	m[ 2] = swab32(input[34]);	m[ 3] = 0;
	m[ 4] = swab32(input[36]);	m[ 5] = swab32(input[37]);
	m[ 6] = swab32(input[38]);	m[ 7] = swab32(input[39]);
	m[ 8] = swab32(input[40]);	m[ 9] = swab32(input[41]);
	m[10] = swab32(input[42]);	m[11] = swab32(input[43]);
	m[12] = swab32(input[44]);	m[13] = 0x80000001;
	m[14] = 0;
	m[15] = 0x000005a0;

	h[ 0] = data[ 8];
	h[ 1] = data[13];

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_h),h, 8, 0, hipMemcpyHostToDevice));

	data[ 0]+= (m[ 0] ^ z[1]) + data[ 4];
	data[12]  = SPH_ROTR32(z[4] ^ SPH_C32(0x5A0) ^ data[ 0], 16);

	data[ 8] = z[0]+data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 12);
	data[ 0]+= (m[ 1] ^ z[0]) + data[ 4];
	data[12] = SPH_ROTR32(data[12] ^ data[ 0],8);
	data[ 8]+= data[12];
	data[ 4] = SPH_ROTR32(data[ 4] ^ data[ 8], 7);

	data[ 1]+= (m[ 2] ^ z[3]) + data[ 5];
	data[13] = SPH_ROTR32((z[5] ^ SPH_C32(0x5A0)) ^ data[ 1], 16);
	data[ 9] = z[1]+data[13];
	data[ 5] = SPH_ROTR32(data[ 5] ^ data[ 9], 12);
	data[ 1]+= data[ 5]; //+nonce ^ ...

	data[ 2]+= (m[ 4] ^ z[5]) + h[ 0];
	data[14] = SPH_ROTR32(z[6] ^ data[ 2],16);
	data[10] = z[2] + data[14];
	data[ 6] = SPH_ROTR32(h[ 0] ^ data[10], 12);
	data[ 2]+= (m[ 5] ^ z[4]) + data[ 6];
	data[14] = SPH_ROTR32(data[14] ^ data[ 2], 8);
	data[10]+= data[14];
	data[ 6] = SPH_ROTR32(data[ 6] ^ data[10], 7);

	data[ 3]+= (m[ 6] ^ z[7]) + h[ 1];
	data[15] = SPH_ROTR32(z[7] ^ data[ 3],16);
	data[11] = z[3] + data[15];
	data[ 7] = SPH_ROTR32(h[ 1] ^ data[11], 12);
	data[ 3]+= (m[ 7] ^ z[6]) + data[ 7];
	data[15] = SPH_ROTR32(data[15] ^ data[ 3],8);
	data[11]+= data[15];
	data[ 7] = SPH_ROTR32(data[11] ^ data[ 7], 7);
	data[ 0]+= m[ 8] ^ z[9];

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, 64, 0, hipMemcpyHostToDevice));

#define precalcXORGS(x,y) { \
	preXOR[i++]= (m[x] ^ z[y]); \
	preXOR[i++]= (m[y] ^ z[x]); \
}
#define precalcXORGS2(x,y,x1,y1){\
	preXOR[i++] = (m[ x] ^ z[ y]);\
	preXOR[i++] = (m[x1] ^ z[y1]);\
	preXOR[i++] = (m[ y] ^ z[ x]);\
	preXOR[i++] = (m[y1] ^ z[x1]);\
}
	precalcXORGS(10,11);
	preXOR[ 0]+=data[ 6];
	preXOR[i++] = (m[9] ^ z[8]);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);
	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);
	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);
	precalcXORGS2( 4, 0,15, 8);
	precalcXORGS2( 9, 0, 5, 7);
	precalcXORGS2( 2, 4,10,15);
	precalcXORGS2(14, 1,11,12);
	precalcXORGS2( 6, 8, 3,13);
	precalcXORGS2( 2,12, 6,10);
	precalcXORGS2( 0,11, 8, 3);
	precalcXORGS2( 4,13, 7, 5);
	precalcXORGS2(15,14, 1, 9);
	precalcXORGS2(12, 5, 1,15);
	precalcXORGS2(14,13, 4,10);
	precalcXORGS2( 0, 7, 6, 3);
	precalcXORGS2( 9, 2, 8,11);
	precalcXORGS2(13,11, 7,14);
	precalcXORGS2(12, 1, 3, 9);
	precalcXORGS2( 5, 0,15, 4);
	precalcXORGS2( 8, 6, 2,10);
	precalcXORGS2( 6,15,14, 9);
	precalcXORGS2(11, 3, 0, 8);
	precalcXORGS2(12, 2,13, 7);
	precalcXORGS2( 1, 4,10, 5);
	precalcXORGS2(10, 2, 8, 4);
	precalcXORGS2( 7, 6, 1, 5);
	precalcXORGS2(15,11, 9,14);
	precalcXORGS2( 3,12,13, 0);
	precalcXORGS2( 0, 1, 2, 3);
	precalcXORGS2( 4, 5, 6, 7);
	precalcXORGS2( 8, 9,10,11);
	precalcXORGS2(12,13,14,15);
	precalcXORGS2(14,10, 4, 8);
	precalcXORGS2( 9,15,13, 6);
	precalcXORGS2( 1,12, 0, 2);
	precalcXORGS2(11, 7, 5, 3);
	precalcXORGS2(11, 8,12, 0);
	precalcXORGS2( 5, 2,15,13);
	precalcXORGS2(10,14, 3, 6);
	precalcXORGS2( 7, 1, 9, 4);
	precalcXORGS2( 7, 9, 3, 1);
	precalcXORGS2(13,12,11,14);
	precalcXORGS2( 2, 6, 5,10);
	precalcXORGS( 4, 0);
	precalcXORGS(15, 8);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_xors), preXOR, 215*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

/* ############################################################################################################################### */

static bool init[MAX_GPUS] = { 0 };

// nonce position is different in decred
#define DCR_NONCE_OFT32 35

extern "C" int scanhash_decred(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[48];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *pnonce = &pdata[DCR_NONCE_OFT32];

	const uint32_t first_nonce = *pnonce;
	const uint32_t targetHigh = opt_benchmark ? 0x1ULL : ptarget[6];

	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 29 : 25;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	const dim3 grid((throughput + TPB-1)/(TPB));
	const dim3 block(TPB);

	if (!init[thr_id]) {
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], MAX_RESULTS*sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], MAX_RESULTS*sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}
	memcpy(endiandata, pdata, 180);

	decred_cpu_setBlock_52(endiandata);
	hipMemset(d_resNonce[thr_id], 0x00, sizeof(uint32_t));

	do {
		uint32_t* resNonces = h_resNonce[thr_id];

		if (resNonces[0]) hipMemset(d_resNonce[thr_id], 0x00, sizeof(uint32_t));

		// GPU HASH
		decred_gpu_hash_nonce <<<grid, block>>> (throughput, (*pnonce), d_resNonce[thr_id], targetHigh);

		*hashes_done = (*pnonce) - first_nonce + throughput;

		// first cell contains the valid nonces count
		hipMemcpy(resNonces, d_resNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (resNonces[0])
		{
			uint32_t _ALIGN(64) vhash[8];

			hipMemcpy(resNonces, d_resNonce[thr_id], (resNonces[0]+1)*sizeof(uint32_t), hipMemcpyDeviceToHost);

			be32enc(&endiandata[DCR_NONCE_OFT32], resNonces[1]);
			decred_hash(vhash, endiandata);
			if (vhash[6] <= ptarget[6] && fulltest(vhash, ptarget))
			{
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[0] = swab32(resNonces[1]);
				*pnonce = work->nonces[0];

				// search for another nonce
				for(uint32_t n=2; n <= resNonces[0]; n++)
				{
					be32enc(&endiandata[DCR_NONCE_OFT32], resNonces[n]);
					decred_hash(vhash, endiandata);
					if (vhash[6] <= ptarget[6] && fulltest(vhash, ptarget)) {
						work->nonces[1] = swab32(resNonces[n]);
						if (bn_hash_target_ratio(vhash, ptarget) > work->shareratio[0]) {
							// we really want the best first ? depends...
							work->shareratio[1] = work->shareratio[0];
							work->sharediff[1] = work->sharediff[0];
							xchg(work->nonces[1], work->nonces[0]);
							work_set_target_ratio(work, vhash);
							work->valid_nonces++;
						} else if (work->valid_nonces == 1) {
							bn_set_target_ratio(work, vhash, 1);
							work->valid_nonces++;
						}
						work->valid_nonces = 2; // MAX_NONCES submit limited to 2

						gpulog(LOG_DEBUG, thr_id, "multiple nonces 1:%08x (%g) %u:%08x (%g)",
							work->nonces[0], work->sharediff[0], n, work->nonces[1], work->sharediff[1]);

					} else if (vhash[6] > ptarget[6]) {
						gpu_increment_reject(thr_id);
						if (!opt_quiet)
						gpulog(LOG_WARNING, thr_id, "result %u for %08x does not validate on CPU!", n, resNonces[n]);
					}
				}
				return work->valid_nonces;

			} else if (vhash[6] > ptarget[6]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", resNonces[1]);
			}
		}
		*pnonce += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + (*pnonce));

	*hashes_done = (*pnonce) - first_nonce;
	MyStreamSynchronize(NULL, 0, device_map[thr_id]);
	return 0;
}

// cleanup
extern "C" void free_decred(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();
	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
