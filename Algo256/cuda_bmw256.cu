#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#undef SPH_ROTL32
#define SPH_ROTL32 ROTL32

static uint32_t *d_gnounce[MAX_GPUS];
static uint32_t *d_GNonce[MAX_GPUS];

__constant__ uint64_t pTarget[4];

#define shl(x, n) ((x) << (n))
#define shr(x, n) ((x) >> (n))

#define ss0(x) (shr((x), 1) ^ shl((x), 3) ^ SPH_ROTL32((x),  4) ^ SPH_ROTL32((x), 19))
#define ss1(x) (shr((x), 1) ^ shl((x), 2) ^ SPH_ROTL32((x),  8) ^ SPH_ROTL32((x), 23))
#define ss2(x) (shr((x), 2) ^ shl((x), 1) ^ SPH_ROTL32((x), 12) ^ SPH_ROTL32((x), 25))
#define ss3(x) (shr((x), 2) ^ shl((x), 2) ^ SPH_ROTL32((x), 15) ^ SPH_ROTL32((x), 29))
#define ss4(x) (shr((x), 1) ^ (x))
#define ss5(x) (shr((x), 2) ^ (x))

#define rs1(x) SPH_ROTL32((x),  3)
#define rs2(x) SPH_ROTL32((x),  7)
#define rs3(x) SPH_ROTL32((x), 13)
#define rs4(x) SPH_ROTL32((x), 16)
#define rs5(x) SPH_ROTL32((x), 19)
#define rs6(x) SPH_ROTL32((x), 23)
#define rs7(x) SPH_ROTL32((x), 27)

/* Message expansion function 1 */
__forceinline__ __device__
uint32_t expand32_1(int i, uint32_t *M32, const uint32_t *H, uint32_t *Q)
{
	return (ss1(Q[i - 16]) + ss2(Q[i - 15]) + ss3(Q[i - 14]) + ss0(Q[i - 13])
		+ ss1(Q[i - 12]) + ss2(Q[i - 11]) + ss3(Q[i - 10]) + ss0(Q[i - 9])
		+ ss1(Q[i - 8]) + ss2(Q[i - 7]) + ss3(Q[i - 6]) + ss0(Q[i - 5])
		+ ss1(Q[i - 4]) + ss2(Q[i - 3]) + ss3(Q[i - 2]) + ss0(Q[i - 1])
		+ ((i*(0x05555555ul) + SPH_ROTL32(M32[(i - 16) % 16], ((i - 16) % 16) + 1)
			+ SPH_ROTL32(M32[(i - 13) % 16], ((i - 13) % 16) + 1)
			- SPH_ROTL32(M32[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ H[(i - 16 + 7) % 16]));
}

/* Message expansion function 2 */
__forceinline__ __device__
uint32_t expand32_2(int i, uint32_t *M32, const uint32_t *H, uint32_t *Q)
{
	return (Q[i - 16] + rs1(Q[i - 15]) + Q[i - 14] + rs2(Q[i - 13])
		+ Q[i - 12] + rs3(Q[i - 11]) + Q[i - 10] + rs4(Q[i - 9])
		+ Q[i - 8] + rs5(Q[i - 7]) + Q[i - 6] + rs6(Q[i - 5])
		+ Q[i - 4] + rs7(Q[i - 3]) + ss4(Q[i - 2]) + ss5(Q[i - 1])
		+ ((i*(0x05555555ul) + SPH_ROTL32(M32[(i - 16) % 16], ((i - 16) % 16) + 1)
			+ SPH_ROTL32(M32[(i - 13) % 16], ((i - 13) % 16) + 1)
			- SPH_ROTL32(M32[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ H[(i - 16 + 7) % 16]));
}

__forceinline__ __device__
void Compression256(uint32_t *  M32)
{
	uint32_t Q[32], XL32, XH32;

	const uint32_t H[16] = {
		0x40414243, 0x44454647, 0x48494A4B, 0x4C4D4E4F,
		0x50515253, 0x54555657, 0x58595A5B, 0x5C5D5E5F,
		0x60616263, 0x64656667, 0x68696A6B, 0x6C6D6E6F,
		0x70717273, 0x74757677, 0x78797A7B, 0x7C7D7E7F
	};

	Q[0]  = (M32[5] ^ H[5]) - (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[13] ^ H[13]) + (M32[14] ^ H[14]);
	Q[1]  = (M32[6] ^ H[6]) - (M32[8] ^ H[8]) + (M32[11] ^ H[11]) + (M32[14] ^ H[14]) - (M32[15] ^ H[15]);
	Q[2]  = (M32[0] ^ H[0]) + (M32[7] ^ H[7]) + (M32[9]  ^ H[9])  - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[3]  = (M32[0] ^ H[0]) - (M32[1] ^ H[1]) + (M32[8]  ^ H[8])  - (M32[10] ^ H[10]) + (M32[13] ^ H[13]);
	Q[4]  = (M32[1] ^ H[1]) + (M32[2] ^ H[2]) + (M32[9]  ^ H[9])  - (M32[11] ^ H[11]) - (M32[14] ^ H[14]);
	Q[5]  = (M32[3] ^ H[3]) - (M32[2] ^ H[2]) + (M32[10] ^ H[10]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[6]  = (M32[4] ^ H[4]) - (M32[0] ^ H[0]) - (M32[3]  ^ H[3])  - (M32[11] ^ H[11]) + (M32[13] ^ H[13]);
	Q[7]  = (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[5]  ^ H[5])  - (M32[12] ^ H[12]) - (M32[14] ^ H[14]);
	Q[8]  = (M32[2] ^ H[2]) - (M32[5] ^ H[5]) - (M32[6]  ^ H[6])  + (M32[13] ^ H[13]) - (M32[15] ^ H[15]);
	Q[9]  = (M32[0] ^ H[0]) - (M32[3] ^ H[3]) + (M32[6]  ^ H[6])  - (M32[7]  ^ H[7])  + (M32[14] ^ H[14]);
	Q[10] = (M32[8] ^ H[8]) - (M32[1] ^ H[1]) - (M32[4]  ^ H[4])  - (M32[7]  ^ H[7])  + (M32[15] ^ H[15]);
	Q[11] = (M32[8] ^ H[8]) - (M32[0] ^ H[0]) - (M32[2]  ^ H[2])  - (M32[5]  ^ H[5])  + (M32[9]  ^ H[9]);
	Q[12] = (M32[1] ^ H[1]) + (M32[3] ^ H[3]) - (M32[6]  ^ H[6])  - (M32[9]  ^ H[9])  + (M32[10] ^ H[10]);
	Q[13] = (M32[2] ^ H[2]) + (M32[4] ^ H[4]) + (M32[7]  ^ H[7])  + (M32[10] ^ H[10]) + (M32[11] ^ H[11]);
	Q[14] = (M32[3] ^ H[3]) - (M32[5] ^ H[5]) + (M32[8]  ^ H[8])  - (M32[11] ^ H[11]) - (M32[12] ^ H[12]);
	Q[15] = (M32[12] ^ H[12]) - (M32[4] ^ H[4]) - (M32[6] ^ H[6]) - (M32[9]  ^ H[9])  + (M32[13] ^ H[13]);

	/*  Diffuse the differences in every word in a bijective manner with ssi, and then add the values of the previous double pipe. */
	Q[0]  = ss0(Q[0])  + H[1];
	Q[1]  = ss1(Q[1])  + H[2];
	Q[2]  = ss2(Q[2])  + H[3];
	Q[3]  = ss3(Q[3])  + H[4];
	Q[4]  = ss4(Q[4])  + H[5];
	Q[5]  = ss0(Q[5])  + H[6];
	Q[6]  = ss1(Q[6])  + H[7];
	Q[7]  = ss2(Q[7])  + H[8];
	Q[8]  = ss3(Q[8])  + H[9];
	Q[9]  = ss4(Q[9])  + H[10];
	Q[10] = ss0(Q[10]) + H[11];
	Q[11] = ss1(Q[11]) + H[12];
	Q[12] = ss2(Q[12]) + H[13];
	Q[13] = ss3(Q[13]) + H[14];
	Q[14] = ss4(Q[14]) + H[15];
	Q[15] = ss0(Q[15]) + H[0];

	/* This is the Message expansion or f_1 in the documentation.       */
	/* It has 16 rounds.                                                */
	/* Blue Midnight Wish has two tunable security parameters.          */
	/* The parameters are named EXPAND_1_ROUNDS and EXPAND_2_ROUNDS.    */
	/* The following relation for these parameters should is satisfied: */
	/* EXPAND_1_ROUNDS + EXPAND_2_ROUNDS = 16                           */

	#pragma unroll
	for (int i=16; i<18; i++)
		Q[i] = expand32_1(i, M32, H, Q);

	#pragma nounroll
	for (int i=18; i<32; i++)
		Q[i] = expand32_2(i, M32, H, Q);

	/* Blue Midnight Wish has two temporary cummulative variables that accumulate via XORing */
	/* 16 new variables that are prooduced in the Message Expansion part.                    */
	XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
	XH32 = XL32^Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];


	/*  This part is the function f_2 - in the documentation            */

	/*  Compute the double chaining pipe for the next message block.    */
	M32[0] = (shl(XH32, 5) ^ shr(Q[16], 5) ^ M32[0]) + (XL32    ^ Q[24] ^ Q[0]);
	M32[1] = (shr(XH32, 7) ^ shl(Q[17], 8) ^ M32[1]) + (XL32    ^ Q[25] ^ Q[1]);
	M32[2] = (shr(XH32, 5) ^ shl(Q[18], 5) ^ M32[2]) + (XL32    ^ Q[26] ^ Q[2]);
	M32[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ M32[3]) + (XL32    ^ Q[27] ^ Q[3]);
	M32[4] = (shr(XH32, 3) ^ Q[20] ^ M32[4]) + (XL32    ^ Q[28] ^ Q[4]);
	M32[5] = (shl(XH32, 6) ^ shr(Q[21], 6) ^ M32[5]) + (XL32    ^ Q[29] ^ Q[5]);
	M32[6] = (shr(XH32, 4) ^ shl(Q[22], 6) ^ M32[6]) + (XL32    ^ Q[30] ^ Q[6]);
	M32[7] = (shr(XH32, 11) ^ shl(Q[23], 2) ^ M32[7]) + (XL32    ^ Q[31] ^ Q[7]);

	M32[8] = SPH_ROTL32(M32[4], 9) + (XH32     ^     Q[24] ^ M32[8]) + (shl(XL32, 8) ^ Q[23] ^ Q[8]);
	M32[9] = SPH_ROTL32(M32[5], 10) + (XH32     ^     Q[25] ^ M32[9]) + (shr(XL32, 6) ^ Q[16] ^ Q[9]);
	M32[10] = SPH_ROTL32(M32[6], 11) + (XH32     ^     Q[26] ^ M32[10]) + (shl(XL32, 6) ^ Q[17] ^ Q[10]);
	M32[11] = SPH_ROTL32(M32[7], 12) + (XH32     ^     Q[27] ^ M32[11]) + (shl(XL32, 4) ^ Q[18] ^ Q[11]);
	M32[12] = SPH_ROTL32(M32[0], 13) + (XH32     ^     Q[28] ^ M32[12]) + (shr(XL32, 3) ^ Q[19] ^ Q[12]);
	M32[13] = SPH_ROTL32(M32[1], 14) + (XH32     ^     Q[29] ^ M32[13]) + (shr(XL32, 4) ^ Q[20] ^ Q[13]);
	M32[14] = SPH_ROTL32(M32[2], 15) + (XH32     ^     Q[30] ^ M32[14]) + (shr(XL32, 7) ^ Q[21] ^ Q[14]);
	M32[15] = SPH_ROTL32(M32[3], 16) + (XH32     ^     Q[31] ^ M32[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);
}

__forceinline__ __device__
void Compression256_2(uint32_t *  M32)
{
	uint32_t XL32, XH32, Q[32];

	const uint32_t H[16] = {
		0xaaaaaaa0, 0xaaaaaaa1, 0xaaaaaaa2, 0xaaaaaaa3,
		0xaaaaaaa4, 0xaaaaaaa5, 0xaaaaaaa6, 0xaaaaaaa7,
		0xaaaaaaa8, 0xaaaaaaa9, 0xaaaaaaaa, 0xaaaaaaab,
		0xaaaaaaac, 0xaaaaaaad, 0xaaaaaaae, 0xaaaaaaaf
	};

	Q[0] = (M32[5] ^ H[5]) - (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[13] ^ H[13]) + (M32[14] ^ H[14]);
	Q[1] = (M32[6] ^ H[6]) - (M32[8] ^ H[8]) + (M32[11] ^ H[11]) + (M32[14] ^ H[14]) - (M32[15] ^ H[15]);
	Q[2] = (M32[0] ^ H[0]) + (M32[7] ^ H[7]) + (M32[9] ^ H[9]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[3] = (M32[0] ^ H[0]) - (M32[1] ^ H[1]) + (M32[8] ^ H[8]) - (M32[10] ^ H[10]) + (M32[13] ^ H[13]);
	Q[4] = (M32[1] ^ H[1]) + (M32[2] ^ H[2]) + (M32[9] ^ H[9]) - (M32[11] ^ H[11]) - (M32[14] ^ H[14]);
	Q[5] = (M32[3] ^ H[3]) - (M32[2] ^ H[2]) + (M32[10] ^ H[10]) - (M32[12] ^ H[12]) + (M32[15] ^ H[15]);
	Q[6] = (M32[4] ^ H[4]) - (M32[0] ^ H[0]) - (M32[3] ^ H[3]) - (M32[11] ^ H[11]) + (M32[13] ^ H[13]);
	Q[7] = (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[5] ^ H[5]) - (M32[12] ^ H[12]) - (M32[14] ^ H[14]);
	Q[8] = (M32[2] ^ H[2]) - (M32[5] ^ H[5]) - (M32[6] ^ H[6]) + (M32[13] ^ H[13]) - (M32[15] ^ H[15]);
	Q[9] = (M32[0] ^ H[0]) - (M32[3] ^ H[3]) + (M32[6] ^ H[6]) - (M32[7] ^ H[7]) + (M32[14] ^ H[14]);
	Q[10] = (M32[8] ^ H[8]) - (M32[1] ^ H[1]) - (M32[4] ^ H[4]) - (M32[7] ^ H[7]) + (M32[15] ^ H[15]);
	Q[11] = (M32[8] ^ H[8]) - (M32[0] ^ H[0]) - (M32[2] ^ H[2]) - (M32[5] ^ H[5]) + (M32[9] ^ H[9]);
	Q[12] = (M32[1] ^ H[1]) + (M32[3] ^ H[3]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[10] ^ H[10]);
	Q[13] = (M32[2] ^ H[2]) + (M32[4] ^ H[4]) + (M32[7] ^ H[7]) + (M32[10] ^ H[10]) + (M32[11] ^ H[11]);
	Q[14] = (M32[3] ^ H[3]) - (M32[5] ^ H[5]) + (M32[8] ^ H[8]) - (M32[11] ^ H[11]) - (M32[12] ^ H[12]);
	Q[15] = (M32[12] ^ H[12]) - (M32[4] ^ H[4]) - (M32[6] ^ H[6]) - (M32[9] ^ H[9]) + (M32[13] ^ H[13]);

	/*  Diffuse the differences in every word in a bijective manner with ssi, and then add the values of the previous double pipe.*/
	Q[0] = ss0(Q[0]) + H[1];
	Q[1] = ss1(Q[1]) + H[2];
	Q[2] = ss2(Q[2]) + H[3];
	Q[3] = ss3(Q[3]) + H[4];
	Q[4] = ss4(Q[4]) + H[5];
	Q[5] = ss0(Q[5]) + H[6];
	Q[6] = ss1(Q[6]) + H[7];
	Q[7] = ss2(Q[7]) + H[8];
	Q[8] = ss3(Q[8]) + H[9];
	Q[9] = ss4(Q[9]) + H[10];
	Q[10] = ss0(Q[10]) + H[11];
	Q[11] = ss1(Q[11]) + H[12];
	Q[12] = ss2(Q[12]) + H[13];
	Q[13] = ss3(Q[13]) + H[14];
	Q[14] = ss4(Q[14]) + H[15];
	Q[15] = ss0(Q[15]) + H[0];

	/* This is the Message expansion or f_1 in the documentation.       */
	/* It has 16 rounds.                                                */
	/* Blue Midnight Wish has two tunable security parameters.          */
	/* The parameters are named EXPAND_1_ROUNDS and EXPAND_2_ROUNDS.    */
	/* The following relation for these parameters should is satisfied: */
	/* EXPAND_1_ROUNDS + EXPAND_2_ROUNDS = 16                           */

	#pragma unroll
	for (int i = 16; i<18; i++)
		Q[i] = expand32_1(i, M32, H, Q);

	#pragma nounroll
	for (int i = 18; i<32; i++)
		Q[i] = expand32_2(i, M32, H, Q);

	/* Blue Midnight Wish has two temporary cummulative variables that accumulate via XORing */
	/* 16 new variables that are prooduced in the Message Expansion part.                    */
	XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
	XH32 = XL32 ^ Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];

	M32[2] = (shr(XH32, 5) ^ shl(Q[18], 5) ^ M32[2]) + (XL32 ^ Q[26] ^ Q[2]);
	M32[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ M32[3]) + (XL32 ^ Q[27] ^ Q[3]);
	M32[14] = SPH_ROTL32(M32[2], 15) + (XH32 ^ Q[30] ^ M32[14]) + (shr(XL32, 7) ^ Q[21] ^ Q[14]);
	M32[15] = SPH_ROTL32(M32[3], 16) + (XH32 ^ Q[31] ^ M32[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);
}

#define TPB 512
__global__ __launch_bounds__(TPB, 2)
void bmw256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *const __restrict__ nonceVector)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t message[16] = { 0 };

		LOHI(message[0], message[1], __ldg(&g_hash[thread]));
		LOHI(message[2], message[3], __ldg(&g_hash[thread + 1 * threads]));
		LOHI(message[4], message[5], __ldg(&g_hash[thread + 2 * threads]));
		LOHI(message[6], message[7], __ldg(&g_hash[thread + 3 * threads]));

		message[8]=0x80;
		message[14]=0x100;
		Compression256(message);
		Compression256_2(message);

		if (((uint64_t*)message)[7] <= pTarget[3])
		{
			uint32_t tmp = atomicExch(&nonceVector[0], startNounce + thread);
			if (tmp != 0)
				nonceVector[1] = tmp;
		}
	}
}

__host__
void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces)
{
	const uint32_t threadsperblock = TPB;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	hipMemset(d_GNonce[thr_id], 0, 2 * sizeof(uint32_t));

	bmw256_gpu_hash_32 << <grid, block >> >(threads, startNounce, g_hash, d_GNonce[thr_id]);
	hipMemcpy(d_gnounce[thr_id], d_GNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	resultnonces[0] = *(d_gnounce[thr_id]);
	resultnonces[1] = *(d_gnounce[thr_id] + 1);
}


__host__
void bmw256_cpu_init(int thr_id, uint32_t threads)
{
	hipMalloc(&d_GNonce[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&d_gnounce[thr_id], 2 * sizeof(uint32_t));
}

__host__
void bmw256_cpu_free(int thr_id)
{
	hipFree(d_GNonce[thr_id]);
	hipHostFree(d_gnounce[thr_id]);
}

__host__
void bmw256_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);
}
