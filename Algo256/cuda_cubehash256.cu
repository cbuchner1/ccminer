#include "hip/hip_runtime.h"
#include "cuda_helper.h"

#define CUBEHASH_ROUNDS 16 /* this is r for CubeHashr/b */
#define CUBEHASH_BLOCKBYTES 32 /* this is b for CubeHashr/b */

#if __CUDA_ARCH__ < 350
#define LROT(x,bits) ((x << bits) | (x >> (32 - bits)))
#else
#define LROT(x, bits) __funnelshift_l(x, x, bits)
#endif

#if __CUDA_ARCH__ < 500
#define TPB 576
#else
#define TPB 1024
#endif

#define ROTATEUPWARDS7(a)  LROT(a,7)
#define ROTATEUPWARDS11(a) LROT(a,11)

//#define SWAP(a,b) { uint32_t u = a; a = b; b = u; }
#define SWAP(a,b) { a ^= b; b ^= a; a ^= b; }

__device__ __forceinline__ void rrounds(uint32_t x[2][2][2][2][2])
{
	int r;
	int j;
	int k;
	int l;
	int m;

	#pragma unroll 2
	for (r = 0; r < CUBEHASH_ROUNDS; ++r) {

		/* "add x_0jklm into x_1jklmn modulo 2^32" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[1][j][k][l][m] += x[0][j][k][l][m];

		/* "rotate x_0jklm upwards by 7 bits" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[0][j][k][l][m] = ROTATEUPWARDS7(x[0][j][k][l][m]);

		/* "swap x_00klm with x_01klm" */
#pragma unroll 2
		for (k = 0; k < 2; ++k)
#pragma unroll 2
			for (l = 0; l < 2; ++l)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[0][0][k][l][m], x[0][1][k][l][m])

					/* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[0][j][k][l][m] ^= x[1][j][k][l][m];

		/* "swap x_1jk0m with x_1jk1m" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[1][j][k][0][m], x[1][j][k][1][m])

					/* "add x_0jklm into x_1jklm modulo 2^32" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[1][j][k][l][m] += x[0][j][k][l][m];

		/* "rotate x_0jklm upwards by 11 bits" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
#pragma unroll 2
					for (m = 0; m < 2; ++m)
						x[0][j][k][l][m] = ROTATEUPWARDS11(x[0][j][k][l][m]);

		/* "swap x_0j0lm with x_0j1lm" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (l = 0; l < 2; ++l)
#pragma unroll 2
				for (m = 0; m < 2; ++m)
					SWAP(x[0][j][0][l][m], x[0][j][1][l][m])

					/* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
					for (j = 0; j < 2; ++j)
#pragma unroll 2
						for (k = 0; k < 2; ++k)
#pragma unroll 2
							for (l = 0; l < 2; ++l)
#pragma unroll 2
								for (m = 0; m < 2; ++m)
									x[0][j][k][l][m] ^= x[1][j][k][l][m];

		/* "swap x_1jkl0 with x_1jkl1" */
#pragma unroll 2
		for (j = 0; j < 2; ++j)
#pragma unroll 2
			for (k = 0; k < 2; ++k)
#pragma unroll 2
				for (l = 0; l < 2; ++l)
					SWAP(x[1][j][k][l][0], x[1][j][k][l][1])

	}
}

__device__ __forceinline__ void block_tox(const uint32_t *in, uint32_t x[2][2][2][2][2])
{
	x[0][0][0][0][0] ^= in[0];
	x[0][0][0][0][1] ^= in[1];
	x[0][0][0][1][0] ^= in[2];
	x[0][0][0][1][1] ^= in[3];
	x[0][0][1][0][0] ^= in[4];
	x[0][0][1][0][1] ^= in[5];
	x[0][0][1][1][0] ^= in[6];
	x[0][0][1][1][1] ^= in[7];
}

__device__ __forceinline__ void hash_fromx(uint32_t *out, uint32_t x[2][2][2][2][2])
{
	out[0] = x[0][0][0][0][0];
	out[1] = x[0][0][0][0][1];
	out[2] = x[0][0][0][1][0];
	out[3] = x[0][0][0][1][1];
	out[4] = x[0][0][1][0][0];
	out[5] = x[0][0][1][0][1];
	out[6] = x[0][0][1][1][0];
	out[7] = x[0][0][1][1][1];

}

__device__ __forceinline__
void Update32(uint32_t x[2][2][2][2][2], const uint32_t *data)
{
	/* "xor the block into the first b bytes of the state" */
	/* "and then transform the state invertibly through r identical rounds" */
	block_tox(data, x);
	rrounds(x);
}

__device__ __forceinline__
void Update32_const(uint32_t x[2][2][2][2][2])
{
	x[0][0][0][0][0] ^= 0x80;
	rrounds(x);
}

__device__ __forceinline__
void Final(uint32_t x[2][2][2][2][2], uint32_t *hashval)
{
	/* "the integer 1 is xored into the last state word x_11111" */
	x[1][1][1][1][1] ^= 1U;

	/* "the state is then transformed invertibly through 10r identical rounds" */
	#pragma unroll 2
	for (int i = 0; i < 10; ++i) rrounds(x);

	/* "output the first h/8 bytes of the state" */
	hash_fromx(hashval, x);
}

#if __CUDA_ARCH__ >= 500

__global__	__launch_bounds__(TPB, 1)
void cubehash256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 Hash[4];

		Hash[0] = __ldg(&g_hash[thread]);
		Hash[1] = __ldg(&g_hash[thread + 1 * threads]);
		Hash[2] = __ldg(&g_hash[thread + 2 * threads]);
		Hash[3] = __ldg(&g_hash[thread + 3 * threads]);

		uint32_t x[2][2][2][2][2] =
		{
			0xEA2BD4B4, 0xCCD6F29F, 0x63117E71, 0x35481EAE,
			0x22512D5B, 0xE5D94E63, 0x7E624131, 0xF4CC12BE,
			0xC2D0B696, 0x42AF2070, 0xD0720C35, 0x3361DA8C,
			0x28CCECA4, 0x8EF8AD83, 0x4680AC00, 0x40E5FBAB,
			0xD89041C3, 0x6107FBD5, 0x6C859D41, 0xF0B26679,
			0x09392549, 0x5FA25603, 0x65C892FD, 0x93CB6285,
			0x2AF2B5AE, 0x9E4B4E60, 0x774ABFDD, 0x85254725,
			0x15815AEB, 0x4AB6AAD6, 0x9CDAF8AF, 0xD6032C0A
		};

		x[0][0][0][0][0] ^= Hash[0].x;
		x[0][0][0][0][1] ^= Hash[0].y;
		x[0][0][0][1][0] ^= Hash[1].x;
		x[0][0][0][1][1] ^= Hash[1].y;
		x[0][0][1][0][0] ^= Hash[2].x;
		x[0][0][1][0][1] ^= Hash[2].y;
		x[0][0][1][1][0] ^= Hash[3].x;
		x[0][0][1][1][1] ^= Hash[3].y;

		rrounds(x);
		x[0][0][0][0][0] ^= 0x80U;
		rrounds(x);

		Final(x, (uint32_t*) Hash);

		g_hash[thread] =               Hash[0];
		g_hash[1 * threads + thread] = Hash[1];
		g_hash[2 * threads + thread] = Hash[2];
		g_hash[3 * threads + thread] = Hash[3];
	}
}

#else

__global__	__launch_bounds__(TPB, 1)
void cubehash256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *d_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t Hash[8];
		uint64_t* g_hash = (uint64_t*) d_hash;

		LOHI(Hash[0], Hash[1], __ldg(&g_hash[thread]));
		LOHI(Hash[2], Hash[3], __ldg(&g_hash[thread + 1 * threads]));
		LOHI(Hash[4], Hash[5], __ldg(&g_hash[thread + 2 * threads]));
		LOHI(Hash[6], Hash[7], __ldg(&g_hash[thread + 3 * threads]));

		uint32_t x[2][2][2][2][2] =
		{
			0xEA2BD4B4, 0xCCD6F29F, 0x63117E71, 0x35481EAE,
			0x22512D5B, 0xE5D94E63, 0x7E624131, 0xF4CC12BE,
			0xC2D0B696, 0x42AF2070, 0xD0720C35, 0x3361DA8C,
			0x28CCECA4, 0x8EF8AD83, 0x4680AC00, 0x40E5FBAB,
			0xD89041C3, 0x6107FBD5, 0x6C859D41, 0xF0B26679,
			0x09392549, 0x5FA25603, 0x65C892FD, 0x93CB6285,
			0x2AF2B5AE, 0x9E4B4E60, 0x774ABFDD, 0x85254725,
			0x15815AEB, 0x4AB6AAD6, 0x9CDAF8AF, 0xD6032C0A
		};

		x[0][0][0][0][0] ^= Hash[0];
		x[0][0][0][0][1] ^= Hash[1];
		x[0][0][0][1][0] ^= Hash[2];
		x[0][0][0][1][1] ^= Hash[3];
		x[0][0][1][0][0] ^= Hash[4];
		x[0][0][1][0][1] ^= Hash[5];
		x[0][0][1][1][0] ^= Hash[6];
		x[0][0][1][1][1] ^= Hash[7];

		rrounds(x);
		x[0][0][0][0][0] ^= 0x80U;
		rrounds(x);

		Final(x, Hash);

		g_hash[thread] =               ((uint64_t*)Hash)[0];
		g_hash[1 * threads + thread] = ((uint64_t*)Hash)[1];
		g_hash[2 * threads + thread] = ((uint64_t*)Hash)[2];
		g_hash[3 * threads + thread] = ((uint64_t*)Hash)[3];
	}
}

#endif

__host__
void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order)
{
	uint32_t tpb = TPB;

	dim3 grid((threads + tpb-1)/tpb);
	dim3 block(tpb);

	cubehash256_gpu_hash_32 <<<grid, block>>> (threads, startNounce, (uint2*) d_hash);
}
