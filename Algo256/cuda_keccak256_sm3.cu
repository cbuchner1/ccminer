#include "hip/hip_runtime.h"
#include "miner.h"

extern "C" {
#include <stdint.h>
#include <memory.h>
}

#include "cuda_helper.h"

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

static uint32_t *d_KNonce[MAX_GPUS];

__constant__ uint32_t pTarget[8];
__constant__ uint64_t keccak_round_constants[24];
__constant__ uint64_t c_PaddedMessage80[10]; // padded message (80 bytes + padding?)

#if __CUDA_ARCH__ >= 350
__device__ __forceinline__
static void keccak_blockv35(uint2 *s, const uint64_t *keccak_round_constants)
{
	size_t i;
	uint2 t[5], u[5], v, w;

	#pragma unroll
	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROL2(t[1], 1);
		u[1] = t[0] ^ ROL2(t[2], 1);
		u[2] = t[1] ^ ROL2(t[3], 1);
		u[3] = t[2] ^ ROL2(t[4], 1);
		u[4] = t[3] ^ ROL2(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1] = ROL2(s[6], 44);
		s[6] = ROL2(s[9], 20);
		s[9] = ROL2(s[22], 61);
		s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);
		s[20] = ROL2(s[2], 62);
		s[2] = ROL2(s[12], 43);
		s[12] = ROL2(s[13], 25);
		s[13] = ROL2(s[19], 8);
		s[19] = ROL2(s[23], 56);
		s[23] = ROL2(s[15], 41);
		s[15] = ROL2(s[4], 27);
		s[4] = ROL2(s[24], 14);
		s[24] = ROL2(s[21], 2);
		s[21] = ROL2(s[8], 55);
		s[8] = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);
		s[5] = ROL2(s[3], 28);
		s[3] = ROL2(s[18], 21);
		s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);
		s[11] = ROL2(s[7], 6);
		s[7] = ROL2(s[10], 3);
		s[10] = ROL2(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= vectorize(keccak_round_constants[i]);
	}
}
#else

__device__ __forceinline__
static void keccak_blockv30(uint64_t *s, const uint64_t *keccak_round_constants)
{
	size_t i;
	uint64_t t[5], u[5], v, w;

	/* absorb input */

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];
	}
}
#endif

__global__ __launch_bounds__(128,5)
void keccak256_sm3_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;

#if __CUDA_ARCH__ >= 350
		uint2 keccak_gpu_state[25];
		#pragma unroll 25
		for (int i=0; i<25; i++) {
			if (i<9) keccak_gpu_state[i] = vectorize(c_PaddedMessage80[i]);
			else     keccak_gpu_state[i] = make_uint2(0, 0);
		}

		keccak_gpu_state[9]= vectorize(c_PaddedMessage80[9]);
		keccak_gpu_state[9].y = cuda_swab32(nounce);
		keccak_gpu_state[10] = make_uint2(1, 0);
		keccak_gpu_state[16] = make_uint2(0, 0x80000000);

		keccak_blockv35(keccak_gpu_state,keccak_round_constants);
		if (devectorize(keccak_gpu_state[3]) <= ((uint64_t*)pTarget)[3]) {resNounce[0] = nounce;}
#else
		uint64_t keccak_gpu_state[25];
		#pragma unroll 25
		for (int i=0; i<25; i++) {
			if (i<9) keccak_gpu_state[i] = c_PaddedMessage80[i];
			else     keccak_gpu_state[i] = 0;
		}
		keccak_gpu_state[9]  = REPLACE_HIDWORD(c_PaddedMessage80[9], cuda_swab32(nounce));
		keccak_gpu_state[10] = 0x0000000000000001;
		keccak_gpu_state[16] = 0x8000000000000000;

		keccak_blockv30(keccak_gpu_state, keccak_round_constants);
		if (keccak_gpu_state[3] <= ((uint64_t*)pTarget)[3]) { resNounce[0] = nounce; }
#endif
	}
}

__host__
void keccak256_sm3_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, int order)
{
	hipMemset(d_KNonce[thr_id], 0xff, 2*sizeof(uint32_t));
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	keccak256_sm3_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_KNonce[thr_id]);

	hipMemcpy(resNonces, d_KNonce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

#if 0
__global__ __launch_bounds__(256,3)
void keccak256_sm3_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
#if __CUDA_ARCH__ >= 350 /* tpr: to double check if faster on SM5+ */
		uint2 keccak_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4) keccak_gpu_state[i] = vectorize(outputHash[i*threads+thread]);
			else     keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_gpu_state[4]  = make_uint2(1, 0);
		keccak_gpu_state[16] = make_uint2(0, 0x80000000);
		keccak_blockv35(keccak_gpu_state, keccak_round_constants);

		#pragma unroll 4
		for (int i=0; i<4; i++)
			outputHash[i*threads+thread] = devectorize(keccak_gpu_state[i]);
#else
		uint64_t keccak_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4)
				keccak_gpu_state[i] = outputHash[i*threads+thread];
			else
				keccak_gpu_state[i] = 0;
		}
		keccak_gpu_state[4]  = 0x0000000000000001;
		keccak_gpu_state[16] = 0x8000000000000000;

		keccak_blockv30(keccak_gpu_state, keccak_round_constants);
		#pragma unroll 4
		for (int i = 0; i<4; i++)
			outputHash[i*threads + thread] = keccak_gpu_state[i];
#endif
	}
}

__host__
void keccak256_sm3_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	keccak256_sm3_gpu_hash_32 <<<grid, block>>> (threads, startNounce, d_outputHash);
	MyStreamSynchronize(NULL, order, thr_id);
}
#endif

__host__
void keccak256_sm3_setBlock_80(void *pdata,const void *pTargetIn)
{
	unsigned char PaddedMessage[80];
	memcpy(PaddedMessage, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 10*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void keccak256_sm3_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(keccak_round_constants), host_keccak_round_constants,
				sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc(&d_KNonce[thr_id], 2*sizeof(uint32_t)));
}

__host__
void keccak256_sm3_free(int thr_id)
{
	hipFree(d_KNonce[thr_id]);
}
