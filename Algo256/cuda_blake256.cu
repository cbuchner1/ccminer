#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 *
 * + merged blake+keccak kernel for lyra2v2
 */
extern "C" {
#include "sph/sph_blake.h"
}

#include "cuda_helper.h"

#include <memory.h>

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
__device__ uint32_t __byte_perm(uint32_t a, uint32_t b, uint32_t c);
#endif

#define UINT2(x,y) make_uint2(x,y)

__device__ __inline__ uint2 ROR8(const uint2 a) {
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x0765);
	result.y = __byte_perm(a.x, a.y, 0x0765);
	return result;
}

static __device__ uint64_t cuda_swab32ll(uint64_t x) {
	return MAKE_ULONGLONG(cuda_swab32(_LODWORD(x)), cuda_swab32(_HIDWORD(x)));
}

__constant__ static uint32_t c_data[3 + 1];

__constant__ static uint32_t sigma[16][16];
static uint32_t  c_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

static const uint32_t  c_IV256[8] = {
	0x6A09E667, 0xBB67AE85,
	0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C,
	0x1F83D9AB, 0x5BE0CD19
};

__device__ __constant__ static uint32_t cpu_h[8];

__device__ __constant__ static  uint32_t  u256[16];
static const uint32_t  c_u256[16] = {
	0x243F6A88, 0x85A308D3,
	0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0,
	0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377,
	0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD,
	0x3F84D5B5, 0xB5470917
};

__constant__ uint2 keccak_round_constants35[24] = {
	{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
	{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
	{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
	{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
	{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
	{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
	{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
	{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
	{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
	{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};


#define GS2(a,b,c,d,x) { \
	const uint32_t idx1 = sigma[r][x]; \
	const uint32_t idx2 = sigma[r][(x)+1]; \
	v[a] += (m[idx1] ^ u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

//#define ROTL32(x, n) ((x) << (n)) | ((x) >> (32 - (n)))
//#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#define hostGS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[r][x]; \
	const uint32_t idx2 = c_sigma[r][(x)+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
						}

/* Second part (64-80) msg never change, store it */
__device__ __constant__ static const uint32_t  c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__host__ __forceinline__
static void blake256_compress1st(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t m[16];
	uint32_t v[16];

	for (int i = 0; i < 16; i++) {
		m[i] = block[i];
	}

	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] = c_u256[0];
	v[9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int r = 0; r < 14; r++) {
		/* column step */
		hostGS(0, 4, 0x8, 0xC, 0x0);
		hostGS(1, 5, 0x9, 0xD, 0x2);
		hostGS(2, 6, 0xA, 0xE, 0x4);
		hostGS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		hostGS(0, 5, 0xA, 0xF, 0x8);
		hostGS(1, 6, 0xB, 0xC, 0xA);
		hostGS(2, 7, 0x8, 0xD, 0xC);
		hostGS(3, 4, 0x9, 0xE, 0xE);
	}

	for (int i = 0; i < 16; i++) {
		int j = i & 7;
		h[j] ^= v[i];
	}
}

__device__ __forceinline__
static void blake256_compress2nd(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	#pragma unroll
	for (int i = 4; i < 16; i++) {
		m[i] = c_Padding[i];
	}

	#pragma unroll 8
	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] = u256[0];
	v[9] = u256[1];
	v[10] = u256[2];
	v[11] = u256[3];

	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

	#pragma unroll 14
	for (int r = 0; r < 14; r++) {
		/* column step */
		GS2(0, 4, 0x8, 0xC, 0x0);
		GS2(1, 5, 0x9, 0xD, 0x2);
		GS2(2, 6, 0xA, 0xE, 0x4);
		GS2(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS2(0, 5, 0xA, 0xF, 0x8);
		GS2(1, 6, 0xB, 0xC, 0xA);
		GS2(2, 7, 0x8, 0xD, 0xC);
		GS2(3, 4, 0x9, 0xE, 0xE);
	}

	#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		int j = i & 7;
		h[j] ^= v[i];
	}
}

static void __forceinline__ __device__ keccak_block(uint2 *s)
{
	uint2 bc[5], tmpxor[5], u, v;
	//	uint2 s[25];

	#pragma unroll 1
	for (int i = 0; i < 24; i++)
	{
		#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		u = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(u, 1);

		u = s[0]; v = s[1]; s[0] ^= (~v) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & u; s[4] ^= (~u) & v;
		u = s[5]; v = s[6]; s[5] ^= (~v) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & u; s[9] ^= (~u) & v;
		u = s[10]; v = s[11]; s[10] ^= (~v) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & u; s[14] ^= (~u) & v;
		u = s[15]; v = s[16]; s[15] ^= (~v) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & u; s[19] ^= (~u) & v;
		u = s[20]; v = s[21]; s[20] ^= (~v) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & u; s[24] ^= (~u) & v;
		s[0] ^= keccak_round_constants35[i];
	}
}


//__launch_bounds__(256)
__global__
void blakeKeccak256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		const uint32_t T0 = 640;

		uint32_t h[8];
		#pragma unroll 8
		for (int i = 0; i<8; i++) { h[i] = cpu_h[i]; }

		uint32_t v[16];

		const uint32_t c_Padding[12] = {
			0x80000000, 0, 0, 0,
			0, 0, 0, 0,
			0, 1, 0, 640
		};

		const uint32_t  u256[16] = {
			0x243F6A88, 0x85A308D3,
			0x13198A2E, 0x03707344,
			0xA4093822, 0x299F31D0,
			0x082EFA98, 0xEC4E6C89,
			0x452821E6, 0x38D01377,
			0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD,
			0x3F84D5B5, 0xB5470917
		};

		uint32_t m[16] = {
			c_data[0], c_data[1], c_data[2], nonce,
			c_Padding[0], c_Padding[1], c_Padding[2], c_Padding[3],
			c_Padding[4], c_Padding[5], c_Padding[6], c_Padding[7],
			c_Padding[8], c_Padding[9], c_Padding[10], c_Padding[11]
		};

		#pragma unroll 8
		for (int i = 0; i < 8; i++)
			v[i] = h[i];

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ T0;
		v[13] = u256[5] ^ T0;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);
		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);
		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		h[0] = cuda_swab32(h[0] ^ v[0] ^ v[8]);
		h[1] = cuda_swab32(h[1] ^ v[1] ^ v[9]);
		h[2] = cuda_swab32(h[2] ^ v[2] ^ v[10]);
		h[3] = cuda_swab32(h[3] ^ v[3] ^ v[11]);
		h[4] = cuda_swab32(h[4] ^ v[4] ^ v[12]);
		h[5] = cuda_swab32(h[5] ^ v[5] ^ v[13]);
		h[6] = cuda_swab32(h[6] ^ v[6] ^ v[14]);
		h[7] = cuda_swab32(h[7] ^ v[7] ^ v[15]);

		uint2 keccak_gpu_state[25] = { 0 };
		keccak_gpu_state[0].x = h[0];
		keccak_gpu_state[0].y = h[1];
		keccak_gpu_state[1].x = h[2];
		keccak_gpu_state[1].y = h[3];
		keccak_gpu_state[2].x = h[4];
		keccak_gpu_state[2].y = h[5];
		keccak_gpu_state[3].x = h[6];
		keccak_gpu_state[3].y = h[7];
		keccak_gpu_state[4] = UINT2(1, 0);

		keccak_gpu_state[16] = UINT2(0, 0x80000000);
		keccak_block(keccak_gpu_state);

		uint64_t *outputHash = (uint64_t *)Hash;
		#pragma unroll 4
		for (int i = 0; i<4; i++)
			outputHash[i*threads + thread] = devectorize(keccak_gpu_state[i]);
	}
}

__global__ __launch_bounds__(256, 3)
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint64_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t h[8];
		uint32_t input[4];

		#pragma unroll
		for (int i = 0; i < 8; i++) h[i] = cpu_h[i];

		#pragma unroll
		for (int i = 0; i < 3; ++i) input[i] = c_data[i];

		input[3] = startNonce + thread;
		blake256_compress2nd(h, input, 640);

		#pragma unroll
		for (int i = 0; i<4; i++) {
			Hash[i*threads + thread] = cuda_swab32ll(MAKE_ULONGLONG(h[2 * i], h[2 * i + 1]));
		}
	}
}

__host__
void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blake256_gpu_hash_80 << <grid, block >> > (threads, startNonce, Hash);
	MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata)
{
	uint32_t h[8], data[20];

	memcpy(data, pdata, 80);
	memcpy(h, c_IV256, sizeof(c_IV256));
	blake256_compress1st(h, pdata, 512);

	hipMemcpyToSymbol(HIP_SYMBOL(cpu_h), h, sizeof(h), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), &data[16], sizeof(c_data), 0, hipMemcpyHostToDevice);
}

__host__
void blake256_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyToSymbol(HIP_SYMBOL(u256), c_u256, sizeof(c_u256), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(sigma), c_sigma, sizeof(c_sigma), 0, hipMemcpyHostToDevice);
}

/** for lyra2v2 **/

__host__
void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blakeKeccak256_gpu_hash_80 <<<grid, block>>> (threads, startNonce, (uint32_t *)Hash);
}

__host__
void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order, hipStream_t stream)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blakeKeccak256_gpu_hash_80 <<<grid, block, 0, stream>>> (threads, startNonce, (uint32_t *)Hash);
}
