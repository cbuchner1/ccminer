#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 */

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

extern int compute_version[8];
#include "cuda_helper.h"

__constant__ static uint32_t  c_data[20];

__constant__ static uint32_t sigma[16][16];
static uint32_t  c_sigma[16][16] = {
		{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};


static const uint32_t  c_IV256[8] = {
	0x6A09E667, 0xBB67AE85,
	0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C,
	0x1F83D9AB, 0x5BE0CD19
};

__device__ __constant__ static uint32_t cpu_h[8];

__device__ __constant__ static  uint32_t  u256[16];
static const uint32_t  c_u256[16] = {
	0x243F6A88, 0x85A308D3,
	0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0,
	0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377,
	0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD,
	0x3F84D5B5, 0xB5470917
};

#define GS2(a,b,c,d,x) { \
	const uint32_t idx1 = sigma[r][x]; \
	const uint32_t idx2 = sigma[r][x+1]; \
	v[a] += (m[idx1] ^ u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}
//#define ROTL32(x, n) ((x) << (n)) | ((x) >> (32 - (n)))
#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#define hostGS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[r][x]; \
	const uint32_t idx2 = c_sigma[r][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

/* Second part (64-80) msg never change, store it */
__device__ __constant__ static const uint32_t  c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__host__ __forceinline__ static void blake256_compress1st(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t m[16];
	uint32_t v[16];


	for (int i = 0; i < 16; i++) {
		m[i] = block[i];
	}


	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] = c_u256[0];
	v[9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];


	for (int r = 0; r < 14; r++) {
		/* column step */
		hostGS(0, 4, 0x8, 0xC, 0x0);
		hostGS(1, 5, 0x9, 0xD, 0x2);
		hostGS(2, 6, 0xA, 0xE, 0x4);
		hostGS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		hostGS(0, 5, 0xA, 0xF, 0x8);
		hostGS(1, 6, 0xB, 0xC, 0xA);
		hostGS(2, 7, 0x8, 0xD, 0xC);
		hostGS(3, 4, 0x9, 0xE, 0xE);
	}

	for (int i = 0; i < 16; i++) {
		int j = i & 7;
		h[j] ^= v[i];
	}
}

void blake256_cpu_init(int thr_id, int threads)
{

	hipMemcpyToSymbol(HIP_SYMBOL(u256), c_u256, sizeof(c_u256), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(sigma), c_sigma, sizeof(c_sigma), 0, hipMemcpyHostToDevice);
}

__device__ __forceinline__ static void blake256_compress2nd(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

#pragma unroll 
	for (int i = 4; i < 16; i++) {
		m[i] = c_Padding[i];
	}

#pragma unroll 8
	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] =  u256[0];
	v[9] =  u256[1];
	v[10] = u256[2];
	v[11] = u256[3];

	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

#pragma unroll 14
	for (int r = 0; r < 14; r++) {
		/* column step */
		GS2(0, 4, 0x8, 0xC, 0x0);
		GS2(1, 5, 0x9, 0xD, 0x2);
		GS2(2, 6, 0xA, 0xE, 0x4);
		GS2(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS2(0, 5, 0xA, 0xF, 0x8);
		GS2(1, 6, 0xB, 0xC, 0xA);
		GS2(2, 7, 0x8, 0xD, 0xC);
		GS2(3, 4, 0x9, 0xE, 0xE);
	}
#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		 int j = i & 7;
		h[j] ^= v[i];
	}
}


__global__ __launch_bounds__(256,3) void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint64_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];

		uint32_t input[4];
        #pragma unroll 8
		for (int i = 0; i<8; i++) { h[i] = cpu_h[i];}
        #pragma unroll 3
		for (int i = 0; i < 3; ++i) input[i] = c_data[16 + i];
		input[3] = nonce;
		blake256_compress2nd(h, input, 640);


        #pragma unroll 
for (int i = 0; i<4; i++) { Hash[i*threads + thread] = cuda_swab32ll(MAKE_ULONGLONG(h[2 * i], h[2*i+1])); }

	}
}

__host__ void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order)
{
	const int threadsperblock = 256;
	

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	
	blake256_gpu_hash_80 << <grid, block, shared_size >> >(threads, startNonce, Hash);
	MyStreamSynchronize(NULL, order, thr_id);

}

__host__ void blake256_cpu_setBlock_80(uint32_t *pdata)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	uint32_t h[8];
	for (int i = 0; i<8; i++) { h[i] = c_IV256[i]; }
	blake256_compress1st(h, pdata, 512);
	hipMemcpyToSymbol(HIP_SYMBOL(cpu_h), h, sizeof(h), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice);
}

