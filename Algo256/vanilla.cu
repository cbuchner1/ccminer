#include "hip/hip_runtime.h"
/**
 * Optimized Blake-256 8-rounds Cuda Kernel (Tested on SM >3.0)
 * Based upon Blake-256 implementation of Tanguy Pruvot - Nov. 2014
 *
 * Provos Alexis - Jan. 2016
 *
 * Fixed CUDA 7.5 flaw
 * minor code changes
 * code cleanup
 * increased nonces per thread
 * removed SSE2 midstate computation
 * Provos Alexis - Mar 2016
 */

#include <stdint.h>
#include <memory.h>

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

/* threads per block and nonces per thread */
#define TPB 768
#define NPT 384
#define NBN 2

__constant__ uint32_t _ALIGN(16) d_data[21];

/* 16 gpu threads max */
static uint32_t		*d_resNonce[MAX_GPUS];
static uint32_t		*h_resNonce[MAX_GPUS];
static hipStream_t	streams[MAX_GPUS];

/* hash by cpu with blake 256 */
extern "C" void vanillahash(void *output, const void *input, int8_t blakerounds){
	uchar hash[64];
	sph_blake256_context ctx;

	sph_blake256_set_rounds(blakerounds);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}
#define GS4(a,b,c,d,x,y,a1,b1,c1,d1,x1,y1,a2,b2,c2,d2,x2,y2,a3,b3,c3,d3,x3,y3) { \
	v[ a]+= (m[ x] ^ z[ y]) + v[ b]; \
	v[a1]+= (m[x1] ^ z[y1]) + v[b1]; \
	v[a2]+= (m[x2] ^ z[y2]) + v[b2]; \
	v[a3]+= (m[x3] ^ z[y3]) + v[b3]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x1032); \
	v[d3] = __byte_perm(v[d3] ^ v[a3], 0, 0x1032); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	v[c2]+= v[d2]; \
	v[c3]+= v[d3]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[b2] = ROTR32(v[b2] ^ v[c2], 12); \
	v[b3] = ROTR32(v[b3] ^ v[c3], 12); \
	\
	v[ a]+= (m[ y] ^ z[ x]) + v[ b]; \
	v[a1]+= (m[y1] ^ z[x1]) + v[b1]; \
	v[a2]+= (m[y2] ^ z[x2]) + v[b2]; \
	v[a3]+= (m[y3] ^ z[x3]) + v[b3]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x0321); \
	v[d3] = __byte_perm(v[d3] ^ v[a3], 0, 0x0321); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	v[c2]+= v[d2]; \
	v[c3]+= v[d3]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
	v[b2] = ROTR32(v[b2] ^ v[c2], 7); \
	v[b3] = ROTR32(v[b3] ^ v[c3], 7); \
}

#define GS3(a,b,c,d,x,y,a1,b1,c1,d1,x1,y1,a2,b2,c2,d2,x2,y2) { \
	v[ a]+= (m[ x] ^ z[ y]) + v[ b]; \
	v[a1]+= (m[x1] ^ z[y1]) + v[b1]; \
	v[a2]+= (m[x2] ^ z[y2]) + v[b2]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x1032); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	v[c2]+= v[d2]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	v[b2] = ROTR32(v[b2] ^ v[c2], 12); \
	\
	v[ a]+= (m[ y] ^ z[ x]) + v[ b]; \
	v[a1]+= (m[y1] ^ z[x1]) + v[b1]; \
	v[a2]+= (m[y2] ^ z[x2]) + v[b2]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x0321); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	v[c2]+= v[d2]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
	v[b2] = ROTR32(v[b2] ^ v[c2], 7); \
}

#define GS2(a,b,c,d,x,y,a1,b1,c1,d1,x1,y1) { \
	v[ a]+= (m[ x] ^ z[ y]) + v[ b]; \
	v[a1]+= (m[x1] ^ z[y1]) + v[b1]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x1032); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 12); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 12); \
	\
	v[ a]+= (m[ y] ^ z[ x]) + v[ b]; \
	v[a1]+= (m[y1] ^ z[x1]) + v[b1]; \
	\
	v[ d] = __byte_perm(v[ d] ^ v[ a], 0, 0x0321); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	\
	v[ c]+= v[ d]; \
	v[c1]+= v[d1]; \
	\
	v[ b] = ROTR32(v[ b] ^ v[ c], 7); \
	v[b1] = ROTR32(v[b1] ^ v[c1], 7); \
}

#define GS(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ z[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ z[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
}

__global__ __launch_bounds__(TPB,1)
void vanilla_gpu_hash_16_8(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,const uint64_t highTarget){
	uint32_t _ALIGN(16) v[16];
	uint32_t _ALIGN(16) tmp[16];

	const size_t thread   = blockDim.x * blockIdx.x + threadIdx.x;
	const uint64_t step     = gridDim.x * blockDim.x;
	const uint64_t maxNonce = startNonce + threads;

	const int8_t r[][16] = {
		{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 },{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 },
		{  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 },{  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 },
		{  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 },{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 },
		{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 }
	};
	const uint32_t z[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344, 0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C, 0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};

//PREFETCH
	#pragma unroll
	for(int i=0;i<16;i++){
		tmp[ i] = d_data[ i];
	}

	uint32_t m[16] = {
				  d_data[16],	d_data[17],	d_data[18],	0,
				  0x80000000UL,	0,		0,		0,
				  0,		0,		0,		0,
				  0,		1,		0,		640
	};

	const uint32_t h7 = d_data[19];
	const uint32_t h6 = d_data[20];
//END OF PREFETCH
	uint64_t m3 = startNonce + thread;
loopstart:
		if(m3>=maxNonce)return;
		m[3]  = m3;

		#pragma unroll
		for(int i=0;i<16;i++)
			v[ i] = tmp[ i];

		v[ 1]+= m[3] ^ z[2];
		v[13] = __byte_perm(v[13] ^ v[1],0, 0x0321);
		v[ 9]+= v[13];
		v[ 5] = ROTR32(v[5] ^ v[9], 7);
		v[ 0]+= v[5];
		v[15] = __byte_perm(v[15] ^ v[0],0, 0x1032);

		v[10]+= v[15];
		v[ 5] = ROTR32(v[5] ^ v[10], 12);
		v[ 0]+= z[8] + v[5];
		v[15] = __byte_perm(v[15] ^ v[0],0, 0x0321);
		v[10]+= v[15];
		v[ 5] = ROTR32(v[5] ^ v[10], 7);

		GS3( 1, 6,11,12,10,11, 2, 7, 8,13,12,13, 3, 4, 9,14,14,15);

		#pragma unroll
		for(int i=0;i<6;i++){
			GS4(0, 4, 8,12,r[i][ 0],r[i][ 1], 1, 5, 9,13,r[i][ 2],r[i][ 3], 2, 6,10,14,r[i][ 4],r[i][ 5], 3, 7,11,15,r[i][ 6],r[i][ 7]);
			GS4(0, 5,10,15,r[i][ 8],r[i][ 9], 1, 6,11,12,r[i][10],r[i][11], 2, 7, 8,13,r[i][12],r[i][13], 3, 4, 9,14,r[i][14],r[i][15]);
		}
		GS4(0, 4, 8,12,r[6][ 0],r[6][ 1], 1, 5, 9,13,r[6][ 2],r[6][ 3], 2, 6,10,14,r[6][ 4],r[6][ 5], 3, 7,11,15,r[6][ 6],r[6][ 7]);

		v[ 0] += (m[ 5] ^ z[0]) + v[5];
		v[ 2] += (m[ 8] ^ z[6]) + v[7];
		v[13] = __byte_perm(v[13] ^ v[2],0, 0x1032);
		v[15] = __byte_perm(v[15] ^ v[0],0, 0x1032);

		v[ 8] += v[13];
		v[10] += v[15];

		v[ 5] = ROTR32(v[ 5] ^ v[10], 12);
		v[ 7] = ROTR32(v[ 7] ^ v[ 8], 12);

		v[ 0] += (m[ 0] ^ z[5]) + v[5];
		v[ 2] += (m[ 6] ^ z[8]) + v[7];

		v[15] = __byte_perm(v[15] ^ v[ 0],0, 0x0321);
		v[13] = __byte_perm(v[13] ^ v[ 2],0, 0x0321);

		v[8] += v[13];
		v[7]  = ROTR32(v[7] ^ v[8], 7);

		// only compute h6 & 7
		if((v[15]^h7)==v[7]){
			v[ 1] += (m[15] ^ z[ 4]) + v[6];
			v[ 3] += (m[2] ^ z[10]) + v[4];
			v[12]  = __byte_perm(v[12] ^ v[ 1],0, 0x1032);
			v[14]  = __byte_perm(v[14] ^ v[3],0, 0x1032);
			v[11] += v[12];
			v[ 9] += v[14];
			v[ 6]  = ROTR32(v[ 6] ^ v[11], 12);

			v[ 1] += (m[ 4] ^ z[15]) + v[ 6];
			v[ 3] += (m[10] ^ z[ 2]) + ROTR32(v[ 4] ^ v[ 9],12);
			v[12]  = __byte_perm(v[12] ^ v[ 1],0, 0x0321);
			v[14]  = __byte_perm(v[14] ^ v[ 3],0, 0x0321);

			v[11] += v[12];
			v[ 6] = ROTR32(v[ 6] ^ v[11], 7);

			if(cuda_swab32(h6^v[6]^v[14]) <= highTarget) {
#if NBN == 2
				/* keep the smallest nonce, + extra one if found */
				if (m[3] < resNonce[0]){
					resNonce[1] = resNonce[0];
					resNonce[0] = m[3];
				}
				else
					resNonce[1] = m[3];
#else
				resNonce[0] = m[3];
#endif
				return; //<-- this may cause a problem on extranonce if the extranonce is on position current_nonce + X * step where X=[1,2,3..,N]
			}
		}
	m3+=step;
	goto loopstart;
}

__host__
void vanilla_cpu_setBlock_16(const int thr_id,const uint32_t* endiandata, uint32_t *penddata){

	const uint32_t _ALIGN(64) z[16] = {
		SPH_C32(0x243F6A88), SPH_C32(0x85A308D3), SPH_C32(0x13198A2E), SPH_C32(0x03707344),
		SPH_C32(0xA4093822), SPH_C32(0x299F31D0), SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
		SPH_C32(0x452821E6), SPH_C32(0x38D01377), SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
		SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD), SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
	};
	uint32_t _ALIGN(64) h[22];

	sph_blake256_context ctx;

	sph_blake256_set_rounds(8);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, endiandata, 64);

	h[ 0] = ctx.H[0];	h[ 1] = ctx.H[1];
	h[ 2] = ctx.H[2];	h[21] = ctx.H[3];
	h[ 4] = ctx.H[4];	h[20] = ctx.H[5];
	h[19] = ctx.H[6];	h[16] = ctx.H[7];

	uint32_t tmp = h[20];
	h[20] = h[19];
	h[19] = h[16];
	h[16] = penddata[ 0];
	h[17] = penddata[ 1];
	h[18] = penddata[ 2];
	h[12] = z[ 4] ^ 640;
	h[ 8] = z[ 0];

	h[ 0] += (h[16] ^ z[ 1]) + h[ 4];
	h[12]  = SPH_ROTR32(h[12] ^ h[0],16);
	h[ 8] += h[12];
	h[ 4]  = SPH_ROTR32(h[ 4] ^ h[ 8], 12);
	h[ 0] += (h[17] ^ z[ 0]) + h[ 4];
	h[12]  = SPH_ROTR32(h[12] ^ h[0],8);
	h[ 8] += h[12];
	h[ 4]  = SPH_ROTR32(h[ 4] ^ h[ 8], 7);

	h[1] += (h[18] ^ z[ 3]) + tmp;

	h[13] = SPH_ROTR32(z[ 5] ^ 640 ^ h[1],16);
	h[ 5] = ROTR32(tmp ^ (z[ 1] + h[13]), 12);

	h[ 1] += h[ 5];
	h[ 2] += (0x80000000UL ^ z[ 5]) + h[20];

	h[14]  = SPH_ROTR32(z[ 6] ^ h[2], 16);
	h[ 6]  = z[ 2] + h[14];
	h[ 6]  = SPH_ROTR32(h[20] ^ h[ 6], 12);

	h[21] += z[ 7] + h[19];
	h[ 0] += z[ 9];

	h[ 2] += z[ 4] + h[ 6];

	h[ 9] = z[ 1] + h[13];
	h[10] = z[ 2] + h[14];

	h[14] = SPH_ROTR32(h[14] ^ h[2],8); //0x0321
	h[10]+=h[14];

	h[ 6] = SPH_ROTR32(h[ 6] ^ h[10],7);
	h[15] = SPH_ROTR32(z[ 7] ^ h[21],16);

	h[11] = z[ 3] + h[15];
	h[ 7] = SPH_ROTR32(h[19] ^ h[11], 12);
	h[ 3] = h[21] + h[ 7] + z[ 6];

	h[15] = SPH_ROTR32(h[15] ^ h[ 3],8);
	h[11]+= h[15];
	h[ 7] = ROTR32(h[ 7] ^ h[11],7);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(d_data), h, 21*sizeof(uint32_t), 0, hipMemcpyHostToDevice, streams[thr_id]);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_vanilla(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, const int8_t blakerounds)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce  = pdata[19];
	const uint32_t targetHigh   = ptarget[6];
	int dev_id = device_map[thr_id];

	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 30 : 24;
	if (device_sm[dev_id] < 350) intensity = 22;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id]) {
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		hipStreamCreate(&streams[thr_id]);
		init[thr_id] = true;
	}

	uint32_t _ALIGN(64) endiandata[20];

	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);

	hipMemsetAsync(d_resNonce[thr_id], 0xff, sizeof(uint32_t),streams[thr_id]);

	vanilla_cpu_setBlock_16(thr_id,endiandata,&pdata[16]);

	const dim3 grid((throughput + (NPT*TPB)-1)/(NPT*TPB));
	const dim3 block(TPB);
	int rc = 0;

	do {
		vanilla_gpu_hash_16_8<<<grid,block, 0, streams[thr_id]>>>(throughput, pdata[19], d_resNonce[thr_id], targetHigh);
		hipMemcpyAsync(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost,streams[thr_id]);
		*hashes_done = pdata[19] - first_nonce + throughput;
		hipStreamSynchronize(streams[thr_id]);

		if (h_resNonce[thr_id][0] != UINT32_MAX){
			uint32_t vhashcpu[8];
			uint32_t Htarg = (uint32_t)targetHigh;

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], h_resNonce[thr_id][0]);
			vanillahash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[0] = h_resNonce[thr_id][0];
				work_set_target_ratio(work, vhashcpu);
#if NBN > 1
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					work->nonces[1] = h_resNonce[thr_id][1];
					be32enc(&endiandata[19], h_resNonce[thr_id][1]);
					vanillahash(vhashcpu, endiandata, blakerounds);
					if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhashcpu);
						xchg(work->nonces[0], work->nonces[1]);
					}
					work->valid_nonces = 2;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#endif
				return work->valid_nonces;
			}
			else if (vhashcpu[6] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	MyStreamSynchronize(NULL, 0, dev_id);
	return rc;
}

// cleanup
extern "C" void free_vanilla(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
