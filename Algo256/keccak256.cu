#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void keccak256_setBlock_80(void *pdata,const void *ptarget);
extern uint32_t keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_keccak_context ctx_keccak;

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21); // 256*256*8*4
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], throughput * 64));
		keccak256_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	keccak256_setBlock_80((void*)endiandata, ptarget);
	do {
		int order = 0;

		*hashes_done = pdata[19] - first_nonce + throughput;

		uint32_t foundNonce = keccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX && bench_algo < 0)
		{
			uint32_t _ALIGN(64) vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			keccak256_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				work_set_target_ratio(work, vhash64);
				pdata[19] = foundNonce;
				return 1;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_keccak256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	keccak256_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
