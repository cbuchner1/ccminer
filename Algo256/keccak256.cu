#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_setBlock_80(void *pdata,const void *ptarget);
extern uint32_t keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_keccak_context ctx_keccak;

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 21); // 256*256*8*4
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0005;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], throughput * 64));
		keccak256_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	keccak256_setBlock_80((void*)endiandata, ptarget);
	do {
		int order = 0;

		*hashes_done = pdata[19] - first_nonce + throughput;

		uint32_t foundNonce = keccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			keccak256_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				pdata[19] = foundNonce;
				return 1;
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		if ((uint64_t) pdata[19] + throughput > max_nonce) {
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	return 0;
}
