#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"

#include "miner.h"
}

#include "cuda_helper.h"

// SM5+ cuda
extern void keccak256_cpu_init(int thr_id);
extern void keccak256_cpu_free(int thr_id);
extern void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t* resNonces, const uint2 highTarget);
extern void keccak256_setBlock_80(uint64_t *endiandata);
extern void keccak256_setOutput(int thr_id);

// compat
extern void keccak256_sm3_init(int thr_id, uint32_t threads);
extern void keccak256_sm3_free(int thr_id);
extern void keccak256_sm3_setBlock_80(void *pdata, const void *ptarget);
extern void keccak256_sm3_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t* resNonces, int order);

// CPU Hash
extern "C" void keccak256_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_keccak_context ctx_keccak;

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_keccak256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	uint32_t throughput;
	uint32_t intensity = 23;
	if(!use_compat_kernels[thr_id]) {
		if (strstr(device_name[dev_id], "GTX 1070")) intensity = 25;
		if (strstr(device_name[dev_id], "GTX 1080")) intensity = 26;
	}
	throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		if(!use_compat_kernels[thr_id]) {
			keccak256_cpu_init(thr_id);
		} else {
			// really useful ?
			keccak256_sm3_init(thr_id, throughput);
		}

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	const uint2 highTarget = make_uint2(ptarget[6], ptarget[7]);
	if(use_compat_kernels[thr_id])
		keccak256_sm3_setBlock_80((void*)endiandata, ptarget);
	else {
		keccak256_setBlock_80((uint64_t*)endiandata);
		keccak256_setOutput(thr_id);
	}

	do {
		int order = 0;

		*hashes_done = pdata[19] - first_nonce + throughput;

		if(use_compat_kernels[thr_id])
			keccak256_sm3_hash_80(thr_id, throughput, pdata[19], work->nonces, order++);
		else {
			keccak256_cpu_hash_80(thr_id, throughput, pdata[19], work->nonces, highTarget);
		}

		if (work->nonces[0] != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			keccak256_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (!use_compat_kernels[thr_id] && work->nonces[1] != UINT32_MAX) {
					be32enc(&endiandata[19], work->nonces[1]);
					keccak256_hash(vhash, endiandata);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						work->valid_nonces++;
						bn_set_target_ratio(work, vhash, 1);
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1;
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				//keccak256_setOutput(thr_id);
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_keccak256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	if(!use_compat_kernels[thr_id])
		keccak256_cpu_free(thr_id);
	else {
		keccak256_sm3_free(thr_id);
	}

	hipDeviceSynchronize();
	init[thr_id] = false;
}
