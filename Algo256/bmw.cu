#include "hip/hip_runtime.h"
/**
 * bmw-256 MDT
 * tpruvot - 2015
 */
extern "C" {
#include "sph/sph_bmw.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint32_t *d_hash[MAX_GPUS];

extern void bmw256_midstate_init(int thr_id, uint32_t threads);
extern void bmw256_midstate_free(int thr_id);
extern void bmw256_setBlock_80(int thr_id, void *pdata);
extern void bmw256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int swap);

extern uint32_t cuda_check_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash);

// CPU Hash
extern "C" void bmw_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_bmw256_context ctx;

	sph_bmw256_init(&ctx);
	sph_bmw256(&ctx, input, 80);
	sph_bmw256_close(&ctx, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_bmw(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x0005;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
                        // reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_check_cpu_init(thr_id, throughput);
		bmw256_midstate_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	hipGetLastError();
	bmw256_setBlock_80(thr_id, (void*)endiandata);

	cuda_check_cpu_setTarget(ptarget);

	do {
		bmw256_cpu_hash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id], 1);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			bmw_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					bmw_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_bmw(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	bmw256_midstate_free(thr_id);
	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
