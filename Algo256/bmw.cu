#include "hip/hip_runtime.h"
/**
 * bmw-256 MDT
 * tpruvot - 2015
 */
extern "C" {
#include "sph/sph_bmw.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint32_t *d_hash[MAX_GPUS];

extern void bmw256_midstate_init(int thr_id, uint32_t threads);
extern void bmw256_midstate_free(int thr_id);
extern void bmw256_setBlock_80(int thr_id, void *pdata);
extern void bmw256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int swap);

extern uint32_t cuda_check_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash);

// CPU Hash
extern "C" void bmw_hash(void *state, const void *input)
{
	uint32_t _ALIGN(64) hash[16];
	sph_bmw256_context ctx;

	sph_bmw256_init(&ctx);
	sph_bmw256(&ctx, input, 80);
	sph_bmw256_close(&ctx, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

extern "C" int scanhash_bmw(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	bool swapnonce = true;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x0005;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		cuda_check_cpu_init(thr_id, throughput);
		bmw256_midstate_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	hipGetLastError();
	bmw256_setBlock_80(thr_id, (void*)endiandata);

	cuda_check_cpu_setTarget(ptarget);

	do {
		bmw256_cpu_hash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id], (int) swapnonce);
		uint32_t foundNonce = cuda_check_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];
			endiandata[19] = swab32_if(foundNonce, swapnonce);
			bmw_hash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				*hashes_done = foundNonce - first_nonce + 1;
				pdata[19] = swab32_if(foundNonce,!swapnonce);
				work_set_target_ratio(work, vhash64);
				return 1;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_bmw(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	bmw256_midstate_free(thr_id);
	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
