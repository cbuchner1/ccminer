#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>


extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];
#include "cuda_helper.h"

static __constant__ uint64_t SKEIN_IV512_256[8] = {
	0xCCD044A12FDB3E13, 0xE83590301A79A9EB,
	0x55AEA0614F816E6F, 0x2A2767A4AE9B94DB,
	0xEC06025E74DD7683, 0xE7A436CDC4746251,
	0xC36FBAF9393AD185, 0x3EEDBA1833EDFC13
};

static __constant__ uint2 vSKEIN_IV512_256[8] = {
	{0x2FDB3E13, 0xCCD044A1 }, 
    {0x1A79A9EB, 0xE8359030 },
	{0x4F816E6F, 0x55AEA061 }, 
    {0xAE9B94DB, 0x2A2767A4 },
	{0x74DD7683, 0xEC06025E }, 
    {0xC4746251, 0xE7A436CD },
	{0x393AD185, 0xC36FBAF9 }, 
    {0x33EDFC13, 0x3EEDBA18 }
};

static __constant__ int ROT256[8][4] =
{
	46,36, 19, 37,
	33,27, 14, 42,
	17,49, 36, 39,
	44, 9, 54, 56,
	39,30, 34, 24,
	13,50, 10, 17,
	25,29, 39, 43,
	8, 35, 56, 22,
};

static __constant__ uint2 skein_ks_parity = { 0xA9FC1A22,0x1BD11BDA};
static __constant__ uint64_t skein_ks_parity64 = 0x1BD11BDAA9FC1A22;
static __constant__ uint2 t12[6] =
{ 
{ 0x20, 0 },
{ 0, 0xf0000000 },
{ 0x20, 0xf0000000 },
{ 0x08, 0 },
{ 0, 0xff000000 },
{ 0x08, 0xff000000 }
};

static __constant__ uint64_t t12_30[6] =
{ 0x20,
0xf000000000000000,
0xf000000000000020,
0x08,
0xff00000000000000,
0xff00000000000008
};


static __forceinline__ __device__ void Round512v35(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7, int ROT)
{
	p0 += p1; p1 = ROL2(p1, ROT256[ROT][0]);  p1 ^= p0;
	p2 += p3; p3 = ROL2(p3, ROT256[ROT][1]);  p3 ^= p2;
	p4 += p5; p5 = ROL2(p5, ROT256[ROT][2]);  p5 ^= p4;
	p6 += p7; p7 = ROL2(p7, ROT256[ROT][3]);  p7 ^= p6;
}


static __forceinline__ __device__ void Round_8_512v35(uint2 *ks,uint2 *ts,uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, 
                                                                          uint2 &p5, uint2 &p6, uint2 &p7, int R)
{
     	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 0);
	    Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 1);
		Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 2);   
		Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 3);   
		p0 += ks[((R)+0) % 9];   /* inject the key schedule value */  
	    p1 += ks[((R)+1) % 9];                                        
		p2 += ks[((R)+2) % 9];                                        
		p3 += ks[((R)+3) % 9];                                        
		p4 += ks[((R)+4) % 9];                                        
		p5 += ks[((R)+5) % 9] + ts[((R)+0) % 3];                      
		p6 += ks[((R)+6) % 9] + ts[((R)+1) % 3];                      
		p7 += ks[((R)+7) % 9] + make_uint2((R),0);                            
		Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 4);   
		Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 5);   
		Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 6);   
		Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 7);   
		p0 += ks[((R)+1) % 9];   /* inject the key schedule value */  
		p1 += ks[((R)+2) % 9];                                        
		p2 += ks[((R)+3) % 9];                                        
		p3 += ks[((R)+4) % 9];                                        
		p4 += ks[((R)+5) % 9];                                        
		p5 += ks[((R)+6) % 9] + ts[((R)+1) % 3];                      
		p6 += ks[((R)+7) % 9] + ts[((R)+2) % 3];                      
		p7 += ks[((R)+8) % 9] + make_uint2((R)+1, 0);  
}


static __forceinline__ __device__ void Round512v30(uint64_t &p0, uint64_t &p1, uint64_t &p2, uint64_t &p3, uint64_t &p4, uint64_t &p5, uint64_t &p6, uint64_t &p7, int ROT)
{
	p0 += p1; p1 = ROTL64(p1, ROT256[ROT][0]);  p1 ^= p0;
	p2 += p3; p3 = ROTL64(p3, ROT256[ROT][1]);  p3 ^= p2;
	p4 += p5; p5 = ROTL64(p5, ROT256[ROT][2]);  p5 ^= p4;
	p6 += p7; p7 = ROTL64(p7, ROT256[ROT][3]);  p7 ^= p6;
}

static __forceinline__ __device__ void Round_8_512v30(uint64_t *ks, uint64_t *ts, uint64_t &p0, uint64_t &p1, uint64_t &p2, uint64_t &p3, uint64_t &p4,
	uint64_t &p5, uint64_t &p6, uint64_t &p7, int R)
{
	Round512v30(p0, p1, p2, p3, p4, p5, p6, p7, 0);
	Round512v30(p2, p1, p4, p7, p6, p5, p0, p3, 1);
	Round512v30(p4, p1, p6, p3, p0, p5, p2, p7, 2);
	Round512v30(p6, p1, p0, p7, p2, p5, p4, p3, 3);
	p0 += ks[((R)+0) % 9];   /* inject the key schedule value */
	p1 += ks[((R)+1) % 9];
	p2 += ks[((R)+2) % 9];
	p3 += ks[((R)+3) % 9];
	p4 += ks[((R)+4) % 9];
	p5 += ks[((R)+5) % 9] + ts[((R)+0) % 3];
	p6 += ks[((R)+6) % 9] + ts[((R)+1) % 3];
	p7 += ks[((R)+7) % 9] + R;
	Round512v30(p0, p1, p2, p3, p4, p5, p6, p7, 4);
	Round512v30(p2, p1, p4, p7, p6, p5, p0, p3, 5);
	Round512v30(p4, p1, p6, p3, p0, p5, p2, p7, 6);
	Round512v30(p6, p1, p0, p7, p2, p5, p4, p3, 7);
	p0 += ks[((R)+1) % 9];   /* inject the key schedule value */
	p1 += ks[((R)+2) % 9];
	p2 += ks[((R)+3) % 9];
	p3 += ks[((R)+4) % 9];
	p4 += ks[((R)+5) % 9];
	p5 += ks[((R)+6) % 9] + ts[((R)+1) % 3];
	p6 += ks[((R)+7) % 9] + ts[((R)+2) % 3];
	p7 += ks[((R)+8) % 9] + (R)+1;
}



__global__ void __launch_bounds__(256,3) skein256_gpu_hash_32(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{

		uint2 h[9];
		uint2 t[3];
        uint2 dt0,dt1,dt2,dt3;
		uint2 p0, p1, p2, p3, p4, p5, p6, p7;
        h[8] = skein_ks_parity;
		for (int i = 0; i<8; i++) {
			h[i] = vSKEIN_IV512_256[i];
			h[8] ^= h[i];}
		    
			t[0]=t12[0];
			t[1]=t12[1];
			t[2]=t12[2];

         
		LOHI(dt0.x,dt0.y,outputHash[thread]);
		LOHI(dt1.x,dt1.y,outputHash[threads+thread]);
		LOHI(dt2.x,dt2.y,outputHash[2*threads+thread]);
		LOHI(dt3.x,dt3.y,outputHash[3*threads+thread]);

		p0 = h[0] + dt0;
		p1 = h[1] + dt1;
		p2 = h[2] + dt2;
		p3 = h[3] + dt3;
		p4 = h[4];
		p5 = h[5] + t[0];
		p6 = h[6] + t[1];
		p7 = h[7];

        #pragma unroll 
		for (int i = 1; i<19; i+=2) {Round_8_512v35(h,t,p0,p1,p2,p3,p4,p5,p6,p7,i);}
        p0 ^= dt0;
        p1 ^= dt1;
        p2 ^= dt2;
        p3 ^= dt3;

		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity;
        #pragma unroll 8
		for (int i = 0; i<8; i++) { h[8] ^= h[i]; }
		
		t[0] = t12[3];
		t[1] = t12[4];
		t[2] = t12[5];
		p5 += t[0];  //p5 already equal h[5] 
		p6 += t[1];
        #pragma unroll 
		for (int i = 1; i<19; i+=2) { Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i); }

		outputHash[thread]           = devectorize(p0);
		outputHash[threads+thread]   = devectorize(p1);
		outputHash[2*threads+thread] = devectorize(p2);
		outputHash[3*threads+thread] = devectorize(p3);

	} //thread
}

__global__ void __launch_bounds__(256, 3) skein256_gpu_hash_32_v30(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{

		uint64_t h[9];
		uint64_t t[3];
		uint64_t dt0, dt1, dt2, dt3;
		uint64_t p0, p1, p2, p3, p4, p5, p6, p7;
		h[8] = skein_ks_parity64;
		for (int i = 0; i<8; i++) {
			h[i] = SKEIN_IV512_256[i];
			h[8] ^= h[i];
		}

		t[0] = devectorize(t12[0]);
		t[1] = devectorize(t12[1]);
		t[2] = devectorize(t12[2]);

		dt0 = outputHash[thread];
		dt1 = outputHash[threads+thread];
		dt2 = outputHash[2*threads+thread];
		dt3 = outputHash[3*threads+thread];
		p0 = h[0] + dt0;
		p1 = h[1] + dt1;
		p2 = h[2] + dt2;
		p3 = h[3] + dt3;
		p4 = h[4];
		p5 = h[5] + t[0];
		p6 = h[6] + t[1];
		p7 = h[7];

#pragma unroll 
		for (int i = 1; i<19; i += 2) { Round_8_512v30(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i); }
		p0 ^= dt0;
		p1 ^= dt1;
		p2 ^= dt2;
		p3 ^= dt3;

		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity64;
#pragma unroll 8
		for (int i = 0; i<8; i++) { h[8] ^= h[i]; }

		t[0] = t12_30[3];
		t[1] = t12_30[4];
		t[2] = t12_30[5];
		p5 += t[0];  //p5 already equal h[5] 
		p6 += t[1];
#pragma unroll 
		for (int i = 1; i<19; i += 2) { Round_8_512v30(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i); }

		outputHash[thread] = p0;
		outputHash[threads + thread] = p1;
		outputHash[2 * threads + thread] = p2;
		outputHash[3 * threads + thread] = p3;


	} //thread
}

   
void skein256_cpu_init(int thr_id, int threads)
{
//empty //mepty //mpety //mptey// mptye	
} 


__host__ void skein256_cpu_hash_32(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;
	if (compute_version[thr_id] >= 35) {
	skein256_gpu_hash_32 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	} else {
	skein256_gpu_hash_32_v30 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	}
	MyStreamSynchronize(NULL, order, thr_id);

}

