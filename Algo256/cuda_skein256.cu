#include "hip/hip_runtime.h"
#include <memory.h>

#include "cuda_helper.h"

static __forceinline__ __device__
void Round512v35(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	const int ROT0, const int ROT1, const int ROT2, const int ROT3)
{
	p0 += p1; p1 = ROL2(p1, ROT0) ^ p0;
	p2 += p3; p3 = ROL2(p3, ROT1) ^ p2;
	p4 += p5; p5 = ROL2(p5, ROT2) ^ p4;
	p6 += p7; p7 = ROL2(p7, ROT3) ^ p6;
}

__forceinline__ __device__
void Round_8_512v35(const uint2 *const __restrict__ ks, const uint2 *const __restrict__ ts,
	uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7, int R)
{
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 46, 36, 19, 37);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 33, 27, 14, 42);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 17, 49, 36, 39);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 44,  9, 54, 56);

	p0 += ks[(R+0) % 9];
	p1 += ks[(R+1) % 9];
	p2 += ks[(R+2) % 9];
	p3 += ks[(R+3) % 9];
	p4 += ks[(R+4) % 9];
	p5 += ks[(R+5) % 9] + ts[(R+0) % 3];
	p6 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p7 += ks[(R+7) % 9] + make_uint2(R, 0);

	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 39, 30, 34, 24);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 13, 50, 10, 17);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 25, 29, 39, 43);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 8,  35, 56, 22);

	p0 += ks[(R+1) % 9];
	p1 += ks[(R+2) % 9];
	p2 += ks[(R+3) % 9];
	p3 += ks[(R+4) % 9];
	p4 += ks[(R+5) % 9];
	p5 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p6 += ks[(R+7) % 9] + ts[(R+2) % 3];
	p7 += ks[(R+8) % 9] + make_uint2(R+1, 0);
}

__forceinline__ __device__
void Round_8_512v35_final(const uint2 *const __restrict__ ks, const uint2 *const __restrict__ ts,
	uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7)
{
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 46, 36, 19, 37);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 33, 27, 14, 42);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 17, 49, 36, 39);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 44, 9, 54, 56);

	p0 += ks[8];
	p1 += ks[0];
	p2 += ks[1];
	p3 += ks[2];
	p4 += ks[3];
	p5 += ks[4] + ts[2];
	p6 += ks[5] + ts[0];
	p7 += ks[6] + make_uint2(17, 0);

	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 39, 30, 34, 24);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 13, 50, 10, 17);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 25, 29, 39, 43);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 8,  35, 56, 22);

	p0 += ks[0];
	p1 += ks[1];
	p2 += ks[2];
	p3 += ks[3];
}

__global__ __launch_bounds__(256,3)
void skein256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		const uint2 skein_ks_parity = { 0xA9FC1A22, 0x1BD11BDA };
		const uint2 t12[6] = {
			{ 0x20, 0 },
			{ 0,    0xf0000000 },
			{ 0x20, 0xf0000000 },
			{ 0x08, 0 },
			{ 0,    0xff000000 },
			{ 0x08, 0xff000000 }
		};

		uint2 h[9] = {
			{ 0x2FDB3E13, 0xCCD044A1 },
			{ 0x1A79A9EB, 0xE8359030 },
			{ 0x4F816E6F, 0x55AEA061 },
			{ 0xAE9B94DB, 0x2A2767A4 },
			{ 0x74DD7683, 0xEC06025E },
			{ 0xC4746251, 0xE7A436CD },
			{ 0x393AD185, 0xC36FBAF9 },
			{ 0x33EDFC13, 0x3EEDBA18 },
			{ 0xC73A4E2A, 0xB69D3CFC }
		};
		uint2 dt0,dt1,dt2,dt3;
		uint2 p0, p1, p2, p3, p4, p5, p6, p7;

		LOHI(dt0.x,dt0.y,outputHash[thread]);
		LOHI(dt1.x,dt1.y,outputHash[threads+thread]);
		LOHI(dt2.x,dt2.y,outputHash[2*threads+thread]);
		LOHI(dt3.x,dt3.y,outputHash[3*threads+thread]);

		p0 = h[0] + dt0;
		p1 = h[1] + dt1;
		p2 = h[2] + dt2;
		p3 = h[3] + dt3;
		p4 = h[4];
		p5 = h[5] + t12[0];
		p6 = h[6] + t12[1];
		p7 = h[7];

		// forced unroll required
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 1);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 3);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 5);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 7);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 9);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 11);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 13);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 15);
		Round_8_512v35(h, t12, p0, p1, p2, p3, p4, p5, p6, p7, 17);

		p0 ^= dt0;
		p1 ^= dt1;
		p2 ^= dt2;
		p3 ^= dt3;

		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity ^ h[0] ^ h[1] ^ h[2] ^ h[3] ^ h[4] ^ h[5] ^ h[6] ^ h[7];

		const uint2 *t = t12+3;
		p5 += t12[3];  //p5 already equal h[5]
		p6 += t12[4];

		// forced unroll
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 1);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 3);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 5);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 7);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 9);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 11);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 13);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 15);
		Round_8_512v35_final(h, t, p0, p1, p2, p3, p4, p5, p6, p7);

		outputHash[thread]           = devectorize(p0);
		outputHash[threads+thread]   = devectorize(p1);
		outputHash[2*threads+thread] = devectorize(p2);
		outputHash[3*threads+thread] = devectorize(p3);
	}
}

static __forceinline__ __device__
void Round512v30(uint64_t &p0, uint64_t &p1, uint64_t &p2, uint64_t &p3,
	uint64_t &p4, uint64_t &p5, uint64_t &p6, uint64_t &p7,
	const int ROT0, const int ROT1, const int ROT2, const int ROT3)
{
	p0 += p1; p1 = ROTL64(p1, ROT0) ^ p0;
	p2 += p3; p3 = ROTL64(p3, ROT1) ^ p2;
	p4 += p5; p5 = ROTL64(p5, ROT2) ^ p4;
	p6 += p7; p7 = ROTL64(p7, ROT3) ^ p6;
}

static __forceinline__ __device__
void Round_8_512v30(uint64_t *ks, uint64_t *ts,
	uint64_t &p0, uint64_t &p1, uint64_t &p2, uint64_t &p3,
	uint64_t &p4, uint64_t &p5, uint64_t &p6, uint64_t &p7, int R)
{
	Round512v30(p0, p1, p2, p3, p4, p5, p6, p7, 46, 36, 19, 37);
	Round512v30(p2, p1, p4, p7, p6, p5, p0, p3, 33, 27, 14, 42);
	Round512v30(p4, p1, p6, p3, p0, p5, p2, p7, 17, 49, 36, 39);
	Round512v30(p6, p1, p0, p7, p2, p5, p4, p3, 44,  9, 54, 56);

	p0 += ks[(R+0) % 9];
	p1 += ks[(R+1) % 9];
	p2 += ks[(R+2) % 9];
	p3 += ks[(R+3) % 9];
	p4 += ks[(R+4) % 9];
	p5 += ks[(R+5) % 9] + ts[(R+0) % 3];
	p6 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p7 += ks[(R+7) % 9] + R;

	Round512v30(p0, p1, p2, p3, p4, p5, p6, p7, 39, 30, 34, 24);
	Round512v30(p2, p1, p4, p7, p6, p5, p0, p3, 13, 50, 10, 17);
	Round512v30(p4, p1, p6, p3, p0, p5, p2, p7, 25, 29, 39, 43);
	Round512v30(p6, p1, p0, p7, p2, p5, p4, p3, 8,  35, 56, 22);

	p0 += ks[(R+1) % 9];
	p1 += ks[(R+2) % 9];
	p2 += ks[(R+3) % 9];
	p3 += ks[(R+4) % 9];
	p4 += ks[(R+5) % 9];
	p5 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p6 += ks[(R+7) % 9] + ts[(R+2) % 3];
	p7 += ks[(R+8) % 9] + R+1;
}

#define skein_ks_parity64 0x1BD11BDAA9FC1A22ull
#include <stdio.h>
__global__  __launch_bounds__(256, 3)
void skein256_gpu_hash_32_v30(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t h[12] = { // SKEIN_IV512_256
			0xCCD044A12FDB3E13, 0xE83590301A79A9EB,
			0x55AEA0614F816E6F, 0x2A2767A4AE9B94DB,
			0xEC06025E74DD7683, 0xE7A436CDC4746251,
			0xC36FBAF9393AD185, 0x3EEDBA1833EDFC13,
			0xb69d3cfcc73a4e2a, // skein_ks_parity64 ^ h[0..7]
			0x20, 0xf000000000000000, 0xf000000000000020 // t0..2
		};
		uint64_t dt0 = outputHash[thread];
		uint64_t dt1 = outputHash[threads   + thread];
		uint64_t dt2 = outputHash[threads*2 + thread];
		uint64_t dt3 = outputHash[threads*3 + thread];

		uint64_t *t = &h[9];
		uint64_t p0 = h[0] + dt0;
		uint64_t p1 = h[1] + dt1;
		uint64_t p2 = h[2] + dt2;
		uint64_t p3 = h[3] + dt3;
		uint64_t p4 = h[4];
		uint64_t p5 = h[5] + t[0];
		uint64_t p6 = h[6] + t[1];
		uint64_t p7 = h[7];

		#pragma unroll 9
		for (int i = 1; i<19; i += 2) {
			Round_8_512v30(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i);
		}

		p0 ^= dt0;
		p1 ^= dt1;
		p2 ^= dt2;
		p3 ^= dt3;

		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity64;

		#pragma unroll 8
		for (int i = 0; i<8; i++) {
			h[8] ^= h[i];
		}

		t[0] = 0x08;
		t[1] = 0xff00000000000000;
		t[2] = 0xff00000000000008;

		p5 += t[0];  //p5 already equal h[5]
		p6 += t[1];

		#pragma unroll 9
		for (int i = 1; i<19; i += 2) {
			Round_8_512v30(h, t, p0, p1, p2, p3, p4, p5, p6, p7, i);
		}

		outputHash[thread] = p0;
		outputHash[threads   + thread] = p1;
		outputHash[threads*2 + thread] = p2;
		outputHash[threads*3 + thread] = p3;
	} //thread
}

__host__
void skein256_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
}

__host__
void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = 256;
	int dev_id = device_map[thr_id];

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	// only 1kH/s perf change between kernels on a 960...
	if (device_sm[dev_id] > 300 && cuda_arch[dev_id] > 300)
		skein256_gpu_hash_32<<<grid, block>>>(threads, startNounce, d_outputHash);
	else
		skein256_gpu_hash_32_v30<<<grid, block>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

