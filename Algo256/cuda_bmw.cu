#include "hip/hip_runtime.h"
/**
 * BMW-256 CUDA Implementation - tpruvot 2015
 *
 * Not optimal but close to the sph version and easier to adapt.
 */

#include <stdio.h>
#include <memory.h>

#define SPH_64 1
#define USE_MIDSTATE

extern "C" {
#include "sph/sph_bmw.h"
}

#include "cuda_helper.h"

__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

#ifndef USE_MIDSTATE
__constant__ static sph_u32 IV256[16] = {
	0x40414243, 0x44454647, 0x48494A4B, 0x4C4D4E4F,
	0x50515253, 0x54555657, 0x58595A5B, 0x5C5D5E5F,
	0x60616263, 0x64656667, 0x68696A6B, 0x6C6D6E6F,
	0x70717273, 0x74757677, 0x78797A7B, 0x7C7D7E7F
};
#endif

__constant__ static sph_u32 final_s[16] = {
	0xaaaaaaa0, 0xaaaaaaa1, 0xaaaaaaa2, 0xaaaaaaa3,
	0xaaaaaaa4, 0xaaaaaaa5, 0xaaaaaaa6, 0xaaaaaaa7,
	0xaaaaaaa8, 0xaaaaaaa9, 0xaaaaaaaa, 0xaaaaaaab,
	0xaaaaaaac, 0xaaaaaaad, 0xaaaaaaae, 0xaaaaaaaf
};

static sph_bmw_small_context* d_midstate[MAX_GPUS];

#define I16_16    0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15
#define I16_17    1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16
#define I16_18    2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17
#define I16_19    3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18
#define I16_20    4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19
#define I16_21    5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20
#define I16_22    6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21
#define I16_23    7,  8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22
#define I16_24    8,  9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23
#define I16_25    9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24
#define I16_26   10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25
#define I16_27   11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26
#define I16_28   12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27
#define I16_29   13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28
#define I16_30   14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29
#define I16_31   15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30

//#define M16_16    0,  1,  3,  4,  7, 10, 11
//#define M16_17    1,  2,  4,  5,  8, 11, 12
#define M16_18    2,  3,  5,  6,  9, 12, 13
#define M16_19    3,  4,  6,  7, 10, 13, 14
#define M16_20    4,  5,  7,  8, 11, 14, 15
#define M16_21    5,  6,  8,  9, 12, 15, 16
#define M16_22    6,  7,  9, 10, 13,  0,  1
#define M16_23    7,  8, 10, 11, 14,  1,  2
#define M16_24    8,  9, 11, 12, 15,  2,  3
#define M16_25    9, 10, 12, 13,  0,  3,  4
#define M16_26   10, 11, 13, 14,  1,  4,  5
#define M16_27   11, 12, 14, 15,  2,  5,  6
#define M16_28   12, 13, 15, 16,  3,  6,  7
#define M16_29   13, 14,  0,  1,  4,  7,  8
#define M16_30   14, 15,  1,  2,  5,  8,  9
#define M16_31   15, 16,  2,  3,  6,  9, 10

#define ss0(x)    (((x) >> 1) ^ ((x) << 3) ^ ROTL32(x,  4) ^ ROTL32(x, 19))
#define ss1(x)    (((x) >> 1) ^ ((x) << 2) ^ ROTL32(x,  8) ^ ROTL32(x, 23))
#define ss2(x)    (((x) >> 2) ^ ((x) << 1) ^ ROTL32(x, 12) ^ ROTL32(x, 25))
#define ss3(x)    (((x) >> 2) ^ ((x) << 2) ^ ROTL32(x, 15) ^ ROTL32(x, 29))
#define ss4(x)    (((x) >> 1) ^ (x))
#define ss5(x)    (((x) >> 2) ^ (x))

#define rs1(x)    ROTL32(x,  3)
#define rs2(x)    ROTL32(x,  7)
#define rs3(x)    ROTL32(x, 13)
#define rs4(x)    ROTL32(x, 16)
#define rs5(x)    ROTL32(x, 19)
#define rs6(x)    ROTL32(x, 23)
#define rs7(x)    ROTL32(x, 27)

#define MAKE_W(tt, i0, op01, i1, op12, i2, op23, i3, op34, i4) \
	tt((data[i0] ^ h[i0]) op01 (data[i1] ^ h[i1]) op12 (data[i2] ^ h[i2]) op23 (data[i3] ^ h[i3]) op34 (data[i4] ^ h[i4]))
//#define Ws0    MAKE_W(SPH_T32,  5, -,  7, +, 10, +, 13, +, 14)
//#define Ws1    MAKE_W(SPH_T32,  6, -,  8, +, 11, +, 14, -, 15)
//#define Ws2    MAKE_W(SPH_T32,  0, +,  7, +,  9, -, 12, +, 15)
//#define Ws3    MAKE_W(SPH_T32,  0, -,  1, +,  8, -, 10, +, 13)
//#define Ws4    MAKE_W(SPH_T32,  1, +,  2, +,  9, -, 11, -, 14)
//#define Ws5    MAKE_W(SPH_T32,  3, -,  2, +, 10, -, 12, +, 15)
//#define Ws6    MAKE_W(SPH_T32,  4, -,  0, -,  3, -, 11, +, 13)
//#define Ws7    MAKE_W(SPH_T32,  1, -,  4, -,  5, -, 12, -, 14)
//#define Ws8    MAKE_W(SPH_T32,  2, -,  5, -,  6, +, 13, -, 15)
//#define Ws9    MAKE_W(SPH_T32,  0, -,  3, +,  6, -,  7, +, 14)
//#define Ws10   MAKE_W(SPH_T32,  8, -,  1, -,  4, -,  7, +, 15)
//#define Ws11   MAKE_W(SPH_T32,  8, -,  0, -,  2, -,  5, +,  9)
//#define Ws12   MAKE_W(SPH_T32,  1, +,  3, -,  6, -,  9, +, 10)
//#define Ws13   MAKE_W(SPH_T32,  2, +,  4, +,  7, +, 10, +, 11)
//#define Ws14   MAKE_W(SPH_T32,  3, -,  5, +,  8, -, 11, -, 12)
//#define Ws15   MAKE_W(SPH_T32, 12, -,  4, -,  6, -,  9, +, 13)

__device__
static void gpu_compress_small(const sph_u32 *data, const sph_u32 h[16], sph_u32 dh[16])
{
		// FOLD MAKE_Qas;

		sph_u32 dx[16];
		for (int i=0; i<16; i++)
			dx[i] = data[i] ^ h[i];

		sph_u32 qt[32];
		qt[ 0] = dx[ 5] - dx[7] + dx[10] + dx[13] + dx[14]; // Ws0
		qt[ 1] = dx[ 6] - dx[8] + dx[11] + dx[14] - dx[15]; // Ws1
		qt[ 2] = dx[ 0] + dx[7] + dx[ 9] - dx[12] + dx[15]; // Ws2
		qt[ 3] = dx[ 0] - dx[1] + dx[ 8] - dx[10] + dx[13]; // Ws3
		qt[ 4] = dx[ 1] + dx[2] + dx[ 9] - dx[11] - dx[14]; // Ws4;
		qt[ 5] = dx[ 3] - dx[2] + dx[10] - dx[12] + dx[15]; // Ws5;
		qt[ 6] = dx[ 4] - dx[0] - dx[ 3] - dx[11] + dx[13]; // Ws6;
		qt[ 7] = dx[ 1] - dx[4] - dx[ 5] - dx[12] - dx[14]; // Ws7;
		qt[ 8] = dx[ 2] - dx[5] - dx[ 6] + dx[13] - dx[15]; // Ws8;
		qt[ 9] = dx[ 0] - dx[3] + dx[ 6] - dx[ 7] + dx[14]; // Ws9;
		qt[10] = dx[ 8] - dx[1] - dx[ 4] - dx[ 7] + dx[15]; // Ws10;
		qt[11] = dx[ 8] - dx[0] - dx[ 2] - dx[ 5] + dx[ 9]; // Ws11;
		qt[12] = dx[ 1] + dx[3] - dx[ 6] - dx[ 9] + dx[10]; // Ws12;
		qt[13] = dx[ 2] + dx[4] + dx[ 7] + dx[10] + dx[11]; // Ws13;
		qt[14] = dx[ 3] - dx[5] + dx[ 8] - dx[11] - dx[12]; // Ws14;
		qt[15] = dx[12] - dx[4] - dx[ 6] - dx[ 9] + dx[13]; // Ws15;

		qt[ 0] = ss0(qt[ 0]) + h[ 1];
		qt[ 1] = ss1(qt[ 1]) + h[ 2];
		qt[ 2] = ss2(qt[ 2]) + h[ 3];
		qt[ 3] = ss3(qt[ 3]) + h[ 4];
		qt[ 4] = ss4(qt[ 4]) + h[ 5];

		qt[ 5] = ss0(qt[ 5]) + h[ 6];
		qt[ 6] = ss1(qt[ 6]) + h[ 7];
		qt[ 7] = ss2(qt[ 7]) + h[ 8];
		qt[ 8] = ss3(qt[ 8]) + h[ 9];
		qt[ 9] = ss4(qt[ 9]) + h[10];

		qt[10] = ss0(qt[10]) + h[11];
		qt[11] = ss1(qt[11]) + h[12];
		qt[12] = ss2(qt[12]) + h[13];
		qt[13] = ss3(qt[13]) + h[14];
		qt[14] = ss4(qt[14]) + h[15];

		qt[15] = ss0(qt[15]) + h[ 0];

		//MAKE_Qbs;
		#define Ks(j)   ((sph_u32)(0x05555555UL * j))
		#define Qs(j)   (qt[j])

		#define expand1s_in(i16, \
				i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, \
				i0m, i1m, i3m, i4m, i7m, i10m, i11m) \
			(ss1(qt[i0]) + ss2(qt[i1]) + ss3(qt[i2]) + ss0(qt[i3]) + ss1(qt[i4]) + ss2(qt[i5]) + ss3(qt[i6]) + ss0(qt[i7]) \
				+ ss1(qt[i8]) + ss2(qt[i9]) + ss3(qt[i10]) + ss0(qt[i11]) + ss1(qt[i12]) + ss2(qt[i13]) + ss3(qt[i14]) + ss0(qt[i15]) \
				+ ((ROTL32(data[i0m], i1m) + ROTL32(data[i3m], i4m)  - ROTL32(data[i10m], i11m) + Ks(i16)) ^ h[i7m]))

		qt[16] = expand1s_in(16,  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15,  0, 1, 3, 4, 7, 10, 11);
		qt[17] = expand1s_in(17,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15, 16,  1, 2, 4, 5, 8, 11, 12);

		#define expand2s_inner(qf, i16, \
				i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, \
				i0m, i1m, i3m, i4m, i7m, i10m, i11m) \
			(qf(i0) + rs1(qf(i1)) + qf(i2) + rs2(qf(i3)) \
				+ qf(i4) + rs3(qf(i5)) + qf(i6) + rs4(qf(i7)) + qf(i8) + rs5(qf(i9)) + qf(i10) + rs6(qf(i11)) \
				+ qf(i12) + rs7(qf(i13)) + ss4(qf(i14)) + ss5(qf(i15)) \
				+ ((ROTL32(data[i0m], i1m) + ROTL32(data[i3m], i4m) - ROTL32(data[i10m], i11m) + Ks(i16)) ^ h[i7m]))

#ifdef _MSC_VER
		#define LPAR   (
		#define expand2s(i16) \
			expand2s_(Qs, i16, I16_ ## i16, M16_ ## i16)
		#define expand2s_(qf, i16, ix, iy) \
			expand2s_inner LPAR qf, i16, ix, iy)
#else
		#define expand2s_(i16, ix, iy) \
			expand2s_inner(Qs, i16, ix, iy)
		#define expand2s(i16) \
			expand2s_(i16, I16_ ## i16, M16_ ## i16)
#endif

		qt[18] = expand2s(18);
		qt[19] = expand2s(19);
		qt[20] = expand2s(20);
		qt[21] = expand2s(21);
		qt[22] = expand2s(22);
		qt[23] = expand2s(23);
		qt[24] = expand2s(24);
		qt[25] = expand2s(25);
		qt[26] = expand2s(26);
		qt[27] = expand2s(27);
		qt[28] = expand2s(28);
		qt[29] = expand2s(29);
		qt[30] = expand2s(30);
		qt[31] = expand2s(31);

		sph_u32 xl, xh;
		xl = Qs(16) ^ Qs(17) ^ Qs(18) ^ Qs(19) ^ Qs(20) ^ Qs(21) ^ Qs(22) ^ Qs(23);

		xh = xl ^ Qs(24) ^ Qs(25) ^ Qs(26) ^ Qs(27)	^ Qs(28) ^ Qs(29) ^ Qs(30) ^ Qs(31);

		dh[ 0] = ((xh <<  5) ^ (Qs(16) >>  5) ^ data[ 0]) + (xl ^ Qs(24) ^ Qs(0));
		dh[ 1] = ((xh >>  7) ^ (Qs(17) <<  8) ^ data[ 1]) + (xl ^ Qs(25) ^ Qs(1));
		dh[ 2] = ((xh >>  5) ^ (Qs(18) <<  5) ^ data[ 2]) + (xl ^ Qs(26) ^ Qs(2));
		dh[ 3] = ((xh >>  1) ^ (Qs(19) <<  5) ^ data[ 3]) + (xl ^ Qs(27) ^ Qs(3));
		dh[ 4] = ((xh >>  3) ^ (Qs(20) <<  0) ^ data[ 4]) + (xl ^ Qs(28) ^ Qs(4));
		dh[ 5] = ((xh <<  6) ^ (Qs(21) >>  6) ^ data[ 5]) + (xl ^ Qs(29) ^ Qs(5));
		dh[ 6] = ((xh >>  4) ^ (Qs(22) <<  6) ^ data[ 6]) + (xl ^ Qs(30) ^ Qs(6));
		dh[ 7] = ((xh >> 11) ^ (Qs(23) <<  2) ^ data[ 7]) + (xl ^ Qs(31) ^ Qs(7));

		dh[ 8] = ROTL32(dh[4],  9) + (xh ^ Qs(24) ^ data[ 8]) + ((xl << 8) ^ Qs(23) ^ Qs( 8));
		dh[ 9] = ROTL32(dh[5], 10) + (xh ^ Qs(25) ^ data[ 9]) + ((xl >> 6) ^ Qs(16) ^ Qs( 9));
		dh[10] = ROTL32(dh[6], 11) + (xh ^ Qs(26) ^ data[10]) + ((xl << 6) ^ Qs(17) ^ Qs(10));
		dh[11] = ROTL32(dh[7], 12) + (xh ^ Qs(27) ^ data[11]) + ((xl << 4) ^ Qs(18) ^ Qs(11));
		dh[12] = ROTL32(dh[0], 13) + (xh ^ Qs(28) ^ data[12]) + ((xl >> 3) ^ Qs(19) ^ Qs(12));
		dh[13] = ROTL32(dh[1], 14) + (xh ^ Qs(29) ^ data[13]) + ((xl >> 4) ^ Qs(20) ^ Qs(13));
		dh[14] = ROTL32(dh[2], 15) + (xh ^ Qs(30) ^ data[14]) + ((xl >> 7) ^ Qs(21) ^ Qs(14));
		dh[15] = ROTL32(dh[3], 16) + (xh ^ Qs(31) ^ data[15]) + ((xl >> 2) ^ Qs(22) ^ Qs(15));
}

#ifndef USE_MIDSTATE

__device__
static void gpu_bmw256_init(sph_bmw_small_context *sc)
{
	memcpy(sc->H, IV256, sizeof sc->H);
	sc->ptr = 0;
	sc->bit_count = 0;
}

__device__
static void gpu_bmw256(sph_bmw_small_context *sc, const void *data, size_t len)
{
	sph_u32 htmp[16];
	sph_u32 *h1, *h2;
	unsigned char *buf = sc->buf;
	size_t ptr = sc->ptr;

	sc->bit_count += (sph_u64)len << 3;

	h1 = sc->H;
	h2 = htmp;
	while (len > 0) {
		size_t clen;

		clen = (sizeof sc->buf) - ptr;
		if (clen > len)
			clen = len;
		memcpy(buf + ptr, data, clen);
		data = (const unsigned char *)data + clen;
		len -= clen;
		ptr += clen;
		if (ptr == sizeof sc->buf) {
			sph_u32 *ht;

			gpu_compress_small((sph_u32 *) buf, h1, h2);
			ht = h1;
			h1 = h2;
			h2 = ht;
			ptr = 0;
		}
	}
	sc->ptr = ptr;
	if (h1 != sc->H)
		memcpy(sc->H, h1, sizeof sc->H);
}

#endif

#define sph_enc64le(ptr, x) \
	*((uint64_t*)(ptr)) = x
#define sph_enc64le_aligned sph_enc64le

__device__
static void gpu_bmw256_close(sph_bmw_small_context *sc, uint2 *out)
{
	unsigned char *buf = sc->buf;
	size_t ptr = sc->ptr;

	buf[ptr ++] = 0x80;
	sph_u32 *h = sc->H;

	sph_u32 h1[16];
	if (ptr > (sizeof sc->buf) - 8) {
		memset(buf + ptr, 0, (sizeof sc->buf) - ptr);
		gpu_compress_small((sph_u32 *) buf, h, h1);
		ptr = 0;
		h = h1;
	}
	memset(buf + ptr, 0, sizeof(sc->buf) - 8 - ptr);

	sph_enc64le_aligned(buf + sizeof(sc->buf) - 8, SPH_T64(sc->bit_count));

	sph_u32 h2[16];
	gpu_compress_small((sph_u32 *) buf, h, h2);
	gpu_compress_small(h2, final_s, h1);

	uint64_t* h64 = (uint64_t*) (&h1[8]);
	#pragma unroll
	for (int i = 0; i < 4; i++) {
		out[i] = vectorize(h64[i]);
	}
}

__global__ /* __launch_bounds__(256, 3) */
void bmw256_gpu_hash_80(uint32_t threads, uint32_t startNonce, uint64_t *g_hash, sph_bmw256_context *d_midstate, int swap)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nonce = startNonce + thread;
		nonce = swap ? cuda_swab32(nonce): nonce;

#ifndef USE_MIDSTATE
		uint2 hash[10];
		#pragma unroll
		for(int i=0;i<9;i++)
			hash[i] = vectorize(c_PaddedMessage80[i]);
		hash[9] = make_uint2(c_PaddedMessage80[9], nonce);

		sph_bmw256_context ctx;
		gpu_bmw256_init(&ctx);
		gpu_bmw256(&ctx, (void*) hash, 80);
#else
		sph_bmw256_context ctx;
		ctx.ptr = 16; ctx.bit_count = 640;
		uint2 *buf = (uint2 *) ctx.buf;
		buf[0] = vectorize(c_PaddedMessage80[8]);
		buf[1] = make_uint2(c_PaddedMessage80[9], nonce);
		#pragma unroll
		for(int i=0;i<16;i++)
			ctx.H[i] = d_midstate->H[i];
#endif
		gpu_bmw256_close(&ctx, (uint2*) &g_hash[thread << 2]);
	}
}

__host__
void bmw256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash, int swap)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	bmw256_gpu_hash_80<<<grid, block>>>(threads, startNonce, (uint64_t*)d_outputHash, d_midstate[thr_id], swap);
}

__host__
void bmw256_setBlock_80(int thr_id, void *pdata)
{
	uint64_t PaddedMessage[16];
	memcpy(PaddedMessage, pdata, 80);
	memset(&PaddedMessage[10], 0, 48);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, sizeof(PaddedMessage), 0, hipMemcpyHostToDevice));

	sph_bmw256_context ctx;
	sph_bmw256_init(&ctx);
	sph_bmw256(&ctx, (void*) PaddedMessage, 80);
	CUDA_SAFE_CALL(hipMemcpy(d_midstate[thr_id], &ctx, sizeof(sph_bmw256_context), hipMemcpyHostToDevice));
}

__host__
void bmw256_midstate_init(int thr_id, uint32_t threads)
{
	hipMalloc(&d_midstate[thr_id], sizeof(sph_bmw256_context));
}

__host__
void bmw256_midstate_free(int thr_id)
{
	hipFree(d_midstate[thr_id]);
}
