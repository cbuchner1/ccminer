#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5/5.2)
 *
 * Tanguy Pruvot / SP - Jan 2016
 */

#include <stdint.h>
#include <memory.h>

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

/* threads per block */
#define TPB 512

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int8_t rounds = 14)
{
	uchar hash[64];
	sph_blake256_context ctx;

	sph_blake256_set_rounds(rounds);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(32) d_data[12];

/* 8 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* max count of found nonces in one call */
#define NBN 2
static __thread uint32_t extra_results[NBN] = { UINT32_MAX };

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ c_u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ c_u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
	}

__device__ __forceinline__
void blake256_compress_14(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t /*_ALIGN(8)*/ m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	const uint32_t c_u256[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
		0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
		0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};

	const uint32_t c_Padding[12] = {
		0x80000000UL, 0, 0, 0,
		0, 0, 0, 0,
		0, 1, 0, 640,
	};

	#pragma unroll
	for (uint32_t i = 0; i < 12; i++) {
		m[i+4] = c_Padding[i];
	}

	//#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC,0,1);
	GSPREC(1, 5, 0x9, 0xD,2,3);
	GSPREC(2, 6, 0xA, 0xE, 4,5);
	GSPREC(3, 7, 0xB, 0xF, 6,7);
	GSPREC(0, 5, 0xA, 0xF, 8,9);
	GSPREC(1, 6, 0xB, 0xC, 10,11);
	GSPREC(2, 7, 0x8, 0xD, 12,13);
	GSPREC(3, 4, 0x9, 0xE, 14,15);
	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);
	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);
	//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	GSPREC(0, 4, 0x8, 0xC, 9, 0);
	GSPREC(1, 5, 0x9, 0xD, 5, 7);
	GSPREC(2, 6, 0xA, 0xE, 2, 4);
	GSPREC(3, 7, 0xB, 0xF, 10, 15);
	GSPREC(0, 5, 0xA, 0xF, 14, 1);
	GSPREC(1, 6, 0xB, 0xC, 11, 12);
	GSPREC(2, 7, 0x8, 0xD, 6, 8);
	GSPREC(3, 4, 0x9, 0xE, 3, 13);
	//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	GSPREC(0, 4, 0x8, 0xC, 2, 12);
	GSPREC(1, 5, 0x9, 0xD, 6, 10);
	GSPREC(2, 6, 0xA, 0xE, 0, 11);
	GSPREC(3, 7, 0xB, 0xF, 8, 3);
	GSPREC(0, 5, 0xA, 0xF, 4, 13);
	GSPREC(1, 6, 0xB, 0xC, 7, 5);
	GSPREC(2, 7, 0x8, 0xD, 15, 14);
	GSPREC(3, 4, 0x9, 0xE, 1, 9);
	//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	GSPREC(0, 4, 0x8, 0xC, 12, 5);
	GSPREC(1, 5, 0x9, 0xD, 1, 15);
	GSPREC(2, 6, 0xA, 0xE, 14, 13);
	GSPREC(3, 7, 0xB, 0xF, 4, 10);
	GSPREC(0, 5, 0xA, 0xF, 0, 7);
	GSPREC(1, 6, 0xB, 0xC, 6, 3);
	GSPREC(2, 7, 0x8, 0xD, 9, 2);
	GSPREC(3, 4, 0x9, 0xE, 8, 11);
	//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	GSPREC(0, 4, 0x8, 0xC, 13, 11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3, 9);
	GSPREC(0, 5, 0xA, 0xF, 5, 0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8, 6);
	GSPREC(3, 4, 0x9, 0xE, 2, 10);
	//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	GSPREC(0, 4, 0x8, 0xC, 6, 15);
	GSPREC(1, 5, 0x9, 0xD, 14, 9);
	GSPREC(2, 6, 0xA, 0xE, 11, 3);
	GSPREC(3, 7, 0xB, 0xF, 0, 8);
	GSPREC(0, 5, 0xA, 0xF, 12, 2);
	GSPREC(1, 6, 0xB, 0xC, 13, 7);
	GSPREC(2, 7, 0x8, 0xD, 1, 4);
	GSPREC(3, 4, 0x9, 0xE, 10, 5);
	//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	GSPREC(0, 4, 0x8, 0xC, 10, 2);
	GSPREC(1, 5, 0x9, 0xD, 8, 4);
	GSPREC(2, 6, 0xA, 0xE, 7, 6);
	GSPREC(3, 7, 0xB, 0xF, 1, 5);
	GSPREC(0, 5, 0xA, 0xF, 15, 11);
	GSPREC(1, 6, 0xB, 0xC, 9, 14);
	GSPREC(2, 7, 0x8, 0xD, 3, 12);
	GSPREC(3, 4, 0x9, 0xE, 13, 0);
	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC, 0, 1);
	GSPREC(1, 5, 0x9, 0xD, 2, 3);
	GSPREC(2, 6, 0xA, 0xE, 4, 5);
	GSPREC(3, 7, 0xB, 0xF, 6, 7);
	GSPREC(0, 5, 0xA, 0xF, 8, 9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);
	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);
	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);

	// only compute h6 & 7
	h[6U] ^= v[6U] ^ v[14U];
	h[7U] ^= v[7U] ^ v[15U];
}

/* ############################################################################################################################### */
/* Precalculated 1st 64-bytes block (midstate) method */

__global__ __launch_bounds__(1024,1)
void blake256_gpu_hash_16(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint64_t highTarget)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t _ALIGN(16) h[8];

		#pragma unroll
		for(int i=0; i < 8; i++) {
			h[i] = d_data[i];
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t _ALIGN(16) ending[4];
		ending[0] = d_data[8];
		ending[1] = d_data[9];
		ending[2] = d_data[10];
		ending[3] = nonce; /* our tested value */

		blake256_compress_14(h, ending, 640);

		if (h[7] == 0 && cuda_swab32(h[6]) <= highTarget) {
#if NBN == 2
			if (resNonce[0] != UINT32_MAX)
				resNonce[1] = nonce;
			else
				resNonce[0] = nonce;
#else
			resNonce[0] = nonce;
#endif
		}
	}
}

__global__
#if __CUDA_ARCH__ >= 500
__launch_bounds__(512, 3) /* 40 regs */
#endif
void blake256_gpu_hash_16_8(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint64_t highTarget)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t h[8];
		const uint32_t nonce = startNonce + thread;

		#pragma unroll
		for (int i = 0; i < 8; i++) {
			h[i] = d_data[i];
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t m[16] = {
			d_data[8], d_data[9], d_data[10], nonce,
			0x80000000UL, 0, 0, 0,
			0, 0, 0, 0,
			0, 1, 0, 640,
		};

		const uint32_t c_u256[16] = {
			0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
			0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
			0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
		};

		uint32_t v[16];

		#pragma unroll
		for (uint32_t i = 0; i < 8; i++)
			v[i] = h[i];

		v[8]  = c_u256[0];
		v[9]  = c_u256[1];
		v[10] = c_u256[2];
		v[11] = c_u256[3];

		v[12] = c_u256[4] ^ 640U;
		v[13] = c_u256[5] ^ 640U;
		v[14] = c_u256[6];
		v[15] = c_u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);
		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);
		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		//GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },

		// only compute h6 & 7
		//h[6] ^= v[6] ^ v[14];
		//h[7] ^= v[7] ^ v[15];

		if ((h[7]^v[7]^v[15]) == 0) // h7
		{
			GSPREC(3, 4, 0x9, 0xE, 2, 10);
			if (cuda_swab32(h[6]^v[6]^v[14]) <= highTarget) {
#if NBN == 2
				if (resNonce[0] != UINT32_MAX)
					resNonce[1] = nonce;
				else
					resNonce[0] = nonce;
#else
				resNonce[0] = nonce;
#endif
			}
		}
	}
}

__host__
static uint32_t blake256_cpu_hash_16(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget,
	const int8_t rounds)
{
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	if (rounds == 8)
		blake256_gpu_hash_16_8 <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id], highTarget);
	else
		blake256_gpu_hash_16  <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id], highTarget);

	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = h_resNonce[thr_id][0];
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
	}
	return result;
}

__host__
static void blake256mid(uint32_t *output, const uint32_t *input, int8_t rounds = 14)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(rounds);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 64);

	memcpy(output, (void*)ctx.H, 32);
}

__host__
void blake256_cpu_setBlock_16(uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	uint32_t _ALIGN(64) data[11];
	memcpy(data, midstate, 32);
	data[8] = penddata[0];
	data[9] = penddata[1];
	data[10]= penddata[2];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 32 + 12, 0, hipMemcpyHostToDevice));
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake256(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, int8_t blakerounds=14)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t _ALIGN(64) midstate[8];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 30 : 26;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (opt_benchmark) {
		targetHigh = 0x1ULL << 32;
		ptarget[6] = swab32(0xff);
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256mid(midstate, endiandata, blakerounds);
	blake256_cpu_setBlock_16(&pdata[16], midstate, ptarget);

	do {
		// GPU HASH (second block only, first is midstate)
		work->nonces[0] = blake256_cpu_hash_16(thr_id, throughput, pdata[19], targetHigh, blakerounds);

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhashcpu[8];
			const uint32_t Htarg = ptarget[6];

			for (int k=16; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], work->nonces[0]);
			blake256hash(vhashcpu, endiandata, blakerounds);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhashcpu);
#if NBN > 1
				if (extra_results[0] != UINT32_MAX) {
					work->nonces[1] = extra_results[0];
					be32enc(&endiandata[19], work->nonces[1]);
					blake256hash(vhashcpu, endiandata, blakerounds);
					if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio[0]) {
							work_set_target_ratio(work, vhashcpu);
							xchg(work->nonces[0], work->nonces[1]);
						} else {
							bn_set_target_ratio(work, vhashcpu, 1);
						}
						work->valid_nonces = 2;
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
					extra_results[0] = UINT32_MAX;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#endif
				return work->valid_nonces;
			}
			else if (vhashcpu[6] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + pdata[19]);

	*hashes_done = pdata[19] - first_nonce;

	MyStreamSynchronize(NULL, 0, device_map[thr_id]);
	return rc;
}

// cleanup
extern "C" void free_blake256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

