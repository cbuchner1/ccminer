#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 */

#define PRECALC64 1

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block and throughput (intensity) */
#define TPB 128

/* added in sph_blake.c */
extern "C" int blake256_rounds = 14;

/* hash by cpu with blake 256 */
extern "C" void blake256hash(void *output, const void *input, int8_t rounds = 14)
{
	uchar hash[64];
	sph_blake256_context ctx;

	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#if PRECALC64
__constant__ uint32_t _ALIGN(32) d_data[12];
#else
__constant__ static uint32_t _ALIGN(32) c_data[20];
/* midstate hash cache, this algo is run on 2 parts */
__device__ static uint32_t cache[8];
__device__ static uint32_t prevsum = 0;
/* crc32.c */
extern "C" uint32_t crc32_u32t(const uint32_t *buf, size_t size);
#endif

/* 8 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* max count of found nonces in one call */
#define NBN 2
static uint32_t extra_results[NBN] = { UINT32_MAX };

/* prefer uint32_t to prevent size conversions = speed +5/10 % */
__constant__
static uint32_t _ALIGN(32) c_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

#if !PRECALC64
__device__ __constant__
static const uint32_t __align__(32) c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};
#endif

__device__ __constant__
static const uint32_t __align__(32) c_u256[16] = {
	SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
	SPH_C32(0x13198A2E), SPH_C32(0x03707344),
	SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
	SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
	SPH_C32(0x452821E6), SPH_C32(0x38D01377),
	SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
	SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
	SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
};

#define GS(a,b,c,d,x) { \
	const uint32_t idx1 = c_sigma[r][x]; \
	const uint32_t idx2 = c_sigma[r][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = SPH_ROTL32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

/* Second part (64-80) msg never change, store it */
__device__ __constant__
static const uint32_t __align__(32) c_Padding[16] = {
	0, 0, 0, 0,
	0x80000000UL, 0, 0, 0,
	0, 0, 0, 0,
	0, 1, 0, 640,
};

__device__ static
void blake256_compress(uint32_t *h, const uint32_t *block, const uint32_t T0, const int rounds)
{
	uint32_t /*_ALIGN(8)*/ m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	for (uint32_t i = 4; i < 16; i++) {
#if PRECALC64
		m[i] = c_Padding[i];
#else
		m[i] = (T0 == 0x200) ? block[i] : c_Padding[i];
#endif
	}

	//#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int r = 0; r < rounds; r++) {
		/* column step */
		GS(0, 4, 0x8, 0xC, 0x0);
		GS(1, 5, 0x9, 0xD, 0x2);
		GS(2, 6, 0xA, 0xE, 0x4);
		GS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		GS(0, 5, 0xA, 0xF, 0x8);
		GS(1, 6, 0xB, 0xC, 0xA);
		GS(2, 7, 0x8, 0xD, 0xC);
		GS(3, 4, 0x9, 0xE, 0xE);
	}
#if PRECALC64
	// only compute h6 & 7
	h[6U] ^= v[6U] ^ v[14U];
	h[7U] ^= v[7U] ^ v[15U];
#else
	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i % 8U;
		h[j] ^= v[i];
	}
#endif
}

#if !PRECALC64 /* original method */
__global__
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,
	const uint64_t highTarget, const int crcsum, const int rounds)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++) {
			h[i] = c_IV256[i];
		}

		if (crcsum != prevsum) {
			prevsum = crcsum;
			blake256_compress(h, c_data, 512, rounds);
			#pragma unroll
			for(int i=0; i<8; i++) {
				cache[i] = h[i];
			}
		} else {
			#pragma unroll
			for(int i=0; i<8; i++) {
				h[i] = cache[i];
			}
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t ending[4];
		ending[0] = c_data[16];
		ending[1] = c_data[17];
		ending[2] = c_data[18];
		ending[3] = nonce; /* our tested value */

		blake256_compress(h, ending, 640, rounds);

		// not sure why, h[7] is ok
		h[6] = cuda_swab32(h[6]);

		// compare count of leading zeros h[6] + h[7]
		uint64_t high64 = ((uint64_t*)h)[3];
		if (high64 <= highTarget)
#if NBN == 2
		/* keep the smallest nonce, + extra one if found */
		if (resNonce[0] > nonce) {
			// printf("%llx %llx \n", high64, highTarget);
			resNonce[1] = resNonce[0];
			resNonce[0] = nonce;
		}
		else
			resNonce[1] = nonce;
#else
		resNonce[0] = nonce;
#endif
	}
}

__host__
uint32_t blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget,
	const uint32_t crcsum, const int8_t rounds)
{
	const uint32_t threadsperblock = TPB;
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNonce, d_resNonce[thr_id], highTarget, crcsum, (int) rounds);
	MyStreamSynchronize(NULL, 0, thr_id);
	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = h_resNonce[thr_id][0];
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata, const uint32_t *ptarget)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
}
#else

/* ############################################################################################################################### */
/* Precalculated 1st 64-bytes block (midstate) method */

__global__
void blake256_gpu_hash_16(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,
	const uint64_t highTarget, const int rounds, const bool trace)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t _ALIGN(16) h[8];

		#pragma unroll
		for(int i=0; i < 8; i++) {
			h[i] = d_data[i];
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t _ALIGN(16) ending[4];
		ending[0] = d_data[8];
		ending[1] = d_data[9];
		ending[2] = d_data[10];
		ending[3] = nonce; /* our tested value */

		blake256_compress(h, ending, 640, rounds);

		if (h[7] == 0 && cuda_swab32(h[6]) <= highTarget) {
#if NBN == 2
			/* keep the smallest nonce, + extra one if found */
			if (resNonce[0] > nonce) {
				resNonce[1] = resNonce[0];
				resNonce[0] = nonce;
			}
			else
				resNonce[1] = nonce;
#else
			resNonce[0] = nonce;
#endif
#ifdef _DEBUG
			if (trace) {
				uint64_t high64 = ((uint64_t*)h)[3];
				printf("gpu:  %16llx\n", high64);
				printf("gpu: %08x.%08x\n", h[7], h[6]);
				printf("tgt:  %16llx\n", highTarget);
			}
#endif
		}
	}
}

__host__
static uint32_t blake256_cpu_hash_16(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget,
	const int8_t rounds)
{
	const uint32_t threadsperblock = TPB;
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_16 <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id], highTarget, (int) rounds, opt_tracegpu);
	MyStreamSynchronize(NULL, 0, thr_id);
	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = h_resNonce[thr_id][0];
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
	}
	return result;
}

__host__
static void blake256mid(uint32_t *output, const uint32_t *input, int8_t rounds = 14)
{
	sph_blake256_context ctx;

	/* in sph_blake.c */
	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 64);

	memcpy(output, (void*)ctx.H, 32);
}

__host__
void blake256_cpu_setBlock_16(uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	uint32_t _ALIGN(64) data[11];
	memcpy(data, midstate, 32);
	data[8] = penddata[0];
	data[9] = penddata[1];
	data[10]= penddata[2];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 32 + 12, 0, hipMemcpyHostToDevice));
}
#endif

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_blake256(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done, int8_t blakerounds=14)
{
	const uint32_t first_nonce = pdata[19];
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];
	uint32_t _ALIGN(64) endiandata[20];
#if PRECALC64
	uint32_t _ALIGN(64) midstate[8];
#else
	uint32_t crcsum;
#endif
	int intensity = (device_sm[device_map[thr_id]] > 500) ? 22 : 20;
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << intensity);
	throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (opt_benchmark) {
		targetHigh = 0x1ULL << 32;
		((uint32_t*)ptarget)[6] = swab32(0xff);
	}

	if (opt_tracegpu) {
		/* test call from util.c */
		throughput = 1;
		for (int k = 0; k < 20; k++)
			pdata[k] = swab32(pdata[k]);
	}

	if (!init[thr_id]) {
		if (active_gpus > 1)
			hipSetDevice(device_map[thr_id]);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), 0);
		init[thr_id] = true;
	}

#if PRECALC64
	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);
	blake256mid(midstate, endiandata, blakerounds);
	blake256_cpu_setBlock_16(&pdata[16], midstate, ptarget);
#else
	blake256_cpu_setBlock_80(pdata, ptarget);
	crcsum = crc32_u32t(pdata, 64);
#endif /* PRECALC64 */

	do {
		uint32_t foundNonce =
#if PRECALC64
		// GPU HASH (second block only, first is midstate)
		blake256_cpu_hash_16(thr_id, throughput, pdata[19], targetHigh, blakerounds);
#else
		// GPU FULL HASH
		blake256_cpu_hash_80(thr_id, throughput, pdata[19], targetHigh, crcsum, blakerounds);
#endif
		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhashcpu[8];
			uint32_t Htarg = (uint32_t)targetHigh;

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			be32enc(&endiandata[19], foundNonce);
			blake256hash(vhashcpu, endiandata, blakerounds);

			//applog(LOG_BLUE, "%08x %16llx", vhashcpu[6], targetHigh);
			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				rc = 1;
				pdata[19] = foundNonce;
				*hashes_done = pdata[19] - first_nonce + 1;
#if NBN > 1
				if (extra_results[0] != UINT32_MAX) {
					be32enc(&endiandata[19], extra_results[0]);
					blake256hash(vhashcpu, endiandata, blakerounds);
					if (vhashcpu[6] <= Htarg /* && fulltest(vhashcpu, ptarget) */) {
						pdata[21] = extra_results[0];
						applog(LOG_BLUE, "1:%x 2:%x", foundNonce, extra_results[0]);
						*hashes_done = max(*hashes_done, extra_results[0] - first_nonce + 1);
						rc = 2;
					}
					extra_results[0] = UINT32_MAX;
				}
#endif
				//applog_hash((uint8_t*)ptarget);
				//applog_compare_hash((uint8_t*)vhashcpu,(uint8_t*)ptarget);
				return rc;
			}
			else if (opt_debug) {
				applog_hash((uchar*)ptarget);
				applog_compare_hash((uchar*)vhashcpu, (uchar*)ptarget);
				applog(LOG_WARNING, "GPU #%d: result for nonce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		if ((uint64_t) pdata[19] + throughput > (uint64_t) max_nonce) {
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return rc;
}
