#include "hip/hip_runtime.h"
/**
 * KECCAK-256 CUDA optimised implementation, based on ccminer-alexis code
 */

#include <miner.h>

extern "C" {
#include <stdint.h>
#include <memory.h>
}

#include <cuda_helper.h>
#include <cuda_vectors.h>

#define TPB52 1024
#define TPB50 384
#define NPT 2
#define NBN 2

static uint32_t *d_nonces[MAX_GPUS];
static uint32_t *h_nonces[MAX_GPUS];

__constant__ uint2 c_message48[6];
__constant__ uint2 c_mid[17];

__constant__ uint2 keccak_round_constants[24] = {
	{ 0x00000001, 0x00000000 }, { 0x00008082, 0x00000000 },	{ 0x0000808a, 0x80000000 }, { 0x80008000, 0x80000000 },
	{ 0x0000808b, 0x00000000 }, { 0x80000001, 0x00000000 },	{ 0x80008081, 0x80000000 }, { 0x00008009, 0x80000000 },
	{ 0x0000008a, 0x00000000 }, { 0x00000088, 0x00000000 },	{ 0x80008009, 0x00000000 }, { 0x8000000a, 0x00000000 },
	{ 0x8000808b, 0x00000000 }, { 0x0000008b, 0x80000000 },	{ 0x00008089, 0x80000000 }, { 0x00008003, 0x80000000 },
	{ 0x00008002, 0x80000000 }, { 0x00000080, 0x80000000 },	{ 0x0000800a, 0x00000000 }, { 0x8000000a, 0x80000000 },
	{ 0x80008081, 0x80000000 }, { 0x00008080, 0x80000000 },	{ 0x80000001, 0x00000000 }, { 0x80008008, 0x80000000 }
};

__device__ __forceinline__
uint2 xor3x(const uint2 a,const uint2 b,const uint2 c) {
	uint2 result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.x) : "r"(a.x), "r"(b.x),"r"(c.x)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.y) : "r"(a.y), "r"(b.y),"r"(c.y)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

__device__ __forceinline__
uint2 chi(const uint2 a,const uint2 b,const uint2 c) { // keccak chi
	uint2 result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.x) : "r"(a.x), "r"(b.x),"r"(c.x)); //0xD2 = 0xF0 ^ ((~0xCC) & 0xAA)
	asm ("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.y) : "r"(a.y), "r"(b.y),"r"(c.y)); //0xD2 = 0xF0 ^ ((~0xCC) & 0xAA)
#else
	result = a ^ (~b) & c;
#endif
	return result;
}

__device__ __forceinline__
uint64_t xor5(uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e)
{
	uint64_t result;
	asm("xor.b64 %0, %1, %2;" : "=l"(result) : "l"(d) ,"l"(e));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(c));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(b));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(a));
	return result;
}

#if __CUDA_ARCH__ <= 500
__global__ __launch_bounds__(TPB50, 2)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void keccak256_gpu_hash_80(uint32_t threads, uint32_t startNonce, uint32_t *resNounce, const uint2 highTarget)
{
	uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	uint2 s[25], t[5], v, w, u[5];
#if __CUDA_ARCH__ > 500
	uint64_t step     = gridDim.x * blockDim.x;
	uint64_t maxNonce = startNonce + threads;
	for(uint64_t nounce = startNonce + thread; nounce<maxNonce;nounce+=step) {
#else
	uint32_t nounce = startNonce+thread;
	if(thread<threads) {
#endif
		s[ 9] = make_uint2(c_message48[0].x,cuda_swab32(nounce));
		s[10] = keccak_round_constants[0];

		t[ 4] = c_message48[1]^s[ 9];
		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[ 0] = t[4] ^ c_mid[ 0];
		u[ 1] = c_mid[ 1] ^ ROL2(t[4],1);
		u[ 2] = c_mid[ 2];
		/* thetarho pi: b[..] = rotl(a[..] ^ d[...], ..)*/
		s[ 7] = ROL2(s[10]^u[0], 3);
		s[10] = c_mid[ 3];
		    w = c_mid[ 4];
		s[20] = c_mid[ 5];
		s[ 6] = ROL2(s[ 9]^u[2],20);
		s[ 9] = c_mid[ 6];
		s[22] = c_mid[ 7];
		s[14] = ROL2(u[0],18);
		s[ 2] = c_mid[ 8];
		s[12] = ROL2(u[1],25);
		s[13] = c_mid[ 9];
		s[19] = ROR8(u[1]);
		s[23] = ROR2(u[0],23);
		s[15] = c_mid[10];
		s[ 4] = c_mid[11];
		s[24] = c_mid[12];
		s[21] = ROR2(c_message48[2]^u[1], 9);
		s[ 8] = c_mid[13];
		s[16] = ROR2(c_message48[3]^u[0],28);
		s[ 5] = ROL2(c_message48[4]^u[1],28);
		s[ 3] = ROL2(u[1],21);
		s[18] = c_mid[14];
		s[17] = c_mid[15];
		s[11] = c_mid[16];

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = c_message48[5]^u[0];
		s[ 0] = chi(v,w,s[ 2]);
		s[ 1] = chi(w,s[ 2],s[ 3]);
		s[ 2] = chi(s[ 2],s[ 3],s[ 4]);
		s[ 3] = chi(s[ 3],s[ 4],v);
		s[ 4] = chi(s[ 4],v,w);
		v = s[ 5]; w = s[ 6]; s[ 5] = chi(v,w,s[ 7]); s[ 6] = chi(w,s[ 7],s[ 8]); s[ 7] = chi(s[ 7],s[ 8],s[ 9]); s[ 8] = chi(s[ 8],s[ 9],v);s[ 9] = chi(s[ 9],v,w);
		v = s[10]; w = s[11]; s[10] = chi(v,w,s[12]); s[11] = chi(w,s[12],s[13]); s[12] = chi(s[12],s[13],s[14]); s[13] = chi(s[13],s[14],v);s[14] = chi(s[14],v,w);
		v = s[15]; w = s[16]; s[15] = chi(v,w,s[17]); s[16] = chi(w,s[17],s[18]); s[17] = chi(s[17],s[18],s[19]); s[18] = chi(s[18],s[19],v);s[19] = chi(s[19],v,w);
		v = s[20]; w = s[21]; s[20] = chi(v,w,s[22]); s[21] = chi(w,s[22],s[23]); s[22] = chi(s[22],s[23],s[24]); s[23] = chi(s[23],s[24],v);s[24] = chi(s[24],v,w);

		/* iota: a[0,0] ^= round constant */
		s[ 0] ^=keccak_round_constants[ 0];

		#if __CUDA_ARCH__ > 500
			#pragma unroll 22
		#else
			#pragma unroll 4
		#endif
		for (int i = 1; i < 23; i++) {
			#pragma unroll
			for(int j=0;j<5;j++) {
				t[ j] = vectorize(xor5(devectorize(s[ j]),devectorize(s[j+5]),devectorize(s[j+10]),devectorize(s[j+15]),devectorize(s[j+20])));
			}
			/*theta*/
			#pragma unroll
			for(int j=0;j<5;j++) {
				u[j] = ROL2(t[j], 1);
			}
			s[ 4] = xor3x(s[ 4], t[3], u[0]);s[ 9] = xor3x(s[ 9], t[3], u[0]);s[14] = xor3x(s[14], t[3], u[0]);s[19] = xor3x(s[19], t[3], u[0]);s[24] = xor3x(s[24], t[3], u[0]);
			s[ 0] = xor3x(s[ 0], t[4], u[1]);s[ 5] = xor3x(s[ 5], t[4], u[1]);s[10] = xor3x(s[10], t[4], u[1]);s[15] = xor3x(s[15], t[4], u[1]);s[20] = xor3x(s[20], t[4], u[1]);
			s[ 1] = xor3x(s[ 1], t[0], u[2]);s[ 6] = xor3x(s[ 6], t[0], u[2]);s[11] = xor3x(s[11], t[0], u[2]);s[16] = xor3x(s[16], t[0], u[2]);s[21] = xor3x(s[21], t[0], u[2]);
			s[ 2] = xor3x(s[ 2], t[1], u[3]);s[ 7] = xor3x(s[ 7], t[1], u[3]);s[12] = xor3x(s[12], t[1], u[3]);s[17] = xor3x(s[17], t[1], u[3]);s[22] = xor3x(s[22], t[1], u[3]);
			s[ 3] = xor3x(s[ 3], t[2], u[4]);s[ 8] = xor3x(s[ 8], t[2], u[4]);s[13] = xor3x(s[13], t[2], u[4]);s[18] = xor3x(s[18], t[2], u[4]);s[23] = xor3x(s[23], t[2], u[4]);
			/*rho pi: b[..] = rotl(a[..] ^ d[...], ..)*/
			v = s[ 1];
			s[ 1] = ROL2(s[ 6],44);	s[ 6] = ROL2(s[ 9],20);	s[ 9] = ROL2(s[22],61);	s[22] = ROL2(s[14],39);
			s[14] = ROL2(s[20],18);	s[20] = ROL2(s[ 2],62);	s[ 2] = ROL2(s[12],43);	s[12] = ROL2(s[13],25);
			s[13] = ROL8(s[19]);	s[19] = ROR8(s[23]);	s[23] = ROL2(s[15],41);	s[15] = ROL2(s[ 4],27);
			s[ 4] = ROL2(s[24],14);	s[24] = ROL2(s[21], 2);	s[21] = ROL2(s[ 8],55);	s[ 8] = ROL2(s[16],45);
			s[16] = ROL2(s[ 5],36);	s[ 5] = ROL2(s[ 3],28);	s[ 3] = ROL2(s[18],21);	s[18] = ROL2(s[17],15);
			s[17] = ROL2(s[11],10);	s[11] = ROL2(s[ 7], 6);	s[ 7] = ROL2(s[10], 3);	s[10] = ROL2(v, 1);
			/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
			#pragma unroll
			for(int j=0;j<25;j+=5) {
				v=s[j];w=s[j + 1];s[j] = chi(s[j],s[j+1],s[j+2]);s[j+1] = chi(s[j+1],s[j+2],s[j+3]);s[j+2]=chi(s[j+2],s[j+3],s[j+4]);s[j+3]=chi(s[j+3],s[j+4],v);s[j+4]=chi(s[j+4],v,w);
			}
			/* iota: a[0,0] ^= round constant */
			s[ 0] ^=keccak_round_constants[ i];
		}
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		#pragma unroll 5
		for(int j=0;j<5;j++) {
			t[ j] = xor3x(xor3x(s[j+0],s[j+5],s[j+10]), s[j+15], s[j+20]);
		}
		s[24] = xor3x(s[24],t[3],ROL2(t[0],1));
		s[18] = xor3x(s[18],t[2],ROL2(t[4],1));
		s[ 0] = xor3x(s[ 0],t[4],ROL2(t[1],1));
		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		s[24] = ROL2(s[24],14);
		s[18] = ROL2(s[18],21);
		if (devectorize(chi(s[18],s[24],s[ 0])) <= devectorize(highTarget)) {
//		if(chi(s[18].x,s[24].x,s[0].x)<=highTarget.x) {
//			if(chi(s[18].y,s[24].y,s[0].y)<=highTarget.y) {
				const uint32_t tmp = atomicExch(&resNounce[0], nounce);
				if (tmp != UINT32_MAX)
					resNounce[1] = tmp;
	//			return;
//			}
		}
	}
}

__host__
void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t* resNonces, const uint2 highTarget)
{
	uint32_t tpb;
	dim3 grid;
	if (device_sm[device_map[thr_id]] <= 500) {
		tpb = TPB50;
		grid.x = (threads + tpb-1)/tpb;
	} else {
		tpb = TPB52;
		grid.x = (threads + (NPT*tpb)-1)/(NPT*tpb);
	}
	const dim3 block(tpb);

	keccak256_gpu_hash_80<<<grid, block>>>(threads, startNonce, d_nonces[thr_id], highTarget);
//	hipDeviceSynchronize();
	hipMemcpy(h_nonces[thr_id], d_nonces[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
	memcpy(resNonces, h_nonces[thr_id], NBN*sizeof(uint32_t));
}

#if 0
#if __CUDA_ARCH__ <= 500
__global__ __launch_bounds__(TPB50, 2)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void keccak256_gpu_hash_32(uint32_t threads, uint2* outputHash)
{
	uint32_t thread   = blockDim.x * blockIdx.x + threadIdx.x;
	uint2 s[25], t[5], v, w, u[5];

	if(thread < threads) {
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4) s[i] = __ldg(&outputHash[i*threads+thread]);
			else     s[i] = make_uint2(0, 0);
		}
		s[4]  = keccak_round_constants[ 0];
		s[16] = make_uint2(0, 0x80000000);
		#if __CUDA_ARCH__ > 500
			#pragma unroll
		#else
			#pragma unroll 4
		#endif
		for (uint32_t i = 0; i < 23; i++) {
			/*theta*/
			#pragma unroll 5
			for(int j=0; j<5; j++) {
				t[ j] = vectorize(xor5(devectorize(s[ j]),devectorize(s[j+5]),devectorize(s[j+10]),devectorize(s[j+15]),devectorize(s[j+20])));
			}
			/*theta*/
			#pragma unroll 5
			for(int j=0; j<5; j++) {
				u[j] = ROL2(t[j], 1);
			}
			s[ 4] = xor3x(s[ 4], t[3], u[0]);s[ 9] = xor3x(s[ 9], t[3], u[0]);s[14] = xor3x(s[14], t[3], u[0]);s[19] = xor3x(s[19], t[3], u[0]);s[24] = xor3x(s[24], t[3], u[0]);
			s[ 0] = xor3x(s[ 0], t[4], u[1]);s[ 5] = xor3x(s[ 5], t[4], u[1]);s[10] = xor3x(s[10], t[4], u[1]);s[15] = xor3x(s[15], t[4], u[1]);s[20] = xor3x(s[20], t[4], u[1]);
			s[ 1] = xor3x(s[ 1], t[0], u[2]);s[ 6] = xor3x(s[ 6], t[0], u[2]);s[11] = xor3x(s[11], t[0], u[2]);s[16] = xor3x(s[16], t[0], u[2]);s[21] = xor3x(s[21], t[0], u[2]);
			s[ 2] = xor3x(s[ 2], t[1], u[3]);s[ 7] = xor3x(s[ 7], t[1], u[3]);s[12] = xor3x(s[12], t[1], u[3]);s[17] = xor3x(s[17], t[1], u[3]);s[22] = xor3x(s[22], t[1], u[3]);
			s[ 3] = xor3x(s[ 3], t[2], u[4]);s[ 8] = xor3x(s[ 8], t[2], u[4]);s[13] = xor3x(s[13], t[2], u[4]);s[18] = xor3x(s[18], t[2], u[4]);s[23] = xor3x(s[23], t[2], u[4]);
			/*rho pi: b[..] = rotl(a[..] ^ d[...], ..)*/
			v = s[ 1];
			s[ 1] = ROL2(s[ 6],44); s[ 6] = ROL2(s[ 9],20); s[ 9] = ROL2(s[22],61); s[22] = ROL2(s[14],39);
			s[14] = ROL2(s[20],18); s[20] = ROL2(s[ 2],62); s[ 2] = ROL2(s[12],43); s[12] = ROL2(s[13],25);
			s[13] = ROL8(s[19]);    s[19] = ROR8(s[23]);    s[23] = ROL2(s[15],41); s[15] = ROL2(s[ 4],27);
			s[ 4] = ROL2(s[24],14); s[24] = ROL2(s[21], 2); s[21] = ROL2(s[ 8],55); s[ 8] = ROL2(s[16],45);
			s[16] = ROL2(s[ 5],36); s[ 5] = ROL2(s[ 3],28); s[ 3] = ROL2(s[18],21); s[18] = ROL2(s[17],15);
			s[17] = ROL2(s[11],10); s[11] = ROL2(s[ 7], 6); s[ 7] = ROL2(s[10], 3); s[10] = ROL2(v, 1);
			/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
			#pragma unroll 5
			for(int j=0; j<25; j+=5) {
				v=s[j];w=s[j + 1]; s[j] = chi(v,w,s[j+2]); s[j+1] = chi(w,s[j+2],s[j+3]); s[j+2]=chi(s[j+2],s[j+3],s[j+4]); s[j+3]=chi(s[j+3],s[j+4],v); s[j+4]=chi(s[j+4],v,w);
			}
			/* iota: a[0,0] ^= round constant */
			s[ 0] ^=keccak_round_constants[ i];
		}
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		#pragma unroll 5
		for(int j=0;j<5;j++) {
			t[ j] = xor3x(xor3x(s[j+0],s[j+5],s[j+10]), s[j+15], s[j+20]);
		}
		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		#pragma unroll 5
		for(int j=0;j<5;j++) {
			u[j] = ROL2(t[j],1);
		}
		/* thetarho pi: b[..] = rotl(a[..] ^ d[...], ..) //There's no need to perform theta and -store- the result since it's unique for each a[..]*/
		s[ 4] = xor3x(s[24], t[3], u[0]);
		s[ 0] = xor3x(s[ 0], t[4], u[1]);
		s[ 1] = xor3x(s[ 6], t[0], u[2]);
		s[ 2] = xor3x(s[12], t[1], u[3]);
		s[ 3] = xor3x(s[18], t[2], u[4]);
		s[ 1] = ROR2(s[ 1],20);
		s[ 2] = ROR2(s[ 2],21);
		s[ 3] = ROL2(s[ 3],21);
		s[ 4] = ROL2(s[ 4],14);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		outputHash[0*threads+thread] = chi(s[ 0],s[ 1],s[ 2]) ^ keccak_round_constants[23];
		outputHash[1*threads+thread] = chi(s[ 1],s[ 2],s[ 3]);
		outputHash[2*threads+thread] = chi(s[ 2],s[ 3],s[ 4]);
		outputHash[3*threads+thread] = chi(s[ 3],s[ 4],s[ 0]);
	}
}

__host__
void keccak256_cpu_hash_32(const int thr_id,const uint32_t threads, uint2* d_hash)
{
	uint32_t tpb = TPB52;
	if (device_sm[device_map[thr_id]] == 500) tpb = TPB50;
	const dim3 grid((threads + tpb-1)/tpb);
	const dim3 block(tpb);

	keccak256_gpu_hash_32 <<<grid, block>>> (threads, d_hash);
}
#endif

__host__
void keccak256_setBlock_80(uint64_t *endiandata)
{
	uint64_t midstate[17], s[25];
	uint64_t t[5], u[5];

	s[10] = 1; //(uint64_t)make_uint2(1, 0);
	s[16] = ((uint64_t)1)<<63; //(uint64_t)make_uint2(0, 0x80000000);

	t[0] = endiandata[0] ^ endiandata[5] ^ s[10];
	t[1] = endiandata[1] ^ endiandata[6] ^ s[16];
	t[2] = endiandata[2] ^ endiandata[7];
	t[3] = endiandata[3] ^ endiandata[8];

	midstate[ 0] = ROTL64(t[1], 1);         //u[0] -partial
	       u[1] = t[ 0] ^ ROTL64(t[2], 1);  //u[1]
	       u[2] = t[ 1] ^ ROTL64(t[3], 1);  //u[2]
	midstate[ 1] = t[ 2];                   //u[3] -partial
	midstate[ 2] = t[ 3] ^ ROTL64(t[0], 1); //u[4]
	midstate[ 3] = ROTL64(endiandata[1]^u[1], 1); //v
	midstate[ 4] = ROTL64(endiandata[6]^u[1], 44);
	midstate[ 5] = ROTL64(endiandata[2]^u[2], 62);
	midstate[ 6] = ROTL64(u[2], 61);
	midstate[ 7] = ROTL64(midstate[2], 39);
	midstate[ 8] = ROTL64(u[2], 43);
	midstate[ 9] = ROTL64(midstate[2], 8);
	midstate[10] = ROTL64(endiandata[4]^midstate[ 2],27);
	midstate[11] = ROTL64(midstate[2], 14);
	midstate[12] = ROTL64(u[1], 2);
	midstate[13] = ROTL64(s[16] ^ u[1], 45);
	midstate[14] = ROTL64(u[2],15);
	midstate[15] = ROTL64(u[1],10);
	midstate[16] = ROTL64(endiandata[7]^u[2], 6);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mid), midstate,17*sizeof(uint64_t), 0, hipMemcpyHostToDevice));

	// pass only what's needed
	uint64_t message48[6];
	message48[0] = endiandata[9];
	message48[1] = endiandata[4];
	message48[2] = endiandata[8];
	message48[3] = endiandata[5];
	message48[4] = endiandata[3];
	message48[5] = endiandata[0];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_message48), message48, 6*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void keccak256_cpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], NBN*sizeof(uint32_t)));
	//CUDA_SAFE_CALL(hipHostMalloc(&h_nonces[thr_id], NBN*sizeof(uint32_t)));
	h_nonces[thr_id] = (uint32_t*) malloc(NBN * sizeof(uint32_t));
	if(h_nonces[thr_id] == NULL) {
		gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
		exit(EXIT_FAILURE);
	}
}

__host__
void keccak256_setOutput(int thr_id)
{
	CUDA_SAFE_CALL(hipMemset(d_nonces[thr_id], 0xff, NBN*sizeof(uint32_t)));
}

__host__
void keccak256_cpu_free(int thr_id)
{
	hipFree(d_nonces[thr_id]);
	//hipHostFree(h_nonces[thr_id]);
	free(h_nonces[thr_id]);
}
