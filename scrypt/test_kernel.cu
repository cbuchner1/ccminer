#include "hip/hip_runtime.h"
/* Copyright (C) 2013 David G. Andersen. All rights reserved.
 * with modifications by Christian Buchner
 *
 * Use of this code is covered under the Apache 2.0 license, which
 * can be found in the file "LICENSE"
 *
 * The array notation for b[] and bx[] arrays was converted to uint4,
 * in preparation for some experimental changes to memory access patterns.
 * Also this kernel is going to be a testbed for adaptation to Fermi devices.
 */

// TODO: experiment with different memory access patterns in write/read_keys_direct functions
// TODO: attempt V.Volkov style ILP (factor 4)

#include <map>

#include <hip/hip_runtime.h>
#include "miner.h"

#include "salsa_kernel.h"
#include "test_kernel.h"

#define TEXWIDTH 32768
#define THREADS_PER_WU 4  // four threads per hash

typedef enum
{
		ANDERSEN,
		SIMPLE
} MemoryAccess;


// scratchbuf constants (pointers to scratch buffer for each warp, i.e. 32 hashes)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1;                   // N-1
// scratch buffer size SCRATCH
__constant__ uint32_t c_SCRATCH;
__constant__ uint32_t c_SCRATCH_WU_PER_WARP;   // (SCRATCH * WU_PER_WARP)
__constant__ uint32_t c_SCRATCH_WU_PER_WARP_1; // (SCRATCH * WU_PER_WARP) - 1

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

template <int ALGO> __device__  __forceinline__ void block_mixer(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3);

static __device__ uint4& operator^=(uint4& left, const uint4& right) {
	left.x ^= right.x;
	left.y ^= right.y;
	left.z ^= right.z;
	left.w ^= right.w;
	return left;
}

static __device__ uint4& operator+=(uint4& left, const uint4& right) {
	left.x += right.x;
	left.y += right.y;
	left.z += right.z;
	left.w += right.w;
	return left;
}

/* write_keys writes the 8 keys being processed by a warp to the global
 * scratchpad. To effectively use memory bandwidth, it performs the writes
 * (and reads, for read_keys) 128 bytes at a time per memory location
 * by __shfl'ing the 4 entries in bx to the threads in the next-up
 * thread group. It then has eight threads together perform uint4
 * (128 bit) writes to the destination region. This seems to make
 * quite effective use of memory bandwidth. An approach that spread
 * uint32s across more threads was slower because of the increased
 * computation it required.
 *
 * "start" is the loop iteration producing the write - the offset within
 * the block's memory.
 *
 * Internally, this algorithm first __shfl's the 4 bx entries to
 * the next up thread group, and then uses a conditional move to
 * ensure that odd-numbered thread groups exchange the b/bx ordering
 * so that the right parts are written together.
 *
 * Thanks to Babu for helping design the 128-bit-per-write version.
 *
 * _direct lets the caller specify the absolute start location instead of
 * the relative start location, as an attempt to reduce some recomputation.
 */

template <MemoryAccess SCHEME> __device__ __forceinline__
void write_keys_direct(const uint4 &b, const uint4 &bx, uint32_t start)
{
	uint32_t *scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/32];

	if (SCHEME == ANDERSEN) {
		uint4 t=b, t2;
		extern __shared__ unsigned char shared[];
		uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);
		uint32_t *s = &tmp[threadIdx.x/32][threadIdx.x%32];
		uint32_t *st = &tmp[threadIdx.x/32][(threadIdx.x + 4)%32];
		*s = bx.x; t2.x = *st;
		*s = bx.y; t2.y = *st;
		*s = bx.z; t2.z = *st;
		*s = bx.w; t2.w = *st;
		*s = start; int t2_start = *st + 4;
		bool c = (threadIdx.x & 0x4);
		*((uint4 *)(&scratch[c ? t2_start : start])) = (c ? t2 : t);
		*((uint4 *)(&scratch[c ? start : t2_start])) = (c ? t : t2);
	} else {
		*((uint4 *)(&scratch[start   ])) = b;
		*((uint4 *)(&scratch[start+16])) = bx;
	}
}

template <MemoryAccess SCHEME, int TEX_DIM> __device__  __forceinline__
void read_keys_direct(uint4 &b, uint4 &bx, uint32_t start)
{
	uint32_t *scratch;

	if (TEX_DIM == 0) scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/32];
	if (SCHEME == ANDERSEN) {
		extern __shared__ unsigned char shared[];
		uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);
		uint32_t *s = &tmp[threadIdx.x/32][threadIdx.x%32];
		*s = start; int t2_start = tmp[threadIdx.x/32][(threadIdx.x + 4)%32] + 4;
		if (TEX_DIM > 0) { start /= 4; t2_start /= 4; }
		bool c = (threadIdx.x & 0x4);
		if (TEX_DIM == 0) {
				b  = *((uint4 *)(&scratch[c ? t2_start : start]));
				bx = *((uint4 *)(&scratch[c ? start : t2_start]));
		} else if (TEX_DIM == 1) {
				b  = tex1Dfetch(texRef1D_4_V, c ? t2_start : start);
				bx = tex1Dfetch(texRef1D_4_V, c ? start : t2_start);
		} else if (TEX_DIM == 2) {
				b  = tex2D(texRef2D_4_V, 0.5f + ((c ? t2_start : start)%TEXWIDTH), 0.5f + ((c ? t2_start : start)/TEXWIDTH));
				bx = tex2D(texRef2D_4_V, 0.5f + ((c ? start : t2_start)%TEXWIDTH), 0.5f + ((c ? start : t2_start)/TEXWIDTH));
		}
		uint4 temp = b; b = (c ? bx : b); bx = (c ? temp : bx);
		uint32_t *st = &tmp[threadIdx.x/32][(threadIdx.x + 28)%32];
		*s = bx.x; bx.x = *st;
		*s = bx.y; bx.y = *st;
		*s = bx.z; bx.z = *st;
		*s = bx.w; bx.w = *st;
	} else {
				 if (TEX_DIM == 0) b = *((uint4 *)(&scratch[start]));
		else if (TEX_DIM == 1) b = tex1Dfetch(texRef1D_4_V, start/4);
		else if (TEX_DIM == 2) b = tex2D(texRef2D_4_V, 0.5f + ((start/4)%TEXWIDTH), 0.5f + ((start/4)/TEXWIDTH));
				 if (TEX_DIM == 0) bx = *((uint4 *)(&scratch[start+16]));
		else if (TEX_DIM == 1) bx = tex1Dfetch(texRef1D_4_V, (start+16)/4);
		else if (TEX_DIM == 2) bx = tex2D(texRef2D_4_V, 0.5f + (((start+16)/4)%TEXWIDTH), 0.5f + (((start+16)/4)/TEXWIDTH));
	}
}


__device__  __forceinline__
void primary_order_shuffle(uint4 &b, uint4 &bx)
{
	/* Inner loop shuffle targets */
	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	extern __shared__ unsigned char shared[];
	uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);
	unsigned int wrp  = threadIdx.x/32, lane = threadIdx.x%32;
	uint32_t *s = &tmp[wrp][lane];
	uint32_t *s1 = &tmp[wrp][x1];
	uint32_t *s2 = &tmp[wrp][x2];
	uint32_t *s3 = &tmp[wrp][x3];

	*s = b.w; b.w = *s1;
	*s = b.z; b.z = *s2;
	*s = b.y; b.y = *s3;
	uint32_t temp = b.y; b.y = b.w; b.w = temp;

	*s = bx.w; bx.w = *s1;
	*s = bx.z; bx.z = *s2;
	*s = bx.y; bx.y = *s3;
	temp = bx.y; bx.y = bx.w; bx.w = temp;
}

/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__
void load_key_salsa(const uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	// Read in permuted order. Key loads are not our bottleneck right now.
	b.x = B[key_offset + 4*thread_in_block + (thread_in_block+0)%4];
	b.y = B[key_offset + 4*thread_in_block + (thread_in_block+1)%4];
	b.z = B[key_offset + 4*thread_in_block + (thread_in_block+2)%4];
	b.w = B[key_offset + 4*thread_in_block + (thread_in_block+3)%4];
	bx.x = B[key_offset + 4*thread_in_block + (thread_in_block+0)%4 + 16];
	bx.y = B[key_offset + 4*thread_in_block + (thread_in_block+1)%4 + 16];
	bx.z = B[key_offset + 4*thread_in_block + (thread_in_block+2)%4 + 16];
	bx.w = B[key_offset + 4*thread_in_block + (thread_in_block+3)%4 + 16];

	primary_order_shuffle(b, bx);

}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__
void store_key_salsa(uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	primary_order_shuffle(b, bx);

	B[key_offset + 4*thread_in_block + (thread_in_block+0)%4] = b.x;
	B[key_offset + 4*thread_in_block + (thread_in_block+1)%4] = b.y;
	B[key_offset + 4*thread_in_block + (thread_in_block+2)%4] = b.z;
	B[key_offset + 4*thread_in_block + (thread_in_block+3)%4] = b.w;
	B[key_offset + 4*thread_in_block + (thread_in_block+0)%4 + 16] = bx.x;
	B[key_offset + 4*thread_in_block + (thread_in_block+1)%4 + 16] = bx.y;
	B[key_offset + 4*thread_in_block + (thread_in_block+2)%4 + 16] = bx.z;
	B[key_offset + 4*thread_in_block + (thread_in_block+3)%4 + 16] = bx.w;
}


/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__ 
void load_key_chacha(const uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	// Read in permuted order. Key loads are not our bottleneck right now.
	b.x = B[key_offset + 4*0 + thread_in_block%4];
	b.y = B[key_offset + 4*1 + thread_in_block%4];
	b.z = B[key_offset + 4*2 + thread_in_block%4];
	b.w = B[key_offset + 4*3 + thread_in_block%4];
	bx.x = B[key_offset + 4*0 + thread_in_block%4 + 16];
	bx.y = B[key_offset + 4*1 + thread_in_block%4 + 16];
	bx.z = B[key_offset + 4*2 + thread_in_block%4 + 16];
	bx.w = B[key_offset + 4*3 + thread_in_block%4 + 16];
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__
void store_key_chacha(uint32_t *B, const uint4 &b, const uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	B[key_offset + 4*0 + thread_in_block%4] = b.x;
	B[key_offset + 4*1 + thread_in_block%4] = b.y;
	B[key_offset + 4*2 + thread_in_block%4] = b.z;
	B[key_offset + 4*3 + thread_in_block%4] = b.w;
	B[key_offset + 4*0 + thread_in_block%4 + 16] = bx.x;
	B[key_offset + 4*1 + thread_in_block%4 + 16] = bx.y;
	B[key_offset + 4*2 + thread_in_block%4 + 16] = bx.z;
	B[key_offset + 4*3 + thread_in_block%4 + 16] = bx.w;
}


template <int ALGO> __device__  __forceinline__
void load_key(const uint32_t *B, uint4 &b, uint4 &bx)
{
	switch(ALGO) {
	case A_SCRYPT:      load_key_salsa(B, b, bx); break;
	case A_SCRYPT_JANE: load_key_chacha(B, b, bx); break;
	}
}

template <int ALGO> __device__  __forceinline__
void store_key(uint32_t *B, uint4 &b, uint4 &bx)
{
	switch(ALGO) {
	case A_SCRYPT:      store_key_salsa(B, b, bx); break;
	case A_SCRYPT_JANE: store_key_chacha(B, b, bx); break;
	}
}


/*
 * salsa_xor_core (Salsa20/8 cypher)
 * The original scrypt called:
 * xor_salsa8(&X[0], &X[16]); <-- the "b" loop
 * xor_salsa8(&X[16], &X[0]); <-- the "bx" loop
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 */

#define XOR_ROTATE_ADD(dst, s1, s2, amt) { uint32_t tmp = s1+s2; dst ^= ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__
void salsa_xor_core(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	extern __shared__ unsigned char shared[];
	uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);
	unsigned int wrp  = threadIdx.x/32, lane = threadIdx.x%32;
	uint32_t *s = &tmp[wrp][lane];
	uint32_t *s1 = &tmp[wrp][x1];
	uint32_t *s2 = &tmp[wrp][x2];
	uint32_t *s3 = &tmp[wrp][x3];

	uint4 x;

	b ^= bx;
	x = b;

	// Enter in "primary order" (t0 has  0,  4,  8, 12)
	//                          (t1 has  5,  9, 13,  1)
	//                          (t2 has 10, 14,  2,  6)
	//                          (t3 has 15,  3,  7, 11)

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Mixing phase of salsa
		XOR_ROTATE_ADD(x.y, x.x, x.w, 7);
		XOR_ROTATE_ADD(x.z, x.y, x.x, 9);
		XOR_ROTATE_ADD(x.w, x.z, x.y, 13);
		XOR_ROTATE_ADD(x.x, x.w, x.z, 18);

		/* Transpose rows and columns. */
		/* Unclear if this optimization is needed: These are ordered based
		 * upon the dependencies needed in the later xors. Compiler should be
		 * able to figure this out, but might as well give it a hand. */
		*s = x.y; x.y = *s3;
		*s = x.w; x.w = *s1;
		*s = x.z; x.z = *s2;

		/* The next XOR_ROTATE_ADDS could be written to be a copy-paste of the first,
		 * but the register targets are rewritten here to swap x[1] and x[3] so that
		 * they can be directly shuffled to and from our peer threads without
		 * reassignment. The reverse shuffle then puts them back in the right place.
		 */

		XOR_ROTATE_ADD(x.w, x.x, x.y, 7);
		XOR_ROTATE_ADD(x.z, x.w, x.x, 9);
		XOR_ROTATE_ADD(x.y, x.z, x.w, 13);
		XOR_ROTATE_ADD(x.x, x.y, x.z, 18);

		*s = x.w; x.w = *s3;
		*s = x.y; x.y = *s1;
		*s = x.z; x.z = *s2;
	}

	b += x;
	// The next two lines are the beginning of the BX-centric loop iteration
	bx ^= b;
	x = bx;

	// This is a copy of the same loop above, identical but stripped of comments.
	// Duplicated so that we can complete a bx-based loop with fewer register moves.
	#pragma unroll
	for (int j = 0; j < 4; j++) {
		XOR_ROTATE_ADD(x.y, x.x, x.w, 7);
		XOR_ROTATE_ADD(x.z, x.y, x.x, 9);
		XOR_ROTATE_ADD(x.w, x.z, x.y, 13);
		XOR_ROTATE_ADD(x.x, x.w, x.z, 18);

		*s = x.y; x.y = *s3;
		*s = x.w; x.w = *s1;
		*s = x.z; x.z = *s2;

		XOR_ROTATE_ADD(x.w, x.x, x.y, 7);
		XOR_ROTATE_ADD(x.z, x.w, x.x, 9);
		XOR_ROTATE_ADD(x.y, x.z, x.w, 13);
		XOR_ROTATE_ADD(x.x, x.y, x.z, 18);

		*s = x.w; x.w = *s3;
		*s = x.y; x.y = *s1;
		*s = x.z; x.z = *s2;
	}

	// At the end of these iterations, the data is in primary order again.
#undef XOR_ROTATE_ADD

	bx += x;
}


/*
 * chacha_xor_core (ChaCha20/8 cypher)
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 *
 * load_key and store_key must not use primary order when
 * using ChaCha20/8, but rather the basic transposed order
 * (referred to as "column mode" below)
 */

#define CHACHA_PRIMITIVE(pt, rt, ps, amt) { uint32_t tmp = rt ^ (pt += ps); rt = ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__
void chacha_xor_core(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	extern __shared__ unsigned char shared[];
	uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);
	unsigned int wrp  = threadIdx.x/32, lane = threadIdx.x%32;
	uint32_t *s = &tmp[wrp][lane];
	uint32_t *s1 = &tmp[wrp][x1];
	uint32_t *s2 = &tmp[wrp][x2];
	uint32_t *s3 = &tmp[wrp][x3];

	uint4 x;

	b ^= bx;
	x = b;

	// Enter in "column" mode (t0 has 0, 4,  8, 12)
	//                        (t1 has 1, 5,  9, 13)
	//                        (t2 has 2, 6, 10, 14)
	//                        (t3 has 3, 7, 11, 15)

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Column Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		*s = x.y; x.y = *s1;
		*s = x.z; x.z = *s2;
		*s = x.w; x.w = *s3;

		// Diagonal Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		*s = x.y; x.y = *s3;
		*s = x.z; x.z = *s2;
		*s = x.w; x.w = *s1;
	}

	b += x;
	// The next two lines are the beginning of the BX-centric loop iteration
	bx ^= b;
	x = bx;

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Column Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		*s = x.y; x.y = *s1;
		*s = x.z; x.z = *s2;
		*s = x.w; x.w = *s3;

		// Diagonal Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		*s = x.y; x.y = *s3;
		*s = x.z; x.z = *s2;
		*s = x.w; x.w = *s1;
	}

#undef CHACHA_PRIMITIVE

	bx += x;
}

template <int ALGO> __device__  __forceinline__
void block_mixer(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	switch(ALGO) {
	case A_SCRYPT:      salsa_xor_core(b, bx, x1, x2, x3); break;
	case A_SCRYPT_JANE: chacha_xor_core(b, bx, x1, x2, x3); break;
	}
}


/*
 * The hasher_gen_kernel operates on a group of 1024-bit input keys
 * in B, stored as:
 * B = { k1B k1Bx k2B k2Bx ... }
 * and fills up the scratchpad with the iterative hashes derived from
 * those keys:
 * scratch { k1h1B k1h1Bx K1h2B K1h2Bx ... K2h1B K2h1Bx K2h2B K2h2Bx ... }
 * scratch is 1024 times larger than the input keys B.
 * It is extremely important to stream writes effectively into scratch;
 * less important to coalesce the reads from B.
 *
 * Key ordering note: Keys are input from B in "original" order:
 * K = {k1, k2, k3, k4, k5, ..., kx15, kx16, kx17, ..., kx31 }
 * After inputting into kernel_gen, each component k and kx of the
 * key is transmuted into a permuted internal order to make processing faster:
 * K = k, kx with:
 * k = 0, 4, 8, 12, 5, 9, 13, 1, 10, 14, 2, 6, 15, 3, 7, 11
 * and similarly for kx.
 */

template <int ALGO, MemoryAccess SCHEME> __global__
void test_scrypt_core_kernelA(const uint32_t *d_idata, int begin, int end)
{
	uint4 b, bx;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4)) % c_SCRATCH_WU_PER_WARP;

	int i=begin;

	if (i == 0) {
		load_key<ALGO>(d_idata, b, bx);
		write_keys_direct<SCHEME>(b, bx, start);
		++i;
	} else read_keys_direct<SCHEME,0>(b, bx, start+32*(i-1));

	while (i < end) {
		block_mixer<ALGO>(b, bx, x1, x2, x3);
		write_keys_direct<SCHEME>(b, bx, start+32*i);
		++i;
	}
}

template <int ALGO, MemoryAccess SCHEME> __global__
void test_scrypt_core_kernelA_LG(const uint32_t *d_idata, int begin, int end, unsigned int LOOKUP_GAP)
{
	uint4 b, bx;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4)) % c_SCRATCH_WU_PER_WARP;

	int i=begin;

	if (i == 0) {
		load_key<ALGO>(d_idata, b, bx);
		write_keys_direct<SCHEME>(b, bx, start);
		++i;
	} else {
		int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
		read_keys_direct<SCHEME,0>(b, bx, start+32*pos);
		while(loop--) block_mixer<ALGO>(b, bx, x1, x2, x3);
	}

	while (i < end) {
		block_mixer<ALGO>(b, bx, x1, x2, x3);
		if (i % LOOKUP_GAP == 0)
			write_keys_direct<SCHEME>(b, bx, start+32*(i/LOOKUP_GAP));
		++i;
	}
}


/*
 * hasher_hash_kernel runs the second phase of scrypt after the scratch
 * buffer is filled with the iterative hashes: It bounces through
 * the scratch buffer in pseudorandom order, mixing the key as it goes.
 */

template <int ALGO, MemoryAccess SCHEME, int TEX_DIM> __global__
void test_scrypt_core_kernelB(uint32_t *d_odata, int begin, int end)
{
	extern __shared__ unsigned char shared[];
	uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);

	uint4 b, bx;

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH) + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4);
	if (TEX_DIM == 0) start %= c_SCRATCH_WU_PER_WARP;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	if (begin == 0) {
		read_keys_direct<SCHEME,TEX_DIM>(b, bx, start+32*c_N_1);
		block_mixer<ALGO>(b, bx, x1, x2, x3);
	} else load_key<ALGO>(d_odata, b, bx);

	for (int i = begin; i < end; i++) {
		tmp[threadIdx.x/32][threadIdx.x%32] = bx.x;
		int j = (tmp[threadIdx.x/32][(threadIdx.x & 0x1c)] & (c_N_1));
		uint4 t, tx; read_keys_direct<SCHEME,TEX_DIM>(t, tx, start+32*j);
		b ^= t; bx ^= tx;
		block_mixer<ALGO>(b, bx, x1, x2, x3);
	}

	store_key<ALGO>(d_odata, b, bx);
}

template <int ALGO, MemoryAccess SCHEME, int TEX_DIM> __global__
void test_scrypt_core_kernelB_LG(uint32_t *d_odata, int begin, int end, unsigned int LOOKUP_GAP)
{
	extern __shared__ unsigned char shared[];
	uint32_t (*tmp)[32+1] = (uint32_t (*)[32+1])(shared);

	uint4 b, bx;

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH) + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4);
	if (TEX_DIM == 0) start %= c_SCRATCH_WU_PER_WARP;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	if (begin == 0) {
		int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
		read_keys_direct<SCHEME,TEX_DIM>(b, bx, start+32*pos);
		while(loop--) block_mixer<ALGO>(b, bx, x1, x2, x3);
	} else load_key<ALGO>(d_odata, b, bx);

	for (int i = begin; i < end; i++) {
		tmp[threadIdx.x/32][threadIdx.x%32] = bx.x;
		int j = (tmp[threadIdx.x/32][(threadIdx.x & 0x1c)] & (c_N_1));
		int pos = j/LOOKUP_GAP, loop = j-pos*LOOKUP_GAP;
		uint4 t, tx; read_keys_direct<SCHEME,TEX_DIM>(t, tx, start+32*pos);
		while(loop--) block_mixer<ALGO>(t, tx, x1, x2, x3);
		b ^= t; bx ^= tx;
		block_mixer<ALGO>(b, bx, x1, x2, x3);
	}

	store_key<ALGO>(d_odata, b, bx);
}


TestKernel::TestKernel() : KernelInterface()
{
}

bool TestKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef1D_4_V.normalized = 0;
	texRef1D_4_V.filterMode = hipFilterModePoint;
	texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
	checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
	return true;
}

bool TestKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef2D_4_V.normalized = 0;
	texRef2D_4_V.filterMode = hipFilterModePoint;
	texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
	texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
	// maintain texture width of TEXWIDTH (max. limit is 65000)
	while (width > TEXWIDTH) { width /= 2; height *= 2; pitch /= 2; }
	while (width < TEXWIDTH) { width *= 2; height = (height+1)/2; pitch *= 2; }
	checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
	return true;
}

bool TestKernel::unbindtexture_1D()
{
	checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
	return true;
}

bool TestKernel::unbindtexture_2D()
{
	checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
	return true;
}

void TestKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool TestKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;

	// compute required shared memory per block for __shfl() emulation
	size_t shared = ((threads.x + 31) / 32) * (32+1) * sizeof(uint32_t);

	// make some constants available to kernel, update only initially and when changing
	static uint32_t prev_N[MAX_GPUS] = { 0 };

	if (N != prev_N[thr_id]) {
		uint32_t h_N = N;
		uint32_t h_N_1 = N-1;
		uint32_t h_SCRATCH = SCRATCH;
		uint32_t h_SCRATCH_WU_PER_WARP = (SCRATCH * WU_PER_WARP);
		uint32_t h_SCRATCH_WU_PER_WARP_1 = (SCRATCH * WU_PER_WARP) - 1;

		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH), &h_SCRATCH, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH_WU_PER_WARP), &h_SCRATCH_WU_PER_WARP, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH_WU_PER_WARP_1), &h_SCRATCH_WU_PER_WARP_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);

		prev_N[thr_id] = N;
	}

	// First phase: Sequential writes to scratchpad.

	int batch = device_batchsize[thr_id];

	unsigned int pos = 0;
	do {
		if (LOOKUP_GAP == 1) {
			if (IS_SCRYPT())      test_scrypt_core_kernelA<A_SCRYPT,    ANDERSEN> <<< grid, threads, shared, stream >>>(d_idata, pos, min(pos+batch, N));
			if (IS_SCRYPT_JANE()) test_scrypt_core_kernelA<A_SCRYPT_JANE, SIMPLE> <<< grid, threads, shared, stream >>>(d_idata, pos, min(pos+batch, N));
		} else {
			if (IS_SCRYPT())      test_scrypt_core_kernelA_LG<A_SCRYPT,    ANDERSEN> <<< grid, threads, shared, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (IS_SCRYPT_JANE())	test_scrypt_core_kernelA_LG<A_SCRYPT_JANE, SIMPLE> <<< grid, threads, shared, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
		}
		pos += batch;
	} while (pos < N);

	// Second phase: Random read access from scratchpad.
	pos = 0;
	do {
		if (LOOKUP_GAP == 1) {
			if (texture_cache == 0) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB<A_SCRYPT,    ANDERSEN, 0><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB<A_SCRYPT_JANE, SIMPLE, 0><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
			}
			else if (texture_cache == 1) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB<A_SCRYPT,    ANDERSEN, 1><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB<A_SCRYPT_JANE, SIMPLE, 1><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
			}
			else if (texture_cache == 2) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB<A_SCRYPT,    ANDERSEN, 2><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB<A_SCRYPT_JANE, SIMPLE, 2><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N));
			}
		} else {
			if (texture_cache == 0) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB_LG<A_SCRYPT,    ANDERSEN, 0><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB_LG<A_SCRYPT_JANE, SIMPLE, 0><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
			else if (texture_cache == 1) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB_LG<A_SCRYPT,    ANDERSEN, 1><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB_LG<A_SCRYPT_JANE, SIMPLE, 1><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
			else if (texture_cache == 2) {
				if (IS_SCRYPT())      test_scrypt_core_kernelB_LG<A_SCRYPT,    ANDERSEN, 2><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE()) test_scrypt_core_kernelB_LG<A_SCRYPT_JANE, SIMPLE, 2><<< grid, threads, shared, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
		}

		pos += batch;
	} while (pos < N);

	return success;
}
