#include "hip/hip_runtime.h"
//
//  =============== KECCAK part on nVidia GPU ======================
//
// The keccak512 (SHA-3) is used in the PBKDF2 for scrypt-jane coins
// in place of the SHA2 based PBKDF2 used in scrypt coins.
//
// NOTE: compile this .cu module for compute_20,sm_20 with --maxrregcount=64
//

#include <map>

#include "miner.h"
#include "cuda_helper.h"

#include "keccak.h"
#include "salsa_kernel.h"

// define some error checking macros
#define DELIMITER '/'
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#undef checkCudaErrors
#define checkCudaErrors(x) \
{ \
	hipGetLastError(); \
	x; \
	hipError_t err = hipGetLastError(); \
	if (err != hipSuccess && !abort_flag) \
		applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

#ifndef ROTL64
#define ROTL64(a,b) (((a) << (b)) | ((a) >> (64 - b)))
#endif

// CB
#define U32TO64_LE(p) \
	(((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
	*p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static __device__ void mycpy64(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
	for (int k=0; k < 16; ++k) d[k] = s[k];
}

static __device__ void mycpy56(uint32_t *d, const uint32_t *s) {
#pragma unroll 14
	for (int k=0; k < 14; ++k) d[k] = s[k];
}

static __device__ void mycpy32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
	for (int k=0; k < 8; ++k) d[k] = s[k];
}

static __device__ void mycpy8(uint32_t *d, const uint32_t *s) {
#pragma unroll 2
	for (int k=0; k < 2; ++k) d[k] = s[k];
}

static __device__ void mycpy4(uint32_t *d, const uint32_t *s) {
	*d = *s;
}

// ---------------------------- BEGIN keccak functions ------------------------------------

#define KECCAK_HASH "Keccak-512"

typedef struct keccak_hash_state_t {
	uint64_t state[25];                        // 25*2
	uint32_t buffer[72/4];                     // 72
} keccak_hash_state;

__device__ void statecopy0(keccak_hash_state *d, keccak_hash_state *s)
{
#pragma unroll 25
	for (int i=0; i < 25; ++i)
		d->state[i] = s->state[i];
}

__device__ void statecopy8(keccak_hash_state *d, keccak_hash_state *s)
{
#pragma unroll 25
	for (int i=0; i < 25; ++i)
		d->state[i] = s->state[i];
#pragma unroll 2
	for (int i=0; i < 2; ++i)
		d->buffer[i] = s->buffer[i];
}

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];
__constant__ uint32_t c_data[20];

__device__
void keccak_block(keccak_hash_state *S, const uint32_t *in)
{
	uint64_t *s = S->state, t[5], u[5], v, w;

	/* absorb input */
	#pragma unroll 9
	for (int i = 0; i < 72 / 8; i++, in += 2)
		s[i] ^= U32TO64_LE(in);

	for (int i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= c_keccak_round_constants[i];
	}
}

__device__
void keccak_hash_init(keccak_hash_state *S)
{
	#pragma unroll 25
	for (int i=0; i<25; ++i)
		S->state[i] = 0ULL;
}

// assuming there is no leftover data and exactly 72 bytes are incoming
// we can directly call into the block hashing function
__device__ void keccak_hash_update72(keccak_hash_state *S, const uint32_t *in) {
	keccak_block(S, in);
}

__device__ void keccak_hash_update8(keccak_hash_state *S, const uint32_t *in) {
	mycpy8(S->buffer, in);
}

__device__ void keccak_hash_update4_8(keccak_hash_state *S, const uint32_t *in) {
	mycpy4(S->buffer+8/4, in);
}

__device__ void keccak_hash_update4_56(keccak_hash_state *S, const uint32_t *in) {
	mycpy4(S->buffer+56/4, in);
}

__device__ void keccak_hash_update56(keccak_hash_state *S, const uint32_t *in) {
	mycpy56(S->buffer, in);
}

__device__ void keccak_hash_update64(keccak_hash_state *S, const uint32_t *in) {
	mycpy64(S->buffer, in);
}

__device__
void keccak_hash_finish8(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[8/4] = 0x01;
	#pragma unroll 15
	for (int i=8/4+1; i < 72/4; ++i) S->buffer[i] = 0;
	S->buffer[72/4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

	#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i/4]), S->state[i / 8]);
	}
}

__device__
void keccak_hash_finish12(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[12/4] = 0x01;
	#pragma unroll 14
	for (int i=12/4+1; i < 72/4; ++i) S->buffer[i] = 0;
	S->buffer[72/4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

	#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i/4]), S->state[i / 8]);
	}
}

__device__
void keccak_hash_finish60(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[60/4] = 0x01;
	#pragma unroll
	for (int i=60/4+1; i < 72/4; ++i) S->buffer[i] = 0;
	S->buffer[72/4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

	#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i/4]), S->state[i / 8]);
	}
}

__device__
void keccak_hash_finish64(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[64/4] = 0x01;
	#pragma unroll
	for (int i=64/4+1; i < 72/4; ++i) S->buffer[i] = 0;
	S->buffer[72/4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

	#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i/4]), S->state[i / 8]);
	}
}

// ---------------------------- END keccak functions ------------------------------------

// ---------------------------- BEGIN PBKDF2 functions ------------------------------------

typedef struct pbkdf2_hmac_state_t {
	keccak_hash_state inner, outer;
} pbkdf2_hmac_state;


__device__ void pbkdf2_hash(uint32_t *hash, const uint32_t *m)
{
	keccak_hash_state st;
	keccak_hash_init(&st);
	keccak_hash_update72(&st, m);
	keccak_hash_update8(&st, m+72/4);
	keccak_hash_finish8(&st, hash);
}

/* hmac */
__device__
void pbkdf2_hmac_init80(pbkdf2_hmac_state *st, const uint32_t *key)
{
	uint32_t pad[72/4] = { 0 };
	//#pragma unroll 18
	//for (int i = 0; i < 72/4; i++)
	//	pad[i] = 0;

	keccak_hash_init(&st->inner);
	keccak_hash_init(&st->outer);

	/* key > blocksize bytes, hash it */
	pbkdf2_hash(pad, key);

	/* inner = (key ^ 0x36) */
	/* h(inner || ...) */
	#pragma unroll 18
	for (int i = 0; i < 72/4; i++)
		pad[i] ^= 0x36363636U;
	keccak_hash_update72(&st->inner, pad);

	/* outer = (key ^ 0x5c) */
	/* h(outer || ...) */
	#pragma unroll 18
	for (int i = 0; i < 72/4; i++)
		pad[i] ^= 0x6a6a6a6aU;
	keccak_hash_update72(&st->outer, pad);
}

// assuming there is no leftover data and exactly 72 bytes are incoming
// we can directly call into the block hashing function
__device__ void pbkdf2_hmac_update72(pbkdf2_hmac_state *st, const uint32_t *m) {
	/* h(inner || m...) */
	keccak_hash_update72(&st->inner, m);
}

__device__ void pbkdf2_hmac_update8(pbkdf2_hmac_state *st, const uint32_t *m) {
	/* h(inner || m...) */
	keccak_hash_update8(&st->inner, m);
}

__device__ void pbkdf2_hmac_update4_8(pbkdf2_hmac_state *st, const uint32_t *m) {
	/* h(inner || m...) */
	keccak_hash_update4_8(&st->inner, m);
}

__device__ void pbkdf2_hmac_update4_56(pbkdf2_hmac_state *st, const uint32_t *m) {
	/* h(inner || m...) */
	keccak_hash_update4_56(&st->inner, m);
}

__device__ void pbkdf2_hmac_update56(pbkdf2_hmac_state *st, const uint32_t *m) {
	/* h(inner || m...) */
	keccak_hash_update56(&st->inner, m);
}

__device__ void pbkdf2_hmac_finish12(pbkdf2_hmac_state *st, uint32_t *mac) {
	/* h(inner || m) */
	uint32_t innerhash[16];
	keccak_hash_finish12(&st->inner, innerhash);

	/* h(outer || h(inner || m)) */
	keccak_hash_update64(&st->outer, innerhash);
	keccak_hash_finish64(&st->outer, mac);
}

__device__ void pbkdf2_hmac_finish60(pbkdf2_hmac_state *st, uint32_t *mac) {
	/* h(inner || m) */
	uint32_t innerhash[16];
	keccak_hash_finish60(&st->inner, innerhash);

	/* h(outer || h(inner || m)) */
	keccak_hash_update64(&st->outer, innerhash);
	keccak_hash_finish64(&st->outer, mac);
}

__device__ void pbkdf2_statecopy8(pbkdf2_hmac_state *d, pbkdf2_hmac_state *s) {
	statecopy8(&d->inner, &s->inner);
	statecopy0(&d->outer, &s->outer);
}

// ---------------------------- END PBKDF2 functions ------------------------------------

__global__ __launch_bounds__(128)
void cuda_pre_keccak512(uint32_t *g_idata, uint32_t nonce)
{
	uint32_t data[20];

	const uint32_t thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	nonce   += thread;
	g_idata += thread * 32;

	#pragma unroll
	for (int i=0; i<19; i++)
		data[i] = cuda_swab32(c_data[i]);
	data[19] = cuda_swab32(nonce);

//    scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)data, 80, (uint8_t*)g_idata, 128);

	pbkdf2_hmac_state hmac_pw;

	/* hmac(password, ...) */
	pbkdf2_hmac_init80(&hmac_pw, data);

	/* hmac(password, salt...) */
	pbkdf2_hmac_update72(&hmac_pw, data);
	pbkdf2_hmac_update8(&hmac_pw, data+72/4);

	pbkdf2_hmac_state work;
	uint32_t ti[16];

	/* U1 = hmac(password, salt || be(i)) */
	uint32_t be = 0x01000000U;//cuda_swab32(1);
	pbkdf2_statecopy8(&work, &hmac_pw);
	pbkdf2_hmac_update4_8(&work, &be);
	pbkdf2_hmac_finish12(&work, ti);
	mycpy64(g_idata, ti);

	be = 0x02000000U;//cuda_swab32(2);
	pbkdf2_statecopy8(&work, &hmac_pw);
	pbkdf2_hmac_update4_8(&work, &be);
	pbkdf2_hmac_finish12(&work, ti);
	mycpy64(g_idata+16, ti);
}


__global__ __launch_bounds__(128)
void cuda_post_keccak512(uint32_t *g_odata, uint32_t *g_hash, uint32_t nonce)
{
	uint32_t data[20];

	const uint32_t thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	g_hash  += thread * 8;
	g_odata += thread * 32;
	nonce   += thread;

	#pragma unroll
	for (int i=0; i<19; i++)
		data[i] = cuda_swab32(c_data[i]);
	data[19] = cuda_swab32(nonce);

//	scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)g_odata, 128, (uint8_t*)g_hash, 32);

	pbkdf2_hmac_state hmac_pw;

	/* hmac(password, ...) */
	pbkdf2_hmac_init80(&hmac_pw, data);

	/* hmac(password, salt...) */
	pbkdf2_hmac_update72(&hmac_pw, g_odata);
	pbkdf2_hmac_update56(&hmac_pw, g_odata+72/4);

	uint32_t ti[16];

	/* U1 = hmac(password, salt || be(i)) */
	uint32_t be = 0x01000000U;//cuda_swab32(1);
	pbkdf2_hmac_update4_56(&hmac_pw, &be);
	pbkdf2_hmac_finish60(&hmac_pw, ti);
	mycpy32(g_hash, ti);
}

//
// callable host code to initialize constants and to call kernels
//

extern "C" void prepare_keccak512(int thr_id, const uint32_t host_pdata[20])
{
	static bool init[MAX_GPUS] = { 0 };

	if (!init[thr_id])
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_keccak_round_constants), host_keccak_round_constants, sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice));
		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_data), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

extern "C" void pre_keccak512(int thr_id, int stream, uint32_t nonce, int throughput)
{
	dim3 block(128);
	dim3 grid((throughput+127)/128);

	cuda_pre_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_idata[stream][thr_id], nonce);
}

extern "C" void post_keccak512(int thr_id, int stream, uint32_t nonce, int throughput)
{
	dim3 block(128);
	dim3 grid((throughput+127)/128);

	cuda_post_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_odata[stream][thr_id], context_hash[stream][thr_id], nonce);
}
