#include "hip/hip_runtime.h"
//
// Kernel that runs best on Fermi devices
//
// - shared memory use reduced by nearly factor 2 over legacy kernel
//   by transferring only half work units (16 x uint32_t) at once.
// - uses ulong2/uint4 based memory transfers (each thread moves 16 bytes),
//   allowing for shorter unrolled loops. This relies on Fermi's better
//   memory controllers to get high memory troughput.
//
// NOTE: compile this .cu module for compute_20,sm_20 with --maxrregcount=63
//
// TODO: batch-size support for this kernel
//

#include <map>

#include <hip/hip_runtime.h>
#include "miner.h"

#include "salsa_kernel.h"
#include "fermi_kernel.h"

#define THREADS_PER_WU 1  // single thread per hash

#define TEXWIDTH 32768

// forward references
template <int ALGO> __global__ void fermi_scrypt_core_kernelA(uint32_t *g_idata, unsigned int N);
template <int ALGO> __global__ void fermi_scrypt_core_kernelB(uint32_t *g_odata, unsigned int N);
template <int ALGO, int TEX_DIM> __global__ void fermi_scrypt_core_kernelB_tex(uint32_t *g_odata, unsigned int N);
template <int ALGO> __global__ void fermi_scrypt_core_kernelA_LG(uint32_t *g_idata, unsigned int N, unsigned int LOOKUP_GAP);
template <int ALGO> __global__ void fermi_scrypt_core_kernelB_LG(uint32_t *g_odata, unsigned int N, unsigned int LOOKUP_GAP);
template <int ALGO, int TEX_DIM> __global__ void fermi_scrypt_core_kernelB_LG_tex(uint32_t *g_odata, unsigned int N, unsigned int LOOKUP_GAP);

// scratchbuf constants (pointers to scratch buffer for each warp, i.e. 32 hashes)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

FermiKernel::FermiKernel() : KernelInterface()
{
}

bool FermiKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef1D_4_V.normalized = 0;
	texRef1D_4_V.filterMode = hipFilterModePoint;
	texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
	checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
	return true;
}

bool FermiKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef2D_4_V.normalized = 0;
	texRef2D_4_V.filterMode = hipFilterModePoint;
	texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
	texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
	// maintain texture width of TEXWIDTH (max. limit is 65000)
	while (width > TEXWIDTH) { width /= 2; height *= 2; pitch /= 2; }
	while (width < TEXWIDTH) { width *= 2; height = (height+1)/2; pitch *= 2; }
//    fprintf(stderr, "total size: %u, %u bytes\n", pitch * height, width * sizeof(uint32_t) * 4 * height);
//    fprintf(stderr, "binding width width=%d, height=%d, pitch=%d\n", width, height,pitch);
	checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
	return true;
}

bool FermiKernel::unbindtexture_1D()
{
	checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
	return true;
}

bool FermiKernel::unbindtexture_2D()
{
	checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
	return true;
}

void FermiKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool FermiKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;

	int shared = WARPS_PER_BLOCK * WU_PER_WARP * (16+4) * sizeof(uint32_t);

	// First phase: Sequential writes to scratchpad.

	if (LOOKUP_GAP == 1) {
		  if (IS_SCRYPT())      fermi_scrypt_core_kernelA<A_SCRYPT><<< grid, threads, shared, stream >>>(d_idata, N);
		  if (IS_SCRYPT_JANE()) fermi_scrypt_core_kernelA<A_SCRYPT_JANE><<< grid, threads, shared, stream >>>(d_idata, N);
	} else {
		  if (IS_SCRYPT())      fermi_scrypt_core_kernelA_LG<A_SCRYPT><<< grid, threads, shared, stream >>>(d_idata, N, LOOKUP_GAP);
		  if (IS_SCRYPT_JANE()) fermi_scrypt_core_kernelA_LG<A_SCRYPT_JANE><<< grid, threads, shared, stream >>>(d_idata, N, LOOKUP_GAP);
	}

	// Second phase: Random read access from scratchpad.

	if (LOOKUP_GAP == 1) {
		if (texture_cache) {
			if (texture_cache == 1) {
				if (IS_SCRYPT())      fermi_scrypt_core_kernelB_tex<A_SCRYPT,1><<< grid, threads, shared, stream >>>(d_odata, N);
				if (IS_SCRYPT_JANE()) fermi_scrypt_core_kernelB_tex<A_SCRYPT_JANE,1><<< grid, threads, shared, stream >>>(d_odata, N);
			} else if (texture_cache == 2) {
				if (IS_SCRYPT())      fermi_scrypt_core_kernelB_tex<A_SCRYPT,2><<< grid, threads, shared, stream >>>(d_odata, N);
				if (IS_SCRYPT_JANE()) fermi_scrypt_core_kernelB_tex<A_SCRYPT_JANE,2><<< grid, threads, shared, stream >>>(d_odata, N);
			}
			else success = false;
		} else {
			if (IS_SCRYPT())      fermi_scrypt_core_kernelB<A_SCRYPT><<< grid, threads, shared, stream >>>(d_odata, N);
			if (IS_SCRYPT_JANE()) fermi_scrypt_core_kernelB<A_SCRYPT_JANE><<< grid, threads, shared, stream >>>(d_odata, N);
		}
	} else {
		if (texture_cache) {
			if (texture_cache == 1) {
				if (IS_SCRYPT())       fermi_scrypt_core_kernelB_LG_tex<A_SCRYPT,1><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
				if (IS_SCRYPT_JANE())  fermi_scrypt_core_kernelB_LG_tex<A_SCRYPT_JANE,1><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
			} else if (texture_cache == 2) {
				if (IS_SCRYPT())       fermi_scrypt_core_kernelB_LG_tex<A_SCRYPT,2><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
				if (IS_SCRYPT_JANE())  fermi_scrypt_core_kernelB_LG_tex<A_SCRYPT_JANE,2><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
			}
			else success = false;
		} else {
			if (IS_SCRYPT())       fermi_scrypt_core_kernelB_LG<A_SCRYPT><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
			if (IS_SCRYPT_JANE())  fermi_scrypt_core_kernelB_LG<A_SCRYPT_JANE><<< grid, threads, shared, stream >>>(d_odata, N, LOOKUP_GAP);
		}
	}

	return success;
}

#if 0

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

#define QUARTER(a,b,c,d) \
	a += b; d ^= a; d = ROTL(d,16); \
	c += d; b ^= c; b = ROTL(b,12); \
	a += b; d ^= a; d = ROTL(d,8); \
	c += d; b ^= c; b = ROTL(b,7);

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#else

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

#define ADD4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 += s1; d2 += s2; d3 += s3; d4 += s4;

#define XOR4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 ^= s1; d2 ^= s2; d3 ^= s3; d4 ^= s4;

#define ROTL4(d1,d2,d3,d4,amt) \
	d1 = ROTL(d1, amt); d2 = ROTL(d2, amt); d3 = ROTL(d3, amt); d4 = ROTL(d4, amt);

#define QROUND(a1,a2,a3,a4, b1,b2,b3,b4, c1,c2,c3,c4, amt) \
	ADD4 (a1,a2,a3,a4, c1,c2,c3,c4) \
	XOR4 (b1,b2,b3,b4, a1,a2,a3,a4) \
	ROTL4(b1,b2,b3,b4, amt)

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#endif

#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<7) | ((a00)>>25) );\
a1^=(((a10)<<7) | ((a10)>>25) );\
a2^=(((a20)<<7) | ((a20)>>25) );\
a3^=(((a30)<<7) | ((a30)>>25) );\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<9) | ((a00)>>23) );\
a1^=(((a10)<<9) | ((a10)>>23) );\
a2^=(((a20)<<9) | ((a20)>>23) );\
a3^=(((a30)<<9) | ((a30)>>23) );\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<13) | ((a00)>>19) );\
a1^=(((a10)<<13) | ((a10)>>19) );\
a2^=(((a20)<<13) | ((a20)>>19) );\
a3^=(((a30)<<13) | ((a30)>>19) );\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<18) | ((a00)>>14) );\
a1^=(((a10)<<18) | ((a10)>>14) );\
a2^=(((a20)<<18) | ((a20)>>14) );\
a3^=(((a30)<<18) | ((a30)>>14) );\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

static __device__ __forceinline__ uint4& operator^=(uint4& left, const uint4& right)
{
	left.x ^= right.x;
	left.y ^= right.y;
	left.z ^= right.z;
	left.w ^= right.w;
	return left;
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel for Fermi class devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int ALGO> __global__
void fermi_scrypt_core_kernelA(uint32_t *g_idata, unsigned int N)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;
	const unsigned int LOOKUP_GAP = 1;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / WU_PER_WARP]  + SCRATCH*Y + Z;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&V[SCRATCH*wu])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&V[SCRATCH*wu+16])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	for (int i = 1; i < N; i++) {

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C); xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}

#pragma unroll 4
		for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)(&V[SCRATCH*wu + i*32])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
		for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)(&V[SCRATCH*wu + i*32 + 16])) = *((ulonglong2*)XB[wu]);
	}
}

template <int ALGO> __global__
void fermi_scrypt_core_kernelB(uint32_t *g_odata, unsigned int N)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;
	const unsigned int LOOKUP_GAP = 1;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + (N-1)*32]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + (N-1)*32 + 16]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	switch(ALGO) {
	case A_SCRYPT:      xor_salsa8(B, C); xor_salsa8(C, B); break;
	case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
	}

	for (int i = 0; i < N; i++) {

		XX[16] = 32 * (C[0].x & (N-1));

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z]]));
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) B[idx] ^= *((uint4*)&XX[4*idx]);

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z] + 16]));
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) C[idx] ^= *((uint4*)&XX[4*idx]);

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C); xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}
	}

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);

}

template <int ALGO, int TEX_DIM> __global__ void
fermi_scrypt_core_kernelB_tex(uint32_t *g_odata, unsigned int N)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;
	const unsigned int LOOKUP_GAP = 1;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_odata += 32 * offset;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + (N-1)*32 + Z)/4;
		*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
					tex1Dfetch(texRef1D_4_V, loc) :
					tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + (N-1)*32 + 16+Z)/4;
		*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
					tex1Dfetch(texRef1D_4_V, loc) :
					tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	switch(ALGO) {
	case A_SCRYPT:      xor_salsa8(B, C); xor_salsa8(C, B); break;
	case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
	}

	for (int i = 0; i < N; i++) {

		XX[16] = 32 * (C[0].x & (N-1));

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + Z)/4;
			*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
						tex1Dfetch(texRef1D_4_V, loc) :
						tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) B[idx] ^= *((uint4*)&XX[4*idx]);

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + 16+Z)/4;
			*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
						tex1Dfetch(texRef1D_4_V, loc) :
						tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) C[idx] ^= *((uint4*)&XX[4*idx]);

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}
	}

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);
}

//
// Lookup-Gap variations of the above functions
//

template <int ALGO> __global__ void
fermi_scrypt_core_kernelA_LG(uint32_t *g_idata, unsigned int N, unsigned int LOOKUP_GAP)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / WU_PER_WARP]  + SCRATCH*Y + Z;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&V[SCRATCH*wu])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+Z]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&V[SCRATCH*wu+16])) = *((ulonglong2*)XB[wu]) = *((ulonglong2*)(&g_idata[32*(wu+Y)+16+Z]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	for (int i = 1; i < N; i++) {

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}

		if (i % LOOKUP_GAP == 0) {
#pragma unroll 4
			for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
			for (int wu=0; wu < 32; wu+=8)
				*((ulonglong2*)(&V[SCRATCH*wu + (i/LOOKUP_GAP)*32])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
			for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
			for (int wu=0; wu < 32; wu+=8)
				*((ulonglong2*)(&V[SCRATCH*wu + (i/LOOKUP_GAP)*32 + 16])) = *((ulonglong2*)XB[wu]);
		}
	}
}

template <int ALGO> __global__ void
fermi_scrypt_core_kernelB_LG(uint32_t *g_odata, unsigned int N, unsigned int LOOKUP_GAP)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

	uint32_t pos = (N-1)/LOOKUP_GAP; uint32_t loop = 1 + (N-1)-pos*LOOKUP_GAP;
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + pos*32]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + pos*32 + 16]));
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	while (loop--)
		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}

	for (int i = 0; i < N; i++) {

		uint32_t j = C[0].x & (N-1);
		uint32_t pos = j / LOOKUP_GAP; uint32_t loop = j - pos*LOOKUP_GAP;
		XX[16] = 32 * pos;

		uint4 b[4], c[4];
#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z]]));
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) b[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8)
			*((ulonglong2*)XB[wu]) = *((ulonglong2*)(&V[SCRATCH*wu + XB[wu][16-Z] + 16]));
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) c[idx] = *((uint4*)&XX[4*idx]);

		while (loop--)
			switch(ALGO) {
			case A_SCRYPT:      xor_salsa8(b, c);  xor_salsa8(c, b); break;
			case A_SCRYPT_JANE: xor_chacha8(b, c); xor_chacha8(c, b); break;
			}

#pragma unroll 4
		for (int idx=0; idx < 4; idx++) B[idx] ^= b[idx];
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) C[idx] ^= c[idx];

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}
	}

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);

}

template <int ALGO, int TEX_DIM> __global__ void
fermi_scrypt_core_kernelB_LG_tex(uint32_t *g_odata, unsigned int N, unsigned int LOOKUP_GAP)
{
	extern __shared__ unsigned char x[];
	uint32_t ((*X)[WU_PER_WARP][16+4]) = (uint32_t (*)[WU_PER_WARP][16+4]) x;

	int warpIdx        = threadIdx.x / warpSize;
	int warpThread     = threadIdx.x % warpSize;

	// variables supporting the large memory transaction magic
	unsigned int Y = warpThread/4;
	unsigned int Z = 4*(warpThread%4);

	// add block specific offsets
	int WARPS_PER_BLOCK = blockDim.x / 32;
	int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
	g_odata += 32 * offset;

	// registers to store an entire work unit
	uint4 B[4], C[4];

	uint32_t ((*XB)[16+4]) = (uint32_t (*)[16+4])&X[warpIdx][Y][Z];
	uint32_t *XX = X[warpIdx][warpThread];

	uint32_t pos = (N-1)/LOOKUP_GAP; uint32_t loop = 1 + (N-1)-pos*LOOKUP_GAP;
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + pos*32 + Z)/4;
		*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
					tex1Dfetch(texRef1D_4_V, loc) :
					tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) B[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + pos*32 + 16+Z)/4;
		*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
					tex1Dfetch(texRef1D_4_V, loc) :
					tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
	for (int idx=0; idx < 4; idx++) C[idx] = *((uint4*)&XX[4*idx]);

	while (loop--)
		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}

	for (int i = 0; i < N; i++) {

		uint32_t j = C[0].x & (N-1);
		uint32_t pos = j / LOOKUP_GAP; uint32_t loop = j - pos*LOOKUP_GAP;
		XX[16] = 32 * pos;

		uint4 b[4], c[4];
#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + Z)/4;
			*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
						tex1Dfetch(texRef1D_4_V, loc) :
						tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) b[idx] = *((uint4*)&XX[4*idx]);

#pragma unroll 4
		for (int wu=0; wu < 32; wu+=8) { unsigned int loc = (SCRATCH*(offset+wu+Y) + XB[wu][16-Z] + 16+Z)/4;
			*((uint4*)XB[wu]) = ((TEX_DIM == 1) ?
						tex1Dfetch(texRef1D_4_V, loc) :
						tex2D(texRef2D_4_V, 0.5f + (loc%TEXWIDTH), 0.5f + (loc/TEXWIDTH))); }
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) c[idx] = *((uint4*)&XX[4*idx]);

		while (loop--)
			switch(ALGO) {
			case A_SCRYPT:      xor_salsa8(b, c);  xor_salsa8(c, b); break;
			case A_SCRYPT_JANE: xor_chacha8(b, c); xor_chacha8(c, b); break;
			}

#pragma unroll 4
		for (int idx=0; idx < 4; idx++) B[idx] ^= b[idx];
#pragma unroll 4
		for (int idx=0; idx < 4; idx++) C[idx] ^= c[idx];

		switch(ALGO) {
		case A_SCRYPT:      xor_salsa8(B, C);  xor_salsa8(C, B); break;
		case A_SCRYPT_JANE: xor_chacha8(B, C); xor_chacha8(C, B); break;
		}
	}

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = B[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+Z])) = *((ulonglong2*)XB[wu]);

#pragma unroll 4
	for (int idx=0; idx < 4; idx++) *((uint4*)&XX[4*idx]) = C[idx];
#pragma unroll 4
	for (int wu=0; wu < 32; wu+=8)
		*((ulonglong2*)(&g_odata[32*(wu+Y)+16+Z])) = *((ulonglong2*)XB[wu]);
}
