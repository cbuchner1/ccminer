#include "hip/hip_runtime.h"
/* Copyright (C) 2013 David G. Andersen. All rights reserved.
 * with modifications by Christian Buchner
 *
 * Use of this code is covered under the Apache 2.0 license, which
 * can be found in the file "LICENSE"
 */

// TODO: attempt V.Volkov style ILP (factor 4)

#include <map>

#include <hip/hip_runtime.h>
#include <cuda_helper.h>

#include "miner.h"

#include "salsa_kernel.h"
#include "kepler_kernel.h"

#define TEXWIDTH 32768
#define THREADS_PER_WU 4  // four threads per hash

#if CUDA_VERSION >= 9000 && __CUDA_ARCH__ >= 300
#define __shfl2(var, srcLane)  __shfl_sync(0xFFFFFFFFu, var, srcLane)
#else
#define __shfl2 __shfl
#endif

typedef enum
{
		ANDERSEN,
		SIMPLE
} MemoryAccess;

// scratchbuf constants (pointers to scratch buffer for each warp, i.e. 32 hashes)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1;                   // N-1
// scratch buffer size SCRATCH
__constant__ uint32_t c_SCRATCH;
__constant__ uint32_t c_SCRATCH_WU_PER_WARP;   // (SCRATCH * WU_PER_WARP)
__constant__ uint32_t c_SCRATCH_WU_PER_WARP_1; // (SCRATCH * WU_PER_WARP) - 1

// using texture references for the "tex" variants of the B kernels
texture<uint4, 1, hipReadModeElementType> texRef1D_4_V;
texture<uint4, 2, hipReadModeElementType> texRef2D_4_V;

template <int ALGO> __device__  __forceinline__ void block_mixer(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3);

static __host__ __device__ uint4& operator ^= (uint4& left, const uint4& right) {
	left.x ^= right.x;
	left.y ^= right.y;
	left.z ^= right.z;
	left.w ^= right.w;
	return left;
}

static __host__ __device__ uint4& operator += (uint4& left, const uint4& right) {
	left.x += right.x;
	left.y += right.y;
	left.z += right.z;
	left.w += right.w;
	return left;
}

static __device__ uint4 shfl4(const uint4 bx, int target_thread) {
	return make_uint4(
		__shfl2((int)bx.x, target_thread),
		__shfl2((int)bx.y, target_thread),
		__shfl2((int)bx.z, target_thread),
		__shfl2((int)bx.w, target_thread)
	);
}

/* write_keys writes the 8 keys being processed by a warp to the global
 * scratchpad. To effectively use memory bandwidth, it performs the writes
 * (and reads, for read_keys) 128 bytes at a time per memory location
 * by __shfl'ing the 4 entries in bx to the threads in the next-up
 * thread group. It then has eight threads together perform uint4
 * (128 bit) writes to the destination region. This seems to make
 * quite effective use of memory bandwidth. An approach that spread
 * uint32s across more threads was slower because of the increased
 * computation it required.
 *
 * "start" is the loop iteration producing the write - the offset within
 * the block's memory.
 *
 * Internally, this algorithm first __shfl's the 4 bx entries to
 * the next up thread group, and then uses a conditional move to
 * ensure that odd-numbered thread groups exchange the b/bx ordering
 * so that the right parts are written together.
 *
 * Thanks to Babu for helping design the 128-bit-per-write version.
 *
 * _direct lets the caller specify the absolute start location instead of
 * the relative start location, as an attempt to reduce some recomputation.
 */

template <MemoryAccess SCHEME> __device__ __forceinline__
void write_keys_direct(const uint4 &b, const uint4 &bx, uint32_t start)
{
	uint32_t *scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/32];

	if (SCHEME == ANDERSEN) {
		int target_thread = (threadIdx.x + 4)%32;
		uint4 t = b, t2 = shfl4(bx, target_thread);
		int t2_start = __shfl2((int)start, target_thread) + 4;
		bool c = (threadIdx.x & 0x4);
		*((uint4 *)(&scratch[c ? t2_start : start])) = (c ? t2 : t);
		*((uint4 *)(&scratch[c ? start : t2_start])) = (c ? t : t2);
	} else if (SCHEME == SIMPLE) {
		*((uint4 *)(&scratch[start   ])) = b;
		*((uint4 *)(&scratch[start+16])) = bx;
	}
}

template <MemoryAccess SCHEME, int TEX_DIM> __device__  __forceinline__
void read_keys_direct(uint4 &b, uint4 &bx, uint32_t start)
{
	uint32_t *scratch;

	if (TEX_DIM == 0) scratch = c_V[(blockIdx.x*blockDim.x + threadIdx.x)/32];
	if (SCHEME == ANDERSEN) {
		int t2_start = __shfl2((int)start, (threadIdx.x + 4)%32) + 4;
		if (TEX_DIM > 0) { start /= 4; t2_start /= 4; }
		bool c = (threadIdx.x & 0x4);
		if (TEX_DIM == 0) {
				b  = *((uint4 *)(&scratch[c ? t2_start : start]));
				bx = *((uint4 *)(&scratch[c ? start : t2_start]));
		} else if (TEX_DIM == 1) {
				b  = tex1Dfetch(texRef1D_4_V, c ? t2_start : start);
				bx = tex1Dfetch(texRef1D_4_V, c ? start : t2_start);
		} else if (TEX_DIM == 2) {
				b  = tex2D(texRef2D_4_V, 0.5f + ((c ? t2_start : start)%TEXWIDTH), 0.5f + ((c ? t2_start : start)/TEXWIDTH));
				bx = tex2D(texRef2D_4_V, 0.5f + ((c ? start : t2_start)%TEXWIDTH), 0.5f + ((c ? start : t2_start)/TEXWIDTH));
		}
		uint4 tmp = b; b = (c ? bx : b); bx = (c ? tmp : bx);
		bx = shfl4(bx, (threadIdx.x + 28)%32);
	} else {
				 if (TEX_DIM == 0) b = *((uint4 *)(&scratch[start]));
		else if (TEX_DIM == 1) b = tex1Dfetch(texRef1D_4_V, start/4);
		else if (TEX_DIM == 2) b = tex2D(texRef2D_4_V, 0.5f + ((start/4)%TEXWIDTH), 0.5f + ((start/4)/TEXWIDTH));
				 if (TEX_DIM == 0) bx = *((uint4 *)(&scratch[start+16]));
		else if (TEX_DIM == 1) bx = tex1Dfetch(texRef1D_4_V, (start+16)/4);
		else if (TEX_DIM == 2) bx = tex2D(texRef2D_4_V, 0.5f + (((start+16)/4)%TEXWIDTH), 0.5f + (((start+16)/4)/TEXWIDTH));
	}
}


__device__  __forceinline__
void primary_order_shuffle(uint4 &b, uint4 &bx)
{
	/* Inner loop shuffle targets */
	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	b.w = __shfl2((int)b.w, x1);
	b.z = __shfl2((int)b.z, x2);
	b.y = __shfl2((int)b.y, x3);

	uint32_t tmp = b.y; b.y = b.w; b.w = tmp;

	bx.w = __shfl2((int)bx.w, x1);
	bx.z = __shfl2((int)bx.z, x2);
	bx.y = __shfl2((int)bx.y, x3);
	tmp = bx.y; bx.y = bx.w; bx.w = tmp;
}

/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__
void load_key_salsa(const uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	// Read in permuted order. Key loads are not our bottleneck right now.
	b.x = B[key_offset + 4*thread_in_block + (thread_in_block+0)%4];
	b.y = B[key_offset + 4*thread_in_block + (thread_in_block+1)%4];
	b.z = B[key_offset + 4*thread_in_block + (thread_in_block+2)%4];
	b.w = B[key_offset + 4*thread_in_block + (thread_in_block+3)%4];
	bx.x = B[key_offset + 4*thread_in_block + (thread_in_block+0)%4 + 16];
	bx.y = B[key_offset + 4*thread_in_block + (thread_in_block+1)%4 + 16];
	bx.z = B[key_offset + 4*thread_in_block + (thread_in_block+2)%4 + 16];
	bx.w = B[key_offset + 4*thread_in_block + (thread_in_block+3)%4 + 16];

	primary_order_shuffle(b, bx);
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__
void store_key_salsa(uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	primary_order_shuffle(b, bx);

	B[key_offset + 4*thread_in_block + (thread_in_block+0)%4] = b.x;
	B[key_offset + 4*thread_in_block + (thread_in_block+1)%4] = b.y;
	B[key_offset + 4*thread_in_block + (thread_in_block+2)%4] = b.z;
	B[key_offset + 4*thread_in_block + (thread_in_block+3)%4] = b.w;
	B[key_offset + 4*thread_in_block + (thread_in_block+0)%4 + 16] = bx.x;
	B[key_offset + 4*thread_in_block + (thread_in_block+1)%4 + 16] = bx.y;
	B[key_offset + 4*thread_in_block + (thread_in_block+2)%4 + 16] = bx.z;
	B[key_offset + 4*thread_in_block + (thread_in_block+3)%4 + 16] = bx.w;
}


/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__
void load_key_chacha(const uint32_t *B, uint4 &b, uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	// Read in permuted order. Key loads are not our bottleneck right now.
	b.x = B[key_offset + 4*0 + thread_in_block%4];
	b.y = B[key_offset + 4*1 + thread_in_block%4];
	b.z = B[key_offset + 4*2 + thread_in_block%4];
	b.w = B[key_offset + 4*3 + thread_in_block%4];
	bx.x = B[key_offset + 4*0 + thread_in_block%4 + 16];
	bx.y = B[key_offset + 4*1 + thread_in_block%4 + 16];
	bx.z = B[key_offset + 4*2 + thread_in_block%4 + 16];
	bx.w = B[key_offset + 4*3 + thread_in_block%4 + 16];
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__
void store_key_chacha(uint32_t *B, const uint4 &b, const uint4 &bx)
{
	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int key_offset = scrypt_block * 32;
	uint32_t thread_in_block = threadIdx.x % 4;

	B[key_offset + 4*0 + thread_in_block%4] = b.x;
	B[key_offset + 4*1 + thread_in_block%4] = b.y;
	B[key_offset + 4*2 + thread_in_block%4] = b.z;
	B[key_offset + 4*3 + thread_in_block%4] = b.w;
	B[key_offset + 4*0 + thread_in_block%4 + 16] = bx.x;
	B[key_offset + 4*1 + thread_in_block%4 + 16] = bx.y;
	B[key_offset + 4*2 + thread_in_block%4 + 16] = bx.z;
	B[key_offset + 4*3 + thread_in_block%4 + 16] = bx.w;
}


template <int ALGO> __device__  __forceinline__
void load_key(const uint32_t *B, uint4 &b, uint4 &bx)
{
		switch(ALGO) {
		case A_SCRYPT:      load_key_salsa(B, b, bx); break;
		case A_SCRYPT_JANE: load_key_chacha(B, b, bx); break;
		}
}

template <int ALGO> __device__  __forceinline__
void store_key(uint32_t *B, uint4 &b, uint4 &bx)
{
		switch(ALGO) {
		case A_SCRYPT:      store_key_salsa(B, b, bx); break;
		case A_SCRYPT_JANE: store_key_chacha(B, b, bx); break;
		}
}


/*
 * salsa_xor_core (Salsa20/8 cypher)
 * The original scrypt called:
 * xor_salsa8(&X[0], &X[16]); <-- the "b" loop
 * xor_salsa8(&X[16], &X[0]); <-- the "bx" loop
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 */

#define XOR_ROTATE_ADD(dst, s1, s2, amt) { uint32_t tmp = s1+s2; dst ^= ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__
void salsa_xor_core(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	uint4 x;

	b ^= bx;
	x = b;

	// Enter in "primary order" (t0 has  0,  4,  8, 12)
	//                          (t1 has  5,  9, 13,  1)
	//                          (t2 has 10, 14,  2,  6)
	//                          (t3 has 15,  3,  7, 11)

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Mixing phase of salsa
		XOR_ROTATE_ADD(x.y, x.x, x.w, 7);
		XOR_ROTATE_ADD(x.z, x.y, x.x, 9);
		XOR_ROTATE_ADD(x.w, x.z, x.y, 13);
		XOR_ROTATE_ADD(x.x, x.w, x.z, 18);

		/* Transpose rows and columns. */
		/* Unclear if this optimization is needed: These are ordered based
		 * upon the dependencies needed in the later xors. Compiler should be
		 * able to figure this out, but might as well give it a hand. */
		x.y = __shfl2((int)x.y, x3);
		x.w = __shfl2((int)x.w, x1);
		x.z = __shfl2((int)x.z, x2);

		/* The next XOR_ROTATE_ADDS could be written to be a copy-paste of the first,
		 * but the register targets are rewritten here to swap x[1] and x[3] so that
		 * they can be directly shuffled to and from our peer threads without
		 * reassignment. The reverse shuffle then puts them back in the right place.
		 */

		XOR_ROTATE_ADD(x.w, x.x, x.y, 7);
		XOR_ROTATE_ADD(x.z, x.w, x.x, 9);
		XOR_ROTATE_ADD(x.y, x.z, x.w, 13);
		XOR_ROTATE_ADD(x.x, x.y, x.z, 18);

		x.w = __shfl2((int)x.w, x3);
		x.y = __shfl2((int)x.y, x1);
		x.z = __shfl2((int)x.z, x2);
	}

	b += x;
	// The next two lines are the beginning of the BX-centric loop iteration
	bx ^= b;
	x = bx;

	// This is a copy of the same loop above, identical but stripped of comments.
	// Duplicated so that we can complete a bx-based loop with fewer register moves.
	#pragma unroll
	for (int j = 0; j < 4; j++) {
		XOR_ROTATE_ADD(x.y, x.x, x.w, 7);
		XOR_ROTATE_ADD(x.z, x.y, x.x, 9);
		XOR_ROTATE_ADD(x.w, x.z, x.y, 13);
		XOR_ROTATE_ADD(x.x, x.w, x.z, 18);

		x.y = __shfl2((int)x.y, x3);
		x.w = __shfl2((int)x.w, x1);
		x.z = __shfl2((int)x.z, x2);

		XOR_ROTATE_ADD(x.w, x.x, x.y, 7);
		XOR_ROTATE_ADD(x.z, x.w, x.x, 9);
		XOR_ROTATE_ADD(x.y, x.z, x.w, 13);
		XOR_ROTATE_ADD(x.x, x.y, x.z, 18);

		x.w = __shfl2((int)x.w, x3);
		x.y = __shfl2((int)x.y, x1);
		x.z = __shfl2((int)x.z, x2);
	}

	// At the end of these iterations, the data is in primary order again.
#undef XOR_ROTATE_ADD

	bx += x;
}


/*
 * chacha_xor_core (ChaCha20/8 cypher)
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 *
 * load_key and store_key must not use primary order when
 * using ChaCha20/8, but rather the basic transposed order
 * (referred to as "column mode" below)
 */

#define CHACHA_PRIMITIVE(pt, rt, ps, amt) { uint32_t tmp = rt ^ (pt += ps); rt = ((tmp<<amt)|(tmp>>(32-amt))); }

__device__  __forceinline__
void chacha_xor_core(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	uint4 x;

	b ^= bx;
	x = b;

	// Enter in "column" mode (t0 has 0, 4,  8, 12)
	//                        (t1 has 1, 5,  9, 13)
	//                        (t2 has 2, 6, 10, 14)
	//                        (t3 has 3, 7, 11, 15)

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Column Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		x.y = __shfl2((int)x.y, x1);
		x.z = __shfl2((int)x.z, x2);
		x.w = __shfl2((int)x.w, x3);

		// Diagonal Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		x.y = __shfl2((int)x.y, x3);
		x.z = __shfl2((int)x.z, x2);
		x.w = __shfl2((int)x.w, x1);
	}

	b += x;
	// The next two lines are the beginning of the BX-centric loop iteration
	bx ^= b;
	x = bx;

	#pragma unroll
	for (int j = 0; j < 4; j++) {

		// Column Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		x.y = __shfl2((int)x.y, x1);
		x.z = __shfl2((int)x.z, x2);
		x.w = __shfl2((int)x.w, x3);

		// Diagonal Mixing phase of chacha
		CHACHA_PRIMITIVE(x.x ,x.w, x.y, 16)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w, 12)
		CHACHA_PRIMITIVE(x.x ,x.w, x.y,  8)
		CHACHA_PRIMITIVE(x.z ,x.y, x.w,  7)

		x.y = __shfl2((int)x.y, x3);
		x.z = __shfl2((int)x.z, x2);
		x.w = __shfl2((int)x.w, x1);
	}

#undef CHACHA_PRIMITIVE

	bx += x;
}


template <int ALGO> __device__  __forceinline__
void block_mixer(uint4 &b, uint4 &bx, const int x1, const int x2, const int x3)
{
	switch(ALGO) {
	case A_SCRYPT:      salsa_xor_core(b, bx, x1, x2, x3);  break;
	case A_SCRYPT_JANE: chacha_xor_core(b, bx, x1, x2, x3); break;
	}
}


/*
 * The hasher_gen_kernel operates on a group of 1024-bit input keys
 * in B, stored as:
 * B = { k1B k1Bx k2B k2Bx ... }
 * and fills up the scratchpad with the iterative hashes derived from
 * those keys:
 * scratch { k1h1B k1h1Bx K1h2B K1h2Bx ... K2h1B K2h1Bx K2h2B K2h2Bx ... }
 * scratch is 1024 times larger than the input keys B.
 * It is extremely important to stream writes effectively into scratch;
 * less important to coalesce the reads from B.
 *
 * Key ordering note: Keys are input from B in "original" order:
 * K = {k1, k2, k3, k4, k5, ..., kx15, kx16, kx17, ..., kx31 }
 * After inputting into kernel_gen, each component k and kx of the
 * key is transmuted into a permuted internal order to make processing faster:
 * K = k, kx with:
 * k = 0, 4, 8, 12, 5, 9, 13, 1, 10, 14, 2, 6, 15, 3, 7, 11
 * and similarly for kx.
 */

template <int ALGO, MemoryAccess SCHEME> __global__
void kepler_scrypt_core_kernelA(const uint32_t *d_idata, int begin, int end)
{
	uint4 b, bx;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4)) % c_SCRATCH_WU_PER_WARP;

	int i=begin;

	if (i == 0) {
		load_key<ALGO>(d_idata, b, bx);
		write_keys_direct<SCHEME>(b, bx, start);
		++i;
	} else read_keys_direct<SCHEME,0>(b, bx, start+32*(i-1));

	while (i < end) {
		block_mixer<ALGO>(b, bx, x1, x2, x3);
		write_keys_direct<SCHEME>(b, bx, start+32*i);
		++i;
	}
}

template <int ALGO, MemoryAccess SCHEME> __global__
void kepler_scrypt_core_kernelA_LG(const uint32_t *d_idata, int begin, int end, unsigned int LOOKUP_GAP)
{
	uint4 b, bx;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4)) % c_SCRATCH_WU_PER_WARP;

	int i=begin;

	if (i == 0) {
		load_key<ALGO>(d_idata, b, bx);
		write_keys_direct<SCHEME>(b, bx, start);
		++i;
	} else {
		int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
		read_keys_direct<SCHEME,0>(b, bx, start+32*pos);
		while(loop--) block_mixer<ALGO>(b, bx, x1, x2, x3);
	}

	while (i < end) {
		block_mixer<ALGO>(b, bx, x1, x2, x3);
		if (i % LOOKUP_GAP == 0)
			write_keys_direct<SCHEME>(b, bx, start+32*(i/LOOKUP_GAP));
		++i;
	}
}


/*
 * hasher_hash_kernel runs the second phase of scrypt after the scratch
 * buffer is filled with the iterative hashes: It bounces through
 * the scratch buffer in pseudorandom order, mixing the key as it goes.
 */

template <int ALGO, MemoryAccess SCHEME, int TEX_DIM> __global__
void kepler_scrypt_core_kernelB(uint32_t *d_odata, int begin, int end)
{
	uint4 b, bx;

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH) + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4);
	if (TEX_DIM == 0) start %= c_SCRATCH_WU_PER_WARP;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	if (begin == 0) {
		read_keys_direct<SCHEME, TEX_DIM>(b, bx, start+32*c_N_1);
		block_mixer<ALGO>(b, bx, x1, x2, x3);
	} else load_key<ALGO>(d_odata, b, bx);

	for (int i = begin; i < end; i++) {
		int j = (__shfl2((int)bx.x, (threadIdx.x & 0x1c)) & (c_N_1));
		uint4 t, tx; read_keys_direct<SCHEME, TEX_DIM>(t, tx, start+32*j);
		b ^= t; bx ^= tx;
		block_mixer<ALGO>(b, bx, x1, x2, x3);
	}

	store_key<ALGO>(d_odata, b, bx);
}

template <int ALGO, MemoryAccess SCHEME, int TEX_DIM> __global__
void kepler_scrypt_core_kernelB_LG(uint32_t *d_odata, int begin, int end, unsigned int LOOKUP_GAP)
{
	uint4 b, bx;

	int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_WU;
	int start = (scrypt_block*c_SCRATCH) + (SCHEME==ANDERSEN?8:4)*(threadIdx.x%4);
	if (TEX_DIM == 0) start %= c_SCRATCH_WU_PER_WARP;

	int x1 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+1)&0x3);
	int x2 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+2)&0x3);
	int x3 = (threadIdx.x & 0x1c) + (((threadIdx.x & 0x03)+3)&0x3);

	if (begin == 0) {
		int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
		read_keys_direct<SCHEME,TEX_DIM>(b, bx, start+32*pos);
		while(loop--) block_mixer<ALGO>(b, bx, x1, x2, x3);
	} else load_key<ALGO>(d_odata, b, bx);

	if (SCHEME == SIMPLE)
	{
		// better divergent thread handling submitted by nVidia engineers, but
		// supposedly this does not run with the ANDERSEN memory access scheme
		int j = (__shfl2((int)bx.x, (threadIdx.x & 0x1c)) & (c_N_1));
		int pos = j/LOOKUP_GAP;
		int loop = -1;
		uint4 t, tx;

		int i = begin;
		while(i < end) {
			if (loop==-1) {
				j = (__shfl2((int)bx.x, (threadIdx.x & 0x1c)) & (c_N_1));
				pos = j/LOOKUP_GAP;
				loop = j-pos*LOOKUP_GAP;
				read_keys_direct<SCHEME,TEX_DIM>(t, tx, start+32*pos);
			}
			if (loop==0) {
				b ^= t; bx ^= tx;
				t=b;tx=bx;
			}
			block_mixer<ALGO>(t, tx, x1, x2, x3);
			if (loop==0) {
				b=t;bx=tx;
				i++;
			}
			loop--;
		}
	}
	else
	{
		// this is my original implementation, now used with the ANDERSEN
		// memory access scheme only.
		for (int i = begin; i < end; i++) {
			int j = (__shfl2((int)bx.x, (threadIdx.x & 0x1c)) & (c_N_1));
			int pos = j/LOOKUP_GAP, loop = j-pos*LOOKUP_GAP;
			uint4 t, tx; read_keys_direct<SCHEME,TEX_DIM>(t, tx, start+32*pos);
			while(loop--) block_mixer<ALGO>(t, tx, x1, x2, x3);
			b ^= t; bx ^= tx;
			block_mixer<ALGO>(b, bx, x1, x2, x3);
		}
	}

//for (int i = begin; i < end; i++) {
//	int j = (__shfl2((int)bx.x, (threadIdx.x & 0x1c)) & (c_N_1));
//	int pos = j/LOOKUP_GAP, loop = j-pos*LOOKUP_GAP;
//	uint4 t, tx; read_keys_direct<SCHEME,TEX_DIM>(t, tx, start+32*pos);
//	while(loop--) block_mixer<ALGO>(t, tx, x1, x2, x3);
//	b ^= t; bx ^= tx;
//	block_mixer<ALGO>(b, bx, x1, x2, x3);
//}

	store_key<ALGO>(d_odata, b, bx);
}

KeplerKernel::KeplerKernel() : KernelInterface()
{
}

bool KeplerKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef1D_4_V.normalized = 0;
	texRef1D_4_V.filterMode = hipFilterModePoint;
	texRef1D_4_V.addressMode[0] = hipAddressModeClamp;
	checkCudaErrors(hipBindTexture(NULL, &texRef1D_4_V, d_V, &channelDesc4, size));
	return true;
}

bool KeplerKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc<uint4>();
	texRef2D_4_V.normalized = 0;
	texRef2D_4_V.filterMode = hipFilterModePoint;
	texRef2D_4_V.addressMode[0] = hipAddressModeClamp;
	texRef2D_4_V.addressMode[1] = hipAddressModeClamp;
	// maintain texture width of TEXWIDTH (max. limit is 65000)
	while (width > TEXWIDTH) { width /= 2; height *= 2; pitch /= 2; }
	while (width < TEXWIDTH) { width *= 2; height = (height+1)/2; pitch *= 2; }
	checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_4_V, d_V, &channelDesc4, width, height, pitch));
	return true;
}

bool KeplerKernel::unbindtexture_1D()
{
	checkCudaErrors(hipUnbindTexture(texRef1D_4_V));
	return true;
}

bool KeplerKernel::unbindtexture_2D()
{
	checkCudaErrors(hipUnbindTexture(texRef2D_4_V));
	return true;
}

void KeplerKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool KeplerKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream,
	uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;

	// make some constants available to kernel, update only initially and when changing
	static uint32_t prev_N[MAX_GPUS] = { 0 };

	if (N != prev_N[thr_id]) {
		uint32_t h_N = N;
		uint32_t h_N_1 = N-1;
		uint32_t h_SCRATCH = SCRATCH;
		uint32_t h_SCRATCH_WU_PER_WARP = (SCRATCH * WU_PER_WARP);
		uint32_t h_SCRATCH_WU_PER_WARP_1 = (SCRATCH * WU_PER_WARP) - 1;

		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH), &h_SCRATCH, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH_WU_PER_WARP), &h_SCRATCH_WU_PER_WARP, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_SCRATCH_WU_PER_WARP_1), &h_SCRATCH_WU_PER_WARP_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);

		prev_N[thr_id] = N;
	}

	// First phase: Sequential writes to scratchpad.

	int batch = device_batchsize[thr_id];
	//int num_sleeps = 2* ((N + (batch-1)) / batch);
	//int sleeptime = 100;

	unsigned int pos = 0;
	do
	{
		if (LOOKUP_GAP == 1) {
			if (IS_SCRYPT())      kepler_scrypt_core_kernelA<A_SCRYPT,    ANDERSEN> <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
			if (IS_SCRYPT_JANE()) kepler_scrypt_core_kernelA<A_SCRYPT_JANE, SIMPLE> <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
		} else {
			if (IS_SCRYPT())      kepler_scrypt_core_kernelA_LG<A_SCRYPT,    ANDERSEN> <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (IS_SCRYPT_JANE()) kepler_scrypt_core_kernelA_LG<A_SCRYPT_JANE, SIMPLE> <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
		}
		pos += batch;
	} while (pos < N);

	// Second phase: Random read access from scratchpad.

	pos = 0;
	do
	{
		if (LOOKUP_GAP == 1) {

			if (texture_cache == 0) {
				if (IS_SCRYPT())      kepler_scrypt_core_kernelB<A_SCRYPT     ,ANDERSEN, 0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) kepler_scrypt_core_kernelB<A_SCRYPT_JANE,SIMPLE,   0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			} else if (texture_cache == 1) {
				if (IS_SCRYPT())      kepler_scrypt_core_kernelB<A_SCRYPT     ,ANDERSEN,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) kepler_scrypt_core_kernelB<A_SCRYPT_JANE,SIMPLE,  1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			} else if (texture_cache == 2) {
				if (IS_SCRYPT())      kepler_scrypt_core_kernelB<A_SCRYPT     ,ANDERSEN,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
				if (IS_SCRYPT_JANE()) kepler_scrypt_core_kernelB<A_SCRYPT_JANE,SIMPLE,  2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			}

		} else {

			if (texture_cache == 0) {
				if (IS_SCRYPT())       kepler_scrypt_core_kernelB_LG<A_SCRYPT     ,ANDERSEN,0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE())  kepler_scrypt_core_kernelB_LG<A_SCRYPT_JANE,SIMPLE,  0><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			} else if (texture_cache == 1) {
				if (IS_SCRYPT())       kepler_scrypt_core_kernelB_LG<A_SCRYPT     ,ANDERSEN,1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE())  kepler_scrypt_core_kernelB_LG<A_SCRYPT_JANE,SIMPLE,  1><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			} else if (texture_cache == 2) {
				if (IS_SCRYPT())       kepler_scrypt_core_kernelB_LG<A_SCRYPT     ,ANDERSEN,2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
				if (IS_SCRYPT_JANE())  kepler_scrypt_core_kernelB_LG<A_SCRYPT_JANE,SIMPLE,  2><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			}
		}

		pos += batch;
	} while (pos < N);

	return success;
}
