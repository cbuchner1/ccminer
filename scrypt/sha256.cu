#include "hip/hip_runtime.h"
//
//  =============== SHA256 part on nVidia GPU ======================
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//

#include <map>

#include <hip/hip_runtime.h>
#include "miner.h"

#include "salsa_kernel.h"

#include "sha256.h"

// define some error checking macros
#define DELIMITER '/'
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#undef checkCudaErrors
#define checkCudaErrors(x) { \
	hipGetLastError(); \
	x; \
	hipError_t err = hipGetLastError(); \
	if (err != hipSuccess && !abort_flag) \
		applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) (%s line %d)\n", (int) device_map[thr_id], err, hipGetErrorString(err), __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_tstate[2];
extern std::map<int, uint32_t *> context_ostate[2];
extern std::map<int, uint32_t *> context_hash[2];

static const uint32_t host_sha256_h[8] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
	0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

static const uint32_t host_sha256_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

/* Elementary functions used by SHA256 */
#define Ch(x, y, z)     ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define ROTR(x, n)      ((x >> n) | (x << (32 - n)))
#define S0(x)           (ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22))
#define S1(x)           (ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25))
#define s0(x)           (ROTR(x, 7) ^ ROTR(x, 18) ^ (x >> 3))
#define s1(x)           (ROTR(x, 17) ^ ROTR(x, 19) ^ (x >> 10))

/* SHA256 round function */
#define RND(a, b, c, d, e, f, g, h, k) \
	do { \
		t0 = h + S1(e) + Ch(e, f, g) + k; \
		t1 = S0(a) + Maj(a, b, c); \
		d += t0; \
		h  = t0 + t1; \
	} while (0)

/* Adjusted round function for rotating state */
#define RNDr(S, W, i) \
	RND(S[(64 - i) % 8], S[(65 - i) % 8], \
		S[(66 - i) % 8], S[(67 - i) % 8], \
		S[(68 - i) % 8], S[(69 - i) % 8], \
		S[(70 - i) % 8], S[(71 - i) % 8], \
		W[i] + sha256_k[i])

static const uint32_t host_keypad[12] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000280
};

static const uint32_t host_innerpad[11] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x000004a0
};

static const uint32_t host_outerpad[8] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0x00000300
};

static const uint32_t host_finalblk[16] = {
	0x00000001, 0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000620
};

//
// CUDA code
//

__constant__ uint32_t sha256_h[8];
__constant__ uint32_t sha256_k[64];
__constant__ uint32_t keypad[12];
__constant__ uint32_t innerpad[11];
__constant__ uint32_t outerpad[8];
__constant__ uint32_t finalblk[16];
__constant__ uint32_t pdata[20];
__constant__ uint32_t midstate[8];

__device__ void mycpy12(uint32_t *d, const uint32_t *s) {
#pragma unroll 3
	for (int k=0; k < 3; k++) d[k] = s[k];
}

__device__ void mycpy16(uint32_t *d, const uint32_t *s) {
#pragma unroll 4
	for (int k=0; k < 4; k++) d[k] = s[k];
}

__device__ void mycpy32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
	for (int k=0; k < 8; k++) d[k] = s[k];
}

__device__ void mycpy44(uint32_t *d, const uint32_t *s) {
#pragma unroll 11
	for (int k=0; k < 11; k++) d[k] = s[k];
}

__device__ void mycpy48(uint32_t *d, const uint32_t *s) {
#pragma unroll 12
	for (int k=0; k < 12; k++) d[k] = s[k];
}

__device__ void mycpy64(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
	for (int k=0; k < 16; k++) d[k] = s[k];
}

__device__ uint32_t cuda_swab32(uint32_t x)
{
	return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
		  | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__device__ void mycpy32_swab32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
	for (int k=0; k < 8; k++) d[k] = cuda_swab32(s[k]);
}

__device__ void mycpy64_swab32(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
	for (int k=0; k < 16; k++) d[k] = cuda_swab32(s[k]);
}

__device__ void cuda_sha256_init(uint32_t *state)
{
	mycpy32(state, sha256_h);
}

/*
 * SHA256 block compression function.  The 256-bit state is transformed via
 * the 512-bit input block to produce a new state. Modified for lower register use.
 */
__device__ void cuda_sha256_transform(uint32_t *state, const uint32_t *block)
{
	uint32_t W[64]; // only 4 of these are accessed during each partial Mix
	uint32_t S[8];
	uint32_t t0, t1;
	int i;

	/* 1. Initialize working variables. */
	mycpy32(S, state);

	/* 2. Prepare message schedule W and Mix. */
	mycpy16(W, block);
	RNDr(S, W,  0); RNDr(S, W,  1); RNDr(S, W,  2); RNDr(S, W,  3);

	mycpy16(W+4, block+4);
	RNDr(S, W,  4); RNDr(S, W,  5); RNDr(S, W,  6); RNDr(S, W,  7);

	mycpy16(W+8, block+8);
	RNDr(S, W,  8); RNDr(S, W,  9); RNDr(S, W, 10); RNDr(S, W, 11);

	mycpy16(W+12, block+12);
	RNDr(S, W, 12); RNDr(S, W, 13); RNDr(S, W, 14); RNDr(S, W, 15);

#pragma unroll 2
	for (i = 16; i < 20; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 16); RNDr(S, W, 17); RNDr(S, W, 18); RNDr(S, W, 19);

#pragma unroll 2
	for (i = 20; i < 24; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 20); RNDr(S, W, 21); RNDr(S, W, 22); RNDr(S, W, 23);

#pragma unroll 2
	for (i = 24; i < 28; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 24); RNDr(S, W, 25); RNDr(S, W, 26); RNDr(S, W, 27);

#pragma unroll 2
	for (i = 28; i < 32; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 28); RNDr(S, W, 29); RNDr(S, W, 30); RNDr(S, W, 31);

#pragma unroll 2
	for (i = 32; i < 36; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 32); RNDr(S, W, 33); RNDr(S, W, 34); RNDr(S, W, 35);

#pragma unroll 2
	for (i = 36; i < 40; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 36); RNDr(S, W, 37); RNDr(S, W, 38); RNDr(S, W, 39);

#pragma unroll 2
	for (i = 40; i < 44; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 40); RNDr(S, W, 41); RNDr(S, W, 42); RNDr(S, W, 43);

#pragma unroll 2
	for (i = 44; i < 48; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 44); RNDr(S, W, 45); RNDr(S, W, 46); RNDr(S, W, 47);

#pragma unroll 2
	for (i = 48; i < 52; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 48); RNDr(S, W, 49); RNDr(S, W, 50); RNDr(S, W, 51);

#pragma unroll 2
	for (i = 52; i < 56; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 52); RNDr(S, W, 53); RNDr(S, W, 54); RNDr(S, W, 55);

#pragma unroll 2
	for (i = 56; i < 60; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 56); RNDr(S, W, 57); RNDr(S, W, 58); RNDr(S, W, 59);

#pragma unroll 2
	for (i = 60; i < 64; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15]; }
	RNDr(S, W, 60); RNDr(S, W, 61); RNDr(S, W, 62); RNDr(S, W, 63);

	/* 3. Mix local working variables into global state */
#pragma unroll 8
	for (i = 0; i < 8; i++)
		state[i] += S[i];
}

//
// HMAC SHA256 functions, modified to work with pdata and nonce directly
//

__device__ void cuda_HMAC_SHA256_80_init(uint32_t *tstate, uint32_t *ostate, uint32_t nonce)
{
	uint32_t ihash[8];
	uint32_t pad[16];
	int i;

	/* tstate is assumed to contain the midstate of key */
	mycpy12(pad, pdata + 16);
	pad[3] = nonce;
	mycpy48(pad + 4, keypad);
	cuda_sha256_transform(tstate, pad);
	mycpy32(ihash, tstate);

	cuda_sha256_init(ostate);
#pragma unroll 8
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x5c5c5c5c;
#pragma unroll 8
	for (i=8; i < 16; i++)
		pad[i] = 0x5c5c5c5c;
	cuda_sha256_transform(ostate, pad);

	cuda_sha256_init(tstate);
#pragma unroll 8
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x36363636;
#pragma unroll 8
	for (i=8; i < 16; i++)
		pad[i] = 0x36363636;
	cuda_sha256_transform(tstate, pad);
}

__device__ void cuda_PBKDF2_SHA256_80_128(const uint32_t *tstate,
	const uint32_t *ostate, uint32_t *output, uint32_t nonce)
{
	uint32_t istate[8], ostate2[8];
	uint32_t ibuf[16], obuf[16];

	mycpy32(istate, tstate);
	cuda_sha256_transform(istate, pdata);

	mycpy12(ibuf, pdata + 16);
	ibuf[3] = nonce;
	ibuf[4] = 1;
	mycpy44(ibuf + 5, innerpad);

	mycpy32(obuf, istate);
	mycpy32(obuf + 8, outerpad);
	cuda_sha256_transform(obuf, ibuf);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf);
	mycpy32_swab32(output, ostate2);       // TODO: coalescing would be desired

	mycpy32(obuf, istate);
	ibuf[4] = 2;
	cuda_sha256_transform(obuf, ibuf);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf);
	mycpy32_swab32(output+8, ostate2);     // TODO: coalescing would be desired

	mycpy32(obuf, istate);
	ibuf[4] = 3;
	cuda_sha256_transform(obuf, ibuf);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf);
	mycpy32_swab32(output+16, ostate2);    // TODO: coalescing would be desired

	mycpy32(obuf, istate);
	ibuf[4] = 4;
	cuda_sha256_transform(obuf, ibuf);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf);
	mycpy32_swab32(output+24, ostate2);    // TODO: coalescing would be desired
}

__global__ void cuda_pre_sha256(uint32_t g_inp[32], uint32_t g_tstate_ext[8], uint32_t g_ostate_ext[8], uint32_t nonce)
{
	nonce        +=       (blockIdx.x * blockDim.x) + threadIdx.x;
	g_inp        += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	g_tstate_ext +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	g_ostate_ext +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);

	uint32_t tstate[8], ostate[8];
	mycpy32(tstate, midstate);

	cuda_HMAC_SHA256_80_init(tstate, ostate, nonce);

	mycpy32(g_tstate_ext, tstate);            // TODO: coalescing would be desired
	mycpy32(g_ostate_ext, ostate);            // TODO: coalescing would be desired

	cuda_PBKDF2_SHA256_80_128(tstate, ostate, g_inp, nonce);
}

__global__ void cuda_post_sha256(uint32_t g_output[8], uint32_t g_tstate_ext[8], uint32_t g_ostate_ext[8], uint32_t g_salt_ext[32])
{
	g_output     +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	g_tstate_ext +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	g_ostate_ext +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	g_salt_ext   += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);

	uint32_t tstate[16];
	mycpy32(tstate, g_tstate_ext);            // TODO: coalescing would be desired

	uint32_t halfsalt[16];
	mycpy64_swab32(halfsalt, g_salt_ext);     // TODO: coalescing would be desired
	cuda_sha256_transform(tstate, halfsalt);
	mycpy64_swab32(halfsalt, g_salt_ext+16);  // TODO: coalescing would be desired
	cuda_sha256_transform(tstate, halfsalt);
	cuda_sha256_transform(tstate, finalblk);

	uint32_t buf[16];
	mycpy32(buf, tstate);
	mycpy32(buf + 8, outerpad);

	uint32_t ostate[16];
	mycpy32(ostate, g_ostate_ext);

	cuda_sha256_transform(ostate, buf);
	mycpy32_swab32(g_output, ostate);        // TODO: coalescing would be desired
}

//
// callable host code to initialize constants and to call kernels
//

void prepare_sha256(int thr_id, uint32_t host_pdata[20], uint32_t host_midstate[8])
{
	static bool init[MAX_GPUS] = { 0 };

	if (!init[thr_id])
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sha256_h), host_sha256_h, sizeof(host_sha256_h), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sha256_k), host_sha256_k, sizeof(host_sha256_k), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(keypad), host_keypad, sizeof(host_keypad), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(innerpad), host_innerpad, sizeof(host_innerpad), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(outerpad), host_outerpad, sizeof(host_outerpad), 0, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(finalblk), host_finalblk, sizeof(host_finalblk), 0, hipMemcpyHostToDevice));
		init[thr_id] = true;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(midstate), host_midstate, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

void pre_sha256(int thr_id, int stream, uint32_t nonce, int throughput)
{
	dim3 block(128);
	dim3 grid((throughput+127)/128);

	cuda_pre_sha256<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_idata[stream][thr_id], context_tstate[stream][thr_id], context_ostate[stream][thr_id], nonce);
}

void post_sha256(int thr_id, int stream, int throughput)
{
	dim3 block(128);
	dim3 grid((throughput+127)/128);

	cuda_post_sha256<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_hash[stream][thr_id], context_tstate[stream][thr_id], context_ostate[stream][thr_id], context_odata[stream][thr_id]);
}
