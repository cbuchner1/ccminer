#include "hip/hip_runtime.h"
//
// Contains the autotuning logic and some utility functions.
// Note that all CUDA kernels have been moved to other .cu files
//

#include <stdio.h>
#include <map>
#include <algorithm>
#include <unistd.h> // usleep
#include <ctype.h> // tolower
#include "cuda_helper.h"

#include "salsa_kernel.h"

#include "nv_kernel2.h"
#include "titan_kernel.h"
#include "nv_kernel.h"
#include "kepler_kernel.h"
#include "fermi_kernel.h"
#include "test_kernel.h"

#include "miner.h"

#if defined(_WIN64) || defined(__x86_64__) || defined(__64BIT__)
#define MAXMEM 0x300000000ULL  // 12 GB (the largest Kepler)
#else
#define MAXMEM  0xFFFFFFFFULL  // nearly 4 GB (32 bit limitations)
#endif

// require CUDA 5.5 driver API
#define DMAJ 5
#define DMIN 5

// define some error checking macros
#define DELIMITER '/'
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#undef checkCudaErrors
#define checkCudaErrors(x) \
{ \
	hipGetLastError(); \
	x; \
	hipError_t err = hipGetLastError(); \
	if (err != hipSuccess && !abort_flag) \
		applog(LOG_ERR, "GPU #%d: Err %d: %s (%s:%d)", device_map[thr_id], err, hipGetErrorString(err), __FILENAME__, __LINE__); \
}

// some globals containing pointers to device memory (for chunked allocation)
// [MAX_GPUS] indexes up to MAX_GPUS threads (0...MAX_GPUS-1)
int       MAXWARPS[MAX_GPUS];
uint32_t* h_V[MAX_GPUS][TOTAL_WARP_LIMIT*64];          // NOTE: the *64 prevents buffer overflow for --keccak
uint32_t  h_V_extra[MAX_GPUS][TOTAL_WARP_LIMIT*64];    //       with really large kernel launch configurations

KernelInterface *Best_Kernel_Heuristics(hipDeviceProp_t *props)
{
	KernelInterface *kernel = NULL;
	uint64_t N = 1UL << (opt_nfactor+1);

	if (IS_SCRYPT() || (IS_SCRYPT_JANE() && N <= 8192))
	{
		// high register count kernels (scrypt, low N-factor scrypt-jane)
		if (props->major > 3 || (props->major == 3 && props->minor >= 5))
			kernel = new NV2Kernel(); // we don't want this for Keccak though
		else if (props->major == 3 && props->minor == 0)
			kernel = new NVKernel();
		else
			kernel = new FermiKernel();
	}
	else
	{
	   // high N-factor scrypt-jane = low registers count kernels
	   if (props->major > 3 || (props->major == 3 && props->minor >= 5))
			kernel = new TitanKernel();
		else if (props->major == 3 && props->minor == 0)
			kernel = new KeplerKernel();
		else
			kernel = new TestKernel();
	}
	return kernel;
}


bool validate_config(char *config, int &b, int &w, KernelInterface **kernel = NULL, hipDeviceProp_t *props = NULL)
{
	bool success = false;
	char kernelid = ' ';
	if (config != NULL)
	{
		if (config[0] == 'T' || config[0] == 'K' || config[0] == 'F' || config[0] == 'L' ||
			config[0] == 't' || config[0] == 'k' || config[0] == 'f' ||
			config[0] == 'Z' || config[0] == 'Y' || config[0] == 'X') {
			kernelid = config[0];
			config++;
		}

		if (config[0] >= '0' && config[0] <= '9')
			if (sscanf(config, "%dx%d", &b, &w) == 2)
				success = true;

		if (success && kernel != NULL)
		{
			switch (kernelid)
			{
				case 'T': case 'Z': *kernel = new NV2Kernel(); break;
				case 't':           *kernel = new TitanKernel(); break;
				case 'K': case 'Y': *kernel = new NVKernel(); break;
				case 'k':           *kernel = new KeplerKernel(); break;
				case 'F': case 'L': *kernel = new FermiKernel(); break;
				case 'f': case 'X': *kernel = new TestKernel(); break;
				case ' ': // choose based on device architecture
					*kernel = Best_Kernel_Heuristics(props);
				break;
			}
		}
	}
	return success;
}

std::map<int, int> context_blocks;
std::map<int, int> context_wpb;
std::map<int, bool> context_concurrent;
std::map<int, KernelInterface *> context_kernel;
std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_X[2];
std::map<int, uint32_t *> context_H[2];
std::map<int, hipEvent_t> context_serialize[2];

// for SHA256 hashing on GPU
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &wpb);

int cuda_throughput(int thr_id)
{
	int GRID_BLOCKS, WARPS_PER_BLOCK;
	if (context_blocks.find(thr_id) == context_blocks.end())
	{
#if 0
		hipCtx_t ctx;
		hipCtxCreate( &ctx, hipDeviceScheduleYield, device_map[thr_id] );
		hipCtxSetCurrent(ctx);
#else
		checkCudaErrors(hipSetDevice(device_map[thr_id]));
		checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleYield));
#endif

		KernelInterface *kernel;
		bool concurrent;
		GRID_BLOCKS = find_optimal_blockcount(thr_id, kernel, concurrent, WARPS_PER_BLOCK);

		if(GRID_BLOCKS == 0)
			return 0;

		unsigned int THREADS_PER_WU = kernel->threads_per_wu();
		unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;
		unsigned int state_size = WU_PER_LAUNCH * sizeof(uint32_t) * 8;

		// allocate device memory for scrypt_core inputs and outputs
		uint32_t *tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[1][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[0][thr_id] = tmp;
		checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[1][thr_id] = tmp;

		// allocate pinned host memory for scrypt hashes
		checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[0][thr_id] = tmp;
		checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[1][thr_id] = tmp;

		if (IS_SCRYPT())
		{
			if (parallel < 2)
			{
				// allocate pinned host memory for scrypt_core input/output
				checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
				checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;
			}
			else
			{
				// allocate tstate, ostate, scrypt hash device memory
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[0][thr_id] = tmp;
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[1][thr_id] = tmp;
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[0][thr_id] = tmp;
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[1][thr_id] = tmp;
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
				checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
			}
		}
		else /* if (IS_SCRYPT_JANE()) */
		{
			// allocate pinned host memory for scrypt_core input/output
			checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
			checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;

			checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
			checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
		}

		// create two CUDA streams
		hipStream_t tmp2;
		checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[0][thr_id] = tmp2;
		checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[1][thr_id] = tmp2;

		// events used to serialize the kernel launches (we don't want any overlapping of kernels)
		hipEvent_t tmp4;
		checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[0][thr_id] = tmp4;
		checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[1][thr_id] = tmp4;
		checkCudaErrors(hipEventRecord(context_serialize[1][thr_id]));

		context_kernel[thr_id] = kernel;
		context_concurrent[thr_id] = concurrent;
		context_blocks[thr_id] = GRID_BLOCKS;
		context_wpb[thr_id] = WARPS_PER_BLOCK;
	}

	GRID_BLOCKS = context_blocks[thr_id];
	WARPS_PER_BLOCK = context_wpb[thr_id];
	unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
	return WU_PER_LAUNCH;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{ 0x10, 8   }, // Tesla Generation (SM 1.0) G80 class
		{ 0x11, 8   }, // Tesla Generation (SM 1.1) G8x class
		{ 0x12, 8   }, // Tesla Generation (SM 1.2) G9x class
		{ 0x13, 8   }, // Tesla Generation (SM 1.3) GT200 class
		{ 0x20, 32  }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48  }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class - GK104 = 1536 cores / 8 SMs
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x50, 128 }, // Maxwell First Generation (SM 5.0) GTX750/750Ti
		{ 0x52, 128 }, // Maxwell Second Generation (SM 5.2) GTX980 = 2048 cores / 16 SMs - GTX970 1664 cores / 13 SMs
		{ 0x61, 128 }, // Pascal GeForce (SM 6.1)
		{ -1, -1 },
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	applog(LOG_WARNING, "MapSMtoCores for SM %d.%d is undefined. Default to use %d Cores/SM", major, minor, 128);
	return 128;
}

#ifdef WIN32
#include <windows.h>
static int console_width() {
	CONSOLE_SCREEN_BUFFER_INFO csbi;
	GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), &csbi);
	return csbi.srWindow.Right - csbi.srWindow.Left + 1;
}
#else
static inline int console_width() {
	return 999;
}
#endif

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &WARPS_PER_BLOCK)
{
	int cw = console_width();
	int optimal_blocks = 0;

	hipDeviceProp_t props;
	checkCudaErrors(hipGetDeviceProperties(&props, device_map[thr_id]));
	concurrent = (props.concurrentKernels > 0);

	WARPS_PER_BLOCK = -1;

	// if not specified, use interactive mode for devices that have the watchdog timer enabled
	if (device_interactive[thr_id] == -1)
		device_interactive[thr_id] = props.kernelExecTimeoutEnabled;

	// turn off texture cache if not otherwise specified
	if (device_texturecache[thr_id] == -1)
		device_texturecache[thr_id] = 0;

	// if not otherwise specified or required, turn single memory allocations off as they reduce
	// the amount of memory that we can allocate on Windows Vista, 7 and 8 (WDDM driver model issue)
	if (device_singlememory[thr_id] == -1) device_singlememory[thr_id] = 0;

	// figure out which kernel implementation to use
	if (!validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK, &kernel, &props)) {
		kernel = NULL;
		if (device_config[thr_id] != NULL) {
				 if (device_config[thr_id][0] == 'T' || device_config[thr_id][0] == 'Z')
				kernel = new NV2Kernel();
			else if (device_config[thr_id][0] == 't')
				kernel = new TitanKernel();
			else if (device_config[thr_id][0] == 'K' || device_config[thr_id][0] == 'Y')
				kernel = new NVKernel();
			else if (device_config[thr_id][0] == 'k')
				kernel = new KeplerKernel();
			else if (device_config[thr_id][0] == 'F' || device_config[thr_id][0] == 'L')
				kernel = new FermiKernel();
			else if (device_config[thr_id][0] == 'f' || device_config[thr_id][0] == 'X')
				kernel = new TestKernel();
		}
		if (kernel == NULL) kernel = Best_Kernel_Heuristics(&props);
	}

	if (kernel->get_major_version() > props.major || kernel->get_major_version() == props.major && kernel->get_minor_version() > props.minor)
	{
		applog(LOG_ERR, "GPU #%d: FATAL: the '%c' kernel requires %d.%d capability!", device_map[thr_id], kernel->get_identifier(), kernel->get_major_version(), kernel->get_minor_version());
		return 0;
	}

	// set whatever cache configuration and shared memory bank mode the kernel prefers
	checkCudaErrors(hipDeviceSetCacheConfig(kernel->cache_config()));
	checkCudaErrors(hipDeviceSetSharedMemConfig(kernel->shared_mem_config()));

	// some kernels (e.g. Titan) do not support the texture cache
	if (kernel->no_textures() && device_texturecache[thr_id]) {
		applog(LOG_WARNING, "GPU #%d: the '%c' kernel ignores the texture cache argument", device_map[thr_id], kernel->get_identifier());
		device_texturecache[thr_id] = 0;
	}

	// Texture caching only works with single memory allocation
	if (device_texturecache[thr_id]) device_singlememory[thr_id] = 1;

	if (kernel->single_memory() && !device_singlememory[thr_id]) {
		applog(LOG_WARNING, "GPU #%d: the '%c' kernel requires single memory allocation", device_map[thr_id], kernel->get_identifier());
		device_singlememory[thr_id] = 1;
	}

	if (device_lookup_gap[thr_id] == 0) device_lookup_gap[thr_id] = 1;
	if (!kernel->support_lookup_gap() && device_lookup_gap[thr_id] > 1)
	{
		applog(LOG_WARNING, "GPU #%d: the '%c' kernel does not support a lookup gap", device_map[thr_id], kernel->get_identifier());
		device_lookup_gap[thr_id] = 1;
	}

	if (opt_debug) {
		applog(LOG_INFO, "GPU #%d: interactive: %d, tex-cache: %d%s, single-alloc: %d", device_map[thr_id],
		   (device_interactive[thr_id]  != 0) ? 1 : 0,
		   (device_texturecache[thr_id] != 0) ? device_texturecache[thr_id] : 0, (device_texturecache[thr_id] != 0) ? "D" : "",
		   (device_singlememory[thr_id] != 0) ? 1 : 0 );
	}

	// number of threads collaborating on one work unit (hash)
	unsigned int THREADS_PER_WU = kernel->threads_per_wu();
	unsigned int LOOKUP_GAP = device_lookup_gap[thr_id];
	unsigned int BACKOFF = device_backoff[thr_id];
	unsigned int N = (1 << (opt_nfactor+1));
	double szPerWarp = (double)(SCRATCH * WU_PER_WARP * sizeof(uint32_t));
	//applog(LOG_INFO, "WU_PER_WARP=%u, THREADS_PER_WU=%u, LOOKUP_GAP=%u, BACKOFF=%u, SCRATCH=%u", WU_PER_WARP, THREADS_PER_WU, LOOKUP_GAP, BACKOFF, SCRATCH);
	applog(LOG_INFO, "GPU #%d: %d hashes / %.1f MB per warp.", device_map[thr_id], WU_PER_WARP, szPerWarp / (1024.0 * 1024.0));

	// compute highest MAXWARPS numbers for kernels allowing hipBindTexture to succeed
	int MW_1D_4 = 134217728 / (SCRATCH * WU_PER_WARP / 4); // for uint4_t textures
	int MW_1D_2 = 134217728 / (SCRATCH * WU_PER_WARP / 2); // for uint2_t textures
	int MW_1D = kernel->get_texel_width() == 2 ? MW_1D_2 : MW_1D_4;

	uint32_t *d_V = NULL;
	if (device_singlememory[thr_id])
	{
		// if no launch config was specified, we simply
		// allocate the single largest memory chunk on the device that we can get
		if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK)) {
			MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
		}
		else {
			// compute no. of warps to allocate the largest number producing a single memory block
			// PROBLEM: one some devices, ALL allocations will fail if the first one failed. This sucks.
			size_t MEM_LIMIT = (size_t)min((unsigned long long)MAXMEM, (unsigned long long)props.totalGlobalMem);
			int warpmax = (int)min((unsigned long long)TOTAL_WARP_LIMIT, (unsigned long long)(MEM_LIMIT / szPerWarp));

			// run a bisection algorithm for memory allocation (way more reliable than the previous approach)
			int best = 0;
			int warp = (warpmax+1)/2;
			int interval = (warpmax+1)/2;
			while (interval > 0)
			{
				hipGetLastError(); // clear the error state
				hipMalloc((void **)&d_V, (size_t)(szPerWarp * warp));
				if (hipGetLastError() == hipSuccess) {
					checkCudaErrors(hipFree(d_V)); d_V = NULL;
					if (warp > best) best = warp;
					if (warp == warpmax) break;
					interval = (interval+1)/2;
					warp += interval;
					if (warp > warpmax) warp = warpmax;
				}
				else
				{
					interval = interval/2;
					warp -= interval;
					if (warp < 1) warp = 1;
				}
			}
			// back off a bit from the largest possible allocation size
			MAXWARPS[thr_id] = ((100-BACKOFF)*best+50)/100;
		}

		// now allocate a buffer for determined MAXWARPS setting
		hipGetLastError(); // clear the error state
		hipMalloc((void **)&d_V, (size_t)SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
		if (hipGetLastError() == hipSuccess) {
			for (int i=0; i < MAXWARPS[thr_id]; ++i)
				h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

			if (device_texturecache[thr_id] == 1)
			{
				if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
				{
					if ( optimal_blocks * WARPS_PER_BLOCK > MW_1D ) {
						applog(LOG_ERR, "GPU #%d: '%s' exceeds limits for 1D cache. Using 2D cache instead.", device_map[thr_id], device_config[thr_id]);
						device_texturecache[thr_id] = 2;
					}
				}
				// bind linear memory to a 1D texture reference
				if (kernel->get_texel_width() == 2)
					kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * min(MAXWARPS[thr_id],MW_1D_2) * sizeof(uint32_t));
				else
					kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * min(MAXWARPS[thr_id],MW_1D_4) * sizeof(uint32_t));
			}
			else if (device_texturecache[thr_id] == 2)
			{
				// bind pitch linear memory to a 2D texture reference
				if (kernel->get_texel_width() == 2)
					kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
				else
					kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
			}
		}
		else
		{
			applog(LOG_ERR, "GPU #%d: FATAL: Launch config '%s' requires too much memory!", device_map[thr_id], device_config[thr_id]);
			return 0;
		}
	}
	else
	{
		if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
			MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
		else
			MAXWARPS[thr_id] = TOTAL_WARP_LIMIT;

		// chunked memory allocation up to device limits
		int warp;
		for (warp = 0; warp < MAXWARPS[thr_id]; ++warp) {
			// work around partition camping problems by adding a random start address offset to each allocation
			h_V_extra[thr_id][warp] = (props.major == 1) ? (16 * (rand()%(16384/16))) : 0;
			hipGetLastError(); // clear the error state
			hipMalloc((void **) &h_V[thr_id][warp], (SCRATCH * WU_PER_WARP + h_V_extra[thr_id][warp])*sizeof(uint32_t));
			if (hipGetLastError() == hipSuccess) h_V[thr_id][warp] += h_V_extra[thr_id][warp];
			else {
				h_V_extra[thr_id][warp] = 0;

				// back off by several warp allocations to have some breathing room
				int remove = (BACKOFF*warp+50)/100;
				for (int i=0; warp > 0 && i < remove; ++i) {
					warp--;
					checkCudaErrors(hipFree(h_V[thr_id][warp]-h_V_extra[thr_id][warp]));
					h_V[thr_id][warp] = NULL; h_V_extra[thr_id][warp] = 0;
				}

				break;
			}
		}
		MAXWARPS[thr_id] = warp;
	}
	kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);

	if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
	{
		if (optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
		{
			applog(LOG_ERR, "GPU #%d: FATAL: Given launch config '%s' requires too much memory.", device_map[thr_id], device_config[thr_id]);
			return 0;
		}

		if (WARPS_PER_BLOCK > kernel->max_warps_per_block())
		{
			applog(LOG_ERR, "GPU #%d: FATAL: Given launch config '%s' exceeds warp limit for '%c' kernel.", device_map[thr_id], device_config[thr_id], kernel->get_identifier());
			return 0;
		}
	}
	else
	{
		if (device_config[thr_id] != NULL && strcasecmp("auto", device_config[thr_id]))
			applog(LOG_WARNING, "GPU #%d: Given launch config '%s' does not validate.", device_map[thr_id], device_config[thr_id]);

		if (opt_autotune)
		{
			applog(LOG_INFO, "GPU #%d: Performing auto-tuning, please wait 2 minutes...", device_map[thr_id]);

			// allocate device memory
			uint32_t *d_idata = NULL, *d_odata = NULL;
			unsigned int mem_size = MAXWARPS[thr_id] * WU_PER_WARP * sizeof(uint32_t) * 32;
			checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
			checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

			// pre-initialize some device memory
			uint32_t *h_idata = (uint32_t*)malloc(mem_size);
			for (unsigned int i=0; i < mem_size/sizeof(uint32_t); ++i) h_idata[i] = i*2654435761UL; // knuth's method
			checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
			free(h_idata);

			double best_hash_sec = 0.0;
			int best_wpb = 0;

			// auto-tuning loop
			{
				// we want to have enough total warps for half the multiprocessors at least
				// compute highest MAXWARPS number that we can support based on texture cache mode
				int MINTW = props.multiProcessorCount / 2;
				int MAXTW = (device_texturecache[thr_id] == 1) ? min(MAXWARPS[thr_id],MW_1D) : MAXWARPS[thr_id];

				// we want to have blocks for half the multiprocessors at least
				int MINB = props.multiProcessorCount / 2;
				int MAXB = MAXTW;

				double tmin = 0.05;

				applog(LOG_INFO, "GPU #%d: maximum total warps (BxW): %d", (int) device_map[thr_id], MAXTW);

				for (int GRID_BLOCKS = MINB; !abort_flag && GRID_BLOCKS <= MAXB; ++GRID_BLOCKS)
				{
					double Hash[32+1] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
					for (WARPS_PER_BLOCK = 1; !abort_flag && WARPS_PER_BLOCK <= kernel->max_warps_per_block(); ++WARPS_PER_BLOCK)
					{
						double hash_sec = 0;
						if (GRID_BLOCKS * WARPS_PER_BLOCK >= MINTW &&
							GRID_BLOCKS * WARPS_PER_BLOCK <= MAXTW)
						{
							// setup execution parameters
							dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
							dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

							struct timeval tv_start, tv_end;
							double tdelta = 0;

							checkCudaErrors(hipDeviceSynchronize());
							gettimeofday(&tv_start, NULL);
							int repeat = 0;
							do  // average several measurements for better exactness
							{
								kernel->run_kernel(
									grid, threads, WARPS_PER_BLOCK, thr_id, NULL, d_idata, d_odata, N,
									LOOKUP_GAP, device_interactive[thr_id], true, device_texturecache[thr_id]
								);
								if(hipDeviceSynchronize() != hipSuccess)
									break;
								++repeat;
								gettimeofday(&tv_end, NULL);
								// for a better result averaging, measure for at least 50ms (10ms for Keccak)
							} while ((tdelta=(1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec))) < tmin);
							if (hipGetLastError() != hipSuccess) continue;

							tdelta /= repeat; // BUGFIX: this averaging over multiple measurements was missing

							// for scrypt: in interactive mode only find launch configs where kernel launch times are short enough
							// TODO: instead we could reduce the batchsize parameter to meet the launch time requirement.
							if (IS_SCRYPT() && device_interactive[thr_id]
								&& GRID_BLOCKS > 2*props.multiProcessorCount && tdelta > 1.0/30)
							{
								if (WARPS_PER_BLOCK == 1) goto skip; else goto skip2;
							}

							hash_sec = (double)WU_PER_LAUNCH / tdelta;
							Hash[WARPS_PER_BLOCK] = hash_sec;
							if (hash_sec > best_hash_sec) {
								optimal_blocks = GRID_BLOCKS;
								best_hash_sec = hash_sec;
								best_wpb = WARPS_PER_BLOCK;
							}
						}
					}
skip2:
					if (opt_debug) {

						if (GRID_BLOCKS == MINB) {
							char line[512] = "    ";
							for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
								char tmp[16]; sprintf(tmp, i < 10 ? "   x%-2d" : "  x%-2d ", i);
								strcat(line, tmp);
								if (cw == 80 && (i % 8 == 0 && i != kernel->max_warps_per_block()))
									strcat(line, "\n                          ");
							}
							applog(LOG_DEBUG, line);
						}

						char kMGT = ' '; bool flag;
						for (int j=0; j < 4; ++j) {
							flag=false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 1000, i++);
							if (flag)   for (int i=1; i<=kernel->max_warps_per_block(); Hash[i] /= 1000, i++);
							else break;
								 if (kMGT == ' ') kMGT = 'k';
							else if (kMGT == 'k') kMGT = 'M';
							else if (kMGT == 'M') kMGT = 'G';
							else if (kMGT == 'G') kMGT = 'T';
						}
						const char *format = "%5.4f%c";
						flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 1, i++); if (flag) format = "%5.3f%c";
						flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 10, i++); if (flag) format = "%5.2f%c";
						flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 100, i++); if (flag) format = "%5.1f%c";

						char line[512]; sprintf(line, "%3d:", GRID_BLOCKS);
						for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
							char tmp[16];
							if (Hash[i]>0)
								sprintf(tmp, format, Hash[i], (i<kernel->max_warps_per_block())?'|':' ');
							else
								sprintf(tmp, "     %c", (i<kernel->max_warps_per_block())?'|':' ');
							strcat(line, tmp);
							if (cw == 80 && (i % 8 == 0 && i != kernel->max_warps_per_block()))
								strcat(line, "\n                          ");
						}
						int n = strlen(line)-1; line[n++] = '|'; line[n++] = ' '; line[n++] = kMGT; line[n++] = '\0';
						strcat(line, "H/s");
						applog(LOG_DEBUG, line);
					}
				}
skip:           ;
			}

			checkCudaErrors(hipFree(d_odata));
			checkCudaErrors(hipFree(d_idata));

			WARPS_PER_BLOCK = best_wpb;
			applog(LOG_INFO, "GPU #%d: %7.2f hash/s with configuration %c%dx%d", device_map[thr_id], best_hash_sec, kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);
		}
		else
		{
			// Heuristics to find a good kernel launch configuration

			// base the initial block estimate on the number of multiprocessors
			int device_cores = props.multiProcessorCount * _ConvertSMVer2Cores(props.major, props.minor);

			// defaults, in case nothing else is chosen below
			optimal_blocks = 4 * device_cores / WU_PER_WARP;
			WARPS_PER_BLOCK = 2;

			// Based on compute capability, pick a known good block x warp configuration.
			if (props.major >= 3)
			{
				if (props.major == 3 && props.minor == 5) // GK110 (Tesla K20X, K20, GeForce GTX TITAN)
				{
					// TODO: what to do with Titan and Tesla K20(X)?
					// for now, do the same as for GTX 660Ti (2GB)
					optimal_blocks = (int)(optimal_blocks * 0.8809524);
					WARPS_PER_BLOCK = 2;
				}
				else // GK104, GK106, GK107 ...
				{
					if (MAXWARPS[thr_id] > (int)(optimal_blocks * 1.7261905) * 2)
					{
						// this results in 290x2 configuration on GTX 660Ti (3GB)
						// but it requires 3GB memory on the card!
						optimal_blocks = (int)(optimal_blocks * 1.7261905);
						WARPS_PER_BLOCK = 2;
					}
					else
					{
						// this results in 148x2 configuration on GTX 660Ti (2GB)
						optimal_blocks = (int)(optimal_blocks * 0.8809524);
						WARPS_PER_BLOCK = 2;
					}
				}
			}
			// 1st generation Fermi (compute 2.0) GF100, GF110
			else if (props.major == 2 && props.minor == 0)
			{
				// this results in a 60x4 configuration on GTX 570
				optimal_blocks = 4 * device_cores / WU_PER_WARP;
				WARPS_PER_BLOCK = 4;
			}
			// 2nd generation Fermi (compute 2.1) GF104,106,108,114,116
			else if (props.major == 2 && props.minor == 1)
			{
				// this results in a 56x2 configuration on GTX 460
				optimal_blocks = props.multiProcessorCount * 8;
				WARPS_PER_BLOCK = 2;
			}

			// in case we run out of memory with the automatically chosen configuration,
			// first back off with WARPS_PER_BLOCK, then reduce optimal_blocks.
			if (WARPS_PER_BLOCK==3 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
				WARPS_PER_BLOCK = 2;
			while (optimal_blocks > 0 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
				optimal_blocks--;
		}
	}

	applog(LOG_INFO, "GPU #%d: using launch configuration %c%dx%d", device_map[thr_id], kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);

	if (device_singlememory[thr_id])
	{
		if (MAXWARPS[thr_id] != optimal_blocks * WARPS_PER_BLOCK)
		{
			MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
			if (device_texturecache[thr_id] == 1)
				kernel->unbindtexture_1D();
			else if (device_texturecache[thr_id] == 2)
				kernel->unbindtexture_2D();
			checkCudaErrors(hipFree(d_V)); d_V = NULL;

			hipGetLastError(); // clear the error state
			hipMalloc((void **)&d_V, (size_t)SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
			if (hipGetLastError() == hipSuccess) {
				for (int i=0; i < MAXWARPS[thr_id]; ++i)
					h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

				if (device_texturecache[thr_id] == 1)
				{
					// bind linear memory to a 1D texture reference
					if (kernel->get_texel_width() == 2)
						kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
					else
						kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
				}
				else if (device_texturecache[thr_id] == 2)
				{
					// bind pitch linear memory to a 2D texture reference
					if (kernel->get_texel_width() == 2)
						kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
					else
						kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
				}

				// update pointers to scratch buffer in constant memory after reallocation
				kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);
			}
			else
			{
				applog(LOG_ERR, "GPU #%d: Unable to allocate enough memory for launch config '%s'.", device_map[thr_id], device_config[thr_id]);
			}
		}
	}
	else
	{
		// back off unnecessary memory allocations to have some breathing room
		while (MAXWARPS[thr_id] > 0 && MAXWARPS[thr_id] > optimal_blocks * WARPS_PER_BLOCK) {
			(MAXWARPS[thr_id])--;
			checkCudaErrors(hipFree(h_V[thr_id][MAXWARPS[thr_id]]-h_V_extra[thr_id][MAXWARPS[thr_id]]));
			h_V[thr_id][MAXWARPS[thr_id]] = NULL; h_V_extra[thr_id][MAXWARPS[thr_id]] = 0;
		}
	}

	return optimal_blocks;
}

void cuda_scrypt_HtoD(int thr_id, uint32_t *X, int stream)
{
	unsigned int GRID_BLOCKS = context_blocks[thr_id];
	unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
	unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
	unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

	// copy host memory to device
	hipMemcpyAsync(context_idata[stream][thr_id], X, mem_size, hipMemcpyHostToDevice, context_streams[stream][thr_id]);
}

void cuda_scrypt_serialize(int thr_id, int stream)
{
	// if the device can concurrently execute multiple kernels, then we must
	// wait for the serialization event recorded by the other stream
	if (context_concurrent[thr_id] || device_interactive[thr_id])
		hipStreamWaitEvent(context_streams[stream][thr_id], context_serialize[(stream+1)&1][thr_id], 0);
}

void cuda_scrypt_done(int thr_id, int stream)
{
	// record the serialization event in the current stream
	hipEventRecord(context_serialize[stream][thr_id], context_streams[stream][thr_id]);
}

void cuda_scrypt_flush(int thr_id, int stream)
{
	// flush the work queue (required for WDDM drivers)
	hipStreamSynchronize(context_streams[stream][thr_id]);
}

void cuda_scrypt_core(int thr_id, int stream, unsigned int N)
{
	unsigned int GRID_BLOCKS = context_blocks[thr_id];
	unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
	unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
	unsigned int LOOKUP_GAP = device_lookup_gap[thr_id];

	// setup execution parameters
	dim3 grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
	dim3 threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

	context_kernel[thr_id]->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id,
		context_streams[stream][thr_id], context_idata[stream][thr_id], context_odata[stream][thr_id],
		N, LOOKUP_GAP, device_interactive[thr_id], opt_benchmark, device_texturecache[thr_id]
	);
}

void cuda_scrypt_DtoH(int thr_id, uint32_t *X, int stream, bool postSHA)
{
	unsigned int GRID_BLOCKS = context_blocks[thr_id];
	unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
	unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
	unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * (postSHA ? 8 : 32);
	// copy result from device to host (asynchronously)
	checkCudaErrors(hipMemcpyAsync(X, postSHA ? context_hash[stream][thr_id] : context_odata[stream][thr_id], mem_size, hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}

bool cuda_scrypt_sync(int thr_id, int stream)
{
	hipError_t err;
	uint32_t wait_us = 0;

	if (device_interactive[thr_id] && !opt_benchmark)
	{
		// For devices that also do desktop rendering or compositing, we want to free up some time slots.
		// That requires making a pause in work submission when there is no active task on the GPU,
		// and Device Synchronize ensures that.

		// this call was replaced by the loop below to workaround the high CPU usage issue
		//err = hipDeviceSynchronize();

		while((err = hipStreamQuery(context_streams[0][thr_id])) == hipErrorNotReady ||
			  (err == hipSuccess && (err = hipStreamQuery(context_streams[1][thr_id])) == hipErrorNotReady)) {
			usleep(50); wait_us+=50;
		}

		usleep(50); wait_us+=50;
	} else {
		// this call was replaced by the loop below to workaround the high CPU usage issue
		//err = hipStreamSynchronize(context_streams[stream][thr_id]);

		while((err = hipStreamQuery(context_streams[stream][thr_id])) == hipErrorNotReady) {
			usleep(50); wait_us+=50;
		}
	}

	if (err != hipSuccess) {
		if (!abort_flag)
			applog(LOG_ERR, "GPU #%d: CUDA error `%s` while waiting the kernel.", device_map[thr_id], hipGetErrorString(err));
		return false;
	}

	//if (opt_debug) {
	//	applog(LOG_DEBUG, "GPU #%d: %s %u us", device_map[thr_id], __FUNCTION__, wait_us);
	//}

	return true;
}

uint32_t* cuda_transferbuffer(int thr_id, int stream)
{
	return context_X[stream][thr_id];
}

uint32_t* cuda_hashbuffer(int thr_id, int stream)
{
	return context_H[stream][thr_id];
}
