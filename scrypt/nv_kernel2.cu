#include "hip/hip_runtime.h"
//
// Experimental Kernel for Kepler (Compute 3.5) devices
// code submitted by nVidia performance engineer Alexey Panteleev
// with modifications by Christian Buchner
//
// for Compute 3.5
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=80
// for Compute 3.0
// NOTE: compile this .cu module for compute_30,sm_30 with --maxrregcount=63
//

#include <map>

#include <hip/hip_runtime.h>
#include <cuda_helper.h>
#include "miner.h"

#include "salsa_kernel.h"
#include "nv_kernel2.h"

#define THREADS_PER_WU 1  // single thread per hash

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define __ldg(x) (*(x))
#endif

#if !defined(__CUDA_ARCH__) ||  __CUDA_ARCH__ >= 300

// grab lane ID
static __device__ __inline__ unsigned int __laneId() { unsigned int laneId; asm( "mov.u32 %0, %%laneid;" : "=r"( laneId ) ); return laneId; }

// forward references
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1; // N - 1
__constant__ uint32_t c_spacing; // (N+LOOKUP_GAP-1)/LOOKUP_GAP


NV2Kernel::NV2Kernel() : KernelInterface()
{
}

void NV2Kernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool NV2Kernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
	bool success = true;
	bool scrypt = IS_SCRYPT();
	bool chacha = IS_SCRYPT_JANE();

	// make some constants available to kernel, update only initially and when changing
	static uint32_t prev_N[MAX_GPUS] = { 0 };

	if (N != prev_N[thr_id]) {
		uint32_t h_N = N;
		uint32_t h_N_1 = N-1;
		uint32_t h_spacing = (N+LOOKUP_GAP-1)/LOOKUP_GAP;

		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(c_spacing), &h_spacing, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream);

		prev_N[thr_id] = N;
	}

	// First phase: Sequential writes to scratchpad.
	const int batch = device_batchsize[thr_id];
	unsigned int pos = 0;

	do
	{
		if (LOOKUP_GAP == 1) {
			if (scrypt) nv2_scrypt_core_kernelA<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
			if (chacha) nv2_scrypt_core_kernelA<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N));
		} else {
			if (scrypt) nv2_scrypt_core_kernelA_LG<A_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (chacha) nv2_scrypt_core_kernelA_LG<A_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP);
		}
		pos += batch;
	} while (pos < N);

	// Second phase: Random read access from scratchpad.
	pos = 0;
	do
	{
		if (LOOKUP_GAP == 1) {
			if (scrypt) nv2_scrypt_core_kernelB<A_SCRYPT     > <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
			if (chacha) nv2_scrypt_core_kernelB<A_SCRYPT_JANE> <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N));
		} else {
			if (scrypt) nv2_scrypt_core_kernelB_LG<A_SCRYPT     > <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
			if (chacha) nv2_scrypt_core_kernelB_LG<A_SCRYPT_JANE> <<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP);
		}

		pos += batch;
	} while (pos < N);

	return success;
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
	left.x ^= right.x;
	left.y ^= right.y;
	left.z ^= right.z;
	left.w ^= right.w;
	return left;
}

__device__ __forceinline__ uint4 shfl4(const uint4 val, unsigned int lane, unsigned int width)
{
	return make_uint4(
		(unsigned int)__shfl((int)val.x, lane, width),
		(unsigned int)__shfl((int)val.y, lane, width),
		(unsigned int)__shfl((int)val.z, lane, width),
		(unsigned int)__shfl((int)val.w, lane, width)
	);
}

__device__ __forceinline__ void __transposed_write_BC(uint4 (&B)[4], uint4 (&C)[4], uint4 *D, int spacing)
{
	unsigned int laneId = __laneId();

	unsigned int lane8 = laneId%8;
	unsigned int tile  = laneId/8;

	uint4 T1[8], T2[8];

	/* Source matrix, A-H are threads, 0-7 are data items, thread A is marked with `*`:

	   *A0  B0  C0  D0  E0  F0  G0  H0
	   *A1  B1  C1  D1  E1  F1  G1  H1
	   *A2  B2  C2  D2  E2  F2  G2  H2
	   *A3  B3  C3  D3  E3  F3  G3  H3
	   *A4  B4  C4  D4  E4  F4  G4  H4
	   *A5  B5  C5  D5  E5  F5  G5  H5
	   *A6  B6  C6  D6  E6  F6  G6  H6
	   *A7  B7  C7  D7  E7  F7  G7  H7
	*/

	// rotate rows
	T1[0] = B[0];
	T1[1] = shfl4(B[1], lane8 + 7, 8);
	T1[2] = shfl4(B[2], lane8 + 6, 8);
	T1[3] = shfl4(B[3], lane8 + 5, 8);
	T1[4] = shfl4(C[0], lane8 + 4, 8);
	T1[5] = shfl4(C[1], lane8 + 3, 8);
	T1[6] = shfl4(C[2], lane8 + 2, 8);
	T1[7] = shfl4(C[3], lane8 + 1, 8);

	/* Matrix after row rotates:

	   *A0  B0  C0  D0  E0  F0  G0  H0
		H1 *A1  B1  C1  D1  E1  F1  G1
		G2  H2 *A2  B2  C2  D2  E2  F2
		F3  G3  H3 *A3  B3  C3  D3  E3
		E4  F4  G4  H4 *A4  B4  C4  D4
		D5  E5  F5  G5  H5 *A5  B5  C5
		C6  D6  E6  F6  G6  H6 *A6  B6
		B7  C7  D7  E7  F7  G7  H7 *A7
	*/

	// rotate columns up using a barrel shifter simulation
	// column X is rotated up by (X+1) items
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T1[n] = ((lane8+1) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 4) ? T1[(n+4) % 8] : T1[n];

	/* Matrix after column rotates:

		H1  H2  H3  H4  H5  H6  H7  H0
		G2  G3  G4  G5  G6  G7  G0  G1
		F3  F4  F5  F6  F7  F0  F1  F2
		E4  E5  E6  E7  E0  E1  E2  E3
		D5  D6  D7  D0  D1  D2  D3  D4
		C6  C7  C0  C1  C2  C3  C4  C5
		B7  B0  B1  B2  B3  B4  B5  B6
	   *A0 *A1 *A2 *A3 *A4 *A5 *A6 *A7
	*/

	// rotate rows again using address math and write to D, in reverse row order
	D[spacing*2*(32*tile   )+ lane8     ] = T2[7];
	D[spacing*2*(32*tile+4 )+(lane8+7)%8] = T2[6];
	D[spacing*2*(32*tile+8 )+(lane8+6)%8] = T2[5];
	D[spacing*2*(32*tile+12)+(lane8+5)%8] = T2[4];
	D[spacing*2*(32*tile+16)+(lane8+4)%8] = T2[3];
	D[spacing*2*(32*tile+20)+(lane8+3)%8] = T2[2];
	D[spacing*2*(32*tile+24)+(lane8+2)%8] = T2[1];
	D[spacing*2*(32*tile+28)+(lane8+1)%8] = T2[0];
}

__device__ __forceinline__ void __transposed_read_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	unsigned int laneId = __laneId();

	unsigned int lane8 = laneId%8;
	unsigned int tile  = laneId/8;

	// Perform the same transposition as in __transposed_write_BC, but in reverse order.
	// See the illustrations in comments for __transposed_write_BC.

	// read and rotate rows, in reverse row order
	uint4 T1[8], T2[8];
	T1[7] = __ldg(&S[(spacing*2*(32*tile   ) +  lane8      + 8*__shfl(row, 0, 8))]);
	T1[6] = __ldg(&S[(spacing*2*(32*tile+4 ) + (lane8+7)%8 + 8*__shfl(row, 1, 8))]);
	T1[5] = __ldg(&S[(spacing*2*(32*tile+8 ) + (lane8+6)%8 + 8*__shfl(row, 2, 8))]);
	T1[4] = __ldg(&S[(spacing*2*(32*tile+12) + (lane8+5)%8 + 8*__shfl(row, 3, 8))]);
	T1[3] = __ldg(&S[(spacing*2*(32*tile+16) + (lane8+4)%8 + 8*__shfl(row, 4, 8))]);
	T1[2] = __ldg(&S[(spacing*2*(32*tile+20) + (lane8+3)%8 + 8*__shfl(row, 5, 8))]);
	T1[1] = __ldg(&S[(spacing*2*(32*tile+24) + (lane8+2)%8 + 8*__shfl(row, 6, 8))]);
	T1[0] = __ldg(&S[(spacing*2*(32*tile+28) + (lane8+1)%8 + 8*__shfl(row, 7, 8))]);

	// rotate columns down using a barrel shifter simulation
	// column X is rotated down by (X+1) items, or up by (8-(X+1)) = (7-X) items
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T1[n] = ((7-lane8) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
	for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 4) ? T1[(n+4) % 8] : T1[n];

	// rotate rows
	B[0] = T2[0];
	B[1] = shfl4(T2[1], lane8 + 1, 8);
	B[2] = shfl4(T2[2], lane8 + 2, 8);
	B[3] = shfl4(T2[3], lane8 + 3, 8);
	C[0] = shfl4(T2[4], lane8 + 4, 8);
	C[1] = shfl4(T2[5], lane8 + 5, 8);
	C[2] = shfl4(T2[6], lane8 + 6, 8);
	C[3] = shfl4(T2[7], lane8 + 7, 8);

}

__device__ __forceinline__ void __transposed_xor_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
	uint4 BT[4], CT[4];
	__transposed_read_BC(S, BT, CT, spacing, row);

#pragma unroll 4
	for(int n = 0; n < 4; n++)
	{
		B[n] ^= BT[n];
		C[n] ^= CT[n];
	}
}

#if __CUDA_ARCH__ < 350
	// Kepler (Compute 3.0)
	#define ROTL(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
	// Kepler (Compute 3.5)
	#define ROTL(a, b) __funnelshift_l( a, a, b );
#endif



#if 0

#define QUARTER(a,b,c,d) \
	a += b; d ^= a; d = ROTL(d,16); \
	c += d; b ^= c; b = ROTL(b,12); \
	a += b; d ^= a; d = ROTL(d,8); \
	c += d; b ^= c; b = ROTL(b,7);

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	/* Operate on columns. */
	QUARTER( x[0], x[4], x[ 8], x[12] )
	QUARTER( x[1], x[5], x[ 9], x[13] )
	QUARTER( x[2], x[6], x[10], x[14] )
	QUARTER( x[3], x[7], x[11], x[15] )

	/* Operate on diagonals */
	QUARTER( x[0], x[5], x[10], x[15] )
	QUARTER( x[1], x[6], x[11], x[12] )
	QUARTER( x[2], x[7], x[ 8], x[13] )
	QUARTER( x[3], x[4], x[ 9], x[14] )

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#else

#define ADD4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 += s1; d2 += s2; d3 += s3; d4 += s4;

#define XOR4(d1,d2,d3,d4,s1,s2,s3,s4) \
	d1 ^= s1; d2 ^= s2; d3 ^= s3; d4 ^= s4;

#define ROTL4(d1,d2,d3,d4,amt) \
	d1 = ROTL(d1, amt); d2 = ROTL(d2, amt); d3 = ROTL(d3, amt); d4 = ROTL(d4, amt);

#define QROUND(a1,a2,a3,a4, b1,b2,b3,b4, c1,c2,c3,c4, amt) \
	ADD4 (a1,a2,a3,a4, c1,c2,c3,c4) \
	XOR4 (b1,b2,b3,b4, a1,a2,a3,a4) \
	ROTL4(b1,b2,b3,b4, amt)

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	/* Operate on columns. */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
	QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

	/* Operate on diagonals */
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
	QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
	QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#endif


#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 7); a1^=ROTL(a10, 7); a2^=ROTL(a20, 7); a3^=ROTL(a30, 7);\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 9); a1^=ROTL(a10, 9); a2^=ROTL(a20, 9); a3^=ROTL(a30, 9);\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 13); a1^=ROTL(a10, 13); a2^=ROTL(a20, 13); a3^=ROTL(a30, 13);\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 18); a1^=ROTL(a10, 18); a2^=ROTL(a20, 18); a3^=ROTL(a30, 18);\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
	uint32_t x[16];
	x[0]=(B[0].x ^= C[0].x);
	x[1]=(B[0].y ^= C[0].y);
	x[2]=(B[0].z ^= C[0].z);
	x[3]=(B[0].w ^= C[0].w);
	x[4]=(B[1].x ^= C[1].x);
	x[5]=(B[1].y ^= C[1].y);
	x[6]=(B[1].z ^= C[1].z);
	x[7]=(B[1].w ^= C[1].w);
	x[8]=(B[2].x ^= C[2].x);
	x[9]=(B[2].y ^= C[2].y);
	x[10]=(B[2].z ^= C[2].z);
	x[11]=(B[2].w ^= C[2].w);
	x[12]=(B[3].x ^= C[3].x);
	x[13]=(B[3].y ^= C[3].y);
	x[14]=(B[3].z ^= C[3].z);
	x[15]=(B[3].w ^= C[3].w);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	/* Operate on columns. */
	ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
	ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
	ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
	ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

	/* Operate on rows. */
	ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
	ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
	ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
	ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

	B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
	B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}


template <int ALGO> static __device__ void block_mixer(uint4 *B, uint4 *C)
{
  switch (ALGO)
  {
	case A_SCRYPT:      xor_salsa8(B, C); break;
	case A_SCRYPT_JANE: xor_chacha8(B, C); break;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Experimental Scrypt core kernel for Titan devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_N);
		++i;
	} else
		__transposed_read_BC((uint4*)(V + (i-1)*32), B, C, c_N, 0);

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		__transposed_write_BC(B, C, (uint4*)(V + i*32), c_N);
		++i;
	}
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_idata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];
	int i = begin;

	if(i == 0) {
		__transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
		__transposed_write_BC(B, C, (uint4*)V, c_spacing);
		++i;
	} else {
		int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
		__transposed_read_BC((uint4*)(V + pos*32), B, C, c_spacing, 0);
		while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	}

	while(i < end) {
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
		if (i % LOOKUP_GAP == 0)
		  __transposed_write_BC(B, C, (uint4*)(V + (i/LOOKUP_GAP)*32), c_spacing);
		++i;
	}
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
		__transposed_read_BC((uint4*)V, B, C, c_N, c_N_1);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	} else
		__transposed_read_BC((uint4*)g_odata, B, C, 1, 0);

	for (int i = begin; i < end; i++)  {
		int slot = C[0].x & c_N_1;
		__transposed_xor_BC((uint4*)(V), B, C, c_N, slot);
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
	g_odata += 32 * offset;
	uint32_t * V = c_V[offset / warpSize];
	uint4 B[4], C[4];

	if(begin == 0) {
	  int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
	  __transposed_read_BC((uint4*)V, B, C, c_spacing, pos);
	  while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
	} else {
		__transposed_read_BC((uint4*)g_odata, B, C, 1, 0);
	}

	for (int i = begin; i < end; i++)  {
		int slot = C[0].x & c_N_1;
		int pos = slot/LOOKUP_GAP, loop = slot-pos*LOOKUP_GAP;
		uint4 b[4], c[4]; __transposed_read_BC((uint4*)(V), b, c, c_spacing, pos);
		while(loop--) { block_mixer<ALGO>(b, c); block_mixer<ALGO>(c, b); }
#pragma unroll 4
		for(int n = 0; n < 4; n++) { B[n] ^= b[n]; C[n] ^= c[n]; }
		block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
	}

	__transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}

#endif /* prevent SM 2 */

