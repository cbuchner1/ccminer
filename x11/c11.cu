#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

void tribus_echo512_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target);

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

// Flax/Chaincoin C11 CPU Hash
extern "C" void c11hash(void *output, const void *input)
{
	unsigned char hash[128] = { 0 };

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(output, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("X11 %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_c11(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x5;

	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		if (use_compat_kernels[thr_id])
			x11_echo512_cpu_init(thr_id, throughput);
		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
			return 0;
		}
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 64 * throughput), 0);
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	if (use_compat_kernels[thr_id])
		cuda_check_cpu_setTarget(ptarget);
	else
		hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("groestl:");
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("jh512  :");
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("keccak :");
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("skein  :");
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		TRACE("luffa+c:");
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("shavite:");
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("simd   :");

		if (use_compat_kernels[thr_id]) {
			x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[1] = UINT32_MAX;
		} else {
			tribus_echo512_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id], AS_U64(&ptarget[6]));
			hipMemcpy(&work->nonces[0], d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			if (!use_compat_kernels[thr_id]) work->nonces[0] += startNounce;
			be32enc(&endiandata[19], work->nonces[0]);
			c11hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != UINT32_MAX) {
					work->nonces[1] += startNounce;
					be32enc(&endiandata[19], work->nonces[1]);
					c11hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_c11(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_resNonce[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
