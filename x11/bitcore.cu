#include "hip/hip_runtime.h"
/**
 * Timetravel-10 (bitcore) CUDA implementation
 *  by tpruvot@github - May 2017
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>

#define HASH_FUNC_BASE_TIMESTAMP 1492973331U
#define HASH_FUNC_COUNT 10
#define HASH_FUNC_COUNT_PERMUTATIONS 40320U

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#if HASH_FUNC_COUNT > 10
#include "sph/sph_echo.h"
#endif
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS];

enum Algo {
	BLAKE = 0,
	BMW,
	GROESTL,
	SKEIN,
	JH,
	KECCAK,
	LUFFA,
	CUBEHASH,
	SHAVITE,
	SIMD,
#if HASH_FUNC_COUNT > 10
	ECHO,
#endif
	MAX_ALGOS_COUNT
};

inline void swap8(uint8_t *a, uint8_t *b)
{
	uint8_t t = *a;
	*a = *b;
	*b = t;
}

inline void initPerm(uint8_t n[], int count)
{
	for (int i = 0; i < count; i++)
		n[i] = i;
}

static int nextPerm(uint8_t n[], int count)
{
	int tail, i, j;

	if (count <= 1)
		return 0;

	for (i = count - 1; i>0 && n[i - 1] >= n[i]; i--);
	tail = i;

	if (tail > 0) {
		for (j = count - 1; j>tail && n[j] <= n[tail - 1]; j--);
		swap8(&n[tail - 1], &n[j]);
	}

	for (i = tail, j = count - 1; i<j; i++, j--)
		swap8(&n[i], &n[j]);

	return (tail != 0);
}

static void getAlgoString(char *str, int seq)
{
	uint8_t algoList[HASH_FUNC_COUNT];
	char *sptr;

	initPerm(algoList, HASH_FUNC_COUNT);

	for (int k = 0; k < seq; k++) {
		nextPerm(algoList, HASH_FUNC_COUNT);
	}

	sptr = str;
	for (int j = 0; j < HASH_FUNC_COUNT; j++) {
		if (algoList[j] >= 10)
			sprintf(sptr, "%c", 'A' + (algoList[j] - 10));
		else
			sprintf(sptr, "%u", (uint32_t) algoList[j]);
		sptr++;
	}
	*sptr = '\0';
}

static __thread uint32_t s_ntime = 0;
static uint32_t s_sequence = UINT32_MAX;
static uint8_t s_firstalgo = 0xFF;
static char hashOrder[HASH_FUNC_COUNT + 1] = { 0 };

#define INITIAL_DATE HASH_FUNC_BASE_TIMESTAMP
static inline uint32_t getCurrentAlgoSeq(uint32_t ntime)
{
	// unlike x11evo, the permutation changes often (with ntime)
	return (uint32_t) (ntime - INITIAL_DATE) % HASH_FUNC_COUNT_PERMUTATIONS;
}

// To finish...
static void get_travel_order(uint32_t ntime, char *permstr)
{
	uint32_t seq = getCurrentAlgoSeq(ntime);
	if (s_sequence != seq) {
		getAlgoString(permstr, seq);
		s_sequence = seq;
	}
}

// CPU Hash
extern "C" void bitcore_hash(void *output, const void *input)
{
	uint32_t _ALIGN(64) hash[64/4] = { 0 };

	sph_blake512_context     ctx_blake;
	sph_bmw512_context       ctx_bmw;
	sph_groestl512_context   ctx_groestl;
	sph_skein512_context     ctx_skein;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_luffa512_context     ctx_luffa1;
	sph_cubehash512_context  ctx_cubehash1;
	sph_shavite512_context   ctx_shavite1;
	sph_simd512_context      ctx_simd1;
#if HASH_FUNC_COUNT > 10
	sph_echo512_context      ctx_echo1;
#endif

	if (s_sequence == UINT32_MAX) {
		uint32_t *data = (uint32_t*) input;
		const uint32_t ntime = (opt_benchmark || !data[17]) ? (uint32_t) time(NULL) : data[17];
		get_travel_order(ntime, hashOrder);
	}

	void *in = (void*) input;
	int size = 80;

	const int hashes = (int) strlen(hashOrder);

	for (int i = 0; i < hashes; i++)
	{
		const char elem = hashOrder[i];
		uint8_t algo = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

		if (i > 0) {
			in = (void*) hash;
			size = 64;
		}

		switch (algo) {
		case BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, in, size);
			sph_blake512_close(&ctx_blake, hash);
			break;
		case BMW:
			sph_bmw512_init(&ctx_bmw);
			sph_bmw512(&ctx_bmw, in, size);
			sph_bmw512_close(&ctx_bmw, hash);
			break;
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, in, size);
			sph_groestl512_close(&ctx_groestl, hash);
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, in, size);
			sph_skein512_close(&ctx_skein, hash);
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, in, size);
			sph_jh512_close(&ctx_jh, hash);
			break;
		case KECCAK:
			sph_keccak512_init(&ctx_keccak);
			sph_keccak512(&ctx_keccak, in, size);
			sph_keccak512_close(&ctx_keccak, hash);
			break;
		case LUFFA:
			sph_luffa512_init(&ctx_luffa1);
			sph_luffa512(&ctx_luffa1, in, size);
			sph_luffa512_close(&ctx_luffa1, hash);
			break;
		case CUBEHASH:
			sph_cubehash512_init(&ctx_cubehash1);
			sph_cubehash512(&ctx_cubehash1, in, size);
			sph_cubehash512_close(&ctx_cubehash1, hash);
			break;
		case SHAVITE:
			sph_shavite512_init(&ctx_shavite1);
			sph_shavite512(&ctx_shavite1, in, size);
			sph_shavite512_close(&ctx_shavite1, hash);
			break;
		case SIMD:
			sph_simd512_init(&ctx_simd1);
			sph_simd512(&ctx_simd1, in, size);
			sph_simd512_close(&ctx_simd1, hash);
			break;
#if HASH_FUNC_COUNT > 10
		case ECHO:
			sph_echo512_init(&ctx_echo1);
			sph_echo512(&ctx_echo1, in, size);
			sph_echo512_close(&ctx_echo1, hash);
			break;
#endif
		}
	}

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "tt-"
#include "cuda_debug.cuh"

void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_bitcore(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) pdata[17] = swab32(0x59090909);

	if (opt_debug || s_ntime != pdata[17] || s_sequence == UINT32_MAX) {
		uint32_t ntime = swab32(work->data[17]);
		get_travel_order(ntime, hashOrder);
		s_ntime = pdata[17];
		if (opt_debug && !thr_id) {
			applog(LOG_DEBUG, "timetravel10 hash order %s (%08x)", hashOrder, ntime);
		}
	}

	if (opt_benchmark)
		ptarget[7] = 0x5;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
			return 0;
		}
#if HASH_FUNC_COUNT > 10
		x11_echo512_cpu_init(thr_id, throughput);
#endif
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMemset(d_hash[thr_id], 0, (size_t) 64 * throughput), -1);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	cuda_check_cpu_setTarget(ptarget);

	const int hashes = (int) strlen(hashOrder);
	const char first = hashOrder[0];
	const uint8_t algo80 = first >= 'A' ? first - 'A' + 10 : first - '0';
	if (algo80 != s_firstalgo) {
		s_firstalgo = algo80;
	}

	// first algo seems locked to blake in bitcore, fine!
	quark_blake512_cpu_setBlock_80(thr_id, endiandata);

	do {
		// Hash with CUDA

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		TRACE("blake80:");

		for (int i = 1; i < hashes; i++)
		{
			const char elem = hashOrder[i];
			const uint8_t algo64 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

			switch (algo64) {
			case BLAKE:
				quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("blake  :");
				break;
			case BMW:
				quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("bmw    :");
				break;
			case GROESTL:
				quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("groestl:");
				break;
			case SKEIN:
				quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("skein  :");
				break;
			case JH:
				quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("jh512  :");
				break;
			case KECCAK:
				quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("keccak :");
				break;
			case LUFFA:
				x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("luffa  :");
				break;
			case CUBEHASH:
				x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("cube   :");
				break;
			case SHAVITE:
				x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("shavite:");
				break;
			case SIMD:
				x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("simd   :");
				break;
#if HASH_FUNC_COUNT > 10
			case ECHO:
				x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], i);
				TRACE("echo   :");
				break;
#endif
			}
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			be32enc(&endiandata[19], work->nonces[0]);
			bitcore_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				pdata[19] = work->nonces[0];
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					bitcore_hash(vhash, endiandata);
					if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(pdata[19], work->nonces[1]) + 1;
				}
				return work->valid_nonces;
			} else if (vhash[7] > Htarg) {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_bitcore(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
