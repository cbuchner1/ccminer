#include "hip/hip_runtime.h"
#include "cuda_helper.h"

typedef unsigned char BitSequence;

#define CUBEHASH_ROUNDS 16 /* this is r for CubeHashr/b */
#define CUBEHASH_BLOCKBYTES 32 /* this is b for CubeHashr/b */

#if __CUDA_ARCH__ < 350
#define LROT(x,bits) ((x << bits) | (x >> (32 - bits)))
#else
#define LROT(x, bits) __funnelshift_l(x, x, bits)
#endif

#define ROTATEUPWARDS7(a)  LROT(a,7)
#define ROTATEUPWARDS11(a) LROT(a,11)

#define SWAP(a,b) { uint32_t u = a; a = b; b = u; }

__device__ __constant__
static const uint32_t c_IV_512[32] = {
	0x2AEA2A61, 0x50F494D4, 0x2D538B8B,
	0x4167D83E, 0x3FEE2313, 0xC701CF8C,
	0xCC39968E, 0x50AC5695, 0x4D42C787,
	0xA647A8B3, 0x97CF0BEF, 0x825B4537,
	0xEEF864D2, 0xF22090C4, 0xD0E5CD33,
	0xA23911AE, 0xFCD398D9, 0x148FE485,
	0x1B017BEF, 0xB6444532, 0x6A536159,
	0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
	0xD65C8A2B, 0xA5A70E75, 0xB1C62456,
	0xBC796576, 0x1921C8F7, 0xE7989AF1,
	0x7795D246, 0xD43E3B44
};

__device__ __forceinline__
static void rrounds(uint32_t x[2][2][2][2][2])
{
    int r;
    int j;
    int k;
    int l;
    int m;

//#pragma unroll 16
    for (r = 0;r < CUBEHASH_ROUNDS;++r) {

        /* "add x_0jklm into x_1jklmn modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 7 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS7(x[0][j][k][l][m]);

        /* "swap x_00klm with x_01klm" */
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][0][k][l][m],x[0][1][k][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jk0m with x_1jk1m" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[1][j][k][0][m],x[1][j][k][1][m])

        /* "add x_0jklm into x_1jklm modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 11 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS11(x[0][j][k][l][m]);

        /* "swap x_0j0lm with x_0j1lm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][j][0][l][m],x[0][j][1][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jkl0 with x_1jkl1" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
                    SWAP(x[1][j][k][l][0],x[1][j][k][l][1])

    }
}

__device__ __forceinline__
static void block_tox(uint32_t block[16], uint32_t x[2][2][2][2][2])
{
    int k;
    int l;
    int m;
    uint32_t *in = block;

#pragma unroll 2
    for (k = 0;k < 2;++k)
#pragma unroll 2
        for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
                x[0][0][k][l][m] ^= *in++;
}

__device__ __forceinline__
static void hash_fromx(uint32_t hash[16], uint32_t x[2][2][2][2][2])
{
    int j;
    int k;
    int l;
    int m;
    uint32_t *out = hash;

#pragma unroll 2
    for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    *out++ = x[0][j][k][l][m];
}

__device__
void Init(uint32_t x[2][2][2][2][2])
{
    int i,j,k,l,m;
#if 0
    /* "the first three state words x_00000, x_00001, x_00010" */
    /* "are set to the integers h/8, b, r respectively." */
    /* "the remaining state words are set to 0." */
#pragma unroll 2
    for (i = 0;i < 2;++i)
#pragma unroll 2
      for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
          for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
              x[i][j][k][l][m] = 0;
    x[0][0][0][0][0] = 512/8;
    x[0][0][0][0][1] = CUBEHASH_BLOCKBYTES;
    x[0][0][0][1][0] = CUBEHASH_ROUNDS;

    /* "the state is then transformed invertibly through 10r identical rounds */
    for (i = 0;i < 10;++i) rrounds(x);
#else
    const uint32_t *iv = c_IV_512;

#pragma unroll 2
    for (i = 0;i < 2;++i)
#pragma unroll 2
      for (j = 0;j < 2;++j)
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
          for (l = 0;l < 2;++l)
#pragma unroll 2
            for (m = 0;m < 2;++m)
              x[i][j][k][l][m] = *iv++;
#endif
}

__device__ __forceinline__
static void Update32(uint32_t x[2][2][2][2][2], const BitSequence *data)
{
    /* "xor the block into the first b bytes of the state" */
    /* "and then transform the state invertibly through r identical rounds" */
    block_tox((uint32_t*)data, x);
    rrounds(x);
}

__device__ __forceinline__
static void Final(uint32_t x[2][2][2][2][2], BitSequence *hashval)
{
    int i;

    /* "the integer 1 is xored into the last state word x_11111" */
    x[1][1][1][1][1] ^= 1;

    /* "the state is then transformed invertibly through 10r identical rounds" */
#pragma unroll 10
    for (i = 0;i < 10;++i) rrounds(x);

    /* "output the first h/8 bytes of the state" */
    hash_fromx((uint32_t*)hashval, x);
}


/***************************************************/
// GPU Hash Function
__global__
void x11_cubehash512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[8 * hashPosition];

        uint32_t x[2][2][2][2][2];
        Init(x);

        // erste Hälfte des Hashes (32 bytes)
        Update32(x, (const BitSequence*)Hash);

        // zweite Hälfte des Hashes (32 bytes)
        Update32(x, (const BitSequence*)(Hash+8));

        // Padding Block
        uint32_t last[8];
        last[0] = 0x80;
#pragma unroll 7
        for (int i=1; i < 8; i++) last[i] = 0;
        Update32(x, (const BitSequence*)last);

        Final(x, (BitSequence*)Hash);
    }
}


// Setup-Funktionen
__host__
void x11_cubehash512_cpu_init(int thr_id, uint32_t threads)
{
}

__host__
void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Größe des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    x11_cubehash512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    //MyStreamSynchronize(NULL, order, thr_id);
}

