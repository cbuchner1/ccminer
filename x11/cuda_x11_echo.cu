#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

extern __device__ __device_builtin__ void __threadfence_block(void);

#include "cuda_x11_aes.cuh"

__device__ __forceinline__ void AES_2ROUND(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3,
	uint32_t &k0)
{
	uint32_t y0, y1, y2, y3;

	aes_round(sharedMemory,
		x0, x1, x2, x3,
		k0,
		y0, y1, y2, y3);

	aes_round(sharedMemory,
		y0, y1, y2, y3,
		x0, x1, x2, x3);

	// hier werden wir ein carry brauchen (oder auch nicht)
	k0++;
}

__device__ __forceinline__
void cuda_echo_round(
	const uint32_t *const __restrict__ sharedMemory, uint32_t *const __restrict__  hash)
{
	const uint32_t P[48] = {
	0xe7e9f5f5, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	0xa4213d7e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	//8-12
	0x01425eb8,	0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	0x65978b09, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	//21-25
	0x2cb6b661, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751,
	0x9ac2dea3, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	//34-38
	0x579f9f33, 0xfbfbfbfb, 0xfbfbfbfb, 0xefefd3c7,
	0xdbfde1dd, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	0x34514d9e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	0xb134347e, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
	0x14b8a457, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
	0x265f4382, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af
	//58-61
	};
	uint32_t k0;
	uint32_t h[16];

	#pragma unroll 16
	for (int i = 0; i < 16; i++)
	{
		h[i] = hash[i];
	}

	k0 = 512 + 8;

	#pragma unroll 4
	for (int idx = 0; idx < 16; idx += 4)
	{
		AES_2ROUND(sharedMemory,
			h[idx + 0], h[idx + 1], h[idx + 2], h[idx + 3], k0);
	}
	k0 += 4;

	uint32_t W[64];

	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		uint32_t a = P[i];
		uint32_t b = P[i + 4];
		uint32_t c = h[i + 8];
		uint32_t d = P[i + 8];

		uint32_t ab = a ^ b;
		uint32_t bc = b ^ c;
		uint32_t cd = c ^ d;


		uint32_t t =  (ab & 0x80808080);
		uint32_t t2 = (bc & 0x80808080);
		uint32_t t3 = (cd & 0x80808080);

		uint32_t abx = (t >> 7) * 27 ^ ((ab^t) << 1);
		uint32_t bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
		uint32_t cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

		W[0 + i] = abx ^ bc ^ d;
		W[0 + i + 4] = bcx ^ a ^ cd;
		W[0 + i + 8] = cdx ^ ab ^ d;
		W[0 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

		a = P[12 + i];
		b = h[i + 4];
		c = P[12 + i + 4];
		d = P[12 + i + 8];

		ab = a ^ b;
		bc = b ^ c;
		cd = c ^ d;


		t = (ab & 0x80808080);
		t2 = (bc & 0x80808080);
		t3 = (cd & 0x80808080);

		abx = (t >> 7) * 27 ^ ((ab^t) << 1);
		bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
		cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

		W[16 + i] = abx ^ bc ^ d;
		W[16 + i + 4] = bcx ^ a ^ cd;
		W[16 + i + 8] = cdx ^ ab ^ d;
		W[16 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

		a = h[i];
		b = P[24 + i + 0];
		c = P[24 + i + 4];
		d = P[24 + i + 8];

		ab = a ^ b;
		bc = b ^ c;
		cd = c ^ d;


		t = (ab & 0x80808080);
		t2 = (bc & 0x80808080);
		t3 = (cd & 0x80808080);

		abx = (t >> 7) * 27 ^ ((ab^t) << 1);
		bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
		cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

		W[32 + i] = abx ^ bc ^ d;
		W[32 + i + 4] = bcx ^ a ^ cd;
		W[32 + i + 8] = cdx ^ ab ^ d;
		W[32 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

		a = P[36 + i ];
		b = P[36 + i + 4];
		c = P[36 + i + 8];
		d = h[i + 12];

		ab = a ^ b;
		bc = b ^ c;
		cd = c ^ d;

		t = (ab & 0x80808080);
		t2 = (bc & 0x80808080);
		t3 = (cd & 0x80808080);

		abx = (t >> 7) * 27 ^ ((ab^t) << 1);
		bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
		cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

		W[48 + i] = abx ^ bc ^ d;
		W[48 + i + 4] = bcx ^ a ^ cd;
		W[48 + i + 8] = cdx ^ ab ^ d;
		W[48 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;

	}

	for (int k = 1; k < 10; k++)
	{

		// Big Sub Words
		#pragma unroll 4
		for (int idx = 0; idx < 64; idx += 16)
		{
			AES_2ROUND(sharedMemory,
				W[idx + 0], W[idx + 1], W[idx + 2], W[idx + 3],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 4], W[idx + 5], W[idx + 6], W[idx + 7],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 8], W[idx + 9], W[idx + 10], W[idx + 11],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 12], W[idx + 13], W[idx + 14], W[idx + 15],
				k0);
		}

		// Shift Rows
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			uint32_t t;

			/// 1, 5, 9, 13
			t = W[4 + i];
			W[4 + i] = W[20 + i];
			W[20 + i] = W[36 + i];
			W[36 + i] = W[52 + i];
			W[52 + i] = t;

			// 2, 6, 10, 14
			t = W[8 + i];
			W[8 + i] = W[40 + i];
			W[40 + i] = t;
			t = W[24 + i];
			W[24 + i] = W[56 + i];
			W[56 + i] = t;

			// 15, 11, 7, 3
			t = W[60 + i];
			W[60 + i] = W[44 + i];
			W[44 + i] = W[28 + i];
			W[28 + i] = W[12 + i];
			W[12 + i] = t;
		}

		// Mix Columns
		#pragma unroll 4
		for (int i = 0; i < 4; i++) // Schleife über je 2*uint32_t
		{
			#pragma unroll 4
			for (int idx = 0; idx < 64; idx += 16) // Schleife über die elemnte
			{
				uint32_t a = W[idx + i];
				uint32_t b = W[idx + i + 4];
				uint32_t c = W[idx + i + 8];
				uint32_t d = W[idx + i + 12];

				uint32_t ab = a ^ b;
				uint32_t bc = b ^ c;
				uint32_t cd = c ^ d;

				uint32_t t, t2, t3;
				t = (ab & 0x80808080);
				t2 = (bc & 0x80808080);
				t3 = (cd & 0x80808080);

				uint32_t abx = (t >> 7) * 27 ^ ((ab^t) << 1);
				uint32_t bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
				uint32_t cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

				W[idx + i] = abx ^ bc ^ d;
				W[idx + i + 4] = bcx ^ a ^ cd;
				W[idx + i + 8] = cdx ^ ab ^ d;
				W[idx + i + 12] = abx ^ bcx ^ cdx ^ ab ^ c;
			}
		}
	}

	#pragma unroll
	for (int i = 0; i < 16; i += 4)
	{
		W[i] ^= W[32 + i] ^ 512;
		W[i + 1] ^= W[32 + i + 1];
		W[i + 2] ^= W[32 + i + 2];
		W[i + 3] ^= W[32 + i + 3];
	}

	#pragma unroll
	for (int i = 0; i < 16; i++)
		hash[i] ^= W[i];
}

__device__ __forceinline__
void echo_gpu_init(uint32_t *const __restrict__ sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 128) {
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = d_AES1[threadIdx.x];
		sharedMemory[threadIdx.x + 512] = d_AES2[threadIdx.x];
		sharedMemory[threadIdx.x + 768] = d_AES3[threadIdx.x];

		sharedMemory[threadIdx.x + 64 * 2] = d_AES0[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 256] = d_AES1[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 512] = d_AES2[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 768] = d_AES3[threadIdx.x + 64 * 2];
	}
}

__global__ __launch_bounds__(128, 7) /* will force 72 registers */
void x11_echo512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__shared__ uint32_t sharedMemory[1024];

	echo_gpu_init(sharedMemory);
	__threadfence_block();

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *Hash = (uint32_t*)&g_hash[hashPosition<<3];

		cuda_echo_round(sharedMemory, Hash);
	}
}

__host__
void x11_echo512_cpu_init(int thr_id, uint32_t threads)
{
	aes_cpu_init(thr_id);
}

__host__
void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x11_echo512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	MyStreamSynchronize(NULL, order, thr_id);
}
