#include "hip/hip_runtime.h"
/*
 * Merged LUFFA512 64 + CUBE512 64 - from sp
 */

#include "cuda_helper.h"

#define MULT0(a) {\
	tmp = a[7]; \
	a[7] = a[6]; \
	a[6] = a[5]; \
	a[5] = a[4]; \
	a[4] = a[3] ^ tmp; \
	a[3] = a[2] ^ tmp; \
	a[2] = a[1]; \
	a[1] = a[0] ^ tmp; \
	a[0] = tmp; \
}

#define MULT2(a,j) { \
	tmp = a[(j<<3)+7]; \
	a[(j*8)+7] = a[(j*8)+6]; \
	a[(j*8)+6] = a[(j*8)+5]; \
	a[(j*8)+5] = a[(j*8)+4]; \
	a[(j*8)+4] = a[(j*8)+3] ^ tmp; \
	a[(j*8)+3] = a[(j*8)+2] ^ tmp; \
	a[(j*8)+2] = a[(j*8)+1]; \
	a[(j*8)+1] = a[(j*8)+0] ^ tmp; \
	a[j*8] = tmp; \
}

#define TWEAK(a0,a1,a2,a3,j) { \
	a0 = ROTL32(a0,j); \
	a1 = ROTL32(a1,j); \
	a2 = ROTL32(a2,j); \
	a3 = ROTL32(a3,j); \
}

#define STEP(c0,c1) { \
	SUBCRUMB(chainv[0],chainv[1],chainv[2],chainv[3],tmp); \
	SUBCRUMB(chainv[5],chainv[6],chainv[7],chainv[4],tmp); \
	MIXWORD(chainv[0],chainv[4]); \
	MIXWORD(chainv[1],chainv[5]); \
	MIXWORD(chainv[2],chainv[6]); \
	MIXWORD(chainv[3],chainv[7]); \
	ADD_CONSTANT(chainv[0],chainv[4],c0,c1); \
}

#define SUBCRUMB(a0,a1,a2,a3,a4) { \
	a4  = a0; \
	a0 |= a1; \
	a2 ^= a3; \
	a1  = ~a1;\
	a0 ^= a3; \
	a3 &= a4; \
	a1 ^= a3; \
	a3 ^= a2; \
	a2 &= a0; \
	a0  = ~a0;\
	a2 ^= a1; \
	a1 |= a3; \
	a4 ^= a1; \
	a3 ^= a2; \
	a2 &= a1; \
	a1 ^= a0; \
	a0  = a4; \
}

#define MIXWORD(a0,a4) { \
	a4 ^= a0; \
	a0  = ROTL32(a0,2); \
	a0 ^= a4; \
	a4  = ROTL32(a4,14); \
	a4 ^= a0; \
	a0  = ROTL32(a0,10); \
	a0 ^= a4; \
	a4  = ROTL32(a4,1); \
}

#define ADD_CONSTANT(a0,b0,c0,c1) { \
	a0 ^= c0; \
	b0 ^= c1; \
}

__device__ __constant__ uint32_t c_CNS[80] = {
	0x303994a6,0xe0337818,0xc0e65299,0x441ba90d,
	0x6cc33a12,0x7f34d442,0xdc56983e,0x9389217f,
	0x1e00108f,0xe5a8bce6,0x7800423d,0x5274baf4,
	0x8f5b7882,0x26889ba7,0x96e1db12,0x9a226e9d,
	0xb6de10ed,0x01685f3d,0x70f47aae,0x05a17cf4,
	0x0707a3d4,0xbd09caca,0x1c1e8f51,0xf4272b28,
	0x707a3d45,0x144ae5cc,0xaeb28562,0xfaa7ae2b,
	0xbaca1589,0x2e48f1c1,0x40a46f3e,0xb923c704,
	0xfc20d9d2,0xe25e72c1,0x34552e25,0xe623bb72,
	0x7ad8818f,0x5c58a4a4,0x8438764a,0x1e38e2e7,
	0xbb6de032,0x78e38b9d,0xedb780c8,0x27586719,
	0xd9847356,0x36eda57f,0xa2c78434,0x703aace7,
	0xb213afa5,0xe028c9bf,0xc84ebe95,0x44756f91,
	0x4e608a22,0x7e8fce32,0x56d858fe,0x956548be,
	0x343b138f,0xfe191be2,0xd0ec4e3d,0x3cb226e5,
	0x2ceb4882,0x5944a28e,0xb3ad2208,0xa1c4c355,
	0xf0d2e9e3,0x5090d577,0xac11d7fa,0x2d1925ab,
	0x1bcb66f2,0xb46496ac,0x6f2d9bc9,0xd1925ab0,
	0x78602649,0x29131ab6,0x8edae952,0x0fc053c3,
	0x3b6ba548,0x3f014f0c,0xedae9520,0xfc053c31
};

// Precalculated chaining values
__device__ __constant__ uint32_t c_IV[40] = {
	0x8bb0a761, 0xc2e4aa8b, 0x2d539bc9, 0x381408f8,
	0x478f6633, 0x255a46ff, 0x581c37f7, 0x601c2e8e,
	0x266c5f9d, 0xc34715d8, 0x8900670e, 0x51a540be,
	0xe4ce69fb, 0x5089f4d4, 0x3cc0a506, 0x609bcb02,
	0xa4e3cd82, 0xd24fd6ca, 0xc0f196dc, 0xcf41eafe,
	0x0ff2e673, 0x303804f2, 0xa7b3cd48, 0x677addd4,
	0x66e66a8a, 0x2303208f, 0x486dafb4, 0xc0d37dc6,
	0x634d15af, 0xe5af6747, 0x10af7e38, 0xee7e6428,
	0x01262e5d, 0xc92c2e64, 0x82fee966, 0xcea738d3,
	0x867de2b0, 0xe0714818, 0xda6e831f, 0xa7062529
};

/***************************************************/
__device__ __forceinline__
static void rnd512(uint32_t *statebuffer, uint32_t *statechainv)
{
	uint32_t t[40];
	uint32_t chainv[8];
	uint32_t tmp;
	int i,j;

	#pragma unroll
	for(i=0;i<8;i++) {
		t[i] = 0;
		#pragma unroll 5
		for(j=0;j<5;j++)
		   t[i] ^= statechainv[i+8*j];
	}

	MULT0(t);

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll
		for(i=0;i<8;i++)
			statechainv[i+8*j] ^= t[i];
	}

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll
		for(i=0;i<8;i++)
			t[i+8*j] = statechainv[i+8*j];
	}

	MULT0(statechainv);
	#pragma unroll 4
	for(j=1;j<5;j++) {
		MULT2(statechainv, j);
	}

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll
		for(i=0;i<8;i++)
			statechainv[8*j+i] ^= t[8*((j+1)%5)+i];
	}

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll
		for(i=0;i<8;i++)
			t[i+8*j] = statechainv[i+8*j];
	}

	MULT0(statechainv);
	#pragma unroll 4
	for(j=1;j<5;j++) {
		MULT2(statechainv, j);
	}

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll
		for(i=0;i<8;i++)
			statechainv[8*j+i] ^= t[8*((j+4)%5)+i];
	}

	#pragma unroll
	for(j=0;j<5;j++) {
		#pragma unroll 8
		for(i=0;i<8;i++)
			statechainv[i+8*j] ^= statebuffer[i];
		MULT0(statebuffer);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		chainv[i] = statechainv[i];
	}

	#pragma unroll 1
	for(i=0;i<8;i++) {
		STEP(c_CNS[(2*i)],c_CNS[(2*i)+1]);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		statechainv[i] = chainv[i];
		chainv[i] = statechainv[i+8];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],1);

	#pragma unroll 1
	for(i=0;i<8;i++) {
		STEP(c_CNS[(2*i)+16],c_CNS[(2*i)+16+1]);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		statechainv[i+8] = chainv[i];
		chainv[i] = statechainv[i+16];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],2);

	#pragma unroll 1
	for(i=0;i<8;i++) {
		STEP(c_CNS[(2*i)+32],c_CNS[(2*i)+32+1]);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		statechainv[i+16] = chainv[i];
		chainv[i] = statechainv[i+24];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],3);

	#pragma unroll 1
	for(i=0;i<8;i++) {
		STEP(c_CNS[(2*i)+48],c_CNS[(2*i)+48+1]);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		statechainv[i+24] = chainv[i];
		chainv[i] = statechainv[i+32];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],4);

	#pragma unroll 1
	for(i=0;i<8;i++) {
		STEP(c_CNS[(2*i)+64],c_CNS[(2*i)+64+1]);
	}

	#pragma unroll
	for(i=0;i<8;i++) {
		statechainv[i+32] = chainv[i];
	}
}

__device__ __forceinline__
static void rnd512_first(uint32_t state[40], uint32_t buffer[8])
{
	uint32_t chainv[8];
	uint32_t tmp;
	int i, j;

	for (j = 0; j<5; j++) {
		state[8 * j] ^= buffer[0];

		#pragma unroll 7
		for (i = 1; i<8; i++)
			state[i + 8 * j] ^= buffer[i];
		MULT0(buffer);
	}

	#pragma unroll
	for (i = 0; i<8; i++)
		chainv[i] = state[i];

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i)], c_CNS[(2 * i) + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i] = chainv[i];
		chainv[i] = state[i + 8];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 1);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 16], c_CNS[(2 * i) + 16 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 8] = chainv[i];
		chainv[i] = state[i + 16];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 2);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 32], c_CNS[(2 * i) + 32 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 16] = chainv[i];
		chainv[i] = state[i + 24];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 3);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 48], c_CNS[(2 * i) + 48 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 24] = chainv[i];
		chainv[i] = state[i + 32];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 4);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 64], c_CNS[(2 * i) + 64 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++)
		state[i + 32] = chainv[i];
}

/***************************************************/
__device__ __forceinline__
static void rnd512_nullhash(uint32_t *state)
{
	uint32_t t[40];
	uint32_t chainv[8];
	uint32_t tmp;
	int i, j;

	#pragma unroll
	for (i = 0; i<8; i++) {
		t[i] = state[i + 8 * 0];
		#pragma unroll 4
		for (j = 1; j<5; j++)
			t[i] ^= state[i + 8 * j];
	}

	MULT0(t);

	#pragma unroll
	for (j = 0; j<5; j++) {
		#pragma unroll
		for (i = 0; i<8; i++)
			state[i + 8 * j] ^= t[i];
	}

	#pragma unroll
	for (j = 0; j<5; j++) {
		#pragma unroll
		for (i = 0; i<8; i++)
			t[i + 8 * j] = state[i + 8 * j];
	}

	MULT0(state);
	#pragma unroll 4
	for(j=1; j<5; j++) {
		MULT2(state, j);
	}

	#pragma unroll
	for (j = 0; j<5; j++) {
		#pragma unroll
		for (i = 0; i<8; i++)
			state[8 * j + i] ^= t[8 * ((j + 1) % 5) + i];
	}

	#pragma unroll
	for (j = 0; j<5; j++) {
		#pragma unroll 8
		for (i = 0; i<8; i++)
			t[i + 8 * j] = state[i + 8 * j];
	}

	MULT0(state);
	#pragma unroll 4
	for(j=1; j<5; j++) {
		MULT2(state, j);
	}

	#pragma unroll
	for (j = 0; j<5; j++) {
		#pragma unroll
		for (i = 0; i<8; i++)
			state[8 * j + i] ^= t[8 * ((j + 4) % 5) + i];
	}

	#pragma unroll
	for (i = 0; i<8; i++)
		chainv[i] = state[i];

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i)], c_CNS[(2 * i) + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i] = chainv[i];
		chainv[i] = state[i + 8];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 1);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 16], c_CNS[(2 * i) + 16 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 8] = chainv[i];
		chainv[i] = state[i + 16];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 2);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 32], c_CNS[(2 * i) + 32 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 16] = chainv[i];
		chainv[i] = state[i + 24];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 3);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 48], c_CNS[(2 * i) + 48 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 24] = chainv[i];
		chainv[i] = state[i + 32];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 4);

	#pragma unroll 1
	for (i = 0; i<8; i++) {
		STEP(c_CNS[(2 * i) + 64], c_CNS[(2 * i) + 64 + 1]);
	}

	#pragma unroll
	for (i = 0; i<8; i++) {
		state[i + 32] = chainv[i];
	}
}

__device__ __forceinline__
static void Update512(uint32_t *statebuffer, uint32_t *statechainv, const uint32_t *data)
{
	#pragma unroll
	for (int i = 0; i < 8; i++) statebuffer[i] = cuda_swab32(data[i]);
	rnd512_first(statechainv, statebuffer);

	#pragma unroll
	for (int i = 0; i < 8; i++) statebuffer[i] = cuda_swab32(data[i + 8]);
	rnd512(statebuffer, statechainv);
}

/***************************************************/
__device__ __forceinline__
static void finalization512(uint32_t *statebuffer, uint32_t *statechainv, uint32_t *b)
{
	int i,j;

	statebuffer[0] = 0x80000000;
	#pragma unroll 7
	for(int i=1;i<8;i++) statebuffer[i] = 0;
	rnd512(statebuffer, statechainv);

	/*---- blank round with m=0 ----*/
	rnd512_nullhash(statechainv);

	#pragma unroll
	for(i=0;i<8;i++) {
		b[i] = statechainv[i];
		#pragma unroll 4
		for(j=1;j<5;j++) {
			b[i] ^= statechainv[i+8*j];
		}
		b[i] = cuda_swab32((b[i]));
	}

	rnd512_nullhash(statechainv);

	#pragma unroll
	for(i=0;i<8;i++) {
		b[8 + i] = statechainv[i];
		#pragma unroll 4
		for(j=1;j<5;j++) {
			b[8+i] ^= statechainv[i+8*j];
		}
		b[8 + i] = cuda_swab32((b[8 + i]));
	}
}

#define ROUND_EVEN { \
	xg = (x0 + xg); \
	x0 = ROTL32(x0, 7); \
	xh = (x1 + xh); \
	x1 = ROTL32(x1, 7); \
	xi = (x2 + xi); \
	x2 = ROTL32(x2, 7); \
	xj = (x3 + xj); \
	x3 = ROTL32(x3, 7); \
	xk = (x4 + xk); \
	x4 = ROTL32(x4, 7); \
	xl = (x5 + xl); \
	x5 = ROTL32(x5, 7); \
	xm = (x6 + xm); \
	x6 = ROTL32(x6, 7); \
	xn = (x7 + xn); \
	x7 = ROTL32(x7, 7); \
	xo = (x8 + xo); \
	x8 = ROTL32(x8, 7); \
	xp = (x9 + xp); \
	x9 = ROTL32(x9, 7); \
	xq = (xa + xq); \
	xa = ROTL32(xa, 7); \
	xr = (xb + xr); \
	xb = ROTL32(xb, 7); \
	xs = (xc + xs); \
	xc = ROTL32(xc, 7); \
	xt = (xd + xt); \
	xd = ROTL32(xd, 7); \
	xu = (xe + xu); \
	xe = ROTL32(xe, 7); \
	xv = (xf + xv); \
	xf = ROTL32(xf, 7); \
	x8 ^= xg; \
	x9 ^= xh; \
	xa ^= xi; \
	xb ^= xj; \
	xc ^= xk; \
	xd ^= xl; \
	xe ^= xm; \
	xf ^= xn; \
	x0 ^= xo; \
	x1 ^= xp; \
	x2 ^= xq; \
	x3 ^= xr; \
	x4 ^= xs; \
	x5 ^= xt; \
	x6 ^= xu; \
	x7 ^= xv; \
	xi = (x8 + xi); \
	x8 = ROTL32(x8, 11); \
	xj = (x9 + xj); \
	x9 = ROTL32(x9, 11); \
	xg = (xa + xg); \
	xa = ROTL32(xa, 11); \
	xh = (xb + xh); \
	xb = ROTL32(xb, 11); \
	xm = (xc + xm); \
	xc = ROTL32(xc, 11); \
	xn = (xd + xn); \
	xd = ROTL32(xd, 11); \
	xk = (xe + xk); \
	xe = ROTL32(xe, 11); \
	xl = (xf + xl); \
	xf = ROTL32(xf, 11); \
	xq = (x0 + xq); \
	x0 = ROTL32(x0, 11); \
	xr = (x1 + xr); \
	x1 = ROTL32(x1, 11); \
	xo = (x2 + xo); \
	x2 = ROTL32(x2, 11); \
	xp = (x3 + xp); \
	x3 = ROTL32(x3, 11); \
	xu = (x4 + xu); \
	x4 = ROTL32(x4, 11); \
	xv = (x5 + xv); \
	x5 = ROTL32(x5, 11); \
	xs = (x6 + xs); \
	x6 = ROTL32(x6, 11); \
	xt = (x7 + xt); \
	x7 = ROTL32(x7, 11); \
	xc ^= xi; \
	xd ^= xj; \
	xe ^= xg; \
	xf ^= xh; \
	x8 ^= xm; \
	x9 ^= xn; \
	xa ^= xk; \
	xb ^= xl; \
	x4 ^= xq; \
	x5 ^= xr; \
	x6 ^= xo; \
	x7 ^= xp; \
	x0 ^= xu; \
	x1 ^= xv; \
	x2 ^= xs; \
	x3 ^= xt; \
}

#define ROUND_ODD { \
	xj = (xc + xj); \
	xc = ROTL32(xc, 7); \
	xi = (xd + xi); \
	xd = ROTL32(xd, 7); \
	xh = (xe + xh); \
	xe = ROTL32(xe, 7); \
	xg = (xf + xg); \
	xf = ROTL32(xf, 7); \
	xn = (x8 + xn); \
	x8 = ROTL32(x8, 7); \
	xm = (x9 + xm); \
	x9 = ROTL32(x9, 7); \
	xl = (xa + xl); \
	xa = ROTL32(xa, 7); \
	xk = (xb + xk); \
	xb = ROTL32(xb, 7); \
	xr = (x4 + xr); \
	x4 = ROTL32(x4, 7); \
	xq = (x5 + xq); \
	x5 = ROTL32(x5, 7); \
	xp = (x6 + xp); \
	x6 = ROTL32(x6, 7); \
	xo = (x7 + xo); \
	x7 = ROTL32(x7, 7); \
	xv = (x0 + xv); \
	x0 = ROTL32(x0, 7); \
	xu = (x1 + xu); \
	x1 = ROTL32(x1, 7); \
	xt = (x2 + xt); \
	x2 = ROTL32(x2, 7); \
	xs = (x3 + xs); \
	x3 = ROTL32(x3, 7); \
	x4 ^= xj; \
	x5 ^= xi; \
	x6 ^= xh; \
	x7 ^= xg; \
	x0 ^= xn; \
	x1 ^= xm; \
	x2 ^= xl; \
	x3 ^= xk; \
	xc ^= xr; \
	xd ^= xq; \
	xe ^= xp; \
	xf ^= xo; \
	x8 ^= xv; \
	x9 ^= xu; \
	xa ^= xt; \
	xb ^= xs; \
	xh = (x4 + xh); \
	x4 = ROTL32(x4, 11); \
	xg = (x5 + xg); \
	x5 = ROTL32(x5, 11); \
	xj = (x6 + xj); \
	x6 = ROTL32(x6, 11); \
	xi = (x7 + xi); \
	x7 = ROTL32(x7, 11); \
	xl = (x0 + xl); \
	x0 = ROTL32(x0, 11); \
	xk = (x1 + xk); \
	x1 = ROTL32(x1, 11); \
	xn = (x2 + xn); \
	x2 = ROTL32(x2, 11); \
	xm = (x3 + xm); \
	x3 = ROTL32(x3, 11); \
	xp = (xc + xp); \
	xc = ROTL32(xc, 11); \
	xo = (xd + xo); \
	xd = ROTL32(xd, 11); \
	xr = (xe + xr); \
	xe = ROTL32(xe, 11); \
	xq = (xf + xq); \
	xf = ROTL32(xf, 11); \
	xt = (x8 + xt); \
	x8 = ROTL32(x8, 11); \
	xs = (x9 + xs); \
	x9 = ROTL32(x9, 11); \
	xv = (xa + xv); \
	xa = ROTL32(xa, 11); \
	xu = (xb + xu); \
	xb = ROTL32(xb, 11); \
	x0 ^= xh; \
	x1 ^= xg; \
	x2 ^= xj; \
	x3 ^= xi; \
	x4 ^= xl; \
	x5 ^= xk; \
	x6 ^= xn; \
	x7 ^= xm; \
	x8 ^= xp; \
	x9 ^= xo; \
	xa ^= xr; \
	xb ^= xq; \
	xc ^= xt; \
	xd ^= xs; \
	xe ^= xv; \
	xf ^= xu; \
}

#define SIXTEEN_ROUNDS \
	for (int j = 0; j < 8; j ++) { \
		ROUND_EVEN; \
		ROUND_ODD; \
	}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(256, 4)
#endif
void x11_luffaCubehash512_gpu_hash_64(uint32_t threads, uint32_t *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t statechainv[40] = {
			0x8bb0a761, 0xc2e4aa8b, 0x2d539bc9, 0x381408f8,
			0x478f6633, 0x255a46ff, 0x581c37f7, 0x601c2e8e,
			0x266c5f9d, 0xc34715d8, 0x8900670e, 0x51a540be,
			0xe4ce69fb, 0x5089f4d4, 0x3cc0a506, 0x609bcb02,
			0xa4e3cd82, 0xd24fd6ca, 0xc0f196dc, 0xcf41eafe,
			0x0ff2e673, 0x303804f2, 0xa7b3cd48, 0x677addd4,
			0x66e66a8a, 0x2303208f, 0x486dafb4, 0xc0d37dc6,
			0x634d15af, 0xe5af6747, 0x10af7e38, 0xee7e6428,
			0x01262e5d, 0xc92c2e64, 0x82fee966, 0xcea738d3,
			0x867de2b0, 0xe0714818, 0xda6e831f, 0xa7062529
		};

		uint32_t statebuffer[8];
		uint32_t *const Hash = &g_hash[thread * 16U];

		Update512(statebuffer, statechainv, Hash);
		finalization512(statebuffer, statechainv, Hash);

		//Cubehash

		uint32_t x0 = 0x2AEA2A61, x1 = 0x50F494D4, x2 = 0x2D538B8B, x3 = 0x4167D83E;
		uint32_t x4 = 0x3FEE2313, x5 = 0xC701CF8C, x6 = 0xCC39968E, x7 = 0x50AC5695;
		uint32_t x8 = 0x4D42C787, x9 = 0xA647A8B3, xa = 0x97CF0BEF, xb = 0x825B4537;
		uint32_t xc = 0xEEF864D2, xd = 0xF22090C4, xe = 0xD0E5CD33, xf = 0xA23911AE;
		uint32_t xg = 0xFCD398D9, xh = 0x148FE485, xi = 0x1B017BEF, xj = 0xB6444532;
		uint32_t xk = 0x6A536159, xl = 0x2FF5781C, xm = 0x91FA7934, xn = 0x0DBADEA9;
		uint32_t xo = 0xD65C8A2B, xp = 0xA5A70E75, xq = 0xB1C62456, xr = 0xBC796576;
		uint32_t xs = 0x1921C8F7, xt = 0xE7989AF1, xu = 0x7795D246, xv = 0xD43E3B44;

		x0 ^= Hash[0];
		x1 ^= Hash[1];
		x2 ^= Hash[2];
		x3 ^= Hash[3];
		x4 ^= Hash[4];
		x5 ^= Hash[5];
		x6 ^= Hash[6];
		x7 ^= Hash[7];

		SIXTEEN_ROUNDS;

		x0 ^= Hash[8];
		x1 ^= Hash[9];
		x2 ^= Hash[10];
		x3 ^= Hash[11];
		x4 ^= Hash[12];
		x5 ^= Hash[13];
		x6 ^= Hash[14];
		x7 ^= Hash[15];

		SIXTEEN_ROUNDS;
		x0 ^= 0x80;

		SIXTEEN_ROUNDS;
		xv ^= 1;

		for (int i = 3; i < 13; i++) {
			SIXTEEN_ROUNDS;
		}

		Hash[0] = x0;
		Hash[1] = x1;
		Hash[2] = x2;
		Hash[3] = x3;
		Hash[4] = x4;
		Hash[5] = x5;
		Hash[6] = x6;
		Hash[7] = x7;
		Hash[8] = x8;
		Hash[9] = x9;
		Hash[10] = xa;
		Hash[11] = xb;
		Hash[12] = xc;
		Hash[13] = xd;
		Hash[14] = xe;
		Hash[15] = xf;
	}
}

__host__
void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x11_luffaCubehash512_gpu_hash_64 <<<grid, block>>> (threads, d_hash);
	MyStreamSynchronize(NULL, order, thr_id);
}

// Setup
__host__
void x11_luffaCubehash512_cpu_init(int thr_id, uint32_t threads) {}
