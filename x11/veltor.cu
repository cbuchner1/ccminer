#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_shabal.h"
#include "sph/sph_streebog.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int swap);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

// for SM3.x
extern void streebog_sm3_set_target(uint32_t* ptarget);
extern void streebog_sm3_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* d_resNonce);

// for latest cards only
extern void skunk_cpu_init(int thr_id, uint32_t threads);
extern void skunk_streebog_set_target(uint32_t* ptarget);
extern void skunk_cuda_streebog(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* d_resNonce);

#include <stdio.h>
#include <memory.h>

#define NBN 2
static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

// veltor CPU Hash
extern "C" void veltorhash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_skein512_context ctx_skein;
	sph_gost512_context ctx_gost;
	sph_shabal512_context ctx_shabal;
	sph_shavite512_context ctx_shavite;

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, (const void*) hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*) hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*) hash, 64);
	sph_gost512_close(&ctx_gost, (void*) hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_veltor(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	int dev_id = device_map[thr_id];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] > 500) ? 20 : 18;
	if (strstr(device_name[dev_id], "GTX 10")) intensity = 21;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xf;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		skunk_cpu_init(thr_id, throughput);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		x11_shavite512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);

		init[thr_id] = true;
	}

	uint32_t _ALIGN(64) h_resNonce[NBN];
	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(endiandata);

	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
	if(use_compat_kernels[thr_id])
		streebog_sm3_set_target(ptarget);
	else
		skunk_streebog_set_target(ptarget);

	do {
		int order = 0;
		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if(use_compat_kernels[thr_id])
			streebog_sm3_hash_64_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id]);
		else
			skunk_cuda_streebog(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id]);

		hipMemcpy(h_resNonce, d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (h_resNonce[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];

			be32enc(&endiandata[19], startNounce + h_resNonce[0]);
			veltorhash(vhash, endiandata);
			if (vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				work->nonces[0] = startNounce + h_resNonce[0];
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (h_resNonce[1] != UINT32_MAX)
				{
					uint32_t secNonce = work->nonces[1] = startNounce + h_resNonce[1];
					be32enc(&endiandata[19], secNonce);
					veltorhash(vhash, endiandata);
					work->nonces[1] = secNonce;
					if (bn_hash_target_ratio(vhash, ptarget) > work->shareratio[0]) {
						work_set_target_ratio(work, vhash);
						xchg(work->nonces[1], work->nonces[0]);
					} else {
						bn_set_target_ratio(work, vhash, work->valid_nonces);
					}
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[0]);
				hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
				pdata[19] = startNounce + h_resNonce[0] + 1;
				continue;
			}
		}
		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_veltor(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
