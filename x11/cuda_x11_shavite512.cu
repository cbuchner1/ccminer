#include "hip/hip_runtime.h"
// aus heavy.cu
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

 __constant__ uint32_t c_PaddedMessage80[32]; // padded message (80 bytes + padding)

static __constant__ uint32_t d_ShaviteInitVector[16];
static const uint32_t h_ShaviteInitVector[] = {
	SPH_C32(0x72FCCDD8), SPH_C32(0x79CA4727), SPH_C32(0x128A077B), SPH_C32(0x40D55AEC),
	SPH_C32(0xD1901A06), SPH_C32(0x430AE307), SPH_C32(0xB29F5CD1), SPH_C32(0xDF07FBFC),
	SPH_C32(0x8E45D73D), SPH_C32(0x681AB538), SPH_C32(0xBDE86578), SPH_C32(0xDD577E47),
	SPH_C32(0xE275EADE), SPH_C32(0x502D9FCD), SPH_C32(0xB9357178), SPH_C32(0x022A4B9A)
};

#include "cuda_x11_aes.cu"

static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return __byte_perm(x, 0, 0x0123);
}

static __device__ __forceinline__ void AES_ROUND_NOKEY(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3)
{
	uint32_t y0, y1, y2, y3;
	aes_round(sharedMemory,
		x0, x1, x2, x3,
		y0, y1, y2, y3);

	x0 = y0;
	x1 = y1;
	x2 = y2;
	x3 = y3;
}

static __device__ __forceinline__ void KEY_EXPAND_ELT(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &k0, uint32_t &k1, uint32_t &k2, uint32_t &k3)
{
	uint32_t y0, y1, y2, y3;
	aes_round(sharedMemory,
		k0, k1, k2, k3,
		y0, y1, y2, y3);

	k0 = y1;
	k1 = y2;
	k2 = y3;
	k3 = y0;
}

static __device__ void
c512(const uint32_t* sharedMemory, uint32_t *state, uint32_t *msg, uint32_t count)
{
	uint32_t p0, p1, p2, p3, p4, p5, p6, p7;
	uint32_t p8, p9, pA, pB, pC, pD, pE, pF;
	uint32_t x0, x1, x2, x3;
	uint32_t rk00, rk01, rk02, rk03, rk04, rk05, rk06, rk07;
	uint32_t rk08, rk09, rk0A, rk0B, rk0C, rk0D, rk0E, rk0F;
	uint32_t rk10, rk11, rk12, rk13, rk14, rk15, rk16, rk17;
	uint32_t rk18, rk19, rk1A, rk1B, rk1C, rk1D, rk1E, rk1F;
	const uint32_t counter = count;

	p0 = state[0x0];
	p1 = state[0x1];
	p2 = state[0x2];
	p3 = state[0x3];
	p4 = state[0x4];
	p5 = state[0x5];
	p6 = state[0x6];
	p7 = state[0x7];
	p8 = state[0x8];
	p9 = state[0x9];
	pA = state[0xA];
	pB = state[0xB];
	pC = state[0xC];
	pD = state[0xD];
	pE = state[0xE];
	pF = state[0xF];
	/* round 0 */
	rk00 = msg[0];
	x0 = p4 ^ rk00;
	rk01 = msg[1];
	x1 = p5 ^ rk01;
	rk02 = msg[2];
	x2 = p6 ^ rk02;
	rk03 = msg[3];
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 = msg[4];
	x0 ^= rk04;
	rk05 = msg[5];
	x1 ^= rk05;
	rk06 = msg[6];
	x2 ^= rk06;
	rk07 = msg[7];
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 = msg[8];
	x0 ^= rk08;
	rk09 = msg[9];
	x1 ^= rk09;
	rk0A = msg[10];
	x2 ^= rk0A;
	rk0B = msg[11];
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C = msg[12];
	x0 ^= rk0C;
	rk0D = msg[13];
	x1 ^= rk0D;
	rk0E = msg[14];
	x2 ^= rk0E;
	rk0F = msg[15];
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	rk10 = msg[16];
	x0 = pC ^ rk10;
	rk11 = msg[17];
	x1 = pD ^ rk11;
	rk12 = msg[18];
	x2 = pE ^ rk12;
	rk13 = msg[19];
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 = msg[20];
	x0 ^= rk14;
	rk15 = msg[21];
	x1 ^= rk15;
	rk16 = msg[22];
	x2 ^= rk16;
	rk17 = msg[23];
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 = msg[24];
	x0 ^= rk18;
	rk19 = msg[25];
	x1 ^= rk19;
	rk1A = msg[26];
	x2 ^= rk1A;
	rk1B = msg[27];
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C = msg[28];
	x0 ^= rk1C;
	rk1D = msg[29];
	x1 ^= rk1D;
	rk1E = msg[30];
	x2 ^= rk1E;
	rk1F = msg[31];
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 1
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	rk00 ^= counter;
	rk03 ^= 0xFFFFFFFF;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	/* round 4, 8, 12 */
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 2
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;	
	rk07 ^= SPH_T32(~counter);
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	/* round 4, 8, 12 */
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 3
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	rk1E ^= counter;
	rk1F ^= 0xFFFFFFFF;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	/* round 4, 8, 12 */
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	/* round 13 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15 ^ counter;
	rk1A ^= rk16;
	rk1B ^= rk17 ^ 0xFFFFFFFF;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	state[0x0] ^= p8;
	state[0x1] ^= p9;
	state[0x2] ^= pA;
	state[0x3] ^= pB;
	state[0x4] ^= pC;
	state[0x5] ^= pD;
	state[0x6] ^= pE;
	state[0x7] ^= pF;
	state[0x8] ^= p0;
	state[0x9] ^= p1;
	state[0xA] ^= p2;
	state[0xB] ^= p3;
	state[0xC] ^= p4;
	state[0xD] ^= p5;
	state[0xE] ^= p6;
	state[0xF] ^= p7;
}



__global__ void x11_shavite512_gpu_hash_80(int threads, uint32_t startNounce, void *outputHash)
{
	__shared__ uint32_t sharedMemory[1024];

	        aes_gpu_init(sharedMemory);	
  

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
		uint32_t nounce = startNounce + thread;
	
		// kopiere init-state
		uint32_t state[16];


#pragma unroll 16
		for(int i=0;i<16;i++) {
			state[i] = d_ShaviteInitVector[i];}

		uint32_t msg[32];

#pragma unroll 32
		for(int i=0;i<32;i++) {			
			msg[i]  = c_PaddedMessage80[i];}
		    msg[19] = cuda_swab32(nounce);
			msg[20] = 0x80;
			msg[27] = 0x2800000;
			msg[31] = 0x2000000;

		c512(sharedMemory, state, msg,640);

uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;

#pragma unroll 16
		for(int i=0;i<16;i++)
			outHash[i] = state[i];


	} //thread < threads
}
// Die Hash-Funktion
__global__ void x11_shavite512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__shared__ uint32_t sharedMemory[1024];

	aes_gpu_init(sharedMemory);


    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[8 * hashPosition];

		// kopiere init-state
		uint32_t state[16];

#pragma unroll 16
		for(int i=0;i<16;i++)
			state[i] = d_ShaviteInitVector[i];

		// nachricht laden
		uint32_t msg[32];

		// f�lle die Nachricht mit 64-byte (vorheriger Hash)
#pragma unroll 16
		for(int i=0;i<16;i++)
			msg[i] = Hash[i];			

		// Nachrichtenende
		msg[16] = 0x80;
#pragma unroll 10
		for(int i=17;i<27;i++)
			msg[i] = 0;

		msg[27] = 0x02000000;
		msg[28] = 0;
		msg[29] = 0;
		msg[30] = 0;
		msg[31] = 0x02000000;

		c512(sharedMemory, state, msg, 512);

#pragma unroll 16
		for(int i=0;i<16;i++)
			Hash[i] = state[i];
    } // thread < threads
}


// Setup-Funktionen
__host__ void x11_shavite512_cpu_init(int thr_id, int threads)
{
	aes_cpu_init();

	hipMemcpyToSymbol(HIP_SYMBOL( d_ShaviteInitVector),
                        h_ShaviteInitVector,
                        sizeof(h_ShaviteInitVector),
                        0, hipMemcpyHostToDevice);
}

__host__ void x11_shavite512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    size_t shared_size = 0;

    x11_shavite512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void x11_shavite512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	x11_shavite512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}
__host__ void x11_shavite512_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 32*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

