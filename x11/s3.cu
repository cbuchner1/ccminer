#include "hip/hip_runtime.h"
/**
 * S3 Hash (Also called 3S - Used by 1Coin)
 */

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdint.h>

static uint32_t *d_hash[MAX_GPUS];

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void x11_shavite512_setBlock_80(void *pdata);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

/* CPU HASH */
extern "C" void s3hash(void *output, const void *input)
{
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_skein512_context ctx_skein;

	unsigned char hash[64];

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

/* Main S3 entry point */
extern "C" int scanhash_s3(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	int intensity = 20; // 256*256*8*2;
#ifdef WIN32
	// reduce by one the intensity on windows
	intensity--;
#endif
	uint32_t throughput =  device_intensity(thr_id, __func__, 1 << intensity);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0xF;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	x11_shavite512_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		const uint32_t Htarg = ptarget[7];
		uint32_t foundNonce;
		int order = 0;

		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			s3hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;

			} else {
				applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
