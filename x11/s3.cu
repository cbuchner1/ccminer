#include "hip/hip_runtime.h"
/**
 * S3 Hash (Also called Triple S - Used by 1Coin)
 */

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

#include <stdint.h>

static uint32_t *d_hash[MAX_GPUS];

/* CPU HASH */
extern "C" void s3hash(void *output, const void *input)
{
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_skein512_context ctx_skein;

	unsigned char hash[64];

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	memcpy(output, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("S3 %s %08x %08x %08x %08x...%08x\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

/* Main S3 entry point */
extern "C" int scanhash_s3(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = 20; // 256*256*8*2;
#ifdef WIN32
	// reduce by one the intensity on windows
	intensity--;
#endif
	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << intensity);
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xF;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	x11_shavite512_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE("shavite:");
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("simd   :");
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("skein  :");

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			s3hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					s3hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_s3(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
