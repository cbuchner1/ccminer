#include "hip/hip_runtime.h"
/***************************************************************************************************
 * SIMD512 SM3+ CUDA IMPLEMENTATION (require cuda_x11_simd512_func.cuh)
 */

#include "miner.h"
#include "cuda_helper.h"

#define TPB 128

uint32_t *d_state[MAX_GPUS];
uint4 *d_temp4[MAX_GPUS];

// texture bound to d_temp4[thr_id], for read access in Compaction kernel
texture<uint4, 1, hipReadModeElementType> texRef1D_128;

#define DEVICE_DIRECT_CONSTANTS

#ifdef DEVICE_DIRECT_CONSTANTS
__constant__ uint8_t c_perm[8][8] = {
#else
__constant__ uint8_t c_perm[8][8];
const uint8_t h_perm[8][8] = {
#endif
	{ 2, 3, 6, 7, 0, 1, 4, 5 },
	{ 6, 7, 2, 3, 4, 5, 0, 1 },
	{ 7, 6, 5, 4, 3, 2, 1, 0 },
	{ 1, 0, 3, 2, 5, 4, 7, 6 },
	{ 0, 1, 4, 5, 6, 7, 2, 3 },
	{ 6, 7, 2, 3, 0, 1, 4, 5 },
	{ 6, 7, 0, 1, 4, 5, 2, 3 },
	{ 4, 5, 2, 3, 6, 7, 0, 1 }
};

/* used in cuda_x11_simd512_func.cuh (SIMD_Compress2) */
#ifdef DEVICE_DIRECT_CONSTANTS
__constant__ uint32_t c_IV_512[32] = {
#else
__constant__ uint32_t c_IV_512[32];
const uint32_t h_IV_512[32] = {
#endif
	0x0ba16b95, 0x72f999ad, 0x9fecc2ae, 0xba3264fc, 0x5e894929, 0x8e9f30e5, 0x2f1daa37, 0xf0f2c558,
	0xac506643, 0xa90635a5, 0xe25b878b, 0xaab7878f, 0x88817f7a, 0x0a02892b, 0x559a7550, 0x598f657e,
	0x7eef60a1, 0x6b70e3e8, 0x9c1714d1, 0xb958e2a8, 0xab02675e, 0xed1c014f, 0xcd8d65bb, 0xfdb7a257,
	0x09254899, 0xd699c7bc, 0x9019b6dc, 0x2b9022e4, 0x8fa14956, 0x21bf9bd3, 0xb94d0943, 0x6ffddc22
};

#ifdef DEVICE_DIRECT_CONSTANTS
__constant__ short c_FFT128_8_16_Twiddle[128] = {
#else
__constant__ short c_FFT128_8_16_Twiddle[128];
static const short h_FFT128_8_16_Twiddle[128] = {
#endif
	1,   1,   1,   1,   1,    1,   1,   1,   1,   1,   1,   1,   1,   1,   1,   1,
	1,  60,   2, 120,   4,  -17,   8, -34,  16, -68,  32, 121,  64, -15, 128, -30,
	1,  46,  60, -67,   2,   92, 120, 123,   4, -73, -17, -11,   8, 111, -34, -22,
	1, -67, 120, -73,   8,  -22, -68, -70,  64,  81, -30, -46,  -2,-123,  17,-111,
	1,-118,  46, -31,  60,  116, -67, -61,   2,  21,  92, -62, 120, -25, 123,-122,
	1, 116,  92,-122, -17,   84, -22,  18,  32, 114, 117, -49, -30, 118,  67,  62,
	1, -31, -67,  21, 120, -122, -73, -50,   8,   9, -22, -89, -68,  52, -70, 114,
	1, -61, 123, -50, -34,   18, -70, -99, 128, -98,  67,  25,  17,  -9,  35, -79
};

#ifdef DEVICE_DIRECT_CONSTANTS
__constant__ short c_FFT256_2_128_Twiddle[128] = {
#else
__constant__ short c_FFT256_2_128_Twiddle[128];
static const short h_FFT256_2_128_Twiddle[128] = {
#endif
	  1,  41,-118,  45,  46,  87, -31,  14,
	 60,-110, 116,-127, -67,  80, -61,  69,
	  2,  82,  21,  90,  92, -83, -62,  28,
	120,  37, -25,   3, 123, -97,-122,-119,
	  4, -93,  42, -77, -73,  91,-124,  56,
	-17,  74, -50,   6, -11,  63,  13,  19,
	  8,  71,  84, 103, 111, -75,   9, 112,
	-34,-109,-100,  12, -22, 126,  26,  38,
	 16,-115, -89, -51, -35, 107,  18, -33,
	-68,  39,  57,  24, -44,  -5,  52,  76,
	 32,  27,  79,-102, -70, -43,  36, -66,
	121,  78, 114,  48, -88, -10, 104,-105,
	 64,  54, -99,  53, 117, -86,  72, 125,
	-15,-101, -29,  96,  81, -20, -49,  47,
	128, 108,  59, 106, -23,  85,-113,  -7,
	-30,  55, -58, -65, -95, -40, -98,  94
};

/************* the round function ****************/
#define IF(x, y, z) (((y ^ z) & x) ^ z)
#define MAJ(x, y, z) ((z &y) | ((z|y) & x))

#include "cuda_x11_simd512_sm2.cuh"
#include "cuda_x11_simd512_func.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#if __CUDA_ARCH__ >= 300

/********************* Message expansion ************************/

/*
 * Reduce modulo 257; result is in [-127; 383]
 * REDUCE(x) := (x&255) - (x>>8)
 */
#define REDUCE(x) \
	(((x)&255) - ((x)>>8))

/*
 * Reduce from [-127; 383] to [-128; 128]
 * EXTRA_REDUCE_S(x) := x<=128 ? x : x-257
 */
#define EXTRA_REDUCE_S(x) \
	((x)<=128 ? (x) : (x)-257)

/*
 * Reduce modulo 257; result is in [-128; 128]
 */
#define REDUCE_FULL_S(x) \
	EXTRA_REDUCE_S(REDUCE(x))

// Parallelization:
//
// FFT_8  wird 2 times 8-fach parallel ausgeführt (in FFT_64)
//        and  1 time 16-fach parallel (in FFT_128_full)
//
// STEP8_IF and STEP8_MAJ beinhalten je 2x 8-fach parallel Operations

/**
 * FFT_8 using w=4 as 8th root of unity
 * Unrolled decimation in frequency (DIF) radix-2 NTT.
 * Output data is in revbin_permuted order.
 */
__device__ __forceinline__
void FFT_8(int *y, int stripe)
{
#define X(i) y[stripe*i]

#define DO_REDUCE(i) \
	X(i) = REDUCE(X(i))

#define DO_REDUCE_FULL_S(i) \
do { \
	X(i) = REDUCE(X(i)); \
	X(i) = EXTRA_REDUCE_S(X(i)); \
} while(0)

#define BUTTERFLY(i,j,n) \
do { \
	int u= X(i); \
	int v= X(j); \
	X(i) = u+v; \
	X(j) = (u-v) << (2*n); \
} while(0)

	BUTTERFLY(0, 4, 0);
	BUTTERFLY(1, 5, 1);
	BUTTERFLY(2, 6, 2);
	BUTTERFLY(3, 7, 3);

	DO_REDUCE(6);
	DO_REDUCE(7);

	BUTTERFLY(0, 2, 0);
	BUTTERFLY(4, 6, 0);
	BUTTERFLY(1, 3, 2);
	BUTTERFLY(5, 7, 2);

	DO_REDUCE(7);

	BUTTERFLY(0, 1, 0);
	BUTTERFLY(2, 3, 0);
	BUTTERFLY(4, 5, 0);
	BUTTERFLY(6, 7, 0);

	DO_REDUCE_FULL_S(0);
	DO_REDUCE_FULL_S(1);
	DO_REDUCE_FULL_S(2);
	DO_REDUCE_FULL_S(3);
	DO_REDUCE_FULL_S(4);
	DO_REDUCE_FULL_S(5);
	DO_REDUCE_FULL_S(6);
	DO_REDUCE_FULL_S(7);

#undef X
#undef DO_REDUCE
#undef DO_REDUCE_FULL_S
#undef BUTTERFLY
}

#if defined(__CUDA_ARCH__)
#if __CUDA_ARCH__ < 300
  #define __shfl(var, srcLane, width) (uint32_t)(var)
  // #error __shfl() not supported by SM 2.x
#endif
#endif

/**
 * FFT_16 using w=2 as 16th root of unity
 * Unrolled decimation in frequency (DIF) radix-2 NTT.
 * Output data is in revbin_permuted order.
 */
__device__ __forceinline__
void FFT_16(int *y)
{
#define DO_REDUCE_FULL_S(i) \
	do { \
		y[i] = REDUCE(y[i]); \
		y[i] = EXTRA_REDUCE_S(y[i]); \
	} while(0)

	int u,v;

	// BUTTERFLY(0, 8, 0);
	// BUTTERFLY(1, 9, 1);
	// BUTTERFLY(2, 10, 2);
	// BUTTERFLY(3, 11, 3);
	// BUTTERFLY(4, 12, 4);
	// BUTTERFLY(5, 13, 5);
	// BUTTERFLY(6, 14, 6);
	// BUTTERFLY(7, 15, 7);
	{
		u = y[0]; // 0..7
		v = y[1]; // 8..15
		y[0] = u+v;
		y[1] = (u-v) << (threadIdx.x&7);
	}

	// DO_REDUCE(11);
	// DO_REDUCE(12);
	// DO_REDUCE(13);
	// DO_REDUCE(14);
	// DO_REDUCE(15);
	if ((threadIdx.x&7) >=3) y[1] = REDUCE(y[1]);  // 11...15

	// BUTTERFLY( 0, 4, 0);
	// BUTTERFLY( 1, 5, 2);
	// BUTTERFLY( 2, 6, 4);
	// BUTTERFLY( 3, 7, 6);
	{
		u = __shfl((int)y[0],  (threadIdx.x&3),8); // 0,1,2,3  0,1,2,3
		v = __shfl((int)y[0],4+(threadIdx.x&3),8); // 4,5,6,7  4,5,6,7
		y[0] = ((threadIdx.x&7) < 4) ? (u+v) : ((u-v) << (2*(threadIdx.x&3)));
	}

	// BUTTERFLY( 8, 12, 0);
	// BUTTERFLY( 9, 13, 2);
	// BUTTERFLY(10, 14, 4);
	// BUTTERFLY(11, 15, 6);
	{
		u = __shfl((int)y[1],  (threadIdx.x&3),8); // 8,9,10,11    8,9,10,11
		v = __shfl((int)y[1],4+(threadIdx.x&3),8); // 12,13,14,15  12,13,14,15
		y[1] = ((threadIdx.x&7) < 4) ? (u+v) : ((u-v) << (2*(threadIdx.x&3)));
	}

	// DO_REDUCE(5);
	// DO_REDUCE(7);
	// DO_REDUCE(13);
	// DO_REDUCE(15);
	if ((threadIdx.x&1) && (threadIdx.x&7) >= 4) {
		y[0] = REDUCE(y[0]);  // 5, 7
		y[1] = REDUCE(y[1]);  // 13, 15
	}

	// BUTTERFLY( 0, 2, 0);
	// BUTTERFLY( 1, 3, 4);
	// BUTTERFLY( 4, 6, 0);
	// BUTTERFLY( 5, 7, 4);
	{
		u = __shfl((int)y[0],  (threadIdx.x&5),8); // 0,1,0,1  4,5,4,5
		v = __shfl((int)y[0],2+(threadIdx.x&5),8); // 2,3,2,3  6,7,6,7
		y[0] = ((threadIdx.x&3) < 2) ? (u+v) : ((u-v) << (4*(threadIdx.x&1)));
	}

	// BUTTERFLY( 8, 10, 0);
	// BUTTERFLY( 9, 11, 4);
	// BUTTERFLY(12, 14, 0);
	// BUTTERFLY(13, 15, 4);
	{
		u = __shfl((int)y[1],  (threadIdx.x&5),8); // 8,9,8,9      12,13,12,13
		v = __shfl((int)y[1],2+(threadIdx.x&5),8); // 10,11,10,11  14,15,14,15
		y[1] = ((threadIdx.x&3) < 2) ? (u+v) : ((u-v) << (4*(threadIdx.x&1)));
	}

	// BUTTERFLY( 0, 1, 0);
	// BUTTERFLY( 2, 3, 0);
	// BUTTERFLY( 4, 5, 0);
	// BUTTERFLY( 6, 7, 0);
	{
		u = __shfl((int)y[0],  (threadIdx.x&6),8); // 0,0,2,2      4,4,6,6
		v = __shfl((int)y[0],1+(threadIdx.x&6),8); // 1,1,3,3      5,5,7,7
		y[0] = ((threadIdx.x&1) < 1) ? (u+v) : (u-v);
	}

	// BUTTERFLY( 8, 9, 0);
	// BUTTERFLY(10, 11, 0);
	// BUTTERFLY(12, 13, 0);
	// BUTTERFLY(14, 15, 0);
	{
		u = __shfl((int)y[1],  (threadIdx.x&6),8); // 8,8,10,10    12,12,14,14
		v = __shfl((int)y[1],1+(threadIdx.x&6),8); // 9,9,11,11    13,13,15,15
		y[1] = ((threadIdx.x&1) < 1) ? (u+v) : (u-v);
	}

	DO_REDUCE_FULL_S( 0); // 0...7
	DO_REDUCE_FULL_S( 1); // 8...15

#undef DO_REDUCE_FULL_S
}

__device__ __forceinline__
void FFT_128_full(int y[128])
{
	int i;

	FFT_8(y+0,2); // eight parallel FFT8's
	FFT_8(y+1,2); // eight parallel FFT8's

#pragma unroll 16
	for (i=0; i<16; i++)
	/*if (i & 7)*/ y[i] = REDUCE(y[i]*c_FFT128_8_16_Twiddle[i*8+(threadIdx.x&7)]);

#pragma unroll 8
	for (i=0; i<8; i++)
		FFT_16(y+2*i);  // eight sequential FFT16's, each one executed in parallel by 8 threads
}

__device__ __forceinline__
void FFT_256_halfzero(int y[256])
{
	/*
	 * FFT_256 using w=41 as 256th root of unity.
	 * Decimation in frequency (DIF) NTT.
	 * Output data is in revbin_permuted order.
	 * In place.
	 */
	const int tmp = y[15];

#pragma unroll 8
	for (int i=0; i<8; i++)
		y[16+i] = REDUCE(y[i] * c_FFT256_2_128_Twiddle[8*i+(threadIdx.x&7)]);
#pragma unroll 8
	for (int i=8; i<16; i++)
		y[16+i] = 0;

	/* handle X^255 with an additional butterfly */
	if ((threadIdx.x&7) == 7)
	{
		y[15] = REDUCE(tmp + 1);
		y[31] = REDUCE((tmp - 1) * c_FFT256_2_128_Twiddle[127]);
	}

	FFT_128_full(y);
	FFT_128_full(y+16);
}

/***************************************************/

__device__ __forceinline__
void Expansion(const uint32_t *data, uint4 *g_temp4)
{
	/* Message Expansion using Number Theoretical Transform similar to FFT */
	int expanded[32];
#pragma unroll 4
	for (int i=0; i < 4; i++) {
		expanded[  i] = __byte_perm(__shfl((int)data[0], 2*i, 8), __shfl((int)data[0], (2*i)+1, 8), threadIdx.x&7)&0xff;
		expanded[4+i] = __byte_perm(__shfl((int)data[1], 2*i, 8), __shfl((int)data[1], (2*i)+1, 8), threadIdx.x&7)&0xff;
	}
#pragma unroll 8
	for (int i=8; i < 16; i++)
		expanded[i] = 0;

	FFT_256_halfzero(expanded);

	// store w matrices in global memory

#define mul_185(x) ( (x)*185 )
#define mul_233(x) ( (x)*233 )

	uint4 vec0;
	int P, Q, P1, Q1, P2, Q2;
	bool even = (threadIdx.x & 1) == 0;

//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         2 2 2 2 2 2 2 2     2 2 2 2 2 2 2 2
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         6 6 6 6 6 6 6 6     6 6 6 6 6 6 6 6
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         0 0 0 0 0 0 0 0     0 0 0 0 0 0 0 0
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         4 4 4 4 4 4 4 4     4 4 4 4 4 4 4 4

	// 2 6 0 4

	P1 = expanded[ 0]; P2 = __shfl(expanded[ 2], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[16]; Q2 = __shfl(expanded[18], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[ 8]; P2 = __shfl(expanded[10], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[24]; Q2 = __shfl(expanded[26], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[ 4]; P2 = __shfl(expanded[ 6], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[20]; Q2 = __shfl(expanded[22], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	P1 = expanded[12]; P2 = __shfl(expanded[14], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[28]; Q2 = __shfl(expanded[30], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[0][threadIdx.x&7], 8);
	g_temp4[threadIdx.x&7] = vec0;

//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         6 6 6 6 6 6 6 6     6 6 6 6 6 6 6 6
//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         2 2 2 2 2 2 2 2     2 2 2 2 2 2 2 2
//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         4 4 4 4 4 4 4 4     4 4 4 4 4 4 4 4
//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         0 0 0 0 0 0 0 0     0 0 0 0 0 0 0 0

	// 6 2 4 0

	P1 = expanded[ 1]; P2 = __shfl(expanded[ 3], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[17]; Q2 = __shfl(expanded[19], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[ 9]; P2 = __shfl(expanded[11], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[25]; Q2 = __shfl(expanded[27], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[ 5]; P2 = __shfl(expanded[ 7], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[21]; Q2 = __shfl(expanded[23], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	P1 = expanded[13]; P2 = __shfl(expanded[15], (threadIdx.x-1)&7, 8); P = even ? P1 : P2;
	Q1 = expanded[29]; Q2 = __shfl(expanded[31], (threadIdx.x-1)&7, 8); Q = even ? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[1][threadIdx.x&7], 8);
	g_temp4[8+(threadIdx.x&7)] = vec0;

//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         7 7 7 7 7 7 7 7     7 7 7 7 7 7 7 7
//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         5 5 5 5 5 5 5 5     5 5 5 5 5 5 5 5
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         3 3 3 3 3 3 3 3     3 3 3 3 3 3 3 3
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         1 1 1 1 1 1 1 1     1 1 1 1 1 1 1 1

	// 7 5 3 1

	bool hi = (threadIdx.x&7)>=4;

	P1 = hi?expanded[ 1]:expanded[ 0]; P2 = __shfl(hi?expanded[ 3]:expanded[ 2], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[17]:expanded[16]; Q2 = __shfl(hi?expanded[19]:expanded[18], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[ 9]:expanded[ 8]; P2 = __shfl(hi?expanded[11]:expanded[10], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[25]:expanded[24]; Q2 = __shfl(hi?expanded[27]:expanded[26], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[ 5]:expanded[ 4]; P2 = __shfl(hi?expanded[ 7]:expanded[ 6], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[21]:expanded[20]; Q2 = __shfl(hi?expanded[23]:expanded[22], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	P1 = hi?expanded[13]:expanded[12]; P2 = __shfl(hi?expanded[15]:expanded[14], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = hi?expanded[29]:expanded[28]; Q2 = __shfl(hi?expanded[31]:expanded[30], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[2][threadIdx.x&7], 8);
	g_temp4[16+(threadIdx.x&7)] = vec0;

//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         1 1 1 1 1 1 1 1     1 1 1 1 1 1 1 1
//  1   9   5  13   3  11   7  15      17  25  21  29  19  27  23  31         3 3 3 3 3 3 3 3     3 3 3 3 3 3 3 3
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         5 5 5 5 5 5 5 5     5 5 5 5 5 5 5 5
//  0   8   4  12   2  10   6  14      16  24  20  28  18  26  22  30         7 7 7 7 7 7 7 7     7 7 7 7 7 7 7 7

  // 1 3 5 7

	bool lo = (threadIdx.x&7)<4;

	P1 = lo?expanded[ 1]:expanded[ 0]; P2 = __shfl(lo?expanded[ 3]:expanded[ 2], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[17]:expanded[16]; Q2 = __shfl(lo?expanded[19]:expanded[18], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[ 9]:expanded[ 8]; P2 = __shfl(lo?expanded[11]:expanded[10], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[25]:expanded[24]; Q2 = __shfl(lo?expanded[27]:expanded[26], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[ 5]:expanded[ 4]; P2 = __shfl(lo?expanded[ 7]:expanded[ 6], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[21]:expanded[20]; Q2 = __shfl(lo?expanded[23]:expanded[22], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	P1 = lo?expanded[13]:expanded[12]; P2 = __shfl(lo?expanded[15]:expanded[14], (threadIdx.x+1)&7, 8); P = !even ? P1 : P2;
	Q1 = lo?expanded[29]:expanded[28]; Q2 = __shfl(lo?expanded[31]:expanded[30], (threadIdx.x+1)&7, 8); Q = !even ? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_185(P),  mul_185(Q) , 0x5410), c_perm[3][threadIdx.x&7], 8);
	g_temp4[24+(threadIdx.x&7)] = vec0;

//  1   9   5  13   3  11   7  15       1   9   5  13   3  11   7  15         0 0 0 0 0 0 0 0     1 1 1 1 1 1 1 1
//  0   8   4  12   2  10   6  14       0   8   4  12   2  10   6  14         4 4 4 4 4 4 4 4     5 5 5 5 5 5 5 5
//  1   9   5  13   3  11   7  15       1   9   5  13   3  11   7  15         6 6 6 6 6 6 6 6     7 7 7 7 7 7 7 7
//  0   8   4  12   2  10   6  14       0   8   4  12   2  10   6  14         2 2 2 2 2 2 2 2     3 3 3 3 3 3 3 3

//{ 8, 72, 40, 104, 24, 88, 56, 120 },   { 9, 73, 41, 105, 25, 89, 57, 121 },
//{ 4, 68, 36, 100, 20, 84, 52, 116 },   { 5, 69, 37, 101, 21, 85, 53, 117 },
//{ 14, 78, 46, 110, 30, 94, 62, 126 },  { 15, 79, 47, 111, 31, 95, 63, 127 },
//{ 2, 66, 34, 98, 18, 82, 50, 114 },    { 3, 67, 35, 99, 19, 83, 51, 115 },

	bool sel = ((threadIdx.x+2)&7) >= 4;  // 2,3,4,5

	P1 = sel?expanded[0]:expanded[1]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[2]:expanded[3]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[8]:expanded[9]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[10]:expanded[11]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[4]:expanded[5]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[6]:expanded[7]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);
	P1 = sel?expanded[12]:expanded[13]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[14]:expanded[15]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[4][threadIdx.x&7], 8);

	g_temp4[32+(threadIdx.x&7)] = vec0;

//  0   8   4  12   2  10   6  14       0   8   4  12   2  10   6  14         6 6 6 6 6 6 6 6     7 7 7 7 7 7 7 7
//  1   9   5  13   3  11   7  15       1   9   5  13   3  11   7  15         2 2 2 2 2 2 2 2     3 3 3 3 3 3 3 3
//  0   8   4  12   2  10   6  14       0   8   4  12   2  10   6  14         0 0 0 0 0 0 0 0     1 1 1 1 1 1 1 1
//  1   9   5  13   3  11   7  15       1   9   5  13   3  11   7  15         4 4 4 4 4 4 4 4     5 5 5 5 5 5 5 5

	P1 = sel?expanded[1]:expanded[0]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[3]:expanded[2]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[9]:expanded[8]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[11]:expanded[10]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[5]:expanded[4]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[7]:expanded[6]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);
	P1 = sel?expanded[13]:expanded[12]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	Q2 = sel?expanded[15]:expanded[14]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[5][threadIdx.x&7], 8);

	g_temp4[40+(threadIdx.x&7)] = vec0;

// 16  24  20  28  18  26  22  30      16  24  20  28  18  26  22  30         6 6 6 6 6 6 6 6     7 7 7 7 7 7 7 7
// 16  24  20  28  18  26  22  30      16  24  20  28  18  26  22  30         0 0 0 0 0 0 0 0     1 1 1 1 1 1 1 1
// 17  25  21  29  19  27  23  31      17  25  21  29  19  27  23  31         0 0 0 0 0 0 0 0     1 1 1 1 1 1 1 1
// 17  25  21  29  19  27  23  31      17  25  21  29  19  27  23  31         6 6 6 6 6 6 6 6     7 7 7 7 7 7 7 7

	// sel markiert threads 2,3,4,5

	int t;
	t = __shfl(expanded[17],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[16]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[19],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[18]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = __shfl(expanded[25],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[24]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[27],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[26]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = __shfl(expanded[21],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[20]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[23],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[22]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);
	t = __shfl(expanded[29],(threadIdx.x+4)&7,8); P1 = sel?t:expanded[28]; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[31],(threadIdx.x+4)&7,8); Q2 = sel?t:expanded[30]; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[6][threadIdx.x&7], 8);

	g_temp4[48+(threadIdx.x&7)] = vec0;

// 17  25  21  29  19  27  23  31      17  25  21  29  19  27  23  31         4 4 4 4 4 4 4 4     5 5 5 5 5 5 5 5
// 17  25  21  29  19  27  23  31      17  25  21  29  19  27  23  31         2 2 2 2 2 2 2 2     3 3 3 3 3 3 3 3
// 16  24  20  28  18  26  22  30      16  24  20  28  18  26  22  30         2 2 2 2 2 2 2 2     3 3 3 3 3 3 3 3
// 16  24  20  28  18  26  22  30      16  24  20  28  18  26  22  30         4 4 4 4 4 4 4 4     5 5 5 5 5 5 5 5

	// sel markiert threads 2,3,4,5

	t = __shfl(expanded[16],(threadIdx.x+4)&7,8); P1 = sel?expanded[17]:t; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[18],(threadIdx.x+4)&7,8); Q2 = sel?expanded[19]:t; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.x = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = __shfl(expanded[24],(threadIdx.x+4)&7,8); P1 = sel?expanded[25]:t; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[26],(threadIdx.x+4)&7,8); Q2 = sel?expanded[27]:t; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.y = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = __shfl(expanded[20],(threadIdx.x+4)&7,8); P1 = sel?expanded[21]:t; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[22],(threadIdx.x+4)&7,8); Q2 = sel?expanded[23]:t; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.z = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);
	t = __shfl(expanded[28],(threadIdx.x+4)&7,8); P1 = sel?expanded[29]:t; Q1 = __shfl(P1, threadIdx.x^1, 8);
	t = __shfl(expanded[30],(threadIdx.x+4)&7,8); Q2 = sel?expanded[31]:t; P2 = __shfl(Q2, threadIdx.x^1, 8);
	P = even? P1 : P2; Q = even? Q1 : Q2;
	vec0.w = __shfl((int)__byte_perm(mul_233(P),  mul_233(Q) , 0x5410), c_perm[7][threadIdx.x&7], 8);

	g_temp4[56+(threadIdx.x&7)] = vec0;

#undef mul_185
#undef mul_233
}

/***************************************************/

__global__ __launch_bounds__(TPB, 4)
void x11_simd512_gpu_expand_64(uint32_t threads, uint32_t *g_hash, uint4 *g_temp4)
{
	int threadBloc = (blockDim.x * blockIdx.x + threadIdx.x) / 8;
	if (threadBloc < threads)
	{
		int hashPosition = threadBloc * 16;
		uint32_t *inpHash = &g_hash[hashPosition];

		// Read hash per 8 threads
		uint32_t Hash[2];
		int ndx = threadIdx.x & 7;
		Hash[0] = inpHash[ndx];
		Hash[1] = inpHash[ndx + 8];

		// Puffer für expandierte Nachricht
		uint4 *temp4 = &g_temp4[hashPosition * 4];

		Expansion(Hash, temp4);
	}
}

__global__ __launch_bounds__(TPB, 1)
void x11_simd512_gpu_compress1_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Compression1(Hash, thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 1)
void x11_simd512_gpu_compress2_64(uint32_t threads, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		Compression2(thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 2)
void x11_simd512_gpu_compress_64_maxwell(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Compression1(Hash, thread, g_fft4, g_state);
		Compression2(thread, g_fft4, g_state);
	}
}

__global__ __launch_bounds__(TPB, 2)
void x11_simd512_gpu_final_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *Hash = &g_hash[thread * 16];
		Final(Hash, thread, g_fft4, g_state);
	}
}

#else
__global__ void x11_simd512_gpu_expand_64(uint32_t threads, uint32_t *g_hash, uint4 *g_temp4) {}
__global__ void x11_simd512_gpu_compress1_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state) {}
__global__ void x11_simd512_gpu_compress2_64(uint32_t threads, uint4 *g_fft4, uint32_t *g_state) {}
__global__ void x11_simd512_gpu_compress_64_maxwell(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state) {}
__global__ void x11_simd512_gpu_final_64(uint32_t threads, uint32_t *g_hash, uint4 *g_fft4, uint32_t *g_state) {}
#endif /* SM3+ */

__host__
int x11_simd512_cpu_init(int thr_id, uint32_t threads)
{
	int dev_id = device_map[thr_id];
	cuda_get_arch(thr_id);
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
		x11_simd512_cpu_init_sm2(thr_id);
		return 0;
	}

	CUDA_CALL_OR_RET_X(hipMalloc(&d_temp4[thr_id], 64*sizeof(uint4)*threads), (int) err); /* todo: prevent -i 21 */
	CUDA_CALL_OR_RET_X(hipMalloc(&d_state[thr_id], 32*sizeof(int)*threads), (int) err);

#ifndef DEVICE_DIRECT_CONSTANTS
	hipMemcpyToSymbol(HIP_SYMBOL(c_perm), h_perm, sizeof(h_perm), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_IV_512), h_IV_512, sizeof(h_IV_512), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_FFT128_8_16_Twiddle), h_FFT128_8_16_Twiddle, sizeof(h_FFT128_8_16_Twiddle), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_FFT256_2_128_Twiddle), h_FFT256_2_128_Twiddle, sizeof(h_FFT256_2_128_Twiddle), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_cw0), h_cw0, sizeof(h_cw0), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cw1), h_cw1, sizeof(h_cw1), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cw2), h_cw2, sizeof(h_cw2), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cw3), h_cw3, sizeof(h_cw3), 0, hipMemcpyHostToDevice);
#endif

	// Texture for 128-Bit Zugriffe
	hipChannelFormatDesc channelDesc128 = hipCreateChannelDesc<uint4>();
	texRef1D_128.normalized = 0;
	texRef1D_128.filterMode = hipFilterModePoint;
	texRef1D_128.addressMode[0] = hipAddressModeClamp;

	CUDA_CALL_OR_RET_X(hipBindTexture(NULL, &texRef1D_128, d_temp4[thr_id], &channelDesc128, 64*sizeof(uint4)*threads), (int) err);

	return 0;
}

__host__
void x11_simd512_cpu_free(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300) {
		hipFree(d_temp4[thr_id]);
		hipFree(d_state[thr_id]);
	}
}

__host__
void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = TPB;
	int dev_id = device_map[thr_id];

	dim3 block(threadsperblock);
	dim3 grid((threads + threadsperblock-1) / threadsperblock);
	dim3 gridX8(grid.x * 8);

	if (d_nonceVector != NULL || device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
		x11_simd512_cpu_hash_64_sm2(thr_id, threads, startNounce, d_nonceVector, d_hash, order);
		return;
	}

	x11_simd512_gpu_expand_64 <<<gridX8, block>>> (threads, d_hash, d_temp4[thr_id]);

	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500) {
		x11_simd512_gpu_compress_64_maxwell <<< grid, block >>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);
	} else {
		x11_simd512_gpu_compress1_64 <<< grid, block >>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);
		x11_simd512_gpu_compress2_64 <<< grid, block >>> (threads, d_temp4[thr_id], d_state[thr_id]);
	}

	x11_simd512_gpu_final_64 <<<grid, block>>> (threads, d_hash, d_temp4[thr_id], d_state[thr_id]);

	//MyStreamSynchronize(NULL, order, thr_id);
}
