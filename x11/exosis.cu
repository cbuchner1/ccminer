#include "hip/hip_runtime.h"
/**
 * Timetravel (exosis) CUDA implementation
 *  by tpruvot@github, exosis
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>

#define HASH_FUNC_BASE_TIMESTAMP 1538556426U
#define HASH_FUNC_COUNT 8
#define HASH_FUNC_COUNT_PERMUTATIONS 40320U

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS];

enum Algo {
	BLAKE = 0,
	BMW,
	GROESTL,
	SKEIN,
	JH,
	KECCAK,
	LUFFA,
	CUBEHASH,
	MAX_ALGOS_COUNT
};

static const char* algo_strings[] = {
	"blake",
	"bmw512",
	"groestl",
	"skein",
	"jh512",
	"keccak",
	"luffa",
	"cube",
	NULL
};

inline void swap8(uint8_t *a, uint8_t *b)
{
	uint8_t t = *a;
	*a = *b;
	*b = t;
}

inline void initPerm(uint8_t n[], int count)
{
	for (int i = 0; i < count; i++)
		n[i] = i;
}

static int nextPerm(uint8_t n[], int count)
{
	int tail, i, j;

	if (count <= 1)
		return 0;

	for (i = count - 1; i>0 && n[i - 1] >= n[i]; i--);
	tail = i;

	if (tail > 0) {
		for (j = count - 1; j>tail && n[j] <= n[tail - 1]; j--);
		swap8(&n[tail - 1], &n[j]);
	}

	for (i = tail, j = count - 1; i<j; i++, j--)
		swap8(&n[i], &n[j]);

	return (tail != 0);
}

static void getAlgoString(char *str, int seq)
{
	uint8_t algoList[HASH_FUNC_COUNT];
	char *sptr;

	initPerm(algoList, HASH_FUNC_COUNT);

	for (int k = 0; k < seq; k++) {
		nextPerm(algoList, HASH_FUNC_COUNT);
	}

	sptr = str;
	for (int j = 0; j < HASH_FUNC_COUNT; j++) {
		if (algoList[j] >= 10)
			sprintf(sptr, "%c", 'A' + (algoList[j] - 10));
		else
			sprintf(sptr, "%u", (uint32_t) algoList[j]);
		sptr++;
	}
	*sptr = '\0';
}

static __thread uint32_t s_ntime = 0;
static uint32_t s_sequence = UINT32_MAX;
static uint8_t s_firstalgo = 0xFF;
static char hashOrder[HASH_FUNC_COUNT + 1] = { 0 };

#define INITIAL_DATE HASH_FUNC_BASE_TIMESTAMP
static inline uint32_t getCurrentAlgoSeq(uint32_t ntime)
{
	// unlike x11evo, the permutation changes often (with ntime)
	return (uint32_t) (ntime - INITIAL_DATE) % HASH_FUNC_COUNT_PERMUTATIONS;
}

// To finish...
static void get_travel_order(uint32_t ntime, char *permstr)
{
	uint32_t seq = getCurrentAlgoSeq(ntime);
	if (s_sequence != seq) {
		getAlgoString(permstr, seq);
		s_sequence = seq;
	}
}

// CPU Hash
extern "C" void exosis_hash(void *output, const void *input)
{
	uint32_t _ALIGN(64) hash[64/4] = { 0 };

	sph_blake512_context     ctx_blake;
	sph_bmw512_context       ctx_bmw;
	sph_groestl512_context   ctx_groestl;
	sph_skein512_context     ctx_skein;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_luffa512_context     ctx_luffa1;
	sph_cubehash512_context  ctx_cubehash1;

	if (s_sequence == UINT32_MAX) {
		uint32_t *data = (uint32_t*) input;
		const uint32_t ntime = (opt_benchmark || !data[17]) ? (uint32_t) time(NULL) : data[17];
		get_travel_order(ntime, hashOrder);
	}

	void *in = (void*) input;
	int size = 80;

	const int hashes = (int) strlen(hashOrder);

	for (int i = 0; i < hashes; i++)
	{
		const char elem = hashOrder[i];
		uint8_t algo = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

		switch (algo) {
		case BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, in, size);
			sph_blake512_close(&ctx_blake, hash);
			break;
		case BMW:
			sph_bmw512_init(&ctx_bmw);
			sph_bmw512(&ctx_bmw, in, size);
			sph_bmw512_close(&ctx_bmw, hash);
			break;
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, in, size);
			sph_groestl512_close(&ctx_groestl, hash);
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, in, size);
			sph_skein512_close(&ctx_skein, hash);
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, in, size);
			sph_jh512_close(&ctx_jh, hash);
			break;
		case KECCAK:
			sph_keccak512_init(&ctx_keccak);
			sph_keccak512(&ctx_keccak, in, size);
			sph_keccak512_close(&ctx_keccak, hash);
			break;
		case LUFFA:
			sph_luffa512_init(&ctx_luffa1);
			sph_luffa512(&ctx_luffa1, in, size);
			sph_luffa512_close(&ctx_luffa1, hash);
			break;
		case CUBEHASH:
			sph_cubehash512_init(&ctx_cubehash1);
			sph_cubehash512(&ctx_cubehash1, in, size);
			sph_cubehash512_close(&ctx_cubehash1, hash);
			break;
		}

		in = (void*) hash;
		size = 64;
	}

	memcpy(output, hash, 32);
}

static uint32_t get_next_time(uint32_t ntime, char* curOrder)
{
	char nextOrder[HASH_FUNC_COUNT + 1] = { 0 };
	uint32_t secs = 15;
	do {
		uint32_t nseq = getCurrentAlgoSeq(ntime+secs);
		getAlgoString(nextOrder, nseq);
		secs += 15;
	} while (curOrder[0] == nextOrder[0]);
	return secs;
}

//#define _DEBUG
#define _DEBUG_PREFIX "tt-"
#include "cuda_debug.cuh"

void quark_bmw512_cpu_setBlock_80(void *pdata);
void quark_bmw512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

void groestl512_setBlock_80(int thr_id, uint32_t *endiandata);
void groestl512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

void skein512_cpu_setBlock_80(void *pdata);
void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int swap);

void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
void qubit_luffa512_cpu_setBlock_80(void *pdata);
void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

void jh512_setBlock_80(int thr_id, uint32_t *endiandata);
void jh512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

void keccak512_setBlock_80(int thr_id, uint32_t *endiandata);
void keccak512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

void cubehash512_setBlock_80(int thr_id, uint32_t* endiandata);
void cubehash512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_exosis(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	// if (opt_benchmark) pdata[17] = swab32(0x5886a4be); // TO DEBUG GROESTL 80

	if (opt_debug || s_ntime != pdata[17] || s_sequence == UINT32_MAX) {
		uint32_t ntime = swab32(work->data[17]);
		get_travel_order(ntime, hashOrder);
		s_ntime = pdata[17];
		if (opt_debug && !thr_id) {
			applog(LOG_DEBUG, "exosis hash order %s (%08x)", hashOrder, ntime);
		}
	}

	if (opt_benchmark)
		ptarget[7] = 0x5;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		qubit_luffa512_cpu_init(thr_id, throughput); // only constants (480 bytes)
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMemset(d_hash[thr_id], 0, (size_t) 64 * throughput), -1);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	cuda_check_cpu_setTarget(ptarget);

	const int hashes = (int) strlen(hashOrder);
	const char first = hashOrder[0];
	const uint8_t algo80 = first >= 'A' ? first - 'A' + 10 : first - '0';
	if (algo80 != s_firstalgo) {
		s_firstalgo = algo80;
		applog(LOG_INFO, "Exosis first algo is now %s", algo_strings[algo80 % HASH_FUNC_COUNT]);
	}

	switch (algo80) {
		case BLAKE:
			quark_blake512_cpu_setBlock_80(thr_id, endiandata);
			break;
		case BMW:
			quark_bmw512_cpu_setBlock_80(endiandata);
			break;
		case GROESTL:
			groestl512_setBlock_80(thr_id, endiandata);
			break;
		case SKEIN:
			skein512_cpu_setBlock_80((void*)endiandata);
			break;
		case JH:
			jh512_setBlock_80(thr_id, endiandata);
			break;
		case KECCAK:
			keccak512_setBlock_80(thr_id, endiandata);
			break;
		case LUFFA:
			qubit_luffa512_cpu_setBlock_80((void*)endiandata);
			break;
		case CUBEHASH:
			cubehash512_setBlock_80(thr_id, endiandata);
			break;
		default: {
			uint32_t next = get_next_time(swab32(s_ntime), hashOrder);
			if (!thr_id)
				applog(LOG_WARNING, "kernel %c unimplemented, next in %u mn", first, next/60);
			sleep(next > 30 ? 60 : 10);
			return -1;
		}
	}

	do {
		int order = 0;

		// Hash with CUDA

		switch (algo80) {
			case BLAKE:
				quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("blake80:");
				break;
			case BMW:
				quark_bmw512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
				TRACE("bmw80  :");
				break;
			case GROESTL:
				groestl512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("grstl80:");
				break;
			case SKEIN:
				skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
				TRACE("skein80:");
				break;
			case JH:
				jh512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("jh51280:");
				break;
			case KECCAK:
				keccak512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("kecck80:");
				break;
			case LUFFA:
				qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
				TRACE("luffa80:");
				break;
			case CUBEHASH:
				cubehash512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("cube 80:");
				break;
		}

		for (int i = 1; i < hashes; i++)
		{
			const char elem = hashOrder[i];
			const uint8_t algo64 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

			switch (algo64) {
			case BLAKE:
				quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("blake  :");
				break;
			case BMW:
				quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("bmw    :");
				break;
			case GROESTL:
				quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("groestl:");
				break;
			case SKEIN:
				quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("skein  :");
				break;
			case JH:
				quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("jh512  :");
				break;
			case KECCAK:
				quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("keccak :");
				break;
			case LUFFA:
				x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("luffa  :");
				break;
			case CUBEHASH:
				x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("cube   :");
				break;
			}
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			be32enc(&endiandata[19], work->nonces[0]);
			exosis_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				pdata[19] = work->nonces[0];
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					exosis_hash(vhash, endiandata);
					if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						work->valid_nonces++;
					}
					pdata[19] = max(pdata[19], work->nonces[1]) + 1;
				}
				return work->valid_nonces;
			} else if (vhash[7] > Htarg) {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_exosis(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
