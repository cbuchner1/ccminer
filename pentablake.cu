#include "hip/hip_runtime.h"
/**
 * Penta Blake-512 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Aug. 2014
 */

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
#include <stdint.h>
#include <memory.h>
}

/* threads per block */
#define TPB 192

/* hash by cpu with blake 256 */
extern "C" void pentablakehash(void *output, const void *input)
{
	unsigned char hash[128];
	#define hashB hash + 64
	sph_blake512_context ctx;

	sph_blake512_init(&ctx);
	sph_blake512(&ctx, input, 80);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hashB);

	sph_blake512(&ctx, hashB, 64);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hashB);

	sph_blake512(&ctx, hashB, 64);
	sph_blake512_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

__constant__
static uint32_t __align__(32) c_Target[8];

__constant__
static uint64_t __align__(32) c_data[32];

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNounce[MAX_GPUS];
static uint32_t *h_resNounce[MAX_GPUS];
static uint32_t extra_results[2] = { UINT32_MAX, UINT32_MAX };

/* prefer uint32_t to prevent size conversions = speed +5/10 % */
__constant__
static uint32_t __align__(32) c_sigma[16][16];
const uint32_t host_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint64_t __align__(32) c_IV512[8] = {
	0x6a09e667f3bcc908ULL,
	0xbb67ae8584caa73bULL,
	0x3c6ef372fe94f82bULL,
	0xa54ff53a5f1d36f1ULL,
	0x510e527fade682d1ULL,
	0x9b05688c2b3e6c1fULL,
	0x1f83d9abfb41bd6bULL,
	0x5be0cd19137e2179ULL
};

__device__ __constant__
const uint64_t c_u512[16] =
{
	0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,
	0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
	0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,
	0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
	0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,
	0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
	0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,
	0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};

#define G(a,b,c,d,x) { \
	uint32_t idx1 = c_sigma[i][x]; \
	uint32_t idx2 = c_sigma[i][x+1]; \
	v[a] += (m[idx1] ^ c_u512[idx2]) + v[b]; \
	v[d] = SWAPDWORDS(v[d] ^ v[a]); \
	v[c] += v[d]; \
	v[b] = ROTR64(v[b] ^ v[c], 25); \
	v[a] += (m[idx2] ^ c_u512[idx1]) + v[b]; \
	v[d] = ROTR64(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR64(v[b] ^ v[c], 11); \
}

// Hash-Padding
__device__ __constant__
static const uint64_t d_constHashPadding[8] = {
	0x0000000000000080ull,
	0,
	0,
	0,
	0,
	0x0100000000000000ull,
	0,
	0x0002000000000000ull
};

__device__ static
void pentablake_compress(uint64_t *h, const uint64_t *block, const uint64_t T0)
{
	uint64_t v[16], m[16], i;

	#pragma unroll 16
	for(i = 0; i < 16; i++) {
		m[i] = cuda_swab64(block[i]);
	}

	#pragma unroll 8
	for (i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u512[0];
	v[ 9] = c_u512[1];
	v[10] = c_u512[2];
	v[11] = c_u512[3];
	v[12] = c_u512[4] ^ T0;
	v[13] = c_u512[5] ^ T0;
	v[14] = c_u512[6];
	v[15] = c_u512[7];

	//#pragma unroll 16
	for( i = 0; i < 16; i++)
	{
		/* column step */
		G(0, 4, 0x8, 0xC, 0x0);
		G(1, 5, 0x9, 0xD, 0x2);
		G(2, 6, 0xA, 0xE, 0x4);
		G(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		G(0, 5, 0xA, 0xF, 0x8);
		G(1, 6, 0xB, 0xC, 0xA);
		G(2, 7, 0x8, 0xD, 0xC);
		G(3, 4, 0x9, 0xE, 0xE);
	}

	//#pragma unroll 16
	for (i = 0; i < 16; i++) {
		uint32_t idx = i % 8;
		h[idx] ^= v[i];
	}
}

__global__
void pentablake_gpu_hash_80(uint32_t threads, const uint32_t startNounce, void *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t h[8];
		uint64_t buf[16];
		uint32_t nounce = startNounce + thread;

		//#pragma unroll 8
		for(int i=0; i<8; i++)
			h[i] = c_IV512[i];

		//#pragma unroll 16
		for (int i=0; i < 16; i++)
			buf[i] = c_data[i];

		// The test Nonce
		((uint32_t*)buf)[19] = cuda_swab32(nounce);

		pentablake_compress(h, buf, 640ULL);

#if __CUDA_ARCH__ < 300
		uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;
		#pragma unroll 8
		for (uint32_t i=0; i < 8; i++) {
			outHash[2*i]   = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = (uint64_t *)outputHash + 8 * thread;
		for (uint32_t i=0; i < 8; i++) {
			outHash[i] = cuda_swab64( h[i] );
		}
#endif

	}
}

__host__
void pentablake_cpu_hash_80(int thr_id, uint32_t threads, const uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = TPB;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	pentablake_gpu_hash_80 <<<grid, block, shared_size>>> (threads, startNounce, d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}


__global__
void pentablake_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint64_t *inpHash = &g_hash[thread<<3]; // hashPosition * 8
		uint64_t buf[16]; // 128 Bytes
		uint64_t h[8]; // State

		#pragma unroll 8
		for (int i=0; i<8; i++)
			h[i] = c_IV512[i];

		// Message for first round
		#pragma unroll 8
		for (int i=0; i < 8; ++i)
			buf[i] = inpHash[i];

		#pragma unroll 8
		for (int i=0; i < 8; i++)
			buf[i+8] = d_constHashPadding[i];

		// Ending round
		pentablake_compress(h, buf, 512);

#if __CUDA_ARCH__ < 300
		uint32_t *outHash = (uint32_t*)&g_hash[thread<<3];
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			outHash[2*i+0] = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = &g_hash[thread<<3];
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab64(h[i]);
		}
#endif
	}
}

__host__
void pentablake_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = TPB;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	pentablake_gpu_hash_64 <<<grid, block, shared_size>>> (threads, startNounce, (uint64_t*)d_outputHash);

	MyStreamSynchronize(NULL, order, thr_id);
}

__global__
void pentablake_gpu_check_hash(uint32_t threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;
		uint32_t *inpHash = &g_hash[thread<<4];
		uint32_t h[8];

		#pragma unroll 8
		for (int i=0; i < 8; i++)
			h[i] = inpHash[i];

		for (int i = 7; i >= 0; i--) {
			uint32_t hash = h[i]; // cuda_swab32(h[i]);
			if (hash > c_Target[i]) {
				return;
			}
			if (hash < c_Target[i]) {
				break;
			}
		}

		/* keep the smallest nounce, + extra one if found */
		if (resNounce[0] > nounce) {
			resNounce[1] = resNounce[0];
			resNounce[0] = nounce;
		}
		else
			resNounce[1] = nounce;
	}
}

__host__ static
uint32_t pentablake_check_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash, int order)
{
	const uint32_t threadsperblock = TPB;
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNounce[thr_id], 0xff, 2*sizeof(uint32_t)) != hipSuccess)
		return result;

	pentablake_gpu_check_hash <<<grid, block, shared_size>>> (threads, startNounce, d_inputHash, d_resNounce[thr_id]);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	if (hipSuccess == hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		hipDeviceSynchronize();
		result = h_resNounce[thr_id][0];
		extra_results[0] = h_resNounce[thr_id][1];
	}
	return result;
}


__host__
void pentablake_cpu_setBlock_80(uint32_t *pdata, const uint32_t *ptarget)
{
	uint8_t data[128];
	memcpy((void*) data, (void*) pdata, 80);
	memset(data+80, 0, 48);

	// to swab...
	data[80] = 0x80;
	data[111] = 1;
	data[126] = 0x02;
	data[127] = 0x80;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_sigma), host_sigma, sizeof(host_sigma), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Target), ptarget, 32, 0, hipMemcpyHostToDevice));
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_pentablake(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int rc = 0;
	uint32_t throughput =  device_intensity(thr_id, __func__, 128U * 2560); // 18.5
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x000F;

	if (!init[thr_id]) {
		if (active_gpus > 1) {
			hipSetDevice(device_map[thr_id]);
		}
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 64 * throughput));
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNounce[thr_id], 2*sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNounce[thr_id], 2*sizeof(uint32_t)));

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	pentablake_cpu_setBlock_80(endiandata, ptarget);

	do {
		int order = 0;

		// GPU HASH
		pentablake_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		pentablake_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		pentablake_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		pentablake_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		pentablake_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		uint32_t foundNonce = pentablake_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhashcpu[8];

			be32enc(&endiandata[19], foundNonce);
			pentablakehash(vhashcpu, endiandata);

			if (vhashcpu[7] <= Htarg && fulltest(vhashcpu, ptarget)) {
				rc = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (extra_results[0] != UINT32_MAX) {
					// Rare but possible if the throughput is big
					applog(LOG_NOTICE, "GPU found more than one result yippee!");
					pdata[21] = extra_results[0];
					extra_results[0] = UINT32_MAX;
					rc++;
				}
				pdata[19] = foundNonce;
				return rc;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for nounce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return rc;
}
