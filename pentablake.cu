#include "hip/hip_runtime.h"
/**
 * Penta Blake
 */

#include <stdint.h>
#include <memory.h>
#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

/* hash by cpu with blake 256 */
extern "C" void pentablakehash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[64];

	sph_blake512_context ctx;

	sph_blake512_init(&ctx);
	sph_blake512(&ctx, input, 80);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hash);

	sph_blake512(&ctx, hash, 64);
	sph_blake512_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_free(int thr_id);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_pentablake(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int rc = 0;
	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000F;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		quark_blake512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);
		CUDA_LOG_ERROR();

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// GPU HASH
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			pentablakehash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					pentablakehash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	return rc;
}

// cleanup
void free_pentablake(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();

	init[thr_id] = false;
}
