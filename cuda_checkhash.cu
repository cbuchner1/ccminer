#include "hip/hip_runtime.h"
/**
 * This code compares final hash against target
 */
#include <stdio.h>
#include <memory.h>

#include "miner.h"

#include "cuda_helper.h"

__constant__ uint32_t pTarget[8]; // 32 bytes

// store MAX_GPUS device arrays of 8 nonces
static uint32_t* h_resNonces[MAX_GPUS] = { NULL };
static uint32_t* d_resNonces[MAX_GPUS] = { NULL };
static __thread bool init_done = false;

__host__
void cuda_check_cpu_init(int thr_id, uint32_t threads)
{
    CUDA_CALL_OR_RET(hipMalloc(&d_resNonces[thr_id], 32));
    CUDA_SAFE_CALL(hipHostMalloc(&h_resNonces[thr_id], 32));
    init_done = true;
}

__host__
void cuda_check_cpu_free(int thr_id)
{
	if (!init_done) return;
	hipFree(d_resNonces[thr_id]);
	hipHostFree(h_resNonces[thr_id]);
	d_resNonces[thr_id] = NULL;
	h_resNonces[thr_id] = NULL;
	init_done = false;
}

// Target Difficulty
__host__
void cuda_check_cpu_setTarget(const void *ptarget)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 32, 0, hipMemcpyHostToDevice));
}

/* --------------------------------------------------------------------------------------------- */

__device__ __forceinline__
static bool hashbelowtarget(const uint32_t *const __restrict__ hash, const uint32_t *const __restrict__ target)
{
	if (hash[7] > target[7])
		return false;
	if (hash[7] < target[7])
		return true;
	if (hash[6] > target[6])
		return false;
	if (hash[6] < target[6])
		return true;

	if (hash[5] > target[5])
		return false;
	if (hash[5] < target[5])
		return true;
	if (hash[4] > target[4])
		return false;
	if (hash[4] < target[4])
		return true;

	if (hash[3] > target[3])
		return false;
	if (hash[3] < target[3])
		return true;
	if (hash[2] > target[2])
		return false;
	if (hash[2] < target[2])
		return true;

	if (hash[1] > target[1])
		return false;
	if (hash[1] < target[1])
		return true;
	if (hash[0] > target[0])
		return false;

	return true;
}

__global__ __launch_bounds__(512, 4)
void cuda_checkhash_64(uint32_t threads, uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// shl 4 = *16 x 4 (uint32) = 64 bytes
		// todo: use only 32 bytes * threads if possible
		uint32_t *inpHash = &hash[thread << 4];

		if (resNonces[0] == UINT32_MAX) {
			if (hashbelowtarget(inpHash, pTarget))
				resNonces[0] = (startNounce + thread);
		}
	}
}

__global__ __launch_bounds__(512, 4)
void cuda_checkhash_32(uint32_t threads, uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *inpHash = &hash[thread << 3];

		if (resNonces[0] == UINT32_MAX) {
			if (hashbelowtarget(inpHash, pTarget))
				resNonces[0] = (startNounce + thread);
		}
	}
}

__host__
uint32_t cuda_check_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash)
{
	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	if (bench_algo >= 0) // dont interrupt the global benchmark
		return UINT32_MAX;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return UINT32_MAX;
	}

	cuda_checkhash_64 <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	return h_resNonces[thr_id][0];
}

__host__
uint32_t cuda_check_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash)
{
	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	if (bench_algo >= 0) // dont interrupt the global benchmark
		return UINT32_MAX;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return UINT32_MAX;
	}

	cuda_checkhash_32 <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	return h_resNonces[thr_id][0];
}

/* --------------------------------------------------------------------------------------------- */

__global__ __launch_bounds__(512, 4)
void cuda_checkhash_64_suppl(uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint32_t *inpHash = &hash[thread << 4];

	if (hashbelowtarget(inpHash, pTarget)) {
		int resNum = ++resNonces[0];
		__threadfence();
		if (resNum < 8)
			resNonces[resNum] = (startNounce + thread);
	}
}

__host__
uint32_t cuda_check_hash_suppl(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash, uint8_t numNonce)
{
	uint32_t rescnt, result = 0;

	const uint32_t threadsperblock = 512;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return 0;
	}

	// first element stores the count of found nonces
	hipMemset(d_resNonces[thr_id], 0, sizeof(uint32_t));

	cuda_checkhash_64_suppl <<<grid, block>>> (startNounce, d_inputHash, d_resNonces[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], 32, hipMemcpyDeviceToHost);
	rescnt = h_resNonces[thr_id][0];
	if (rescnt > numNonce) {
		if (numNonce <= rescnt) {
			result = h_resNonces[thr_id][numNonce+1];
		}
		if (opt_debug)
			applog(LOG_WARNING, "Found %d nonces: %x + %x", rescnt, h_resNonces[thr_id][1], result);
	}

	return result;
}

/* --------------------------------------------------------------------------------------------- */

__global__
void cuda_check_hash_branch_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = g_nonceVector[thread];
		uint32_t hashPosition = (nounce - startNounce) << 4;
		uint32_t *inpHash = &g_hash[hashPosition];

		for (int i = 7; i >= 0; i--) {
			if (inpHash[i] > pTarget[i]) {
				return;
			}
			if (inpHash[i] < pTarget[i]) {
				break;
			}
		}
		if (resNounce[0] > nounce)
			resNounce[0] = nounce;
	}
}

__host__
uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order)
{
	const uint32_t threadsperblock = 256;

	uint32_t result = UINT32_MAX;

	if (bench_algo >= 0) // dont interrupt the global benchmark
		return result;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return result;
	}

	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_hash_branch_64 <<<grid, block>>> (threads, startNounce, d_nonceVector, d_inputHash, d_resNonces[thr_id]);

	MyStreamSynchronize(NULL, order, thr_id);

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	result = *h_resNonces[thr_id];

	return result;
}

/* Function to get the compiled Shader Model version */
int cuda_arch[MAX_GPUS] = { 0 };
__global__ void nvcc_get_arch(int *d_version)
{
	*d_version = 0;
#ifdef __CUDA_ARCH__
	*d_version = __CUDA_ARCH__;
#endif
}

__host__
int cuda_get_arch(int thr_id)
{
	int *d_version;
	int dev_id = device_map[thr_id];
	if (cuda_arch[dev_id] == 0) {
		// only do it once...
		hipMalloc(&d_version, sizeof(int));
		nvcc_get_arch <<< 1, 1 >>> (d_version);
		hipMemcpy(&cuda_arch[dev_id], d_version, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(d_version);
	}
	return cuda_arch[dev_id];
}
