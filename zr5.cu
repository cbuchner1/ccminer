#include "hip/hip_runtime.h"
/* Ziftrcoin ZR5 CUDA Implementation, (c) tpruvot 2015 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

#define ZR_BLAKE   0
#define ZR_GROESTL 1
#define ZR_JH512   2
#define ZR_SKEIN   3

#define POK_BOOL_MASK 0x00008000
#define POK_DATA_MASK 0xFFFF0000

static uint32_t* d_hash[MAX_GPUS];
static uint16_t* d_poks[MAX_GPUS];

static uint32_t**d_buffers[MAX_GPUS];
static uint8_t*  d_permut[MAX_GPUS];

static uint32_t* d_blake[MAX_GPUS];
static uint32_t* d_groes[MAX_GPUS];
static uint32_t* d_jh512[MAX_GPUS];
static uint32_t* d_skein[MAX_GPUS];

static uint8_t*  d_txs[MAX_GPUS];
__constant__ uint16_t c_txlens[POK_MAX_TXS];

__constant__ uint8_t c_permut[24][4];
static const uint8_t permut[24][4] = {
	{0, 1, 2, 3},
	{0, 1, 3, 2},
	{0, 2, 1, 3},
	{0, 2, 3, 1},
	{0, 3, 1, 2},
	{0, 3, 2, 1},
	{1, 0, 2, 3},
	{1, 0, 3, 2},
	{1, 2, 0, 3},
	{1, 2, 3, 0},
	{1, 3, 0, 2},
	{1, 3, 2, 0},
	{2, 0, 1, 3},
	{2, 0, 3, 1},
	{2, 1, 0, 3},
	{2, 1, 3, 0},
	{2, 3, 0, 1},
	{2, 3, 1, 0},
	{3, 0, 1, 2},
	{3, 0, 2, 1},
	{3, 1, 0, 2},
	{3, 1, 2, 0},
	{3, 2, 0, 1},
	{3, 2, 1, 0}
};

// CPU HASH
extern "C" void zr5hash(void *output, const void *input)
{
	sph_keccak512_context ctx_keccak;
	sph_blake512_context ctx_blake;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_skein512_context ctx_skein;

	uchar _ALIGN(64) hash[64];
	uint32_t *phash = (uint32_t *) hash;
	uint32_t norder;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*) input, 80);
	sph_keccak512_close(&ctx_keccak, (void*) phash);

	norder = phash[0] % ARRAY_SIZE(permut); /* % 24 */

	for(int i = 0; i < 4; i++)
	{
		switch (permut[norder][i]) {
		case ZR_BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, (const void*) phash, 64);
			sph_blake512_close(&ctx_blake, phash);
			break;
		case ZR_GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, (const void*) phash, 64);
			sph_groestl512_close(&ctx_groestl, phash);
			break;
		case ZR_JH512:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, (const void*) phash, 64);
			sph_jh512_close(&ctx_jh, phash);
			break;
		case ZR_SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, (const void*) phash, 64);
			sph_skein512_close(&ctx_skein, phash);
			break;
		default:
			break;
		}
	}
	memcpy(output, phash, 32);
}

extern "C" void zr5hash_pok(void *output, uint32_t *pdata)
{
	uint32_t _ALIGN(64) hash[8];
	const uint32_t version = (pdata[0] & (~POK_DATA_MASK)) | (use_pok ? POK_BOOL_MASK : 0);

	pdata[0] = version;
	zr5hash(hash, pdata);

	// fill PoK
	pdata[0] = version | (hash[0] & POK_DATA_MASK);
	zr5hash(hash, pdata);

	memcpy(output, hash, 32);
}

// ------------------------------------------------------------------------------------------------

__global__ __launch_bounds__(128, 8)
void zr5_init_vars_gpu(uint32_t threads, uint32_t* d_hash, uint8_t* d_permut, uint32_t** d_buffers,
        uint32_t* d_blake, uint32_t* d_groes, uint32_t* d_jh512, uint32_t* d_skein)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t offset = thread * 16U; // 64U / sizeof(uint32_t);
		uint32_t *phash = &d_hash[offset];

		// store the algos order for other procs
		const uint8_t norder = (phash[0] % ARRAY_SIZE(permut));
		const uint8_t algo = c_permut[norder][0];
		d_permut[thread] = norder;

		// init array for other procs
		d_buffers[0] = d_blake;
		d_buffers[1] = d_groes;
		d_buffers[2] = d_jh512;
		d_buffers[3] = d_skein;

		// Copy From d_hash to the first algo buffer
		// uint4 = 4x uint32_t = 16 bytes
		uint4 *psrc = (uint4*) phash;
		uint4 *pdst = (uint4*) (d_buffers[algo] + offset);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__host__
void zr5_init_vars(int thr_id, uint32_t threads)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	zr5_init_vars_gpu <<<grid, block>>> (
		threads, d_hash[thr_id], d_permut[thr_id], d_buffers[thr_id],
		d_blake[thr_id], d_groes[thr_id], d_jh512[thr_id], d_skein[thr_id]
	);
}


__global__ __launch_bounds__(128, 8)
void zr5_move_data_to_hash_gpu(const uint32_t threads, const int rnd, uint32_t** const d_buffers, uint8_t *d_permut, uint32_t *d_hash)
{
	// copy 64 bytes hash from/to the right algo buffers
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint8_t norder = d_permut[thread];
		const uint8_t algodst = c_permut[norder][rnd];
		const uint8_t algosrc = c_permut[norder][rnd-1];

		const uint32_t offset = thread * (64 / 4);

		// uint4 = 4x uint32_t = 16 bytes
		uint4 *psrc = (uint4*) (d_buffers[algosrc] + offset);
		uint4 *pdst = (uint4*) (d_buffers[algodst] + offset);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__host__
void zr5_move_data_to_hash(int thr_id, uint32_t threads, int rnd)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	zr5_move_data_to_hash_gpu <<<grid, block>>> (threads, rnd, d_buffers[thr_id], d_permut[thr_id], d_hash[thr_id]);
}


__global__ __launch_bounds__(128, 8)
void zr5_get_poks_gpu(uint32_t threads, uint32_t** const d_buffers, uint8_t* const d_permut, uint16_t *d_poks)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint8_t norder = d_permut[thread];
		const uint8_t algosrc = c_permut[norder][3];

		// copy only pok
		const uint32_t offset = thread * 16U; // 64 / 4;
		uint16_t* hash0 = (uint16_t*) (d_buffers[algosrc] + offset);
		d_poks[thread] = hash0[1];
	}
}

__global__ __launch_bounds__(128, 4)
void zr5_get_poks_xor_gpu(uint32_t threads, uint32_t** const d_buffers, uint8_t* d_permut, uint16_t* d_poks, uint8_t* d_txs, uint8_t txs)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint8_t norder = d_permut[thread];
		const uint8_t algo = c_permut[norder][3];
		const uint8_t ntx = norder % txs; // generally 0 on testnet...
		const uint32_t offset = thread * 16U; // 64 / 4;
		uint32_t* hash = (uint32_t*) (d_buffers[algo] + offset);
		uint32_t randNdx = hash[1] % c_txlens[ntx];
		uint8_t* ptx = &d_txs[POK_MAX_TX_SZ*ntx] + randNdx;
		uint32_t x = 0x100UL * ptx[3] + ptx[2];

		d_poks[thread] = x ^ (hash[2] >> 16);
	}
}

__host__
void zr5_get_poks(int thr_id, uint32_t threads, uint16_t* d_poks, struct work* work)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	uint8_t txs = (uint8_t) work->tx_count;

	if (txs && use_pok)
	{
		uint32_t txlens[POK_MAX_TXS];
		uint8_t* txdata = (uint8_t*) calloc(POK_MAX_TXS, POK_MAX_TX_SZ);
		if (!txdata) {
			applog(LOG_ERR, "%s: error, memory alloc failure", __func__);
			return;
		}
		// create blocs to copy on device
		for (uint8_t tx=0; tx < txs; tx++) {
			txlens[tx] = (uint32_t) (work->txs[tx].len - 3U);
			memcpy(&txdata[POK_MAX_TX_SZ*tx], work->txs[tx].data, min(POK_MAX_TX_SZ, txlens[tx]+3U));
		}
		hipMemcpy(d_txs[thr_id], txdata, txs * POK_MAX_TX_SZ, hipMemcpyHostToDevice);
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_txlens), txlens, txs * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
		zr5_get_poks_xor_gpu <<<grid, block>>> (threads, d_buffers[thr_id], d_permut[thr_id], d_poks, d_txs[thr_id], txs);
		free(txdata);
	} else {
		zr5_get_poks_gpu <<<grid, block>>> (threads, d_buffers[thr_id], d_permut[thr_id], d_poks);
	}
}


__global__ __launch_bounds__(128, 8)
void zr5_final_round_data_gpu(uint32_t threads, uint32_t** const d_buffers, uint8_t* const d_permut, uint32_t *d_hash, uint16_t *d_poks)
{
	// after the 4 algos rounds, copy back hash to d_hash
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint8_t norder = d_permut[thread];
		const uint8_t algosrc = c_permut[norder][3];
		const uint32_t offset = thread * 16U; // 64 / 4;

		// copy only hash[4..7]
		uint2 *psrc = (uint2*) (d_buffers[algosrc] + offset);
		uint2 *phash = (uint2*) (&d_hash[offset]);

		phash[2] = psrc[2];
		phash[3] = psrc[3];
	}
}

__host__
void zr5_final_round(int thr_id, uint32_t threads)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	zr5_final_round_data_gpu <<<grid, block>>> (threads, d_buffers[thr_id], d_permut[thr_id], d_hash[thr_id], d_poks[thr_id]);
}


extern void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);

extern void zr5_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void zr5_keccak512_cpu_hash_pok(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t* pdata, uint32_t *d_hash, uint16_t *d_poks);

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void quark_blake512_cpu_free(int thr_id);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void quark_groestl512_cpu_free(int thr_id);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_zr5(int thr_id, struct work *work,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) tmpdata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t oldp0 = pdata[0];
	const uint32_t version = (oldp0 & (~POK_DATA_MASK)) | (use_pok ? POK_BOOL_MASK : 0);
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 18);
	throughput = min(throughput, (1U << 20)-1024);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x0000ff;

	memcpy(tmpdata, pdata, 80);

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		// constants
		hipMemcpyToSymbol(HIP_SYMBOL(c_permut), permut, 24*4, 0, hipMemcpyHostToDevice);

		// hash buffer = keccak hash 64 required
		hipMalloc(&d_hash[thr_id], 64 * throughput);
		hipMalloc(&d_poks[thr_id], sizeof(uint16_t) * throughput);
		hipMalloc(&d_permut[thr_id], sizeof(uint8_t) * throughput);
		hipMalloc(&d_buffers[thr_id], 4 * sizeof(uint32_t*));

		// data buffers for the 4 rounds
		hipMalloc(&d_blake[thr_id], 64 * throughput);
		hipMalloc(&d_groes[thr_id], 64 * throughput);
		hipMalloc(&d_jh512[thr_id], 64 * throughput);
		hipMalloc(&d_skein[thr_id], 64 * throughput);

		hipMalloc(&d_txs[thr_id], POK_MAX_TXS * POK_MAX_TX_SZ);

		jackpot_keccak512_cpu_init(thr_id, throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipDeviceSynchronize());

		init[thr_id] = true;
	}

	tmpdata[0] = version;
	jackpot_keccak512_cpu_setBlock((void*)tmpdata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Keccak512 Hash with CUDA
		zr5_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		zr5_init_vars(thr_id, throughput);

		for (int rnd=0; rnd<4; rnd++) {
			if (rnd > 0)
				zr5_move_data_to_hash(thr_id, throughput, rnd);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_blake[thr_id], order++);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_groes[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_jh512[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_skein[thr_id], order++);
		}

		// store on device d_poks all hash[0] prefixes
		zr5_get_poks(thr_id, throughput, d_poks[thr_id], work);

		// Keccak512 with pok
		zr5_keccak512_cpu_hash_pok(thr_id, throughput, pdata[19], pdata, d_hash[thr_id], d_poks[thr_id]);
		zr5_init_vars(thr_id, throughput);

		for (int rnd=0; rnd<4; rnd++) {
			if (rnd > 0)
				zr5_move_data_to_hash(thr_id, throughput, rnd);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_blake[thr_id], order++);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_groes[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_jh512[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_skein[thr_id], order++);
		}
		zr5_final_round(thr_id, throughput);

		// do not scan results on interuption
		if (work_restart[thr_id].restart)
			return -1;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			uint32_t oldp19 = pdata[19];
			uint32_t offset = work->nonces[0] - pdata[19];
			uint32_t pok = 0;
			uint16_t h_pok;

			*hashes_done = pdata[19] - first_nonce + throughput;

			hipMemcpy(&h_pok, d_poks[thr_id] + offset, sizeof(uint16_t), hipMemcpyDeviceToHost);
			pok = version | (0x10000UL * h_pok);
			pdata[0] = pok; pdata[19] = work->nonces[0];
			zr5hash(vhash, pdata);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, oldp19, d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					offset = work->nonces[1] - oldp19;
					hipMemcpy(&h_pok, d_poks[thr_id] + offset, sizeof(uint16_t), hipMemcpyDeviceToHost);
					pok = version | (0x10000UL * h_pok);
					memcpy(tmpdata, pdata, 80);
					tmpdata[0] = pok; tmpdata[19] = work->nonces[1];
					zr5hash(vhash, tmpdata);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						bn_set_target_ratio(work, vhash, 1);
						pdata[19] = max(pdata[19], work->nonces[1]); // cursor
						pdata[20] = pok; // second nonce "pok"
						work->valid_nonces++;
					}
					pdata[19]++;
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[0] = oldp0;
			}
		} else
			pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	pdata[0] = oldp0;

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}

// cleanup
extern "C" void free_zr5(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	hipFree(d_poks[thr_id]);
	hipFree(d_permut[thr_id]);
	hipFree(d_buffers[thr_id]);

	hipFree(d_blake[thr_id]);
	hipFree(d_groes[thr_id]);
	hipFree(d_jh512[thr_id]);
	hipFree(d_skein[thr_id]);

	hipFree(d_txs[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
