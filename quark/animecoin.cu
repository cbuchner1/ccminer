#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}
#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

// Speicher zur Generierung der Noncevektoren f�r die bedingten Hashes
static uint32_t *d_branch1Nonces[MAX_GPUS];
static uint32_t *d_branch2Nonces[MAX_GPUS];
static uint32_t *d_branch3Nonces[MAX_GPUS];

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_setBlock_80(void *pdata);
extern void quark_bmw512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int order);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void quark_doublegroestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1,
											uint32_t *d_nonces2, uint32_t *nrm2,
											int order);
extern void quark_compactTest_single_false_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1,
											int order);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

/* CPU Hash */
extern "C" void animehash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_bmw512_context ctx_bmw;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    unsigned char hash[64];

    sph_bmw512_init(&ctx_bmw);
    // ZBMW;
    sph_bmw512 (&ctx_bmw, (const void*) input, 80);
    sph_bmw512_close(&ctx_bmw, (void*) hash);

    sph_blake512_init(&ctx_blake);
    // ZBLAKE;
    sph_blake512 (&ctx_blake, hash, 64);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    if (hash[0] & 0x8)
    {
        sph_groestl512_init(&ctx_groestl);
        // ZGROESTL;
        sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
        sph_groestl512_close(&ctx_groestl, (void*) hash);
    }
    else
    {
        sph_skein512_init(&ctx_skein);
        // ZSKEIN;
        sph_skein512 (&ctx_skein, (const void*) hash, 64);
        sph_skein512_close(&ctx_skein, (void*) hash);
    }
    
    sph_groestl512_init(&ctx_groestl);
    // ZGROESTL;
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    // ZJH;
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_blake512_init(&ctx_blake);
        // ZBLAKE;
        sph_blake512 (&ctx_blake, (const void*) hash, 64);
        sph_blake512_close(&ctx_blake, (void*) hash);
    }
    else
    {
        sph_bmw512_init(&ctx_bmw);
        // ZBMW;
        sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
        sph_bmw512_close(&ctx_bmw, (void*) hash);
    }

    sph_keccak512_init(&ctx_keccak);
    // ZKECCAK;
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    // SKEIN;
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_keccak512_init(&ctx_keccak);
        // ZKECCAK;
        sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
        sph_keccak512_close(&ctx_keccak, (void*) hash);
    }
    else
    {
        sph_jh512_init(&ctx_jh);
        // ZJH;
        sph_jh512 (&ctx_jh, (const void*) hash, 64);
        sph_jh512_close(&ctx_jh, (void*) hash);
    }

    memcpy(state, hash, 32);
}

/*
struct HashPredicate
{    
    HashPredicate(uint32_t *hashes, uint32_t startNonce) :
        m_hashes(hashes),
        m_startNonce(startNonce)
        { }

    __device__
    bool operator()(const uint32_t x)
    {
        uint32_t *hash = &m_hashes[(x - m_startNonce)*16];
        return hash[0] & 0x8;
    }

    uint32_t *m_hashes;
    uint32_t  m_startNonce;
};
*/

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_anime(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput =  device_intensity(thr_id, __func__, 1 << 19); // 256*256*8
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);
		quark_compactTest_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_bmw512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t nrm1=0, nrm2=0, nrm3=0;

		// erstes BMW512 Hash mit CUDA
		quark_bmw512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Blake512
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch3Nonces[thr_id], &nrm3,
				order++);
		
		// nur den Skein Branch weiterverfolgen
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Groestl512
		quark_groestl512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r JH512
		quark_jh512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// quarkNonces in branch1 und branch2 aufsplitten gem�ss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		// das ist der bedingte Branch f�r Blake512
		quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der bedingte Branch f�r Bmw512
		quark_bmw512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Keccak512
		quark_keccak512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der unbedingte Branch f�r Skein512
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// quarkNonces in branch1 und branch2 aufsplitten gem�ss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		// das ist der bedingte Branch f�r Keccak512
		quark_keccak512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);

		// das ist der bedingte Branch f�r JH512
		quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = cuda_check_hash_branch(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			animehash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for nonce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		if ((uint64_t)pdata[19] + throughput > (uint64_t)max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
