#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#define WANT_BMW512_80

#include "cuda_helper.h"

__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

#include "cuda_bmw512_sm3.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#undef SHL
#undef SHR
#undef CONST_EXP2

#define SHR(x, n) SHR2(x, n)
#define SHL(x, n) SHL2(x, n)
#define ROL(x, n) ROL2(x, n)

#define CONST_EXP2(i) \
	q[i+0] + ROL(q[i+1], 5)  + q[i+2] + ROL(q[i+3], 11) + \
	q[i+4] + ROL(q[i+5], 27) + q[i+6] + SWAPUINT2(q[i+7]) + \
	q[i+8] + ROL(q[i+9], 37) + q[i+10] + ROL(q[i+11], 43) + \
	q[i+12] + ROL(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

__device__
void Compression512_64_first(uint2 *msg, uint2 *hash)
{
	// Compression ref. implementation
	uint2 q[32];
	uint2 tmp;

	tmp = (msg[5] ^ hash[5]) - (msg[7] ^ hash[7]) + (hash[10]) + (hash[13]) + (hash[14]);
	q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp, 4) ^ ROL(tmp, 37)) + hash[1];

	tmp = (msg[6] ^ hash[6]) - (msg[8] ^ hash[8]) + (hash[11]) + (hash[14]) - (msg[15] ^ hash[15]);
	q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[2];
	tmp = (msg[0] ^ hash[0]) + (msg[7] ^ hash[7]) + (hash[9]) - (hash[12]) + (msg[15] ^ hash[15]);
	q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[3];
	tmp = (msg[0] ^ hash[0]) - (msg[1] ^ hash[1]) + (msg[8] ^ hash[8]) - (hash[10]) + (hash[13]);
	q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[4];
	tmp = (msg[1] ^ hash[1]) + (msg[2] ^ hash[2]) + (hash[9]) - (hash[11]) - (hash[14]);
	q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
	tmp = (msg[3] ^ hash[3]) - (msg[2] ^ hash[2]) + (hash[10]) - (hash[12]) + (msg[15] ^ hash[15]);
	q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp, 4) ^ ROL(tmp, 37)) + hash[6];
	tmp = (msg[4] ^ hash[4]) - (msg[0] ^ hash[0]) - (msg[3] ^ hash[3]) - (hash[11]) + (hash[13]);
	q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[7];
	tmp = (msg[1] ^ hash[1]) - (msg[4] ^ hash[4]) - (msg[5] ^ hash[5]) - (hash[12]) - (hash[14]);
	q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[8];

	tmp = (msg[2] ^ hash[2]) - (msg[5] ^ hash[5]) - (msg[6] ^ hash[6]) + (hash[13]) - (msg[15] ^ hash[15]);
	q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[9];
	tmp = (msg[0] ^ hash[0]) - (msg[3] ^ hash[3]) + (msg[6] ^ hash[6]) - (msg[7] ^ hash[7]) + (hash[14]);
	q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
	tmp = (msg[8] ^ hash[8]) - (msg[1] ^ hash[1]) - (msg[4] ^ hash[4]) - (msg[7] ^ hash[7]) + (msg[15] ^ hash[15]);
	q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp, 4) ^ ROL(tmp, 37)) + hash[11];
	tmp = (msg[8] ^ hash[8]) - (msg[0] ^ hash[0]) - (msg[2] ^ hash[2]) - (msg[5] ^ hash[5]) + (hash[9]);
	q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[12];
	tmp = (msg[1] ^ hash[1]) + (msg[3] ^ hash[3]) - (msg[6] ^ hash[6]) - (hash[9]) + (hash[10]);
	q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[13];
	tmp = (msg[2] ^ hash[2]) + (msg[4] ^ hash[4]) + (msg[7] ^ hash[7]) + (hash[10]) + (hash[11]);
	q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[14];
	tmp = (msg[3] ^ hash[3]) - (msg[5] ^ hash[5]) + (msg[8] ^ hash[8]) - (hash[11]) - (hash[12]);
	q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
	tmp = (msg[12] ^ hash[12]) - (msg[4] ^ hash[4]) - (msg[6] ^ hash[6]) - (hash[9]) + (hash[13]);
	q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp, 4) ^ ROL(tmp, 37)) + hash[0];

	q[0 + 16] =
		(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROL(q[0], 13) ^ ROL(q[0], 43)) +
		(SHR(q[0 + 1], 2) ^ SHL(q[0 + 1], 1) ^ ROL(q[0 + 1], 19) ^ ROL(q[0 + 1], 53)) +
		(SHR(q[0 + 2], 2) ^ SHL(q[0 + 2], 2) ^ ROL(q[0 + 2], 28) ^ ROL(q[0 + 2], 59)) +
		(SHR(q[0 + 3], 1) ^ SHL(q[0 + 3], 3) ^ ROL(q[0 + 3], 4) ^ ROL(q[0 + 3], 37)) +
		(SHR(q[0 + 4], 1) ^ SHL(q[0 + 4], 2) ^ ROL(q[0 + 4], 13) ^ ROL(q[0 + 4], 43)) +
		(SHR(q[0 + 5], 2) ^ SHL(q[0 + 5], 1) ^ ROL(q[0 + 5], 19) ^ ROL(q[0 + 5], 53)) +
		(SHR(q[0 + 6], 2) ^ SHL(q[0 + 6], 2) ^ ROL(q[0 + 6], 28) ^ ROL(q[0 + 6], 59)) +
		(SHR(q[0 + 7], 1) ^ SHL(q[0 + 7], 3) ^ ROL(q[0 + 7], 4) ^ ROL(q[0 + 7], 37)) +
		(SHR(q[0 + 8], 1) ^ SHL(q[0 + 8], 2) ^ ROL(q[0 + 8], 13) ^ ROL(q[0 + 8], 43)) +
		(SHR(q[0 + 9], 2) ^ SHL(q[0 + 9], 1) ^ ROL(q[0 + 9], 19) ^ ROL(q[0 + 9], 53)) +
		(SHR(q[0 + 10], 2) ^ SHL(q[0 + 10], 2) ^ ROL(q[0 + 10], 28) ^ ROL(q[0 + 10], 59)) +
		(SHR(q[0 + 11], 1) ^ SHL(q[0 + 11], 3) ^ ROL(q[0 + 11], 4) ^ ROL(q[0 + 11], 37)) +
		(SHR(q[0 + 12], 1) ^ SHL(q[0 + 12], 2) ^ ROL(q[0 + 12], 13) ^ ROL(q[0 + 12], 43)) +
		(SHR(q[0 + 13], 2) ^ SHL(q[0 + 13], 1) ^ ROL(q[0 + 13], 19) ^ ROL(q[0 + 13], 53)) +
		(SHR(q[0 + 14], 2) ^ SHL(q[0 + 14], 2) ^ ROL(q[0 + 14], 28) ^ ROL(q[0 + 14], 59)) +
		(SHR(q[0 + 15], 1) ^ SHL(q[0 + 15], 3) ^ ROL(q[0 + 15], 4) ^ ROL(q[0 + 15], 37)) +
		((make_uint2(0x55555550ul,0x55555555) + ROL(msg[0], 0 + 1) +
		ROL(msg[0 + 3], 0 + 4)) ^ hash[0 + 7]);

	q[1 + 16] =
		(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROL(q[1], 13) ^ ROL(q[1], 43)) +
		(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROL(q[1 + 1], 19) ^ ROL(q[1 + 1], 53)) +
		(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROL(q[1 + 2], 28) ^ ROL(q[1 + 2], 59)) +
		(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROL(q[1 + 3], 4) ^ ROL(q[1 + 3], 37)) +
		(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROL(q[1 + 4], 13) ^ ROL(q[1 + 4], 43)) +
		(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROL(q[1 + 5], 19) ^ ROL(q[1 + 5], 53)) +
		(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROL(q[1 + 6], 28) ^ ROL(q[1 + 6], 59)) +
		(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROL(q[1 + 7], 4) ^ ROL(q[1 + 7], 37)) +
		(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROL(q[1 + 8], 13) ^ ROL(q[1 + 8], 43)) +
		(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROL(q[1 + 9], 19) ^ ROL(q[1 + 9], 53)) +
		(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROL(q[1 + 10], 28) ^ ROL(q[1 + 10], 59)) +
		(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROL(q[1 + 11], 4) ^ ROL(q[1 + 11], 37)) +
		(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROL(q[1 + 12], 13) ^ ROL(q[1 + 12], 43)) +
		(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROL(q[1 + 13], 19) ^ ROL(q[1 + 13], 53)) +
		(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROL(q[1 + 14], 28) ^ ROL(q[1 + 14], 59)) +
		(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROL(q[1 + 15], 4) ^ ROL(q[1 + 15], 37)) +
		((make_uint2(0xAAAAAAA5, 0x5AAAAAAA) + ROL(msg[1], 1 + 1) +
		ROL(msg[1 + 3], 1 + 4)) ^ hash[1 + 7]);

	q[2 + 16] = CONST_EXP2(2) +
		((make_uint2(0xFFFFFFFA, 0x5FFFFFFF) + ROL(msg[2], 2 + 1) +
		ROL(msg[2 + 3], 2 + 4) - ROL(msg[2 + 10], 2 + 11)) ^ hash[2 + 7]);
	q[3 + 16] = CONST_EXP2(3) +
		((make_uint2(0x5555554F, 0x65555555) + ROL(msg[3], 3 + 1) +
		ROL(msg[3 + 3], 3 + 4) - ROL(msg[3 + 10], 3 + 11)) ^ hash[3 + 7]);
	q[4 + 16] = CONST_EXP2(4) +
		((make_uint2(0xAAAAAAA4, 0x6AAAAAAA) +ROL(msg[4], 4 + 1) +
		ROL(msg[4 + 3], 4 + 4) - ROL(msg[4 + 10], 4 + 11)) ^ hash[4 + 7]);
	q[5 + 16] = CONST_EXP2(5) +
		((make_uint2(0xFFFFFFF9, 0x6FFFFFFF) + ROL(msg[5], 5 + 1) +
		ROL(msg[5 + 3], 5 + 4) - ROL(msg[5 + 10], 5 + 11)) ^ hash[5 + 7]);

	#pragma unroll 3
	for (int i = 6; i<9; i++) {
		q[i + 16] = CONST_EXP2(i) +
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROL(msg[i], i + 1) -
			ROL(msg[i - 6], (i - 6) + 1)) ^ hash[i + 7]);
	}

	#pragma unroll 4
	for (int i = 9; i<13; i++) {
		q[i + 16] = CONST_EXP2(i) +
			((vectorize((i + 16)*(0x0555555555555555ull)) +
			ROL(msg[i + 3], i + 4) - ROL(msg[i - 6], (i - 6) + 1)) ^ hash[i - 9]);
	}

	q[13 + 16] = CONST_EXP2(13) +
		((make_uint2(0xAAAAAAA1, 0x9AAAAAAA) + ROL(msg[13], 13 + 1) +
		ROL(msg[13 - 13], (13 - 13) + 1) - ROL(msg[13 - 6], (13 - 6) + 1)) ^ hash[13 - 9]);
	q[14 + 16] = CONST_EXP2(14) +
		((make_uint2(0xFFFFFFF6, 0x9FFFFFFF) + ROL(msg[14], 14 + 1) +
		ROL(msg[14 - 13], (14 - 13) + 1) - ROL(msg[14 - 6], (14 - 6) + 1)) ^ hash[14 - 9]);
	q[15 + 16] = CONST_EXP2(15) +
		((make_uint2(0x5555554B, 0xA5555555) + ROL(msg[15], 15 + 1) +
		ROL(msg[15 - 13], (15 - 13) + 1) - ROL(msg[15 - 6], (15 - 6) + 1)) ^ hash[15 - 9]);


	uint2 XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
	uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

	hash[0] = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ msg[0]) + (XL64 ^ q[24] ^ q[0]);
	hash[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ msg[1]) + (XL64 ^ q[25] ^ q[1]);
	hash[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ msg[2]) + (XL64 ^ q[26] ^ q[2]);
	hash[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ msg[3]) + (XL64 ^ q[27] ^ q[3]);
	hash[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + (XL64 ^ q[28] ^ q[4]);
	hash[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ msg[5]) + (XL64 ^ q[29] ^ q[5]);
	hash[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ msg[6]) + (XL64 ^ q[30] ^ q[6]);
	hash[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ msg[7]) + (XL64 ^ q[31] ^ q[7]);

	hash[8] =  ROL(hash[4], 9)  + (XH64 ^ q[24] ^ msg[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);
	hash[9] =  ROL(hash[5], 10) + (XH64 ^ q[25]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
	hash[10] = ROL(hash[6], 11) + (XH64 ^ q[26]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
	hash[11] = ROL(hash[7], 12) + (XH64 ^ q[27]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
	hash[12] = ROL(hash[0], 13) + (XH64 ^ q[28]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
	hash[13] = ROL(hash[1], 14) + (XH64 ^ q[29]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
	hash[14] = ROL(hash[2], 15) + (XH64 ^ q[30]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
	hash[15] = ROL(hash[3], 16) + (XH64 ^ q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);
}

__device__
void Compression512(uint2 *msg, uint2 *hash)
{
	// Compression ref. implementation
	uint2 q[32];
	uint2 tmp;

	tmp = (msg[ 5] ^ hash[ 5]) - (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]) + (msg[14] ^ hash[14]);
	q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp,  4) ^ ROL(tmp, 37)) + hash[1];
	tmp = (msg[ 6] ^ hash[ 6]) - (msg[ 8] ^ hash[ 8]) + (msg[11] ^ hash[11]) + (msg[14] ^ hash[14]) - (msg[15] ^ hash[15]);
	q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[2];
	tmp = (msg[ 0] ^ hash[ 0]) + (msg[ 7] ^ hash[ 7]) + (msg[ 9] ^ hash[ 9]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
	q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[3];
	tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 1] ^ hash[ 1]) + (msg[ 8] ^ hash[ 8]) - (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]);
	q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[4];
	tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 2] ^ hash[ 2]) + (msg[ 9] ^ hash[ 9]) - (msg[11] ^ hash[11]) - (msg[14] ^ hash[14]);
	q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
	tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 2] ^ hash[ 2]) + (msg[10] ^ hash[10]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
	q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp,  4) ^ ROL(tmp, 37)) + hash[6];
	tmp = (msg[ 4] ^ hash[ 4]) - (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) - (msg[11] ^ hash[11]) + (msg[13] ^ hash[13]);
	q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[7];
	tmp = (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 5] ^ hash[ 5]) - (msg[12] ^ hash[12]) - (msg[14] ^ hash[14]);
	q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[8];
	tmp = (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) - (msg[ 6] ^ hash[ 6]) + (msg[13] ^ hash[13]) - (msg[15] ^ hash[15]);
	q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[9];
	tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) + (msg[ 6] ^ hash[ 6]) - (msg[ 7] ^ hash[ 7]) + (msg[14] ^ hash[14]);
	q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
	tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 7] ^ hash[ 7]) + (msg[15] ^ hash[15]);
	q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp,  4) ^ ROL(tmp, 37)) + hash[11];
	tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 0] ^ hash[ 0]) - (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) + (msg[ 9] ^ hash[ 9]);
	q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROL(tmp, 13) ^ ROL(tmp, 43)) + hash[12];
	tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 3] ^ hash[ 3]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[10] ^ hash[10]);
	q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROL(tmp, 19) ^ ROL(tmp, 53)) + hash[13];
	tmp = (msg[ 2] ^ hash[ 2]) + (msg[ 4] ^ hash[ 4]) + (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[11] ^ hash[11]);
	q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROL(tmp, 28) ^ ROL(tmp, 59)) + hash[14];
	tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 5] ^ hash[ 5]) + (msg[ 8] ^ hash[ 8]) - (msg[11] ^ hash[11]) - (msg[12] ^ hash[12]);
	q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
	tmp = (msg[12] ^ hash[12]) - (msg[ 4] ^ hash[ 4]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[13] ^ hash[13]);
	q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROL(tmp, 4) ^ ROL(tmp, 37)) + hash[0];

	q[0+16] =
		(SHR(q[0], 1) ^ SHL(q[0], 2) ^ ROL(q[0], 13) ^ ROL(q[0], 43)) +
		(SHR(q[0+1], 2) ^ SHL(q[0+1], 1) ^ ROL(q[0+1], 19) ^ ROL(q[0+1], 53)) +
		(SHR(q[0+2], 2) ^ SHL(q[0+2], 2) ^ ROL(q[0+2], 28) ^ ROL(q[0+2], 59)) +
		(SHR(q[0+3], 1) ^ SHL(q[0+3], 3) ^ ROL(q[0+3],  4) ^ ROL(q[0+3], 37)) +
		(SHR(q[0+4], 1) ^ SHL(q[0+4], 2) ^ ROL(q[0+4], 13) ^ ROL(q[0+4], 43)) +
		(SHR(q[0+5], 2) ^ SHL(q[0+5], 1) ^ ROL(q[0+5], 19) ^ ROL(q[0+5], 53)) +
		(SHR(q[0+6], 2) ^ SHL(q[0+6], 2) ^ ROL(q[0+6], 28) ^ ROL(q[0+6], 59)) +
		(SHR(q[0+7], 1) ^ SHL(q[0+7], 3) ^ ROL(q[0+7],  4) ^ ROL(q[0+7], 37)) +
		(SHR(q[0+8], 1) ^ SHL(q[0+8], 2) ^ ROL(q[0+8], 13) ^ ROL(q[0+8], 43)) +
		(SHR(q[0+9], 2) ^ SHL(q[0+9], 1) ^ ROL(q[0+9], 19) ^ ROL(q[0+9], 53)) +
		(SHR(q[0+10], 2) ^ SHL(q[0+10], 2) ^ ROL(q[0+10], 28) ^ ROL(q[0+10], 59)) +
		(SHR(q[0+11], 1) ^ SHL(q[0+11], 3) ^ ROL(q[0+11],  4) ^ ROL(q[0+11], 37)) +
		(SHR(q[0+12], 1) ^ SHL(q[0+12], 2) ^ ROL(q[0+12], 13) ^ ROL(q[0+12], 43)) +
		(SHR(q[0+13], 2) ^ SHL(q[0+13], 1) ^ ROL(q[0+13], 19) ^ ROL(q[0+13], 53)) +
		(SHR(q[0+14], 2) ^ SHL(q[0+14], 2) ^ ROL(q[0+14], 28) ^ ROL(q[0+14], 59)) +
		(SHR(q[0+15], 1) ^ SHL(q[0+15], 3) ^ ROL(q[0+15],  4) ^ ROL(q[0+15], 37)) +
		((make_uint2(0x55555550ul, 0x55555555) + ROL(msg[0], 0 + 1) +
		ROL(msg[0+3], 0+4) - ROL(msg[0+10], 0+11) ) ^ hash[0+7]);

	q[1 + 16] =
		(SHR(q[1], 1) ^ SHL(q[1], 2) ^ ROL(q[1], 13) ^ ROL(q[1], 43)) +
		(SHR(q[1 + 1], 2) ^ SHL(q[1 + 1], 1) ^ ROL(q[1 + 1], 19) ^ ROL(q[1 + 1], 53)) +
		(SHR(q[1 + 2], 2) ^ SHL(q[1 + 2], 2) ^ ROL(q[1 + 2], 28) ^ ROL(q[1 + 2], 59)) +
		(SHR(q[1 + 3], 1) ^ SHL(q[1 + 3], 3) ^ ROL(q[1 + 3], 4) ^ ROL(q[1 + 3], 37)) +
		(SHR(q[1 + 4], 1) ^ SHL(q[1 + 4], 2) ^ ROL(q[1 + 4], 13) ^ ROL(q[1 + 4], 43)) +
		(SHR(q[1 + 5], 2) ^ SHL(q[1 + 5], 1) ^ ROL(q[1 + 5], 19) ^ ROL(q[1 + 5], 53)) +
		(SHR(q[1 + 6], 2) ^ SHL(q[1 + 6], 2) ^ ROL(q[1 + 6], 28) ^ ROL(q[1 + 6], 59)) +
		(SHR(q[1 + 7], 1) ^ SHL(q[1 + 7], 3) ^ ROL(q[1 + 7], 4) ^ ROL(q[1 + 7], 37)) +
		(SHR(q[1 + 8], 1) ^ SHL(q[1 + 8], 2) ^ ROL(q[1 + 8], 13) ^ ROL(q[1 + 8], 43)) +
		(SHR(q[1 + 9], 2) ^ SHL(q[1 + 9], 1) ^ ROL(q[1 + 9], 19) ^ ROL(q[1 + 9], 53)) +
		(SHR(q[1 + 10], 2) ^ SHL(q[1 + 10], 2) ^ ROL(q[1 + 10], 28) ^ ROL(q[1 + 10], 59)) +
		(SHR(q[1 + 11], 1) ^ SHL(q[1 + 11], 3) ^ ROL(q[1 + 11], 4) ^ ROL(q[1 + 11], 37)) +
		(SHR(q[1 + 12], 1) ^ SHL(q[1 + 12], 2) ^ ROL(q[1 + 12], 13) ^ ROL(q[1 + 12], 43)) +
		(SHR(q[1 + 13], 2) ^ SHL(q[1 + 13], 1) ^ ROL(q[1 + 13], 19) ^ ROL(q[1 + 13], 53)) +
		(SHR(q[1 + 14], 2) ^ SHL(q[1 + 14], 2) ^ ROL(q[1 + 14], 28) ^ ROL(q[1 + 14], 59)) +
		(SHR(q[1 + 15], 1) ^ SHL(q[1 + 15], 3) ^ ROL(q[1 + 15], 4) ^ ROL(q[1 + 15], 37)) +
		((make_uint2(0xAAAAAAA5, 0x5AAAAAAA) + ROL(msg[1], 1 + 1) +
		ROL(msg[1 + 3], 1 + 4) - ROL(msg[1 + 10], 1 + 11)) ^ hash[1 + 7]);

	q[2 + 16] = CONST_EXP2(2) +
		((make_uint2(0xFFFFFFFA, 0x5FFFFFFF) + ROL(msg[2], 2 + 1) +
		ROL(msg[2+3], 2+4) - ROL(msg[2+10], 2+11) ) ^ hash[2+7]);
	q[3 + 16] = CONST_EXP2(3) +
		((make_uint2(0x5555554F, 0x65555555) + ROL(msg[3], 3 + 1) +
		ROL(msg[3 + 3], 3 + 4) - ROL(msg[3 + 10], 3 + 11)) ^ hash[3 + 7]);
	q[4 + 16] = CONST_EXP2(4) +
		((make_uint2(0xAAAAAAA4, 0x6AAAAAAA) + ROL(msg[4], 4 + 1) +
		ROL(msg[4 + 3], 4 + 4) - ROL(msg[4 + 10], 4 + 11)) ^ hash[4 + 7]);
	q[5 + 16] = CONST_EXP2(5) +
		((make_uint2(0xFFFFFFF9, 0x6FFFFFFF) + ROL(msg[5], 5 + 1) +
		ROL(msg[5 + 3], 5 + 4) - ROL(msg[5 + 10], 5 + 11)) ^ hash[5 + 7]);
	q[6 + 16] = CONST_EXP2(6) +
		((make_uint2(0x5555554E, 0x75555555)+ ROL(msg[6], 6 + 1) +
		ROL(msg[6 + 3], 6 + 4) - ROL(msg[6 - 6], (6 - 6) + 1)) ^ hash[6 + 7]);
	q[7 + 16] = CONST_EXP2(7) +
		((make_uint2(0xAAAAAAA3, 0x7AAAAAAA) + ROL(msg[7], 7 + 1) +
		ROL(msg[7 + 3], 7 + 4) - ROL(msg[7 - 6], (7 - 6) + 1)) ^ hash[7 + 7]);
	q[8 + 16] = CONST_EXP2(8) +
		((make_uint2(0xFFFFFFF8, 0x7FFFFFFF) + ROL(msg[8], 8 + 1) +
		ROL(msg[8 + 3], 8 + 4) - ROL(msg[8 - 6], (8 - 6) + 1)) ^ hash[8 + 7]);
	q[9 + 16] = CONST_EXP2(9) +
		((make_uint2(0x5555554D, 0x85555555) + ROL(msg[9], 9 + 1) +
		ROL(msg[9 + 3], 9 + 4) - ROL(msg[9 - 6], (9 - 6) + 1)) ^ hash[9 - 9]);
	q[10 + 16] = CONST_EXP2(10) +
		((make_uint2(0xAAAAAAA2, 0x8AAAAAAA) + ROL(msg[10], 10 + 1) +
		ROL(msg[10 + 3], 10 + 4) - ROL(msg[10 - 6], (10 - 6) + 1)) ^ hash[10 - 9]);
	q[11 + 16] = CONST_EXP2(11) +
		((make_uint2(0xFFFFFFF7, 0x8FFFFFFF) + ROL(msg[11], 11 + 1) +
		ROL(msg[11 + 3], 11 + 4) - ROL(msg[11 - 6], (11 - 6) + 1)) ^ hash[11 - 9]);
	q[12 + 16] = CONST_EXP2(12) +
		((make_uint2(0x5555554C, 0x95555555) + ROL(msg[12], 12 + 1) +
		ROL(msg[12 + 3], 12 + 4) - ROL(msg[12 - 6], (12 - 6) + 1)) ^ hash[12 - 9]);
	q[13 + 16] = CONST_EXP2(13) +
		((make_uint2(0xAAAAAAA1, 0x9AAAAAAA) + ROL(msg[13], 13 + 1) +
		ROL(msg[13 - 13], (13 - 13) + 1) - ROL(msg[13 - 6], (13 - 6) + 1)) ^ hash[13 - 9]);
	q[14 + 16] = CONST_EXP2(14) +
		((make_uint2(0xFFFFFFF6, 0x9FFFFFFF) + ROL(msg[14], 14 + 1) +
		ROL(msg[14 - 13], (14 - 13) + 1) - ROL(msg[14 - 6], (14 - 6) + 1)) ^ hash[14 - 9]);
	q[15 + 16] = CONST_EXP2(15) +
		((make_uint2(0x5555554B, 0xA5555555) + ROL(msg[15], 15 + 1) +
		ROL(msg[15 - 13], (15 - 13) + 1) - ROL(msg[15 - 6], (15 - 6) + 1)) ^ hash[15 - 9]);

	uint2 XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

	hash[0] = (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[ 0]) + (XL64 ^ q[24] ^ q[ 0]);
	hash[1] = (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[ 1]) + (XL64 ^ q[25] ^ q[ 1]);
	hash[2] = (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[ 2]) + (XL64 ^ q[26] ^ q[ 2]);
	hash[3] = (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[ 3]) + (XL64 ^ q[27] ^ q[ 3]);
	hash[4] = (SHR(XH64, 3) ^     q[20]    ^ msg[ 4]) + (XL64 ^ q[28] ^ q[ 4]);
	hash[5] = (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[ 5]) + (XL64 ^ q[29] ^ q[ 5]);
	hash[6] = (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[ 6]) + (XL64 ^ q[30] ^ q[ 6]);
	hash[7] = (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[ 7]) + (XL64 ^ q[31] ^ q[ 7]);

	hash[ 8] = ROL(hash[4], 9) + (XH64 ^ q[24] ^ msg[ 8]) + (SHL(XL64,8) ^ q[23] ^ q[ 8]);
	hash[ 9] = ROL(hash[5],10) + (XH64 ^ q[25] ^ msg[ 9]) + (SHR(XL64,6) ^ q[16] ^ q[ 9]);
	hash[10] = ROL(hash[6],11) + (XH64 ^ q[26] ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
	hash[11] = ROL(hash[7],12) + (XH64 ^ q[27] ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
	hash[12] = ROL(hash[0],13) + (XH64 ^ q[28] ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
	hash[13] = ROL(hash[1],14) + (XH64 ^ q[29] ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
	hash[14] = ROL(hash[2],15) + (XH64 ^ q[30] ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	hash[15] = ROL(hash[3],16) + (XH64 ^ q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);
}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(32, 16)
#else
__launch_bounds__(64, 8)
#endif
void quark_bmw512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		uint32_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition * 8];

		// Init
		uint2 h[16] = {
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};

		// Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
		// BMW arbeitet mit 128 Byte!!!
		uint2 message[16];
		#pragma unroll
		for(int i=0;i<8;i++)
			message[i] = vectorize(inpHash[i]);

		#pragma unroll 6
		for(int i=9;i<15;i++)
			message[i] = make_uint2(0,0);

		// Padding einf�gen (Byteorder?!?)
		message[8] = make_uint2(0x80,0);
		// L�nge (in Bits, d.h. 64 Byte * 8 = 512 Bits
		message[15] = make_uint2(512,0);

		// Compression 1
		Compression512_64_first(message, h);

		// Final
		#pragma unroll
		for(int i=0;i<16;i++)
		{
			message[i].y = 0xaaaaaaaa;
			message[i].x = 0xaaaaaaa0ul + (uint32_t)i;
		}
		Compression512(h, message);

		// fertig
		uint64_t *outpHash = &g_hash[hashPosition * 8];

		#pragma unroll
		for(int i=0;i<8;i++)
			outpHash[i] = devectorize(message[i+8]);
	}
}

__global__ __launch_bounds__(256, 2)
void quark_bmw512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;

		// Init
		uint2 h[16] = {
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};
		// Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
		// BMW arbeitet mit 128 Byte!!!
		uint2 message[16];
#pragma unroll 16
		for(int i=0;i<16;i++)
			message[i] = vectorize(c_PaddedMessage80[i]);

		// die Nounce durch die thread-spezifische ersetzen
		message[9].y = cuda_swab32(nounce);	//REPLACE_HIDWORD(message[9], cuda_swab32(nounce));

		// Compression 1
		Compression512(message, h);

#pragma unroll 16
		for(int i=0;i<16;i++)
			message[i] = make_uint2(0xaaaaaaa0+i,0xaaaaaaaa);


		Compression512(h, message);

		// fertig
		uint64_t *outpHash = &g_hash[thread * 8];

#pragma unroll 8
		for(int i=0;i<8;i++)
			outpHash[i] = devectorize(message[i+8]);
	}
}

__host__
void quark_bmw512_cpu_setBlock_80(void *pdata)
{
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	uint64_t *message = (uint64_t*)PaddedMessage;
	message[10] = SPH_C64(0x80);
	message[15] = SPH_C64(640);
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__
void quark_bmw512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] > 300 && cuda_arch[dev_id] > 300)
		quark_bmw512_gpu_hash_80<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
	else
		quark_bmw512_gpu_hash_80_30<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
}

__host__
void quark_bmw512_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
}

__host__
void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 32;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] > 300 && cuda_arch[dev_id] > 300)
		quark_bmw512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	else
		quark_bmw512_gpu_hash_64_30<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}
