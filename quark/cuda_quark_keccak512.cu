#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "cuda_helper.h"

#define U32TO64_LE(p) \
	(((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
	*p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t d_keccak_round_constants[24];

__device__ __forceinline__
static void keccak_block(uint2 *s)
{
	size_t i;
	uint2 t[5], u[5], v, w;

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROL2(t[1], 1);
		u[1] = t[0] ^ ROL2(t[2], 1);
		u[2] = t[1] ^ ROL2(t[3], 1);
		u[3] = t[2] ^ ROL2(t[4], 1);
		u[4] = t[3] ^ ROL2(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1]  = ROL2(s[6], 44);
		s[6]  = ROL2(s[9], 20);
		s[9]  = ROL2(s[22], 61);
		s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);
		s[20] = ROL2(s[2], 62);
		s[2]  = ROL2(s[12], 43);
		s[12] = ROL2(s[13], 25);
		s[13] = ROL2(s[19], 8);
		s[19] = ROL2(s[23], 56);
		s[23] = ROL2(s[15], 41);
		s[15] = ROL2(s[4], 27);
		s[4]  = ROL2(s[24], 14);
		s[24] = ROL2(s[21], 2);
		s[21] = ROL2(s[8], 55);
		s[8]  = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);
		s[5]  = ROL2(s[3], 28);
		s[3]  = ROL2(s[18], 21);
		s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);
		s[11] = ROL2(s[7], 6);
		s[7]  = ROL2(s[10], 3);
		s[10] = ROL2(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= vectorize(d_keccak_round_constants[i]);
	}
}

__global__
void quark_keccak512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		off_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition * 8];
		uint2 keccak_gpu_state[25];

		for (int i = 0; i<8; i++) {
			keccak_gpu_state[i] = vectorize(inpHash[i]);
		}
		keccak_gpu_state[8] = vectorize(0x8000000000000001ULL);

		for (int i=9; i<25; i++) {
			keccak_gpu_state[i] = make_uint2(0, 0);
		}
		keccak_block(keccak_gpu_state);

		for(int i=0; i<8; i++) {
			inpHash[i] = devectorize(keccak_gpu_state[i]);
		}
	}
}

__device__ __forceinline__
static void keccak_block_v30(uint64_t *s, const uint32_t *in)
{
	size_t i;
	uint64_t t[5], u[5], v, w;

	#pragma unroll 9
	for (i = 0; i < 72 / 8; i++, in += 2)
		s[i] ^= U32TO64_LE(in);

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= d_keccak_round_constants[i];
	}
}

__global__
void quark_keccak512_gpu_hash_64_v30(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		off_t hashPosition = nounce - startNounce;
		uint32_t *inpHash = (uint32_t*)&g_hash[hashPosition * 8];

		uint32_t message[18];
		#pragma unroll 16
		for(int i=0;i<16;i++)
			message[i] = inpHash[i];

		message[16] = 0x01;
		message[17] = 0x80000000;

		uint64_t keccak_gpu_state[25];
		#pragma unroll 25
		for (int i=0; i<25; i++)
			keccak_gpu_state[i] = 0;

		keccak_block_v30(keccak_gpu_state, message);

		uint32_t hash[16];
		#pragma unroll 8
		for (size_t i = 0; i < 64; i += 8) {
			U64TO32_LE((&hash[i/4]), keccak_gpu_state[i / 8]);
		}

		uint32_t *outpHash = (uint32_t*)&g_hash[hashPosition * 8];
		#pragma unroll 16
		for(int i=0; i<16; i++)
			outpHash[i] = hash[i];
	}
}

__host__
void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] >= 320)
		quark_keccak512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	else
		quark_keccak512_gpu_hash_64_v30<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	MyStreamSynchronize(NULL, order, thr_id);
}

void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
void jackpot_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

__host__
void quark_keccak512_cpu_init(int thr_id, uint32_t threads)
{
	// required for the 64 bytes one
	hipMemcpyToSymbol(HIP_SYMBOL(d_keccak_round_constants), host_keccak_round_constants,
			sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice);

	jackpot_keccak512_cpu_init(thr_id, threads);
}

__host__
void keccak512_setBlock_80(int thr_id, uint32_t *endiandata)
{
	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
}

__host__
void keccak512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	jackpot_keccak512_cpu_hash(thr_id, threads, startNounce, d_hash, 0);
}
