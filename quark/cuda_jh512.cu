#include "hip/hip_runtime.h"
/**
 * JH512 64 and 80 kernels
 *
 * JH80 by tpruvot - 2017 - under GPLv3
 **/
#include <cuda_helper.h>

// #include <stdio.h>  // printf
// #include <unistd.h> // sleep

/* 1344 bytes, align 16 is there to allow ld.const.v4 (made auto. by the compiler) */
__constant__ static __align__(16) uint32_t c_E8_bslice32[42][8] = {
	// Round 0 (Function0)
	{ 0xa2ded572, 0x90d6ab81, 0x67f815df, 0xf6875a4d, 0x0a15847b, 0xc54f9f4e, 0x571523b7, 0x402bd1c3 },
	{ 0xe03a98ea, 0xb4960266, 0x9cfa455c, 0x8a53bbf2, 0x99d2c503, 0x1a1456b5, 0x9a99b266, 0x31a2db88 }, // 1
	{ 0x5c5aa303, 0x8019051c, 0xdb0e199a, 0x1d959e84, 0x0ab23f40, 0xadeb336f, 0x1044c187, 0xdccde75e }, // 2
	{ 0x9213ba10, 0x39812c0a, 0x416bbf02, 0x5078aa37, 0x156578dc, 0xd2bf1a3f, 0xd027bbf7, 0xd3910041 }, // 3
	{ 0x0d5a2d42, 0x0ba75c18, 0x907eccf6, 0xac442bc7, 0x9c9f62dd, 0xd665dfd1, 0xce97c092, 0x23fcc663 }, // 4
	{ 0x036c6e97, 0xbb03f1ee, 0x1ab8e09e, 0xfa618e5d, 0x7e450521, 0xb29796fd, 0xa8ec6c44, 0x97818394 }, // 5
	{ 0x37858e4a, 0x8173fe8a, 0x2f3003db, 0x6c69b8f8, 0x2d8d672a, 0x4672c78a, 0x956a9ffb, 0x14427fc0 }, // 6
	// Round 7 (Function0)
	{ 0x8f15f4c5, 0xb775de52, 0xc45ec7bd, 0xbc88e4ae, 0xa76f4475, 0x1e00b882, 0x80bb118f, 0xf4a3a698 },
	{ 0x338ff48e, 0x20edf1b6, 0x1563a3a9, 0xfde05a7c, 0x24565faa, 0x5ae9ca36, 0x89f9b7d5, 0x362c4206 },
	{ 0x433529ce, 0x591ff5d0, 0x3d98fe4e, 0x86814e6f, 0x74f93a53, 0x81ad9d0e, 0xa74b9a73, 0x9f5ad8af },
	{ 0x670605a7, 0x26077447, 0x6a6234ee, 0x3f1080c6, 0xbe280b8b, 0x6f7ea0e0, 0x2717b96e, 0x7b487ec6 },
	{ 0xa50a550d, 0x81727686, 0xc0a4f84a, 0xd48d6050, 0x9fe7e391, 0x415a9e7e, 0x9ef18e97, 0x62b0e5f3 },
	{ 0xec1f9ffc, 0xf594d74f, 0x7a205440, 0xd895fa9d, 0x001ae4e3, 0x117e2e55, 0x84c9f4ce, 0xa554c324 },
	{ 0x2872df5b, 0xef7c8905, 0x286efebd, 0x2ed349ee, 0xe27ff578, 0x85937e44, 0xb2c4a50f, 0x7f5928eb },
	// Round 14 (Function0)
	{ 0x37695f70, 0x04771bc7, 0x4a3124b3, 0xe720b951, 0xf128865e, 0xe843fe74, 0x65e4d61d, 0x8a87d423 },
	{ 0xa3e8297d, 0xfb301b1d, 0xf2947692, 0xe01bdc5b, 0x097acbdd, 0x4f4924da, 0xc1d9309b, 0xbf829cf2 },
	{ 0x31bae7a4, 0x32fcae3b, 0xffbf70b4, 0x39d3bb53, 0x0544320d, 0xc1c39f45, 0x48bcf8de, 0xa08b29e0 },
	{ 0xfd05c9e5, 0x01b771a2, 0x0f09aef7, 0x95ed44e3, 0x12347094, 0x368e3be9, 0x34f19042, 0x4a982f4f },
	{ 0x631d4088, 0xf14abb7e, 0x15f66ca0, 0x30c60ae2, 0x4b44c147, 0xc5b67046, 0xffaf5287, 0xe68c6ecc },
	{ 0x56a4d5a4, 0x45ce5773, 0x00ca4fbd, 0xadd16430, 0x4b849dda, 0x68cea6e8, 0xae183ec8, 0x67255c14 },
	{ 0xf28cdaa3, 0x20b2601f, 0x16e10ecb, 0x7b846fc2, 0x5806e933, 0x7facced1, 0x9a99949a, 0x1885d1a0 },
	// Round 21 (Function0)
	{ 0xa15b5932, 0x67633d9f, 0xd319dd8d, 0xba6b04e4, 0xc01c9a50, 0xab19caf6, 0x46b4a5aa, 0x7eee560b },
	{ 0xea79b11f, 0x5aac571d, 0x742128a9, 0x76d35075, 0x35f7bde9, 0xfec2463a, 0xee51363b, 0x01707da3 },
	{ 0xafc135f7, 0x15638341, 0x42d8a498, 0xa8db3aea, 0x20eced78, 0x4d3bc3fa, 0x79676b9e, 0x832c8332 },
	{ 0x1f3b40a7, 0x6c4e3ee7, 0xf347271c, 0xfd4f21d2, 0x34f04059, 0x398dfdb8, 0x9a762db7, 0xef5957dc },
	{ 0x490c9b8d, 0xd0ae3b7d, 0xdaeb492b, 0x84558d7a, 0x49d7a25b, 0xf0e9a5f5, 0x0d70f368, 0x658ef8e4 },
	{ 0xf4a2b8a0, 0x92946891, 0x533b1036, 0x4f88e856, 0x9e07a80c, 0x555cb05b, 0x5aec3e75, 0x4cbcbaf8 },
	{ 0x993bbbe3, 0x28acae64, 0x7b9487f3, 0x6db334dc, 0xd6f4da75, 0x50a5346c, 0x5d1c6b72, 0x71db28b8 },
	// Round 28 (Function0)
	{ 0xf2e261f8, 0xf1bcac1c, 0x2a518d10, 0xa23fce43, 0x3364dbe3, 0x3cd1bb67, 0xfc75dd59, 0xb043e802 },
	{ 0xca5b0a33, 0xc3943b92, 0x75a12988, 0x1e4d790e, 0x4d19347f, 0xd7757479, 0x5c5316b4, 0x3fafeeb6 },
	{ 0xf7d4a8ea, 0x5324a326, 0x21391abe, 0xd23c32ba, 0x097ef45c, 0x4a17a344, 0x5127234c, 0xadd5a66d },
	{ 0xa63e1db5, 0xa17cf84c, 0x08c9f2af, 0x4d608672, 0x983d5983, 0xcc3ee246, 0x563c6b91, 0xf6c76e08 },
	{ 0xb333982f, 0xe8b6f406, 0x5e76bcb1, 0x36d4c1be, 0xa566d62b, 0x1582ee74, 0x2ae6c4ef, 0x6321efbc },
	{ 0x0d4ec1fd, 0x1614c17e, 0x69c953f4, 0x16fae006, 0xc45a7da7, 0x3daf907e, 0x26585806, 0x3f9d6328 },
	{ 0xe3f2c9d2, 0x16512a74, 0x0cd29b00, 0x9832e0f2, 0x30ceaa5f, 0xd830eb0d, 0x300cd4b7, 0x9af8cee3 },
	// Round 35 (Function0)
	{ 0x7b9ec54b, 0x574d239b, 0x9279f1b5, 0x316796e6, 0x6ee651ff, 0xf3a6e6cc, 0xd3688604, 0x05750a17 },
	{ 0xd98176b1, 0xb3cb2bf4, 0xce6c3213, 0x47154778, 0x8452173c, 0x825446ff, 0x62a205f8, 0x486a9323 },
	{ 0x0758df38, 0x442e7031, 0x65655e4e, 0x86ca0bd0, 0x897cfcf2, 0xa20940f0, 0x8e5086fc, 0x4e477830 },
	{ 0x39eea065, 0x26b29721, 0x8338f7d1, 0x6ff81301, 0x37e95ef7, 0xd1ed44a3, 0xbd3a2ce4, 0xe7de9fef },
	{ 0x15dfa08b, 0x7ceca7d8, 0xd9922576, 0x7eb027ab, 0xf6f7853c, 0xda7d8d53, 0xbe42dc12, 0xdea83eaa },
	{ 0x93ce25aa, 0xdaef5fc0, 0xd86902bd, 0xa5194a17, 0xfd43f65a, 0x33664d97, 0xf908731a, 0x6a21fd4c },
	{ 0x3198b435, 0xa163d09a, 0x701541db, 0x72409751, 0xbb0f1eea, 0xbf9d75f6, 0x9b54cded, 0xe26f4791 }
	// 42 rounds...
};

/*swapping bits 32i||32i+1||......||32i+15 with bits 32i+16||32i+17||......||32i+31 of 32-bit x*/
//#define SWAP16(x)  (x) = ((((x) & 0x0000ffffUL) << 16) | (((x) & 0xffff0000UL) >> 16));
#define SWAP16(x) (x) = __byte_perm(x, 0, 0x1032);

/*swapping bits 16i||16i+1||......||16i+7  with bits 16i+8||16i+9||......||16i+15 of 32-bit x*/
//#define SWAP8(x)   (x) = ((((x) & 0x00ff00ffUL) << 8) | (((x) & 0xff00ff00UL) >> 8));
#define SWAP8(x) (x) = __byte_perm(x, 0, 0x2301);

/*
__device__ __forceinline__
static void SWAP4(uint32_t &x) {
	uint32_t y = x & 0xF0F0F0F0;
	x = (x ^ y) << 4;
	x |= y >> 4;
}
__device__ __forceinline__
static void SWAP2(uint32_t &x) {
	uint32_t y = (x & 0xCCCCCCCC);
	x = (x ^ y) << 2;
	x |= y >> 2;
}
__device__ __forceinline__
static void SWAP1(uint32_t &x) {
	uint32_t y = (x & 0xAAAAAAAA);
	x = (x ^ y) << 1;
	x |= y >> 1;
}
*/

__device__ __forceinline__
static void SWAP4x4(uint32_t *x) {
	#pragma nounroll
	// y is used as tmp register too
	for (uint32_t y=0; y<4; y++, ++x) {
		asm("and.b32 %1, %0, 0xF0F0F0F0;\n\t"
		"xor.b32 %0, %0, %1; shr.b32 %1, %1, 4;\n\t"
		"vshl.u32.u32.u32.clamp.add %0, %0, 4, %1;"
		: "+r"(*x) : "r"(y));
	}
}

__device__ __forceinline__
static void SWAP2x4(uint32_t *x) {
	#pragma nounroll
	// y is used as tmp register too
	for (uint32_t y=0; y<4; y++, ++x) {
		asm("and.b32 %1, %0, 0xCCCCCCCC;\n\t"
		"xor.b32 %0, %0, %1; shr.b32 %1, %1, 2; \n\t"
		"vshl.u32.u32.u32.clamp.add %0, %0, 2, %1;"
		: "+r"(*x) : "r"(y));
	}
}

__device__ __forceinline__
static void SWAP1x4(uint32_t *x) {
	#pragma nounroll
	// y is used as tmp register too
	for (uint32_t y=0; y<4; y++, ++x) {
		asm("and.b32 %1, %0, 0xAAAAAAAA;\n\t"
		"xor.b32 %0, %0, %1; shr.b32 %1, %1, 1; \n\t"
		"vshl.u32.u32.u32.clamp.add %0, %0, 1, %1;"
		: "+r"(*x) : "r"(y));
	}
}

/* The MDS transform */
#define L(m0,m1,m2,m3,m4,m5,m6,m7) \
      m4 ^= m1;                    \
      m5 ^= m2;                    \
      m6 ^= m0 ^ m3;               \
      m7 ^= m0;                    \
      m0 ^= m5;                    \
      m1 ^= m6;                    \
      m2 ^= m4 ^ m7;               \
      m3 ^= m4;

/* The Sbox */
#define Sbox(m0, m1, m2, m3, cc)   \
      m3  = ~(m3);                 \
      m0 ^= (~(m2)) & cc;          \
      temp0 = cc ^ (m0 & m1);      \
      m0 ^= m2 & m3;               \
      m3 ^= (~(m1)) & m2;          \
      m1 ^= m0 & m2;               \
      m2 ^= m0 & (~(m3));          \
      m0 ^= m1 | m3;               \
      m3 ^= m1 & m2;               \
      m1 ^= temp0 & m0;            \
      m2 ^= temp0;

__device__ __forceinline__
static void Sbox_and_MDS_layer(uint32_t x[8][4], const int rnd)
{
	uint2* cc = (uint2*) &c_E8_bslice32[rnd];

	// Sbox and MDS layer
	#pragma unroll
	for (int i = 0; i < 4; i++, ++cc) {
		uint32_t temp0;
		Sbox(x[0][i], x[2][i], x[4][i], x[6][i], cc->x);
		Sbox(x[1][i], x[3][i], x[5][i], x[7][i], cc->y);
		L(x[0][i], x[2][i], x[4][i], x[6][i], x[1][i], x[3][i], x[5][i], x[7][i]);
	}
}

__device__ __forceinline__
static void RoundFunction0(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 0); // 0, 7, 14 .. 35
	#pragma unroll 4
	for (int j = 1; j < 8; j += 2) { // 1, 3, 5, 7 (Even)
		SWAP1x4(x[j]);
		// SWAP1(x[j][0]); SWAP1(x[j][1]); SWAP1(x[j][2]); SWAP1(x[j][3]);
	}
}

__device__ __forceinline__
static void RoundFunction1(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 1);

	#pragma unroll 4
	for (int j = 1; j < 8; j += 2) {
		SWAP2x4(x[j]);
		// SWAP2(x[j][0]); SWAP2(x[j][1]); SWAP2(x[j][2]); SWAP2(x[j][3]);
	}
}

__device__ __forceinline__
static void RoundFunction2(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 2);

	#pragma unroll 4
	for (int j = 1; j < 8; j += 2) {
		SWAP4x4(x[j]);
		// SWAP4(x[j][0]); SWAP4(x[j][1]); SWAP4(x[j][2]); SWAP4(x[j][3]);
	}
}

__device__ __forceinline__
static void RoundFunction3(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 3);

	//uint32_t* xj = x[j];
	#pragma unroll 4
	for (int j = 1; j < 8; j += 2) {
		SWAP8(x[j][0]);
		SWAP8(x[j][1]);
		SWAP8(x[j][2]);
		SWAP8(x[j][3]);
	}
}

__device__ __forceinline__
static void RoundFunction4(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 4);

	#pragma unroll 4
	for (int j = 1; j < 8; j += 2)
	{
		//uint32_t* xj = x[j];
		#pragma unroll
		for (int i = 0; i < 4; i++)
			SWAP16(x[j][i]);
	}
}

__device__ __forceinline__
static void RoundFunction5(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 5);

	#pragma unroll 4
	for (int j = 1; j < 8; j += 2)
	{
		xchg(x[j][0], x[j][1]);
		xchg(x[j][2], x[j][3]);
	}
}

__device__ __forceinline__
static void RoundFunction6(uint32_t x[8][4], const int rnd)
{
	Sbox_and_MDS_layer(x, rnd + 6);

	#pragma unroll 4
	for (int j = 1; j < 8; j += 2)
	{
		xchg(x[j][0], x[j][2]);
		xchg(x[j][1], x[j][3]);
	}
}

/* The bijective function E8, in bitslice form */
__device__
static void E8(uint32_t x[8][4])
{
	/* perform 6 loops of 7 rounds */
	for (int r = 0; r < 42; r += 7)
	{
		RoundFunction0(x, r);
		RoundFunction1(x, r);
		RoundFunction2(x, r);
		RoundFunction3(x, r);
		RoundFunction4(x, r);
		RoundFunction5(x, r);
		RoundFunction6(x, r);
	}
}

__global__
//__launch_bounds__(256,2)
void quark_jh512_gpu_hash_64(const uint32_t threads, const uint32_t startNounce, uint32_t* g_hash, uint32_t * g_nonceVector)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);
		const uint32_t hashPosition = nounce - startNounce;
		uint32_t *Hash = &g_hash[(size_t)16 * hashPosition];

		uint32_t h[16];
		AS_UINT4(&h[ 0]) = AS_UINT4(&Hash[ 0]);
		AS_UINT4(&h[ 4]) = AS_UINT4(&Hash[ 4]);
		AS_UINT4(&h[ 8]) = AS_UINT4(&Hash[ 8]);
		AS_UINT4(&h[12]) = AS_UINT4(&Hash[12]);

		uint32_t x[8][4] = { /* init */
			{ 0x964bd16f, 0x17aa003e, 0x052e6a63, 0x43d5157a },
			{ 0x8d5e228a, 0x0bef970c, 0x591234e9, 0x61c3b3f2 },
			{ 0xc1a01d89, 0x1e806f53, 0x6b05a92a, 0x806d2bea },
			{ 0xdbcc8e58, 0xa6ba7520, 0x763a0fa9, 0xf73bf8ba },
			{ 0x05e66901, 0x694ae341, 0x8e8ab546, 0x5ae66f2e },
			{ 0xd0a74710, 0x243c84c1, 0xb1716e3b, 0x99c15a2d },
			{ 0xecf657cf, 0x56f8b19d, 0x7c8806a7, 0x56b11657 },
			{ 0xdffcc2e3, 0xfb1785e6, 0x78465a54, 0x4bdd8ccc }
		};

		#pragma unroll
		for (int i = 0; i < 16; i++)
			x[i/4][i & 3] ^= h[i];

		E8(x);

		#pragma unroll
		for (int i = 0; i < 16; i++)
			x[(i+16)/4][(i+16) & 3] ^= h[i];

		x[0][0] ^= 0x80U;
		x[3][3] ^= 0x00020000U;

		E8(x);

		x[4][0] ^= 0x80U;
		x[7][3] ^= 0x00020000U;

		AS_UINT4(&Hash[ 0]) = AS_UINT4(&x[4][0]);
		AS_UINT4(&Hash[ 4]) = AS_UINT4(&x[5][0]);
		AS_UINT4(&Hash[ 8]) = AS_UINT4(&x[6][0]);
		AS_UINT4(&Hash[12]) = AS_UINT4(&x[7][0]);
	}
}

__host__
void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	quark_jh512_gpu_hash_64<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
}

// Setup function
__host__ void  quark_jh512_cpu_init(int thr_id, uint32_t threads) {}

#define WANT_JH80_MIDSTATE
#ifdef WANT_JH80

__constant__
static uint32_t c_PaddedMessage80[20]; // padded message (80 bytes)

__host__
void jh512_setBlock_80(int thr_id, uint32_t *endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__
void jh512_gpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint32_t * g_outhash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t h[20];
		AS_UINT4(&h[ 0]) = AS_UINT4(&c_PaddedMessage80[ 0]);
		AS_UINT4(&h[ 4]) = AS_UINT4(&c_PaddedMessage80[ 4]);
		AS_UINT4(&h[ 8]) = AS_UINT4(&c_PaddedMessage80[ 8]);
		AS_UINT4(&h[12]) = AS_UINT4(&c_PaddedMessage80[12]);
		AS_UINT2(&h[16]) = AS_UINT2(&c_PaddedMessage80[16]);
		h[18] = c_PaddedMessage80[18];
		h[19] = cuda_swab32(startNounce + thread);

		uint32_t x[8][4] = { /* init */
			{ 0x964bd16f, 0x17aa003e, 0x052e6a63, 0x43d5157a },
			{ 0x8d5e228a, 0x0bef970c, 0x591234e9, 0x61c3b3f2 },
			{ 0xc1a01d89, 0x1e806f53, 0x6b05a92a, 0x806d2bea },
			{ 0xdbcc8e58, 0xa6ba7520, 0x763a0fa9, 0xf73bf8ba },
			{ 0x05e66901, 0x694ae341, 0x8e8ab546, 0x5ae66f2e },
			{ 0xd0a74710, 0x243c84c1, 0xb1716e3b, 0x99c15a2d },
			{ 0xecf657cf, 0x56f8b19d, 0x7c8806a7, 0x56b11657 },
			{ 0xdffcc2e3, 0xfb1785e6, 0x78465a54, 0x4bdd8ccc }
		};

		// 1 (could be precomputed)
		#pragma unroll
		for (int i = 0; i < 16; i++)
			x[i/4][i & 3] ^= h[i];
		E8(x);
		#pragma unroll
		for (int i = 0; i < 16; i++)
			x[(i+16)/4][(i+16) & 3] ^= h[i];

		// 2 (16 bytes with nonce)
		#pragma unroll
		for (int i = 0; i < 4; i++)
			x[0][i] ^= h[16+i];
		x[1][0] ^= 0x80U;
		E8(x);
		#pragma unroll
		for (int i = 0; i < 4; i++)
			x[4][i] ^= h[16+i];
		x[5][0] ^= 0x80U;

		// 3 close
		x[3][3] ^= 0x80020000U; // 80 bytes = 640bits (0x280)
		E8(x);
		x[7][3] ^= 0x80020000U;

		uint32_t *Hash = &g_outhash[(size_t)16 * thread];
		AS_UINT4(&Hash[ 0]) = AS_UINT4(&x[4][0]);
		AS_UINT4(&Hash[ 4]) = AS_UINT4(&x[5][0]);
		AS_UINT4(&Hash[ 8]) = AS_UINT4(&x[6][0]);
		AS_UINT4(&Hash[12]) = AS_UINT4(&x[7][0]);
	}
}

__host__
void jh512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	jh512_gpu_hash_80 <<<grid, block>>> (threads, startNounce, d_hash);
}

#endif

#ifdef WANT_JH80_MIDSTATE

__constant__ static uint32_t c_JHState[32];
__constant__ static uint32_t c_Message[4];

__global__
void jh512_gpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint32_t * g_outhash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// 1 (precomputed state)
		uint32_t x[8][4];
		AS_UINT4(&x[0][0]) = AS_UINT4(&c_JHState[ 0]);
		AS_UINT4(&x[1][0]) = AS_UINT4(&c_JHState[ 4]);
		AS_UINT4(&x[2][0]) = AS_UINT4(&c_JHState[ 8]);
		AS_UINT4(&x[3][0]) = AS_UINT4(&c_JHState[12]);

		AS_UINT4(&x[4][0]) = AS_UINT4(&c_JHState[16]);
		AS_UINT4(&x[5][0]) = AS_UINT4(&c_JHState[20]);
		AS_UINT4(&x[6][0]) = AS_UINT4(&c_JHState[24]);
		AS_UINT4(&x[7][0]) = AS_UINT4(&c_JHState[28]);

		// 2 (16 bytes with nonce)
		uint32_t h[4];
		AS_UINT2(&h[0]) = AS_UINT2(&c_Message[0]);
		h[2] = c_Message[2];
		h[3] = cuda_swab32(startNounce + thread);

		#pragma unroll
		for (int i = 0; i < 4; i++)
			x[0][i] ^= h[i];
		x[1][0] ^= 0x80U;
		E8(x);
		#pragma unroll
		for (int i = 0; i < 4; i++)
			x[4][i] ^= h[i];
		x[5][0] ^= 0x80U;

		// 3 close
		x[3][3] ^= 0x80020000U; // 80 bytes = 640bits (0x280)
		E8(x);
		x[7][3] ^= 0x80020000U;

		uint32_t *Hash = &g_outhash[(size_t)16 * thread];
		AS_UINT4(&Hash[ 0]) = AS_UINT4(&x[4][0]);
		AS_UINT4(&Hash[ 4]) = AS_UINT4(&x[5][0]);
		AS_UINT4(&Hash[ 8]) = AS_UINT4(&x[6][0]);
		AS_UINT4(&Hash[12]) = AS_UINT4(&x[7][0]);
	}
}

__host__
void jh512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	jh512_gpu_hash_80 <<<grid, block>>> (threads, startNounce, d_hash);
}

extern "C" {
#undef SPH_C32
#undef SPH_T32
#undef SPH_C64
#undef SPH_T64
#include <sph/sph_jh.h>
}

__host__
void jh512_setBlock_80(int thr_id, uint32_t *endiandata)
{
	sph_jh512_context ctx_jh;

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, endiandata, 64);

	hipMemcpyToSymbol(HIP_SYMBOL(c_JHState), ctx_jh.H.narrow, 128, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_Message), &endiandata[16], sizeof(c_Message), 0, hipMemcpyHostToDevice);
}

#endif
