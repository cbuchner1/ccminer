#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#endif

#define TPB 256
#define THF 4U

#if __CUDA_ARCH__ >= 300
#include "groestl_functions_quad.h"
#include "groestl_transf_quad.h"
#endif

#define WANT_GROESTL80
#ifdef WANT_GROESTL80
__constant__ static uint32_t c_Message80[20];
#endif

#include "cuda_quark_groestl512_sm2.cuh"

__global__ __launch_bounds__(TPB, THF)
void quark_groestl512_gpu_hash_64_quad(const uint32_t threads, const uint32_t startNounce, uint32_t * g_hash, uint32_t * __restrict g_nonceVector)
{
#if __CUDA_ARCH__ >= 300

	// BEWARE : 4-WAY CODE (one hash need 4 threads)
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;

	if (thread < threads)
	{
		uint32_t message[8];
		uint32_t state[8];

		uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
		off_t hashPosition = nounce - startNounce;
		uint32_t *pHash = &g_hash[hashPosition << 4];

		const uint32_t thr = threadIdx.x & 0x3; // % THF

		/*| M0 M1 M2 M3 | M4 M5 M6 M7 | (input)
		--|-------------|-------------|
		T0|  0  4  8 12 | 80          |
		T1|  1  5    13 |             |
		T2|  2  6    14 |             |
		T3|  3  7    15 |          01 |
		--|-------------|-------------| */

		#pragma unroll
		for(int k=0;k<4;k++) message[k] = pHash[thr + (k * THF)];

		#pragma unroll
		for(int k=4;k<8;k++) message[k] = 0;

		if (thr == 0) message[4] = 0x80U; // end of data tag
		if (thr == 3) message[7] = 0x01000000U;

		uint32_t msgBitsliced[8];
		to_bitslice_quad(message, msgBitsliced);

		groestl512_progressMessage_quad(state, msgBitsliced);

		uint32_t hash[16];
		from_bitslice_quad(state, hash);

		// uint4 = 4x4 uint32_t = 16 bytes
		if (thr == 0) {
			uint4 *phash = (uint4*) hash;
			uint4 *outpt = (uint4*) pHash;
			outpt[0] = phash[0];
			outpt[1] = phash[1];
			outpt[2] = phash[2];
			outpt[3] = phash[3];
		}
	}
#endif
}

__host__
void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
	int dev_id = device_map[thr_id];
	cuda_get_arch(thr_id);
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
		quark_groestl512_sm20_init(thr_id, threads);
}

__host__
void quark_groestl512_cpu_free(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
		quark_groestl512_sm20_free(thr_id);
}

__host__
void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	uint32_t threadsperblock = TPB;

	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	const uint32_t factor = THF;

	dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300)
		quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
	else
		quark_groestl512_sm20_hash_64(thr_id, threads, startNounce, d_nonceVector, d_hash, order);
}

// --------------------------------------------------------------------------------------------------------------------------------------------

#ifdef WANT_GROESTL80

__host__
void groestl512_setBlock_80(int thr_id, uint32_t *endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_Message80), endiandata, sizeof(c_Message80), 0, hipMemcpyHostToDevice);
}

__global__ __launch_bounds__(TPB, THF)
void groestl512_gpu_hash_80_quad(const uint32_t threads, const uint32_t startNounce, uint32_t * g_outhash)
{
#if __CUDA_ARCH__ >= 300
	// BEWARE : 4-WAY CODE (one hash need 4 threads)
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	if (thread < threads)
	{
		const uint32_t thr = threadIdx.x & 0x3; // % THF

		/*| M0 M1 M2 M3 M4 | M5 M6 M7 | (input)
		--|----------------|----------|
		T0|  0  4  8 12 16 | 80       |
		T1|  1  5       17 |          |
		T2|  2  6       18 |          |
		T3|  3  7       Nc |       01 |
		--|----------------|----------| TPR */

		uint32_t message[8];

		#pragma unroll 5
		for(int k=0; k<5; k++) message[k] = c_Message80[thr + (k * THF)];

		#pragma unroll 3
		for(int k=5; k<8; k++) message[k] = 0;

		if (thr == 0) message[5] = 0x80U;
		if (thr == 3) {
			message[4] = cuda_swab32(startNounce + thread);
			message[7] = 0x01000000U;
		}

		uint32_t msgBitsliced[8];
		to_bitslice_quad(message, msgBitsliced);

		uint32_t state[8];
		groestl512_progressMessage_quad(state, msgBitsliced);

		uint32_t hash[16];
		from_bitslice_quad(state, hash);

		if (thr == 0) { /* 4 threads were done */
			const off_t hashPosition = thread;
			//if (!thread) hash[15] = 0xFFFFFFFF;
			uint4 *outpt = (uint4*) &g_outhash[hashPosition << 4];
			uint4 *phash = (uint4*) hash;
			outpt[0] = phash[0];
			outpt[1] = phash[1];
			outpt[2] = phash[2];
			outpt[3] = phash[3];
		}
	}
#endif
}

__host__
void groestl512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	int dev_id = device_map[thr_id];

	if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300) {
		const uint32_t threadsperblock = TPB;
		const uint32_t factor = THF;

		dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
		dim3 block(threadsperblock);

		groestl512_gpu_hash_80_quad <<<grid, block>>> (threads, startNounce, d_hash);

	} else {

		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);

		groestl512_gpu_hash_80_sm2 <<<grid, block>>> (threads, startNounce, d_hash);
	}
}

#endif
