#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "cuda_helper.h"

#define TPB 256
#define THF 4

#if __CUDA_ARCH__ >= 300
#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"
#endif

#include "quark/cuda_quark_groestl512_sm20.cu"

__global__ __launch_bounds__(TPB, THF)
void quark_groestl512_gpu_hash_64_quad(uint32_t threads, uint32_t startNounce, uint32_t * __restrict g_hash, uint32_t * __restrict g_nonceVector)
{
#if __CUDA_ARCH__ >= 300
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
        off_t hashPosition = nounce - startNounce;
        uint32_t *pHash = &g_hash[hashPosition << 4];

        const uint32_t thr = threadIdx.x % THF;

        #pragma unroll
        for(int k=0;k<4;k++) message[k] = pHash[thr + (k * THF)];

        #pragma unroll
        for(int k=4;k<8;k++) message[k] = 0;

        if (thr == 0) message[4] = 0x80U;
        if (thr == 3) message[7] = 0x01000000U;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        groestl512_progressMessage_quad(state, msgBitsliced);

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t hash[16];
        from_bitslice_quad(state, hash);

        // uint4 = 4x4 uint32_t = 16 bytes
        if (thr == 0) {
            uint4 *phash = (uint4*) hash;
            uint4 *outpt = (uint4*) pHash;
            outpt[0] = phash[0];
            outpt[1] = phash[1];
            outpt[2] = phash[2];
            outpt[3] = phash[3];
        }
/*
        if (thr == 0) {
            #pragma unroll
            for(int k=0;k<16;k++) outpHash[k] = hash[k];
        }
*/
    }
#endif
}

__global__ void __launch_bounds__(TPB, THF)
 quark_doublegroestl512_gpu_hash_64_quad(uint32_t threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *g_nonceVector)
{
#if __CUDA_ARCH__ >= 300
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x)>>2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);

        off_t hashPosition = nounce - startNounce;
        uint32_t * inpHash = &g_hash[hashPosition<<4];
        const uint16_t thr = threadIdx.x % THF;

        #pragma unroll
        for(int k=0;k<4;k++) message[k] = inpHash[(k * THF) + thr];

        #pragma unroll
        for(int k=4;k<8;k++) message[k] = 0;

        if (thr == 0) message[4] = 0x80;
        if (thr == 3) message[7] = 0x01000000;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                // Verkettung zweier Runden inclusive Padding.
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + (((threadIdx.x%4)==3)<<13));
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
                msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + (((threadIdx.x%4)==0)<<4));
            }
        }

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t *outpHash = inpHash;
        uint32_t hash[16];
        from_bitslice_quad(state, hash);

        if (thr == 0)
        {
            #pragma unroll
            for(int k=0;k<16;k++) outpHash[k] = hash[k];
        }
    }
#endif
}

__host__
void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
    int dev_id = device_map[thr_id];
    cuda_get_arch(thr_id);
    if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
        quark_groestl512_sm20_init(thr_id, threads);
}

__host__
void quark_groestl512_cpu_free(int thr_id)
{
    int dev_id = device_map[thr_id];
    if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
        quark_groestl512_sm20_free(thr_id);
}

__host__
void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    int threadsperblock = TPB;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const int factor = THF;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    int dev_id = device_map[thr_id];

    if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300)
        quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
    else
        quark_groestl512_sm20_hash_64(thr_id, threads, startNounce, d_nonceVector, d_hash, order);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void quark_doublegroestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int factor = THF;
    int threadsperblock = TPB;

    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    int dev_id = device_map[thr_id];

    if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300)
        quark_doublegroestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
    else
        quark_doublegroestl512_sm20_hash_64(thr_id, threads, startNounce, d_nonceVector, d_hash, order);

    MyStreamSynchronize(NULL, order, thr_id);
}
