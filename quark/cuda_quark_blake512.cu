#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "miner.h"
#include "cuda_helper.h"

#define ROTR(x,n) ROTR64(x,n)

// use sp kernel on SM 5+
#define SP_KERNEL

#define USE_SHUFFLE 0

__constant__
static uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

// ---------------------------- BEGIN CUDA quark_blake512 functions ------------------------------------

__device__ __constant__
static const uint8_t c_sigma_big[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },

	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },

	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__
static const uint64_t c_u512[16] =
{
	0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,
	0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
	0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,
	0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
	0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,
	0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
	0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,
	0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};

#define G(a,b,c,d,x) { \
	uint32_t idx1 = sigma[i][x]; \
	uint32_t idx2 = sigma[i][x+1]; \
	v[a] += (m[idx1] ^ u512[idx2]) + v[b]; \
	v[d] = SWAPDWORDS(v[d] ^ v[a]); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 25); \
	v[a] += (m[idx2] ^ u512[idx1]) + v[b]; \
	v[d] = ROTR( v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR( v[b] ^ v[c], 11); \
}

__device__ __forceinline__
void quark_blake512_compress(uint64_t *h, const uint64_t *block, const uint8_t ((*sigma)[16]), const uint64_t *u512, const int T0)
{
	uint64_t v[16];
	uint64_t m[16];

	#pragma unroll
	for(int i=0; i < 16; i++) {
		m[i] = cuda_swab64(block[i]);
	}

	//#pragma unroll 8
	for(int i=0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = u512[0];
	v[ 9] = u512[1];
	v[10] = u512[2];
	v[11] = u512[3];
	v[12] = u512[4] ^ T0;
	v[13] = u512[5] ^ T0;
	v[14] = u512[6];
	v[15] = u512[7];

	//#pragma unroll 16
	for(int i=0; i < 16; i++)
	{
		/* column step */
		G( 0, 4, 8, 12, 0 );
		G( 1, 5, 9, 13, 2 );
		G( 2, 6, 10, 14, 4 );
		G( 3, 7, 11, 15, 6 );
		/* diagonal step */
		G( 0, 5, 10, 15, 8 );
		G( 1, 6, 11, 12, 10 );
		G( 2, 7, 8, 13, 12 );
		G( 3, 4, 9, 14, 14 );
	}

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}

__global__ __launch_bounds__(256, 4)
void quark_blake512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint64_t *g_hash)
{
#if !defined(SP_KERNEL) || __CUDA_ARCH__ < 500
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

#if USE_SHUFFLE
	const uint32_t warpBlockID = (thread + 15)>>4; // aufrunden auf volle Warp-Blöcke

	if (warpBlockID < ( (threads+15)>>4 ))
#else
	if (thread < threads)
#endif
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		off_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition<<3]; // hashPosition * 8

		// 128 Bytes
		uint64_t buf[16];

		// State
		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		// Message for first round
		#pragma unroll 8
		for (int i=0; i < 8; ++i)
			buf[i] = inpHash[i];

		// Hash Pad
		buf[8]  = 0x0000000000000080ull;
		buf[9]  = 0;
		buf[10] = 0;
		buf[11] = 0;
		buf[12] = 0;
		buf[13] = 0x0100000000000000ull;
		buf[14] = 0;
		buf[15] = 0x0002000000000000ull;

		// Ending round
		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 512);

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t*)&g_hash[hashPosition * 8U];
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			outHash[2*i+0] = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = &g_hash[hashPosition * 8U];
		for (int i=0; i < 8; i++) {
			outHash[i] = cuda_swab64(h[i]);
		}
#endif
	}
#endif /* SP */
}

__global__ __launch_bounds__(256,4)
void quark_blake512_gpu_hash_80(uint32_t threads, uint32_t startNounce, void *outputHash)
{
//#if !defined(SP_KERNEL) || __CUDA_ARCH__ < 500
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t buf[16];
		#pragma unroll
		for (int i=0; i < 16; ++i)
			buf[i] = c_PaddedMessage80[i];

		// The test Nonce
		const uint32_t nounce = startNounce + thread;
		((uint32_t*)buf)[19] = cuda_swab32(nounce);

		uint64_t h[8] = {
			0x6a09e667f3bcc908ULL,
			0xbb67ae8584caa73bULL,
			0x3c6ef372fe94f82bULL,
			0xa54ff53a5f1d36f1ULL,
			0x510e527fade682d1ULL,
			0x9b05688c2b3e6c1fULL,
			0x1f83d9abfb41bd6bULL,
			0x5be0cd19137e2179ULL
		};

		quark_blake512_compress(h, buf, c_sigma_big, c_u512, 640);

#if __CUDA_ARCH__ <= 350
		uint32_t *outHash = (uint32_t*)outputHash + (thread * 16U);
		#pragma unroll 8
		for (uint32_t i=0; i < 8; i++) {
			outHash[2*i]   = cuda_swab32( _HIDWORD(h[i]) );
			outHash[2*i+1] = cuda_swab32( _LODWORD(h[i]) );
		}
#else
		uint64_t *outHash = (uint64_t*)outputHash + (thread * 8U);
		for (uint32_t i=0; i < 8; i++) {
			outHash[i] = cuda_swab64( h[i] );
		}
#endif
	}
//#endif
}

#ifdef SP_KERNEL
#include "cuda_quark_blake512_sp.cuh"
#endif

__host__
void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash, int order)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_hash_64_sp(threads, startNounce, d_nonceVector, d_outputHash);
	else
#endif
	{
		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);
		quark_blake512_gpu_hash_64<<<grid, block>>>(threads, startNounce, d_nonceVector, (uint64_t*)d_outputHash);
	}
	MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_hash_80_sp(threads, startNounce, d_outputHash);
	else
#endif
	{
		const uint32_t threadsperblock = 256;
		dim3 grid((threads + threadsperblock-1)/threadsperblock);
		dim3 block(threadsperblock);

		quark_blake512_gpu_hash_80<<<grid, block>>>(threads, startNounce, d_outputHash);
	}
}

// ---------------------------- END CUDA quark_blake512 functions ------------------------------------

__host__
void quark_blake512_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
}

__host__
void quark_blake512_cpu_free(int thr_id)
{
}

// ----------------------------- Host midstate for 80-bytes input ------------------------------------

#undef SPH_C32
#undef SPH_T32
#undef SPH_C64
#undef SPH_T64

extern "C" {
#include "sph/sph_blake.h"
}

__host__
void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *endiandata)
{
#ifdef SP_KERNEL
	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_blake512_cpu_setBlock_80_sp(thr_id, (uint64_t*) endiandata);
	else
#endif
	{
		uint64_t message[16];

		memcpy(message, endiandata, 80);
		message[10] = 0x80;
		message[11] = 0;
		message[12] = 0;
		message[13] = 0x0100000000000000ull;
		message[14] = 0;
		message[15] = 0x8002000000000000ull; // 0x280

		hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), message, sizeof(message), 0, hipMemcpyHostToDevice);
	}
	CUDA_LOG_ERROR();
}
