#include "hip/hip_runtime.h"
#define SP_KERNEL

#ifdef SP_KERNEL
#include "cuda_skein512_sp.cuh"
#undef TFBIG_KINIT
#undef TFBIG_ADDKEY
#undef TFBIG_MIX
#else

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"

#endif

static __constant__ uint64_t c_PaddedMessage80[20]; // padded message (80 bytes + 72 bytes midstate + align)

// Take a look at: https://www.schneier.com/skein1.3.pdf

#define SHL(x, n)			((x) << (n))
#define SHR(x, n)			((x) >> (n))

#if __CUDA_ARCH__ > 300
__device__
uint64_t skein_rotl64(const uint64_t x, const int offset)
{
	uint64_t res;
	asm("{\n\t"
		".reg .u32 tl,th,vl,vh;\n\t"
		".reg .pred p;\n\t"
		"mov.b64 {tl,th}, %1;\n\t"
		"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
		"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
		"setp.lt.u32 p, %2, 32;\n\t"
		"@!p mov.b64 %0, {vl,vh};\n\t"
		"@p  mov.b64 %0, {vh,vl};\n\t"
	"}"
		: "=l"(res) : "l"(x) , "r"(offset)
	);
	return res;
}
#undef ROTL64
#define ROTL64 skein_rotl64
#endif

/*
 * M9_ ## s ## _ ## i  evaluates to s+i mod 9 (0 <= s <= 18, 0 <= i <= 7).
 */

#define M9_0_0    0
#define M9_0_1    1
#define M9_0_2    2
#define M9_0_3    3
#define M9_0_4    4
#define M9_0_5    5
#define M9_0_6    6
#define M9_0_7    7

#define M9_1_0    1
#define M9_1_1    2
#define M9_1_2    3
#define M9_1_3    4
#define M9_1_4    5
#define M9_1_5    6
#define M9_1_6    7
#define M9_1_7    8

#define M9_2_0    2
#define M9_2_1    3
#define M9_2_2    4
#define M9_2_3    5
#define M9_2_4    6
#define M9_2_5    7
#define M9_2_6    8
#define M9_2_7    0

#define M9_3_0    3
#define M9_3_1    4
#define M9_3_2    5
#define M9_3_3    6
#define M9_3_4    7
#define M9_3_5    8
#define M9_3_6    0
#define M9_3_7    1

#define M9_4_0    4
#define M9_4_1    5
#define M9_4_2    6
#define M9_4_3    7
#define M9_4_4    8
#define M9_4_5    0
#define M9_4_6    1
#define M9_4_7    2

#define M9_5_0    5
#define M9_5_1    6
#define M9_5_2    7
#define M9_5_3    8
#define M9_5_4    0
#define M9_5_5    1
#define M9_5_6    2
#define M9_5_7    3

#define M9_6_0    6
#define M9_6_1    7
#define M9_6_2    8
#define M9_6_3    0
#define M9_6_4    1
#define M9_6_5    2
#define M9_6_6    3
#define M9_6_7    4

#define M9_7_0    7
#define M9_7_1    8
#define M9_7_2    0
#define M9_7_3    1
#define M9_7_4    2
#define M9_7_5    3
#define M9_7_6    4
#define M9_7_7    5

#define M9_8_0    8
#define M9_8_1    0
#define M9_8_2    1
#define M9_8_3    2
#define M9_8_4    3
#define M9_8_5    4
#define M9_8_6    5
#define M9_8_7    6

#define M9_9_0    0
#define M9_9_1    1
#define M9_9_2    2
#define M9_9_3    3
#define M9_9_4    4
#define M9_9_5    5
#define M9_9_6    6
#define M9_9_7    7

#define M9_10_0   1
#define M9_10_1   2
#define M9_10_2   3
#define M9_10_3   4
#define M9_10_4   5
#define M9_10_5   6
#define M9_10_6   7
#define M9_10_7   8

#define M9_11_0   2
#define M9_11_1   3
#define M9_11_2   4
#define M9_11_3   5
#define M9_11_4   6
#define M9_11_5   7
#define M9_11_6   8
#define M9_11_7   0

#define M9_12_0   3
#define M9_12_1   4
#define M9_12_2   5
#define M9_12_3   6
#define M9_12_4   7
#define M9_12_5   8
#define M9_12_6   0
#define M9_12_7   1

#define M9_13_0   4
#define M9_13_1   5
#define M9_13_2   6
#define M9_13_3   7
#define M9_13_4   8
#define M9_13_5   0
#define M9_13_6   1
#define M9_13_7   2

#define M9_14_0   5
#define M9_14_1   6
#define M9_14_2   7
#define M9_14_3   8
#define M9_14_4   0
#define M9_14_5   1
#define M9_14_6   2
#define M9_14_7   3

#define M9_15_0   6
#define M9_15_1   7
#define M9_15_2   8
#define M9_15_3   0
#define M9_15_4   1
#define M9_15_5   2
#define M9_15_6   3
#define M9_15_7   4

#define M9_16_0   7
#define M9_16_1   8
#define M9_16_2   0
#define M9_16_3   1
#define M9_16_4   2
#define M9_16_5   3
#define M9_16_6   4
#define M9_16_7   5

#define M9_17_0   8
#define M9_17_1   0
#define M9_17_2   1
#define M9_17_3   2
#define M9_17_4   3
#define M9_17_5   4
#define M9_17_6   5
#define M9_17_7   6

#define M9_18_0   0
#define M9_18_1   1
#define M9_18_2   2
#define M9_18_3   3
#define M9_18_4   4
#define M9_18_5   5
#define M9_18_6   6
#define M9_18_7   7

/*
 * M3_ ## s ## _ ## i  evaluates to s+i mod 3 (0 <= s <= 18, 0 <= i <= 1).
 */

#define M3_0_0    0
#define M3_0_1    1
#define M3_1_0    1
#define M3_1_1    2
#define M3_2_0    2
#define M3_2_1    0
#define M3_3_0    0
#define M3_3_1    1
#define M3_4_0    1
#define M3_4_1    2
#define M3_5_0    2
#define M3_5_1    0
#define M3_6_0    0
#define M3_6_1    1
#define M3_7_0    1
#define M3_7_1    2
#define M3_8_0    2
#define M3_8_1    0
#define M3_9_0    0
#define M3_9_1    1
#define M3_10_0   1
#define M3_10_1   2
#define M3_11_0   2
#define M3_11_1   0
#define M3_12_0   0
#define M3_12_1   1
#define M3_13_0   1
#define M3_13_1   2
#define M3_14_0   2
#define M3_14_1   0
#define M3_15_0   0
#define M3_15_1   1
#define M3_16_0   1
#define M3_16_1   2
#define M3_17_0   2
#define M3_17_1   0
#define M3_18_0   0
#define M3_18_1   1

#define XCAT(x, y)     XCAT_(x, y)
#define XCAT_(x, y)    x ## y

#define SKBI(k, s, i)   XCAT(k, XCAT(XCAT(XCAT(M9_, s), _), i))
#define SKBT(t, s, v)   XCAT(t, XCAT(XCAT(XCAT(M3_, s), _), v))

#define TFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ SPH_C64(0x1BD11BDAA9FC1A22); \
		t2 = t0 ^ t1; \
	}

#define TFBIG_ADDKEY(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + (uint64_t)s); \
	}

#define TFBIG_MIX(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROTL64(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX(w0, w1, rc0); \
		TFBIG_MIX(w2, w3, rc1); \
		TFBIG_MIX(w4, w5, rc2); \
		TFBIG_MIX(w6, w7, rc3); \
	}

#define TFBIG_4e(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}

/* uint64_t midstate for skein 80 */

#define TFBIG_ADDKEY_PRE(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + (s)); \
	}

#define TFBIG_MIX_PRE(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROTL64(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8_PRE(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX_PRE(w0, w1, rc0); \
		TFBIG_MIX_PRE(w2, w3, rc1); \
		TFBIG_MIX_PRE(w4, w5, rc2); \
		TFBIG_MIX_PRE(w6, w7, rc3); \
	}

#define TFBIG_4e_PRE(s) { \
		TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8_PRE(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8_PRE(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8_PRE(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o_PRE(s) { \
		TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8_PRE(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8_PRE(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8_PRE(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}

/* uint2 variant for SM3.2+ */

#define TFBIG_KINIT_UI2(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ vectorize(SPH_C64(0x1BD11BDAA9FC1A22)); \
		t2 = t0 ^ t1; \
	}

#define TFBIG_ADDKEY_UI2(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + vectorize(s)); \
	}

#define TFBIG_MIX_UI2(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROL2(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8_UI2(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX_UI2(w0, w1, rc0); \
		TFBIG_MIX_UI2(w2, w3, rc1); \
		TFBIG_MIX_UI2(w4, w5, rc2); \
		TFBIG_MIX_UI2(w6, w7, rc3); \
	}

#define TFBIG_4e_UI2(s)  { \
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8_UI2(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8_UI2(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8_UI2(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o_UI2(s)  { \
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8_UI2(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8_UI2(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8_UI2(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}


__global__
void quark_skein512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t * const __restrict__ g_hash, uint32_t *g_nonceVector)
{
#if !defined(SP_KERNEL) || __CUDA_ARCH__ < 500
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint2 h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint2 t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		uint32_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition * 8U];

		// Initialisierung
		h0 = vectorize(0x4903ADFF749C51CEull);
		h1 = vectorize(0x0D95DE399746DF03ull);
		h2 = vectorize(0x8FD1934127C79BCEull);
		h3 = vectorize(0x9A255629FF352CB1ull);
		h4 = vectorize(0x5DB62599DF6CA7B0ull);
		h5 = vectorize(0xEABE394CA9D5C3F4ull);
		h6 = vectorize(0x991112C71A75B523ull);
		h7 = vectorize(0xAE18A40B660FCC33ull);

		uint2 p[8];
		// 1st Round -> etype = 480, ptr = 64, bcount = 0, data = msg
		#pragma unroll 8
		for (int i = 0; i < 8; i++)
			p[i] = vectorize(inpHash[i]);

		t0 = vectorize(64); // ptr
		// t1 = vectorize(480ull << 55); // etype
		t1 = vectorize(0xf000000000000000ULL);

//#if CUDA_VERSION >= 7000
		// doesnt really affect x11 perfs.
		__threadfence();
//#endif
		TFBIG_KINIT_UI2(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e_UI2(0);
		TFBIG_4o_UI2(1);
		TFBIG_4e_UI2(2);
		TFBIG_4o_UI2(3);
		TFBIG_4e_UI2(4);
		TFBIG_4o_UI2(5);
		TFBIG_4e_UI2(6);
		TFBIG_4o_UI2(7);
		TFBIG_4e_UI2(8);
		TFBIG_4o_UI2(9);
		TFBIG_4e_UI2(10);
		TFBIG_4o_UI2(11);
		TFBIG_4e_UI2(12);
		TFBIG_4o_UI2(13);
		TFBIG_4e_UI2(14);
		TFBIG_4o_UI2(15);
		TFBIG_4e_UI2(16);
		TFBIG_4o_UI2(17);
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = vectorize(inpHash[0]) ^ p[0];
		h1 = vectorize(inpHash[1]) ^ p[1];
		h2 = vectorize(inpHash[2]) ^ p[2];
		h3 = vectorize(inpHash[3]) ^ p[3];
		h4 = vectorize(inpHash[4]) ^ p[4];
		h5 = vectorize(inpHash[5]) ^ p[5];
		h6 = vectorize(inpHash[6]) ^ p[6];
		h7 = vectorize(inpHash[7]) ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
		#pragma unroll 8
		for(int i=0; i<8; i++)
			p[i] = vectorize(0);

		t0 = vectorize(8); // ptr
		//t1 = vectorize(510ull << 55); // etype
		t1 = vectorize(0xff00000000000000ULL);

		TFBIG_KINIT_UI2(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e_UI2(0);
		TFBIG_4o_UI2(1);
		TFBIG_4e_UI2(2);
		TFBIG_4o_UI2(3);
		TFBIG_4e_UI2(4);
		TFBIG_4o_UI2(5);
		TFBIG_4e_UI2(6);
		TFBIG_4o_UI2(7);
		TFBIG_4e_UI2(8);
		TFBIG_4o_UI2(9);
		TFBIG_4e_UI2(10);
		TFBIG_4o_UI2(11);
		TFBIG_4e_UI2(12);
		TFBIG_4o_UI2(13);
		TFBIG_4e_UI2(14);
		TFBIG_4o_UI2(15);
		TFBIG_4e_UI2(16);
		TFBIG_4o_UI2(17);
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// output
		uint64_t *outpHash = &g_hash[hashPosition * 8U];
		#pragma unroll 8
		for(int i=0; i<8; i++)
			outpHash[i] = devectorize(p[i]);
	}
#endif /* SM < 5.0 */
}

__global__
void quark_skein512_gpu_hash_64_sm3(uint32_t threads, uint32_t startNounce, uint64_t * const __restrict__ g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint64_t p[8];
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		uint32_t hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[hashPosition * 8];

		// Init
		h0 = 0x4903ADFF749C51CEull;
		h1 = 0x0D95DE399746DF03ull;
		h2 = 0x8FD1934127C79BCEull;
		h3 = 0x9A255629FF352CB1ull;
		h4 = 0x5DB62599DF6CA7B0ull;
		h5 = 0xEABE394CA9D5C3F4ull;
		h6 = 0x991112C71A75B523ull;
		h7 = 0xAE18A40B660FCC33ull;

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg
		#pragma unroll 8
		for(int i=0; i<8; i++)
			p[i] = inpHash[i];

		t0 = 64; // ptr
		// t1 = 480ull << 55; // etype
		t1 = 0xf000000000000000ULL;

		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = inpHash[0] ^ p[0];
		h1 = inpHash[1] ^ p[1];
		h2 = inpHash[2] ^ p[2];
		h3 = inpHash[3] ^ p[3];
		h4 = inpHash[4] ^ p[4];
		h5 = inpHash[5] ^ p[5];
		h6 = inpHash[6] ^ p[6];
		h7 = inpHash[7] ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
		#pragma unroll 8
		for(int i=0; i<8; i++)
			p[i] = 0ull;

		t0 = 8; // ptr
		t1 = 510ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// output
		uint64_t *outpHash = &g_hash[hashPosition * 8];

		#pragma unroll 8
		for(int i=0; i<8; i++)
			outpHash[i] = p[i];
	}
}

__global__ __launch_bounds__(128,5)
void skein512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint64_t *output64, int swap)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint2 h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint2 t0, t1, t2;

		h0 = vectorize(c_PaddedMessage80[10]);
		h1 = vectorize(c_PaddedMessage80[11]);
		h2 = vectorize(c_PaddedMessage80[12]);
		h3 = vectorize(c_PaddedMessage80[13]);
		h4 = vectorize(c_PaddedMessage80[14]);
		h5 = vectorize(c_PaddedMessage80[15]);
		h6 = vectorize(c_PaddedMessage80[16]);
		h7 = vectorize(c_PaddedMessage80[17]);

		t2 = vectorize(c_PaddedMessage80[18]);

		uint32_t nonce = swap ? cuda_swab32(startNounce + thread) : startNounce + thread;
		uint2 nonce2 = make_uint2(_LODWORD(c_PaddedMessage80[9]), nonce);

		uint2 p[8];
		p[0] = vectorize(c_PaddedMessage80[8]);
		p[1] = nonce2;

		#pragma unroll
		for (int i = 2; i < 8; i++)
			p[i] = vectorize(0ull);

		t0 = vectorize(0x50ull);
		t1 = vectorize(0xB000000000000000ull);
		TFBIG_KINIT_UI2(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e_UI2(0);
		TFBIG_4o_UI2(1);
		TFBIG_4e_UI2(2);
		TFBIG_4o_UI2(3);
		TFBIG_4e_UI2(4);
		TFBIG_4o_UI2(5);
		TFBIG_4e_UI2(6);
		TFBIG_4o_UI2(7);
		TFBIG_4e_UI2(8);
		TFBIG_4o_UI2(9);
		TFBIG_4e_UI2(10);
		TFBIG_4o_UI2(11);
		TFBIG_4e_UI2(12);
		TFBIG_4o_UI2(13);
		TFBIG_4e_UI2(14);
		TFBIG_4o_UI2(15);
		TFBIG_4e_UI2(16);
		TFBIG_4o_UI2(17);
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		uint64_t *outpHash = &output64[thread * 8];
		outpHash[0] = c_PaddedMessage80[8] ^ devectorize(p[0]);
		outpHash[1] = devectorize(nonce2 ^ p[1]);
		#pragma unroll
		for(int i=2; i<8; i++)
			outpHash[i] = devectorize(p[i]);
	}
}

__global__
void skein512_gpu_hash_80_sm3(uint32_t threads, uint32_t startNounce, uint64_t *output64, int swap)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		// Init
		h0 = 0x4903ADFF749C51CEull;
		h1 = 0x0D95DE399746DF03ull;
		h2 = 0x8FD1934127C79BCEull;
		h3 = 0x9A255629FF352CB1ull;
		h4 = 0x5DB62599DF6CA7B0ull;
		h5 = 0xEABE394CA9D5C3F4ull;
		h6 = 0x991112C71A75B523ull;
		h7 = 0xAE18A40B660FCC33ull;

		t0 = 64; // ptr
		//t1 = vectorize(0xE0ull << 55); // etype
		t1 = 0x7000000000000000ull;
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);

		uint64_t p[8];
		#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = c_PaddedMessage80[i];

		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = c_PaddedMessage80[0] ^ p[0];
		h1 = c_PaddedMessage80[1] ^ p[1];
		h2 = c_PaddedMessage80[2] ^ p[2];
		h3 = c_PaddedMessage80[3] ^ p[3];
		h4 = c_PaddedMessage80[4] ^ p[4];
		h5 = c_PaddedMessage80[5] ^ p[5];
		h6 = c_PaddedMessage80[6] ^ p[6];
		h7 = c_PaddedMessage80[7] ^ p[7];

		uint32_t nonce = swap ? cuda_swab32(startNounce + thread) : startNounce + thread;
		uint64_t nonce64 = MAKE_ULONGLONG(_LODWORD(c_PaddedMessage80[9]), nonce);

		// skein_big_close -> etype = 0x160, ptr = 16, bcount = 1, extra = 16
		p[0] = c_PaddedMessage80[8];
		p[1] = nonce64;

		#pragma unroll
		for (int i = 2; i < 8; i++)
			p[i] = 0ull;

		t0 = 0x50ull; // SPH_T64(bcount << 6) + (sph_u64)(extra);
		t1 = 0xB000000000000000ull; // (bcount >> 58) + ((sph_u64)(etype) << 55);

		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// skein_big_close 2nd loop -> etype = 0x1fe, ptr = 8, bcount = 0
		// output
		uint64_t *outpHash = &output64[thread * 8];
		outpHash[0] = c_PaddedMessage80[8] ^ p[0];
		outpHash[1] = nonce64 ^ p[1];
		#pragma unroll
		for(int i=2; i<8; i++)
			outpHash[i] = p[i];
	}
}

__global__ __launch_bounds__(128,6)
void skein512_gpu_hash_close(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 t0 = vectorize(8); // extra
		uint2 t1 = vectorize(0xFF00000000000000ull); // etype
		uint2 t2 = vectorize(0xB000000000000050ull);

		uint64_t *state = &g_hash[thread * 8];
		uint2 h0 = vectorize(state[0]);
		uint2 h1 = vectorize(state[1]);
		uint2 h2 = vectorize(state[2]);
		uint2 h3 = vectorize(state[3]);
		uint2 h4 = vectorize(state[4]);
		uint2 h5 = vectorize(state[5]);
		uint2 h6 = vectorize(state[6]);
		uint2 h7 = vectorize(state[7]);
		uint2 h8;
		TFBIG_KINIT_UI2(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);

		uint2 p[8] = { 0 };

		TFBIG_4e_UI2(0);
		TFBIG_4o_UI2(1);
		TFBIG_4e_UI2(2);
		TFBIG_4o_UI2(3);
		TFBIG_4e_UI2(4);
		TFBIG_4o_UI2(5);
		TFBIG_4e_UI2(6);
		TFBIG_4o_UI2(7);
		TFBIG_4e_UI2(8);
		TFBIG_4o_UI2(9);
		TFBIG_4e_UI2(10);
		TFBIG_4o_UI2(11);
		TFBIG_4e_UI2(12);
		TFBIG_4o_UI2(13);
		TFBIG_4e_UI2(14);
		TFBIG_4o_UI2(15);
		TFBIG_4e_UI2(16);
		TFBIG_4o_UI2(17);
		TFBIG_ADDKEY_UI2(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		uint64_t *outpHash = state;
		#pragma unroll 8
		for (int i = 0; i < 8; i++)
			outpHash[i] = devectorize(p[i]);
	}
}

__global__ __launch_bounds__(128,6)
void skein512_gpu_hash_close_sm3(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t t0 = 8ull; // extra
		uint64_t t1 = 0xFF00000000000000ull; // etype
		uint64_t t2 = 0xB000000000000050ull;

		uint64_t *state = &g_hash[thread * 8];

		uint64_t h0 = state[0];
		uint64_t h1 = state[1];
		uint64_t h2 = state[2];
		uint64_t h3 = state[3];
		uint64_t h4 = state[4];
		uint64_t h5 = state[5];
		uint64_t h6 = state[6];
		uint64_t h7 = state[7];
		uint64_t h8;
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);

		uint64_t p[8] = { 0 };

		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		uint64_t *outpHash = state;
		#pragma unroll 8
		for (int i = 0; i < 8; i++)
			outpHash[i] = p[i];
	}
}

__host__
void quark_skein512_cpu_init(int thr_id, uint32_t threads)
{
	// store the binary SM version
	cuda_get_arch(thr_id);
}

__host__
void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];

	// uint2 uint64 variants for SM 3.2+
#ifdef SP_KERNEL
	if (device_sm[dev_id] >= 500 && cuda_arch[dev_id] >= 500)
		quark_skein512_cpu_hash_64(threads, startNounce, d_nonceVector, d_hash); /* sp.cuh */
	else
#endif
	if (device_sm[dev_id] > 300 && cuda_arch[dev_id] > 300)
		quark_skein512_gpu_hash_64 <<<grid, block>>> (threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	else
		quark_skein512_gpu_hash_64_sm3 <<<grid, block>>> (threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	MyStreamSynchronize(NULL, order, thr_id);
}

/* skein / skein2 */

__host__
static void skein512_precalc_80(uint64_t* message)
{
	uint64_t p[8];
	uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
	uint64_t t0, t1, t2;

	h0 = 0x4903ADFF749C51CEull;
	h1 = 0x0D95DE399746DF03ull;
	h2 = 0x8FD1934127C79BCEull;
	h3 = 0x9A255629FF352CB1ull;
	h4 = 0x5DB62599DF6CA7B0ull;
	h5 = 0xEABE394CA9D5C3F4ull;
	h6 = 0x991112C71A75B523ull;
	h7 = 0xAE18A40B660FCC33ull;
	// h8 = h0 ^ h1 ^ h2 ^ h3 ^ h4 ^ h5 ^ h6 ^ h7 ^ SPH_C64(0x1BD11BDAA9FC1A22);
	h8 = 0xcab2076d98173ec4ULL;

	t0 = 64; // ptr
	t1 = 0x7000000000000000ull;
	t2 = 0x7000000000000040ull;

	memcpy(&p[0], &message[0], 64);

	TFBIG_4e_PRE(0);
	TFBIG_4o_PRE(1);
	TFBIG_4e_PRE(2);
	TFBIG_4o_PRE(3);
	TFBIG_4e_PRE(4);
	TFBIG_4o_PRE(5);
	TFBIG_4e_PRE(6);
	TFBIG_4o_PRE(7);
	TFBIG_4e_PRE(8);
	TFBIG_4o_PRE(9);
	TFBIG_4e_PRE(10);
	TFBIG_4o_PRE(11);
	TFBIG_4e_PRE(12);
	TFBIG_4o_PRE(13);
	TFBIG_4e_PRE(14);
	TFBIG_4o_PRE(15);
	TFBIG_4e_PRE(16);
	TFBIG_4o_PRE(17);
	TFBIG_ADDKEY_PRE(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

	message[10] = message[0] ^ p[0];
	message[11] = message[1] ^ p[1];
	message[12] = message[2] ^ p[2];
	message[13] = message[3] ^ p[3];
	message[14] = message[4] ^ p[4];
	message[15] = message[5] ^ p[5];
	message[16] = message[6] ^ p[6];
	message[17] = message[7] ^ p[7];

	message[18] = t2;
}

__host__
void skein512_cpu_setBlock_80(void *pdata)
{
	uint64_t message[20];
	memcpy(&message[0], pdata, 80);
	skein512_precalc_80(message);
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), message, sizeof(message), 0, hipMemcpyHostToDevice);

	CUDA_SAFE_CALL(hipGetLastError());
}

__host__
void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *g_hash, int swap)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];
	uint64_t *d_hash = (uint64_t*) g_hash;

	if (device_sm[dev_id] > 300 && cuda_arch[dev_id] > 300) {
		// hash function is cut in 2 parts to reduce kernel size
		skein512_gpu_hash_80 <<< grid, block >>> (threads, startNounce, d_hash, swap);
		skein512_gpu_hash_close <<< grid, block >>> (threads, startNounce, d_hash);
	} else {
		// variant without uint2 variables
		skein512_gpu_hash_80_sm3 <<< grid, block >>> (threads, startNounce, d_hash, swap);
		skein512_gpu_hash_close_sm3 <<< grid, block >>> (threads, startNounce, d_hash);
	}
}
