#include "hip/hip_runtime.h"
/**
 * __shfl_up require SM 3.0 arch!
 *
 * SM 2 alternative method by tpruvot@github 2015
 */
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 300
#define __shfl_up(var, delta, width) (0)
#endif

static uint32_t *h_numValid[MAX_GPUS];
static uint32_t *d_tempBranch1Nonces[MAX_GPUS];
static uint32_t *d_partSum[2][MAX_GPUS]; // für bis zu vier partielle Summen

// True/False tester
typedef uint32_t(*cuda_compactTestFunction_t)(uint32_t *inpHash);

__device__ uint32_t QuarkTrueTest(uint32_t *inpHash)
{
	return ((inpHash[0] & 0x08) == 0x08);
}

__device__ uint32_t QuarkFalseTest(uint32_t *inpHash)
{
	return ((inpHash[0] & 0x08) == 0);
}

__device__ cuda_compactTestFunction_t d_QuarkTrueFunction = QuarkTrueTest, d_QuarkFalseFunction = QuarkFalseTest;

cuda_compactTestFunction_t h_QuarkTrueFunction[MAX_GPUS], h_QuarkFalseFunction[MAX_GPUS];

// Setup/Alloc Function
__host__
void quark_compactTest_cpu_init(int thr_id, uint32_t threads)
{
	int dev_id = device_map[thr_id];
	cuda_get_arch(thr_id);

	hipMemcpyFromSymbol(&h_QuarkTrueFunction[thr_id], HIP_SYMBOL(d_QuarkTrueFunction), sizeof(cuda_compactTestFunction_t));
	hipMemcpyFromSymbol(&h_QuarkFalseFunction[thr_id], HIP_SYMBOL(d_QuarkFalseFunction), sizeof(cuda_compactTestFunction_t));

	if (cuda_arch[dev_id] >= 300) {
		uint32_t s1 = (threads / 256) * 2;
		CUDA_SAFE_CALL(hipMalloc(&d_tempBranch1Nonces[thr_id], sizeof(uint32_t) * threads * 2));
		CUDA_SAFE_CALL(hipMalloc(&d_partSum[0][thr_id], sizeof(uint32_t) * s1)); // BLOCKSIZE (Threads/Block)
		CUDA_SAFE_CALL(hipMalloc(&d_partSum[1][thr_id], sizeof(uint32_t) * s1)); // BLOCKSIZE (Threads/Block)
	} else {
		CUDA_SAFE_CALL(hipMalloc(&d_tempBranch1Nonces[thr_id], sizeof(uint32_t) * threads));
	}

	hipHostMalloc(&h_numValid[thr_id], 2*sizeof(uint32_t));
}

// Because all alloc should have a free...
__host__
void quark_compactTest_cpu_free(int thr_id)
{
	int dev_id = device_map[thr_id];

	hipHostFree(h_numValid[thr_id]);

	if (cuda_arch[dev_id] >= 300) {
		hipFree(d_tempBranch1Nonces[thr_id]);
		hipFree(d_partSum[0][thr_id]);
		hipFree(d_partSum[1][thr_id]);
	} else {
		hipFree(d_tempBranch1Nonces[thr_id]);
	}
}

__global__
void quark_compactTest_gpu_SCAN(uint32_t *data, const int width, uint32_t *partial_sums=NULL, cuda_compactTestFunction_t testFunc=NULL,
	uint32_t threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
#if __CUDA_ARCH__ >= 300
	__shared__ uint32_t sums[32];
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	//int lane_id = id % warpSize;
	int lane_id = id % width;
	// determine a warp_id within a block
	 //int warp_id = threadIdx.x / warpSize;
	int warp_id = threadIdx.x / width;

	sums[lane_id] = 0;

	// Below is the basic structure of using a shfl instruction
	// for a scan.
	// Record "value" as a variable - we accumulate it along the way
	uint32_t value;
	if(testFunc != NULL)
	{
		if (id < threads)
		{
			uint32_t *inpHash;
			if(d_validNonceTable == NULL)
			{
				// keine Nonce-Liste
				inpHash = &inpHashes[id<<4];
			} else {
				// Nonce-Liste verfügbar
				int nonce = d_validNonceTable[id] - startNounce;
				inpHash = &inpHashes[nonce<<4];
			}			
			value = (*testFunc)(inpHash);
		} else {
			value = 0;
		}
	} else {
		value = data[id];
	}

	__syncthreads();

	// Now accumulate in log steps up the chain
	// compute sums, with another thread's value who is
	// distance delta away (i).  Note
	// those threads where the thread 'i' away would have
	// been out of bounds of the warp are unaffected.  This
	// creates the scan sum.

	#pragma unroll
	for (int i=1; i<=width; i*=2)
	{
		uint32_t n = __shfl_up((int)value, i, width);
		if (lane_id >= i) value += n;
	}

	// value now holds the scan value for the individual thread
	// next sum the largest values for each warp

	// write the sum of the warp to smem
	//if (threadIdx.x % warpSize == warpSize-1)
	if (threadIdx.x % width == width-1)
	{
		sums[warp_id] = value;
	}

	__syncthreads();

	//
	// scan sum the warp sums
	// the same shfl scan operation, but performed on warp sums
	//
	if (warp_id == 0)
	{
		uint32_t warp_sum = sums[lane_id];

		for (int i=1; i<=width; i*=2)
		{
			uint32_t n = __shfl_up((int)warp_sum, i, width);
			if (lane_id >= i) warp_sum += n;
		}

		sums[lane_id] = warp_sum;
	}

	__syncthreads();

	// perform a uniform add across warps in the block
	// read neighbouring warp's sum and add it to threads value
	uint32_t blockSum = 0;

	if (warp_id > 0)
	{
		blockSum = sums[warp_id-1];
	}

	value += blockSum;

	// Now write out our result
	data[id] = value;

	// last thread has sum, write write out the block's sum
	if (partial_sums != NULL && threadIdx.x == blockDim.x-1)
	{
		partial_sums[blockIdx.x] = value;
	}
#endif // SM3+
}

// Uniform add: add partial sums array
__global__
void quark_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
	__shared__ uint32_t buf;
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

	if (id > len) return;

	if (threadIdx.x == 0)
	{
		buf = partial_sums[blockIdx.x];
	}

	__syncthreads();
	data[id] += buf;
}

__global__
void quark_compactTest_gpu_SCATTER(uint32_t *sum, uint32_t *outp, cuda_compactTestFunction_t testFunc,
	uint32_t threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	uint32_t actNounce = id;
	uint32_t value;
	if (id < threads)
	{
		uint32_t *inpHash;
		if(d_validNonceTable == NULL)
		{
			// keine Nonce-Liste
			inpHash = &inpHashes[id<<4];
		} else {
			// Nonce-Liste verfügbar
			int nonce = d_validNonceTable[id] - startNounce;
			actNounce = nonce;
			inpHash = &inpHashes[nonce<<4];
		}

		value = (*testFunc)(inpHash);
	} else {
		value = 0;
	}

	if (value) {
		int idx = sum[id];
		if(idx > 0)
			outp[idx-1] = startNounce + actNounce;
	}
}

__host__ static uint32_t quark_compactTest_roundUpExp(uint32_t val)
{
	if(val == 0)
		return 0;

	uint32_t mask = 0x80000000;
	while( (val & mask) == 0 ) mask = mask >> 1;

	if( (val & (~mask)) != 0 )
		return mask << 1;

	return mask;
}

__host__
void quark_compactTest_cpu_singleCompaction(int thr_id, uint32_t threads, uint32_t *nrm,uint32_t *d_nonces1,
	cuda_compactTestFunction_t function, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	int orgThreads = threads;
	threads = quark_compactTest_roundUpExp(threads);
	// threadsPerBlock ausrechnen
	int blockSize = 256;
	int nSummen = threads / blockSize;

	int thr1 = (threads+blockSize-1) / blockSize;
	int thr2 = threads / (blockSize*blockSize);
	int blockSize2 = (nSummen < blockSize) ? nSummen : blockSize;
	int thr3 = (nSummen + blockSize2-1) / blockSize2;

	bool callThrid = (thr2 > 0) ? true : false;

	// Erster Initialscan
	quark_compactTest_gpu_SCAN <<<thr1,blockSize>>>(
		d_tempBranch1Nonces[thr_id], 32, d_partSum[0][thr_id], function, orgThreads, startNounce, inpHashes, d_validNonceTable);	

	// weitere Scans
	if(callThrid) {
		quark_compactTest_gpu_SCAN<<<thr2,blockSize>>>(d_partSum[0][thr_id], 32, d_partSum[1][thr_id]);
		quark_compactTest_gpu_SCAN<<<1, thr2>>>(d_partSum[1][thr_id], (thr2>32) ? 32 : thr2);
	} else {
		quark_compactTest_gpu_SCAN<<<thr3,blockSize2>>>(d_partSum[0][thr_id], (blockSize2>32) ? 32 : blockSize2);
	}

	// Sync + Anzahl merken
	hipStreamSynchronize(NULL);

	if(callThrid)
		hipMemcpy(nrm, &(d_partSum[1][thr_id])[thr2-1], sizeof(uint32_t), hipMemcpyDeviceToHost);
	else
		hipMemcpy(nrm, &(d_partSum[0][thr_id])[nSummen-1], sizeof(uint32_t), hipMemcpyDeviceToHost);

	if(callThrid) {
		quark_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum[0][thr_id]+blockSize, d_partSum[1][thr_id], blockSize*thr2);
	}
	quark_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch1Nonces[thr_id]+blockSize, d_partSum[0][thr_id], threads);

	quark_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch1Nonces[thr_id], d_nonces1, 
		function, orgThreads, startNounce, inpHashes, d_validNonceTable);

	// Sync
	hipStreamSynchronize(NULL);
}

#if __CUDA_ARCH__ < 300
__global__ __launch_bounds__(128, 8)
void quark_filter_gpu_sm2(const uint32_t threads, const uint32_t* d_hash, uint32_t* d_branch2, uint32_t* d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t offset = thread * 16U; // 64U / sizeof(uint32_t);
		uint4 *psrc = (uint4*) (&d_hash[offset]);
		d_NonceBranch[thread] = ((uint8_t*)psrc)[0] & 0x8;
		if (d_NonceBranch[thread]) return;
		// uint4 = 4x uint32_t = 16 bytes
		uint4 *pdst = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__global__ __launch_bounds__(128, 8)
void quark_merge_gpu_sm2(const uint32_t threads, uint32_t* d_hash, uint32_t* d_branch2, uint32_t* const d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads && !d_NonceBranch[thread])
	{
		const uint32_t offset = thread * 16U;
		uint4 *pdst = (uint4*) (&d_hash[offset]);
		uint4 *psrc = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}
#else
__global__ void quark_filter_gpu_sm2(const uint32_t threads, const uint32_t* d_hash, uint32_t* d_branch2, uint32_t* d_NonceBranch) {}
__global__ void quark_merge_gpu_sm2(const uint32_t threads, uint32_t* d_hash, uint32_t* d_branch2, uint32_t* const d_NonceBranch) {}
#endif

__host__
uint32_t quark_filter_cpu_sm2(const int thr_id, const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// extract algo permution hashes to a second branch buffer
	quark_filter_gpu_sm2 <<<grid, block>>> (threads, inpHashes, d_branch2, d_tempBranch1Nonces[thr_id]);
	return threads;
}

__host__
void quark_merge_cpu_sm2(const int thr_id, const uint32_t threads, uint32_t *outpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// put back second branch hashes to the common buffer d_hash
	quark_merge_gpu_sm2 <<<grid, block>>> (threads, outpHashes, d_branch2, d_tempBranch1Nonces[thr_id]);
}

////// ACHTUNG: Diese funktion geht aktuell nur mit threads > 65536 (Am besten 256 * 1024 oder 256*2048)
__host__
void quark_compactTest_cpu_dualCompaction(int thr_id, uint32_t threads, uint32_t *nrm, uint32_t *d_nonces1,
	 uint32_t *d_nonces2, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	quark_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[0], d_nonces1, h_QuarkTrueFunction[thr_id], startNounce, inpHashes, d_validNonceTable);
	quark_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[1], d_nonces2, h_QuarkFalseFunction[thr_id], startNounce, inpHashes, d_validNonceTable);
}

__host__
void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
	uint32_t *d_validNonceTable, uint32_t *d_nonces1, uint32_t *nrm1, uint32_t *d_nonces2, uint32_t *nrm2, int order)
{
	// Wenn validNonceTable genutzt wird, dann werden auch nur die Nonces betrachtet, die dort enthalten sind
	// "threads" ist in diesem Fall auf die Länge dieses Array's zu setzen!
	
	quark_compactTest_cpu_dualCompaction(thr_id, threads,
		h_numValid[thr_id], d_nonces1, d_nonces2,
		startNounce, inpHashes, d_validNonceTable);

	hipStreamSynchronize(NULL); // Das original braucht zwar etwas CPU-Last, ist an dieser Stelle aber evtl besser
	*nrm1 = h_numValid[thr_id][0];
	*nrm2 = h_numValid[thr_id][1];
}

__host__
void quark_compactTest_single_false_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
	uint32_t *d_validNonceTable, uint32_t *d_nonces1, uint32_t *nrm1, int order)
{
	// Wenn validNonceTable genutzt wird, dann werden auch nur die Nonces betrachtet, die dort enthalten sind
	// "threads" ist in diesem Fall auf die Länge dieses Array's zu setzen!

	quark_compactTest_cpu_singleCompaction(thr_id, threads, h_numValid[thr_id], d_nonces1, h_QuarkFalseFunction[thr_id], startNounce, inpHashes, d_validNonceTable);

	hipStreamSynchronize(NULL);
	*nrm1 = h_numValid[thr_id][0];
}