#include "hip/hip_runtime.h"
// SM 2.1 variant

// #include "cuda_helper.h"

#define MAXWELL_OR_FERMI 0
#define USE_SHARED 1

static unsigned int *d_textures[MAX_GPUS][8];

// #define SPH_C32(x)    ((uint32_t)(x ## U))
// #define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)

// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) tex1Dfetch(t3dn1, x)

texture<unsigned int, 1, hipReadModeElementType> t0up1;
texture<unsigned int, 1, hipReadModeElementType> t0dn1;
texture<unsigned int, 1, hipReadModeElementType> t1up1;
texture<unsigned int, 1, hipReadModeElementType> t1dn1;
texture<unsigned int, 1, hipReadModeElementType> t2up1;
texture<unsigned int, 1, hipReadModeElementType> t2dn1;
texture<unsigned int, 1, hipReadModeElementType> t3up1;
texture<unsigned int, 1, hipReadModeElementType> t3dn1;

extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

#if __CUDA_ARCH__ < 300 || defined(_DEBUG)

__device__ __forceinline__
void quark_groestl512_perm_P(uint32_t *a, char *mixtabs)
{
	uint32_t t[32];

	for(int r=0; r<14; r++)
	{
		switch(r) {
		case 0:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 0); break;
		case 1:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 1); break;
		case 2:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 2); break;
		case 3:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 3); break;
		case 4:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 4); break;
		case 5:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 5); break;
		case 6:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 6); break;
		case 7:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 7); break;
		case 8:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 8); break;
		case 9:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 9); break;
		case 10:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 10); break;
		case 11:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 11); break;
		case 12:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 12); break;
		case 13:
			#pragma unroll 16
			for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 13); break;
		}

		// RBTT
		#pragma unroll 16
		for(int k=0;k<32;k+=2) {
			uint32_t t0_0 = B32_0(a[(k     ) & 0x1f]), t9_0  = B32_0(a[(k +  9) & 0x1f]);
			uint32_t t2_1 = B32_1(a[(k +  2) & 0x1f]), t11_1 = B32_1(a[(k + 11) & 0x1f]);
			uint32_t t4_2 = B32_2(a[(k +  4) & 0x1f]), t13_2 = B32_2(a[(k + 13) & 0x1f]);
			uint32_t t6_3 = B32_3(a[(k +  6) & 0x1f]), t23_3 = B32_3(a[(k + 23) & 0x1f]);

			t[k + 0] =  T0up( t0_0 ) ^ T1up(  t2_1 ) ^ T2up(  t4_2 ) ^ T3up(  t6_3 ) ^
						T0dn( t9_0 ) ^ T1dn( t11_1 ) ^ T2dn( t13_2 ) ^ T3dn( t23_3 );

			t[k + 1] =  T0dn( t0_0 ) ^ T1dn(  t2_1 ) ^ T2dn(  t4_2 ) ^ T3dn(  t6_3 ) ^
						T0up( t9_0 ) ^ T1up( t11_1 ) ^ T2up( t13_2 ) ^ T3up( t23_3 );
		}

		#pragma unroll 32
		for(int k=0; k<32; k++) {
			a[k] = t[k];
		}
	}
}

__device__ __forceinline__
void quark_groestl512_perm_Q(uint32_t *a, char *mixtabs)
{
	for(int r=0; r<14; r++)
	{
		uint32_t t[32];

		switch(r) {
		case 0:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 0); a[(k*2)+1] ^= QC32dn(k<< 4, 0);} break;
		case 1:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 1); a[(k*2)+1] ^= QC32dn(k<< 4, 1);} break;
		case 2:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 2); a[(k*2)+1] ^= QC32dn(k<< 4, 2);} break;
		case 3:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 3); a[(k*2)+1] ^= QC32dn(k<< 4, 3);} break;
		case 4:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 4); a[(k*2)+1] ^= QC32dn(k<< 4, 4);} break;
		case 5:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 5); a[(k*2)+1] ^= QC32dn(k<< 4, 5);} break;
		case 6:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 6); a[(k*2)+1] ^= QC32dn(k<< 4, 6);} break;
		case 7:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 7); a[(k*2)+1] ^= QC32dn(k<< 4, 7);} break;
		case 8:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 8); a[(k*2)+1] ^= QC32dn(k<< 4, 8);} break;
		case 9:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 9); a[(k*2)+1] ^= QC32dn(k<< 4, 9);} break;
		case 10:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 10); a[(k*2)+1] ^= QC32dn(k<< 4, 10);} break;
		case 11:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 11); a[(k*2)+1] ^= QC32dn(k<< 4, 11);} break;
		case 12:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 12); a[(k*2)+1] ^= QC32dn(k<< 4, 12);} break;
		case 13:
			#pragma unroll 16
			for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 13); a[(k*2)+1] ^= QC32dn(k<< 4, 13);} break;
		}

		// RBTT
		#pragma unroll 16
		for(int k=0;k<32;k+=2)
		{
			uint32_t t2_0  = B32_0(a[(k +  2) & 0x1f]), t1_0  = B32_0(a[(k +  1) & 0x1f]);
			uint32_t t6_1  = B32_1(a[(k +  6) & 0x1f]), t5_1  = B32_1(a[(k +  5) & 0x1f]);
			uint32_t t10_2 = B32_2(a[(k + 10) & 0x1f]), t9_2  = B32_2(a[(k +  9) & 0x1f]);
			uint32_t t22_3 = B32_3(a[(k + 22) & 0x1f]), t13_3 = B32_3(a[(k + 13) & 0x1f]);

			t[k + 0] =  T0up( t2_0 ) ^ T1up( t6_1 ) ^ T2up( t10_2 ) ^ T3up( t22_3 ) ^
						T0dn( t1_0 ) ^ T1dn( t5_1 ) ^ T2dn(  t9_2 ) ^ T3dn( t13_3 );

			t[k + 1] =  T0dn( t2_0 ) ^ T1dn( t6_1 ) ^ T2dn( t10_2 ) ^ T3dn( t22_3 ) ^
						T0up( t1_0 ) ^ T1up( t5_1 ) ^ T2up(  t9_2 ) ^ T3up( t13_3 );
		}
		#pragma unroll 32
		for(int k=0;k<32;k++)
			a[k] = t[k];
	}
}

#endif

__global__
void quark_groestl512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *g_nonceVector)
{
#if __CUDA_ARCH__ < 300 || defined(_DEBUG)
	extern __shared__ char mixtabs[];

	if (threadIdx.x < 256)
	{
		*((uint32_t*)mixtabs + (    threadIdx.x)) = tex1Dfetch(t0up1, threadIdx.x);
		*((uint32_t*)mixtabs + (256+threadIdx.x)) = tex1Dfetch(t0dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (512+threadIdx.x)) = tex1Dfetch(t1up1, threadIdx.x);
		*((uint32_t*)mixtabs + (768+threadIdx.x)) = tex1Dfetch(t1dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1024+threadIdx.x)) = tex1Dfetch(t2up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1280+threadIdx.x)) = tex1Dfetch(t2dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1536+threadIdx.x)) = tex1Dfetch(t3up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1792+threadIdx.x)) = tex1Dfetch(t3dn1, threadIdx.x);
	}

	__syncthreads();

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// GROESTL
		uint32_t message[32];
		uint32_t state[32];

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		off_t hashPosition = nounce - startNounce;
		uint32_t *inpHash = &g_hash[hashPosition * 16];

		#pragma unroll 16
		for(int k=0; k<16; k++)
			message[k] = inpHash[k];

		#pragma unroll 14
		for(int k=1; k<15; k++)
			message[k+16] = 0;

		message[16] = 0x80;
		message[31] = 0x01000000;

		#pragma unroll 32
		for(int u=0; u<32; u++)
			state[u] = message[u];
		state[31] ^= 0x20000;

		// Perm
		quark_groestl512_perm_P(state, mixtabs);
		state[31] ^= 0x20000;
		quark_groestl512_perm_Q(message, mixtabs);

		#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

		#pragma unroll 32
		for(int u=0;u<32;u++) message[u] = state[u];

		quark_groestl512_perm_P(message, mixtabs);

		#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

		// Erzeugten Hash rausschreiben
		uint32_t *outpHash = &g_hash[hashPosition * 16];

		#pragma unroll 16
		for(int k=0;k<16;k++) outpHash[k] = state[k+16];
	}
#endif
}

#define texDef(id, texname, texmem, texsource, texsize) { \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	d_textures[thr_id][id] = texmem; \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); \
	} \
}

__host__
void quark_groestl512_sm20_init(int thr_id, uint32_t threads)
{
	// Texturen mit obigem Makro initialisieren
	texDef(0, t0up1, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
	texDef(1, t0dn1, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
	texDef(2, t1up1, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
	texDef(3, t1dn1, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
	texDef(4, t2up1, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
	texDef(5, t2dn1, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
	texDef(6, t3up1, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
	texDef(7, t3dn1, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);
}

__host__
void quark_groestl512_sm20_free(int thr_id)
{
	for (int i=0; i<8; i++)
		hipFree(d_textures[thr_id][i]);
}

__host__
void quark_groestl512_sm20_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	int threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 8 * 256 * sizeof(uint32_t);

	quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

	// MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void quark_doublegroestl512_sm20_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	int threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 8 * 256 * sizeof(uint32_t);

	quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);
	quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

	// MyStreamSynchronize(NULL, order, thr_id);
}
