#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "cuda_quark.h"

#include <stdio.h>

extern uint32_t quark_filter_cpu_sm2(const int thr_id, const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_branch2);
extern void quark_merge_cpu_sm2(const int thr_id, const uint32_t threads, uint32_t *outpHashes, uint32_t* d_branch2);

static uint32_t *d_hash[MAX_GPUS];
static uint32_t* d_hash_br2[MAX_GPUS];  // SM 2

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_branch1Nonces[MAX_GPUS];
static uint32_t *d_branch2Nonces[MAX_GPUS];
static uint32_t *d_branch3Nonces[MAX_GPUS];

// Original Quarkhash Funktion aus einem miner Quelltext
extern "C" void quarkhash(void *state, const void *input)
{
	unsigned char _ALIGN(128) hash[64];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
		sph_groestl512_close(&ctx_groestl, (void*) hash);
	}
	else
	{
		sph_skein512_init(&ctx_skein);
		sph_skein512 (&ctx_skein, (const void*) hash, 64);
		sph_skein512_close(&ctx_skein, (void*) hash);
	}

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_blake512_init(&ctx_blake);
		sph_blake512 (&ctx_blake, (const void*) hash, 64);
		sph_blake512_close(&ctx_blake, (void*) hash);
	}
	else
	{
		sph_bmw512_init(&ctx_bmw);
		sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
		sph_bmw512_close(&ctx_bmw, (void*) hash);
	}

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
		sph_keccak512_close(&ctx_keccak, (void*) hash);
	}
	else
	{
		sph_jh512_init(&ctx_jh);
		sph_jh512 (&ctx_jh, (const void*) hash, 64);
		sph_jh512_close(&ctx_jh, (void*) hash);
	}

	memcpy(state, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("quark %s %08x %08x %08x %08x...%08x... \n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_quark(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	int dev_id = device_map[thr_id];
	uint32_t def_thr = 1U << 20; // 256*4096
	uint32_t throughput = cuda_default_throughput(thr_id, def_thr);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x00F;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		hipGetLastError();
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_compactTest_cpu_init(thr_id, throughput);

		if (cuda_arch[dev_id] >= 300) {
			hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput);
		} else {
			hipMalloc(&d_hash_br2[thr_id], (size_t) 64 * throughput);
		}

		cuda_check_cpu_init(thr_id, throughput);
		CUDA_SAFE_CALL(hipGetLastError());

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t nrm1=0, nrm2=0, nrm3=0;

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");

		if (cuda_arch[dev_id] >= 300) {

			quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch3Nonces[thr_id], &nrm3, order++);

			// nur den Skein Branch weiterverfolgen
			quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Groestl512
			quark_groestl512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für JH512
			quark_jh512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
				d_branch1Nonces[thr_id], &nrm1,
				d_branch2Nonces[thr_id], &nrm2,
				order++);

			// das ist der bedingte Branch für Blake512
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der bedingte Branch für Bmw512
			quark_bmw512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Keccak512
			quark_keccak512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Skein512
			quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
				d_branch1Nonces[thr_id], &nrm1,
				d_branch2Nonces[thr_id], &nrm2,
				order++);

			quark_keccak512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

			work->nonces[0] = cuda_check_hash_branch(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);
			work->nonces[1] = 0;
		} else {
			/* algo permutations are made with 2 different buffers */

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm1  :");

			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("groestl:");
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("jh512  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm2  :");

			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("keccak :");
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("skein  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm3  :");

			CUDA_LOG_ERROR();
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			quarkhash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					quarkhash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	return 0;
}

// cleanup
extern "C" void free_quark(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	if (cuda_arch[dev_id] >= 300) {
		hipFree(d_branch1Nonces[thr_id]);
		hipFree(d_branch2Nonces[thr_id]);
		hipFree(d_branch3Nonces[thr_id]);
	} else {
		hipFree(d_hash_br2[thr_id]);
	}

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	quark_compactTest_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
