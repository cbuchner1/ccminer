#include "hip/hip_runtime.h"
// originally from djm34 - github.com/djm34/ccminer-sp-neoscrypt
// kernel code from Nanashi Meiyo-Meijin 1.7.6-r10 (July 2016)

#include <stdio.h>
#include <memory.h>

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>
#include "cuda_vectors.h"

typedef uint48 uint4x2;

#include "miner.h"

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#define __byte_perm(x,y,c) x
#define __shfl(x,y,c) x
#define atomicExch(p,x) x
#endif

static uint32_t* d_NNonce[MAX_GPUS];

__device__ uint2x4* W;
__device__ uint2x4* Tr;
__device__ uint2x4* Tr2;
__device__ uint2x4* Input;

__constant__ uint32_t c_data[64];
__constant__ uint32_t c_target[2];
__constant__ uint32_t key_init[16];
__constant__ uint32_t input_init[16];

static const __constant__ uint8 BLAKE2S_IV_Vec = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint8 BLAKE2S_IV_Vechost = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

__constant__ uint32_t BLAKE2S_SIGMA[10][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

#define BLOCK_SIZE         64U
#define BLAKE2S_BLOCK_SIZE 64U
#define BLAKE2S_OUT_SIZE   32U

#define SALSA(a,b,c,d) { \
	t = rotateL(a+d,  7U); b ^= t; \
	t = rotateL(b+a,  9U); c ^= t; \
	t = rotateL(c+b, 13U); d ^= t; \
	t = rotateL(d+c, 18U); a ^= t; \
}

#define shf_r_clamp32(out,a,b,shift) \
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(out) : "r"(a), "r"(b), "r"(shift));

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl(a, b, c);
}

__device__ __forceinline__ void WarpShuffle3(uint32_t &a1, uint32_t &a2, uint32_t &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	__shared__ uint32_t shared_mem[32];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	shared_mem[thread] = a;
	__threadfence_block();

	uint32_t result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];
	__threadfence_block();

	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint32_t &a1, uint32_t &a2, uint32_t &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	__shared__ uint32_t shared_mem[32];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	shared_mem[thread] = a1;
	__threadfence_block();

	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();

	shared_mem[thread] = a2;
	__threadfence_block();

	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();

	shared_mem[thread] = a3;
	__threadfence_block();

	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];
	__threadfence_block();
}

#endif

#define CHACHA_STEP(a,b,c,d) { \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateL(b^c, 12); \
	a += b; d = __byte_perm(d^a, 0, 0x2103); \
	c += d; b = rotateL(b^c, 7); \
}

#if __CUDA_ARCH__ < 500

__device__ __forceinline__
static void shift256R4(uint32_t* ret, const uint8 &vec4, const uint32_t shift2)
{
#if __CUDA_ARCH__ >= 320
	uint32_t shift = 32U - shift2;
	asm("shf.r.clamp.b32 %0, 0, %1, %2;" : "=r"(ret[0]) : "r"(vec4.s0), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[1]) : "r"(vec4.s0), "r"(vec4.s1), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[2]) : "r"(vec4.s1), "r"(vec4.s2), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[3]) : "r"(vec4.s2), "r"(vec4.s3), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[4]) : "r"(vec4.s3), "r"(vec4.s4), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[5]) : "r"(vec4.s4), "r"(vec4.s5), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[6]) : "r"(vec4.s5), "r"(vec4.s6), "r"(shift));
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(ret[7]) : "r"(vec4.s6), "r"(vec4.s7), "r"(shift));
	asm("shr.b32         %0, %1, %2;"     : "=r"(ret[8]) : "r"(vec4.s7), "r"(shift));
#else
	// to check
	shift256R(ret, vec4, shift2);
#endif
}

#define BLAKE(a, b, c, d, key1, key2) { \
	a += key1; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += key2; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE0(idx0, idx1, a, b, c, d, key) { \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE1(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE2(idx0, idx1, a, b, c, d, key) { \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}

static __forceinline__ __device__
void Blake2S(uint32_t *out, const uint32_t* const __restrict__  inout, const  uint32_t * const __restrict__ TheKey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi ^ tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	for (uint32_t x = 4U; x < 10U; x++)
	{
		BLAKE_G(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)out)[0] = V.lo;
}
#endif

#if __CUDA_ARCH__ >= 500

#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE(a, b, c, d, key1,key2) { \
	a += key1; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += key2; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE(idx0,idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE0(idx0,idx1, a, b, c, d, key) { \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE1(idx0,idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE2(idx0,idx1, a, b, c, d, key) { \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

static __forceinline__ __device__
void Blake2S_v2(uint32_t *out, const uint32_t* __restrict__  inout, const  uint32_t * __restrict__ TheKey)
{
	uint16 V;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[9], inout[0]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[5], inout[7]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[2], inout[4]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[10], inout[15]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[14], inout[1]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[11], inout[12]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[6], inout[8]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[3], inout[13]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[2], inout[12]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[6], inout[10]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[0], inout[11]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[8], inout[3]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[4], inout[13]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[7], inout[5]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[15], inout[14]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[1], inout[9]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[12], inout[5]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[1], inout[15]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[14], inout[13]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[4], inout[10]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[0], inout[7]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[6], inout[3]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[9], inout[2]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[8], inout[11]);
	// 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[13], inout[11]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[7], inout[14]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[12], inout[1]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[3], inout[9]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[5], inout[0]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[15], inout[4]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[8], inout[6]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[2], inout[10]);
	// 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[6], inout[15]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[14], inout[9]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[11], inout[3]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[0], inout[8]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[12], inout[2]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[13], inout[7]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[1], inout[4]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[10], inout[5]);
	// 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[10], inout[2]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[8], inout[4]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[7], inout[6]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[1], inout[5]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[15], inout[11]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[9], inout[14]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[3], inout[12]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[13], inout[0]);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	((uint8*)out)[0] = V.lo;
}

#endif /* __CUDA_ARCH__ >= 500 */

#define SALSA_CORE(state) { \
	uint32_t t; \
	SALSA(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w, threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1,4); \
	SALSA(state.x, state.w, state.z, state.y); \
	WarpShuffle3(state.y, state.z, state.w, threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3,4); \
}

#define CHACHA_CORE_PARALLEL(state)	{ \
	CHACHA_STEP(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w, threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3,4); \
	CHACHA_STEP(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w, threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1,4); \
}

__forceinline__ __device__
uint4 salsa_small_scalar_rnd(const uint4 X)
{
	uint4 state = X;

	#pragma nounroll
	for (int i = 0; i < 10; i++) {
		SALSA_CORE(state);
	}

	return (X + state);
}

__device__ __forceinline__
uint4 chacha_small_parallel_rnd(const uint4 X)
{
	uint4 state = X;

	#pragma nounroll
	for (int i = 0; i < 10; i++) {
		CHACHA_CORE_PARALLEL(state);
	}
	return (X + state);
}

__device__ __forceinline__
void neoscrypt_chacha(uint4 XV[4])
{
	uint4 temp;

	XV[0] = chacha_small_parallel_rnd(XV[0] ^ XV[3]);
	temp = chacha_small_parallel_rnd(XV[1] ^ XV[0]);
	XV[1] = chacha_small_parallel_rnd(XV[2] ^ temp);
	XV[3] = chacha_small_parallel_rnd(XV[3] ^ XV[1]);
	XV[2] = temp;
}

__device__ __forceinline__
void neoscrypt_salsa(uint4 XV[4])
{
	uint4 temp;

	XV[0] = salsa_small_scalar_rnd(XV[0] ^ XV[3]);
	temp = salsa_small_scalar_rnd(XV[1] ^ XV[0]);
	XV[1] = salsa_small_scalar_rnd(XV[2] ^ temp);
	XV[3] = salsa_small_scalar_rnd(XV[3] ^ XV[1]);
	XV[2] = temp;
}


#if __CUDA_ARCH__ < 500
static __forceinline__ __device__
void fastkdf256_v1(const uint32_t thread, const uint32_t nonce, uint32_t* const s_data)
{
	uint2x4 output[8];
	uint32_t* B = (uint32_t*)&s_data[threadIdx.x * 64U];
	uint32_t qbuf, rbuf, bitbuf;
	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = { 0 };

	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	((uintx64*)(B))[0] = ((uintx64*)c_data)[0];
	((uint32_t*)B)[19] = nonce;
	((uint32_t*)B)[39] = nonce;
	((uint32_t*)B)[59] = nonce;

	((uint816*)input)[0] = ((uint816*)input_init)[0];
	((uint4x2*)key)[0] = ((uint4x2*)key_init)[0];

	#pragma unroll 1
	for (int i = 0; i < 31; i++)
	{
		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t shifted[9];
		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		uint32_t temp[9];
		//#pragma unroll
		for (int k = 0; k < 9; k++)
		{
			uint32_t indice = (k + qbuf) & 0x3f;
			temp[k] = B[indice] ^ shifted[k];
			B[indice] = temp[k];
		}
#if __CUDA_ARCH__ >= 320  || !defined(__CUDA_ARCH__)
		uint32_t a = c_data[qbuf & 0x3f], b;
		//#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = c_data[(qbuf + k + 1) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = c_data[(qbuf + k + 2) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}

		const uint32_t noncepos = 19U - qbuf % 20U;
		if (noncepos <= 16U && qbuf < 60U)
		{
			if (noncepos != 0)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16U)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		for (int k = 0; k<8; k++)
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[k]) : "r"(temp[k]), "r"(temp[k + 1]), "r"(bitbuf));
#else
		//#error SM 3.0 code missing here
		printf("", data18, data20);
#endif
		Blake2S(input, input, key);
	}

	uint32_t bufidx = 0;
	#pragma unroll
	for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

#if __CUDA_ARCH__ >= 320
	for (int i = 0; i<64; i++)
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(((uint32_t*)output)[i]) : "r"(B[(qbuf + i) & 0x3f]), "r"(B[(qbuf + i + 1) & 0x3f4]), "r"(bitbuf));
#endif

	((ulonglong4*)output)[0] ^= ((ulonglong4*)input)[0];
	((uintx64*)output)[0] ^= ((uintx64*)c_data)[0];
	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;

	for (int i = 0; i<8; i++)
		(Input + 8U * thread)[i] = output[i];
}
#endif

#if __CUDA_ARCH__ >= 500
static __forceinline__ __device__
void fastkdf256_v2(const uint32_t thread, const uint32_t nonce, uint32_t* const s_data)
{
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];
	uint32_t input[16];
	uint32_t key[16] = { 0 };
	uint32_t qbuf, rbuf, bitbuf;

	uint32_t* B = (uint32_t*)&s_data[threadIdx.x * 64U];
	((uintx64*)(B))[0] = ((uintx64*)c_data)[0];

	B[19] = nonce;
	B[39] = nonce;
	B[59] = nonce;

	{
		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input_init[x] & 0x00ff00ff) + ((input_init[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t temp[9];

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		asm("shl.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input_init[0]), "r"(bitbuf));
		temp[0] = B[(0 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[0]), "r"(input_init[1]), "r"(shift));
		temp[1] = B[(1 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[1]), "r"(input_init[2]), "r"(shift));
		temp[2] = B[(2 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[2]), "r"(input_init[3]), "r"(shift));
		temp[3] = B[(3 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[3]), "r"(input_init[4]), "r"(shift));
		temp[4] = B[(4 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[4]), "r"(input_init[5]), "r"(shift));
		temp[5] = B[(5 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[5]), "r"(input_init[6]), "r"(shift));
		temp[6] = B[(6 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input_init[6]), "r"(input_init[7]), "r"(shift));
		temp[7] = B[(7 + qbuf) & 0x3f] ^ shifted;
		asm("shr.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input_init[7]), "r"(shift));
		temp[8] = B[(8 + qbuf) & 0x3f] ^ shifted;

		uint32_t a = c_data[qbuf & 0x3f], b;

		#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = c_data[(qbuf + k + 1) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = c_data[(qbuf + k + 2) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}

		const uint32_t noncepos = 19 - qbuf % 20U;
		if (noncepos <= 16U && qbuf < 60U)
		{
			if (noncepos)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16U)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));

		Blake2S_v2(input, input, key);

		#pragma unroll
		for (int k = 0; k < 9; k++)
			B[(k + qbuf) & 0x3f] = temp[k];
	}

	for (int i = 1; i < 31; i++)
	{
		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t temp[9];

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		asm("shl.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input[0]), "r"(bitbuf));
		temp[0] = B[(0 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[0]), "r"(input[1]), "r"(shift));
		temp[1] = B[(1 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[1]), "r"(input[2]), "r"(shift));
		temp[2] = B[(2 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[2]), "r"(input[3]), "r"(shift));
		temp[3] = B[(3 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[3]), "r"(input[4]), "r"(shift));
		temp[4] = B[(4 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[4]), "r"(input[5]), "r"(shift));
		temp[5] = B[(5 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[5]), "r"(input[6]), "r"(shift));
		temp[6] = B[(6 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[6]), "r"(input[7]), "r"(shift));
		temp[7] = B[(7 + qbuf) & 0x3f] ^ shifted;
		asm("shr.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input[7]), "r"(shift));
		temp[8] = B[(8 + qbuf) & 0x3f] ^ shifted;

		uint32_t a = c_data[qbuf & 0x3f], b;

		#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = c_data[(qbuf + k + 1) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = c_data[(qbuf + k + 2) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}

		const uint32_t noncepos = 19 - qbuf % 20U;
		if (noncepos <= 16U && qbuf < 60U)
		{
			if (noncepos)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16U)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));

		Blake2S_v2(input, input, key);

		#pragma unroll
		for (int k = 0; k < 9; k++)
			B[(k + qbuf) & 0x3f] = temp[k];
	}

	{
		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
	}

	uint2x4 output[8];
	for (int i = 0; i<64; i++) {
		const uint32_t a = (qbuf + i) & 0x3f, b = (qbuf + i + 1) & 0x3f;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(((uint32_t*)output)[i]) : "r"(B[a]), "r"(B[b]), "r"(bitbuf));
	}

	output[0] ^= ((uint2x4*)input)[0];
	#pragma unroll
	for (int i = 0; i<8; i++)
		output[i] ^= ((uint2x4*)c_data)[i];

	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;;
	((ulonglong16 *)(Input + 8U * thread))[0] = ((ulonglong16*)output)[0];
}
#endif

#if __CUDA_ARCH__ < 500
static __forceinline__ __device__
uint32_t fastkdf32_v1(uint32_t thread, const uint32_t nonce, uint32_t* const salt, uint32_t* const s_data)
{
	const uint32_t cdata7 = c_data[7];
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	uint32_t* B0 = (uint32_t*)&s_data[threadIdx.x * 64U];
	((uintx64*)B0)[0] = ((uintx64*)salt)[0];

	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	((uint816*)input)[0] = ((uint816*)c_data)[0];

	uint32_t key[BLAKE2S_BLOCK_SIZE / 4];
	((uint4x2*)key)[0] = ((uint4x2*)salt)[0];
	((uint4*)key)[2] = make_uint4(0, 0, 0, 0);
	((uint4*)key)[3] = make_uint4(0, 0, 0, 0);

	uint32_t qbuf, rbuf, bitbuf;
	uint32_t temp[9];

	#pragma nounroll
	for (int i = 0; i < 31; i++)
	{
		Blake2S(input, input, key);

		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; k++) {
			temp[k] = B0[(k + qbuf) & 0x3f];
		}

		((uint2x4*)temp)[0] ^= ((uint2x4*)shifted)[0];
		temp[8] ^= shifted[8];

#if __CUDA_ARCH__ >= 320 || !defined(__CUDA_ARCH__)
		uint32_t a = c_data[qbuf & 0x3f], b;
		//#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = c_data[(qbuf + k + 1) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = c_data[(qbuf + k + 2) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}

		const uint32_t noncepos = 19U - qbuf % 20U;
		if (noncepos <= 16U && qbuf < 60U)
		{
			if (noncepos != 0)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16U)	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));
#else
		//#error SM 3.0 code missing here
		printf("", data18, data20);
#endif
		for (int k = 0; k < 9; k++) {
			B0[(k + qbuf) & 0x3f] = temp[k];
		}
	}

	Blake2S(input, input, key);

	uint32_t bufidx = 0;
	#pragma unroll
	for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	for (int k = 7; k < 9; k++) {
		temp[k] = B0[(k + qbuf) & 0x3f];
	}

	uint32_t output;
#if __CUDA_ARCH__ >= 320
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(output) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));
#else
	output = (MAKE_ULONGLONG(temp[7], temp[8]) >> bitbuf); // to check maybe 7/8 reversed
#endif
	output ^= input[7] ^ cdata7;
	return output;
}
#endif

#if __CUDA_ARCH__ >= 500
static __forceinline__ __device__
uint32_t fastkdf32_v3(uint32_t thread, const uint32_t nonce, uint32_t* const salt, uint32_t* const s_data)
{
	const uint32_t cdata7 = c_data[7];
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	uint32_t* B0 = (uint32_t*)&s_data[threadIdx.x * 64U];
	((uintx64*)B0)[0] = ((uintx64*)salt)[0];

	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	((uint816*)input)[0] = ((uint816*)c_data)[0];

	uint32_t key[BLAKE2S_BLOCK_SIZE / 4];
	((uint4x2*)key)[0] = ((uint4x2*)salt)[0];
	((uint4*)key)[2] = make_uint4(0, 0, 0, 0);
	((uint4*)key)[3] = make_uint4(0, 0, 0, 0);

	uint32_t qbuf, rbuf, bitbuf;
	uint32_t temp[9];

	#pragma nounroll
	for (int i = 0; i < 31; i++)
	{
		Blake2S_v2(input, input, key);

		uint32_t bufidx = 0;
		#pragma unroll
		for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		asm("shl.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input[0]), "r"(bitbuf));
		temp[0] = B0[(0 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[0]), "r"(input[1]), "r"(shift));
		temp[1] = B0[(1 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[1]), "r"(input[2]), "r"(shift));
		temp[2] = B0[(2 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[2]), "r"(input[3]), "r"(shift));
		temp[3] = B0[(3 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[3]), "r"(input[4]), "r"(shift));
		temp[4] = B0[(4 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[4]), "r"(input[5]), "r"(shift));
		temp[5] = B0[(5 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[5]), "r"(input[6]), "r"(shift));
		temp[6] = B0[(6 + qbuf) & 0x3f] ^ shifted;
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(shifted) : "r"(input[6]), "r"(input[7]), "r"(shift));
		temp[7] = B0[(7 + qbuf) & 0x3f] ^ shifted;
		asm("shr.b32         %0, %1, %2;"     : "=r"(shifted) : "r"(input[7]), "r"(shift));
		temp[8] = B0[(8 + qbuf) & 0x3f] ^ shifted;

		uint32_t a = c_data[qbuf & 0x3f], b;
		#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = c_data[(qbuf + k + 1) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k]) : "r"(a), "r"(b), "r"(bitbuf));
			a = c_data[(qbuf + k + 2) & 0x3f];
			asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[k + 1]) : "r"(b), "r"(a), "r"(bitbuf));
		}

		const uint32_t noncepos = 19U - qbuf % 20U;
		if (noncepos <= 16U && qbuf < 60U)
		{
			if (noncepos != 0)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos - 1]) : "r"(data18), "r"(nonce), "r"(bitbuf));
			if (noncepos != 16U)
				asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(input[noncepos]) : "r"(nonce), "r"(data20), "r"(bitbuf));
		}

		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[0]) : "r"(temp[0]), "r"(temp[1]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[1]) : "r"(temp[1]), "r"(temp[2]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[2]) : "r"(temp[2]), "r"(temp[3]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[3]) : "r"(temp[3]), "r"(temp[4]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[4]) : "r"(temp[4]), "r"(temp[5]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[5]) : "r"(temp[5]), "r"(temp[6]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[6]) : "r"(temp[6]), "r"(temp[7]), "r"(bitbuf));
		asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(key[7]) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));

		#pragma unroll
		for (int k = 0; k < 9; k++) {
			B0[(k + qbuf) & 0x3f] = temp[k];
		}
	}

	Blake2S_v2(input, input, key);

	uint32_t bufidx = 0;
	#pragma unroll
	for (int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	temp[7] = B0[(qbuf + 7) & 0x3f];
	temp[8] = B0[(qbuf + 8) & 0x3f];

	uint32_t output;
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(output) : "r"(temp[7]), "r"(temp[8]), "r"(bitbuf));
	output ^= input[7] ^ cdata7;
	return output;
}
#endif


#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,16); \
	c += d; b = ROTR32(b^c, 12); \
	idx = BLAKE2S_SIGMA_host[idx0][idx1+1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,8); \
	c += d; b = ROTR32(b^c, 7); \
}

static void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}


#define SHIFT 128U
#define TPB 32
#define TPB2 64

__global__
__launch_bounds__(TPB2, 1)
void neoscrypt_gpu_hash_start(const int stratum, const uint32_t startNonce)
{
	__shared__ uint32_t s_data[64 * TPB2];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t nonce = startNonce + thread;
	const uint32_t ZNonce = (stratum) ? cuda_swab32(nonce) : nonce; //freaking morons !!!

	__syncthreads();
#if __CUDA_ARCH__ < 500
	fastkdf256_v1(thread, ZNonce, s_data);
#else
	fastkdf256_v2(thread, ZNonce, s_data);
#endif
}

__global__
__launch_bounds__(TPB, 1)
void neoscrypt_gpu_hash_chacha1()
{
	const uint32_t thread = (blockDim.y * blockIdx.x + threadIdx.y);
	const uint32_t shift = SHIFT * 8U * (thread & 8191);
	const uint32_t shiftTr = 8U * thread;

	uint4 X[4];
	for (int i = 0; i < 4; i++)
	{
		X[i].x = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 0 * 4 + threadIdx.x);
		X[i].y = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 1 * 4 + threadIdx.x);
		X[i].z = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 2 * 4 + threadIdx.x);
		X[i].w = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 3 * 4 + threadIdx.x);
	}

	#pragma nounroll
	for (int i = 0; i < 128; i++)
	{
		uint32_t offset = shift + i * 8U;
		for (int j = 0; j < 4; j++)
			((uint4*)(W + offset))[j * 4 + threadIdx.x] = X[j];
		neoscrypt_chacha(X);
	}

	#pragma nounroll
	for (int t = 0; t < 128; t++)
	{
		uint32_t offset = shift + (WarpShuffle(X[3].x, 0, 4) & 0x7F) * 8U;
		for (int j = 0; j < 4; j++)
			X[j] ^= ((uint4*)(W + offset))[j * 4 + threadIdx.x];
		neoscrypt_chacha(X);
	}

	#pragma unroll
	for (int i = 0; i < 4; i++)
	{
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 0 * 4 + threadIdx.x) = X[i].x;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 1 * 4 + threadIdx.x) = X[i].y;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 2 * 4 + threadIdx.x) = X[i].z;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 3 * 4 + threadIdx.x) = X[i].w;
	}
}

__global__
__launch_bounds__(TPB, 1)
void neoscrypt_gpu_hash_salsa1()
{
	const uint32_t thread = (blockDim.y * blockIdx.x + threadIdx.y);
	const uint32_t shift = SHIFT * 8U * (thread & 8191);
	const uint32_t shiftTr = 8U * thread;

	uint4 Z[4];
	for (int i = 0; i < 4; i++)
	{
		Z[i].x = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((0 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].y = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((1 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].z = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((2 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].w = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((3 + threadIdx.x) & 3) * 4 + threadIdx.x);
	}

	#pragma nounroll
	for (int i = 0; i < 128; i++)
	{
		uint32_t offset = shift + i * 8U;
		for (int j = 0; j < 4; j++)
			((uint4*)(W + offset))[j * 4 + threadIdx.x] = Z[j];
		neoscrypt_salsa(Z);
	}

	#pragma nounroll
	for (int t = 0; t < 128; t++)
	{
		uint32_t offset = shift + (WarpShuffle(Z[3].x, 0, 4) & 0x7F) * 8U;
		for (int j = 0; j < 4; j++)
			Z[j] ^= ((uint4*)(W + offset))[j * 4 + threadIdx.x];
		neoscrypt_salsa(Z);
	}
	#pragma unroll
	for (int i = 0; i < 4; i++)
	{
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((0 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].x;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((1 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].y;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((2 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].z;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((3 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].w;
	}
}

__global__
__launch_bounds__(TPB2, 8)
void neoscrypt_gpu_hash_ending(const int stratum, const uint32_t startNonce, uint32_t *resNonces)
{
	__shared__ uint32_t s_data[64 * TPB2];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t shiftTr = thread * 8U;
	const uint32_t nonce = startNonce + thread;
	const uint32_t ZNonce = (stratum) ? cuda_swab32(nonce) : nonce;

	__syncthreads();

	uint2x4 Z[8];
	#pragma unroll
	for (int i = 0; i<8; i++)
		Z[i] = __ldg4(&(Tr2 + shiftTr)[i]) ^ __ldg4(&(Tr + shiftTr)[i]);

#if __CUDA_ARCH__ < 500
	uint32_t outbuf = fastkdf32_v1(thread, ZNonce, (uint32_t*)Z, s_data);
#else
	uint32_t outbuf = fastkdf32_v3(thread, ZNonce, (uint32_t*)Z, s_data);
#endif

	if (outbuf <= c_target[1])
	{
		resNonces[0] = nonce;
		//uint32_t tmp = atomicExch(resNonces, nonce);
		//if(tmp != UINT32_MAX)
		//	resNonces[1] = tmp;
	}
}

static __thread uint32_t *hash1 = NULL;
static __thread uint32_t *Trans1 = NULL;
static __thread uint32_t *Trans2 = NULL; // 2 streams
static __thread uint32_t *Trans3 = NULL; // 2 streams

__host__
void neoscrypt_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);

	CUDA_SAFE_CALL(hipMalloc(&d_NNonce[thr_id], 2 * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&hash1, 32 * 128 * sizeof(uint64_t) * min(8192, threads)));
	CUDA_SAFE_CALL(hipMalloc(&Trans1, 32 * sizeof(uint64_t) * threads));
	CUDA_SAFE_CALL(hipMalloc(&Trans2, 32 * sizeof(uint64_t) * threads));
	CUDA_SAFE_CALL(hipMalloc(&Trans3, 32 * sizeof(uint64_t) * threads));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(W), &hash1, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Tr), &Trans1, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Tr2), &Trans2, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Input), &Trans3, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
}

__host__
void neoscrypt_free(int thr_id)
{
	hipFree(d_NNonce[thr_id]);

	hipFree(hash1);
	hipFree(Trans1);
	hipFree(Trans2);
	hipFree(Trans3);
}

__host__
void neoscrypt_hash_k4(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, bool stratum)
{
	CUDA_SAFE_CALL(hipMemset(d_NNonce[thr_id], 0xff, 2 * sizeof(uint32_t)));

	const int threadsperblock2 = TPB2;
	dim3 grid2((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 block2(threadsperblock2);

	const int threadsperblock = TPB;
	dim3 grid3((threads * 4 + threadsperblock - 1) / threadsperblock);
	dim3 block3(4, threadsperblock >> 2);

	neoscrypt_gpu_hash_start <<<grid2, block2>>> (stratum, startNounce); //fastkdf

	neoscrypt_gpu_hash_salsa1 <<<grid3, block3>>> ();
	neoscrypt_gpu_hash_chacha1 <<<grid3, block3>>> ();

	neoscrypt_gpu_hash_ending <<<grid2, block2>>> (stratum, startNounce, d_NNonce[thr_id]); //fastkdf+end

	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_NNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

__host__
void neoscrypt_setBlockTarget(uint32_t* const pdata, uint32_t* const target)
{
	uint32_t PaddedMessage[64];
	uint32_t input[16], key[16] = { 0 };

	for (int i = 0; i < 19; i++)
	{
		PaddedMessage[i] = pdata[i];
		PaddedMessage[i + 20] = pdata[i];
		PaddedMessage[i + 40] = pdata[i];
	}
	for (int i = 0; i<4; i++)
		PaddedMessage[i + 60] = pdata[i];

	PaddedMessage[19] = 0;
	PaddedMessage[39] = 0;
	PaddedMessage[59] = 0;

	((uint16*)input)[0] = ((uint16*)pdata)[0];
	((uint8*)key)[0] = ((uint8*)pdata)[0];

	Blake2Shost(input, key);

	hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, 64, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(key_init), key, 64, 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_target), &target[6], 2 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 64 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipGetLastError());
}

