#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include "cuda_vectors.h" /* NOT COMPATIBLE WITH SM 3.0 !!! */

static uint32_t *d_buffer[MAX_GPUS];
static uint32_t *d_NNonce[MAX_GPUS];
__constant__ uint4* W;
__constant__ uint32_t pTarget[8];
__constant__ uint32_t key_init[16];
__constant__ uint32_t input_init[16];
__constant__ uint32_t c_data[80];

/// constants ///

static const __constant__ uint8 BLAKE2S_IV_Vec = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint8 BLAKE2S_IV_Vechost = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

static __constant__ uint32_t BLAKE2S_SIGMA[10][16];

#define FASTKDF_BUFFER_SIZE  256U

// Blake2S

#define BLAKE2S_BLOCK_SIZE    64U
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U

#if __CUDA_ARCH__ >= 500
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}
#else
#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}
#endif

#if __CUDA_ARCH__ >= 500
#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = __byte_perm(d^a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}
#else
#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = rotateL(d^a, 16); \
	c += d; b = rotateR(b^c, 12); \
	a += key[idx1]; \
	a += b; d = rotateR(d^a, 8); \
	c += d; b = rotateR(b^c, 7); \
}
#endif

#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,16); \
	c += d; b = ROTR32(b^c, 12); \
	idx = BLAKE2S_SIGMA_host[idx0][idx1+1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,8); \
	c += d; b = ROTR32(b^c, 7); \
}

static __forceinline__ __device__
void Blake2S(uint32_t * inout, const uint32_t * TheKey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

#if 0
	for (int x = 0; x < 10; ++x)
	{
		BLAKE_G(x, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
		BLAKE_G(x, 0x2, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
		BLAKE_G(x, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
		BLAKE_G(x, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
		BLAKE_G(x, 0x8, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
		BLAKE_G(x, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
		BLAKE_G(x, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
		BLAKE_G(x, 0xE, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	}
#else
	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0x0, 0x1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x2, 0x3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x4, 0x5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x6, 0x7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x8, 0x9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xA, 0xB, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0xC, 0xD, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xE, 0xF, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(0xE, 0xA, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x4, 0x8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x9, 0xF, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0xD, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x1, 0xC, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x0, 0x2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0xB, 0x7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x5, 0x3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(0xB, 0x8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0xC, 0x0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x5, 0x2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0xF, 0xD, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xA, 0xE, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x3, 0x6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x7, 0x1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x9, 0x4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(0x7, 0x9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x3, 0x1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xD, 0xC, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0xB, 0xE, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x2, 0x6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x5, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x4, 0x0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xF, 0x8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE(0x9, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x5, 0x7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x2, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0xA, 0xF, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xE, 0x1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xB, 0xC, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x6, 0x8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x3, 0xD, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE(0x2, 0xC, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x6, 0xA, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x0, 0xB, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x8, 0x3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x4, 0xD, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x7, 0x5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0xF, 0xE, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x1, 0x9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE(0xC, 0x5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x1, 0xF, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xE, 0xD, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x4, 0xA, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x0, 0x7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x6, 0x3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x9, 0x2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x8, 0xB, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE(0xD, 0xB, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x7, 0xE, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xC, 0x1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x3, 0x9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x5, 0x0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xF, 0x4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x8, 0x6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x2, 0xA, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE(0x6, 0xF, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0xE, 0x9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xB, 0x3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x0, 0x8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xC, 0x2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xD, 0x7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x1, 0x4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xA, 0x5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE(0xA, 0x2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x8, 0x4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0x7, 0x6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0x1, 0x5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0xF, 0xB, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0x9, 0xE, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(0x3, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(0xD, 0x0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
#endif

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

#if 0
	for (int x = 0; x < 10; ++x)
	{
		BLAKE_G(x, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x2, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x8, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0xE, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}
#else
	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0x0, 0x1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(0x2, 0x3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(0x4, 0x5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(0x6, 0x7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(0x8, 0x9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0xA, 0xB, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(0xC, 0xD, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(0xE, 0xF, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(0xE, 0xA, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(0x4, 0x8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(0x9, 0xF, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(0xD, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(0x1, 0xC, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0x0, 0x2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(0xB, 0x7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(0x5, 0x3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(0xB, 0x8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(0xC, 0x0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(0x5, 0x2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(0xF, 0xD, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(0xA, 0xE, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0x3, 0x6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(0x7, 0x1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(0x9, 0x4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(0x7, 0x9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(0x3, 0x1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(0xD, 0xC, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(0xB, 0xE, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(0x2, 0x6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0x5, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(0x4, 0x0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(0xF, 0x8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	for (int x = 4; x < 10; ++x)
	{
		BLAKE_G(x, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x2, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x8, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0xE, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}
#endif

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0]=V.lo;
}

static __forceinline__ __host__
void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x2, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x8, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0xE, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x2, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x8, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0xA, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0xC, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 0xE, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}

static __forceinline__ __device__
void fastkdf256(const uint32_t* password, uint8_t* output)
{
	uint8_t bufidx = 0;
	uchar4 bufhelper;
	uint8_t A[320],B[288];

	((uintx64*)A)[0] = ((uintx64*)password)[0];
	((uint816 *)A)[4] =  ((uint816 *)password)[0];

	((uintx64*)B)[0] = ((uintx64*)password)[0];
	((uint48 *)B)[8] = ((uint48 *)password)[0];

	uint32_t input[BLAKE2S_BLOCK_SIZE/4]; uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = { 0 };

	((uint816*)input)[0] = ((uint816*)input_init)[0];
	((uint48*)key)[0] = ((uint48*)key_init)[0];

	for (int i = 0; i < 32; ++i)
	{
		bufhelper = ((uchar4*)input)[0];
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x)
			bufhelper += ((uchar4*)input)[x];
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;

		int qbuf = bufidx/4;
		int rbuf = bufidx&3;
		int bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; ++k) {
			((uint32_t *)B)[k + qbuf] ^= ((uint32_t *)shifted)[k];
		}

		if (bufidx < BLAKE2S_KEY_SIZE)                          {((uint8*)B)[8] = ((uint8*)B)[0];}
		else if (bufidx > FASTKDF_BUFFER_SIZE-BLAKE2S_OUT_SIZE) {((uint8*)B)[0] = ((uint8*)B)[8];}

		if (i<31) {
			for (int k = 0; k <BLAKE2S_BLOCK_SIZE / 4; k++) {
				((uchar4*)(input))[k] = make_uchar4(
					(A + bufidx)[4 * k], (A + bufidx)[4 * k + 1],
					(A + bufidx)[4 * k + 2], (A + bufidx)[4 * k + 3]
				);
			}

			for (int k = 0; k <BLAKE2S_KEY_SIZE / 4; k++) {
				((uchar4*)(key))[k] = make_uchar4(
					(B + bufidx)[4 * k], (B + bufidx)[4 * k + 1],
					(B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]
				);
			}

			Blake2S((uint32_t*)input, key);
		}
	}

	int left = FASTKDF_BUFFER_SIZE - bufidx;
	int qleft =left/4;
	int rleft =left&3;
	for (int k = 0; k < qleft; ++k) {
		((uchar4*)output)[k] = make_uchar4(
			(B + bufidx)[4 * k], (B + bufidx)[4 * k + 1],
			(B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]
		) ^ ((uchar4*)A)[k];
	}
	for (int i = 4*qleft; i < 4*qleft+rleft; ++i) {
		output[i] = (B + bufidx)[i] ^ A[i];
	}
	for (int i = qleft*4+rleft; i < (qleft+1)*4; ++i) {
		((uint8_t *)output)[i] = ((uint8_t *)B)[i - left] ^ ((uint8_t *)A)[i];
	}
	for (int i = qleft+1; i < FASTKDF_BUFFER_SIZE/4; ++i) {
		((uchar4 *)output)[i] = make_uchar4(B[4*i - left],B[4*i+1-left],
		 B[4*i+2-left],B[4*i+3-left]) ^ ((uchar4 *)A)[i];
	}
}

static __forceinline__ __device__
void fastkdf32(const uint32_t * password, const uint32_t * salt, uint32_t * output)
{
	uint8_t bufidx = 0;
	uchar4 bufhelper;

	uint8_t A[320];
	uint8_t B[288];

	// Initialize the password buffer
	((uintx64*)A)[0] = ((uintx64*)password)[0];
	((uint816*)A)[4] = ((uint816*)password)[0];
	((uintx64*)B)[0] = ((uintx64*)salt)[0];
	((uintx64*)B)[1] = ((uintx64*)salt)[0];

	uint32_t input[BLAKE2S_BLOCK_SIZE/4];
	uint32_t key[BLAKE2S_BLOCK_SIZE/4] = { 0 };

	((uint816*)input)[0] = ((uint816*)password)[0];
	((uint48*)key)[0] = ((uint48*)salt)[0];

	for (int i = 0; i < 32; ++i)
	{
		Blake2S((uint32_t*)input, key);

		bufidx = 0;
		bufhelper = ((uchar4*)input)[0];

		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x)
			bufhelper += ((uchar4*)input)[x];

		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
		int qbuf = bufidx / 4;
		int rbuf = bufidx & 3;
		int bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R(shifted, ((uint8*)input)[0], bitbuf);

		for (int k = 0; k < 9; ++k) {
			((uint32_t *)B)[k + qbuf] ^= ((uint32_t *)shifted)[k];
		}

		if (i<31) {
			if (bufidx < BLAKE2S_KEY_SIZE)                            {((uint8*)B)[8] = ((uint8*)B)[0];}
			else if (bufidx > FASTKDF_BUFFER_SIZE - BLAKE2S_OUT_SIZE) {((uint8*)B)[0] = ((uint8*)B)[8];}

			for (uint8_t k = 0; k < BLAKE2S_BLOCK_SIZE/4; k++) {
				((uchar4*)(input))[k] = make_uchar4(
					(A + bufidx)[4 * k], (A + bufidx)[4 * k + 1],
					(A + bufidx)[4 * k + 2], (A + bufidx)[4 * k + 3]
				);
			}
			for (uint8_t k = 0; k < BLAKE2S_KEY_SIZE / 4; k++) {
				((uchar4*)(key))[k] = make_uchar4(
					(B + bufidx)[4 * k], (B + bufidx)[4 * k + 1],
					(B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]
				);
			}
		}
	}

	uchar4 unfucked[1];
	unfucked[0] = make_uchar4(B[28 + bufidx], B[29 + bufidx],B[30 + bufidx], B[31 + bufidx]);
	((uint32_t*)output)[7] = ((uint32_t*)unfucked)[0] ^ ((uint32_t*)A)[7];
}


#define SALSA(a,b,c,d) { \
    t =a+d; b^=rotateL(t,  7); \
    t =b+a; c^=rotateL(t,  9); \
    t =c+b; d^=rotateL(t, 13); \
    t =d+c; a^=rotateL(t, 18); \
}

#define SALSA_CORE(state) { \
    SALSA(state.s0,state.s4,state.s8,state.sc); \
    SALSA(state.s5,state.s9,state.sd,state.s1); \
    SALSA(state.sa,state.se,state.s2,state.s6); \
    SALSA(state.sf,state.s3,state.s7,state.sb); \
    SALSA(state.s0,state.s1,state.s2,state.s3); \
    SALSA(state.s5,state.s6,state.s7,state.s4); \
    SALSA(state.sa,state.sb,state.s8,state.s9); \
    SALSA(state.sf,state.sc,state.sd,state.se); \
}

#if __CUDA_ARCH__ >=500
#define CHACHA_STEP(a,b,c,d) { \
    a += b; d = __byte_perm(d^a,0,0x1032); \
    c += d; b = rotateL(b^c, 12); \
    a += b; d = __byte_perm(d^a,0,0x2103); \
    c += d; b = rotateL(b^c, 7); \
}
#else
#define CHACHA_STEP(a,b,c,d) { \
    a += b; d = rotateL(d^a,16); \
    c += d; b = rotateL(b^c, 12); \
    a += b; d = rotateL(d^a,8); \
    c += d; b = rotateL(b^c, 7); \
}
#endif

#define CHACHA_CORE_PARALLEL(state) { \
    CHACHA_STEP(state.lo.s0, state.lo.s4, state.hi.s0, state.hi.s4); \
    CHACHA_STEP(state.lo.s1, state.lo.s5, state.hi.s1, state.hi.s5); \
    CHACHA_STEP(state.lo.s2, state.lo.s6, state.hi.s2, state.hi.s6); \
    CHACHA_STEP(state.lo.s3, state.lo.s7, state.hi.s3, state.hi.s7); \
    CHACHA_STEP(state.lo.s0, state.lo.s5, state.hi.s2, state.hi.s7); \
    CHACHA_STEP(state.lo.s1, state.lo.s6, state.hi.s3, state.hi.s4); \
    CHACHA_STEP(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5); \
    CHACHA_STEP(state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \
}


static __forceinline__ __device__ uint16 salsa_small_scalar_rnd(const uint16 &X)
{
	uint16 state = X;
	uint32_t t;

	for (int i = 0; i < 10; ++i) { SALSA_CORE(state);}

	return(X + state);
}

static __device__ __forceinline__ uint16 chacha_small_parallel_rnd(const uint16 &X)
{
	uint16 st = X;

	for (int i = 0; i < 10; ++i) {CHACHA_CORE_PARALLEL(st);}
	return(X + st);
}

static __device__ __forceinline__ void neoscrypt_chacha(uint16 *XV)
{
	XV[0] ^= XV[3];
	uint16 temp;

	XV[0] = chacha_small_parallel_rnd(XV[0]); XV[1] ^= XV[0];
	 temp = chacha_small_parallel_rnd(XV[1]); XV[2] ^= temp;
	XV[1] = chacha_small_parallel_rnd(XV[2]); XV[3] ^= XV[1];
	XV[3] = chacha_small_parallel_rnd(XV[3]);
	XV[2] = temp;
}

static __device__ __forceinline__ void neoscrypt_salsa(uint16 *XV)
{
	XV[0] ^= XV[3];
	uint16 temp;

	XV[0] = salsa_small_scalar_rnd(XV[0]); XV[1] ^= XV[0];
	 temp = salsa_small_scalar_rnd(XV[1]); XV[2] ^= temp;
	XV[1] = salsa_small_scalar_rnd(XV[2]); XV[3] ^= XV[1];
	XV[3] = salsa_small_scalar_rnd(XV[3]);
	XV[2] = temp;
}


#define SHIFT 130

__global__ __launch_bounds__(128, 1)
void neoscrypt_gpu_hash_k0(uint32_t threads, uint32_t startNonce, int stratum)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint32_t data[80];
		uint16 X[4];
		uint32_t shift = thread * SHIFT * 16;
		const uint32_t nonce = startNonce + thread;

		for (int i = 0; i<20; i++) {
			((uint4*)data)[i] = ((uint4 *)c_data)[i];
		}  //ld.local.v4
		data[19] = (stratum) ? cuda_swab32(nonce) : nonce; //freaking morons !!!
		data[39] = data[19];
		data[59] = data[19];

		fastkdf256(data, (uint8_t*)X);

		((uintx64 *)(W + shift))[0] = ((uintx64 *)X)[0];
//		((ulonglong16 *)(W + shift))[0] = ((ulonglong16 *)X)[0];
	}
}

__global__ __launch_bounds__(128, 1)
void neoscrypt_gpu_hash_k01(uint32_t threads, uint32_t startNonce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint16 X[4];
		uint32_t shift = thread * SHIFT * 16;
		((uintx64 *)X)[0]= ldg256(&(W + shift)[0]);

		//#pragma unroll
		for (int i = 0; i < 128; ++i)
		{
			neoscrypt_chacha(X);
			((ulonglong16 *)(W + shift))[i+1] = ((ulonglong16 *)X)[0];
//			((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)X)[0];
		}
	}
}

__global__ __launch_bounds__(128, 1)
void neoscrypt_gpu_hash_k2(uint32_t threads, uint32_t startNonce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint16 X[4];
		uint32_t shift = thread * SHIFT * 16;
		((uintx64 *)X)[0] = ldg256(&(W + shift)[2048]);

		for (int t = 0; t < 128; t++)
		{
			int idx = X[3].lo.s0 & 0x7F;
			((uintx64 *)X)[0] ^= ldg256(&(W + shift)[idx << 4]);
			neoscrypt_chacha(X);

		}
		((uintx64 *)(W + shift))[129] = ((uintx64*)X)[0];  // best checked

	}
}

__global__ __launch_bounds__(128, 1)
void neoscrypt_gpu_hash_k3(uint32_t threads, uint32_t startNonce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint32_t shift = thread * SHIFT * 16;
		uint16 Z[4];

		((uintx64*)Z)[0] = ldg256(&(W + shift)[0]);

		//#pragma unroll
		for (int i = 0; i < 128; ++i) {
			neoscrypt_salsa(Z);
			((ulonglong16 *)(W + shift))[i+1] = ((ulonglong16 *)Z)[0];
//			((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)Z)[0];
		}
	}
}

__global__ __launch_bounds__(128, 1)
void neoscrypt_gpu_hash_k4(uint32_t threads, uint32_t startNonce, uint32_t *nonceRes, int stratum)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		uint32_t shift = thread * SHIFT * 16;
		uint16 Z[4];
		uint32_t outbuf[8];
		uint32_t data[80];

		for (int i=0; i<20; i++) {
			((uint4*)data)[i] = ((uint4 *)c_data)[i];
		}

		data[19] = (stratum) ? cuda_swab32(nonce) : nonce;
		data[39] = data[19];
		data[59] = data[19];
		((uintx64 *)Z)[0] = ldg256(&(W + shift)[2048]);
		for (int t = 0; t < 128; t++)
		{
			int idx = Z[3].lo.s0 & 0x7F;
			((uintx64 *)Z)[0] ^= ldg256(&(W + shift)[idx << 4]);
			neoscrypt_salsa(Z);
		}
		((uintx64 *)Z)[0] ^= ldg256(&(W + shift)[2064]);
		fastkdf32(data, (uint32_t*)Z, outbuf);
#if __CUDA_ARCH__ < 320
		// workaround required when using SM 3.0 shift256R() func (tested on SM 5.0)
		if (thread == 0)
			printf("", outbuf[7]);
#endif
		if (outbuf[7] <= pTarget[7]) {
			atomicMin(nonceRes, nonce); // init val is UINT32_MAX
		}
	}
}

__host__
void neoscrypt_cpu_init(int thr_id, uint32_t threads)
{
	cuda_get_arch(thr_id);
	hipMalloc(&d_NNonce[thr_id], sizeof(uint32_t));
	CUDA_SAFE_CALL(hipMalloc(&d_buffer[thr_id], threads * 256 * SHIFT));
	hipMemcpyToSymbol(HIP_SYMBOL(W), &d_buffer[thr_id], sizeof(uint4*), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(BLAKE2S_SIGMA), BLAKE2S_SIGMA_host, sizeof(BLAKE2S_SIGMA_host), 0, hipMemcpyHostToDevice);
}

__host__
uint32_t neoscrypt_cpu_hash_k4(int thr_id, uint32_t threads, uint32_t startNounce, int have_stratum, int order)
{
	uint32_t result[MAX_GPUS];
	memset(result, 0xff, sizeof(result));
	hipMemset(d_NNonce[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	neoscrypt_gpu_hash_k0  <<< grid, block >>>(threads, startNounce, have_stratum);
	neoscrypt_gpu_hash_k01 <<< grid, block >>>(threads, startNounce);
	neoscrypt_gpu_hash_k2  <<< grid, block >>>(threads, startNounce);
	neoscrypt_gpu_hash_k3  <<< grid, block >>>(threads, startNounce);
	neoscrypt_gpu_hash_k4  <<< grid, block >>>(threads, startNounce, d_NNonce[thr_id], have_stratum);

	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(&result[thr_id], d_NNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	return result[thr_id];
}

__host__
void neoscrypt_setBlockTarget(uint32_t* pdata, const void *target)
{
	unsigned char PaddedMessage[80*4]; //bring balance to the force
	uint32_t input[16], key[16] = { 0 };

	memcpy(PaddedMessage,     pdata, 80);
	memcpy(PaddedMessage + 80, pdata, 80);
	memcpy(PaddedMessage + 160, pdata, 80);
	memcpy(PaddedMessage + 240, pdata, 80);

	((uint16*)input)[0] = ((uint16*)pdata)[0];
	((uint8*)key)[0] = ((uint8*)pdata)[0];

	Blake2Shost(input,key);

	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), target, 32, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, sizeof(input), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(key_init), key, sizeof(key), 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 80*4, 0, hipMemcpyHostToDevice);
}

