#include "hip/hip_runtime.h"
/*
 * Polytimos algorithm
 */
extern "C"
{
#include "sph/sph_skein.h"
#include "sph/sph_shabal.h"
#include "sph/sph_echo.h"
#include "sph/sph_luffa.h"
#include "sph/sph_fugue.h"
#include "sph/sph_streebog.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "x11/cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int swap);
extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_free(int thr_id);
extern void streebog_sm3_set_target(uint32_t* ptarget);
extern void streebog_sm3_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* d_resNonce);
extern void skunk_streebog_set_target(uint32_t* ptarget);
extern void skunk_cuda_streebog(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* d_resNonce);

// CPU Hash
extern "C" void polytimos_hash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;
	sph_shabal512_context ctx_shabal;
	sph_echo512_context ctx_echo;
	sph_luffa512_context ctx_luffa;
	sph_fugue512_context ctx_fugue;
	sph_gost512_context ctx_gost;

	uint32_t _ALIGN(128) hash[16];
	memset(hash, 0, sizeof hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, 64);
	sph_shabal512_close(&ctx_shabal, hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hash, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hash, 64);
	sph_luffa512_close(&ctx_luffa, hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hash, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*) hash, 64);
	sph_gost512_close(&ctx_gost, (void*) hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_polytimos(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	int dev_id = device_map[thr_id];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		quark_skein512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)), -1);

		init[thr_id] = true;
	}


	uint32_t _ALIGN(64) h_resNonce[2];
	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);


	hipMemset(d_resNonce[thr_id], 0xff, 2*sizeof(uint32_t));
	skein512_cpu_setBlock_80(endiandata);
	if (use_compat_kernels[thr_id]) {
		streebog_sm3_set_target(ptarget);
	} else {
		skunk_streebog_set_target(ptarget);
	}

	do {
		int order = 0;

		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (use_compat_kernels[thr_id]) {
			streebog_sm3_hash_64_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id]);
		} else {
			skunk_cuda_streebog(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id]);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		hipMemcpy(h_resNonce, d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
		CUDA_LOG_ERROR();

		if (h_resNonce[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], startNounce + h_resNonce[0]);
			polytimos_hash(vhash, endiandata);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[0] = startNounce + h_resNonce[0];
				work_set_target_ratio(work, vhash);
				if (h_resNonce[1] != UINT32_MAX) {
					uint32_t secNonce = work->nonces[1] = startNounce + h_resNonce[1];
					be32enc(&endiandata[19], secNonce);
					polytimos_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				hipMemset(d_resNonce[thr_id], 0xff, 2*sizeof(uint32_t));
				pdata[19] = startNounce + h_resNonce[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	CUDA_LOG_ERROR();

	return 0;
}

// cleanup
extern "C" void free_polytimos(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	x13_fugue512_cpu_free(thr_id);
	hipFree(d_resNonce[thr_id]);

	CUDA_LOG_ERROR();

	hipDeviceSynchronize();
	init[thr_id] = false;
}
