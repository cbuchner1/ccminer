#include "hip/hip_runtime.h"
/**
 * SHA256 3x
 * by tpruvot@github - 2017
 */

#include <miner.h>
#include <cuda_helper.h>
#include <openssl/sha.h>

// CPU Check
extern "C" void sha256t_hash(void *output, const void *input)
{
	unsigned char _ALIGN(64) hash[64];
	SHA256_CTX sha256;

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char *)input, 80);
	SHA256_Final(hash, &sha256);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, hash, 32);
	SHA256_Final(hash, &sha256);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, hash, 32);
	SHA256_Final((unsigned char *)output, &sha256);
}

static bool init[MAX_GPUS] = { 0 };
extern void sha256t_init(int thr_id);
extern void sha256t_free(int thr_id);
extern void sha256t_setBlock_80(uint32_t *pdata, uint32_t *ptarget);
extern void sha256t_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces);

extern "C" int scanhash_sha256t(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 23);
	if (init[thr_id]) throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x03;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		sha256t_init(thr_id);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	sha256t_setBlock_80(endiandata, ptarget);

	do {
		// Hash with CUDA
		*hashes_done = pdata[19] - first_nonce + throughput;

		sha256t_hash_80(thr_id, throughput, pdata[19], work->nonces);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			endiandata[19] = swab32(work->nonces[0]);
			sha256t_hash(vhash, endiandata);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != UINT32_MAX) {
					endiandata[19] = swab32(work->nonces[1]);
					sha256t_hash(vhash, endiandata);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						work->valid_nonces++;
						bn_set_target_ratio(work, vhash, 1);
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1;
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_sha256t(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	sha256t_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
