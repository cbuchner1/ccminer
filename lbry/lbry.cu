#include "hip/hip_runtime.h"
/**
 * Lbry Algo (sha-256 / sha-512 / ripemd)
 *
 * tpruvot and Provos Alexis - Jan 2017
 *
 * Sponsored by LBRY.IO team
 */

#include <string.h>
#include <stdint.h>

extern "C" {
#include <sph/sph_sha2.h>
#include <sph/sph_ripemd.h>
}

#include <cuda_helper.h>
#include <miner.h>

#define A 64
#define debug_cpu 0

extern "C" void lbry_hash(void* output, const void* input)
{
	uint32_t _ALIGN(A) hashA[16];
	uint32_t _ALIGN(A) hashB[8];
	uint32_t _ALIGN(A) hashC[8];

	sph_sha256_context ctx_sha256;
	sph_sha512_context ctx_sha512;
	sph_ripemd160_context ctx_ripemd;

	sph_sha256_init(&ctx_sha256);
	sph_sha256(&ctx_sha256, input, 112);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512, hashA, 32);
	sph_sha512_close(&ctx_sha512, hashA);

	sph_ripemd160_init(&ctx_ripemd);
	sph_ripemd160(&ctx_ripemd, hashA, 32);  // sha512 low
	sph_ripemd160_close(&ctx_ripemd, hashB);
	if (debug_cpu) applog_hex(hashB, 20);

	sph_ripemd160(&ctx_ripemd, &hashA[8], 32); // sha512 high
	sph_ripemd160_close(&ctx_ripemd, hashC);
	if (debug_cpu) applog_hex(hashC, 20);

	sph_sha256(&ctx_sha256, hashB, 20);
	sph_sha256(&ctx_sha256, hashC, 20);
	sph_sha256_close(&ctx_sha256, hashA);
	if (debug_cpu) applog_hex(hashA,32);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	memcpy(output, hashA, 32);
}

/* ############################################################################################################################### */

extern void lbry_sha256_init(int thr_id);
extern void lbry_sha256_free(int thr_id);
extern void lbry_sha256_setBlock_112(uint32_t *pdata);
extern void lbry_sha256d_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash);
extern void lbry_sha512_init(int thr_id);
extern void lbry_sha512_hash_32(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void lbry_sha256d_hash_final(int thr_id, uint32_t threads, uint32_t *d_inputHash, uint32_t *d_resNonce, const uint64_t target64);

extern void lbry_sha256_setBlock_112_merged(uint32_t *pdata);
extern void lbry_merged(int thr_id,uint32_t startNonce, uint32_t threads, uint32_t *d_resNonce, const uint64_t target64);

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

static bool init[MAX_GPUS] = { 0 };

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];
// nonce position is different
#define LBC_NONCE_OFT32 27

extern "C" int scanhash_lbry(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(A) endiandata[28];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[LBC_NONCE_OFT32];
	const int swap = 0; // to toggle nonce endian (need kernel change)

	const int dev_id = device_map[thr_id];
	const bool merged_kernel = (device_sm[dev_id] > 500);

	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 22 : 20;
	if (device_sm[dev_id] >= 600) intensity = 23;
	if (device_sm[dev_id] < 350) intensity = 18;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) {
		ptarget[7] = 0xf;
	}

	if (!init[thr_id]){
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		if (CUDART_VERSION == 6050) {
			applog(LOG_ERR, "This lbry kernel is not compatible with CUDA 6.5!");
			proper_exit(EXIT_FAILURE);
		}

		if (!merged_kernel)
			CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)64 * throughput));

		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));
		CUDA_LOG_ERROR();

		init[thr_id] = true;
	}

	for (int i=0; i < LBC_NONCE_OFT32; i++) {
		be32enc(&endiandata[i], pdata[i]);
	}

	if (merged_kernel)
		lbry_sha256_setBlock_112_merged(endiandata);
	else
		lbry_sha256_setBlock_112(endiandata);

	hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));

	do {
		uint32_t resNonces[2] = { UINT32_MAX, UINT32_MAX };

		// Hash with CUDA
		if (merged_kernel) {
			lbry_merged(thr_id, pdata[LBC_NONCE_OFT32], throughput, d_resNonce[thr_id], AS_U64(&ptarget[6]));
		} else {
			lbry_sha256d_hash_112(thr_id, throughput, pdata[LBC_NONCE_OFT32], d_hash[thr_id]);
			lbry_sha512_hash_32(thr_id, throughput, d_hash[thr_id]);
			lbry_sha256d_hash_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id], AS_U64(&ptarget[6]));
		}

		*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce + throughput;

		hipMemcpy(resNonces, d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (resNonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(A) vhash[8];
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNonce = pdata[LBC_NONCE_OFT32];
			resNonces[0] += startNonce;

			endiandata[LBC_NONCE_OFT32] = swab32_if(resNonces[0], !swap);
			lbry_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				work->nonces[0] = swab32_if(resNonces[0], swap);
				work_set_target_ratio(work, vhash);
				work->valid_nonces = 1;

				if (resNonces[1] != UINT32_MAX)
				{
					resNonces[1] += startNonce;
					endiandata[LBC_NONCE_OFT32] = swab32_if(resNonces[1], !swap);
					lbry_hash(vhash, endiandata);
					work->nonces[1] = swab32_if(resNonces[1], swap);

					if (bn_hash_target_ratio(vhash, ptarget) > work->shareratio[0]) {
						// best first
						xchg(work->nonces[1], work->nonces[0]);
						work->sharediff[1] = work->sharediff[0];
						work->shareratio[1] = work->shareratio[0];
						work_set_target_ratio(work, vhash);
					} else {
						bn_set_target_ratio(work, vhash, 1);
					}
					work->valid_nonces++;
				}

				pdata[LBC_NONCE_OFT32] = max(work->nonces[0], work->nonces[1]); // next scan start

				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", resNonces[0]);
				hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
			}
		}

		if ((uint64_t) throughput + pdata[LBC_NONCE_OFT32] >= max_nonce) {
			pdata[LBC_NONCE_OFT32] = max_nonce;
			break;
		}

		pdata[LBC_NONCE_OFT32] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce;

	return 0;
}

// cleanup
void free_lbry(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	if(device_sm[device_map[thr_id]] <= 500)
		hipFree(d_hash[thr_id]);

	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
