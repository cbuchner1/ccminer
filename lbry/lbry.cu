#include "hip/hip_runtime.h"
/**
 * Lbry CUDA Implementation
 *
 * by tpruvot@github - July 2016
 *
 */

#include <string.h>
#include <stdint.h>

extern "C" {
#include <sph/sph_sha2.h>
#include <sph/sph_ripemd.h>
}

#include <cuda_helper.h>
#include <miner.h>

#define A 64
#define debug_cpu 0

extern "C" void lbry_hash(void* output, const void* input)
{
	uint32_t _ALIGN(A) hashA[16];
	uint32_t _ALIGN(A) hashB[8];
	uint32_t _ALIGN(A) hashC[8];

	sph_sha256_context ctx_sha256;
	sph_sha512_context ctx_sha512;
	sph_ripemd160_context ctx_ripemd;

	sph_sha256_init(&ctx_sha256);
	sph_sha256(&ctx_sha256, input, 112);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512, hashA, 32);
	sph_sha512_close(&ctx_sha512, hashA);

	sph_ripemd160_init(&ctx_ripemd);
	sph_ripemd160(&ctx_ripemd, hashA, 32);  // sha512 low
	sph_ripemd160_close(&ctx_ripemd, hashB);
	if (debug_cpu) applog_hex(hashB, 20);

	sph_ripemd160(&ctx_ripemd, &hashA[8], 32); // sha512 high
	sph_ripemd160_close(&ctx_ripemd, hashC);
	if (debug_cpu) applog_hex(hashC, 20);

	sph_sha256(&ctx_sha256, hashB, 20);
	sph_sha256(&ctx_sha256, hashC, 20);
	sph_sha256_close(&ctx_sha256, hashA);
	if (debug_cpu) applog_hex(hashA,32);

	sph_sha256(&ctx_sha256, hashA, 32);
	sph_sha256_close(&ctx_sha256, hashA);

	memcpy(output, hashA, 32);
}

/* ############################################################################################################################### */

extern void lbry_sha256_init(int thr_id);
extern void lbry_sha256_free(int thr_id);
extern void lbry_sha256_setBlock_112(uint32_t *pdata, uint32_t *ptarget);
extern void lbry_sha256d_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash);
extern void lbry_sha512_init(int thr_id);
extern void lbry_sha512_hash_32(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void lbry_sha256d_hash_final(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_inputHash, uint32_t *resNonces);

static __inline uint32_t swab32_if(uint32_t val, bool iftrue) {
	return iftrue ? swab32(val) : val;
}

static bool init[MAX_GPUS] = { 0 };

static uint32_t *d_hash[MAX_GPUS];

// nonce position is different
#define LBC_NONCE_OFT32 27

extern "C" int scanhash_lbry(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(A) vhash[8];
	uint32_t _ALIGN(A) endiandata[28];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[LBC_NONCE_OFT32];
	const int swap = 0; // to toggle nonce endian (need kernel change)

	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 22 : 20;
	if (device_sm[dev_id] >= 600) intensity = 23;
	if (device_sm[dev_id] < 350) intensity = 18;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark) {
		ptarget[7] = 0xf;
	}

	if (!init[thr_id]){
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		lbry_sha256_init(thr_id);
		lbry_sha512_init(thr_id);
		CUDA_LOG_ERROR();

		init[thr_id] = true;
	}

	for (int i=0; i < LBC_NONCE_OFT32; i++) {
		be32enc(&endiandata[i], pdata[i]);
	}

	lbry_sha256_setBlock_112(endiandata, ptarget);

	do {
		// Hash with CUDA
		lbry_sha256d_hash_112(thr_id, throughput, pdata[LBC_NONCE_OFT32], d_hash[thr_id]);
		CUDA_LOG_ERROR();

		lbry_sha512_hash_32(thr_id, throughput, d_hash[thr_id]);
		CUDA_LOG_ERROR();

		uint32_t resNonces[2] = { UINT32_MAX, UINT32_MAX };
		lbry_sha256d_hash_final(thr_id, throughput, pdata[LBC_NONCE_OFT32], d_hash[thr_id], resNonces);
		CUDA_LOG_ERROR();

		uint32_t foundNonce = resNonces[0];
		*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce + throughput;

		if (foundNonce != UINT32_MAX)
		{
			endiandata[LBC_NONCE_OFT32] = swab32_if(foundNonce, !swap);
			lbry_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				int res = 1;
				uint32_t secNonce =  resNonces[1];
				work->nonces[0] = swab32_if(foundNonce, swap);
				work_set_target_ratio(work, vhash);
				if (secNonce != UINT32_MAX) {
					if (opt_debug)
						gpulog(LOG_BLUE, thr_id, "found second nonce %08x", swab32(secNonce));
					endiandata[LBC_NONCE_OFT32] = swab32_if(secNonce, !swap);
					lbry_hash(vhash, endiandata);
					work->nonces[1] = swab32_if(secNonce, swap);
					if (bn_hash_target_ratio(vhash, ptarget) > work->shareratio) {
						work_set_target_ratio(work, vhash);
						xchg(work->nonces[0], work->nonces[1]);
					}
					res++;
				}
				pdata[LBC_NONCE_OFT32] = work->nonces[0];
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU %08x > %08x!", foundNonce, vhash[7], ptarget[7]);
			}
		}

		if ((uint64_t) throughput + pdata[LBC_NONCE_OFT32] >= max_nonce) {
			pdata[LBC_NONCE_OFT32] = max_nonce;
			break;
		}

		pdata[LBC_NONCE_OFT32] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[LBC_NONCE_OFT32] - first_nonce;

	return 0;
}

// cleanup
void free_lbry(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	lbry_sha256_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
