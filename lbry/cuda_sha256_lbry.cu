#include "hip/hip_runtime.h"
/*
 * sha256 + ripemd CUDA implementation for SM 5.0 GPUS (and lower)
 * Tanguy Pruvot and Provos Alexis - Jul / Sep 2016
 * Sponsored by LBRY.IO team
 */

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>

#include <miner.h>

__constant__ static uint32_t _ALIGN(16) c_midstate112[8];
__constant__ static uint32_t _ALIGN(16) c_midbuffer112[8];
__constant__ static uint32_t _ALIGN(16) c_dataEnd112[12];

__constant__  const uint32_t c_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};
__constant__ static uint32_t _ALIGN(16) c_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#ifdef __INTELLISENSE__
#define atomicExch(p,y) y
#define __byte_perm(x,y,c) x
#endif

// ------------------------------------------------------------------------------------------------

static const uint32_t cpu_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};

static const uint32_t cpu_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

__host__
static void sha256_step1_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d,
	uint32_t e, uint32_t f, uint32_t g, uint32_t &h, uint32_t in, const uint32_t Kshared)
{
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);
	uint32_t t1 = h + bsg21 + vxandx + Kshared + in;
	uint32_t t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__host__
static void sha256_step2_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d,
	uint32_t e, uint32_t f, uint32_t g, uint32_t &h, uint32_t* in, uint32_t pc, const uint32_t Kshared)
{
	int pcidx1 = (pc-2)  & 0xF;
	int pcidx2 = (pc-7)  & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];

	uint32_t ssg21 = ROTR32(inx1, 17) ^ ROTR32(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
	uint32_t ssg20 = ROTR32(inx3, 7) ^ ROTR32(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);
	uint32_t t1,t2;

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__host__
static void sha256_round_body_host(uint32_t* in, uint32_t* state, const uint32_t* Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	for (int i=0; i<3; i++)
	{
		sha256_step2_host(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha256_step2_host(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

#define xor3b(a,b,c) (a ^ b ^ c)

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,2),ROTR32(x,13),ROTR32(x,22));
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,6),ROTR32(x,11),ROTR32(x,25));
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	return xor3b(ROTR32(x,7),ROTR32(x,18),(x>>3));
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	return xor3b(ROTR32(x,17),ROTR32(x,19),(x>>10));
}

__device__ __forceinline__ uint2 vectorizeswap(uint64_t v)
{
	uint2 result;
	asm("mov.b64 {%0,%1},%2; // vectorizeswap \n\t"
		: "=r"(result.y), "=r"(result.x) : "l"(v));
	return result;
}

#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define Ch(a, b, c)     (((b^c) & a) ^ c)

__device__ __forceinline__
static void sha2_step(const uint32_t a,const uint32_t b,const uint32_t c, uint32_t &d,const uint32_t e,const uint32_t f,const uint32_t g, uint32_t &h,const uint32_t in, const uint32_t Kshared)
{
	const uint32_t t1 = h + bsg2_1(e) + Ch(e, f, g) + Kshared + in;
	h = t1 + bsg2_0(a) + Maj(a, b, c);
	d+= t1;

}

__device__ __forceinline__
static void sha256_round_first(uint32_t *const __restrict__ in,uint32_t *const __restrict__ buf,const uint32_t *const __restrict__ state,const uint32_t* __restrict__ Kshared)
{
	uint32_t a = buf[0] + in[11];
	uint32_t b = buf[1];
	uint32_t c = buf[2];
	uint32_t d = buf[3];
	uint32_t e = buf[4] + in[11];
	uint32_t f = buf[5];
	uint32_t g = buf[6];
	uint32_t h = buf[7];

	// 10 first steps made on host
	//sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[11]);

	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	//in is partially precomputed on host
	in[2]+= in[11];
	in[4]+= ssg2_1(in[2]);
	in[6]+= ssg2_1(in[4]);
	in[8]+= ssg2_1(in[6]);
	in[9]+= in[ 2];

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25]);

	#pragma unroll 6
	for (uint32_t j = 10; j < 16; j++){
		in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
	}

	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31]);

	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++){
		in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
	}

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16+16]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17+16]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18+16]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19+16]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20+16]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21+16]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22+16]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23+16]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24+16]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25+16]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26+16]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27+16]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28+16]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29+16]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30+16]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31+16]);

	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++){
		in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
	}

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16+16*2]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17+16*2]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18+16*2]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19+16*2]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20+16*2]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21+16*2]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22+16*2]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23+16*2]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24+16*2]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25+16*2]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26+16*2]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27+16*2]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28+16*2]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29+16*2]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30+16*2]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31+16*2]);

	buf[ 0] = state[0] + a;
	buf[ 1] = state[1] + b;
	buf[ 2] = state[2] + c;
	buf[ 3] = state[3] + d;
	buf[ 4] = state[4] + e;
	buf[ 5] = state[5] + f;
	buf[ 6] = state[6] + g;
	buf[ 7] = state[7] + h;
}

__device__ __forceinline__
static void sha256_round_body(uint32_t *const __restrict__ in, uint32_t *const __restrict__ state,const uint32_t *const __restrict__ Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	#pragma unroll 3
	for (uint32_t i=0; i<3; i++)
	{
		#pragma unroll 16
		for (uint32_t j = 0; j < 16; j++){
			in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
		}
		sha2_step(a, b, c, d, e, f, g, h, in[0], Kshared[16 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[1], Kshared[17 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[2], Kshared[18 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[3], Kshared[19 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[4], Kshared[20 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[5], Kshared[21 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[6], Kshared[22 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[7], Kshared[23 + 16 * i]);
		sha2_step(a, b, c, d, e, f, g, h, in[8], Kshared[24 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[9], Kshared[25 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__device__ __forceinline__
static void sha256_round_body_final(uint32_t *const __restrict__ in, uint32_t *const __restrict__ state,const uint32_t *const __restrict__ Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	#pragma unroll 2
	for (uint32_t i=0; i<2; i++)
	{
		#pragma unroll 16
		for (uint32_t j = 0; j < 16; j++){
			in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
		}
		sha2_step(a, b, c, d, e, f, g, h, in[0], Kshared[16 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[1], Kshared[17 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[2], Kshared[18 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[3], Kshared[19 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[4], Kshared[20 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[5], Kshared[21 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[6], Kshared[22 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[7], Kshared[23 + 16 * i]);
		sha2_step(a, b, c, d, e, f, g, h, in[8], Kshared[24 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[9], Kshared[25 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}
	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++){
		in[j] = in[j] + in[(j + 9) & 15] + ssg2_0(in[(j + 1) & 15]) + ssg2_1(in[(j + 14) & 15]);
	}
	sha2_step(a, b, c, d, e, f, g, h, in[0], Kshared[16 + 16 * 2]);
	sha2_step(h, a, b, c, d, e, f, g, in[1], Kshared[17 + 16 * 2]);
	sha2_step(g, h, a, b, c, d, e, f, in[2], Kshared[18 + 16 * 2]);
	sha2_step(f, g, h, a, b, c, d, e, in[3], Kshared[19 + 16 * 2]);
	sha2_step(e, f, g, h, a, b, c, d, in[4], Kshared[20 + 16 * 2]);
	sha2_step(d, e, f, g, h, a, b, c, in[5], Kshared[21 + 16 * 2]);
	sha2_step(c, d, e, f, g, h, a, b, in[6], Kshared[22 + 16 * 2]);
	sha2_step(b, c, d, e, f, g, h, a, in[7], Kshared[23 + 16 * 2]);
	sha2_step(a, b, c, d, e, f, g, h, in[8], Kshared[24 + 16 * 2]);
	sha2_step(h, a, b, c, d, e, f, g, in[9], Kshared[25 + 16 * 2]);
	sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * 2]);
	sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * 2]);
	sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * 2]);
	sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * 2]);

	state[6] += g;
	state[7] += h;
}

 __device__ __forceinline__
uint64_t cuda_swab64ll(const uint32_t x, const uint32_t y) {
	uint64_t r;
	asm("prmt.b32 %1, %1, 0, 0x0123; // swab64ll\n\t"
	    "prmt.b32 %2, %2, 0, 0x0123;\n\t"
	    "mov.b64 %0, {%1,%2};\n\t"
	  : "=l"(r): "r"(x), "r"(y) );
	return r;
}

__global__
#if CUDA_VERSION > 6050
__launch_bounds__(768,2) /* to force 32 regs */
#endif
void lbry_sha256d_gpu_hash_112(const uint32_t threads, const uint32_t startNonce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t buf[8], state[8];
	if (thread < threads)
	{
		uint32_t dat[16];

		#pragma unroll 11
		for (uint32_t i=0; i<11; i++) dat[i] = c_dataEnd112[i];
		dat[11] = startNonce + thread;
		dat[12] = 0x80000000;
		dat[13] = 0;
		dat[14] = 0;
		dat[15] = 0x380;

		*(uint2x4*)&state[0] = *(uint2x4*)&c_midstate112[0];
		*(uint2x4*)&buf[0]   = *(uint2x4*)&c_midbuffer112[0];

		sha256_round_first(dat, buf, state, c_K); // no shared mem here

		// second sha256

		*(uint2x4*)&dat[0] = *(uint2x4*)&buf[0];

		dat[8] = 0x80000000;

		#pragma unroll 6
		for (uint32_t i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		*(uint2x4*)&buf[0] = *(uint2x4*)&c_H256[0];

		sha256_round_body(dat, buf, c_K); //no shared mem at all

		// output
		*(uint2*)&buf[0] = vectorizeswap(((uint64_t*)buf)[0]);
		*(uint2*)&buf[2] = vectorizeswap(((uint64_t*)buf)[1]);
		*(uint2*)&buf[4] = vectorizeswap(((uint64_t*)buf)[2]);
		*(uint2*)&buf[6] = vectorizeswap(((uint64_t*)buf)[3]);

		*(uint2x4*)&outputHash[thread<<3] = *(uint2x4*)&buf[0];
	}
}

__host__
void lbry_sha256d_hash_112(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_outputHash){

	const int threadsperblock = 768;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha256d_gpu_hash_112 <<<grid, block>>> (threads, startNonce, (uint64_t*) d_outputHash);
}

__host__
void lbry_sha256_init(int thr_id)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_K), cpu_K, sizeof(cpu_K), 0, hipMemcpyHostToDevice);
}

__host__
void lbry_sha256_setBlock_112(uint32_t *pdata){

	uint32_t in[16], buf[8], end[16];
	for (int i=0;i<16;i++) in[i] = cuda_swab32(pdata[i]);
	for (int i=0; i<8;i++) buf[i] = cpu_H256[i];
	for (int i=0;i<11;i++) end[i] = cuda_swab32(pdata[16+i]);
	sha256_round_body_host(in, buf, cpu_K);

	hipMemcpyToSymbol(HIP_SYMBOL(c_midstate112), buf, 32, 0, hipMemcpyHostToDevice);

	uint32_t a = buf[0];
	uint32_t b = buf[1];
	uint32_t c = buf[2];
	uint32_t d = buf[3];
	uint32_t e = buf[4];
	uint32_t f = buf[5];
	uint32_t g = buf[6];
	uint32_t h = buf[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,end[0], cpu_K[0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,end[1], cpu_K[1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,end[2], cpu_K[2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,end[3], cpu_K[3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,end[4], cpu_K[4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,end[5], cpu_K[5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,end[6], cpu_K[6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,end[7], cpu_K[7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,end[8], cpu_K[8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,end[9], cpu_K[9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,end[10],cpu_K[10]);
	sha256_step1_host(f, g, h, a, b, c, d, e, 0, cpu_K[11]);

	buf[0] = a;
	buf[1] = b;
	buf[2] = c;
	buf[3] = d;
	buf[4] = e;
	buf[5] = f;
	buf[6] = g;
	buf[7] = h;

	hipMemcpyToSymbol(HIP_SYMBOL(c_midbuffer112), buf, 32, 0, hipMemcpyHostToDevice);

	end[12] = 0x80000000;
	end[13] = 0;
	end[14] = 0;
	end[15] = 0x380;
	uint32_t x2_0,x2_1;

	x2_0 = ROTR32(end[1], 7) ^ ROTR32(end[1], 18) ^ SPH_T32(end[1] >> 3); //ssg2_0(end[1]);
//	x2_1 = ROTR32(end[14], 17) ^ ROTR32(end[14], 19) ^ SPH_T32(end[14] >> 10) + x2_0; //ssg2_1(end[14]) + x2_0;
	end[0] = end[0] + end[9] + x2_0;

	x2_0 = ROTR32(end[2], 7) ^ ROTR32(end[2], 18) ^ SPH_T32(end[2] >> 3);
	x2_1 = (ROTR32(end[15], 17) ^ ROTR32(end[15], 19) ^ SPH_T32(end[15] >> 10)) + x2_0;
	end[1] = end[1] + end[10] + x2_1;

	x2_0 = ROTR32(end[3], 7) ^ ROTR32(end[3], 18) ^ SPH_T32(end[3] >> 3);//ssg2_0(end[3]);
	x2_1 = (ROTR32(end[0], 17) ^ ROTR32(end[0], 19) ^ SPH_T32(end[0] >> 10)) + x2_0;
	end[2]+= x2_1;

	x2_0 = ROTR32(end[4], 7) ^ ROTR32(end[4], 18) ^ SPH_T32(end[4] >> 3);//ssg2_0(end[4]);
	x2_1 = (ROTR32(end[1], 17) ^ ROTR32(end[1], 19) ^ SPH_T32(end[1] >> 10)) + x2_0;
	end[3] = end[3] + end[12] + x2_1;

	x2_0 = ROTR32(end[5], 7) ^ ROTR32(end[5], 18) ^ SPH_T32(end[5] >> 3);//ssg2_0(end[4]);
	end[4] = end[4] + end[13] + x2_0;

	x2_0 = ROTR32(end[6], 7) ^ ROTR32(end[6], 18) ^ SPH_T32(end[6] >> 3);//ssg2_0(end[6]);
	x2_1 = (ROTR32(end[3], 17) ^ ROTR32(end[3], 19) ^ SPH_T32(end[3] >> 10)) + x2_0;
	end[5] = end[5] + end[14] + x2_1;

	x2_0 = ROTR32(end[7], 7) ^ ROTR32(end[7], 18) ^ SPH_T32(end[7] >> 3);//ssg2_0(end[7]);
	end[6] = end[6] + end[15] + x2_0;

	x2_0 = ROTR32(end[8], 7) ^ ROTR32(end[8], 18) ^ SPH_T32(end[8] >> 3);//ssg2_0(end[8]);
	x2_1 = (ROTR32(end[5], 17) ^ ROTR32(end[5], 19) ^ SPH_T32(end[5] >> 10)) + x2_0;
	end[7] = end[7] + end[0] + x2_1;

	x2_0 = ROTR32(end[9], 7) ^ ROTR32(end[9], 18) ^ SPH_T32(end[9] >> 3);//ssg2_0(end[9]);
	end[8] = end[8] + end[1] + x2_0;

	x2_0 = ROTR32(end[10], 7) ^ ROTR32(end[10], 18) ^ SPH_T32(end[10] >> 3);//ssg2_0(end[10]);
	x2_1 = (ROTR32(end[7], 17) ^ ROTR32(end[7], 19) ^ SPH_T32(end[7] >> 10)) + x2_0;
	end[9] = end[9] + x2_1;

	hipMemcpyToSymbol(HIP_SYMBOL(c_dataEnd112),  end, 12*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

// ------------------------------------------------------------------------------------------

static __constant__ const uint32_t c_IV[5] = { 0x67452301u, 0xEFCDAB89u, 0x98BADCFEu, 0x10325476u, 0xC3D2E1F0u };

static __constant__ const uint32_t c_K1[5] = { 0, 0x5A827999, 0x6ED9EBA1, 0x8F1BBCDC, 0xA953FD4E };

static __constant__ const uint32_t c_K2[5] = { 0x50A28BE6, 0x5C4DD124, 0x6D703EF3, 0x7A6D76E9, 0 };

__device__ __forceinline__
static uint32_t ROTATE(const uint32_t x,const uint32_t r){
	if(r==8)
		return __byte_perm(x, 0, 0x2103);
	else
		return ROTL32(x,r);
}

/*
 * Round functions for RIPEMD-160.
 */
#define F1(x, y, z)   (x ^ y ^ z)
#define F2(x, y, z)   ((x & (y ^ z)) ^ z)
#define F3(x, y, z)   ((x | ~y) ^ z)
#define F4(x, y, z)   (y ^ ((x ^ y) & z))
#define F5(x, y, z)   (x ^ (y | ~z))

/*
 * Round constants for RIPEMD-160.
 */
#define RR(a, b, c, d, e, f, s, r, k) { \
	a = e + ROTATE((a + r + k + f(b, c, d)), s); \
	c = ROTL32(c, 10); \
}

#define ROUND1(a, b, c, d, e, f, s, r, k) \
	RR(a[0], b[0], c[0], d[0], e[0], f, s, r, c_K1[k])

#define ROUND2(a, b, c, d, e, f, s, r, k) \
	RR(a[1], b[1], c[1], d[1], e[1], f, s, r, c_K2[k])

#define RIPEMD160_ROUND_BODY(in, h) { \
	uint32_t A[2], B[2], C[2], D[2], E[2]; \
	uint32_t tmp; \
\
	A[0] = A[1] = h[0]; \
	B[0] = B[1] = h[1]; \
	C[0] = C[1] = h[2]; \
	D[0] = D[1] = h[3]; \
	E[0] = E[1] = h[4]; \
\
	ROUND1(A, B, C, D, E, F1, 11, in[ 0],  0); \
	ROUND1(E, A, B, C, D, F1, 14, in[ 1],  0); \
	ROUND1(D, E, A, B, C, F1, 15, in[ 2],  0); \
	ROUND1(C, D, E, A, B, F1, 12, in[ 3],  0); \
	ROUND1(B, C, D, E, A, F1,  5, in[ 4],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[ 5],  0); \
	ROUND1(E, A, B, C, D, F1,  7, in[ 6],  0); \
	ROUND1(D, E, A, B, C, F1,  9, in[ 7],  0); \
	ROUND1(C, D, E, A, B, F1, 11, in[ 8],  0); \
	ROUND1(B, C, D, E, A, F1, 13, in[ 9],  0); \
	ROUND1(A, B, C, D, E, F1, 14, in[10],  0); \
	ROUND1(E, A, B, C, D, F1, 15, in[11],  0); \
	ROUND1(D, E, A, B, C, F1,  6, in[12],  0); \
	ROUND1(C, D, E, A, B, F1,  7, in[13],  0); \
	ROUND1(B, C, D, E, A, F1,  9, in[14],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[15],  0); \
\
	ROUND1(E, A, B, C, D, F2,  7, in[ 7],  1); \
	ROUND1(D, E, A, B, C, F2,  6, in[ 4],  1); \
	ROUND1(C, D, E, A, B, F2,  8, in[13],  1); \
	ROUND1(B, C, D, E, A, F2, 13, in[ 1],  1); \
	ROUND1(A, B, C, D, E, F2, 11, in[10],  1); \
	ROUND1(E, A, B, C, D, F2,  9, in[ 6],  1); \
	ROUND1(D, E, A, B, C, F2,  7, in[15],  1); \
	ROUND1(C, D, E, A, B, F2, 15, in[ 3],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[12],  1); \
	ROUND1(A, B, C, D, E, F2, 12, in[ 0],  1); \
	ROUND1(E, A, B, C, D, F2, 15, in[ 9],  1); \
	ROUND1(D, E, A, B, C, F2,  9, in[ 5],  1); \
	ROUND1(C, D, E, A, B, F2, 11, in[ 2],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[14],  1); \
	ROUND1(A, B, C, D, E, F2, 13, in[11],  1); \
	ROUND1(E, A, B, C, D, F2, 12, in[ 8],  1); \
\
	ROUND1(D, E, A, B, C, F3, 11, in[ 3],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[10],  2); \
	ROUND1(B, C, D, E, A, F3,  6, in[14],  2); \
	ROUND1(A, B, C, D, E, F3,  7, in[ 4],  2); \
	ROUND1(E, A, B, C, D, F3, 14, in[ 9],  2); \
	ROUND1(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[ 8],  2); \
	ROUND1(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND1(A, B, C, D, E, F3, 14, in[ 2],  2); \
	ROUND1(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND1(D, E, A, B, C, F3, 13, in[ 0],  2); \
	ROUND1(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND1(B, C, D, E, A, F3,  5, in[13],  2); \
	ROUND1(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND1(E, A, B, C, D, F3,  7, in[ 5],  2); \
	ROUND1(D, E, A, B, C, F3,  5, in[12],  2); \
\
	ROUND1(C, D, E, A, B, F4, 11, in[ 1],  3); \
	ROUND1(B, C, D, E, A, F4, 12, in[ 9],  3); \
	ROUND1(A, B, C, D, E, F4, 14, in[11],  3); \
	ROUND1(E, A, B, C, D, F4, 15, in[10],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 0],  3); \
	ROUND1(C, D, E, A, B, F4, 15, in[ 8],  3); \
	ROUND1(B, C, D, E, A, F4,  9, in[12],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[ 4],  3); \
	ROUND1(E, A, B, C, D, F4,  9, in[13],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 3],  3); \
	ROUND1(C, D, E, A, B, F4,  5, in[ 7],  3); \
	ROUND1(B, C, D, E, A, F4,  6, in[15],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[14],  3); \
	ROUND1(E, A, B, C, D, F4,  6, in[ 5],  3); \
	ROUND1(D, E, A, B, C, F4,  5, in[ 6],  3); \
	ROUND1(C, D, E, A, B, F4, 12, in[ 2],  3); \
\
	ROUND1(B, C, D, E, A, F5,  9, in[ 4],  4); \
	ROUND1(A, B, C, D, E, F5, 15, in[ 0],  4); \
	ROUND1(E, A, B, C, D, F5,  5, in[ 5],  4); \
	ROUND1(D, E, A, B, C, F5, 11, in[ 9],  4); \
	ROUND1(C, D, E, A, B, F5,  6, in[ 7],  4); \
	ROUND1(B, C, D, E, A, F5,  8, in[12],  4); \
	ROUND1(A, B, C, D, E, F5, 13, in[ 2],  4); \
	ROUND1(E, A, B, C, D, F5, 12, in[10],  4); \
	ROUND1(D, E, A, B, C, F5,  5, in[14],  4); \
	ROUND1(C, D, E, A, B, F5, 12, in[ 1],  4); \
	ROUND1(B, C, D, E, A, F5, 13, in[ 3],  4); \
	ROUND1(A, B, C, D, E, F5, 14, in[ 8],  4); \
	ROUND1(E, A, B, C, D, F5, 11, in[11],  4); \
	ROUND1(D, E, A, B, C, F5,  8, in[ 6],  4); \
	ROUND1(C, D, E, A, B, F5,  5, in[15],  4); \
	ROUND1(B, C, D, E, A, F5,  6, in[13],  4); \
\
	ROUND2(A, B, C, D, E, F5,  8, in[ 5],  0); \
	ROUND2(E, A, B, C, D, F5,  9, in[14],  0); \
	ROUND2(D, E, A, B, C, F5,  9, in[ 7],  0); \
	ROUND2(C, D, E, A, B, F5, 11, in[ 0],  0); \
	ROUND2(B, C, D, E, A, F5, 13, in[ 9],  0); \
	ROUND2(A, B, C, D, E, F5, 15, in[ 2],  0); \
	ROUND2(E, A, B, C, D, F5, 15, in[11],  0); \
	ROUND2(D, E, A, B, C, F5,  5, in[ 4],  0); \
	ROUND2(C, D, E, A, B, F5,  7, in[13],  0); \
	ROUND2(B, C, D, E, A, F5,  7, in[ 6],  0); \
	ROUND2(A, B, C, D, E, F5,  8, in[15],  0); \
	ROUND2(E, A, B, C, D, F5, 11, in[ 8],  0); \
	ROUND2(D, E, A, B, C, F5, 14, in[ 1],  0); \
	ROUND2(C, D, E, A, B, F5, 14, in[10],  0); \
	ROUND2(B, C, D, E, A, F5, 12, in[ 3],  0); \
	ROUND2(A, B, C, D, E, F5,  6, in[12],  0); \
\
	ROUND2(E, A, B, C, D, F4,  9, in[ 6],  1); \
	ROUND2(D, E, A, B, C, F4, 13, in[11],  1); \
	ROUND2(C, D, E, A, B, F4, 15, in[ 3],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[ 7],  1); \
	ROUND2(A, B, C, D, E, F4, 12, in[ 0],  1); \
	ROUND2(E, A, B, C, D, F4,  8, in[13],  1); \
	ROUND2(D, E, A, B, C, F4,  9, in[ 5],  1); \
	ROUND2(C, D, E, A, B, F4, 11, in[10],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[14],  1); \
	ROUND2(A, B, C, D, E, F4,  7, in[15],  1); \
	ROUND2(E, A, B, C, D, F4, 12, in[ 8],  1); \
	ROUND2(D, E, A, B, C, F4,  7, in[12],  1); \
	ROUND2(C, D, E, A, B, F4,  6, in[ 4],  1); \
	ROUND2(B, C, D, E, A, F4, 15, in[ 9],  1); \
	ROUND2(A, B, C, D, E, F4, 13, in[ 1],  1); \
	ROUND2(E, A, B, C, D, F4, 11, in[ 2],  1); \
\
	ROUND2(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND2(C, D, E, A, B, F3,  7, in[ 5],  2); \
	ROUND2(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND2(A, B, C, D, E, F3, 11, in[ 3],  2); \
	ROUND2(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND2(D, E, A, B, C, F3,  6, in[14],  2); \
	ROUND2(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND2(B, C, D, E, A, F3, 14, in[ 9],  2); \
	ROUND2(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND2(E, A, B, C, D, F3, 13, in[ 8],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[12],  2); \
	ROUND2(C, D, E, A, B, F3, 14, in[ 2],  2); \
	ROUND2(B, C, D, E, A, F3, 13, in[10],  2); \
	ROUND2(A, B, C, D, E, F3, 13, in[ 0],  2); \
	ROUND2(E, A, B, C, D, F3,  7, in[ 4],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[13],  2); \
\
	ROUND2(C, D, E, A, B, F2, 15, in[ 8],  3); \
	ROUND2(B, C, D, E, A, F2,  5, in[ 6],  3); \
	ROUND2(A, B, C, D, E, F2,  8, in[ 4],  3); \
	ROUND2(E, A, B, C, D, F2, 11, in[ 1],  3); \
	ROUND2(D, E, A, B, C, F2, 14, in[ 3],  3); \
	ROUND2(C, D, E, A, B, F2, 14, in[11],  3); \
	ROUND2(B, C, D, E, A, F2,  6, in[15],  3); \
	ROUND2(A, B, C, D, E, F2, 14, in[ 0],  3); \
	ROUND2(E, A, B, C, D, F2,  6, in[ 5],  3); \
	ROUND2(D, E, A, B, C, F2,  9, in[12],  3); \
	ROUND2(C, D, E, A, B, F2, 12, in[ 2],  3); \
	ROUND2(B, C, D, E, A, F2,  9, in[13],  3); \
	ROUND2(A, B, C, D, E, F2, 12, in[ 9],  3); \
	ROUND2(E, A, B, C, D, F2,  5, in[ 7],  3); \
	ROUND2(D, E, A, B, C, F2, 15, in[10],  3); \
	ROUND2(C, D, E, A, B, F2,  8, in[14],  3); \
\
	ROUND2(B, C, D, E, A, F1,  8, in[12],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[15],  4); \
	ROUND2(E, A, B, C, D, F1, 12, in[10],  4); \
	ROUND2(D, E, A, B, C, F1,  9, in[ 4],  4); \
	ROUND2(C, D, E, A, B, F1, 12, in[ 1],  4); \
	ROUND2(B, C, D, E, A, F1,  5, in[ 5],  4); \
	ROUND2(A, B, C, D, E, F1, 14, in[ 8],  4); \
	ROUND2(E, A, B, C, D, F1,  6, in[ 7],  4); \
	ROUND2(D, E, A, B, C, F1,  8, in[ 6],  4); \
	ROUND2(C, D, E, A, B, F1, 13, in[ 2],  4); \
	ROUND2(B, C, D, E, A, F1,  6, in[13],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[14],  4); \
	ROUND2(E, A, B, C, D, F1, 15, in[ 0],  4); \
	ROUND2(D, E, A, B, C, F1, 13, in[ 3],  4); \
	ROUND2(C, D, E, A, B, F1, 11, in[ 9],  4); \
	ROUND2(B, C, D, E, A, F1, 11, in[11],  4); \
\
	tmp  = h[1] + C[0] + D[1]; \
	h[1] = h[2] + D[0] + E[1]; \
	h[2] = h[3] + E[0] + A[1]; \
	h[3] = h[4] + A[0] + B[1]; \
	h[4] = h[0] + B[0] + C[1]; \
	h[0] = tmp; \
}

__global__
#if CUDA_VERSION > 6050
__launch_bounds__(1024,2) /* to force 32 regs */
#endif
void lbry_ripemd(const uint32_t threads, uint64_t *Hash512){
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t dat[16];
	uint32_t h[5];
	uint32_t buf[8]; // align for vectorize
	if(thread<threads){

		uint32_t* input = (uint32_t*) (&Hash512[thread<<3]);

		*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[0]);

		dat[8] = 0x80;

		#pragma unroll 7
		for (int i=9;i<16;i++) dat[i] = 0;

		dat[14] = 0x100; // size in bits

		#pragma unroll 5
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(dat, h);

		#pragma unroll 5
		for (int i=0; i<5; i++)
			buf[i] = h[i];

		// second 32 bytes block hash

		*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[8]);

		dat[8] = 0x80;

		#pragma unroll 7
		for (int i=9;i<16;i++) dat[i] = 0;

		dat[14] = 0x100; // size in bits

		#pragma unroll 5
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(dat, h);

		#pragma unroll 5
		for (int i=0;i<5;i++) dat[i] = cuda_swab32(buf[i]);
		#pragma unroll 5
		for (int i=0;i<5;i++) dat[i+5] = cuda_swab32(h[i]);

		*(uint2x4*)&input[ 0] = *(uint2x4*)&dat[ 0];
		*(uint2*)&input[ 8] = *(uint2*)&dat[ 8];
	}
}

__global__
#if CUDA_VERSION > 6050
__launch_bounds__(768,2) /* to force 32 regs */
#endif
void lbry_sha256d_gpu_hash_final(const uint32_t threads, uint64_t *Hash512, uint32_t *resNonces,const uint64_t target64)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t dat[16];
	uint32_t buf[8]; // align for vectorize
	if (thread < threads)
	{
		// first final sha256

		uint32_t* input = (uint32_t*) (&Hash512[thread<<3]);

		*(uint2x4*)&dat[0] = __ldg4((uint2x4*)&input[0]);
		*(uint2*)&dat[8] = __ldg((uint2*)&input[8]);
		dat[10] = 0x80000000;
		#pragma unroll 4
		for (int i=11; i<15; i++) dat[i] = 0;
		dat[15] = 0x140;

		*(uint2x4*)&buf[0] = *(uint2x4*)&c_H256[0];

		sha256_round_body(dat, buf, c_K); // s_K uses too many regs

		// second sha256

		*(uint2x4*)&dat[0] = *(uint2x4*)&buf[0];
		*(uint2x4*)&buf[0] = *(uint2x4*)&c_H256[0];

		dat[8] = 0x80000000;

		#pragma unroll 6
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		sha256_round_body_final(dat, buf, c_K);

		// valid nonces
		if (cuda_swab64ll(buf[ 6],buf[ 7]) <= target64) {
			uint32_t tmp = atomicExch(&resNonces[0], thread);
			if (tmp != UINT32_MAX)
				resNonces[1] = tmp;
		}
	}
}

__host__
void lbry_sha256d_hash_final(int thr_id, uint32_t threads, uint32_t *d_inputHash, uint32_t *d_resNonce, const uint64_t target64)
{
	int threadsperblock;

	threadsperblock = 1024;
	dim3 grid1((threads + threadsperblock - 1) / threadsperblock);
	dim3 block1(threadsperblock);

	threadsperblock = 768;
	dim3 grid2((threads + threadsperblock - 1) / threadsperblock);
	dim3 block2(threadsperblock);
	lbry_ripemd <<<grid1, block1>>> (threads,(uint64_t*) d_inputHash);
	lbry_sha256d_gpu_hash_final <<<grid2, block2>>> (threads, (uint64_t*) d_inputHash, d_resNonce, target64);
}
