#include "hip/hip_runtime.h"
/*
 * LBRY merged kernel CUDA implementation.
 * For compute 5.2 and beyond gpus
 * tpruvot and Provos Alexis - Sep 2016
 * Sponsored by LBRY.IO team
 */

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>

#include <miner.h>

__constant__ static uint32_t _ALIGN(16) c_midstate112[8];
__constant__ static uint32_t _ALIGN(16) c_midbuffer112[8];
__constant__ static uint32_t _ALIGN(16) c_dataEnd112[12];

__constant__ static const uint32_t c_H256[8] = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};
__constant__ static uint32_t _ALIGN(8) c_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#ifdef __INTELLISENSE__
#define atomicExch(p,y) y
#define __byte_perm(x,y,z) x
#endif

// ------------------------------------------------------------------------------------------------

static const uint32_t cpu_H256[8] = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t cpu_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

__host__
static void sha256_step1_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d, uint32_t e, uint32_t f, uint32_t g, uint32_t &h, uint32_t in, const uint32_t Kshared)
{
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);
	uint32_t t1 = h + bsg21 + vxandx + Kshared + in;
	uint32_t t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__host__
static void sha256_step2_host(uint32_t a, uint32_t b, uint32_t c, uint32_t &d,
	uint32_t e, uint32_t f, uint32_t g, uint32_t &h, uint32_t* in, uint32_t pc, const uint32_t Kshared)
{
	int pcidx1 = (pc-2)  & 0xF;
	int pcidx2 = (pc-7)  & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];

	uint32_t ssg21 = ROTR32(inx1, 17) ^ ROTR32(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
	uint32_t ssg20 = ROTR32(inx3, 7) ^ ROTR32(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 = ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25); // bsg2_1(e);
	uint32_t bsg20 = ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22); //bsg2_0(a);
	uint32_t andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);
	uint32_t t1,t2;

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__host__
static void sha256_round_body_host(uint32_t* in, uint32_t* state, const uint32_t* Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,in[0], Kshared[0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[1], Kshared[1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[2], Kshared[2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[3], Kshared[3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[4], Kshared[4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[5], Kshared[5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[6], Kshared[6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[7], Kshared[7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,in[8], Kshared[8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[9], Kshared[9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[10],Kshared[10]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[11],Kshared[11]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	for (int i=0; i<3; i++)
	{
		sha256_step2_host(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha256_step2_host(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__host__
void lbry_sha256_setBlock_112_merged(uint32_t *pdata)
{
	uint32_t in[16], buf[8], end[16];
	for (int i=0;i<16;i++) in[i] = cuda_swab32(pdata[i]);
	for (int i=0; i<8;i++) buf[i] = cpu_H256[i];
	for (int i=0;i<11;i++) end[i] = cuda_swab32(pdata[16+i]);
	sha256_round_body_host(in, buf, cpu_K);

	hipMemcpyToSymbol(HIP_SYMBOL(c_midstate112), buf, 32, 0, hipMemcpyHostToDevice);

	uint32_t a = buf[0];
	uint32_t b = buf[1];
	uint32_t c = buf[2];
	uint32_t d = buf[3];
	uint32_t e = buf[4];
	uint32_t f = buf[5];
	uint32_t g = buf[6];
	uint32_t h = buf[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,end[0], cpu_K[0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,end[1], cpu_K[1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,end[2], cpu_K[2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,end[3], cpu_K[3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,end[4], cpu_K[4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,end[5], cpu_K[5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,end[6], cpu_K[6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,end[7], cpu_K[7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,end[8], cpu_K[8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,end[9], cpu_K[9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,end[10],cpu_K[10]);
	sha256_step1_host(f, g, h, a, b, c, d, e, 0, cpu_K[11]);

	buf[0] = a;
	buf[1] = b;
	buf[2] = c;
	buf[3] = d;
	buf[4] = e;
	buf[5] = f;
	buf[6] = g;
	buf[7] = h;

	hipMemcpyToSymbol(HIP_SYMBOL(c_midbuffer112), buf, 32, 0, hipMemcpyHostToDevice);

	end[12] = 0x80000000;
	end[13] = 0;
	end[14] = 0;
	end[15] = 0x380;
	uint32_t x2_0,x2_1;

	x2_0 = ROTR32(end[1], 7) ^ ROTR32(end[1], 18) ^ SPH_T32(end[1] >> 3); //ssg2_0(inx3);//ssg2_0(end[1]);
//	x2_1 = ROTR32(end[14], 17) ^ ROTR32(end[14], 19) ^ SPH_T32(end[14] >> 10) + x2_0; //ssg2_1(inx1); ssg2_1(end[14]) + x2_0;
	end[0] = end[0] + end[9] + x2_0;

	x2_0 = ROTR32(end[2], 7) ^ ROTR32(end[2], 18) ^ SPH_T32(end[2] >> 3);
	x2_1 = (ROTR32(end[15], 17) ^ ROTR32(end[15], 19) ^ SPH_T32(end[15] >> 10)) + x2_0;
	end[1] = end[1] + end[10] + x2_1;

	x2_0 = ROTR32(end[3], 7) ^ ROTR32(end[3], 18) ^ SPH_T32(end[3] >> 3);//ssg2_0(end[3]);
	x2_1 = (ROTR32(end[0], 17) ^ ROTR32(end[0], 19) ^ SPH_T32(end[0] >> 10)) + x2_0;
	end[2]+= x2_1;

	x2_0 = ROTR32(end[4], 7) ^ ROTR32(end[4], 18) ^ SPH_T32(end[4] >> 3);//ssg2_0(end[4]);
	x2_1 = (ROTR32(end[1], 17) ^ ROTR32(end[1], 19) ^ SPH_T32(end[1] >> 10)) + x2_0;
	end[3] = end[3] + end[12] + x2_1;

	x2_0 = ROTR32(end[5], 7) ^ ROTR32(end[5], 18) ^ SPH_T32(end[5] >> 3);//ssg2_0(end[4]);
	end[4] = end[4] + end[13] + x2_0;

	x2_0 = ROTR32(end[6], 7) ^ ROTR32(end[6], 18) ^ SPH_T32(end[6] >> 3);//ssg2_0(end[6]);
	x2_1 = (ROTR32(end[3], 17) ^ ROTR32(end[3], 19) ^ SPH_T32(end[3] >> 10)) + x2_0;
	end[5] = end[5] + end[14] + x2_1;

	x2_0 = ROTR32(end[7], 7) ^ ROTR32(end[7], 18) ^ SPH_T32(end[7] >> 3);//ssg2_0(end[7]);
	end[6] = end[6] + end[15] + x2_0;

	x2_0 = ROTR32(end[8], 7) ^ ROTR32(end[8], 18) ^ SPH_T32(end[8] >> 3);//ssg2_0(end[8]);
	x2_1 = (ROTR32(end[5], 17) ^ ROTR32(end[5], 19) ^ SPH_T32(end[5] >> 10)) + x2_0;
	end[7] = end[7] + end[0] + x2_1;

	x2_0 = ROTR32(end[9], 7) ^ ROTR32(end[9], 18) ^ SPH_T32(end[9] >> 3);//ssg2_0(end[9]);
	end[8] = end[8] + end[1] + x2_0;

	x2_0 = ROTR32(end[10], 7) ^ ROTR32(end[10], 18) ^ SPH_T32(end[10] >> 3);//ssg2_0(end[10]);
	x2_1 = (ROTR32(end[7], 17) ^ ROTR32(end[7], 19) ^ SPH_T32(end[7] >> 10)) + x2_0;
	end[9] = end[9] + x2_1;

	hipMemcpyToSymbol(HIP_SYMBOL(c_dataEnd112),  end, sizeof(end), 0, hipMemcpyHostToDevice);
}

//END OF HOST FUNCTIONS -------------------------------------------------------------------

//SHA256 MACROS ---------------------------------------------------------------------------

#define xor3b(a,b,c) (a ^ b ^ c)

__device__ __forceinline__ uint32_t bsg2_0(const uint32_t x){
	return xor3b(ROTR32(x,2),ROTR32(x,13),ROTR32(x,22));
}

__device__ __forceinline__ uint32_t bsg2_1(const uint32_t x){
	return xor3b(ROTR32(x,6),ROTR32(x,11),ROTR32(x,25));
}

__device__ __forceinline__ uint32_t ssg2_0(const uint32_t x){
	return xor3b(ROTR32(x,7),ROTR32(x,18),(x>>3));
}

__device__ __forceinline__ uint32_t ssg2_1(const uint32_t x){
	return xor3b(ROTR32(x,17),ROTR32(x,19),(x>>10));
}

__device__ __forceinline__ uint64_t vectorizeswap(const uint64_t v){
	uint2 result;
	asm volatile ("mov.b64 {%0,%1},%2;" : "=r"(result.y), "=r"(result.x) : "l"(v));
	return devectorize(result);
}

#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define Ch(a, b, c)     (((b^c) & a) ^ c)

__device__
static void sha2_step(const uint32_t a, const uint32_t b,const uint32_t c, uint32_t &d,
	const uint32_t e,const uint32_t f,const uint32_t g, uint32_t &h,
	const uint32_t in, const uint32_t Kshared)
{
	const uint32_t t1 = h + bsg2_1(e) + Ch(e, f, g) + Kshared + in;
	h = t1 + Maj(a, b, c) + bsg2_0(a);
	d+= t1;
}

__device__
static void sha256_round_first(uint32_t *in, uint32_t *buf,
	const uint32_t *state, const uint32_t* __restrict__ Kshared)
{
	uint32_t a = buf[0] + in[11];
	uint32_t b = buf[1];
	uint32_t c = buf[2];
	uint32_t d = buf[3];
	uint32_t e = buf[4] + in[11];
	uint32_t f = buf[5];
	uint32_t g = buf[6];
	uint32_t h = buf[7];

	// 10 first steps made on host
	//sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[11]);

	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[15]);

	//in is partially precomputed on host
	in[2]+= in[11];
	in[4]+= ssg2_1(in[2]);
	in[6]+= ssg2_1(in[4]);
	in[8]+= ssg2_1(in[6]);
	in[9]+= in[ 2];

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25]);

	#pragma unroll 6
	for (uint32_t j = 10; j < 16; j++) {
		const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
		const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
		in[j] = in[j] + in[(j + 9) & 15] + x2_1;
	}

	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31]);

	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++) {
		const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
		const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
		in[j] = in[j] + in[(j + 9) & 15] + x2_1;
	}

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16+16]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17+16]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18+16]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19+16]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20+16]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21+16]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22+16]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23+16]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24+16]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25+16]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26+16]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27+16]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28+16]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29+16]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30+16]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31+16]);

	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++) {
		const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
		const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
		in[j] = in[j] + in[(j + 9) & 15] + x2_1;
	}

	sha2_step(a,b,c,d,e,f,g,h,in[0], Kshared[16+16*2]);
	sha2_step(h,a,b,c,d,e,f,g,in[1], Kshared[17+16*2]);
	sha2_step(g,h,a,b,c,d,e,f,in[2], Kshared[18+16*2]);
	sha2_step(f,g,h,a,b,c,d,e,in[3], Kshared[19+16*2]);
	sha2_step(e,f,g,h,a,b,c,d,in[4], Kshared[20+16*2]);
	sha2_step(d,e,f,g,h,a,b,c,in[5], Kshared[21+16*2]);
	sha2_step(c,d,e,f,g,h,a,b,in[6], Kshared[22+16*2]);
	sha2_step(b,c,d,e,f,g,h,a,in[7], Kshared[23+16*2]);
	sha2_step(a,b,c,d,e,f,g,h,in[8], Kshared[24+16*2]);
	sha2_step(h,a,b,c,d,e,f,g,in[9], Kshared[25+16*2]);
	sha2_step(g,h,a,b,c,d,e,f,in[10],Kshared[26+16*2]);
	sha2_step(f,g,h,a,b,c,d,e,in[11],Kshared[27+16*2]);
	sha2_step(e,f,g,h,a,b,c,d,in[12],Kshared[28+16*2]);
	sha2_step(d,e,f,g,h,a,b,c,in[13],Kshared[29+16*2]);
	sha2_step(c,d,e,f,g,h,a,b,in[14],Kshared[30+16*2]);
	sha2_step(b,c,d,e,f,g,h,a,in[15],Kshared[31+16*2]);

	buf[ 0] = state[0] + a;
	buf[ 1] = state[1] + b;
	buf[ 2] = state[2] + c;
	buf[ 3] = state[3] + d;
	buf[ 4] = state[4] + e;
	buf[ 5] = state[5] + f;
	buf[ 6] = state[6] + g;
	buf[ 7] = state[7] + h;
}

__device__
static void sha256_round_body(uint32_t *in, uint32_t *state,const uint32_t* Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step(a,b,c,d,e,f,g,h,in[ 0], Kshared[ 0]);
	sha2_step(h,a,b,c,d,e,f,g,in[ 1], Kshared[ 1]);
	sha2_step(g,h,a,b,c,d,e,f,in[ 2], Kshared[ 2]);
	sha2_step(f,g,h,a,b,c,d,e,in[ 3], Kshared[ 3]);
	sha2_step(e,f,g,h,a,b,c,d,in[ 4], Kshared[ 4]);
	sha2_step(d,e,f,g,h,a,b,c,in[ 5], Kshared[ 5]);
	sha2_step(c,d,e,f,g,h,a,b,in[ 6], Kshared[ 6]);
	sha2_step(b,c,d,e,f,g,h,a,in[ 7], Kshared[ 7]);
	sha2_step(a,b,c,d,e,f,g,h,in[ 8], Kshared[ 8]);
	sha2_step(h,a,b,c,d,e,f,g,in[ 9], Kshared[ 9]);
	sha2_step(g,h,a,b,c,d,e,f,in[10], Kshared[10]);
	sha2_step(f,g,h,a,b,c,d,e,in[11], Kshared[11]);
	sha2_step(e,f,g,h,a,b,c,d,in[12], Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13], Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14], Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15], Kshared[15]);

	#pragma unroll 3
	for (int i=0; i<3; i++)
	{
		#pragma unroll 16
		for (uint32_t j = 0; j < 16; j++) {
			const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
			const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
			in[j] = in[j] + in[(j + 9) & 15] + x2_1;
		}
		sha2_step(a, b, c, d, e, f, g, h, in[ 0], Kshared[16 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[ 1], Kshared[17 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[ 2], Kshared[18 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[ 3], Kshared[19 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[ 4], Kshared[20 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[ 5], Kshared[21 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[ 6], Kshared[22 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[ 7], Kshared[23 + 16 * i]);
		sha2_step(a, b, c, d, e, f, g, h, in[ 8], Kshared[24 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[ 9], Kshared[25 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

__device__
static void sha256_round_body_final(uint32_t *in, uint32_t *state, const uint32_t *Kshared)
{
	uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];
	uint32_t e = state[4];
	uint32_t f = state[5];
	uint32_t g = state[6];
	uint32_t h = state[7];

	sha2_step(a,b,c,d,e,f,g,h,in[ 0], Kshared[0]);
	sha2_step(h,a,b,c,d,e,f,g,in[ 1], Kshared[1]);
	sha2_step(g,h,a,b,c,d,e,f,in[ 2], Kshared[2]);
	sha2_step(f,g,h,a,b,c,d,e,in[ 3], Kshared[3]);
	sha2_step(e,f,g,h,a,b,c,d,in[ 4], Kshared[4]);
	sha2_step(d,e,f,g,h,a,b,c,in[ 5], Kshared[5]);
	sha2_step(c,d,e,f,g,h,a,b,in[ 6], Kshared[6]);
	sha2_step(b,c,d,e,f,g,h,a,in[ 7], Kshared[7]);
	sha2_step(a,b,c,d,e,f,g,h,in[ 8], Kshared[8]);
	sha2_step(h,a,b,c,d,e,f,g,in[ 9], Kshared[9]);
	sha2_step(g,h,a,b,c,d,e,f,in[10], Kshared[10]);
	sha2_step(f,g,h,a,b,c,d,e,in[11], Kshared[11]);
	sha2_step(e,f,g,h,a,b,c,d,in[12], Kshared[12]);
	sha2_step(d,e,f,g,h,a,b,c,in[13], Kshared[13]);
	sha2_step(c,d,e,f,g,h,a,b,in[14], Kshared[14]);
	sha2_step(b,c,d,e,f,g,h,a,in[15], Kshared[15]);

	#pragma unroll 2
	for (int i=0; i<2; i++)
	{
		#pragma unroll 16
		for (uint32_t j = 0; j < 16; j++) {
			const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
			const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
			in[j] = in[j] + in[(j + 9) & 15] + x2_1;
		}
		sha2_step(a, b, c, d, e, f, g, h, in[ 0], Kshared[16 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[ 1], Kshared[17 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[ 2], Kshared[18 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[ 3], Kshared[19 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[ 4], Kshared[20 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[ 5], Kshared[21 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[ 6], Kshared[22 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[ 7], Kshared[23 + 16 * i]);
		sha2_step(a, b, c, d, e, f, g, h, in[ 8], Kshared[24 + 16 * i]);
		sha2_step(h, a, b, c, d, e, f, g, in[ 9], Kshared[25 + 16 * i]);
		sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * i]);
		sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * i]);
		sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * i]);
		sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * i]);
		sha2_step(c, d, e, f, g, h, a, b, in[14], Kshared[30 + 16 * i]);
		sha2_step(b, c, d, e, f, g, h, a, in[15], Kshared[31 + 16 * i]);
	}
	#pragma unroll 16
	for (uint32_t j = 0; j < 16; j++) {
		const uint32_t x2_0 = ssg2_0(in[(j + 1) & 15]);
		const uint32_t x2_1 = ssg2_1(in[(j + 14) & 15]) + x2_0;
		in[j] = in[j] + in[(j + 9) & 15] + x2_1;
	}
	sha2_step(a, b, c, d, e, f, g, h, in[ 0], Kshared[16 + 16 * 2]);
	sha2_step(h, a, b, c, d, e, f, g, in[ 1], Kshared[17 + 16 * 2]);
	sha2_step(g, h, a, b, c, d, e, f, in[ 2], Kshared[18 + 16 * 2]);
	sha2_step(f, g, h, a, b, c, d, e, in[ 3], Kshared[19 + 16 * 2]);
	sha2_step(e, f, g, h, a, b, c, d, in[ 4], Kshared[20 + 16 * 2]);
	sha2_step(d, e, f, g, h, a, b, c, in[ 5], Kshared[21 + 16 * 2]);
	sha2_step(c, d, e, f, g, h, a, b, in[ 6], Kshared[22 + 16 * 2]);
	sha2_step(b, c, d, e, f, g, h, a, in[ 7], Kshared[23 + 16 * 2]);
	sha2_step(a, b, c, d, e, f, g, h, in[ 8], Kshared[24 + 16 * 2]);
	sha2_step(h, a, b, c, d, e, f, g, in[ 9], Kshared[25 + 16 * 2]);
	sha2_step(g, h, a, b, c, d, e, f, in[10], Kshared[26 + 16 * 2]);
	sha2_step(f, g, h, a, b, c, d, e, in[11], Kshared[27 + 16 * 2]);
	sha2_step(e, f, g, h, a, b, c, d, in[12], Kshared[28 + 16 * 2]);
	sha2_step(d, e, f, g, h, a, b, c, in[13], Kshared[29 + 16 * 2]);

	state[6] += g;
	state[7] += h;
}

//END OF SHA256 MACROS --------------------------------------------------------------------

//SHA512 MACROS ---------------------------------------------------------------------------
static __constant__ _ALIGN(8) uint64_t K_512[80] = {
	0x428A2F98D728AE22, 0x7137449123EF65CD, 0xB5C0FBCFEC4D3B2F, 0xE9B5DBA58189DBBC,
	0x3956C25BF348B538, 0x59F111F1B605D019, 0x923F82A4AF194F9B, 0xAB1C5ED5DA6D8118,
	0xD807AA98A3030242, 0x12835B0145706FBE, 0x243185BE4EE4B28C, 0x550C7DC3D5FFB4E2,
	0x72BE5D74F27B896F, 0x80DEB1FE3B1696B1, 0x9BDC06A725C71235, 0xC19BF174CF692694,
	0xE49B69C19EF14AD2, 0xEFBE4786384F25E3, 0x0FC19DC68B8CD5B5, 0x240CA1CC77AC9C65,
	0x2DE92C6F592B0275, 0x4A7484AA6EA6E483, 0x5CB0A9DCBD41FBD4, 0x76F988DA831153B5,
	0x983E5152EE66DFAB, 0xA831C66D2DB43210, 0xB00327C898FB213F, 0xBF597FC7BEEF0EE4,
	0xC6E00BF33DA88FC2, 0xD5A79147930AA725, 0x06CA6351E003826F, 0x142929670A0E6E70,
	0x27B70A8546D22FFC, 0x2E1B21385C26C926, 0x4D2C6DFC5AC42AED, 0x53380D139D95B3DF,
	0x650A73548BAF63DE, 0x766A0ABB3C77B2A8, 0x81C2C92E47EDAEE6, 0x92722C851482353B,
	0xA2BFE8A14CF10364, 0xA81A664BBC423001, 0xC24B8B70D0F89791, 0xC76C51A30654BE30,
	0xD192E819D6EF5218, 0xD69906245565A910, 0xF40E35855771202A, 0x106AA07032BBD1B8,
	0x19A4C116B8D2D0C8, 0x1E376C085141AB53, 0x2748774CDF8EEB99, 0x34B0BCB5E19B48A8,
	0x391C0CB3C5C95A63, 0x4ED8AA4AE3418ACB, 0x5B9CCA4F7763E373, 0x682E6FF3D6B2B8A3,
	0x748F82EE5DEFB2FC, 0x78A5636F43172F60, 0x84C87814A1F0AB72, 0x8CC702081A6439EC,
	0x90BEFFFA23631E28, 0xA4506CEBDE82BDE9, 0xBEF9A3F7B2C67915, 0xC67178F2E372532B,
	0xCA273ECEEA26619C, 0xD186B8C721C0C207, 0xEADA7DD6CDE0EB1E, 0xF57D4F7FEE6ED178,
	0x06F067AA72176FBA, 0x0A637DC5A2C898A6, 0x113F9804BEF90DAE, 0x1B710B35131C471B,
	0x28DB77F523047D84, 0x32CAAB7B40C72493, 0x3C9EBE0A15C9BEBC, 0x431D67C49C100D4C,
	0x4CC5D4BECB3E42B6, 0x597F299CFC657E2A, 0x5FCB6FAB3AD6FAEC, 0x6C44198C4A475817
};

#undef xor3
#define xor3(a,b,c) (a^b^c)

#define bsg5_0(x) xor3(ROTR64(x,28),ROTR64(x,34),ROTR64(x,39))
#define bsg5_1(x) xor3(ROTR64(x,14),ROTR64(x,18),ROTR64(x,41))
#define ssg5_0(x) xor3(ROTR64(x, 1),ROTR64(x, 8),x>>7)
#define ssg5_1(x) xor3(ROTR64(x,19),ROTR64(x,61),x>>6)

#define andor64(a,b,c) ((a & (b | c)) | (b & c))
#define xandx64(e,f,g) (g ^ (e & (g ^ f)))

__device__ __forceinline__
uint64_t cuda_swab64ll(const uint32_t x, const uint32_t y)
{
	uint64_t r;
	asm("prmt.b32 %1, %1, 0, 0x0123; // swab64ll\n\t"
	    "prmt.b32 %2, %2, 0, 0x0123;\n\t"
	    "mov.b64 %0, {%1,%2};\n\t"
	  : "=l"(r): "r"(x), "r"(y) );
	return r;
}

// RIPEMD MACROS-----------------------------------------------------------------------------
static __constant__ const uint32_t c_IV[5] = { 0x67452301u, 0xEFCDAB89u, 0x98BADCFEu, 0x10325476u, 0xC3D2E1F0u };
static __constant__ const uint32_t c_K1[5] = { 0, 0x5A827999, 0x6ED9EBA1, 0x8F1BBCDC, 0xA953FD4E };
static __constant__ const uint32_t c_K2[5] = { 0x50A28BE6, 0x5C4DD124, 0x6D703EF3, 0x7A6D76E9, 0 };

__device__ __forceinline__
static uint32_t ROTATE(const uint32_t x,const uint32_t r) {
	if(r==8)
		return __byte_perm(x, 0, 0x2103);
	else
		return ROTL32(x,r);
}

/*
 * Round functions for RIPEMD-160.
 */
//#define F1(x, y, z)   xor3x(x, y, z)
__device__ __forceinline__
uint32_t F1(const uint32_t a, const uint32_t b, const uint32_t c) {
	uint32_t result;
	#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
		asm volatile ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result) : "r"(a), "r"(b),"r"(c));
	#else
		result = a^b^c;
	#endif
	return result;
}
//#define F2(x, y, z)   ((x & (y ^ z)) ^ z)
__device__ __forceinline__
uint32_t F2(const uint32_t a, const uint32_t b, const uint32_t c) {
	uint32_t result;
	#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
		asm volatile ("lop3.b32 %0, %1, %2, %3, 0xCA;" : "=r"(result) : "r"(a), "r"(b),"r"(c)); //0xCA=((F0∧(CC⊻AA))⊻AA)
	#else
		result = ((a & (b ^ c)) ^ c);
	#endif
	return result;
}
//#define F3(x, y, z)   ((x | ~y) ^ z)
__device__ __forceinline__
uint32_t F3(const uint32_t x, const uint32_t y, const uint32_t z) {
	uint32_t result;
	#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
		asm volatile ("lop3.b32 %0, %1, %2, %3, 0x59;" : "=r"(result) : "r"(x), "r"(y),"r"(z)); //0x59=((F0∨(¬CC))⊻AA)
	#else
		result = ((x | ~y) ^ z);
	#endif
	return result;
}
//#define F4(x, y, z)   (y ^ ((x ^ y) & z))
__device__ __forceinline__
uint32_t F4(const uint32_t x, const uint32_t y, const uint32_t z) {
	uint32_t result;
	#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
		asm volatile ("lop3.b32 %0, %1, %2, %3, 0xE4;" : "=r"(result) : "r"(x), "r"(y),"r"(z)); //0xE4=(CC⊻((F0⊻CC)∧AA))
	#else
		result = (y ^ ((x ^ y) & z));
	#endif
	return result;
}
//#define F5(x, y, z)   (x ^ (y | ~z))
__device__ __forceinline__
uint32_t F5(const uint32_t x, const uint32_t y, const uint32_t z) {
	uint32_t result;
	#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
		asm volatile ("lop3.b32 %0, %1, %2, %3, 0x2D;" : "=r"(result) : "r"(x), "r"(y),"r"(z)); //0x2D=(F0⊻(CC∨(¬AA)))
	#else
		result = (x ^ (y | ~z));
	#endif
	return result;
}

/*
 * Round constants for RIPEMD-160.
 */
#define RR(a, b, c, d, e, f, s, r, k) { \
	a = e + ROTATE((a + k + r + f(b, c, d)), s); \
	c = ROTL32(c, 10); \
}

#define ROUND1(a, b, c, d, e, f, s, r, k) \
	RR(a[0], b[0], c[0], d[0], e[0], f, s, r, c_K1[k])

#define ROUND2(a, b, c, d, e, f, s, r, k) \
	RR(a[1], b[1], c[1], d[1], e[1], f, s, r, c_K2[k])

#define RIPEMD160_ROUND_BODY(in, h) { \
	uint32_t A[2], B[2], C[2], D[2], E[2]; \
	uint32_t tmp; \
\
	A[0] = A[1] = h[0]; \
	B[0] = B[1] = h[1]; \
	C[0] = C[1] = h[2]; \
	D[0] = D[1] = h[3]; \
	E[0] = E[1] = h[4]; \
\
	ROUND1(A, B, C, D, E, F1, 11, in[ 0],  0); \
	ROUND1(E, A, B, C, D, F1, 14, in[ 1],  0); \
	ROUND1(D, E, A, B, C, F1, 15, in[ 2],  0); \
	ROUND1(C, D, E, A, B, F1, 12, in[ 3],  0); \
	ROUND1(B, C, D, E, A, F1,  5, in[ 4],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[ 5],  0); \
	ROUND1(E, A, B, C, D, F1,  7, in[ 6],  0); \
	ROUND1(D, E, A, B, C, F1,  9, in[ 7],  0); \
	ROUND1(C, D, E, A, B, F1, 11, in[ 8],  0); \
	ROUND1(B, C, D, E, A, F1, 13, in[ 9],  0); \
	ROUND1(A, B, C, D, E, F1, 14, in[10],  0); \
	ROUND1(E, A, B, C, D, F1, 15, in[11],  0); \
	ROUND1(D, E, A, B, C, F1,  6, in[12],  0); \
	ROUND1(C, D, E, A, B, F1,  7, in[13],  0); \
	ROUND1(B, C, D, E, A, F1,  9, in[14],  0); \
	ROUND1(A, B, C, D, E, F1,  8, in[15],  0); \
\
	ROUND1(E, A, B, C, D, F2,  7, in[ 7],  1); \
	ROUND1(D, E, A, B, C, F2,  6, in[ 4],  1); \
	ROUND1(C, D, E, A, B, F2,  8, in[13],  1); \
	ROUND1(B, C, D, E, A, F2, 13, in[ 1],  1); \
	ROUND1(A, B, C, D, E, F2, 11, in[10],  1); \
	ROUND1(E, A, B, C, D, F2,  9, in[ 6],  1); \
	ROUND1(D, E, A, B, C, F2,  7, in[15],  1); \
	ROUND1(C, D, E, A, B, F2, 15, in[ 3],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[12],  1); \
	ROUND1(A, B, C, D, E, F2, 12, in[ 0],  1); \
	ROUND1(E, A, B, C, D, F2, 15, in[ 9],  1); \
	ROUND1(D, E, A, B, C, F2,  9, in[ 5],  1); \
	ROUND1(C, D, E, A, B, F2, 11, in[ 2],  1); \
	ROUND1(B, C, D, E, A, F2,  7, in[14],  1); \
	ROUND1(A, B, C, D, E, F2, 13, in[11],  1); \
	ROUND1(E, A, B, C, D, F2, 12, in[ 8],  1); \
\
	ROUND1(D, E, A, B, C, F3, 11, in[ 3],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[10],  2); \
	ROUND1(B, C, D, E, A, F3,  6, in[14],  2); \
	ROUND1(A, B, C, D, E, F3,  7, in[ 4],  2); \
	ROUND1(E, A, B, C, D, F3, 14, in[ 9],  2); \
	ROUND1(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND1(C, D, E, A, B, F3, 13, in[ 8],  2); \
	ROUND1(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND1(A, B, C, D, E, F3, 14, in[ 2],  2); \
	ROUND1(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND1(D, E, A, B, C, F3, 13, in[ 0],  2); \
	ROUND1(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND1(B, C, D, E, A, F3,  5, in[13],  2); \
	ROUND1(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND1(E, A, B, C, D, F3,  7, in[ 5],  2); \
	ROUND1(D, E, A, B, C, F3,  5, in[12],  2); \
\
	ROUND1(C, D, E, A, B, F4, 11, in[ 1],  3); \
	ROUND1(B, C, D, E, A, F4, 12, in[ 9],  3); \
	ROUND1(A, B, C, D, E, F4, 14, in[11],  3); \
	ROUND1(E, A, B, C, D, F4, 15, in[10],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 0],  3); \
	ROUND1(C, D, E, A, B, F4, 15, in[ 8],  3); \
	ROUND1(B, C, D, E, A, F4,  9, in[12],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[ 4],  3); \
	ROUND1(E, A, B, C, D, F4,  9, in[13],  3); \
	ROUND1(D, E, A, B, C, F4, 14, in[ 3],  3); \
	ROUND1(C, D, E, A, B, F4,  5, in[ 7],  3); \
	ROUND1(B, C, D, E, A, F4,  6, in[15],  3); \
	ROUND1(A, B, C, D, E, F4,  8, in[14],  3); \
	ROUND1(E, A, B, C, D, F4,  6, in[ 5],  3); \
	ROUND1(D, E, A, B, C, F4,  5, in[ 6],  3); \
	ROUND1(C, D, E, A, B, F4, 12, in[ 2],  3); \
\
	ROUND1(B, C, D, E, A, F5,  9, in[ 4],  4); \
	ROUND1(A, B, C, D, E, F5, 15, in[ 0],  4); \
	ROUND1(E, A, B, C, D, F5,  5, in[ 5],  4); \
	ROUND1(D, E, A, B, C, F5, 11, in[ 9],  4); \
	ROUND1(C, D, E, A, B, F5,  6, in[ 7],  4); \
	ROUND1(B, C, D, E, A, F5,  8, in[12],  4); \
	ROUND1(A, B, C, D, E, F5, 13, in[ 2],  4); \
	ROUND1(E, A, B, C, D, F5, 12, in[10],  4); \
	ROUND1(D, E, A, B, C, F5,  5, in[14],  4); \
	ROUND1(C, D, E, A, B, F5, 12, in[ 1],  4); \
	ROUND1(B, C, D, E, A, F5, 13, in[ 3],  4); \
	ROUND1(A, B, C, D, E, F5, 14, in[ 8],  4); \
	ROUND1(E, A, B, C, D, F5, 11, in[11],  4); \
	ROUND1(D, E, A, B, C, F5,  8, in[ 6],  4); \
	ROUND1(C, D, E, A, B, F5,  5, in[15],  4); \
	ROUND1(B, C, D, E, A, F5,  6, in[13],  4); \
\
	ROUND2(A, B, C, D, E, F5,  8, in[ 5],  0); \
	ROUND2(E, A, B, C, D, F5,  9, in[14],  0); \
	ROUND2(D, E, A, B, C, F5,  9, in[ 7],  0); \
	ROUND2(C, D, E, A, B, F5, 11, in[ 0],  0); \
	ROUND2(B, C, D, E, A, F5, 13, in[ 9],  0); \
	ROUND2(A, B, C, D, E, F5, 15, in[ 2],  0); \
	ROUND2(E, A, B, C, D, F5, 15, in[11],  0); \
	ROUND2(D, E, A, B, C, F5,  5, in[ 4],  0); \
	ROUND2(C, D, E, A, B, F5,  7, in[13],  0); \
	ROUND2(B, C, D, E, A, F5,  7, in[ 6],  0); \
	ROUND2(A, B, C, D, E, F5,  8, in[15],  0); \
	ROUND2(E, A, B, C, D, F5, 11, in[ 8],  0); \
	ROUND2(D, E, A, B, C, F5, 14, in[ 1],  0); \
	ROUND2(C, D, E, A, B, F5, 14, in[10],  0); \
	ROUND2(B, C, D, E, A, F5, 12, in[ 3],  0); \
	ROUND2(A, B, C, D, E, F5,  6, in[12],  0); \
\
	ROUND2(E, A, B, C, D, F4,  9, in[ 6],  1); \
	ROUND2(D, E, A, B, C, F4, 13, in[11],  1); \
	ROUND2(C, D, E, A, B, F4, 15, in[ 3],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[ 7],  1); \
	ROUND2(A, B, C, D, E, F4, 12, in[ 0],  1); \
	ROUND2(E, A, B, C, D, F4,  8, in[13],  1); \
	ROUND2(D, E, A, B, C, F4,  9, in[ 5],  1); \
	ROUND2(C, D, E, A, B, F4, 11, in[10],  1); \
	ROUND2(B, C, D, E, A, F4,  7, in[14],  1); \
	ROUND2(A, B, C, D, E, F4,  7, in[15],  1); \
	ROUND2(E, A, B, C, D, F4, 12, in[ 8],  1); \
	ROUND2(D, E, A, B, C, F4,  7, in[12],  1); \
	ROUND2(C, D, E, A, B, F4,  6, in[ 4],  1); \
	ROUND2(B, C, D, E, A, F4, 15, in[ 9],  1); \
	ROUND2(A, B, C, D, E, F4, 13, in[ 1],  1); \
	ROUND2(E, A, B, C, D, F4, 11, in[ 2],  1); \
\
	ROUND2(D, E, A, B, C, F3,  9, in[15],  2); \
	ROUND2(C, D, E, A, B, F3,  7, in[ 5],  2); \
	ROUND2(B, C, D, E, A, F3, 15, in[ 1],  2); \
	ROUND2(A, B, C, D, E, F3, 11, in[ 3],  2); \
	ROUND2(E, A, B, C, D, F3,  8, in[ 7],  2); \
	ROUND2(D, E, A, B, C, F3,  6, in[14],  2); \
	ROUND2(C, D, E, A, B, F3,  6, in[ 6],  2); \
	ROUND2(B, C, D, E, A, F3, 14, in[ 9],  2); \
	ROUND2(A, B, C, D, E, F3, 12, in[11],  2); \
	ROUND2(E, A, B, C, D, F3, 13, in[ 8],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[12],  2); \
	ROUND2(C, D, E, A, B, F3, 14, in[ 2],  2); \
	ROUND2(B, C, D, E, A, F3, 13, in[10],  2); \
	ROUND2(A, B, C, D, E, F3, 13, in[ 0],  2); \
	ROUND2(E, A, B, C, D, F3,  7, in[ 4],  2); \
	ROUND2(D, E, A, B, C, F3,  5, in[13],  2); \
\
	ROUND2(C, D, E, A, B, F2, 15, in[ 8],  3); \
	ROUND2(B, C, D, E, A, F2,  5, in[ 6],  3); \
	ROUND2(A, B, C, D, E, F2,  8, in[ 4],  3); \
	ROUND2(E, A, B, C, D, F2, 11, in[ 1],  3); \
	ROUND2(D, E, A, B, C, F2, 14, in[ 3],  3); \
	ROUND2(C, D, E, A, B, F2, 14, in[11],  3); \
	ROUND2(B, C, D, E, A, F2,  6, in[15],  3); \
	ROUND2(A, B, C, D, E, F2, 14, in[ 0],  3); \
	ROUND2(E, A, B, C, D, F2,  6, in[ 5],  3); \
	ROUND2(D, E, A, B, C, F2,  9, in[12],  3); \
	ROUND2(C, D, E, A, B, F2, 12, in[ 2],  3); \
	ROUND2(B, C, D, E, A, F2,  9, in[13],  3); \
	ROUND2(A, B, C, D, E, F2, 12, in[ 9],  3); \
	ROUND2(E, A, B, C, D, F2,  5, in[ 7],  3); \
	ROUND2(D, E, A, B, C, F2, 15, in[10],  3); \
	ROUND2(C, D, E, A, B, F2,  8, in[14],  3); \
\
	ROUND2(B, C, D, E, A, F1,  8, in[12],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[15],  4); \
	ROUND2(E, A, B, C, D, F1, 12, in[10],  4); \
	ROUND2(D, E, A, B, C, F1,  9, in[ 4],  4); \
	ROUND2(C, D, E, A, B, F1, 12, in[ 1],  4); \
	ROUND2(B, C, D, E, A, F1,  5, in[ 5],  4); \
	ROUND2(A, B, C, D, E, F1, 14, in[ 8],  4); \
	ROUND2(E, A, B, C, D, F1,  6, in[ 7],  4); \
	ROUND2(D, E, A, B, C, F1,  8, in[ 6],  4); \
	ROUND2(C, D, E, A, B, F1, 13, in[ 2],  4); \
	ROUND2(B, C, D, E, A, F1,  6, in[13],  4); \
	ROUND2(A, B, C, D, E, F1,  5, in[14],  4); \
	ROUND2(E, A, B, C, D, F1, 15, in[ 0],  4); \
	ROUND2(D, E, A, B, C, F1, 13, in[ 3],  4); \
	ROUND2(C, D, E, A, B, F1, 11, in[ 9],  4); \
	ROUND2(B, C, D, E, A, F1, 11, in[11],  4); \
\
	tmp  = h[1] + C[0] + D[1]; \
	h[1] = h[2] + D[0] + E[1]; \
	h[2] = h[3] + E[0] + A[1]; \
	h[3] = h[4] + A[0] + B[1]; \
	h[4] = h[0] + B[0] + C[1]; \
	h[0] = tmp; \
}
// END OF RIPEMD MACROS----------------------------------------------------------------------

__global__
__launch_bounds__(768,1) /* will force 64 regs max on SM 3+ */
void gpu_lbry_merged(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonces, const uint64_t target64)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	const uint64_t IV512[8] = {
		0x6A09E667F3BCC908, 0xBB67AE8584CAA73B, 0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
		0x510E527FADE682D1, 0x9B05688C2B3E6C1F, 0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
	};

	if (thread < threads)
	{
		uint64_t r[8];
		uint64_t W[16];
		uint32_t dat[16];
		uint32_t buf[8], state[8];
		uint32_t h[5];

		*(uint2x4*)&dat[0] = *(uint2x4*)&c_dataEnd112[0];
		dat[ 8] = c_dataEnd112[ 8];
		dat[ 9] = c_dataEnd112[ 9];
		dat[10] = c_dataEnd112[10];
		dat[11] = startNonce + thread;
		dat[12] = 0x80000000;
		dat[13] = 0;
		dat[14] = 0;
		dat[15] = 0x380;

		*(uint2x4*)&state[0] = *(uint2x4*)&c_midstate112[0];
		*(uint2x4*)&buf[0]   = *(uint2x4*)&c_midbuffer112[0];

		sha256_round_first(dat, buf, state, c_K);

		// second sha256

		#pragma unroll 8
		for(int i=0; i<8; i++){
			dat[i] = buf[i];
		}
		dat[8] = 0x80000000;

		#pragma unroll 6
		for(int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		#pragma unroll 8
		for(int i=0; i<8; i++)
			buf[i] = c_H256[i];

		sha256_round_body(dat, buf, c_K);

// SHA512-------------------------------------------------------------------------------------

		#pragma unroll 8
		for(int i=0; i<8; i++)
			r[i] = IV512[i];

		W[0] = vectorizeswap(((uint64_t*)buf)[0]);
		W[1] = vectorizeswap(((uint64_t*)buf)[1]);
		W[2] = vectorizeswap(((uint64_t*)buf)[2]);
		W[3] = vectorizeswap(((uint64_t*)buf)[3]);
		W[4] = 0x8000000000000000; // end tag

		#pragma unroll 10
		for (int i = 5; i < 15; i++)
			W[i] = 0;

		W[15] = 0x100; // 256 bits

		#pragma unroll 16
		for (uint32_t i = 0; i < 16; i++)
		{
//			sha512_step2(r, W[ i], K_512[ i], i&7);
			const uint32_t ord = i&7;
			const uint64_t T1 = r[(15-ord) & 7] + K_512[ i] + W[ i] + bsg5_1(r[(12-ord) & 7]) +
				xandx64(r[(12-ord) & 7], r[(13-ord) & 7], r[(14-ord) & 7]);

			r[(15-ord)& 7] = andor64(r[( 8-ord) & 7], r[( 9-ord) & 7], r[(10-ord) & 7]) + bsg5_0(r[( 8-ord) & 7]) + T1;
			r[(11-ord)& 7] = r[(11-ord)& 7] + T1;
		}

		#pragma unroll 5
		for (uint32_t i = 16; i < 80; i+=16)
		{
			#pragma unroll 16
			for (uint32_t j = 0; j<16; j++)
				W[(i + j) & 15] = W[((i + j) - 7) & 15] + W[(i + j) & 15] + ssg5_0(W[((i + j) - 15) & 15]) + ssg5_1(W[((i + j) - 2) & 15]);

			#pragma unroll 16
			for (uint32_t j = 0; j<16; j++) {

				const uint32_t ord = (i+j)&7;
				const uint64_t T1 = K_512[i+j] + W[ j] + r[(15-ord) & 7] + bsg5_1(r[(12-ord) & 7]) +
					xandx64(r[(12-ord) & 7], r[(13-ord) & 7], r[(14-ord) & 7]);

				r[(15-ord)& 7] = andor64(r[( 8-ord) & 7], r[( 9-ord) & 7], r[(10-ord) & 7]) + bsg5_0(r[( 8-ord) & 7]) + T1;
				r[(11-ord)& 7] = r[(11-ord)& 7] + T1;
			}
		}

//END OF SHA512------------------------------------------------------------------------------
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
			*(uint64_t*)&dat[i*2] = cuda_swab64(r[i] + IV512[i]);
		dat[8] = 0x80;

		#pragma unroll 7
		for (int i=9; i<16; i++) dat[i] = 0;

		dat[14] = 0x100; // size in bits

		#pragma unroll 5
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(dat, h);

		#pragma unroll 5
		for (int i=0; i<5; i++)
			buf[i] = h[i];

		// second 32 bytes block hash
		#pragma unroll 4
		for (int i=0; i < 4; i++)
			*(uint64_t*)&dat[i*2] = cuda_swab64(r[i+4] + IV512[i+4]);

		dat[8] = 0x80;

		#pragma unroll 7
		for (int i=9; i<16; i++) dat[i] = 0;

		dat[14] = 0x100; // size in bits

		#pragma unroll 5
		for (int i=0; i<5; i++)
			h[i] = c_IV[i];

		RIPEMD160_ROUND_BODY(dat, h);

		// first final sha256

		#pragma unroll 5
		for (int i=0; i<5; i++) dat[i] = cuda_swab32(buf[i]);
		#pragma unroll 5
		for (int i=0; i<5; i++) dat[i+5] = cuda_swab32(h[i]);
		dat[10] = 0x80000000;
		#pragma unroll 4
		for (int i=11; i<15; i++) dat[i] = 0;

		dat[15] = 0x140;

		#pragma unroll 8
		for(int i=0; i<8; i++)
			buf[i] = c_H256[i];

		sha256_round_body(dat, buf, c_K);

		// second sha256

		#pragma unroll 8
		for(int i=0; i<8; i++) {
			dat[i] = buf[i];
		}
		dat[8] = 0x80000000;

		#pragma unroll 8
		for(int i=0; i<8; i++)
			buf[i] = c_H256[i];

		#pragma unroll 6
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		sha256_round_body_final(dat, buf, c_K);

		// valid nonces
		if (cuda_swab64ll(buf[ 6],buf[ 7]) <= target64) {
			uint32_t tmp = atomicExch(&resNonces[0], thread);
			if (tmp != UINT32_MAX)
				resNonces[1] = tmp;
		}
	}
}

__host__
void lbry_merged(int thr_id, uint32_t startNonce, uint32_t threads, uint32_t *d_resNonce, const uint64_t target64)
{
	uint32_t threadsperblock = 768;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	gpu_lbry_merged <<<grid, block>>> (threads,startNonce, d_resNonce, target64);
}
