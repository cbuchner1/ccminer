#include "hip/hip_runtime.h"
// Auf Myriadcoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#define __funnelshift_r(x,y,n) (x >> n)
#define atomicExch(p,x) x
#endif

#if __CUDA_ARCH__ >= 300
// 64 Registers Variant for Compute 3.0
#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"
#endif

// globaler Speicher für alle HeftyHashes aller Threads
static uint32_t *d_outputHashes[MAX_GPUS];
static uint32_t *d_resultNonces[MAX_GPUS];

__constant__ uint32_t pTarget[2]; // Same for all GPU
__constant__ uint32_t myriadgroestl_gpu_msg[32];

// muss expandiert werden
__constant__ uint32_t myr_sha256_gpu_constantTable[64];
__constant__ uint32_t myr_sha256_gpu_constantTable2[64];

const uint32_t myr_sha256_cpu_constantTable[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
};

const uint32_t myr_sha256_cpu_w2Table[] = {
	0x80000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000,
	0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000200,
	0x80000000, 0x01400000, 0x00205000, 0x00005088, 0x22000800, 0x22550014, 0x05089742, 0xa0000020,
	0x5a880000, 0x005c9400, 0x0016d49d, 0xfa801f00, 0xd33225d0, 0x11675959, 0xf6e6bfda, 0xb30c1549,
	0x08b2b050, 0x9d7c4c27, 0x0ce2a393, 0x88e6e1ea, 0xa52b4335, 0x67a16f49, 0xd732016f, 0x4eeb2e91,
	0x5dbf55e5, 0x8eee2335, 0xe2bc5ec2, 0xa83f4394, 0x45ad78f7, 0x36f3d0cd, 0xd99c05e8, 0xb0511dc7,
	0x69bc7ac4, 0xbd11375b, 0xe3ba71e5, 0x3b209ff2, 0x18feee17, 0xe25ad9e7, 0x13375046, 0x0515089d,
	0x4f0d0f04, 0x2627484e, 0x310128d2, 0xc668b434, 0x420841cc, 0x62d311b8, 0xe59ba771, 0x85a7a484
};

#define SWAB32(x) cuda_swab32(x)

#if __CUDA_ARCH__ < 320
	// Kepler (Compute 3.0)
	#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#else
	// Kepler (Compute 3.5)
	#define ROTR32(x, n) __funnelshift_r( (x), (x), (n) )
#endif

#define R(x, n)         ((x) >> (n))
#define Ch(x, y, z)     ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define S0(x)           (ROTR32(x, 2) ^ ROTR32(x, 13) ^ ROTR32(x, 22))
#define S1(x)           (ROTR32(x, 6) ^ ROTR32(x, 11) ^ ROTR32(x, 25))
#define s0(x)           (ROTR32(x, 7) ^ ROTR32(x, 18) ^ R(x, 3))
#define s1(x)           (ROTR32(x, 17) ^ ROTR32(x, 19) ^ R(x, 10))

__device__ __forceinline__
void myriadgroestl_gpu_sha256(uint32_t *message)
{
	uint32_t W1[16];
	#pragma unroll
	for(int k=0; k<16; k++)
		W1[k] = SWAB32(message[k]);

	uint32_t regs[8] = {
		0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
		0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
	};

	// Progress W1
	#pragma unroll
	for(int j=0; j<16; j++)
	{
		uint32_t T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j] + W1[j];
		uint32_t T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	// Progress W2...W3
	uint32_t W2[16];

	////// PART 1
	#pragma unroll
	for(int j=0; j<2; j++)
		W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll 5
	for(int j=2; j<7;j++)
		W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll
	for(int j=7; j<15; j++)
		W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

	W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

	// Round function
	#pragma unroll
	for(int j=0; j<16; j++)
	{
		uint32_t T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j + 16] + W2[j];
		uint32_t T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	////// PART 2
	#pragma unroll
	for(int j=0; j<2; j++)
		W1[j] = s1(W2[14+j]) + W2[9+j] + s0(W2[1+j]) + W2[j];
	#pragma unroll 5
	for(int j=2; j<7; j++)
		W1[j] = s1(W1[j-2]) + W2[9+j] + s0(W2[1+j]) + W2[j];

	#pragma unroll
	for(int j=7; j<15; j++)
		W1[j] = s1(W1[j-2]) + W1[j-7] + s0(W2[1+j]) + W2[j];

	W1[15] = s1(W1[13]) + W1[8] + s0(W1[0]) + W2[15];

	// Round function
	#pragma unroll
	for(int j=0; j<16; j++)
	{
		uint32_t T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j + 32] + W1[j];
		uint32_t T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	////// PART 3
	#pragma unroll
	for(int j=0; j<2; j++)
		W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll 5
	for(int j=2; j<7; j++)
		W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll
	for(int j=7; j<15; j++)
		W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

	W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

	// Round function
	#pragma unroll
	for(int j=0; j<16; j++)
	{
		uint32_t T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j + 48] + W2[j];
		uint32_t T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	uint32_t hash[8] = {
		0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
		0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
	};

	#pragma unroll 8
	for(int k=0; k<8; k++)
		hash[k] += regs[k];

	/////
	///// 2nd Round (wegen Msg-Padding)
	/////
	#pragma unroll
	for(int k=0; k<8; k++)
		regs[k] = hash[k];

	// Progress W1
	#pragma unroll
	for(int j=0; j<64; j++)
	{
		uint32_t T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable2[j];
		uint32_t T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

#if 0
	// Full sha hash
	#pragma unroll
	for(int k=0; k<8; k++)
		hash[k] += regs[k];

	#pragma unroll
	for(int k=0; k<8; k++)
		message[k] = SWAB32(hash[k]);
#else
	message[6] = SWAB32(hash[6] + regs[6]);
	message[7] = SWAB32(hash[7] + regs[7]);
#endif
}

__global__
//__launch_bounds__(256, 6) // we want <= 40 regs
void myriadgroestl_gpu_hash_sha(uint32_t threads, uint32_t startNounce, uint32_t *hashBuffer, uint32_t *resNonces)
{
#if __CUDA_ARCH__ >= 300
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNounce + thread;

		uint32_t out_state[16];
		uint32_t *inpHash = &hashBuffer[16 * thread];

		#pragma unroll 16
		for (int i=0; i < 16; i++)
			out_state[i] = inpHash[i];

		myriadgroestl_gpu_sha256(out_state);

		if (out_state[7] <= pTarget[1] && out_state[6] <= pTarget[0])
		{
			uint32_t tmp = atomicExch(&resNonces[0], nonce);
			if (tmp != UINT32_MAX)
				resNonces[1] = tmp;
		}
	}
#endif
}

__global__
__launch_bounds__(256, 4)
void myriadgroestl_gpu_hash_quad(uint32_t threads, uint32_t startNounce, uint32_t *hashBuffer)
{
#if __CUDA_ARCH__ >= 300
	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) / 4;
	if (thread < threads)
	{
		// GROESTL
		uint32_t paddedInput[8];
		#pragma unroll 8
		for(int k=0; k<8; k++)
			paddedInput[k] = myriadgroestl_gpu_msg[4*k+threadIdx.x%4];

		uint32_t nounce = startNounce + thread;
		if ((threadIdx.x % 4) == 3)
			paddedInput[4] = SWAB32(nounce);  // 4*4+3 = 19

		uint32_t msgBitsliced[8];
		to_bitslice_quad(paddedInput, msgBitsliced);

		uint32_t state[8];
		groestl512_progressMessage_quad(state, msgBitsliced);

		uint32_t out_state[16];
		from_bitslice_quad(state, out_state);

		if ((threadIdx.x & 0x03) == 0)
		{
			uint32_t *outpHash = &hashBuffer[16 * thread];
			#pragma unroll 16
			for(int k=0; k<16; k++) outpHash[k] = out_state[k];
		}
	}
#endif
}

// Setup Function
__host__
void myriadgroestl_cpu_init(int thr_id, uint32_t threads)
{
	uint32_t temp[64];
	for(int i=0; i<64; i++)
		temp[i] = myr_sha256_cpu_w2Table[i] + myr_sha256_cpu_constantTable[i];

	hipMemcpyToSymbol(HIP_SYMBOL( myr_sha256_gpu_constantTable2), temp, sizeof(uint32_t) * 64 );

	hipMemcpyToSymbol(HIP_SYMBOL( myr_sha256_gpu_constantTable),
						myr_sha256_cpu_constantTable,
						sizeof(uint32_t) * 64 );

	// to check if the binary supports SM3+
	cuda_get_arch(thr_id);

	hipMalloc(&d_outputHashes[thr_id], (size_t) 64 * threads);
	hipMalloc(&d_resultNonces[thr_id], 2 * sizeof(uint32_t));
}

__host__
void myriadgroestl_cpu_free(int thr_id)
{
	hipFree(d_outputHashes[thr_id]);
	hipFree(d_resultNonces[thr_id]);
}

__host__
void myriadgroestl_cpu_setBlock(int thr_id, void *data, uint32_t *pTargetIn)
{
	uint32_t msgBlock[32] = { 0 };
	memcpy(&msgBlock[0], data, 80);
	msgBlock[20] = 0x80;
	msgBlock[31] = 0x01000000;

	hipMemcpyToSymbol(HIP_SYMBOL(myriadgroestl_gpu_msg), msgBlock, 128);
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), &pTargetIn[6], 2 * sizeof(uint32_t));
}

__host__
void myriadgroestl_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNounce)
{
	uint32_t threadsperblock = 256;

	hipMemset(d_resultNonces[thr_id], 0xFF, 2 * sizeof(uint32_t));

	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	const int factor = 4;

	dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
		printf("Sorry, This algo is not supported by this GPU arch (SM 3.0 required)");
		return;
	}

	myriadgroestl_gpu_hash_quad <<< grid, block >>> (threads, startNounce, d_outputHashes[thr_id]);

	dim3 grid2((threads + threadsperblock-1)/threadsperblock);
	myriadgroestl_gpu_hash_sha <<< grid2, block >>> (threads, startNounce, d_outputHashes[thr_id], d_resultNonces[thr_id]);

	hipMemcpy(resNounce, d_resultNonces[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
}
