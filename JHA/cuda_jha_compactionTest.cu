#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include <sm_30_intrinsics.h>

#ifdef __INTELLISENSE__
#define __shfl_up(a,b)
#endif

static uint32_t *d_tempBranch1Nonces[MAX_GPUS];
static uint32_t *d_numValid[MAX_GPUS];
static uint32_t *h_numValid[MAX_GPUS];

static uint32_t *d_partSum[2][MAX_GPUS]; // für bis zu vier partielle Summen

// True/False tester
typedef uint32_t(*cuda_compactTestFunction_t)(uint32_t *inpHash);

__device__ uint32_t JackpotTrueTest(uint32_t *inpHash)
{
	uint32_t tmp = inpHash[0] & 0x01;
	return (tmp == 1);
}

__device__ uint32_t JackpotFalseTest(uint32_t *inpHash)
{
	uint32_t tmp = inpHash[0] & 0x01;
	return (tmp == 0);
}

__device__ cuda_compactTestFunction_t d_JackpotTrueFunction = JackpotTrueTest, d_JackpotFalseFunction = JackpotFalseTest;

cuda_compactTestFunction_t h_JackpotTrueFunction[MAX_GPUS], h_JackpotFalseFunction[MAX_GPUS];

// Setup-Function
__host__
void jackpot_compactTest_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyFromSymbol(&h_JackpotTrueFunction[thr_id], HIP_SYMBOL(d_JackpotTrueFunction), sizeof(cuda_compactTestFunction_t));
	hipMemcpyFromSymbol(&h_JackpotFalseFunction[thr_id], HIP_SYMBOL(d_JackpotFalseFunction), sizeof(cuda_compactTestFunction_t));

	// wir brauchen auch Speicherplatz auf dem Device
	hipMalloc(&d_tempBranch1Nonces[thr_id], sizeof(uint32_t) * threads * 2);	
	hipMalloc(&d_numValid[thr_id], 2*sizeof(uint32_t));
	hipHostMalloc(&h_numValid[thr_id], 2*sizeof(uint32_t));

	uint32_t s1;
	s1 = (threads / 256) * 2;

	hipMalloc(&d_partSum[0][thr_id], sizeof(uint32_t) * s1); // BLOCKSIZE (Threads/Block)
	hipMalloc(&d_partSum[1][thr_id], sizeof(uint32_t) * s1); // BLOCKSIZE (Threads/Block)
}

__host__
void jackpot_compactTest_cpu_free(int thr_id)
{
	hipFree(d_tempBranch1Nonces[thr_id]);
	hipFree(d_numValid[thr_id]);

	hipFree(d_partSum[0][thr_id]);
	hipFree(d_partSum[1][thr_id]);

	hipHostFree(h_numValid[thr_id]);
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 300
/**
 * __shfl_up() calculates a source lane ID by subtracting delta from the caller's lane ID, and clamping to the range 0..width-1
 */
#undef __shfl_up
#define __shfl_up(var, delta, width) (0)
#endif

// Die Summenfunktion (vom NVIDIA SDK)
__global__
void jackpot_compactTest_gpu_SCAN(uint32_t *data, int width, uint32_t *partial_sums=NULL, cuda_compactTestFunction_t testFunc=NULL,
	uint32_t threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
	extern __shared__ uint32_t sums[];
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	//int lane_id = id % warpSize;
	int lane_id = id % width;
	// determine a warp_id within a block
	 //int warp_id = threadIdx.x / warpSize;
	int warp_id = threadIdx.x / width;

	sums[lane_id] = 0;

	// Below is the basic structure of using a shfl instruction
	// for a scan.
	// Record "value" as a variable - we accumulate it along the way
	uint32_t value;
	if(testFunc != NULL)
	{
		if (id < threads)
		{
			uint32_t *inpHash;
			if(d_validNonceTable == NULL)
			{
				// keine Nonce-Liste
				inpHash = &inpHashes[id<<4];
			}else
			{
				// Nonce-Liste verfügbar
				int nonce = d_validNonceTable[id] - startNounce;
				inpHash = &inpHashes[nonce<<4];
			}			
			value = (*testFunc)(inpHash);
		}else
		{
			value = 0;
		}
	}else
	{
		value = data[id];
	}

	__syncthreads();

	// Now accumulate in log steps up the chain
	// compute sums, with another thread's value who is
	// distance delta away (i).  Note
	// those threads where the thread 'i' away would have
	// been out of bounds of the warp are unaffected.  This
	// creates the scan sum.
#pragma unroll

	for (int i=1; i<=width; i*=2)
	{
		uint32_t n = __shfl_up((int)value, i, width);

		if (lane_id >= i) value += n;
	}

	// value now holds the scan value for the individual thread
	// next sum the largest values for each warp

	// write the sum of the warp to smem
	//if (threadIdx.x % warpSize == warpSize-1)
	if (threadIdx.x % width == width-1)
	{
		sums[warp_id] = value;
	}

	__syncthreads();

	//
	// scan sum the warp sums
	// the same shfl scan operation, but performed on warp sums
	//
	if (warp_id == 0)
	{
		uint32_t warp_sum = sums[lane_id];

		for (int i=1; i<=width; i*=2)
		{
			uint32_t n = __shfl_up((int)warp_sum, i, width);

		if (lane_id >= i) warp_sum += n;
		}

		sums[lane_id] = warp_sum;
	}

	__syncthreads();

	// perform a uniform add across warps in the block
	// read neighbouring warp's sum and add it to threads value
	uint32_t blockSum = 0;

	if (warp_id > 0)
	{
		blockSum = sums[warp_id-1];
	}

	value += blockSum;

	// Now write out our result
	data[id] = value;

	// last thread has sum, write write out the block's sum
	if (partial_sums != NULL && threadIdx.x == blockDim.x-1)
	{
		partial_sums[blockIdx.x] = value;
	}
}

// Uniform add: add partial sums array
__global__
void jackpot_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
	__shared__ uint32_t buf;
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

	if (id > len) return;

	if (threadIdx.x == 0)
	{
		buf = partial_sums[blockIdx.x];
	}

	__syncthreads();
	data[id] += buf;
}

// Der Scatter
__global__
void jackpot_compactTest_gpu_SCATTER(uint32_t *sum, uint32_t *outp, cuda_compactTestFunction_t testFunc,
	uint32_t threads=0, uint32_t startNounce=0, uint32_t *inpHashes=NULL, uint32_t *d_validNonceTable=NULL)
{
	int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
	uint32_t actNounce = id;
	uint32_t value;
	if (id < threads)
	{
//		uint32_t nounce = startNounce + id;
		uint32_t *inpHash;
		if(d_validNonceTable == NULL)
		{
			// keine Nonce-Liste
			inpHash = &inpHashes[id<<4];
		}else
		{
			// Nonce-Liste verfügbar
			int nonce = d_validNonceTable[id] - startNounce;
			actNounce = nonce;
			inpHash = &inpHashes[nonce<<4];
		}

		value = (*testFunc)(inpHash);
	}else
	{
		value = 0;
	}

	if( value )
	{
		int idx = sum[id];
		if(idx > 0)
			outp[idx-1] = startNounce + actNounce;
	}
}

__host__
static uint32_t jackpot_compactTest_roundUpExp(uint32_t val)
{
	if(val == 0)
		return 0;

	uint32_t mask = 0x80000000;
	while( (val & mask) == 0 ) mask = mask >> 1;

	if( (val & (~mask)) != 0 )
		return mask << 1;

	return mask;
}

__host__
void jackpot_compactTest_cpu_singleCompaction(int thr_id, uint32_t threads, uint32_t *nrm, uint32_t *d_nonces1,
	cuda_compactTestFunction_t function, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	int orgThreads = threads;
	threads = (int)jackpot_compactTest_roundUpExp((uint32_t)threads);
	// threadsPerBlock ausrechnen
	int blockSize = 256;
	int nSummen = threads / blockSize;

	int thr1 = (threads+blockSize-1) / blockSize;
	int thr2 = threads / (blockSize*blockSize);
	int blockSize2 = (nSummen < blockSize) ? nSummen : blockSize;
	int thr3 = (nSummen + blockSize2-1) / blockSize2;

	bool callThrid = (thr2 > 0) ? true : false;

	// Erster Initialscan
	jackpot_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(
		d_tempBranch1Nonces[thr_id], 32, d_partSum[0][thr_id], function, orgThreads, startNounce, inpHashes, d_validNonceTable);	

	// weitere Scans
	if(callThrid)
	{		
		jackpot_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum[0][thr_id], 32, d_partSum[1][thr_id]);
		jackpot_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum[1][thr_id], (thr2>32) ? 32 : thr2);
	}else
	{
		jackpot_compactTest_gpu_SCAN<<<thr3,blockSize2, 32*sizeof(uint32_t)>>>(d_partSum[0][thr_id], (blockSize2>32) ? 32 : blockSize2);
	}

	// Sync + Anzahl merken
	hipStreamSynchronize(NULL);

	if(callThrid)
		hipMemcpy(nrm, &(d_partSum[1][thr_id])[thr2-1], sizeof(uint32_t), hipMemcpyDeviceToHost);
	else
		hipMemcpy(nrm, &(d_partSum[0][thr_id])[nSummen-1], sizeof(uint32_t), hipMemcpyDeviceToHost);

	
	// Addieren
	if(callThrid)
	{
		jackpot_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum[0][thr_id]+blockSize, d_partSum[1][thr_id], blockSize*thr2);
	}
	jackpot_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch1Nonces[thr_id]+blockSize, d_partSum[0][thr_id], threads);
	
	// Scatter
	jackpot_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch1Nonces[thr_id], d_nonces1, 
		function, orgThreads, startNounce, inpHashes, d_validNonceTable);

	// Sync
	hipStreamSynchronize(NULL);
}

////// ACHTUNG: Diese funktion geht aktuell nur mit threads > 65536 (Am besten 256 * 1024 oder 256*2048)
__host__
void jackpot_compactTest_cpu_dualCompaction(int thr_id, uint32_t threads, uint32_t *nrm, uint32_t *d_nonces1,
	uint32_t *d_nonces2, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable)
{
	jackpot_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[0], d_nonces1, h_JackpotTrueFunction[thr_id], startNounce, inpHashes, d_validNonceTable);
	jackpot_compactTest_cpu_singleCompaction(thr_id, threads, &nrm[1], d_nonces2, h_JackpotFalseFunction[thr_id], startNounce, inpHashes, d_validNonceTable);

	/*
	// threadsPerBlock ausrechnen
	int blockSize = 256;
	int thr1 = threads / blockSize;
	int thr2 = threads / (blockSize*blockSize);

	// 1
	jackpot_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(d_tempBranch1Nonces[thr_id], 32, d_partSum1[thr_id], h_JackpotTrueFunction[thr_id], threads, startNounce, inpHashes);
	jackpot_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum1[thr_id], 32, d_partSum2[thr_id]);
	jackpot_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum2[thr_id], (thr2>32) ? 32 : thr2);
	hipStreamSynchronize(NULL);
	hipMemcpy(&nrm[0], &(d_partSum2[thr_id])[thr2-1], sizeof(uint32_t), hipMemcpyDeviceToHost);
	jackpot_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum1[thr_id]+blockSize, d_partSum2[thr_id], blockSize*thr2);
	jackpot_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch1Nonces[thr_id]+blockSize, d_partSum1[thr_id], threads);

	// 2
	jackpot_compactTest_gpu_SCAN<<<thr1,blockSize, 32*sizeof(uint32_t)>>>(d_tempBranch2Nonces[thr_id], 32, d_partSum1[thr_id], h_JackpotFalseFunction[thr_id], threads, startNounce, inpHashes);
	jackpot_compactTest_gpu_SCAN<<<thr2,blockSize, 32*sizeof(uint32_t)>>>(d_partSum1[thr_id], 32, d_partSum2[thr_id]);
	jackpot_compactTest_gpu_SCAN<<<1, thr2, 32*sizeof(uint32_t)>>>(d_partSum2[thr_id], (thr2>32) ? 32 : thr2);
	hipStreamSynchronize(NULL);
	hipMemcpy(&nrm[1], &(d_partSum2[thr_id])[thr2-1], sizeof(uint32_t), hipMemcpyDeviceToHost);	
	jackpot_compactTest_gpu_ADD<<<thr2-1, blockSize>>>(d_partSum1[thr_id]+blockSize, d_partSum2[thr_id], blockSize*thr2);
	jackpot_compactTest_gpu_ADD<<<thr1-1, blockSize>>>(d_tempBranch2Nonces[thr_id]+blockSize, d_partSum1[thr_id], threads);
	
	// Hier ist noch eine Besonderheit: in d_tempBranch1Nonces sind die element von 1...nrm1 die Interessanten
	// Schritt 3: Scatter
	jackpot_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch1Nonces[thr_id], d_nonces1, h_JackpotTrueFunction[thr_id], threads, startNounce, inpHashes);
	jackpot_compactTest_gpu_SCATTER<<<thr1,blockSize,0>>>(d_tempBranch2Nonces[thr_id], d_nonces2, h_JackpotFalseFunction[thr_id], threads, startNounce, inpHashes);
	hipStreamSynchronize(NULL);
	*/
}

__host__
void jackpot_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
	uint32_t *d_nonces1, uint32_t *nrm1, uint32_t *d_nonces2, uint32_t *nrm2, int order)
{
	// Wenn validNonceTable genutzt wird, dann werden auch nur die Nonces betrachtet, die dort enthalten sind
	// "threads" ist in diesem Fall auf die Länge dieses Array's zu setzen!

	jackpot_compactTest_cpu_dualCompaction(thr_id, threads,
		h_numValid[thr_id], d_nonces1, d_nonces2,
		startNounce, inpHashes, d_validNonceTable);

	hipStreamSynchronize(NULL); // Das original braucht zwar etwas CPU-Last, ist an dieser Stelle aber evtl besser
	*nrm1 = h_numValid[thr_id][0];
	*nrm2 = h_numValid[thr_id][1];
}
