#include "hip/hip_runtime.h"
/**
 * JHA v8 algorithm - compatible implementation
 * @author tpruvot@github 05-2017
 */

extern "C" {
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "quark/cuda_quark.h"

static uint32_t *d_hash[MAX_GPUS] = { 0 };
static uint32_t *d_hash_br2[MAX_GPUS];
static uint32_t *d_tempBranch[MAX_GPUS];

extern void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

// CPU HASH
extern "C" void jha_hash(void *output, const void *input)
{
	uint32_t hash[16];

	sph_blake512_context     ctx_blake;
	sph_groestl512_context   ctx_groestl;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_skein512_context     ctx_skein;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, input, 80);
	sph_keccak512_close(&ctx_keccak, hash);

	for (int rnd = 0; rnd < 3; rnd++)
	{
		if (hash[0] & 0x01) {
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512 (&ctx_groestl, (&hash), 64);
			sph_groestl512_close(&ctx_groestl, (&hash));
		}
		else {
			sph_skein512_init(&ctx_skein);
			sph_skein512 (&ctx_skein, (&hash), 64);
			sph_skein512_close(&ctx_skein, (&hash));
		}

		if (hash[0] & 0x01) {
			sph_blake512_init(&ctx_blake);
			sph_blake512 (&ctx_blake, (&hash), 64);
			sph_blake512_close(&ctx_blake, (&hash));
		}
		else {
			sph_jh512_init(&ctx_jh);
			sph_jh512 (&ctx_jh, (&hash), 64);
			sph_jh512_close(&ctx_jh, (&hash));
		}
	}
	memcpy(output, hash, 32);
}

__global__ __launch_bounds__(128, 8)
void jha_filter_gpu(const uint32_t threads, const uint32_t* d_hash, uint32_t* d_branch2, uint32_t* d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t offset = thread * 16U; // 64U / sizeof(uint32_t);
		uint4 *psrc = (uint4*) (&d_hash[offset]);
		d_NonceBranch[thread] = ((uint8_t*)psrc)[0] & 0x01;
		if (d_NonceBranch[thread]) return;
		// uint4 = 4x uint32_t = 16 bytes
		uint4 *pdst = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__global__ __launch_bounds__(128, 8)
void jha_merge_gpu(const uint32_t threads, uint32_t* d_hash, uint32_t* d_branch2, uint32_t* const d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads && !d_NonceBranch[thread])
	{
		const uint32_t offset = thread * 16U;
		uint4 *pdst = (uint4*) (&d_hash[offset]);
		uint4 *psrc = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__host__
uint32_t jha_filter_cpu(const int thr_id, const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// extract algo permution hashes to a second branch buffer
	jha_filter_gpu <<<grid, block>>> (threads, inpHashes, d_branch2, d_tempBranch[thr_id]);
	return threads;
}

__host__
void jha_merge_cpu(const int thr_id, const uint32_t threads, uint32_t *outpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// put back second branch hashes to the common buffer d_hash
	jha_merge_gpu <<<grid, block>>> (threads, outpHashes, d_branch2, d_tempBranch[thr_id]);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_jha(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[22];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 20);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		cuda_get_arch(thr_id);
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_hash_br2[thr_id], (size_t) 64 * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_tempBranch[thr_id], sizeof(uint32_t) * throughput));

		jackpot_keccak512_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], pdata[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		jackpot_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		for (int rnd = 0; rnd < 3; rnd++)
		{
			jha_filter_cpu(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			jha_merge_cpu(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);

			jha_filter_cpu(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			jha_merge_cpu(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		CUDA_LOG_ERROR();

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			jha_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					jha_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	CUDA_LOG_ERROR();

	return 0;
}

// cleanup
extern "C" void free_jha(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_hash_br2[thr_id]);
	hipFree(d_tempBranch[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	CUDA_LOG_ERROR();

	hipDeviceSynchronize();
	init[thr_id] = false;
}
