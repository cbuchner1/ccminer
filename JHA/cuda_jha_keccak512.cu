#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include "miner.h"

// ZR5
__constant__ uint32_t d_OriginalData[20];

__constant__ uint32_t c_PaddedMessage[18];
__constant__ uint64_t c_State[25];

#define U32TO64_LE(p) \
	(((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
	*p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];

static __device__ __forceinline__ void
keccak_block(uint64_t *s, const uint32_t *in, const uint64_t *keccak_round_constants) {
	size_t i;
	uint64_t t[5], u[5], v, w;

	/* absorb input */
	#pragma unroll 9
	for (i = 0; i < 72 / 8; i++, in += 2)
		s[i] ^= U32TO64_LE(in);

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];
	}
}

// Setup-Funktionen
__host__
void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL( c_keccak_round_constants),
						host_keccak_round_constants,
						sizeof(host_keccak_round_constants),
						0, hipMemcpyHostToDevice);
}

#define cKeccakB    1600
#define cKeccakR    576

#define cKeccakR_SizeInBytes    (cKeccakR / 8)
#define crypto_hash_BYTES 64

#if (cKeccakB == 1600)
	typedef unsigned long long UINT64;
	typedef UINT64 tKeccakLane;
	#define cKeccakNumberOfRounds 24
#endif

#define cKeccakLaneSizeInBits   (sizeof(tKeccakLane) * 8)

#define ROL(a, offset) ((((tKeccakLane)a) << ((offset) % cKeccakLaneSizeInBits)) ^ (((tKeccakLane)a) >> (cKeccakLaneSizeInBits-((offset) % cKeccakLaneSizeInBits))))
#if ((cKeccakB/25) == 8)
	#define ROL_mult8(a, offset) ((tKeccakLane)a)
#else
	#define ROL_mult8(a, offset) ROL(a, offset)
#endif

const tKeccakLane KeccakF_RoundConstants[cKeccakNumberOfRounds] = {
	(tKeccakLane)0x0000000000000001ULL,
	(tKeccakLane)0x0000000000008082ULL,
	(tKeccakLane)0x800000000000808aULL,
	(tKeccakLane)0x8000000080008000ULL,
	(tKeccakLane)0x000000000000808bULL,
	(tKeccakLane)0x0000000080000001ULL,
	(tKeccakLane)0x8000000080008081ULL,
	(tKeccakLane)0x8000000000008009ULL,
	(tKeccakLane)0x000000000000008aULL,
	(tKeccakLane)0x0000000000000088ULL,
	(tKeccakLane)0x0000000080008009ULL,
	(tKeccakLane)0x000000008000000aULL,
	(tKeccakLane)0x000000008000808bULL,
	(tKeccakLane)0x800000000000008bULL,
	(tKeccakLane)0x8000000000008089ULL,
	(tKeccakLane)0x8000000000008003ULL,
	(tKeccakLane)0x8000000000008002ULL,
	(tKeccakLane)0x8000000000000080ULL
#if (cKeccakB >= 400)
  , (tKeccakLane)0x000000000000800aULL,
	(tKeccakLane)0x800000008000000aULL
#if (cKeccakB >= 800)
  , (tKeccakLane)0x8000000080008081ULL,
	(tKeccakLane)0x8000000000008080ULL
#if (cKeccakB == 1600)
  , (tKeccakLane)0x0000000080000001ULL,
	(tKeccakLane)0x8000000080008008ULL
#endif
#endif
#endif
};

void KeccakF(tKeccakLane * state, const tKeccakLane *in, int laneCount)
{
	while ( --laneCount >= 0 ) {
		state[laneCount] ^= in[laneCount];
	}

	{
		tKeccakLane Aba, Abe, Abi, Abo, Abu;
		tKeccakLane Aga, Age, Agi, Ago, Agu;
		tKeccakLane Aka, Ake, Aki, Ako, Aku;
		tKeccakLane Ama, Ame, Ami, Amo, Amu;
		tKeccakLane Asa, Ase, Asi, Aso, Asu;
		tKeccakLane BCa, BCe, BCi, BCo, BCu;
		tKeccakLane Da, De, Di, Do, Du;
		tKeccakLane Eba, Ebe, Ebi, Ebo, Ebu;
		tKeccakLane Ega, Ege, Egi, Ego, Egu;
		tKeccakLane Eka, Eke, Eki, Eko, Eku;
		tKeccakLane Ema, Eme, Emi, Emo, Emu;
		tKeccakLane Esa, Ese, Esi, Eso, Esu;
		#define    round    laneCount

		//copyFromState(A, state)
		Aba = state[ 0];
		Abe = state[ 1];
		Abi = state[ 2];
		Abo = state[ 3];
		Abu = state[ 4];
		Aga = state[ 5];
		Age = state[ 6];
		Agi = state[ 7];
		Ago = state[ 8];
		Agu = state[ 9];
		Aka = state[10];
		Ake = state[11];
		Aki = state[12];
		Ako = state[13];
		Aku = state[14];
		Ama = state[15];
		Ame = state[16];
		Ami = state[17];
		Amo = state[18];
		Amu = state[19];
		Asa = state[20];
		Ase = state[21];
		Asi = state[22];
		Aso = state[23];
		Asu = state[24];

		for( round = 0; round < cKeccakNumberOfRounds; round += 2 )
		{
			//    prepareTheta
			BCa = Aba^Aga^Aka^Ama^Asa;
			BCe = Abe^Age^Ake^Ame^Ase;
			BCi = Abi^Agi^Aki^Ami^Asi;
			BCo = Abo^Ago^Ako^Amo^Aso;
			BCu = Abu^Agu^Aku^Amu^Asu;

			//thetaRhoPiChiIotaPrepareTheta(round  , A, E)
			Da = BCu^ROL(BCe, 1);
			De = BCa^ROL(BCi, 1);
			Di = BCe^ROL(BCo, 1);
			Do = BCi^ROL(BCu, 1);
			Du = BCo^ROL(BCa, 1);

			Aba ^= Da;
			BCa = Aba;
			Age ^= De;
			BCe = ROL(Age, 44);
			Aki ^= Di;
			BCi = ROL(Aki, 43);
			Amo ^= Do;
			BCo = ROL(Amo, 21);
			Asu ^= Du;
			BCu = ROL(Asu, 14);
			Eba =   BCa ^((~BCe)&  BCi );
			Eba ^= (tKeccakLane)KeccakF_RoundConstants[round];
			Ebe =   BCe ^((~BCi)&  BCo );
			Ebi =   BCi ^((~BCo)&  BCu );
			Ebo =   BCo ^((~BCu)&  BCa );
			Ebu =   BCu ^((~BCa)&  BCe );

			Abo ^= Do;
			BCa = ROL(Abo, 28);
			Agu ^= Du;
			BCe = ROL(Agu, 20);
			Aka ^= Da;
			BCi = ROL(Aka,  3);
			Ame ^= De;
			BCo = ROL(Ame, 45);
			Asi ^= Di;
			BCu = ROL(Asi, 61);
			Ega =   BCa ^((~BCe)&  BCi );
			Ege =   BCe ^((~BCi)&  BCo );
			Egi =   BCi ^((~BCo)&  BCu );
			Ego =   BCo ^((~BCu)&  BCa );
			Egu =   BCu ^((~BCa)&  BCe );

			Abe ^= De;
			BCa = ROL(Abe,  1);
			Agi ^= Di;
			BCe = ROL(Agi,  6);
			Ako ^= Do;
			BCi = ROL(Ako, 25);
			Amu ^= Du;
			BCo = ROL_mult8(Amu,  8);
			Asa ^= Da;
			BCu = ROL(Asa, 18);
			Eka =   BCa ^((~BCe)&  BCi );
			Eke =   BCe ^((~BCi)&  BCo );
			Eki =   BCi ^((~BCo)&  BCu );
			Eko =   BCo ^((~BCu)&  BCa );
			Eku =   BCu ^((~BCa)&  BCe );

			Abu ^= Du;
			BCa = ROL(Abu, 27);
			Aga ^= Da;
			BCe = ROL(Aga, 36);
			Ake ^= De;
			BCi = ROL(Ake, 10);
			Ami ^= Di;
			BCo = ROL(Ami, 15);
			Aso ^= Do;
			BCu = ROL_mult8(Aso, 56);
			Ema =   BCa ^((~BCe)&  BCi );
			Eme =   BCe ^((~BCi)&  BCo );
			Emi =   BCi ^((~BCo)&  BCu );
			Emo =   BCo ^((~BCu)&  BCa );
			Emu =   BCu ^((~BCa)&  BCe );

			Abi ^= Di;
			BCa = ROL(Abi, 62);
			Ago ^= Do;
			BCe = ROL(Ago, 55);
			Aku ^= Du;
			BCi = ROL(Aku, 39);
			Ama ^= Da;
			BCo = ROL(Ama, 41);
			Ase ^= De;
			BCu = ROL(Ase,  2);
			Esa =   BCa ^((~BCe)&  BCi );
			Ese =   BCe ^((~BCi)&  BCo );
			Esi =   BCi ^((~BCo)&  BCu );
			Eso =   BCo ^((~BCu)&  BCa );
			Esu =   BCu ^((~BCa)&  BCe );

			//    prepareTheta
			BCa = Eba^Ega^Eka^Ema^Esa;
			BCe = Ebe^Ege^Eke^Eme^Ese;
			BCi = Ebi^Egi^Eki^Emi^Esi;
			BCo = Ebo^Ego^Eko^Emo^Eso;
			BCu = Ebu^Egu^Eku^Emu^Esu;

			//thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
			Da = BCu^ROL(BCe, 1);
			De = BCa^ROL(BCi, 1);
			Di = BCe^ROL(BCo, 1);
			Do = BCi^ROL(BCu, 1);
			Du = BCo^ROL(BCa, 1);

			Eba ^= Da;
			BCa = Eba;
			Ege ^= De;
			BCe = ROL(Ege, 44);
			Eki ^= Di;
			BCi = ROL(Eki, 43);
			Emo ^= Do;
			BCo = ROL(Emo, 21);
			Esu ^= Du;
			BCu = ROL(Esu, 14);
			Aba =   BCa ^((~BCe)&  BCi );
			Aba ^= (tKeccakLane)KeccakF_RoundConstants[round+1];
			Abe =   BCe ^((~BCi)&  BCo );
			Abi =   BCi ^((~BCo)&  BCu );
			Abo =   BCo ^((~BCu)&  BCa );
			Abu =   BCu ^((~BCa)&  BCe );

			Ebo ^= Do;
			BCa = ROL(Ebo, 28);
			Egu ^= Du;
			BCe = ROL(Egu, 20);
			Eka ^= Da;
			BCi = ROL(Eka, 3);
			Eme ^= De;
			BCo = ROL(Eme, 45);
			Esi ^= Di;
			BCu = ROL(Esi, 61);
			Aga =   BCa ^((~BCe)&  BCi );
			Age =   BCe ^((~BCi)&  BCo );
			Agi =   BCi ^((~BCo)&  BCu );
			Ago =   BCo ^((~BCu)&  BCa );
			Agu =   BCu ^((~BCa)&  BCe );

			Ebe ^= De;
			BCa = ROL(Ebe, 1);
			Egi ^= Di;
			BCe = ROL(Egi, 6);
			Eko ^= Do;
			BCi = ROL(Eko, 25);
			Emu ^= Du;
			BCo = ROL_mult8(Emu, 8);
			Esa ^= Da;
			BCu = ROL(Esa, 18);
			Aka =   BCa ^((~BCe)&  BCi );
			Ake =   BCe ^((~BCi)&  BCo );
			Aki =   BCi ^((~BCo)&  BCu );
			Ako =   BCo ^((~BCu)&  BCa );
			Aku =   BCu ^((~BCa)&  BCe );

			Ebu ^= Du;
			BCa = ROL(Ebu, 27);
			Ega ^= Da;
			BCe = ROL(Ega, 36);
			Eke ^= De;
			BCi = ROL(Eke, 10);
			Emi ^= Di;
			BCo = ROL(Emi, 15);
			Eso ^= Do;
			BCu = ROL_mult8(Eso, 56);
			Ama =   BCa ^((~BCe)&  BCi );
			Ame =   BCe ^((~BCi)&  BCo );
			Ami =   BCi ^((~BCo)&  BCu );
			Amo =   BCo ^((~BCu)&  BCa );
			Amu =   BCu ^((~BCa)&  BCe );

			Ebi ^= Di;
			BCa = ROL(Ebi, 62);
			Ego ^= Do;
			BCe = ROL(Ego, 55);
			Eku ^= Du;
			BCi = ROL(Eku, 39);
			Ema ^= Da;
			BCo = ROL(Ema, 41);
			Ese ^= De;
			BCu = ROL(Ese, 2);
			Asa =   BCa ^((~BCe)&  BCi );
			Ase =   BCe ^((~BCi)&  BCo );
			Asi =   BCi ^((~BCo)&  BCu );
			Aso =   BCo ^((~BCu)&  BCa );
			Asu =   BCu ^((~BCa)&  BCe );
		}

		//copyToState(state, A)
		state[ 0] = Aba;
		state[ 1] = Abe;
		state[ 2] = Abi;
		state[ 3] = Abo;
		state[ 4] = Abu;
		state[ 5] = Aga;
		state[ 6] = Age;
		state[ 7] = Agi;
		state[ 8] = Ago;
		state[ 9] = Agu;
		state[10] = Aka;
		state[11] = Ake;
		state[12] = Aki;
		state[13] = Ako;
		state[14] = Aku;
		state[15] = Ama;
		state[16] = Ame;
		state[17] = Ami;
		state[18] = Amo;
		state[19] = Amu;
		state[20] = Asa;
		state[21] = Ase;
		state[22] = Asi;
		state[23] = Aso;
		state[24] = Asu;

		#undef    round
	}
}

// inlen kann 72...143 betragen
__host__
void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen)
{
	const unsigned char *in = (const unsigned char*)pdata;

	tKeccakLane state[5 * 5];
	unsigned char temp[cKeccakR_SizeInBytes];

	memset( state, 0, sizeof(state) );

	for ( /* empty */; inlen >= cKeccakR_SizeInBytes; inlen -= cKeccakR_SizeInBytes, in += cKeccakR_SizeInBytes)
	{
		KeccakF( state, (const tKeccakLane*)in, cKeccakR_SizeInBytes / sizeof(tKeccakLane) );
	}

	// Copy state of the first round (72 Bytes)
	// in Constant Memory
	hipMemcpyToSymbol(HIP_SYMBOL( c_State),
						state,
						sizeof(state),
						0, hipMemcpyHostToDevice);

	// second part
	memcpy(temp, in, inlen);
	temp[inlen++] = 1;
	memset(temp + inlen, 0, cKeccakR_SizeInBytes - inlen);
	temp[cKeccakR_SizeInBytes-1] |= 0x80;

	// Copy rest of the message in constant memory
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage),
						temp,
						cKeccakR_SizeInBytes,
						0, hipMemcpyHostToDevice);
}

__global__
void jackpot_keccak512_gpu_hash(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;

		int hashPosition = nounce - startNounce;

		uint32_t message[18];
		#pragma unroll 18
		for(int i=0;i<18;i++)
			message[i] = c_PaddedMessage[i];

		message[1] = cuda_swab32(nounce);

		// State init
		uint64_t keccak_gpu_state[25];
		#pragma unroll 25
		for (int i=0; i<25; i++)
			keccak_gpu_state[i] = c_State[i];

		// den Block einmal gut durchschütteln
		keccak_block(keccak_gpu_state, message, c_keccak_round_constants);

		uint32_t hash[16];

		#pragma unroll 8
		for (size_t i = 0; i < 64; i += 8) {
			U64TO32_LE((&hash[i/4]), keccak_gpu_state[i / 8]);
		}

		// copy hash
		uint32_t *outpHash = (uint32_t*)&g_hash[8 * hashPosition];

		#pragma unroll 16
		for(int i=0;i<16;i++)
			outpHash[i] = hash[i];
	}
}

__host__
void jackpot_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	jackpot_keccak512_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash);
	MyStreamSynchronize(NULL, order, thr_id);
}


/* zr5 keccak, no nonce swab32 */

__global__
void zr5_keccak512_gpu_hash(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;
		uint32_t message[18];

		#pragma unroll 18
		for(int i=0; i<18; i++)
			message[i] = c_PaddedMessage[i];

		message[1] = nounce;

		// Get mid-state
		uint64_t keccak_gpu_state[25];
		#pragma unroll 25
		for (int i=0; i<25; i++)
			keccak_gpu_state[i] = c_State[i];

		keccak_block(keccak_gpu_state, message, c_keccak_round_constants);

		uint32_t hash[16];

		#pragma unroll 8
		for (int i = 0; i < 8; i++) {
			U64TO32_LE((&hash[i*2]), keccak_gpu_state[i]);
		}

		// Output (64 bytes hash required)
		uint32_t hashPosition = nounce - startNounce;
		//uint32_t *outpHash = (uint32_t*) (&g_hash[hashPosition*8]);
		//#pragma unroll 16
		//for(int i=0; i<16; i++)
		//	outpHash[i] = hash[i];

		uint4 *outpHash = (uint4*) (&g_hash[hashPosition*8]);
		uint4 *psrc = (uint4*) hash;
		outpHash[0] = psrc[0];
		outpHash[1] = psrc[1];
		outpHash[2] = psrc[2];
		outpHash[3] = psrc[3];
	}
}

__host__
void zr5_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	zr5_keccak512_gpu_hash<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
	MyStreamSynchronize(NULL, 0, thr_id);
}

/* required for the second hash part of zr5 */

__global__
void zr5_keccak512_gpu_hash_pok(uint32_t threads, uint32_t startNounce, uint32_t *g_hash, uint16_t *d_poks, uint32_t version)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;

		uint32_t message[18]; /* 72 bytes */

		// pok - hash[0] from prev hash
		message[0] = version | (0x10000UL * d_poks[thread]);
		#pragma unroll
		for (int i=1; i<18; i++) {
			message[i]=d_OriginalData[i];
		}

		// first bloc
		uint64_t keccak_gpu_state[25] = { 0 };
		keccak_block(keccak_gpu_state, message, c_keccak_round_constants);

		// second bloc
		message[0] = d_OriginalData[18];
		message[1] = nounce; //cuda_swab32(nounce);
		message[2] = 1;

		#pragma unroll
		for(int i=3; i<17; i++)
			message[i] = 0;

		message[17] = 0x80000000UL;

		keccak_block(keccak_gpu_state, message, c_keccak_round_constants);

		uint32_t hash[16];

		#pragma unroll 8
		for (size_t i = 0; i < 8; i++) {
			U64TO32_LE((&hash[i*2]), keccak_gpu_state[i]);
		}

		//uint32_t *outpHash = &g_hash[thread * 16];
		//#pragma unroll 16
		//for(int i=0; i<16; i++)
		//	outpHash[i] = hash[i];

		uint4 *outpHash = (uint4*) (&g_hash[thread * 16]);
		uint4 *psrc = (uint4*) hash;
		outpHash[0] = psrc[0];
		outpHash[1] = psrc[1];
		outpHash[2] = psrc[2];
		outpHash[3] = psrc[3];
	}
}

__host__
void zr5_keccak512_cpu_hash_pok(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t* pdata, uint32_t *d_hash, uint16_t *d_poks)
{
	const uint32_t threadsperblock = 256;
	const uint32_t version = (pdata[0] & (~POK_DATA_MASK)) | (use_pok ? POK_BOOL_MASK : 0);

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	hipMemcpyToSymbol(HIP_SYMBOL(d_OriginalData), pdata, sizeof(d_OriginalData), 0, hipMemcpyHostToDevice);
	zr5_keccak512_gpu_hash_pok<<<grid, block>>>(threads, startNounce, d_hash, d_poks, version);
	MyStreamSynchronize(NULL, 10, thr_id);
}
