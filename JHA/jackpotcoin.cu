#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "quark/cuda_quark.h"

static uint32_t *d_hash[MAX_GPUS] = { 0 };

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_jackpotNonces[MAX_GPUS] = { 0 };
static uint32_t *d_branch1Nonces[MAX_GPUS] = { 0 };
static uint32_t *d_branch2Nonces[MAX_GPUS] = { 0 };
static uint32_t *d_branch3Nonces[MAX_GPUS] = { 0 };

extern void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void jackpot_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_compactTest_cpu_free(int thr_id);
extern void jackpot_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
                                            uint32_t *d_nonces1, uint32_t *nrm1, uint32_t *d_nonces2, uint32_t *nrm2, int order);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

// CPU HASH JHA v8
extern "C" void jackpothash(void *state, const void *input)
{
	uint32_t hash[16];
	unsigned int rnd;

	sph_blake512_context     ctx_blake;
	sph_groestl512_context   ctx_groestl;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_skein512_context     ctx_skein;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, input, 80);
	sph_keccak512_close(&ctx_keccak, hash);

	for (rnd = 0; rnd < 3; rnd++)
	{
		if (hash[0] & 0x01) {
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512 (&ctx_groestl, (&hash), 64);
			sph_groestl512_close(&ctx_groestl, (&hash));
		}
		else {
			sph_skein512_init(&ctx_skein);
			sph_skein512 (&ctx_skein, (&hash), 64);
			sph_skein512_close(&ctx_skein, (&hash));
		}

		if (hash[0] & 0x01) {
			sph_blake512_init(&ctx_blake);
			sph_blake512 (&ctx_blake, (&hash), 64);
			sph_blake512_close(&ctx_blake, (&hash));
		}
		else {
			sph_jh512_init(&ctx_jh);
			sph_jh512 (&ctx_jh, (&hash), 64);
			sph_jh512_close(&ctx_jh, (&hash));
		}
	}
	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_jackpot(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[22];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 20);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		cuda_get_arch(thr_id);
		if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
			gpulog(LOG_ERR, thr_id, "Sorry, This algo is not supported by this GPU arch (SM 3.0 required)");
			proper_exit(EXIT_CODE_CUDA_ERROR);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		jackpot_keccak512_cpu_init(thr_id, throughput);
		jackpot_compactTest_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		hipMalloc(&d_branch1Nonces[thr_id], (size_t) sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch2Nonces[thr_id], (size_t) sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch3Nonces[thr_id], (size_t) sizeof(uint32_t)*throughput*2);

		CUDA_SAFE_CALL(hipMalloc(&d_jackpotNonces[thr_id], (size_t) sizeof(uint32_t)*throughput*2));

		init[thr_id] = true;
	}

	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], pdata[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// erstes Keccak512 Hash mit CUDA
		jackpot_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		uint32_t nrm1, nrm2, nrm3;

		// Runde 1 (ohne Gröstl)

		jackpot_compactTest_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch1Nonces[thr_id], &nrm1,
				d_branch3Nonces[thr_id], &nrm3,
				order++);

		// verfolge den skein-pfad weiter
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// noch schnell Blake & JH
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		CUDA_LOG_ERROR();

		work->nonces[0] = cuda_check_hash_branch(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);

			// jackpothash function gibt die Zahl der Runden zurück
			jackpothash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
#if 0
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					jackpothash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#else
				pdata[19] = work->nonces[0] + 1; // cursor
#endif
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	CUDA_LOG_ERROR();

	return 0;
}

// cleanup
extern "C" void free_jackpot(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_branch1Nonces[thr_id]);
	hipFree(d_branch2Nonces[thr_id]);
	hipFree(d_branch3Nonces[thr_id]);
	hipFree(d_jackpotNonces[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	jackpot_compactTest_cpu_free(thr_id);

	hipFree(d_hash[thr_id]);

	cuda_check_cpu_free(thr_id);
	CUDA_LOG_ERROR();

	hipDeviceSynchronize();

	init[thr_id] = false;
}
