#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void jackpot_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void jackpot_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void jackpot_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable, 
											uint32_t *d_nonces1, uint32_t *nrm1,
											uint32_t *d_nonces2, uint32_t *nrm2,
											int order);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_jackpotNonces[MAX_GPUS];
static uint32_t *d_branch1Nonces[MAX_GPUS];
static uint32_t *d_branch2Nonces[MAX_GPUS];
static uint32_t *d_branch3Nonces[MAX_GPUS];

// Original jackpothash Funktion aus einem miner Quelltext
extern "C" unsigned int jackpothash(void *state, const void *input)
{
    sph_blake512_context     ctx_blake;
    sph_groestl512_context   ctx_groestl;
    sph_jh512_context        ctx_jh;
    sph_keccak512_context    ctx_keccak;
    sph_skein512_context     ctx_skein;

    uint32_t hash[16];

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, input, 80);
    sph_keccak512_close(&ctx_keccak, hash);

    unsigned int round;
    for (round = 0; round < 3; round++) {
        if (hash[0] & 0x01) {
           sph_groestl512_init(&ctx_groestl);
           sph_groestl512 (&ctx_groestl, (&hash), 64);
           sph_groestl512_close(&ctx_groestl, (&hash));
        }
        else {
           sph_skein512_init(&ctx_skein);
           sph_skein512 (&ctx_skein, (&hash), 64);
           sph_skein512_close(&ctx_skein, (&hash));
        }
        if (hash[0] & 0x01) {
           sph_blake512_init(&ctx_blake);
           sph_blake512 (&ctx_blake, (&hash), 64);
           sph_blake512_close(&ctx_blake, (&hash));
        }
        else {
           sph_jh512_init(&ctx_jh);
           sph_jh512 (&ctx_jh, (&hash), 64);
           sph_jh512_close(&ctx_jh, (&hash));
        }
    }
    memcpy(state, hash, 32);

    return round;
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_jackpot(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  device_intensity(thr_id, __func__, 1U << 20);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		jackpot_keccak512_cpu_init(thr_id, throughput);
		jackpot_compactTest_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput*2);

		CUDA_SAFE_CALL(hipMalloc(&d_jackpotNonces[thr_id], sizeof(uint32_t)*throughput*2));

		init[thr_id] = true;
	}

	uint32_t endiandata[22];
	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], pdata[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// erstes Keccak512 Hash mit CUDA
		jackpot_keccak512_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		uint32_t nrm1, nrm2, nrm3;

		// Runde 1 (ohne Gröstl)

		jackpot_compactTest_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch1Nonces[thr_id], &nrm1,
				d_branch3Nonces[thr_id], &nrm3,
				order++);

		// verfolge den skein-pfad weiter
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

		// noch schnell Blake & JH
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		jackpot_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2,
			order++);

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		uint32_t foundNonce = cuda_check_hash_branch(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);

			// diese jackpothash Funktion gibt die Zahl der Runden zurück
			jackpothash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for nonce %08x does not validate on CPU!",
					device_map[thr_id], foundNonce);
			}
		}

		if ((uint64_t) pdata[19] + throughput > max_nonce) {
			*hashes_done = pdata[19] - first_nonce;
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	return 0;
}
