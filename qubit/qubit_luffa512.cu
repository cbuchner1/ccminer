#include "hip/hip_runtime.h"
/*******************************************************************************
 * luffa512 for 80-bytes input (with midstate precalc by klausT)
 */

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"

static __constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)
static __constant__ uint32_t statebufferpre[8];
static __constant__ uint32_t statechainvpre[40];

#define MULT2(a,j) {\
	tmp = a[7+(8*j)];\
	a[7+(8*j)] = a[6+(8*j)];\
	a[6+(8*j)] = a[5+(8*j)];\
	a[5+(8*j)] = a[4+(8*j)];\
	a[4+(8*j)] = a[3+(8*j)] ^ tmp;\
	a[3+(8*j)] = a[2+(8*j)] ^ tmp;\
	a[2+(8*j)] = a[1+(8*j)];\
	a[1+(8*j)] = a[0+(8*j)] ^ tmp;\
	a[0+(8*j)] = tmp;\
}

#define TWEAK(a0,a1,a2,a3,j) { \
	a0 = (a0<<(j))|(a0>>(32-j));\
	a1 = (a1<<(j))|(a1>>(32-j));\
	a2 = (a2<<(j))|(a2>>(32-j));\
	a3 = (a3<<(j))|(a3>>(32-j));\
}

#define STEP(c0,c1) { \
	SUBCRUMB(chainv[0],chainv[1],chainv[2],chainv[3],tmp);\
	SUBCRUMB(chainv[5],chainv[6],chainv[7],chainv[4],tmp);\
	MIXWORD(chainv[0],chainv[4]);\
	MIXWORD(chainv[1],chainv[5]);\
	MIXWORD(chainv[2],chainv[6]);\
	MIXWORD(chainv[3],chainv[7]);\
	ADD_CONSTANT(chainv[0],chainv[4],c0,c1);\
}

#define SUBCRUMB(a0,a1,a2,a3,a4)\
	a4  = a0;\
	a0 |= a1;\
	a2 ^= a3;\
	a1  = ~a1;\
	a0 ^= a3;\
	a3 &= a4;\
	a1 ^= a3;\
	a3 ^= a2;\
	a2 &= a0;\
	a0  = ~a0;\
	a2 ^= a1;\
	a1 |= a3;\
	a4 ^= a1;\
	a3 ^= a2;\
	a2 &= a1;\
	a1 ^= a0;\
	a0  = a4;

#define MIXWORD(a0,a4)\
	a4 ^= a0;\
	a0  = (a0<<2) | (a0>>(30));\
	a0 ^= a4;\
	a4  = (a4<<14) | (a4>>(18));\
	a4 ^= a0;\
	a0  = (a0<<10) | (a0>>(22));\
	a0 ^= a4;\
	a4  = (a4<<1) | (a4>>(31));

#define ADD_CONSTANT(a0,b0,c0,c1)\
	a0 ^= c0;\
	b0 ^= c1;

/* initial values of chaining variables */
__constant__ uint32_t c_IV[40];
static const uint32_t h_IV[40] = {
	0x6d251e69,0x44b051e0,0x4eaa6fb4,0xdbf78465,
	0x6e292011,0x90152df4,0xee058139,0xdef610bb,
	0xc3b44b95,0xd9d2f256,0x70eee9a0,0xde099fa3,
	0x5d9b0557,0x8fc944b3,0xcf1ccf0e,0x746cd581,
	0xf7efc89d,0x5dba5781,0x04016ce5,0xad659c05,
	0x0306194f,0x666d1836,0x24aa230a,0x8b264ae7,
	0x858075d5,0x36d79cce,0xe571f7d7,0x204b1f67,
	0x35870c6a,0x57e9e923,0x14bcb808,0x7cde72ce,
	0x6c68e9be,0x5ec41e22,0xc825b7c7,0xaffb4363,
	0xf5df3999,0x0fc688f1,0xb07224cc,0x03e86cea};

__constant__ uint32_t c_CNS[80];
static const uint32_t h_CNS[80] = {
	0x303994a6,0xe0337818,0xc0e65299,0x441ba90d,
	0x6cc33a12,0x7f34d442,0xdc56983e,0x9389217f,
	0x1e00108f,0xe5a8bce6,0x7800423d,0x5274baf4,
	0x8f5b7882,0x26889ba7,0x96e1db12,0x9a226e9d,
	0xb6de10ed,0x01685f3d,0x70f47aae,0x05a17cf4,
	0x0707a3d4,0xbd09caca,0x1c1e8f51,0xf4272b28,
	0x707a3d45,0x144ae5cc,0xaeb28562,0xfaa7ae2b,
	0xbaca1589,0x2e48f1c1,0x40a46f3e,0xb923c704,
	0xfc20d9d2,0xe25e72c1,0x34552e25,0xe623bb72,
	0x7ad8818f,0x5c58a4a4,0x8438764a,0x1e38e2e7,
	0xbb6de032,0x78e38b9d,0xedb780c8,0x27586719,
	0xd9847356,0x36eda57f,0xa2c78434,0x703aace7,
	0xb213afa5,0xe028c9bf,0xc84ebe95,0x44756f91,
	0x4e608a22,0x7e8fce32,0x56d858fe,0x956548be,
	0x343b138f,0xfe191be2,0xd0ec4e3d,0x3cb226e5,
	0x2ceb4882,0x5944a28e,0xb3ad2208,0xa1c4c355,
	0xf0d2e9e3,0x5090d577,0xac11d7fa,0x2d1925ab,
	0x1bcb66f2,0xb46496ac,0x6f2d9bc9,0xd1925ab0,
	0x78602649,0x29131ab6,0x8edae952,0x0fc053c3,
	0x3b6ba548,0x3f014f0c,0xedae9520,0xfc053c31};


/***************************************************/
__device__ __forceinline__
void rnd512(uint32_t *statebuffer, uint32_t *statechainv)
{
	int i,j;
	uint32_t t[40];
	uint32_t chainv[8];
	uint32_t tmp;

	#pragma unroll 8
	for(i=0; i<8; i++) {
		t[i]=0;
		#pragma unroll 5
		for(j=0; j<5; j++)
			t[i] ^= statechainv[i+8*j];
	}

	MULT2(t, 0);

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			statechainv[i+8*j] ^= t[i];
	}

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			t[i+8*j] = statechainv[i+8*j];
	}

	#pragma unroll
	for(j=0; j<5; j++)
		MULT2(statechainv, j);

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			statechainv[8*j+i] ^= t[8*((j+1)%5)+i];
	}

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			t[i+8*j] = statechainv[i+8*j];
	}

	#pragma unroll
	for(j=0; j<5; j++)
		MULT2(statechainv, j);

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			statechainv[8*j+i] ^= t[8*((j+4)%5)+i];
	}

	#pragma unroll 5
	for(j=0; j<5; j++) {
		#pragma unroll 8
		for(i=0; i<8; i++)
			statechainv[i+8*j] ^= statebuffer[i];
		MULT2(statebuffer, 0);
	}

	#pragma unroll
	for(i=0; i<8; i++)
		chainv[i] = statechainv[i];

	#pragma unroll
	for(i=0; i<8; i++)
		STEP(c_CNS[(2*i)], c_CNS[(2*i)+1]);

	#pragma unroll
	for(i=0; i<8; i++) {
		statechainv[i] = chainv[i];
		chainv[i] = statechainv[i+8];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],1);

	#pragma unroll
	for(i=0; i<8; i++)
		STEP(c_CNS[(2*i)+16], c_CNS[(2*i)+16+1]);

	#pragma unroll
	for(i=0; i<8; i++) {
		statechainv[i+8] = chainv[i];
		chainv[i] = statechainv[i+16];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],2);

	#pragma unroll
	for(i=0; i<8; i++)
		STEP(c_CNS[(2*i)+32],c_CNS[(2*i)+32+1]);

	#pragma unroll
	for(i=0; i<8; i++) {
		statechainv[i+16] = chainv[i];
		chainv[i] = statechainv[i+24];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],3);

	#pragma unroll
	for(i=0; i<8; i++)
		STEP(c_CNS[(2*i)+48],c_CNS[(2*i)+48+1]);

	#pragma unroll
	for(i=0; i<8; i++) {
		statechainv[i+24] = chainv[i];
		chainv[i] = statechainv[i+32];
	}

	TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],4);

	#pragma unroll
	for(i=0; i<8; i++)
		STEP(c_CNS[(2*i)+64],c_CNS[(2*i)+64+1]);

	#pragma unroll 8
	for(i=0; i<8; i++)
		statechainv[i+32] = chainv[i];
}

static void rnd512_cpu(uint32_t *statebuffer, uint32_t *statechainv)
{
	int i, j;
	uint32_t t[40];
	uint32_t chainv[8];
	uint32_t tmp;

	for (i = 0; i<8; i++) {
		t[i] = statechainv[i];
		for (j = 1; j<5; j++)
			t[i] ^= statechainv[i + 8 * j];
	}

	MULT2(t, 0);

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			statechainv[i + 8 * j] ^= t[i];
	}

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			t[i + 8 * j] = statechainv[i + 8 * j];
	}

	for (j = 0; j<5; j++)
		MULT2(statechainv, j);

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			statechainv[8 * j + i] ^= t[8 * ((j + 1) % 5) + i];
	}

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			t[i + 8 * j] = statechainv[i + 8 * j];
	}

	for (j = 0; j<5; j++)
		MULT2(statechainv, j);

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			statechainv[8 * j + i] ^= t[8 * ((j + 4) % 5) + i];
	}

	for (j = 0; j<5; j++) {
		for (i = 0; i<8; i++)
			statechainv[i + 8 * j] ^= statebuffer[i];
		MULT2(statebuffer, 0);
	}

	for (i = 0; i<8; i++)
		chainv[i] = statechainv[i];

	for (i = 0; i<8; i++)
		STEP(h_CNS[(2 * i)], h_CNS[(2 * i) + 1]);

	for (i = 0; i<8; i++) {
		statechainv[i] = chainv[i];
		chainv[i] = statechainv[i + 8];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 1);

	for (i = 0; i<8; i++)
		STEP(h_CNS[(2 * i) + 16], h_CNS[(2 * i) + 16 + 1]);

	for (i = 0; i<8; i++) {
		statechainv[i + 8] = chainv[i];
		chainv[i] = statechainv[i + 16];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 2);

	for (i = 0; i<8; i++)
		STEP(h_CNS[(2 * i) + 32], h_CNS[(2 * i) + 32 + 1]);

	for (i = 0; i<8; i++) {
		statechainv[i + 16] = chainv[i];
		chainv[i] = statechainv[i + 24];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 3);

	for (i = 0; i<8; i++)
		STEP(h_CNS[(2 * i) + 48], h_CNS[(2 * i) + 48 + 1]);

	for (i = 0; i<8; i++) {
		statechainv[i + 24] = chainv[i];
		chainv[i] = statechainv[i + 32];
	}

	TWEAK(chainv[4], chainv[5], chainv[6], chainv[7], 4);

	for (i = 0; i<8; i++)
		STEP(h_CNS[(2 * i) + 64], h_CNS[(2 * i) + 64 + 1]);

	for (i = 0; i<8; i++)
		statechainv[i + 32] = chainv[i];
}

/***************************************************/
__device__ __forceinline__
void Update512(uint32_t* statebuffer, uint32_t *statechainv, const uint32_t *const __restrict__ data)
{
	#pragma unroll
	for (int i = 0; i<8; i++)
		statebuffer[i] = cuda_swab32((data[i]));
	rnd512(statebuffer, statechainv);

	#pragma unroll
	for(int i=0; i<8; i++)
		statebuffer[i] = cuda_swab32((data[i+8]));
	rnd512(statebuffer, statechainv);

	#pragma unroll
	for(int i=0; i<4; i++)
		statebuffer[i] = cuda_swab32((data[i+16]));
}


/***************************************************/
__device__ __forceinline__
void finalization512(uint32_t* statebuffer, uint32_t *statechainv, uint32_t *b)
{
	int i,j;

	statebuffer[4] = 0x80000000U;

	#pragma unroll 3
	for(int i=5; i<8; i++)
		statebuffer[i] = 0;
	rnd512(statebuffer, statechainv);

	/*---- blank round with m=0 ----*/
	#pragma unroll
	for(i=0; i<8; i++)
		statebuffer[i] =0;
	rnd512(statebuffer, statechainv);

	#pragma unroll
	for(i=0; i<8; i++) {
		b[i] = 0;
		#pragma unroll 5
		for(j=0; j<5; j++)
			b[i] ^= statechainv[i+8*j];
		b[i] = cuda_swab32((b[i]));
	}

	#pragma unroll
	for(i=0; i<8; i++)
		statebuffer[i]=0;
	rnd512(statebuffer, statechainv);

	#pragma unroll
	for(i=0; i<8; i++)
	{
		b[8+i] = 0;
		#pragma unroll 5
		for(j=0; j<5; j++)
			b[8+i] ^= statechainv[i+8*j];
		b[8+i] = cuda_swab32((b[8+i]));
	}
}


/***************************************************/
__global__
void qubit_luffa512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = startNounce + thread;
		union {
		uint64_t buf64[16];
		uint32_t buf32[32];
		} buff;

		#pragma unroll 8
		for (int i=8; i < 16; i++)
			buff.buf64[i] = c_PaddedMessage80[i];

		// die Nounce durch die thread-spezifische ersetzen
		buff.buf64[9] = REPLACE_HIDWORD(buff.buf64[9], cuda_swab32(nounce));

		uint32_t statebuffer[8], statechainv[40];

		#pragma unroll
		for (int i = 0; i<4; i++)
			statebuffer[i] = cuda_swab32(buff.buf32[i + 16]);

		#pragma unroll 4
		for (int i = 4; i<8; i++)
			statebuffer[i] = statebufferpre[i];

		#pragma unroll
		for (int i = 0; i<40; i++)
			statechainv[i] = statechainvpre[i];

		uint32_t *outHash = &outputHash[thread * 16];
		finalization512(statebuffer, statechainv, outHash);
	}
}

__host__
void qubit_luffa512_cpu_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_IV), h_IV, sizeof(h_IV), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_CNS), h_CNS, sizeof(h_CNS), 0, hipMemcpyHostToDevice));
}

__host__
void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash,int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	size_t shared_size = 0;

	qubit_luffa512_gpu_hash_80 <<<grid, block, shared_size>>> (threads, startNounce, d_outputHash);
}

__host__
static void qubit_cpu_precalc(uint32_t* message)
{
	uint32_t statebuffer[8];
	uint32_t statechainv[40] =
	{
		0x6d251e69, 0x44b051e0, 0x4eaa6fb4, 0xdbf78465,
		0x6e292011, 0x90152df4, 0xee058139, 0xdef610bb,
		0xc3b44b95, 0xd9d2f256, 0x70eee9a0, 0xde099fa3,
		0x5d9b0557, 0x8fc944b3, 0xcf1ccf0e, 0x746cd581,
		0xf7efc89d, 0x5dba5781, 0x04016ce5, 0xad659c05,
		0x0306194f, 0x666d1836, 0x24aa230a, 0x8b264ae7,
		0x858075d5, 0x36d79cce, 0xe571f7d7, 0x204b1f67,
		0x35870c6a, 0x57e9e923, 0x14bcb808, 0x7cde72ce,
		0x6c68e9be, 0x5ec41e22, 0xc825b7c7, 0xaffb4363,
		0xf5df3999, 0x0fc688f1, 0xb07224cc, 0x03e86cea
	};

	for (int i = 0; i<8; i++)
		statebuffer[i] = cuda_swab32(message[i]);
	rnd512_cpu(statebuffer, statechainv);

	for (int i = 0; i<8; i++)
		statebuffer[i] = cuda_swab32(message[i+8]);

	rnd512_cpu(statebuffer, statechainv);

	hipMemcpyToSymbol(HIP_SYMBOL(statebufferpre), statebuffer, sizeof(statebuffer), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(statechainvpre), statechainv, sizeof(statechainv), 0, hipMemcpyHostToDevice);
}

__host__
void qubit_luffa512_cpu_setBlock_80(void *pdata)
{
	unsigned char PaddedMessage[128];

	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, sizeof(PaddedMessage), 0, hipMemcpyHostToDevice));
	qubit_cpu_precalc((uint32_t*) PaddedMessage);
}
