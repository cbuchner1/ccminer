#include "hip/hip_runtime.h"
/*
 * luffa_for_32.c
 * Version 2.0 (Sep 15th 2009)
 *
 * Copyright (C) 2008-2009 Hitachi, Ltd. All rights reserved.
 *
 * Hitachi, Ltd. is the owner of this software and hereby grant
 * the U.S. Government and any interested party the right to use
 * this software for the purposes of the SHA-3 evaluation process,
 * notwithstanding that this software is copyrighted.
 *
 * THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES
 * WITH REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR
 * ANY SPECIAL, DIRECT, INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES
 * WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER IN AN
 * ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
 * OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS SOFTWARE.
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>
// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

typedef unsigned char BitSequence;


#include "cuda_helper.h"
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)
__constant__ uint32_t pTarget[8];
uint32_t *d_lnounce[8];
uint32_t *d_LNonce[8];

typedef struct {
    uint32_t buffer[8]; /* Buffer to be hashed */
    uint32_t chainv[40];   /* Chaining values */
} hashState;


static __device__ __forceinline__ uint32_t BYTES_SWAP32(uint32_t x)
{
	return __byte_perm(x, x, 0x0123);
}

#define MULT2(a,j)\
    tmp = a[7+(8*j)];\
    a[7+(8*j)] = a[6+(8*j)];\
    a[6+(8*j)] = a[5+(8*j)];\
    a[5+(8*j)] = a[4+(8*j)];\
    a[4+(8*j)] = a[3+(8*j)] ^ tmp;\
    a[3+(8*j)] = a[2+(8*j)] ^ tmp;\
    a[2+(8*j)] = a[1+(8*j)];\
    a[1+(8*j)] = a[0+(8*j)] ^ tmp;\
    a[0+(8*j)] = tmp;

#define TWEAK(a0,a1,a2,a3,j)\
    a0 = (a0<<(j))|(a0>>(32-j));\
    a1 = (a1<<(j))|(a1>>(32-j));\
    a2 = (a2<<(j))|(a2>>(32-j));\
    a3 = (a3<<(j))|(a3>>(32-j));

#define STEP(c0,c1)\
    SUBCRUMB(chainv[0],chainv[1],chainv[2],chainv[3],tmp);\
    SUBCRUMB(chainv[5],chainv[6],chainv[7],chainv[4],tmp);\
    MIXWORD(chainv[0],chainv[4]);\
    MIXWORD(chainv[1],chainv[5]);\
    MIXWORD(chainv[2],chainv[6]);\
    MIXWORD(chainv[3],chainv[7]);\
    ADD_CONSTANT(chainv[0],chainv[4],c0,c1);

#define SUBCRUMB(a0,a1,a2,a3,a4)\
    a4  = a0;\
    a0 |= a1;\
    a2 ^= a3;\
    a1  = ~a1;\
    a0 ^= a3;\
    a3 &= a4;\
    a1 ^= a3;\
    a3 ^= a2;\
    a2 &= a0;\
    a0  = ~a0;\
    a2 ^= a1;\
    a1 |= a3;\
    a4 ^= a1;\
    a3 ^= a2;\
    a2 &= a1;\
    a1 ^= a0;\
    a0  = a4;

#define MIXWORD(a0,a4)\
    a4 ^= a0;\
    a0  = (a0<<2) | (a0>>(30));\
    a0 ^= a4;\
    a4  = (a4<<14) | (a4>>(18));\
    a4 ^= a0;\
    a0  = (a0<<10) | (a0>>(22));\
    a0 ^= a4;\
    a4  = (a4<<1) | (a4>>(31));

#define ADD_CONSTANT(a0,b0,c0,c1)\
    a0 ^= c0;\
    b0 ^= c1;

/* initial values of chaining variables */
__constant__ uint32_t c_IV[40];
const uint32_t h2_IV[40] = {
    0x6d251e69,0x44b051e0,0x4eaa6fb4,0xdbf78465,
    0x6e292011,0x90152df4,0xee058139,0xdef610bb,
    0xc3b44b95,0xd9d2f256,0x70eee9a0,0xde099fa3,
    0x5d9b0557,0x8fc944b3,0xcf1ccf0e,0x746cd581,
    0xf7efc89d,0x5dba5781,0x04016ce5,0xad659c05,
    0x0306194f,0x666d1836,0x24aa230a,0x8b264ae7,
    0x858075d5,0x36d79cce,0xe571f7d7,0x204b1f67,
    0x35870c6a,0x57e9e923,0x14bcb808,0x7cde72ce,
    0x6c68e9be,0x5ec41e22,0xc825b7c7,0xaffb4363,
    0xf5df3999,0x0fc688f1,0xb07224cc,0x03e86cea};

__constant__ uint32_t c_CNS[80];
uint32_t h2_CNS[80] = {
    0x303994a6,0xe0337818,0xc0e65299,0x441ba90d,
    0x6cc33a12,0x7f34d442,0xdc56983e,0x9389217f,
    0x1e00108f,0xe5a8bce6,0x7800423d,0x5274baf4,
    0x8f5b7882,0x26889ba7,0x96e1db12,0x9a226e9d,
    0xb6de10ed,0x01685f3d,0x70f47aae,0x05a17cf4,
    0x0707a3d4,0xbd09caca,0x1c1e8f51,0xf4272b28,
    0x707a3d45,0x144ae5cc,0xaeb28562,0xfaa7ae2b,
    0xbaca1589,0x2e48f1c1,0x40a46f3e,0xb923c704,
    0xfc20d9d2,0xe25e72c1,0x34552e25,0xe623bb72,
    0x7ad8818f,0x5c58a4a4,0x8438764a,0x1e38e2e7,
    0xbb6de032,0x78e38b9d,0xedb780c8,0x27586719,
    0xd9847356,0x36eda57f,0xa2c78434,0x703aace7,
    0xb213afa5,0xe028c9bf,0xc84ebe95,0x44756f91,
    0x4e608a22,0x7e8fce32,0x56d858fe,0x956548be,
    0x343b138f,0xfe191be2,0xd0ec4e3d,0x3cb226e5,
    0x2ceb4882,0x5944a28e,0xb3ad2208,0xa1c4c355,
    0xf0d2e9e3,0x5090d577,0xac11d7fa,0x2d1925ab,
    0x1bcb66f2,0xb46496ac,0x6f2d9bc9,0xd1925ab0,
    0x78602649,0x29131ab6,0x8edae952,0x0fc053c3,
    0x3b6ba548,0x3f014f0c,0xedae9520,0xfc053c31};


/***************************************************/
__device__ __forceinline__ void rnd512(hashState *state)
{
    int i,j;
    uint32_t t[40];
    uint32_t chainv[8];
    uint32_t tmp;

#pragma unroll 8
    for(i=0;i<8;i++) {
        t[i]=0;
#pragma unroll 5
        for(j=0;j<5;j++) {
            t[i] ^= state->chainv[i+8*j];
        }
    }

    MULT2(t, 0);

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[i+8*j] ^= t[i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            t[i+8*j] = state->chainv[i+8*j];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
        MULT2(state->chainv, j);
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[8*j+i] ^= t[8*((j+1)%5)+i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            t[i+8*j] = state->chainv[i+8*j];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
        MULT2(state->chainv, j);
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[8*j+i] ^= t[8*((j+4)%5)+i];
        }
    }

#pragma unroll 5
    for(j=0;j<5;j++) {
#pragma unroll 8
        for(i=0;i<8;i++) {
            state->chainv[i+8*j] ^= state->buffer[i];
        }
        MULT2(state->buffer, 0);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        chainv[i] = state->chainv[i];
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)],c_CNS[(2*i)+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i] = chainv[i];
        chainv[i] = state->chainv[i+8];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],1);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+16],c_CNS[(2*i)+16+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+8] = chainv[i];
        chainv[i] = state->chainv[i+16];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],2);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+32],c_CNS[(2*i)+32+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+16] = chainv[i];
        chainv[i] = state->chainv[i+24];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],3);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+48],c_CNS[(2*i)+48+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+24] = chainv[i];
        chainv[i] = state->chainv[i+32];
    }

    TWEAK(chainv[4],chainv[5],chainv[6],chainv[7],4);

#pragma unroll 8
    for(i=0;i<8;i++) {
        STEP(c_CNS[(2*i)+64],c_CNS[(2*i)+64+1]);
    }

#pragma unroll 8
    for(i=0;i<8;i++) {
        state->chainv[i+32] = chainv[i];
    }
}


__device__ __forceinline__ void Update512(hashState *state, const BitSequence *data) 
{
#pragma unroll 8
    for(int i=0;i<8;i++) state->buffer[i] = BYTES_SWAP32(((uint32_t*)data)[i]);
    rnd512(state);

#pragma unroll 8
    for(int i=0;i<8;i++) state->buffer[i] = BYTES_SWAP32(((uint32_t*)(data+32))[i]);
    rnd512(state);
#pragma unroll 4
    for(int i=0;i<4;i++) state->buffer[i] = BYTES_SWAP32(((uint32_t*)(data+64))[i]);
}


/***************************************************/
__device__ __forceinline__ void finalization512(hashState *state, uint32_t *b)
{
    int i,j;

    state->buffer[4] = 0x80000000;
#pragma unroll 3
    for(int i=5;i<8;i++) state->buffer[i] = 0;
    rnd512(state);

    /*---- blank round with m=0 ----*/
#pragma unroll 8
    for(i=0;i<8;i++) state->buffer[i] =0;
    rnd512(state);

#pragma unroll 8
    for(i=0;i<8;i++) {
        b[i] = 0;
#pragma unroll 5
        for(j=0;j<5;j++) {
            b[i] ^= state->chainv[i+8*j];
        }
        b[i] = BYTES_SWAP32((b[i]));
    }

#pragma unroll 8
    for(i=0;i<8;i++) state->buffer[i]=0;
    rnd512(state);

#pragma unroll 8
    for(i=0;i<8;i++) {
        b[8+i] = 0;
#pragma unroll 5
        for(j=0;j<5;j++) {
            b[8+i] ^= state->chainv[i+8*j];
        }
        b[8+i] = BYTES_SWAP32((b[8+i]));
    }
}


/***************************************************/
// Die Hash-Funktion
__global__ void qubit_luffa512_gpu_hash_80(int threads, uint32_t startNounce, void *outputHash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = startNounce + thread;
        
union {
uint64_t buf64[16];
uint32_t buf32[32];
} buff;
#pragma unroll 16
		for (int i=0; i < 16; ++i) buff.buf64[i] = c_PaddedMessage80[i];

		// die Nounce durch die thread-spezifische ersetzen
		buff.buf64[9] = REPLACE_HIWORD(buff.buf64[9], cuda_swab32(nounce));


        hashState state;
#pragma unroll 40
        for(int i=0;i<40;i++) state.chainv[i] = c_IV[i];
#pragma unroll 8
        for(int i=0;i<8;i++) state.buffer[i] = 0;
        Update512(&state, (BitSequence*)buff.buf32);
		uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;
        finalization512(&state, (uint32_t*)outHash);
    }
}

__global__ void qubit_luffa512_gpu_finalhash_80(int threads, uint32_t startNounce, void *outputHash, uint32_t *resNounce)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = startNounce + thread;
        
union {
uint64_t buf64[16];
uint32_t buf32[32];
} buff;

uint32_t Hash[16];
#pragma unroll 16
		for (int i=0; i < 16; ++i) buff.buf64[i] = c_PaddedMessage80[i];

		// die Nounce durch die thread-spezifische ersetzen
		buff.buf64[9] = REPLACE_HIWORD(buff.buf64[9], cuda_swab32(nounce));


        hashState state;
#pragma unroll 40
        for(int i=0;i<40;i++) state.chainv[i] = c_IV[i];
#pragma unroll 8
        for(int i=0;i<8;i++) state.buffer[i] = 0;
        Update512(&state, (BitSequence*)buff.buf32);
        finalization512(&state, Hash);

		bool rc = true;
		int position = -1;
#pragma unroll 8	
		for (int i = 7; i >= 0; i--) {
			if (Hash[i] > pTarget[i]) {
				if(position < i) {
                    position = i;
                    rc = false;
                }
				
			}
			if (Hash[i] < pTarget[i]) {
				if(position < i) {
                    position = i;
                    rc = true;
                }
			}
		}

		if(rc == true)
		{
			if(resNounce[0] > nounce)
				resNounce[0] = nounce;
		}

    }
}
// Setup-Funktionen
__host__ void qubit_luffa512_cpu_init(int thr_id, int threads)
{
    hipMemcpyToSymbol(HIP_SYMBOL( c_IV), h2_IV, sizeof(h2_IV), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( c_CNS), h2_CNS, sizeof(h2_CNS), 0, hipMemcpyHostToDevice );
	hipMalloc(&d_LNonce[thr_id], sizeof(uint32_t)); 
	hipHostMalloc(&d_lnounce[thr_id], 1*sizeof(uint32_t));
}

__host__ uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash,int order)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_LNonce[thr_id], 0xffffffff, sizeof(uint32_t));
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    qubit_luffa512_gpu_finalhash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash, d_LNonce[thr_id]);
    MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(d_lnounce[thr_id], d_LNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	result = *d_lnounce[thr_id];
	return result;
}

__host__ void qubit_luffa512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_outputHash,int order)
{
    const int threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    qubit_luffa512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, d_outputHash);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void qubit_luffa512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void qubit_luffa512_cpufinal_setBlock_80(void *pdata, const void *ptarget)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	PaddedMessage[80] = 0x80;
	PaddedMessage[111] = 1;
	PaddedMessage[126] = 0x02;
	PaddedMessage[127] = 0x80;
	hipMemcpyToSymbol(HIP_SYMBOL( pTarget), ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	
	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}