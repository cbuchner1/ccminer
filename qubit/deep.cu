#include "hip/hip_runtime.h"
/*
 * deepcoin algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void x11_cubehash512_cpu_init(int thr_id, uint32_t threads);
extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern "C" void deephash(void *state, const void *input)
{
	uint8_t _ALIGN(64) hash[64];

	// luffa-80 cubehash-64 echo-64
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_echo512_context ctx_echo;

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_deep(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput =  device_intensity(thr_id, __func__, 1U << 19); // 256*256*8
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], throughput * 64));

		qubit_luffa512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			deephash(vhash64, endiandata);

			if (vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nonce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
