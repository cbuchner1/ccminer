#include "hip/hip_runtime.h"
/*
 * luffa 80 algo (Introduced by Doomcoin)
 */
extern "C" {
#include "sph/sph_luffa.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern "C" void luffa_hash(void *state, const void *input)
{
	uint8_t _ALIGN(64) hash[64];

	sph_luffa512_context ctx_luffa;

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_luffa(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 21);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x0000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		qubit_luffa512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		qubit_luffa512_cpu_hash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id], 0);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			luffa_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					luffa_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_luffa(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	cuda_check_cpu_free(thr_id);

	init[thr_id] = false;
	hipDeviceSynchronize();
}
