#include "hip/hip_runtime.h"
/*
 * "pluck" kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2015  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   djm34
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>
 


extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];

__device__  uint8_t *  hashbuffer;
uint32_t *d_PlNonce[8];
__constant__  uint32_t pTarget[8];
__constant__  uint32_t  c_data[20];
#include "cuda_vector.h" 


#define HASH_MEMORY_8bit 131072
#define HASH_MEMORY_32bit 32768
#define HASH_MEMORY 4096

static __constant__  uint32_t H256[8] = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372,
	0xA54FF53A, 0x510E527F, 0x9B05688C,
	0x1F83D9AB, 0x5BE0CD19
};

static  __constant__  uint32_t Ksha[64] = {
	0x428A2F98, 0x71374491,
	0xB5C0FBCF, 0xE9B5DBA5,
	0x3956C25B, 0x59F111F1,
	0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01,
	0x243185BE, 0x550C7DC3,
	0x72BE5D74, 0x80DEB1FE,
	0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786,
	0x0FC19DC6, 0x240CA1CC,
	0x2DE92C6F, 0x4A7484AA,
	0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D,
	0xB00327C8, 0xBF597FC7,
	0xC6E00BF3, 0xD5A79147,
	0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138,
	0x4D2C6DFC, 0x53380D13,
	0x650A7354, 0x766A0ABB,
	0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B,
	0xC24B8B70, 0xC76C51A3,
	0xD192E819, 0xD6990624,
	0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08,
	0x2748774C, 0x34B0BCB5,
	0x391C0CB3, 0x4ED8AA4A,
	0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F,
	0x84C87814, 0x8CC70208,
	0x90BEFFFA, 0xA4506CEB,
	0xBEF9A3F7, 0xC67178F2
};


#define SALSA(a,b,c,d) { \
    t =a+d; b^=rotate(t,  7);    \
    t =b+a; c^=rotate(t,  9);    \
    t =c+b; d^=rotate(t, 13);    \
    t =d+c; a^=rotate(t, 18);     \
}


#define SALSA_CORE(state) { \
\
SALSA(state.s0,state.s4,state.s8,state.sc); \
SALSA(state.s5,state.s9,state.sd,state.s1); \
SALSA(state.sa,state.se,state.s2,state.s6); \
SALSA(state.sf,state.s3,state.s7,state.sb); \
SALSA(state.s0,state.s1,state.s2,state.s3); \
SALSA(state.s5,state.s6,state.s7,state.s4); \
SALSA(state.sa,state.sb,state.s8,state.s9); \
SALSA(state.sf,state.sc,state.sd,state.se); \
	} 


static __device__ __forceinline__ uint16 xor_salsa8(const uint16 &Bx)
{
	uint32_t t;
	uint16 state = Bx;
	SALSA_CORE(state);
	SALSA_CORE(state);
	SALSA_CORE(state);
	SALSA_CORE(state);
	return(state+Bx);
}



// sha256

static __device__ __forceinline__ uint32_t bsg2_0(const uint32_t x)
{
	uint32_t r1 = SPH_ROTR32(x, 2);
	uint32_t r2 = SPH_ROTR32(x, 13);
	uint32_t r3 = SPH_ROTR32(x, 22);
	return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t bsg2_1(const uint32_t x)
{
	uint32_t r1 = SPH_ROTR32(x, 6);
	uint32_t r2 = SPH_ROTR32(x, 11);
	uint32_t r3 = SPH_ROTR32(x, 25);
	return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t ssg2_0(const uint32_t x)
{
	uint64_t r1 = SPH_ROTR32(x, 7);
	uint64_t r2 = SPH_ROTR32(x, 18);
	uint64_t r3 = shr_t32(x, 3);
	return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ uint32_t ssg2_1(const uint32_t x)
{
	uint64_t r1 = SPH_ROTR32(x, 17);
	uint64_t r2 = SPH_ROTR32(x, 19);
	uint64_t r3 = shr_t32(x, 10);
	return xor3b(r1, r2, r3);
}

static __device__ __forceinline__ void sha2_step1(const uint32_t a, const uint32_t b, const uint32_t c, uint32_t &d, const uint32_t e, 
const uint32_t f, const uint32_t g, uint32_t &h, const uint32_t in, const uint32_t Kshared)
{
	uint32_t t1, t2;
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a, b, c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

static __device__ __forceinline__ void sha2_step2(const uint32_t a, const uint32_t b, const uint32_t c, uint32_t &d, const uint32_t e, 
const uint32_t f, const uint32_t g, uint32_t &h, uint32_t* in, const uint32_t pc, const uint32_t Kshared)
{
	uint32_t t1, t2;

	int pcidx1 = (pc - 2) & 0xF;
	int pcidx2 = (pc - 7) & 0xF;
	int pcidx3 = (pc - 15) & 0xF;
	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];


	uint32_t ssg21 = ssg2_1(inx1);
	uint32_t ssg20 = ssg2_0(inx3);
	uint32_t vxandx = xandx(e, f, g);
	uint32_t bsg21 = bsg2_1(e);
	uint32_t bsg20 = bsg2_0(a);
	uint32_t andorv = andor32(a, b, c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;

}


static __device__ __forceinline__ void sha2_round_body(uint32_t* in, uint32_t* r)
{
	uint32_t a = r[0];
	uint32_t b = r[1];
	uint32_t c = r[2];
	uint32_t d = r[3];
	uint32_t e = r[4];
	uint32_t f = r[5];
	uint32_t g = r[6];
	uint32_t h = r[7];

	sha2_step1(a, b, c, d, e, f, g, h, in[0], Ksha[0]);
	sha2_step1(h, a, b, c, d, e, f, g, in[1], Ksha[1]);
	sha2_step1(g, h, a, b, c, d, e, f, in[2], Ksha[2]);
	sha2_step1(f, g, h, a, b, c, d, e, in[3], Ksha[3]);
	sha2_step1(e, f, g, h, a, b, c, d, in[4], Ksha[4]);
	sha2_step1(d, e, f, g, h, a, b, c, in[5], Ksha[5]);
	sha2_step1(c, d, e, f, g, h, a, b, in[6], Ksha[6]);
	sha2_step1(b, c, d, e, f, g, h, a, in[7], Ksha[7]);
	sha2_step1(a, b, c, d, e, f, g, h, in[8], Ksha[8]);
	sha2_step1(h, a, b, c, d, e, f, g, in[9], Ksha[9]);
	sha2_step1(g, h, a, b, c, d, e, f, in[10], Ksha[10]);
	sha2_step1(f, g, h, a, b, c, d, e, in[11], Ksha[11]);
	sha2_step1(e, f, g, h, a, b, c, d, in[12], Ksha[12]);
	sha2_step1(d, e, f, g, h, a, b, c, in[13], Ksha[13]);
	sha2_step1(c, d, e, f, g, h, a, b, in[14], Ksha[14]);
	sha2_step1(b, c, d, e, f, g, h, a, in[15], Ksha[15]);

#pragma unroll 3
	for (int i = 0; i<3; i++) {

		sha2_step2(a, b, c, d, e, f, g, h, in, 0, Ksha[16 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, in, 1, Ksha[17 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, in, 2, Ksha[18 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, in, 3, Ksha[19 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, in, 4, Ksha[20 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, in, 5, Ksha[21 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, in, 6, Ksha[22 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, in, 7, Ksha[23 + 16 * i]);
		sha2_step2(a, b, c, d, e, f, g, h, in, 8, Ksha[24 + 16 * i]);
		sha2_step2(h, a, b, c, d, e, f, g, in, 9, Ksha[25 + 16 * i]);
		sha2_step2(g, h, a, b, c, d, e, f, in, 10, Ksha[26 + 16 * i]);
		sha2_step2(f, g, h, a, b, c, d, e, in, 11, Ksha[27 + 16 * i]);
		sha2_step2(e, f, g, h, a, b, c, d, in, 12, Ksha[28 + 16 * i]);
		sha2_step2(d, e, f, g, h, a, b, c, in, 13, Ksha[29 + 16 * i]);
		sha2_step2(c, d, e, f, g, h, a, b, in, 14, Ksha[30 + 16 * i]);
		sha2_step2(b, c, d, e, f, g, h, a, in, 15, Ksha[31 + 16 * i]);

	}



	r[0] += a;
	r[1] += b;
	r[2] += c;
	r[3] += d;
	r[4] += e;
	r[5] += f;
	r[6] += g;
	r[7] += h;
}


static __device__ __forceinline__ uint8 sha256_64(uint32_t *data)
{

	uint32_t __align__(64) in[16];
    uint32_t __align__(32) buf[8];
	
	((uint16 *)in)[0] = swapvec((uint16*)data);

	((uint8*)buf)[0] = ((uint8*)H256)[0];

	sha2_round_body(in, buf);

#pragma unroll 14
	for (int i = 0; i<14; i++) { in[i + 1] = 0; }
	in[0] = 0x80000000;
	in[15] = 0x200;


	sha2_round_body(in, buf);
	return swapvec((uint8*)buf);
}


static __device__ __forceinline__ uint8 sha256_80(uint32_t nonce)
{

//	uint32_t in[16], buf[8];
	uint32_t __align__(64) in[16];
	uint32_t __align__(32) buf[8];
	((uint16 *)in)[0] = swapvec((uint16*)c_data);

	((uint8*)buf)[0] = ((uint8*)H256)[0];

	sha2_round_body(in, buf);


#pragma unroll 3
	for (int i = 0; i<3; i++) { in[i] = cuda_swab32(c_data[i + 16]); }
//	in[3] = cuda_swab32(nonce);
    in[3] = nonce;
	in[4] = 0x80000000;
	in[15] = 0x280;

#pragma unroll 10
	for (int i = 5; i<15; i++) { in[i] = 0; }

	sha2_round_body(in, buf);
	return swapvec((uint8*)buf);
}


#define SHIFT 32 * 1024 * 4
__global__ __launch_bounds__(256, 1) void pluck_gpu_hash0_v50(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		int shift = SHIFT * thread; //uint32_t
		((uint8*)(hashbuffer + shift))[0] = sha256_80(nonce);
		((uint8*)(hashbuffer + shift))[1] = make_uint8(0, 0, 0, 0, 0, 0, 0, 0);
		for (int i = 2; i < 5; i++)
		{
			uint32_t randmax = i * 32 - 4;
			uint32_t randseed[16];
			uint32_t randbuffer[16];
			uint32_t joint[16];
			uint8 Buffbuffer[2];

			((uint8*)randseed)[0] = __ldg8(&(hashbuffer + shift)[32 * i - 64]);
			((uint8*)randseed)[1] = __ldg8(&(hashbuffer + shift)[32 * i - 32]);

			

			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);

//			((uint8*)joint)[0] = __ldg8(&(hashbuffer + shift)[(i - 1) << 5]);
			((uint8*)joint)[0] = ((uint8*)randseed)[1];
#pragma unroll
			for (int j = 0; j < 8; j++)
			{
				uint32_t rand = randbuffer[j] % (randmax - 32);
				joint[j + 8] = __ldgtoint_unaligned(&(hashbuffer + shift)[rand]); 
			}

			uint8 truc = sha256_64(joint);
			((uint8*)(hashbuffer + shift))[i] = truc;
			((uint8*)randseed)[0] = ((uint8*)joint)[0];
			((uint8*)randseed)[1] = truc;


			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);


			for (int j = 0; j < 32; j += 2)
			{

				uint32_t rand = randbuffer[j / 2] % randmax;
				(hashbuffer + shift)[rand] = __ldg(&(hashbuffer + shift)[randmax + j]);
				(hashbuffer + shift)[rand + 1] = __ldg(&(hashbuffer + shift)[randmax + j + 1]);
				(hashbuffer + shift)[rand + 2] = __ldg(&(hashbuffer + shift)[randmax + j + 2]);
				(hashbuffer + shift)[rand + 3] = __ldg(&(hashbuffer + shift)[randmax + j + 3]);
			}

		} // main loop

} 
}
__global__ __launch_bounds__(256, 1) void pluck_gpu_hash_v50(int threads, uint32_t startNonce, uint32_t *nonceVector)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
 
		int shift = SHIFT * thread; //uint32_t

		for (int i = 5; i < HASH_MEMORY - 1; i++)
		{
			uint32_t randmax = i*32-4;
			uint32_t randseed[16];
			uint32_t randbuffer[16];  
			uint32_t joint[16];
			uint8 Buffbuffer[2];
            
			((uint8*)randseed)[0] = __ldg8(&(hashbuffer + shift)[32*i-64]);
			((uint8*)randseed)[1] = __ldg8(&(hashbuffer + shift)[32*i-32]);           	
			

                Buffbuffer[0] = __ldg8(&(hashbuffer + shift)[32*i - 128]);
				Buffbuffer[1] = __ldg8(&(hashbuffer + shift)[32*i - 96]);
				((uint16*)randseed)[0] ^= ((uint16*)Buffbuffer)[0];
 
			((uint16*)randbuffer)[0]= xor_salsa8(((uint16*)randseed)[0]);

			((uint8*)joint)[0] = __ldg8(&(hashbuffer + shift)[(i-1)<<5]);

#pragma unroll
			for (int j = 0; j < 8; j++)
			{
				uint32_t rand = randbuffer[j] % (randmax - 32); 
				joint[j+8] = __ldgtoint_unaligned(&(hashbuffer + shift)[rand]); 
			}
	
			uint8 truc =  sha256_64(joint);
			((uint8*)(hashbuffer + shift))[i] = truc;
			((uint8*)randseed)[0] = ((uint8*)joint)[0];
			((uint8*)randseed)[1] = truc;


	 ((uint16*)randseed)[0] ^= ((uint16*)Buffbuffer)[0];


 ((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);


			for (int j = 0; j < 32; j += 2)
			{
 
				uint32_t rand = randbuffer[j / 2] % randmax;
				
				(hashbuffer+shift)[rand] =       __ldg(&(hashbuffer+shift)[randmax+j]);
				(hashbuffer + shift)[rand + 1] = __ldg(&(hashbuffer + shift)[randmax + j + 1]);
				(hashbuffer + shift)[rand + 2] = __ldg(&(hashbuffer + shift)[randmax + j + 2]);
				(hashbuffer + shift)[rand + 3] = __ldg(&(hashbuffer + shift)[randmax + j + 3]);
			}
 
		} // main loop

		uint32_t outbuf =  __ldgtoint(&(hashbuffer + shift)[28]);

		if (outbuf <= pTarget[7]) {
			nonceVector[0] = nonce;
		}

	}
}

__global__ __launch_bounds__(128, 3) void pluck_gpu_hash0(int threads, uint32_t startNonce)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		int shift = SHIFT * thread; //uint32_t
		((uint8*)(hashbuffer + shift))[0] = sha256_80(nonce);
		((uint8*)(hashbuffer + shift))[1] = make_uint8(0, 0, 0, 0, 0, 0, 0, 0);
		for (int i = 2; i < 5; i++)
		{
			uint32_t randmax = i * 32 - 4;
			uint32_t randseed[16];
			uint32_t randbuffer[16];
			uint32_t joint[16];
			uint8 Buffbuffer[2];

			((uint8*)randseed)[0] = __ldg8(&(hashbuffer + shift)[32 * i - 64]);
			((uint8*)randseed)[1] = __ldg8(&(hashbuffer + shift)[32 * i - 32]);



			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);

			//			((uint8*)joint)[0] = __ldg8(&(hashbuffer + shift)[(i - 1) << 5]);
			((uint8*)joint)[0] = ((uint8*)randseed)[1];
#pragma unroll
			for (int j = 0; j < 8; j++)
			{
				uint32_t rand = randbuffer[j] % (randmax - 32);
				joint[j + 8] = __ldgtoint_unaligned(&(hashbuffer + shift)[rand]);
			}

			uint8 truc = sha256_64(joint);
			((uint8*)(hashbuffer + shift))[i] = truc;
			((uint8*)randseed)[0] = ((uint8*)joint)[0];
			((uint8*)randseed)[1] = truc;


			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);


			for (int j = 0; j < 32; j += 2)
			{

				uint32_t rand = randbuffer[j / 2] % randmax;
				(hashbuffer + shift)[rand] = __ldg(&(hashbuffer + shift)[randmax + j]);
				(hashbuffer + shift)[rand + 1] = __ldg(&(hashbuffer + shift)[randmax + j + 1]);
				(hashbuffer + shift)[rand + 2] = __ldg(&(hashbuffer + shift)[randmax + j + 2]);
				(hashbuffer + shift)[rand + 3] = __ldg(&(hashbuffer + shift)[randmax + j + 3]);
			}

		} // main loop

	}
}
__global__ __launch_bounds__(128, 3) void pluck_gpu_hash(int threads, uint32_t startNonce, uint32_t *nonceVector)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;

		int shift = SHIFT * thread; //uint32_t

		for (int i = 5; i < HASH_MEMORY - 1; i++)
		{
			uint32_t randmax = i * 32 - 4;
			uint32_t randseed[16];
			uint32_t randbuffer[16];
			uint32_t joint[16];
			uint8 Buffbuffer[2];

			((uint8*)randseed)[0] = __ldg8(&(hashbuffer + shift)[32 * i - 64]);
			((uint8*)randseed)[1] = __ldg8(&(hashbuffer + shift)[32 * i - 32]);


			Buffbuffer[0] = __ldg8(&(hashbuffer + shift)[32 * i - 128]);
			Buffbuffer[1] = __ldg8(&(hashbuffer + shift)[32 * i - 96]);
			((uint16*)randseed)[0] ^= ((uint16*)Buffbuffer)[0];

			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);

			((uint8*)joint)[0] = __ldg8(&(hashbuffer + shift)[(i - 1) << 5]);

#pragma unroll
			for (int j = 0; j < 8; j++)
			{
				uint32_t rand = randbuffer[j] % (randmax - 32);
				joint[j + 8] = __ldgtoint_unaligned(&(hashbuffer + shift)[rand]);
			}

			uint8 truc = sha256_64(joint);
			((uint8*)(hashbuffer + shift))[i] = truc;
			((uint8*)randseed)[0] = ((uint8*)joint)[0];
			((uint8*)randseed)[1] = truc;


			((uint16*)randseed)[0] ^= ((uint16*)Buffbuffer)[0];


			((uint16*)randbuffer)[0] = xor_salsa8(((uint16*)randseed)[0]);


			for (int j = 0; j < 32; j += 2)
			{

				uint32_t rand = randbuffer[j / 2] % randmax;

				(hashbuffer + shift)[rand] = __ldg(&(hashbuffer + shift)[randmax + j]);
				(hashbuffer + shift)[rand + 1] = __ldg(&(hashbuffer + shift)[randmax + j + 1]);
				(hashbuffer + shift)[rand + 2] = __ldg(&(hashbuffer + shift)[randmax + j + 2]);
				(hashbuffer + shift)[rand + 3] = __ldg(&(hashbuffer + shift)[randmax + j + 3]);
			}

		} // main loop

		uint32_t outbuf = __ldgtoint(&(hashbuffer + shift)[28]);

		if (outbuf <= pTarget[7]) {
			nonceVector[0] = nonce;
		}

	}
}


void pluck_cpu_init(int thr_id, int threads, uint32_t* hash)
{
    
	hipMemcpyToSymbol(HIP_SYMBOL(hashbuffer), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_PlNonce[thr_id], sizeof(uint32_t)); 

} 


__host__ uint32_t pluck_cpu_hash(int thr_id, int threads, uint32_t startNounce,  int order)
{
	uint32_t result[8] = {0xffffffff};
	hipMemset(d_PlNonce[thr_id], 0xffffffff, sizeof(uint32_t));

 
	const int threadsperblock = 128;
	
 
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	dim3 grid50((threads + 256 - 1) / 256);
	dim3 block50(256);

	if (compute_version[thr_id]==50) {
	pluck_gpu_hash0_v50 << <grid50, block50 >> >(threads, startNounce);
	pluck_gpu_hash_v50  << <grid50, block50 >> >(threads, startNounce, d_PlNonce[thr_id]);
	}
	else {
		pluck_gpu_hash0 << <grid, block >> >(threads, startNounce);
		pluck_gpu_hash << <grid, block >> >(threads, startNounce, d_PlNonce[thr_id]);
	}

	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(&result[thr_id], d_PlNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

return result[thr_id];
}



__host__ void pluck_setBlockTarget(const void *pdata, const void *ptarget)
{
	unsigned char PaddedMessage[80];
	memcpy(PaddedMessage, pdata, 80);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 10 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}