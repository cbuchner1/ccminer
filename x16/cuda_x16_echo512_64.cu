#include "hip/hip_runtime.h"
/**
 * Echo512-64 kernel for maxwell, based on alexis work
 */

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>
#include <cuda_vectors.h>

#define INTENSIVE_GMF
#include "tribus/cuda_echo512_aes.cuh"

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#define atomicExch(p,y) (*p) = y
#endif

__device__
static void echo_round_alexis(const uint32_t sharedMemory[4][256], uint32_t *W, uint32_t &k0)
{
	// Big Sub Words
	#pragma unroll 16
	for (int idx = 0; idx < 16; idx++)
		AES_2ROUND(sharedMemory,W[(idx<<2) + 0], W[(idx<<2) + 1], W[(idx<<2) + 2], W[(idx<<2) + 3], k0);

	// Shift Rows
	#pragma unroll 4
	for (int i = 0; i < 4; i++){
		uint32_t t[4];
		/// 1, 5, 9, 13
		t[0] = W[i+ 4];
		t[1] = W[i+ 8];
		t[2] = W[i+24];
		t[3] = W[i+60];
		W[i + 4] = W[i + 20];
		W[i + 8] = W[i + 40];
		W[i +24] = W[i + 56];
		W[i +60] = W[i + 44];

		W[i +20] = W[i +36];
		W[i +40] = t[1];
		W[i +56] = t[2];
		W[i +44] = W[i +28];

		W[i +28] = W[i +12];
		W[i +12] = t[3];
		W[i +36] = W[i +52];
		W[i +52] = t[0];
	}
	// Mix Columns
	#pragma unroll 4
	for (int i = 0; i < 4; i++){ // Schleife über je 2*uint32_t
		#pragma unroll 4
		for (int idx = 0; idx < 64; idx += 16){ // Schleife über die elemnte
			uint32_t a[4];
			a[0] = W[idx + i];
			a[1] = W[idx + i + 4];
			a[2] = W[idx + i + 8];
			a[3] = W[idx + i +12];

			uint32_t ab = a[0] ^ a[1];
			uint32_t bc = a[1] ^ a[2];
			uint32_t cd = a[2] ^ a[3];

			uint32_t t, t2, t3;
			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			uint32_t abx = (t  >> 7) * 27U ^ ((ab^t) << 1);
			uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[idx + i] = bc ^ a[3] ^ abx;
			W[idx + i + 4] = a[0] ^ cd ^ bcx;
			W[idx + i + 8] = ab ^ a[3] ^ cdx;
			W[idx + i +12] = ab ^ a[2] ^ (abx ^ bcx ^ cdx);
		}
	}
}

__global__ __launch_bounds__(128, 5) /* will force 80 registers */
static void x16_echo512_gpu_hash_64(uint32_t threads, uint32_t* g_hash, uint32_t* const d_filter, const uint32_t filter_val)
{
	__shared__ uint32_t sharedMemory[4][256];

	aes_gpu_init128(sharedMemory);
	__syncthreads();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t k0;
	uint32_t h[16];
	uint32_t hash[16];
	if (thread < threads)
	{
		// phi2 filter (2 hash chain branches)
		if (d_filter && d_filter[thread] != filter_val) return;

		uint32_t *Hash = &g_hash[thread<<4];

		*(uint2x4*)&h[ 0] = __ldg4((uint2x4*)&Hash[ 0]);
		*(uint2x4*)&h[ 8] = __ldg4((uint2x4*)&Hash[ 8]);

		*(uint2x4*)&hash[ 0] = *(uint2x4*)&h[ 0];
		*(uint2x4*)&hash[ 8] = *(uint2x4*)&h[ 8];

		const uint32_t P[48] = {
			0xe7e9f5f5, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xa4213d7e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//8-12
			0x01425eb8, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x65978b09, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//21-25
			0x2cb6b661, 0x6b23b3b3, 0xcf93a7cf, 0x9d9d3751, 0x9ac2dea3, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			//34-38
			0x579f9f33, 0xfbfbfbfb, 0xfbfbfbfb, 0xefefd3c7, 0xdbfde1dd, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af,
			0x34514d9e, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0xb134347e, 0xea6f7e7e, 0xbd7731bd, 0x8a8a1968,
			0x14b8a457, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af, 0x265f4382, 0xf5e7e9f5, 0xb3b36b23, 0xb3dbe7af
			//58-61
		};

		k0 = 520;

		#pragma unroll 4
		for (uint32_t idx = 0; idx < 16; idx += 4) {
			AES_2ROUND(sharedMemory, h[idx + 0], h[idx + 1], h[idx + 2], h[idx + 3], k0);
		}
		k0 += 4;

		uint32_t W[64];

		#pragma unroll 4
		for (uint32_t i = 0; i < 4; i++)
		{
			uint32_t a = P[i];
			uint32_t b = P[i + 4];
			uint32_t c = h[i + 8];
			uint32_t d = P[i + 8];

			uint32_t ab = a ^ b;
			uint32_t bc = b ^ c;
			uint32_t cd = c ^ d;


			uint32_t t =  (ab & 0x80808080);
			uint32_t t2 = (bc & 0x80808080);
			uint32_t t3 = (cd & 0x80808080);

			uint32_t abx = (t  >> 7) * 27U ^ ((ab^t) << 1);
			uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[i] = abx ^ bc ^ d;
			W[i + 4] = bcx ^ a ^ cd;
			W[i + 8] = cdx ^ ab ^ d;
			W[i +12] = abx ^ bcx ^ cdx ^ ab ^ c;

			a = P[i +12];
			b = h[i + 4];
			c = P[i +16];
			d = P[i +20];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[16 + i] = bc ^ d ^ abx;
			W[16 + i + 4] = a ^ cd ^ bcx;
			W[16 + i + 8] = d ^ ab ^ cdx;
			W[16 + i + 12] = c ^ ab ^ abx ^ bcx ^ cdx;

			a = h[i];
			b = P[24 + i + 0];
			c = P[24 + i + 4];
			d = P[24 + i + 8];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;


			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[32 + i] = bc ^ d ^ abx;
			W[32 + i + 4] = a ^ cd ^ bcx;
			W[32 + i + 8] = d ^ ab ^ cdx;
			W[32 + i + 12] = c ^ ab ^ abx ^ bcx ^ cdx;

			a = P[36 + i ];
			b = P[36 + i + 4];
			c = P[36 + i + 8];
			d = h[i + 12];

			ab = a ^ b;
			bc = b ^ c;
			cd = c ^ d;

			t = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			abx = (t >> 7) * 27U ^ ((ab^t) << 1);
			bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[48 + i] = bc ^ d ^ abx;
			W[48 + i + 4] = a ^ cd ^ bcx;
			W[48 + i + 8] = d ^ ab ^ cdx;
			W[48 + i + 12] = c ^ ab ^ abx ^ bcx ^ cdx;
		}

		for (int k = 1; k < 10; k++)
			echo_round_alexis(sharedMemory,W,k0);

		#pragma unroll 4
		for (int i = 0; i < 16; i += 4)
		{
			W[i] ^= W[32 + i] ^ 512;
			W[i + 1] ^= W[32 + i + 1];
			W[i + 2] ^= W[32 + i + 2];
			W[i + 3] ^= W[32 + i + 3];
		}
		*(uint2x4*)&Hash[ 0] = *(uint2x4*)&hash[ 0] ^ *(uint2x4*)&W[ 0];
		*(uint2x4*)&Hash[ 8] = *(uint2x4*)&hash[ 8] ^ *(uint2x4*)&W[ 8];
	}
}

__host__
void x16_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x16_echo512_gpu_hash_64 <<<grid, block>>> (threads, d_hash, NULL, 0);
}

__host__
void phi_echo512_cpu_hash_64_filtered(int thr_id, const uint32_t threads, uint32_t* g_hash, uint32_t* d_filter)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	x16_echo512_gpu_hash_64 <<<grid, block>>> (threads, g_hash, d_filter, 0);
}