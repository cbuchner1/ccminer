#include "hip/hip_runtime.h"
/**
 * echo512-80 cuda kernel for X16R algorithm
 *
 * tpruvot 2018 - GPL code
 */

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

extern __device__ __device_builtin__ void __threadfence_block(void);

#include "../x11/cuda_x11_aes.cuh"

__device__ __forceinline__ void AES_2ROUND(const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3,
	uint32_t &k0)
{
	uint32_t y0, y1, y2, y3;

	aes_round(sharedMemory,
		x0, x1, x2, x3,
		k0,
		y0, y1, y2, y3);

	aes_round(sharedMemory,
		y0, y1, y2, y3,
		x0, x1, x2, x3);

	k0++;
}

__device__
static void echo_round(uint32_t* const sharedMemory, uint32_t *W, uint32_t &k0)
{
	// Big Sub Words
	#pragma unroll 16
	for (int idx = 0; idx < 16; idx++) {
		AES_2ROUND(sharedMemory, W[(idx << 2) + 0], W[(idx << 2) + 1], W[(idx << 2) + 2], W[(idx << 2) + 3], k0);
	}

	// Shift Rows
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		uint32_t t[4];
		/// 1, 5, 9, 13
		t[0] = W[i +  4];
		t[1] = W[i +  8];
		t[2] = W[i + 24];
		t[3] = W[i + 60];

		W[i +  4] = W[i + 20];
		W[i +  8] = W[i + 40];
		W[i + 24] = W[i + 56];
		W[i + 60] = W[i + 44];

		W[i + 20] = W[i + 36];
		W[i + 40] = t[1];
		W[i + 56] = t[2];
		W[i + 44] = W[i + 28];

		W[i + 28] = W[i + 12];
		W[i + 12] = t[3];
		W[i + 36] = W[i + 52];
		W[i + 52] = t[0];
	}

	// Mix Columns
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		#pragma unroll 4
		for (int idx = 0; idx < 64; idx += 16)
		{
			uint32_t a[4];
			a[0] = W[idx + i];
			a[1] = W[idx + i + 4];
			a[2] = W[idx + i + 8];
			a[3] = W[idx + i + 12];

			uint32_t ab = a[0] ^ a[1];
			uint32_t bc = a[1] ^ a[2];
			uint32_t cd = a[2] ^ a[3];

			uint32_t t, t2, t3;
			t  = (ab & 0x80808080);
			t2 = (bc & 0x80808080);
			t3 = (cd & 0x80808080);

			uint32_t abx = (t  >> 7) * 27U ^ ((ab^t)  << 1);
			uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
			uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

			W[idx + i] = bc ^ a[3] ^ abx;
			W[idx + i +  4] = a[0] ^ cd ^ bcx;
			W[idx + i +  8] = ab ^ a[3] ^ cdx;
			W[idx + i + 12] = ab ^ a[2] ^ (abx ^ bcx ^ cdx);
		}
	}
}

__device__ __forceinline__
void cuda_echo_round_80(uint32_t *const __restrict__ sharedMemory, uint32_t *const __restrict__ data, const uint32_t nonce, uint32_t *hash)
{
	uint32_t h[29]; // <= 127 bytes input

	#pragma unroll 8
	for (int i = 0; i < 18; i += 2)
		AS_UINT2(&h[i]) = AS_UINT2(&data[i]);
	h[18] = data[18];
	h[19] = cuda_swab32(nonce);
	h[20] = 0x80;
	h[21] = h[22] = h[23] = h[24] = h[25] = h[26] = 0;
	//((uint8_t*)h)[80] = 0x80;
	//((uint8_t*)h)[128-17] = 0x02;
	//((uint8_t*)h)[128-16] = 0x80;
	//((uint8_t*)h)[128-15] = 0x02;
	h[27] = 0x2000000;
	h[28] = 0x280;
	//h[29] = h[30] = h[31] = 0;

	uint32_t k0 = 640; // bitlen
	uint32_t W[64];

	#pragma unroll 8
	for (int i = 0; i < 32; i+=4) {
		W[i] = 512; // L
		W[i+1] = 0; // H
		W[i+2] = 0; // X
		W[i+3] = 0;
	}

	uint32_t Z[16];
	#pragma unroll
	for (int i = 0;  i<16; i++) Z[i] = W[i];
	#pragma unroll
	for (int i = 32; i<61; i++) W[i] = h[i - 32];
	#pragma unroll
	for (int i = 61; i<64; i++) W[i] = 0;

	for (int i = 0; i < 10; i++)
		echo_round(sharedMemory, W, k0);

	#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		Z[i] ^= h[i] ^ W[i] ^ W[i + 32];
	}

	#pragma unroll 8
	for (int i = 0; i < 16; i += 2)
		AS_UINT2(&hash[i]) = AS_UINT2(&Z[i]);
}

__device__ __forceinline__
void echo_gpu_init(uint32_t *const __restrict__ sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 128) {
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = d_AES1[threadIdx.x];
		sharedMemory[threadIdx.x + 512] = d_AES2[threadIdx.x];
		sharedMemory[threadIdx.x + 768] = d_AES3[threadIdx.x];

		sharedMemory[threadIdx.x + 64 * 2] = d_AES0[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 256] = d_AES1[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 512] = d_AES2[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 768] = d_AES3[threadIdx.x + 64 * 2];
	}
}

__host__
void x16_echo512_cuda_init(int thr_id, const uint32_t threads)
{
	aes_cpu_init(thr_id);
}

__constant__ static uint32_t c_PaddedMessage80[20];

__host__
void x16_echo512_setBlock_80(void *endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__ __launch_bounds__(128, 7) /* will force 72 registers */
void x16_echo512_gpu_hash_80(uint32_t threads, uint32_t startNonce, uint64_t *g_hash)
{
	__shared__ uint32_t sharedMemory[1024];

	echo_gpu_init(sharedMemory);
	__threadfence_block();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t hashPosition = thread;
		uint32_t *pHash = (uint32_t*)&g_hash[hashPosition<<3];

		cuda_echo_round_80(sharedMemory, c_PaddedMessage80, startNonce + thread, pHash);
	}
}

__host__
void x16_echo512_cuda_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x16_echo512_gpu_hash_80<<<grid, block>>>(threads, startNonce, (uint64_t*)d_hash);
}
