#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// globaler Speicher für alle HeftyHashes aller Threads
extern uint32_t *heavy_heftyHashes[MAX_GPUS];
extern uint32_t *heavy_nonceVector[MAX_GPUS];

// globaler Speicher für unsere Ergebnisse
uint32_t *d_hash2output[MAX_GPUS];


/* Hash-Tabellen */
__constant__ uint32_t sha256_gpu_constantTable[64];

// muss expandiert werden
__constant__ uint32_t sha256_gpu_blockHeader[16]; // 2x512 Bit Message
__constant__ uint32_t sha256_gpu_register[8];

uint32_t sha256_cpu_hashTable[] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };
uint32_t sha256_cpu_constantTable[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
};

#define S(x, n)			(((x) >> (n)) | ((x) << (32 - (n))))
#define R(x, n)			((x) >> (n))
#define Ch(x, y, z)		((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define S0(x)			(S(x, 2) ^ S(x, 13) ^ S(x, 22))
#define S1(x)			(S(x, 6) ^ S(x, 11) ^ S(x, 25))
#define s0(x)			(S(x, 7) ^ S(x, 18) ^ R(x, 3))
#define s1(x)			(S(x, 17) ^ S(x, 19) ^ R(x, 10))

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

// Die Hash-Funktion
template <int BLOCKSIZE> __global__ void sha256_gpu_hash(uint32_t threads, uint32_t startNounce, void *outputHash, uint32_t *heftyHashes, uint32_t *nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Zähler
		uint32_t nounce = startNounce + thread;
		nonceVector[thread] = nounce;

		// jeder thread in diesem  Block bekommt sein eigenes W Array im Shared memory
		uint32_t W1[16];
		uint32_t W2[16];

		// Initialisiere die register a bis h mit der Hash-Tabelle
		uint32_t regs[8];
		uint32_t hash[8];

		// pre
#pragma unroll 8
		for (int k=0; k < 8; k++)
		{
			regs[k] = sha256_gpu_register[k];
			hash[k] = regs[k];
		}

		// 2. Runde
		//memcpy(W, &sha256_gpu_blockHeader[0], sizeof(uint32_t) * 16); // TODO: aufsplitten in zwei Teilblöcke
		//memcpy(&W[5], &heftyHashes[8 * (blockDim.x * blockIdx.x + threadIdx.x)], sizeof(uint32_t) * 8); // den richtigen Hefty1 Hash holen
#pragma unroll 16
		for(int k=0;k<16;k++)
			W1[k] = sha256_gpu_blockHeader[k];

		uint32_t offset = 8 * (blockDim.x * blockIdx.x + threadIdx.x);
#pragma unroll 8
		for(int k=0;k<8;k++)
			W1[((BLOCKSIZE-64)/4)+k] = heftyHashes[offset + k];

#pragma unroll 8
		for (int i=((BLOCKSIZE-64)/4); i < ((BLOCKSIZE-64)/4)+8; ++i) W1[i] = SWAB32(W1[i]); // die Hefty1 Hashes brauchen eine Drehung ;)
		W1[3] = SWAB32(nounce);

// Progress W1
#pragma unroll 16
		for(int j=0;j<16;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j] + W1[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

			#pragma unroll 7
			for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}

// Progress W2...W3
#pragma unroll 3
		for(int k=0;k<3;k++)
		{
	#pragma unroll 2
			for(int j=0;j<2;j++)
				W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];
	#pragma unroll 5
			for(int j=2;j<7;j++)
				W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll 8
			for(int j=7;j<15;j++)
				W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

			W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

			// Rundenfunktion
	#pragma unroll 16
			for(int j=0;j<16;j++)
			{
				uint32_t T1, T2;
				T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j + 16 * (k+1)] + W2[j];
				T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

				#pragma unroll 7
				for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
				regs[0] = T1 + T2;
				regs[4] += T1;
			}

	#pragma unroll 16
			for(int j=0;j<16;j++)
				W1[j] = W2[j];
		}

/*
		for(int j=16;j<64;j++)
			W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];

#pragma unroll 64
		for(int j=0;j<64;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j] + W[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

			#pragma unroll 7
			for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}
*/
#pragma unroll 8
		for(int k=0;k<8;k++)
			hash[k] += regs[k];

#pragma unroll 8
		for(int k=0;k<8;k++)
			((uint32_t*)outputHash)[8*thread+k] = SWAB32(hash[k]);
	}
}

// Setup Function
__host__
void sha256_cpu_init(int thr_id, uint32_t threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL(	sha256_gpu_constantTable),
						sha256_cpu_constantTable,
						sizeof(uint32_t) * 64 );

	// Speicher für alle Ergebnisse belegen
	hipMalloc(&d_hash2output[thr_id], (size_t) 8 * sizeof(uint32_t) * threads);
}

__host__
void sha256_cpu_free(int thr_id)
{
	hipFree(d_hash2output[thr_id]);
}

static int BLOCKSIZE = 84;

__host__ void sha256_cpu_setBlock(void *data, int len)
	// data muss 80/84-Byte haben!
	// heftyHash hat 32-Byte
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, len);
	if (len == 84) {
		memset(&msgBlock[21], 0, 32); // vorläufig  Nullen anstatt der Hefty1 Hashes einfüllen
		msgBlock[29] |= 0x80;
		msgBlock[31] = 928; // bitlen
	} else if (len == 80) {
		memset(&msgBlock[20], 0, 32); // vorläufig  Nullen anstatt der Hefty1 Hashes einfüllen
		msgBlock[28] |= 0x80;
		msgBlock[31] = 896; // bitlen
	}

	for(int i=0;i<31;i++) // Byteorder drehen
		msgBlock[i] = SWAB32(msgBlock[i]);

	// die erste Runde wird auf der CPU durchgeführt, da diese für
	// alle Threads gleich ist. Der Hash wird dann an die Threads
	// übergeben
	uint32_t W[64];

	// Erstelle expandierten Block W
	memcpy(W, &msgBlock[0], sizeof(uint32_t) * 16);
	for(int j=16;j<64;j++)
		W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];

	// Initialisiere die register a bis h mit der Hash-Tabelle
	uint32_t regs[8];
	uint32_t hash[8];

	// pre
	for (int k=0; k < 8; k++)
	{
		regs[k] = sha256_cpu_hashTable[k];
		hash[k] = regs[k];
	}

	// 1. Runde
	for(int j=0;j<64;j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_cpu_constantTable[j] + W[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		//#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		// sollte mal noch durch memmov ersetzt werden!
//		memcpy(&regs[1], &regs[0], sizeof(uint32_t) * 7);
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	for(int k=0;k<8;k++)
		hash[k] += regs[k];

	// hash speichern
	hipMemcpyToSymbol(HIP_SYMBOL(	sha256_gpu_register),
						hash,
						sizeof(uint32_t) * 8 );

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(HIP_SYMBOL(	sha256_gpu_blockHeader),
						&msgBlock[16],
						64);

	BLOCKSIZE = len;
}

__host__ void sha256_cpu_copyHeftyHash(int thr_id, uint32_t threads, void *heftyHashes, int copy)
{
	// Hefty1 Hashes kopieren
	if (copy)
		CUDA_SAFE_CALL(hipMemcpy(heavy_heftyHashes[thr_id], heftyHashes, 8 * sizeof(uint32_t) * threads, hipMemcpyHostToDevice));
	//else hipDeviceSynchronize();
}

__host__ void sha256_cpu_hash(int thr_id, uint32_t threads, int startNounce)
{
	const uint32_t threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Größe des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	if (BLOCKSIZE == 84)
		sha256_gpu_hash<84><<<grid, block, shared_size>>>(threads, startNounce, d_hash2output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
	else if (BLOCKSIZE == 80) {
		sha256_gpu_hash<80><<<grid, block, shared_size>>>(threads, startNounce, d_hash2output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
	}
}
