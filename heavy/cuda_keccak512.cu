#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// globaler Speicher für alle HeftyHashes aller Threads
extern uint32_t *heavy_heftyHashes[MAX_GPUS];
extern uint32_t *heavy_nonceVector[MAX_GPUS];

// globaler Speicher für unsere Ergebnisse
uint32_t *d_hash3output[MAX_GPUS];
extern uint32_t *d_hash4output[MAX_GPUS];
extern uint32_t *d_hash5output[MAX_GPUS];

// der Keccak512 State nach der ersten Runde (72 Bytes)
__constant__ uint64_t c_State[25];

// die Message (72 Bytes) für die zweite Runde auf der GPU
__constant__ uint32_t c_PaddedMessage2[18]; // 44 bytes of remaining message (Nonce at offset 4) plus padding

// ---------------------------- BEGIN CUDA keccak512 functions ------------------------------------

#define U32TO64_LE(p) \
	(((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
	*p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static __device__ void mycpy72(uint32_t *d, const uint32_t *s) {
	#pragma unroll 18
	for (int k=0; k < 18; ++k) d[k] = s[k];
}

static __device__ void mycpy32(uint32_t *d, const uint32_t *s) {
	#pragma unroll 8
	for (int k=0; k < 8; ++k) d[k] = s[k];
}

typedef struct keccak_hash_state_t {
	uint64_t state[25];    // 25*2
	uint32_t buffer[72/4]; // 72
} keccak_hash_state;

__device__ void statecopy(uint64_t *d, uint64_t *s)
{
	#pragma unroll 25
	for (int i=0; i < 25; ++i)
		d[i] = s[i];
}


static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];

__host__ __device__ void
keccak_block(uint64_t *s, const uint32_t *in, const uint64_t *keccak_round_constants) {
	size_t i;
	uint64_t t[5], u[5], v, w;

	/* absorb input */
	for (i = 0; i < 9 /* 72/8 */; i++, in += 2)
		s[i] ^= U32TO64_LE(in);

	for (i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[ 1];
		s[ 1] = ROTL64(s[ 6], 44);
		s[ 6] = ROTL64(s[ 9], 20);
		s[ 9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[ 2], 62);
		s[ 2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19],  8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[ 4], 27);
		s[ 4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21],  2);
		s[21] = ROTL64(s[ 8], 55);
		s[ 8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[ 5], 36);
		s[ 5] = ROTL64(s[ 3], 28);
		s[ 3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[ 7],  6);
		s[ 7] = ROTL64(s[10],  3);
		s[10] = ROTL64(    v,  1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
		v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= keccak_round_constants[i];
	}
}

// Die Hash-Funktion
template <int BLOCKSIZE> __global__ void keccak512_gpu_hash(uint32_t threads, uint32_t startNounce, void *outputHash, uint32_t *heftyHashes, uint32_t *nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Zähler
		//uint32_t nounce = startNounce + thread;
		uint32_t nounce = nonceVector[thread];

		// Index-Position des Hashes in den Hash Puffern bestimmen (Hefty1 und outputHash)
		uint32_t hashPosition = nounce - startNounce;

		// erstmal den State der ersten Runde holen
		uint64_t keccak_gpu_state[25];
#pragma unroll 25
		for (int i=0; i < 25; ++i)
			keccak_gpu_state[i] = c_State[i];

		// Message2 in den Puffer holen
		uint32_t msgBlock[18];
		mycpy72(msgBlock, c_PaddedMessage2);

		// die individuelle Nonce einsetzen
		msgBlock[1] = nounce;

		// den individuellen Hefty1 Hash einsetzen
		mycpy32(&msgBlock[(BLOCKSIZE-72)/sizeof(uint32_t)], &heftyHashes[8 * hashPosition]);

		// den Block einmal gut durchschütteln
		keccak_block(keccak_gpu_state, msgBlock, c_keccak_round_constants);

		// das Hash erzeugen
		uint32_t hash[16];

#pragma unroll 8
		for (size_t i = 0; i < 64; i += 8) {
			U64TO32_LE((&hash[i/4]), keccak_gpu_state[i / 8]);
		}

		// und ins Global Memory rausschreiben
#pragma unroll 16
		for(int k=0;k<16;k++)
			((uint32_t*)outputHash)[16*hashPosition+k] = hash[k];
	}
}

// ---------------------------- END CUDA keccak512 functions ------------------------------------

__host__
void keccak512_cpu_init(int thr_id, uint32_t threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL( c_keccak_round_constants),
						host_keccak_round_constants,
						sizeof(host_keccak_round_constants),
						0, hipMemcpyHostToDevice);

	// Speicher für alle Ergebnisse belegen
	hipMalloc(&d_hash3output[thr_id], (size_t) 64 * threads);
}

__host__
void keccak512_cpu_free(int thr_id)
{
	hipFree(d_hash3output[thr_id]);
}

// ----------------BEGIN keccak512 CPU version from scrypt-jane code --------------------

#define SCRYPT_HASH_DIGEST_SIZE 64
#define SCRYPT_KECCAK_F 1600
#define SCRYPT_KECCAK_C (SCRYPT_HASH_DIGEST_SIZE * 8 * 2) /* 1024 */
#define SCRYPT_KECCAK_R (SCRYPT_KECCAK_F - SCRYPT_KECCAK_C) /* 576 */
#define SCRYPT_HASH_BLOCK_SIZE (SCRYPT_KECCAK_R / 8) /* 72 */

// --------------- END keccak512 CPU version from scrypt-jane code --------------------

static int BLOCKSIZE = 84;

__host__
void keccak512_cpu_setBlock(void *data, int len)
	// data muss 80 oder 84-Byte haben!
	// heftyHash hat 32-Byte
{
	// CH
	// state init
	uint64_t keccak_cpu_state[25];
	memset(keccak_cpu_state, 0, sizeof(keccak_cpu_state));

	// erste Runde
	keccak_block((uint64_t*)&keccak_cpu_state, (const uint32_t*)data, host_keccak_round_constants);

	// state kopieren
	hipMemcpyToSymbol(HIP_SYMBOL( c_State), keccak_cpu_state, 25*sizeof(uint64_t), 0, hipMemcpyHostToDevice);

	// keccak hat 72-Byte blöcke, d.h. in unserem Fall zwei Blöcke
	// zu jeweils
	uint32_t msgBlock[18];
	memset(msgBlock, 0, 18 * sizeof(uint32_t));

	// kopiere die restlichen Daten rein (aber nur alles nach Byte 72)
	if (len == 84)
		memcpy(&msgBlock[0], &((uint8_t*)data)[72], 12);
	else if (len == 80)
		memcpy(&msgBlock[0], &((uint8_t*)data)[72], 8);

	// Nachricht abschließen
	if (len == 84)
		msgBlock[11] = 0x01;
	else if (len == 80)
		msgBlock[10] = 0x01;
	msgBlock[17] = 0x80000000;

	// Message 2 ins Constant Memory kopieren (die variable Nonce und
	// der Hefty1 Anteil muss aber auf der GPU erst noch ersetzt werden)
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage2), msgBlock, 18*sizeof(uint32_t), 0, hipMemcpyHostToDevice );

	BLOCKSIZE = len;
}

__host__
void keccak512_cpu_copyHeftyHash(int thr_id, uint32_t threads, void *heftyHashes, int copy)
{
	// Hefty1 Hashes kopieren
	if (copy)
		CUDA_SAFE_CALL(hipMemcpy(heavy_heftyHashes[thr_id], heftyHashes, 8 * sizeof(uint32_t) * threads, hipMemcpyHostToDevice));
	//else hipDeviceSynchronize();
}

__host__
void keccak512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce)
{
	const uint32_t threadsperblock = 128;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Größe des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	if (BLOCKSIZE==84)
		keccak512_gpu_hash<84><<<grid, block, shared_size>>>(threads, startNounce, d_hash3output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
	else if (BLOCKSIZE==80)
		keccak512_gpu_hash<80><<<grid, block, shared_size>>>(threads, startNounce, d_hash3output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
}
