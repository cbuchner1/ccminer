#include "hip/hip_runtime.h"
#include <stdio.h>
#include <openssl/sha.h>
#include <map>
// include thrust
#include <thrust/remove.h>
#include <thrust/device_vector.h>

#include "miner.h"

extern "C" {
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
}
#include "hefty1.h"
#include "heavy/heavy.h"
#include "cuda_helper.h"

extern uint32_t *d_hash2output[MAX_GPUS];
extern uint32_t *d_hash3output[MAX_GPUS];
extern uint32_t *d_hash4output[MAX_GPUS];
extern uint32_t *d_hash5output[MAX_GPUS];

#define HEAVYCOIN_BLKHDR_SZ 84
#define MNR_BLKHDR_SZ       80

// nonce-array für die threads
uint32_t *heavy_nonceVector[MAX_GPUS];

extern uint32_t *heavy_heftyHashes[MAX_GPUS];

/* Combines top 64-bits from each hash into a single hash */
static void combine_hashes(uint32_t *out, const uint32_t *hash1, const uint32_t *hash2, const uint32_t *hash3, const uint32_t *hash4)
{
    const uint32_t *hash[4] = { hash1, hash2, hash3, hash4 };
    int bits;
    unsigned int i;
    uint32_t mask;
    unsigned int k;

    /* Transpose first 64 bits of each hash into out */
    memset(out, 0, 32);
    bits = 0;
    for (i = 7; i >= 6; i--) {
        for (mask = 0x80000000; mask; mask >>= 1) {
            for (k = 0; k < 4; k++) {
                out[(255 - bits)/32] <<= 1;
                if ((hash[k][i] & mask) != 0)
                    out[(255 - bits)/32] |= 1;
                bits++;
            }
        }
    }
}

#ifdef _MSC_VER
#include <intrin.h>
static uint32_t __inline bitsset( uint32_t x )
{
    DWORD r = 0;
    _BitScanReverse(&r, x);
    return r;
}
#else
static uint32_t bitsset( uint32_t x )
{
    return 31-__builtin_clz(x);
}
#endif

// Finde das high bit in einem Multiword-Integer.
static int findhighbit(const uint32_t *ptarget, int words)
{
    int i;
    int highbit = 0;
    for (i=words-1; i >= 0; --i)
    {
        if (ptarget[i] != 0) {
            highbit = i*32 + bitsset(ptarget[i])+1;
            break;
        }
    }
    return highbit;
}

// Generiere ein Multiword-Integer das die Zahl
// (2 << highbit) - 1 repräsentiert.
static void genmask(uint32_t *ptarget, int words, int highbit)
{
    int i;
    for (i=words-1; i >= 0; --i)
    {
        if ((i+1)*32 <= highbit)
            ptarget[i] = UINT32_MAX;
        else if (i*32 > highbit)
            ptarget[i] = 0x00000000;
        else
            ptarget[i] = (1 << (highbit-i*32)) - 1;
    }
}

struct check_nonce_for_remove
{
    check_nonce_for_remove(uint64_t target, uint32_t *hashes, uint32_t hashlen, uint32_t startNonce) :
        m_target(target),
        m_hashes(hashes),
        m_hashlen(hashlen),
        m_startNonce(startNonce) { }

    uint64_t  m_target;
    uint32_t *m_hashes;
    uint32_t  m_hashlen;
    uint32_t  m_startNonce;

    __device__
    bool operator()(const uint32_t x)
    {
        // Position im Hash Buffer
        uint32_t hashIndex = x - m_startNonce;
        // Wert des Hashes (als uint64_t) auslesen.
        // Steht im 6. und 7. Wort des Hashes (jeder dieser Hashes hat 512 Bits)
        uint64_t hashValue = *((uint64_t*)(&m_hashes[m_hashlen*hashIndex + 6]));
        bool res = (hashValue & m_target) != hashValue;
        //printf("ndx=%x val=%08x target=%lx\n", hashIndex, hashValue, m_target);
        // gegen das Target prüfen. Es dürfen nur Bits aus dem Target gesetzt sein.
        return res;
    }
};

static bool init[MAX_GPUS] = { 0 };

__host__
int scanhash_heavy(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done, uint32_t maxvote, int blocklen)
{
    const uint32_t first_nonce = pdata[19];
    // CUDA will process thousands of threads.
    uint32_t throughput = device_intensity(thr_id, __func__, (1U << 19) - 256);
    throughput = min(throughput, max_nonce - first_nonce);

    int rc = 0;
    uint32_t *hash = NULL;
    uint32_t *cpu_nonceVector = NULL;
    CUDA_SAFE_CALL(hipHostMalloc(&hash, throughput*8*sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipHostMalloc(&cpu_nonceVector, throughput*sizeof(uint32_t)));

    int nrmCalls[6];
    memset(nrmCalls, 0, sizeof(int) * 6);

    if (opt_benchmark)
        ((uint32_t*)ptarget)[7] = 0x00ff;

    // für jeden Hash ein individuelles Target erstellen basierend
    // auf dem höchsten Bit, das in ptarget gesetzt ist.
    int highbit = findhighbit(ptarget, 8);
    uint32_t target2[2], target3[2], target4[2], target5[2];
    genmask(target2, 2, highbit/4+(((highbit%4)>3)?1:0) ); // SHA256
    genmask(target3, 2, highbit/4+(((highbit%4)>2)?1:0) ); // keccak512
    genmask(target4, 2, highbit/4+(((highbit%4)>1)?1:0) ); // groestl512
    genmask(target5, 2, highbit/4+(((highbit%4)>0)?1:0) ); // blake512

    if (!init[thr_id])
    {
        hefty_cpu_init(thr_id, throughput);
        sha256_cpu_init(thr_id, throughput);
        keccak512_cpu_init(thr_id, throughput);
        groestl512_cpu_init(thr_id, throughput);
        blake512_cpu_init(thr_id, throughput);
        combine_cpu_init(thr_id, throughput);

        CUDA_SAFE_CALL(hipMalloc(&heavy_nonceVector[thr_id], sizeof(uint32_t) * throughput));

        init[thr_id] = true;
    }

    if (blocklen == HEAVYCOIN_BLKHDR_SZ)
    {
        uint16_t *ext = (uint16_t *)&pdata[20];

        if (opt_vote > maxvote) {
            applog(LOG_WARNING, "Your block reward vote (%hu) exceeds "
                    "the maxvote reported by the pool (%hu).",
                    opt_vote, maxvote);
        }

        if (opt_trust_pool && opt_vote > maxvote) {
            applog(LOG_WARNING, "Capping block reward vote to maxvote reported by pool.");
            ext[0] = maxvote;
        }
        else
            ext[0] = opt_vote;
    }

    // Setze die Blockdaten
    hefty_cpu_setBlock(thr_id, throughput, pdata, blocklen);
    sha256_cpu_setBlock(pdata, blocklen);
    keccak512_cpu_setBlock(pdata, blocklen);
    groestl512_cpu_setBlock(pdata, blocklen);
    blake512_cpu_setBlock(pdata, blocklen);

    do {

        ////// Compaction init
        thrust::device_ptr<uint32_t> devNoncePtr(heavy_nonceVector[thr_id]);
        thrust::device_ptr<uint32_t> devNoncePtrEnd((heavy_nonceVector[thr_id]) + throughput);
        uint32_t actualNumberOfValuesInNonceVectorGPU = throughput;
        uint64_t *t;

        hefty_cpu_hash(thr_id, throughput, pdata[19]);
        //hipDeviceSynchronize();
        sha256_cpu_hash(thr_id, throughput, pdata[19]);
        //hipDeviceSynchronize();

        // Hier ist die längste CPU Wartephase. Deshalb ein strategisches MyStreamSynchronize() hier.
        MyStreamSynchronize(NULL, 1, thr_id);

        ////// Compaction
        t = (uint64_t*) target2;
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash2output[thr_id], 8, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        keccak512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        t = (uint64_t*) target3;
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash3output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        blake512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        t = (uint64_t*) target5;
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash5output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        groestl512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        t = (uint64_t*) target4;
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash4output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        // combine
        combine_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19], hash);

        if (opt_tracegpu) {
            applog(LOG_BLUE, "heavy GPU hash:");
            applog_hash((uchar*)hash);
        }

        // Ergebnisse kopieren
        if(actualNumberOfValuesInNonceVectorGPU > 0)
        {
            size_t size = sizeof(uint32_t) * actualNumberOfValuesInNonceVectorGPU;
            CUDA_SAFE_CALL(hipMemcpy(cpu_nonceVector, heavy_nonceVector[thr_id], size, hipMemcpyDeviceToHost));
            hipDeviceSynchronize();

            for (uint32_t i=0; i < actualNumberOfValuesInNonceVectorGPU; i++)
            {
                uint32_t nonce = cpu_nonceVector[i];
                uint32_t *foundhash = &hash[8*i];
                if (foundhash[7] <= ptarget[7]) {
                    if (fulltest(foundhash, ptarget)) {
                        uint32_t verification[8];
                        pdata[19] += nonce - pdata[19];
                        heavycoin_hash((uchar*)verification, (uchar*)pdata, blocklen);
                        if (memcmp(verification, foundhash, 8*sizeof(uint32_t))) {
                            applog(LOG_ERR, "hash for nonce=$%08X does not validate on CPU!\n", nonce);
                        } else {
                            *hashes_done = pdata[19] - first_nonce;
                            rc = 1;
                            goto exit;
                        }
                    }
                }
            }
        }

emptyNonceVector:

        pdata[19] += throughput;

    } while (pdata[19] < max_nonce && !work_restart[thr_id].restart);
    *hashes_done = pdata[19] - first_nonce;

exit:
    hipHostFree(cpu_nonceVector);
    hipHostFree(hash);
    return rc;
}

__host__
void heavycoin_hash(uchar* output, const uchar* input, int len)
{
    unsigned char hash1[32];
    unsigned char hash2[32];
    uint32_t hash3[16];
    uint32_t hash4[16];
    uint32_t hash5[16];
    uint32_t *final;
    SHA256_CTX ctx;
    sph_keccak512_context keccakCtx;
    sph_groestl512_context groestlCtx;
    sph_blake512_context blakeCtx;

    HEFTY1(input, len, hash1);

    /* HEFTY1 is new, so take an extra security measure to eliminate
     * the possiblity of collisions:
     *
     *     Hash(x) = SHA256(x + HEFTY1(x))
     *
     * N.B. '+' is concatenation.
     */
    SHA256_Init(&ctx);
    SHA256_Update(&ctx, input, len);
    SHA256_Update(&ctx, hash1, sizeof(hash1));
    SHA256_Final(hash2, &ctx);

    /* Additional security: Do not rely on a single cryptographic hash
     * function.  Instead, combine the outputs of 4 of the most secure
     * cryptographic hash functions-- SHA256, KECCAK512, GROESTL512
     * and BLAKE512.
     */

    sph_keccak512_init(&keccakCtx);
    sph_keccak512(&keccakCtx, input, len);
    sph_keccak512(&keccakCtx, hash1, sizeof(hash1));
    sph_keccak512_close(&keccakCtx, (void *)&hash3);

    sph_groestl512_init(&groestlCtx);
    sph_groestl512(&groestlCtx, input, len);
    sph_groestl512(&groestlCtx, hash1, sizeof(hash1));
    sph_groestl512_close(&groestlCtx, (void *)&hash4);

    sph_blake512_init(&blakeCtx);
    sph_blake512(&blakeCtx, input, len);
    sph_blake512(&blakeCtx, (unsigned char *)&hash1, sizeof(hash1));
    sph_blake512_close(&blakeCtx, (void *)&hash5);

    final = (uint32_t *)output;
    combine_hashes(final, (uint32_t *)hash2, hash3, hash4, hash5);
}
