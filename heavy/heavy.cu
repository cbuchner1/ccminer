#include "hip/hip_runtime.h"
#include <string.h>
#include <openssl/sha.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <map>

#ifndef _WIN32
#include <unistd.h>
#endif

// include thrust
#include <thrust/version.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>

#include "miner.h"

#include "hefty1.h"
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"

#include "heavy/cuda_hefty1.h"
#include "heavy/cuda_sha256.h"
#include "heavy/cuda_keccak512.h"
#include "heavy/cuda_groestl512.h"
#include "heavy/cuda_blake512.h"
#include "heavy/cuda_combine.h"

extern uint32_t *d_hash2output[8];
extern uint32_t *d_hash3output[8];
extern uint32_t *d_hash4output[8];
extern uint32_t *d_hash5output[8];

#define HEAVYCOIN_BLKHDR_SZ        84
#define MNR_BLKHDR_SZ		       80

// nonce-array f�r die threads
uint32_t *d_nonceVector[8];

/* Combines top 64-bits from each hash into a single hash */
static void combine_hashes(uint32_t *out, const uint32_t *hash1, const uint32_t *hash2, const uint32_t *hash3, const uint32_t *hash4)
{
    const uint32_t *hash[4] = { hash1, hash2, hash3, hash4 };
    int bits;
    unsigned int i;
    uint32_t mask;
    unsigned int k;

    /* Transpose first 64 bits of each hash into out */
    memset(out, 0, 32);
    bits = 0;
    for (i = 7; i >= 6; i--) {
        for (mask = 0x80000000; mask; mask >>= 1) {
            for (k = 0; k < 4; k++) {
                out[(255 - bits)/32] <<= 1;
                if ((hash[k][i] & mask) != 0)
                    out[(255 - bits)/32] |= 1;
                bits++;
            }
        }
    }
}

#ifdef _MSC_VER
#include <intrin.h>
static uint32_t __inline bitsset( uint32_t x )
{
   DWORD r = 0;
   _BitScanReverse(&r, x);
   return r;
}
#else
static uint32_t bitsset( uint32_t x )
{
    return 31-__builtin_clz(x);
}
#endif

// Finde das high bit in einem Multiword-Integer.
static int findhighbit(const uint32_t *ptarget, int words)
{
    int i;
    int highbit = 0;
    for (i=words-1; i >= 0; --i)
    {
        if (ptarget[i] != 0) {
            highbit = i*32 + bitsset(ptarget[i])+1;
                break;
        }
    }
    return highbit;
}

// Generiere ein Multiword-Integer das die Zahl
// (2 << highbit) - 1 repr�sentiert.
static void genmask(uint32_t *ptarget, int words, int highbit)
{
    int i;
    for (i=words-1; i >= 0; --i)
    {
        if ((i+1)*32 <= highbit)
            ptarget[i] = 0xffffffff;
        else if (i*32 > highbit)
            ptarget[i] = 0x00000000;
        else
            ptarget[i] = (1 << (highbit-i*32)) - 1;
    }
}

struct check_nonce_for_remove
{    
    check_nonce_for_remove(uint64_t target, uint32_t *hashes, uint32_t hashlen, uint32_t startNonce) :
        m_target(target),
        m_hashes(hashes),
        m_hashlen(hashlen),
        m_startNonce(startNonce) { }

    __device__
    bool operator()(const uint32_t x)
    {
        // Position im Hash Buffer
        uint32_t hashIndex = x - m_startNonce;
        // Wert des Hashes (als uint64_t) auslesen.
        // Steht im 6. und 7. Wort des Hashes (jeder dieser Hashes hat 512 Bits)
        uint64_t hashValue = *((uint64_t*)(&m_hashes[m_hashlen*hashIndex + 6]));
        // gegen das Target pr�fen. Es d�rfen nur Bits aus dem Target gesetzt sein.
        return (hashValue & m_target) != hashValue;
    }

    uint64_t  m_target;
    uint32_t *m_hashes;
    uint32_t  m_hashlen;
    uint32_t  m_startNonce;
};

// Zahl der CUDA Devices im System bestimmen
extern "C" int cuda_num_devices()
{
    int version;
    hipError_t err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < 5 || (maj == 5 && min < 5))
    {
        applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", 5, 5);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

// Ger�tenamen holen
extern char *device_name[8];
extern int device_map[8];
int device_major[8]; 
int device_minor[8];
int compute_version[8];

extern "C" void cuda_devicenames()
{
    hipError_t err;
    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }

    for (int i=0; i < GPU_N; i++)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device_map[i]);

        device_name[i] = strdup(props.name);
		device_major[i] = props.major; 
		device_minor[i] = props.minor;
		compute_version[i]= props.major*10+props.minor;
    }
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
    int hlen = strlen(haystack);
    int nlen = strlen(needle);
    for (int i=0; i < hlen; ++i)
    {
        if (haystack[i] == ' ') continue;
        int j=0, x = 0;
        while(j < nlen)
        {
            if (haystack[i+x] == ' ') {++x; continue;}
            if (needle[j] == ' ') {++j; continue;}
            if (needle[j] == '#') return ++match == needle[j+1]-'0';
            if (tolower(haystack[i+x]) != tolower(needle[j])) break;
            ++j; ++x;
        }
        if (j == nlen) return true;
    }
    return false;
}

// CUDA Ger�t nach Namen finden (gibt Ger�te-Index zur�ck oder -1)
extern "C" int cuda_finddevice(char *name)
{
    int num = cuda_num_devices();
    int match = 0;
    for (int i=0; i < num; ++i)
    {
        hipDeviceProp_t props;
        if (hipGetDeviceProperties(&props, i) == hipSuccess)
            if (substringsearch(props.name, name, match)) return i;
    }
    return -1;
}

// Zeitsynchronisations-Routine von cudaminer mit CPU sleep
typedef struct { double value[8]; } tsumarray;
hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
    hipError_t result = hipSuccess;
    if (situation >= 0)
    {   
        static std::map<int, tsumarray> tsum;

        double a = 0.95, b = 0.05;
        if (tsum.find(situation) == tsum.end()) { a = 0.5; b = 0.5; } // faster initial convergence

        double tsync = 0.0;
        double tsleep = 0.95 * tsum[situation].value[thr_id];
        if (hipStreamQuery(stream) == hipErrorNotReady)
        {
            usleep((useconds_t)(1e6*tsleep));
            struct timeval tv_start, tv_end;
            gettimeofday(&tv_start, NULL);
            result = hipStreamSynchronize(stream);
            gettimeofday(&tv_end, NULL);
            tsync = 1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec);
        }
        if (tsync >= 0) tsum[situation].value[thr_id] = a * tsum[situation].value[thr_id] + b * (tsleep+tsync);
    }
    else
        result = hipStreamSynchronize(stream);
    return result;
}

int scanhash_heavy_cpp(int thr_id, uint32_t *pdata,
 const uint32_t *ptarget, uint32_t max_nonce,
 unsigned long *hashes_done, uint32_t maxvote, int blocklen);

extern "C"
int scanhash_heavy(int thr_id, uint32_t *pdata,
 const uint32_t *ptarget, uint32_t max_nonce,
 unsigned long *hashes_done, uint32_t maxvote, int blocklen)
{
 return scanhash_heavy_cpp(thr_id, pdata,
  ptarget, max_nonce, hashes_done, maxvote, blocklen);
}

extern bool opt_benchmark;

int scanhash_heavy_cpp(int thr_id, uint32_t *pdata,
 const uint32_t *ptarget, uint32_t max_nonce,
 unsigned long *hashes_done, uint32_t maxvote, int blocklen)
{
    // CUDA will process thousands of threads.
    const int throughput = 4096 * 128;

    if (opt_benchmark)
        ((uint32_t*)ptarget)[7] = 0x000000ff;

    int rc = 0;
    uint32_t *hash = NULL;
    hipHostMalloc(&hash, throughput*8*sizeof(uint32_t));
    uint32_t *cpu_nonceVector = NULL;
    hipHostMalloc(&cpu_nonceVector, throughput*sizeof(uint32_t));

    int nrmCalls[6];
    memset(nrmCalls, 0, sizeof(int) * 6);

    uint32_t start_nonce = pdata[19];    

    // f�r jeden Hash ein individuelles Target erstellen basierend
    // auf dem h�chsten Bit, das in ptarget gesetzt ist.
    int highbit = findhighbit(ptarget, 8);
    uint32_t target2[2], target3[2], target4[2], target5[2];
    genmask(target2, 2, highbit/4+(((highbit%4)>3)?1:0) ); // SHA256
    genmask(target3, 2, highbit/4+(((highbit%4)>2)?1:0) ); // keccak512
    genmask(target4, 2, highbit/4+(((highbit%4)>1)?1:0) ); // groestl512
    genmask(target5, 2, highbit/4+(((highbit%4)>0)?1:0) ); // blake512

    static bool init[8] = {0,0,0,0,0,0,0,0};
    if (!init[thr_id])
    {
        hefty_cpu_init(thr_id, throughput);
        sha256_cpu_init(thr_id, throughput);
        keccak512_cpu_init(thr_id, throughput);
        groestl512_cpu_init(thr_id, throughput);
        blake512_cpu_init(thr_id, throughput);
        combine_cpu_init(thr_id, throughput);
        init[thr_id] = true;
        hipMalloc(&d_nonceVector[thr_id], sizeof(uint32_t) * throughput);
    }

    if (blocklen == HEAVYCOIN_BLKHDR_SZ)
    {
        uint16_t *ext = (uint16_t *)&pdata[20];

        if (opt_vote > maxvote) {
            printf("Warning: Your block reward vote (%hu) exceeds "
                    "the maxvote reported by the pool (%hu).\n",
                    opt_vote, maxvote);
        }

        if (opt_trust_pool && opt_vote > maxvote) {
            printf("Warning: Capping block reward vote to maxvote reported by pool.\n");
            ext[0] = maxvote;
        }
        else
            ext[0] = opt_vote;
    }

    // Setze die Blockdaten
    hefty_cpu_setBlock(thr_id, throughput, pdata, blocklen);
    sha256_cpu_setBlock(pdata, blocklen);
    keccak512_cpu_setBlock(pdata, blocklen);
    groestl512_cpu_setBlock(pdata, blocklen);
    blake512_cpu_setBlock(pdata, blocklen);

    do {
        int i;

        ////// Compaction init
        thrust::device_ptr<uint32_t> devNoncePtr(d_nonceVector[thr_id]);
        thrust::device_ptr<uint32_t> devNoncePtrEnd((d_nonceVector[thr_id]) + throughput);
        uint32_t actualNumberOfValuesInNonceVectorGPU = throughput;

        hefty_cpu_hash(thr_id, throughput, pdata[19]);
        //hipDeviceSynchronize();
        sha256_cpu_hash(thr_id, throughput, pdata[19]);
        //hipDeviceSynchronize();

        // Hier ist die l�ngste CPU Wartephase. Deshalb ein strategisches MyStreamSynchronize() hier.
        MyStreamSynchronize(NULL, 1, thr_id);

        ////// Compaction
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*((uint64_t*)target2), d_hash2output[thr_id], 8, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;
        
        keccak512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*((uint64_t*)target3), d_hash3output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        blake512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*((uint64_t*)target5), d_hash5output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;

        groestl512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);
        //hipDeviceSynchronize();

        ////// Compaction
        devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*((uint64_t*)target4), d_hash4output[thr_id], 16, pdata[19]));
        actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
        if(actualNumberOfValuesInNonceVectorGPU == 0)
            goto emptyNonceVector;
        
        // combine
        combine_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19], hash);

        // Ergebnisse kopieren
        if(actualNumberOfValuesInNonceVectorGPU > 0)
        {
            hipMemcpy(cpu_nonceVector, d_nonceVector[thr_id], sizeof(uint32_t) * actualNumberOfValuesInNonceVectorGPU, hipMemcpyDeviceToHost);
        
            for (i=0; i<actualNumberOfValuesInNonceVectorGPU;++i)
            {
                uint32_t nonce = cpu_nonceVector[i];
                //uint32_t index = nonce - pdata[19];
                uint32_t index = i;
                uint32_t *foundhash = &hash[8*index];
                if (foundhash[7] <= ptarget[7]) {
                    if (fulltest(foundhash, ptarget)) {
                        uint32_t verification[8];
                        pdata[19] += nonce - pdata[19];
                        heavycoin_hash((unsigned char *)verification, (const unsigned char *)pdata, blocklen);
                        if (memcmp(verification, foundhash, 8*sizeof(uint32_t))) {
                            applog(LOG_ERR, "hash for nonce=$%08X does not validate on CPU!\n", nonce);
                        }
                        else
                        {
                            *hashes_done = pdata[19] - start_nonce;
                            rc = 1;
                            goto exit;
                        }
                    }
                }
            }
        }

emptyNonceVector:

        pdata[19] += throughput;

    } while (pdata[19] < max_nonce && !work_restart[thr_id].restart);
    *hashes_done = pdata[19] - start_nonce;

exit:
    hipHostFree(cpu_nonceVector);
    hipHostFree(hash);
    return rc;
}

void heavycoin_hash(unsigned char* output, const unsigned char* input, int len)
{
    unsigned char hash1[32];
    unsigned char hash2[32];
    uint32_t hash3[16];
    uint32_t hash4[16];
    uint32_t hash5[16];
    uint32_t *final;
    SHA256_CTX ctx;
    sph_keccak512_context keccakCtx;
    sph_groestl512_context groestlCtx;
    sph_blake512_context blakeCtx;

    HEFTY1(input, len, hash1);

    /* HEFTY1 is new, so take an extra security measure to eliminate
     * the possiblity of collisions:
     *
     *     Hash(x) = SHA256(x + HEFTY1(x))
     *
     * N.B. '+' is concatenation.
     */
    SHA256_Init(&ctx);
    SHA256_Update(&ctx, input, len);
    SHA256_Update(&ctx, hash1, sizeof(hash1));
    SHA256_Final(hash2, &ctx);

    /* Additional security: Do not rely on a single cryptographic hash
     * function.  Instead, combine the outputs of 4 of the most secure
     * cryptographic hash functions-- SHA256, KECCAK512, GROESTL512
     * and BLAKE512.
     */

    sph_keccak512_init(&keccakCtx);
    sph_keccak512(&keccakCtx, input, len);
    sph_keccak512(&keccakCtx, hash1, sizeof(hash1));
    sph_keccak512_close(&keccakCtx, (void *)&hash3);

    sph_groestl512_init(&groestlCtx);
    sph_groestl512(&groestlCtx, input, len);
    sph_groestl512(&groestlCtx, hash1, sizeof(hash1));
    sph_groestl512_close(&groestlCtx, (void *)&hash4);

    sph_blake512_init(&blakeCtx);
    sph_blake512(&blakeCtx, input, len);
    sph_blake512(&blakeCtx, (unsigned char *)&hash1, sizeof(hash1));
    sph_blake512_close(&blakeCtx, (void *)&hash5);

    final = (uint32_t *)output;
    combine_hashes(final, (uint32_t *)hash2, hash3, hash4, hash5);
}
