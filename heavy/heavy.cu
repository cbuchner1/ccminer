#include "hip/hip_runtime.h"
#include <stdio.h>
#include <openssl/sha.h>
#include <hip/hip_runtime.h>
#include <map>

#ifndef WITH_HEAVY_ALGO
#include <unistd.h>
#include "miner.h"
// nonce array also used in other algos
uint32_t *heavy_nonceVector[MAX_GPUS];
int scanhash_heavy(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done, uint32_t maxvote, int blocklen)
{
	applog(LOG_ERR, "heavy algo not included in this build!");
	sleep(3);
	return -1;
}
void free_heavy(int thr_id) {}

#else

// include thrust if possible
#if defined(__GNUC__) && __GNUC__ == 5 && __GNUC_MINOR__ >= 2 && CUDA_VERSION < 7000
#warning "Heavy: incompatible GCC version!"
#define USE_THRUST 0
#else
#define USE_THRUST 1
#endif

#if USE_THRUST
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#endif

#include "miner.h"
#include "cuda_helper.h"

// nonce array also used in other algos
uint32_t *heavy_nonceVector[MAX_GPUS];

extern uint32_t *d_hash2output[MAX_GPUS];
extern uint32_t *d_hash3output[MAX_GPUS];
extern uint32_t *d_hash4output[MAX_GPUS];
extern uint32_t *d_hash5output[MAX_GPUS];

#define HEAVYCOIN_BLKHDR_SZ 84
#define MNR_BLKHDR_SZ       80

extern uint32_t *heavy_heftyHashes[MAX_GPUS];

#ifdef _MSC_VER
#include <intrin.h>
static uint32_t __inline bitsset( uint32_t x )
{
	DWORD r = 0;
	_BitScanReverse(&r, x);
	return r;
}
#else
static uint32_t bitsset( uint32_t x )
{
	return 31-__builtin_clz(x);
}
#endif

// Finde das high bit in einem Multiword-Integer.
static int findhighbit(const uint32_t *ptarget, int words)
{
	int i;
	int highbit = 0;
	for (i=words-1; i >= 0; --i)
	{
		if (ptarget[i] != 0) {
			highbit = i*32 + bitsset(ptarget[i])+1;
			break;
		}
	}
	return highbit;
}

// Generiere ein Multiword-Integer das die Zahl
// (2 << highbit) - 1 repräsentiert.
static void genmask(uint32_t *ptarget, int words, int highbit)
{
	int i;
	for (i=words-1; i >= 0; --i)
	{
		if ((i+1)*32 <= highbit)
			ptarget[i] = UINT32_MAX;
		else if (i*32 > highbit)
			ptarget[i] = 0x00000000;
		else
			ptarget[i] = (1 << (highbit-i*32)) - 1;
	}
}

struct check_nonce_for_remove
{
	check_nonce_for_remove(uint64_t target, uint32_t *hashes, uint32_t hashlen, uint32_t startNonce) :
		m_target(target),
		m_hashes(hashes),
		m_hashlen(hashlen),
		m_startNonce(startNonce) { }

	uint64_t  m_target;
	uint32_t *m_hashes;
	uint32_t  m_hashlen;
	uint32_t  m_startNonce;

	__device__
	bool operator()(const uint32_t x)
	{
		// Position im Hash Buffer
		uint32_t hashIndex = x - m_startNonce;
		// Wert des Hashes (als uint64_t) auslesen.
		// Steht im 6. und 7. Wort des Hashes (jeder dieser Hashes hat 512 Bits)
		uint64_t hashValue = *((uint64_t*)(&m_hashes[m_hashlen*hashIndex + 6]));
		bool res = (hashValue & m_target) != hashValue;
		//printf("ndx=%x val=%08x target=%lx\n", hashIndex, hashValue, m_target);
		// gegen das Target prüfen. Es dürfen nur Bits aus dem Target gesetzt sein.
		return res;
	}
};

static bool init[MAX_GPUS] = { 0 };

__host__
int scanhash_heavy(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done, uint32_t maxvote, int blocklen)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	// CUDA will process thousands of threads.
	uint32_t throughput = cuda_default_throughput(thr_id, (1U << 19) - 256);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;
	uint32_t *hash = NULL;
	uint32_t *cpu_nonceVector = NULL;

	int nrmCalls[6];
	memset(nrmCalls, 0, sizeof(int) * 6);

	if (opt_benchmark)
	   ptarget[7] = 0x000f;

	// für jeden Hash ein individuelles Target erstellen basierend
	// auf dem höchsten Bit, das in ptarget gesetzt ist.
	int highbit = findhighbit(ptarget, 8);
	uint32_t target2[2], target3[2], target4[2], target5[2];
	genmask(target2, 2, highbit/4+(((highbit%4)>3)?1:0) ); // SHA256
	genmask(target3, 2, highbit/4+(((highbit%4)>2)?1:0) ); // keccak512
	genmask(target4, 2, highbit/4+(((highbit%4)>1)?1:0) ); // groestl512
	genmask(target5, 2, highbit/4+(((highbit%4)>0)?1:0) ); // blake512

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		hefty_cpu_init(thr_id, throughput);
		sha256_cpu_init(thr_id, throughput);
		keccak512_cpu_init(thr_id, throughput);
		groestl512_cpu_init(thr_id, throughput);
		blake512_cpu_init(thr_id, throughput);
		combine_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&heavy_nonceVector[thr_id], sizeof(uint32_t) * throughput));

		init[thr_id] = true;
	}

	// weird but require at least one hipSetDevice first
	CUDA_SAFE_CALL(hipHostMalloc(&hash, (size_t) 32 * throughput));
	CUDA_SAFE_CALL(hipHostMalloc(&cpu_nonceVector, sizeof(uint32_t) * throughput));

	if (blocklen == HEAVYCOIN_BLKHDR_SZ)
	{
		uint16_t *ext = (uint16_t*) &pdata[20];

		if (opt_vote > maxvote && !opt_benchmark) {
			applog(LOG_WARNING, "Your block reward vote (%hu) exceeds the maxvote reported by the pool (%hu).",
					opt_vote, maxvote);
		}

		if (opt_trust_pool && opt_vote > maxvote) {
			applog(LOG_WARNING, "Capping block reward vote to maxvote reported by pool.");
			ext[0] = maxvote;
		}
		else
			ext[0] = opt_vote;
	}

	// Setze die Blockdaten
	hefty_cpu_setBlock(thr_id, throughput, pdata, blocklen);
	sha256_cpu_setBlock(pdata, blocklen);
	keccak512_cpu_setBlock(pdata, blocklen);
	groestl512_cpu_setBlock(pdata, blocklen);
	blake512_cpu_setBlock(pdata, blocklen);

	do {
		uint32_t actualNumberOfValuesInNonceVectorGPU = throughput;

		////// Compaction init

		hefty_cpu_hash(thr_id, throughput, pdata[19]);
		sha256_cpu_hash(thr_id, throughput, pdata[19]);

		// Hier ist die längste CPU Wartephase. Deshalb ein strategisches MyStreamSynchronize() hier.
		MyStreamSynchronize(NULL, 1, thr_id);

#if USE_THRUST
		thrust::device_ptr<uint32_t> devNoncePtr(heavy_nonceVector[thr_id]);
		thrust::device_ptr<uint32_t> devNoncePtrEnd((heavy_nonceVector[thr_id]) + throughput);

		////// Compaction
		uint64_t *t = (uint64_t*) target2;
		devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash2output[thr_id], 8, pdata[19]));
		actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
		if(actualNumberOfValuesInNonceVectorGPU == 0)
			goto emptyNonceVector;

		keccak512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);

		////// Compaction
		t = (uint64_t*) target3;
		devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash3output[thr_id], 16, pdata[19]));
		actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
		if(actualNumberOfValuesInNonceVectorGPU == 0)
			goto emptyNonceVector;

		blake512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);

		////// Compaction
		t = (uint64_t*) target5;
		devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash5output[thr_id], 16, pdata[19]));
		actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
		if(actualNumberOfValuesInNonceVectorGPU == 0)
			goto emptyNonceVector;

		groestl512_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19]);

		////// Compaction
		t = (uint64_t*) target4;
		devNoncePtrEnd = thrust::remove_if(devNoncePtr, devNoncePtrEnd, check_nonce_for_remove(*t, d_hash4output[thr_id], 16, pdata[19]));
		actualNumberOfValuesInNonceVectorGPU = (uint32_t)(devNoncePtrEnd - devNoncePtr);
#else
		// todo (nvlabs cub ?)
		actualNumberOfValuesInNonceVectorGPU = 0;
#endif
		if(actualNumberOfValuesInNonceVectorGPU == 0)
			goto emptyNonceVector;

		// combine
		combine_cpu_hash(thr_id, actualNumberOfValuesInNonceVectorGPU, pdata[19], hash);

		if (opt_tracegpu) {
			applog(LOG_BLUE, "heavy GPU hash:");
			applog_hash((uchar*)hash);
		}

		// Ergebnisse kopieren
		if(actualNumberOfValuesInNonceVectorGPU > 0)
		{
			size_t size = sizeof(uint32_t) * actualNumberOfValuesInNonceVectorGPU;
			hipMemcpy(cpu_nonceVector, heavy_nonceVector[thr_id], size, hipMemcpyDeviceToHost);

			for (uint32_t i=0; i < actualNumberOfValuesInNonceVectorGPU; i++)
			{
				uint32_t nonce = cpu_nonceVector[i];
				uint32_t *foundhash = &hash[8*i];
				if (foundhash[7] <= ptarget[7] && fulltest(foundhash, ptarget)) {
					uint32_t vhash[8];
					pdata[19] += nonce - pdata[19];
					heavycoin_hash((uchar*)vhash, (uchar*)pdata, blocklen);
					if (memcmp(vhash, foundhash, 32)) {
						gpu_increment_reject(thr_id);
						if (!opt_quiet)
							gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", nonce);
					} else {
						work_set_target_ratio(work, vhash);
						rc = 1;
						goto exit;
					}
				}
			}
		}

emptyNonceVector:
		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

exit:
	*hashes_done = pdata[19] - first_nonce;

	hipHostFree(cpu_nonceVector);
	hipHostFree(hash);
	CUDA_LOG_ERROR();

	return rc;
}

// cleanup
extern "C" void free_heavy(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(heavy_nonceVector[thr_id]);

	blake512_cpu_free(thr_id);
	groestl512_cpu_free(thr_id);
	hefty_cpu_free(thr_id);
	keccak512_cpu_free(thr_id);
	sha256_cpu_free(thr_id);
	combine_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

#endif

extern "C" {
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
}
#include "hefty1.h"
#include "heavy/heavy.h"

/* Combines top 64-bits from each hash into a single hash */
__host__
static void combine_hashes(uint32_t *out, const uint32_t *hash1, const uint32_t *hash2, const uint32_t *hash3, const uint32_t *hash4)
{
	const uint32_t *hash[4] = { hash1, hash2, hash3, hash4 };
	int bits;
	unsigned int i;
	uint32_t mask;
	unsigned int k;

	/* Transpose first 64 bits of each hash into out */
	memset(out, 0, 32);
	bits = 0;
	for (i = 7; i >= 6; i--) {
		for (mask = 0x80000000; mask; mask >>= 1) {
			for (k = 0; k < 4; k++) {
				out[(255 - bits) / 32] <<= 1;
				if ((hash[k][i] & mask) != 0)
					out[(255 - bits) / 32] |= 1;
				bits++;
			}
		}
	}
}

// CPU hash function
__host__
void heavycoin_hash(uchar* output, const uchar* input, int len)
{
	unsigned char hash1[32];
	unsigned char hash2[32];
	uint32_t hash3[16];
	uint32_t hash4[16];
	uint32_t hash5[16];
	uint32_t *final;
	SHA256_CTX ctx;
	sph_keccak512_context keccakCtx;
	sph_groestl512_context groestlCtx;
	sph_blake512_context blakeCtx;

	HEFTY1(input, len, hash1);

	/* HEFTY1 is new, so take an extra security measure to eliminate
	 * the possiblity of collisions:
	 *
	 *     Hash(x) = SHA256(x + HEFTY1(x))
	 *
	 * N.B. '+' is concatenation.
	 */
	SHA256_Init(&ctx);
	SHA256_Update(&ctx, input, len);
	SHA256_Update(&ctx, hash1, sizeof(hash1));
	SHA256_Final(hash2, &ctx);

	/* Additional security: Do not rely on a single cryptographic hash
	 * function.  Instead, combine the outputs of 4 of the most secure
	 * cryptographic hash functions-- SHA256, KECCAK512, GROESTL512
	 * and BLAKE512.
	 */

	sph_keccak512_init(&keccakCtx);
	sph_keccak512(&keccakCtx, input, len);
	sph_keccak512(&keccakCtx, hash1, sizeof(hash1));
	sph_keccak512_close(&keccakCtx, (void *)&hash3);

	sph_groestl512_init(&groestlCtx);
	sph_groestl512(&groestlCtx, input, len);
	sph_groestl512(&groestlCtx, hash1, sizeof(hash1));
	sph_groestl512_close(&groestlCtx, (void *)&hash4);

	sph_blake512_init(&blakeCtx);
	sph_blake512(&blakeCtx, input, len);
	sph_blake512(&blakeCtx, (unsigned char *)&hash1, sizeof(hash1));
	sph_blake512_close(&blakeCtx, (void *)&hash5);

	final = (uint32_t *)output;
	combine_hashes(final, (uint32_t *)hash2, hash3, hash4, hash5);
}
