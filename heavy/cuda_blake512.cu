#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// globaler Speicher für alle HeftyHashes aller Threads
extern uint32_t *heavy_heftyHashes[MAX_GPUS];
extern uint32_t *heavy_nonceVector[MAX_GPUS];

// globaler Speicher für unsere Ergebnisse
uint32_t *d_hash5output[MAX_GPUS];

// die Message (112 bzw. 116 Bytes) mit Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage[16]; // padded message (80/84+32 bytes + padding)


// ---------------------------- BEGIN CUDA blake512 functions ------------------------------------

__constant__ uint8_t c_sigma[16][16];

const uint8_t host_sigma[16][16] =
{
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

/* in cuda_helper */
#define SWAP32(x) cuda_swab32(x)
#define SWAP64(x) cuda_swab64(x)

__constant__ uint64_t c_SecondRound[15];

const uint64_t host_SecondRound[15] =
{
  0,0,0,0,0,0,0,0,0,0,0,0,0,SWAP64(1),0
};

__constant__ uint64_t c_u512[16];

const uint64_t host_u512[16] =
{
  0x243f6a8885a308d3ULL, 0x13198a2e03707344ULL,
  0xa4093822299f31d0ULL, 0x082efa98ec4e6c89ULL,
  0x452821e638d01377ULL, 0xbe5466cf34e90c6cULL,
  0xc0ac29b7c97c50ddULL, 0x3f84d5b5b5470917ULL,
  0x9216d5d98979fb1bULL, 0xd1310ba698dfb5acULL,
  0x2ffd72dbd01adfb7ULL, 0xb8e1afed6a267e96ULL,
  0xba7c9045f12c7f99ULL, 0x24a19947b3916cf7ULL,
  0x0801f2e2858efc16ULL, 0x636920d871574e69ULL
};


#define G(a,b,c,d,e)          \
    v[a] += (m[sigma[i][e]] ^ u512[sigma[i][e+1]]) + v[b];\
    v[d] = SWAPDWORDS( v[d] ^ v[a]);        \
    v[c] += v[d];           \
    v[b] = ROTR64( v[b] ^ v[c],25);        \
    v[a] += (m[sigma[i][e+1]] ^ u512[sigma[i][e]])+v[b];  \
    v[d] = ROTR64( v[d] ^ v[a],16);        \
    v[c] += v[d];           \
    v[b] = ROTR64( v[b] ^ v[c],11);

template <int BLOCKSIZE> __device__ void blake512_compress( uint64_t *h, const uint64_t *block, int nullt, const uint8_t ((*sigma)[16]), const uint64_t *u512 )
{
    uint64_t v[16], m[16], i;

#pragma unroll 16
    for( i = 0; i < 16; ++i )  m[i] = cuda_swab64(block[i]);

#pragma unroll 8
    for( i = 0; i < 8; ++i )  v[i] = h[i];

    v[ 8] = u512[0];
    v[ 9] = u512[1];
    v[10] = u512[2];
    v[11] = u512[3];
    v[12] = u512[4];
    v[13] = u512[5];
    v[14] = u512[6];
    v[15] = u512[7];

    /* don't xor t when the block is only padding */
    if ( !nullt ) {
        v[12] ^= 8*(BLOCKSIZE+32);
        v[13] ^= 8*(BLOCKSIZE+32);
    }

//#pragma unroll 16
    for( i = 0; i < 16; ++i )
    {
        /* column step */
        G( 0, 4, 8, 12, 0 );
        G( 1, 5, 9, 13, 2 );
        G( 2, 6, 10, 14, 4 );
        G( 3, 7, 11, 15, 6 );
        /* diagonal step */
        G( 0, 5, 10, 15, 8 );
        G( 1, 6, 11, 12, 10 );
        G( 2, 7, 8, 13, 12 );
        G( 3, 4, 9, 14, 14 );
    }

#pragma unroll 16
    for( i = 0; i < 16; ++i )  h[i % 8] ^= v[i];
}

template <int BLOCKSIZE> __global__ void blake512_gpu_hash(uint32_t threads, uint32_t startNounce, void *outputHash, uint32_t *heftyHashes, uint32_t *nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Zähler
		//uint32_t nounce = startNounce + thread;
		uint32_t nounce = nonceVector[thread];

		// Index-Position des Hashes in den Hash Puffern bestimmen (Hefty1 und outputHash)
		uint32_t hashPosition = nounce - startNounce;

		// State vorbereiten
		uint64_t h[8];
		h[0] = 0x6a09e667f3bcc908ULL;
		h[1] = 0xbb67ae8584caa73bULL;
		h[2] = 0x3c6ef372fe94f82bULL;
		h[3] = 0xa54ff53a5f1d36f1ULL;
		h[4] = 0x510e527fade682d1ULL;
		h[5] = 0x9b05688c2b3e6c1fULL;
		h[6] = 0x1f83d9abfb41bd6bULL;
		h[7] = 0x5be0cd19137e2179ULL;

		// 128 Byte für die Message
		uint64_t buf[16];

		// Message für die erste Runde in Register holen
#pragma unroll 16
		for (int i=0; i < 16; ++i) buf[i] = c_PaddedMessage[i];

		// die Nounce durch die thread-spezifische ersetzen
		buf[9] = REPLACE_HIDWORD(buf[9], nounce);

		uint32_t *hefty = heftyHashes + 8 * hashPosition;
		if (BLOCKSIZE == 84) {
			// den thread-spezifischen Hefty1 hash einsetzen
			// aufwändig, weil das nicht mit uint64_t Wörtern aligned ist.
			buf[10] = REPLACE_HIDWORD(buf[10], hefty[0]);
			buf[11] = REPLACE_LODWORD(buf[11], hefty[1]);
			buf[11] = REPLACE_HIDWORD(buf[11], hefty[2]);
			buf[12] = REPLACE_LODWORD(buf[12], hefty[3]);
			buf[12] = REPLACE_HIDWORD(buf[12], hefty[4]);
			buf[13] = REPLACE_LODWORD(buf[13], hefty[5]);
			buf[13] = REPLACE_HIDWORD(buf[13], hefty[6]);
			buf[14] = REPLACE_LODWORD(buf[14], hefty[7]);
		}
		else if (BLOCKSIZE == 80) {
			buf[10] = MAKE_ULONGLONG(hefty[0], hefty[1]);
			buf[11] = MAKE_ULONGLONG(hefty[2], hefty[3]);
			buf[12] = MAKE_ULONGLONG(hefty[4], hefty[5]);
			buf[13] = MAKE_ULONGLONG(hefty[6], hefty[7]);
		}

		// erste Runde
		blake512_compress<BLOCKSIZE>( h, buf, 0, c_sigma, c_u512 );


		// zweite Runde
#pragma unroll 15
		for (int i=0; i < 15; ++i) buf[i] = c_SecondRound[i];
		buf[15] = SWAP64(8*(BLOCKSIZE+32)); // Blocksize in Bits einsetzen
		blake512_compress<BLOCKSIZE>( h, buf, 1, c_sigma, c_u512 );

		// Hash rauslassen
		uint64_t *outHash = (uint64_t *)outputHash + 8 * hashPosition;
#pragma unroll 8
		for (int i=0; i < 8; ++i) outHash[i] = cuda_swab64( h[i] );
	}
}


// ---------------------------- END CUDA blake512 functions ------------------------------------

// Setup Function
__host__
void blake512_cpu_init(int thr_id, uint32_t threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(HIP_SYMBOL( c_sigma),
						host_sigma,
						sizeof(host_sigma),
						0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL( c_u512),
						host_u512,
						sizeof(host_u512),
						0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL( c_SecondRound),
						host_SecondRound,
						sizeof(host_SecondRound),
						0, hipMemcpyHostToDevice);

	// Speicher für alle Ergebnisse belegen
	CUDA_SAFE_CALL(hipMalloc(&d_hash5output[thr_id], (size_t) 64 * threads));
}

__host__
void blake512_cpu_free(int thr_id)
{
	hipFree(d_hash5output[thr_id]);
}

static int BLOCKSIZE = 84;

__host__
void blake512_cpu_setBlock(void *pdata, int len)
	// data muss 84-Byte haben!
	// heftyHash hat 32-Byte
{
	unsigned char PaddedMessage[128];
	if (len == 84) {
		// Message mit Padding für erste Runde bereitstellen
		memcpy(PaddedMessage, pdata, 84);
		memset(PaddedMessage+84, 0, 32); // leeres Hefty Hash einfüllen
		memset(PaddedMessage+116, 0, 12);
		PaddedMessage[116] = 0x80;
	} else if (len == 80) {
		memcpy(PaddedMessage, pdata, 80);
		memset(PaddedMessage+80, 0, 32); // leeres Hefty Hash einfüllen
		memset(PaddedMessage+112, 0, 16);
		PaddedMessage[112] = 0x80;
	}
	// die Message (116 Bytes) ohne Padding zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	BLOCKSIZE = len;
}

__host__ void blake512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce)
{
	const uint32_t threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Größe des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	if (BLOCKSIZE == 80)
		blake512_gpu_hash<80><<<grid, block, shared_size>>>(threads, startNounce, d_hash5output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
	else if (BLOCKSIZE == 84)
		blake512_gpu_hash<84><<<grid, block, shared_size>>>(threads, startNounce, d_hash5output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
}
