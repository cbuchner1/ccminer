#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cuda_helper.h"

// globaler Speicher für unsere Ergebnisse
static uint32_t *d_hashoutput[MAX_GPUS];
extern uint32_t *d_hash2output[MAX_GPUS];
extern uint32_t *d_hash3output[MAX_GPUS];
extern uint32_t *d_hash4output[MAX_GPUS];
extern uint32_t *d_hash5output[MAX_GPUS];

extern uint32_t *heavy_nonceVector[MAX_GPUS];

/* Combines top 64-bits from each hash into a single hash */
__device__
static void combine_hashes(uint32_t *out, uint32_t *hash1, uint32_t *hash2, uint32_t *hash3, uint32_t *hash4)
{
	uint32_t lout[8]; // Combining in Registern machen

#pragma unroll 8
	for (int i=0; i < 8; ++i)
		lout[i] = 0;

	// das Makro setzt jeweils 4 Bits aus vier verschiedenen Hashes zu einem Nibble zusammen
#define MIX(bits, mask, i) \
	lout[(255 - (bits+3))/32] <<= 4; \
	if ((hash1[i] & mask) != 0) lout[(255 - (bits+0))/32] |= 8; \
	if ((hash2[i] & mask) != 0) lout[(255 - (bits+1))/32] |= 4; \
	if ((hash3[i] & mask) != 0) lout[(255 - (bits+2))/32] |= 2; \
	if ((hash4[i] & mask) != 0) lout[(255 - (bits+3))/32] |= 1; \

	/* Transpose first 64 bits of each hash into out */
	MIX(  0, 0x80000000, 7);
	MIX(  4, 0x40000000, 7);
	MIX(  8, 0x20000000, 7);
	MIX( 12, 0x10000000, 7);
	MIX( 16, 0x08000000, 7);
	MIX( 20, 0x04000000, 7);
	MIX( 24, 0x02000000, 7);
	MIX( 28, 0x01000000, 7);
	MIX( 32, 0x00800000, 7);
	MIX( 36, 0x00400000, 7);
	MIX( 40, 0x00200000, 7);
	MIX( 44, 0x00100000, 7);
	MIX( 48, 0x00080000, 7);
	MIX( 52, 0x00040000, 7);
	MIX( 56, 0x00020000, 7);
	MIX( 60, 0x00010000, 7);
	MIX( 64, 0x00008000, 7);
	MIX( 68, 0x00004000, 7);
	MIX( 72, 0x00002000, 7);
	MIX( 76, 0x00001000, 7);
	MIX( 80, 0x00000800, 7);
	MIX( 84, 0x00000400, 7);
	MIX( 88, 0x00000200, 7);
	MIX( 92, 0x00000100, 7);
	MIX( 96, 0x00000080, 7);
	MIX(100, 0x00000040, 7);
	MIX(104, 0x00000020, 7);
	MIX(108, 0x00000010, 7);
	MIX(112, 0x00000008, 7);
	MIX(116, 0x00000004, 7);
	MIX(120, 0x00000002, 7);
	MIX(124, 0x00000001, 7);

	MIX(128, 0x80000000, 6);
	MIX(132, 0x40000000, 6);
	MIX(136, 0x20000000, 6);
	MIX(140, 0x10000000, 6);
	MIX(144, 0x08000000, 6);
	MIX(148, 0x04000000, 6);
	MIX(152, 0x02000000, 6);
	MIX(156, 0x01000000, 6);
	MIX(160, 0x00800000, 6);
	MIX(164, 0x00400000, 6);
	MIX(168, 0x00200000, 6);
	MIX(172, 0x00100000, 6);
	MIX(176, 0x00080000, 6);
	MIX(180, 0x00040000, 6);
	MIX(184, 0x00020000, 6);
	MIX(188, 0x00010000, 6);
	MIX(192, 0x00008000, 6);
	MIX(196, 0x00004000, 6);
	MIX(200, 0x00002000, 6);
	MIX(204, 0x00001000, 6);
	MIX(208, 0x00000800, 6);
	MIX(212, 0x00000400, 6);
	MIX(216, 0x00000200, 6);
	MIX(220, 0x00000100, 6);
	MIX(224, 0x00000080, 6);
	MIX(228, 0x00000040, 6);
	MIX(232, 0x00000020, 6);
	MIX(236, 0x00000010, 6);
	MIX(240, 0x00000008, 6);
	MIX(244, 0x00000004, 6);
	MIX(248, 0x00000002, 6);
	MIX(252, 0x00000001, 6);

#pragma unroll 8
	for (int i=0; i < 8; ++i)
		out[i] = lout[i];
}

__global__
void combine_gpu_hash(uint32_t threads, uint32_t startNounce, uint32_t *out, uint32_t *hash2, uint32_t *hash3, uint32_t *hash4, uint32_t *hash5, uint32_t *nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = nonceVector[thread];
		uint32_t hashPosition = nounce - startNounce;
		// Die Aufgabe der combine-funktion besteht aus zwei Teilen.
		// 1) Komprimiere die hashes zu einem kleinen Array
		// 2) Errechne dort den combines-value

		// Die Kompression wird dadurch verwirklicht, dass im out-array weiterhin mit "thread" indiziert
		// wird. Die anderen Werte werden mit der nonce indiziert

		combine_hashes(&out[8 * thread], &hash2[8 * hashPosition], &hash3[16 * hashPosition], &hash4[16 * hashPosition], &hash5[16 * hashPosition]);
	}
}

__host__
void combine_cpu_init(int thr_id, uint32_t threads)
{
	// Speicher für alle Ergebnisse belegen
	CUDA_SAFE_CALL(hipMalloc(&d_hashoutput[thr_id], (size_t) 32 * threads));
}

__host__
void combine_cpu_free(int thr_id)
{
	hipFree(d_hashoutput[thr_id]);
}

__host__
void combine_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *hash)
{
	// diese Kopien sind optional, da die Hashes jetzt bereits auf der GPU liegen sollten

	const uint32_t threadsperblock = 128;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	combine_gpu_hash <<<grid, block>>> (threads, startNounce, d_hashoutput[thr_id], d_hash2output[thr_id], d_hash3output[thr_id], d_hash4output[thr_id], d_hash5output[thr_id], heavy_nonceVector[thr_id]);

	// da die Hash Auswertung noch auf der CPU erfolgt, müssen die Ergebnisse auf jeden Fall zum Host kopiert werden
	CUDA_SAFE_CALL(hipMemcpy(hash, d_hashoutput[thr_id], 8 * sizeof(uint32_t) * threads, hipMemcpyDeviceToHost));
}
