#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#define SPH_C32(x) ((uint32_t)(x ## U))
#define SPH_T32(x) ((x) & SPH_C32(0xFFFFFFFF))

#include "cuda_helper.h"

// globaler Speicher für alle HeftyHashes aller Threads
extern uint32_t *heavy_heftyHashes[MAX_GPUS];
extern uint32_t *heavy_nonceVector[MAX_GPUS];
static unsigned int *d_textures[MAX_GPUS][8];

// globaler Speicher für unsere Ergebnisse
uint32_t *d_hash4output[MAX_GPUS];

__constant__ uint32_t groestl_gpu_state[32];
__constant__ uint32_t groestl_gpu_msg[32];

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    ((x) & 0xFF)
#define B32_1(x)    (((x) >> 8) & 0xFF)
#define B32_2(x)    (((x) >> 16) & 0xFF)
#define B32_3(x)    ((x) >> 24)

#define C32e(x)     ((SPH_C32(x) >> 24) \
                    | ((SPH_C32(x) >>  8) & SPH_C32(0x0000FF00)) \
                    | ((SPH_C32(x) <<  8) & SPH_C32(0x00FF0000)) \
                    | ((SPH_C32(x) << 24) & SPH_C32(0xFF000000)))

#define T0up(x) tex1Dfetch(t0up, x)
#define T0dn(x) tex1Dfetch(t0dn, x)
#define T1up(x) tex1Dfetch(t1up, x)
#define T1dn(x) tex1Dfetch(t1dn, x)
#define T2up(x) tex1Dfetch(t2up, x)
#define T2dn(x) tex1Dfetch(t2dn, x)
#define T3up(x) tex1Dfetch(t3up, x)
#define T3dn(x) tex1Dfetch(t3dn, x)

texture<unsigned int, 1, hipReadModeElementType> t0up;
texture<unsigned int, 1, hipReadModeElementType> t0dn;
texture<unsigned int, 1, hipReadModeElementType> t1up;
texture<unsigned int, 1, hipReadModeElementType> t1dn;
texture<unsigned int, 1, hipReadModeElementType> t2up;
texture<unsigned int, 1, hipReadModeElementType> t2dn;
texture<unsigned int, 1, hipReadModeElementType> t3up;
texture<unsigned int, 1, hipReadModeElementType> t3dn;

uint32_t T0up_cpu[] = {
	C32e(0xc632f4a5), C32e(0xf86f9784), C32e(0xee5eb099), C32e(0xf67a8c8d),
	C32e(0xffe8170d), C32e(0xd60adcbd), C32e(0xde16c8b1), C32e(0x916dfc54),
	C32e(0x6090f050), C32e(0x02070503), C32e(0xce2ee0a9), C32e(0x56d1877d),
	C32e(0xe7cc2b19), C32e(0xb513a662), C32e(0x4d7c31e6), C32e(0xec59b59a),
	C32e(0x8f40cf45), C32e(0x1fa3bc9d), C32e(0x8949c040), C32e(0xfa689287),
	C32e(0xefd03f15), C32e(0xb29426eb), C32e(0x8ece40c9), C32e(0xfbe61d0b),
	C32e(0x416e2fec), C32e(0xb31aa967), C32e(0x5f431cfd), C32e(0x456025ea),
	C32e(0x23f9dabf), C32e(0x535102f7), C32e(0xe445a196), C32e(0x9b76ed5b),
	C32e(0x75285dc2), C32e(0xe1c5241c), C32e(0x3dd4e9ae), C32e(0x4cf2be6a),
	C32e(0x6c82ee5a), C32e(0x7ebdc341), C32e(0xf5f30602), C32e(0x8352d14f),
	C32e(0x688ce45c), C32e(0x515607f4), C32e(0xd18d5c34), C32e(0xf9e11808),
	C32e(0xe24cae93), C32e(0xab3e9573), C32e(0x6297f553), C32e(0x2a6b413f),
	C32e(0x081c140c), C32e(0x9563f652), C32e(0x46e9af65), C32e(0x9d7fe25e),
	C32e(0x30487828), C32e(0x37cff8a1), C32e(0x0a1b110f), C32e(0x2febc4b5),
	C32e(0x0e151b09), C32e(0x247e5a36), C32e(0x1badb69b), C32e(0xdf98473d),
	C32e(0xcda76a26), C32e(0x4ef5bb69), C32e(0x7f334ccd), C32e(0xea50ba9f),
	C32e(0x123f2d1b), C32e(0x1da4b99e), C32e(0x58c49c74), C32e(0x3446722e),
	C32e(0x3641772d), C32e(0xdc11cdb2), C32e(0xb49d29ee), C32e(0x5b4d16fb),
	C32e(0xa4a501f6), C32e(0x76a1d74d), C32e(0xb714a361), C32e(0x7d3449ce),
	C32e(0x52df8d7b), C32e(0xdd9f423e), C32e(0x5ecd9371), C32e(0x13b1a297),
	C32e(0xa6a204f5), C32e(0xb901b868), C32e(0x00000000), C32e(0xc1b5742c),
	C32e(0x40e0a060), C32e(0xe3c2211f), C32e(0x793a43c8), C32e(0xb69a2ced),
	C32e(0xd40dd9be), C32e(0x8d47ca46), C32e(0x671770d9), C32e(0x72afdd4b),
	C32e(0x94ed79de), C32e(0x98ff67d4), C32e(0xb09323e8), C32e(0x855bde4a),
	C32e(0xbb06bd6b), C32e(0xc5bb7e2a), C32e(0x4f7b34e5), C32e(0xedd73a16),
	C32e(0x86d254c5), C32e(0x9af862d7), C32e(0x6699ff55), C32e(0x11b6a794),
	C32e(0x8ac04acf), C32e(0xe9d93010), C32e(0x040e0a06), C32e(0xfe669881),
	C32e(0xa0ab0bf0), C32e(0x78b4cc44), C32e(0x25f0d5ba), C32e(0x4b753ee3),
	C32e(0xa2ac0ef3), C32e(0x5d4419fe), C32e(0x80db5bc0), C32e(0x0580858a),
	C32e(0x3fd3ecad), C32e(0x21fedfbc), C32e(0x70a8d848), C32e(0xf1fd0c04),
	C32e(0x63197adf), C32e(0x772f58c1), C32e(0xaf309f75), C32e(0x42e7a563),
	C32e(0x20705030), C32e(0xe5cb2e1a), C32e(0xfdef120e), C32e(0xbf08b76d),
	C32e(0x8155d44c), C32e(0x18243c14), C32e(0x26795f35), C32e(0xc3b2712f),
	C32e(0xbe8638e1), C32e(0x35c8fda2), C32e(0x88c74fcc), C32e(0x2e654b39),
	C32e(0x936af957), C32e(0x55580df2), C32e(0xfc619d82), C32e(0x7ab3c947),
	C32e(0xc827efac), C32e(0xba8832e7), C32e(0x324f7d2b), C32e(0xe642a495),
	C32e(0xc03bfba0), C32e(0x19aab398), C32e(0x9ef668d1), C32e(0xa322817f),
	C32e(0x44eeaa66), C32e(0x54d6827e), C32e(0x3bdde6ab), C32e(0x0b959e83),
	C32e(0x8cc945ca), C32e(0xc7bc7b29), C32e(0x6b056ed3), C32e(0x286c443c),
	C32e(0xa72c8b79), C32e(0xbc813de2), C32e(0x1631271d), C32e(0xad379a76),
	C32e(0xdb964d3b), C32e(0x649efa56), C32e(0x74a6d24e), C32e(0x1436221e),
	C32e(0x92e476db), C32e(0x0c121e0a), C32e(0x48fcb46c), C32e(0xb88f37e4),
	C32e(0x9f78e75d), C32e(0xbd0fb26e), C32e(0x43692aef), C32e(0xc435f1a6),
	C32e(0x39dae3a8), C32e(0x31c6f7a4), C32e(0xd38a5937), C32e(0xf274868b),
	C32e(0xd5835632), C32e(0x8b4ec543), C32e(0x6e85eb59), C32e(0xda18c2b7),
	C32e(0x018e8f8c), C32e(0xb11dac64), C32e(0x9cf16dd2), C32e(0x49723be0),
	C32e(0xd81fc7b4), C32e(0xacb915fa), C32e(0xf3fa0907), C32e(0xcfa06f25),
	C32e(0xca20eaaf), C32e(0xf47d898e), C32e(0x476720e9), C32e(0x10382818),
	C32e(0x6f0b64d5), C32e(0xf0738388), C32e(0x4afbb16f), C32e(0x5cca9672),
	C32e(0x38546c24), C32e(0x575f08f1), C32e(0x732152c7), C32e(0x9764f351),
	C32e(0xcbae6523), C32e(0xa125847c), C32e(0xe857bf9c), C32e(0x3e5d6321),
	C32e(0x96ea7cdd), C32e(0x611e7fdc), C32e(0x0d9c9186), C32e(0x0f9b9485),
	C32e(0xe04bab90), C32e(0x7cbac642), C32e(0x712657c4), C32e(0xcc29e5aa),
	C32e(0x90e373d8), C32e(0x06090f05), C32e(0xf7f40301), C32e(0x1c2a3612),
	C32e(0xc23cfea3), C32e(0x6a8be15f), C32e(0xaebe10f9), C32e(0x69026bd0),
	C32e(0x17bfa891), C32e(0x9971e858), C32e(0x3a536927), C32e(0x27f7d0b9),
	C32e(0xd9914838), C32e(0xebde3513), C32e(0x2be5ceb3), C32e(0x22775533),
	C32e(0xd204d6bb), C32e(0xa9399070), C32e(0x07878089), C32e(0x33c1f2a7),
	C32e(0x2decc1b6), C32e(0x3c5a6622), C32e(0x15b8ad92), C32e(0xc9a96020),
	C32e(0x875cdb49), C32e(0xaab01aff), C32e(0x50d88878), C32e(0xa52b8e7a),
	C32e(0x03898a8f), C32e(0x594a13f8), C32e(0x09929b80), C32e(0x1a233917),
	C32e(0x651075da), C32e(0xd7845331), C32e(0x84d551c6), C32e(0xd003d3b8),
	C32e(0x82dc5ec3), C32e(0x29e2cbb0), C32e(0x5ac39977), C32e(0x1e2d3311),
	C32e(0x7b3d46cb), C32e(0xa8b71ffc), C32e(0x6d0c61d6), C32e(0x2c624e3a)
};

uint32_t T0dn_cpu[] = {
	C32e(0xf497a5c6), C32e(0x97eb84f8), C32e(0xb0c799ee), C32e(0x8cf78df6),
	C32e(0x17e50dff), C32e(0xdcb7bdd6), C32e(0xc8a7b1de), C32e(0xfc395491),
	C32e(0xf0c05060), C32e(0x05040302), C32e(0xe087a9ce), C32e(0x87ac7d56),
	C32e(0x2bd519e7), C32e(0xa67162b5), C32e(0x319ae64d), C32e(0xb5c39aec),
	C32e(0xcf05458f), C32e(0xbc3e9d1f), C32e(0xc0094089), C32e(0x92ef87fa),
	C32e(0x3fc515ef), C32e(0x267febb2), C32e(0x4007c98e), C32e(0x1ded0bfb),
	C32e(0x2f82ec41), C32e(0xa97d67b3), C32e(0x1cbefd5f), C32e(0x258aea45),
	C32e(0xda46bf23), C32e(0x02a6f753), C32e(0xa1d396e4), C32e(0xed2d5b9b),
	C32e(0x5deac275), C32e(0x24d91ce1), C32e(0xe97aae3d), C32e(0xbe986a4c),
	C32e(0xeed85a6c), C32e(0xc3fc417e), C32e(0x06f102f5), C32e(0xd11d4f83),
	C32e(0xe4d05c68), C32e(0x07a2f451), C32e(0x5cb934d1), C32e(0x18e908f9),
	C32e(0xaedf93e2), C32e(0x954d73ab), C32e(0xf5c45362), C32e(0x41543f2a),
	C32e(0x14100c08), C32e(0xf6315295), C32e(0xaf8c6546), C32e(0xe2215e9d),
	C32e(0x78602830), C32e(0xf86ea137), C32e(0x11140f0a), C32e(0xc45eb52f),
	C32e(0x1b1c090e), C32e(0x5a483624), C32e(0xb6369b1b), C32e(0x47a53ddf),
	C32e(0x6a8126cd), C32e(0xbb9c694e), C32e(0x4cfecd7f), C32e(0xbacf9fea),
	C32e(0x2d241b12), C32e(0xb93a9e1d), C32e(0x9cb07458), C32e(0x72682e34),
	C32e(0x776c2d36), C32e(0xcda3b2dc), C32e(0x2973eeb4), C32e(0x16b6fb5b),
	C32e(0x0153f6a4), C32e(0xd7ec4d76), C32e(0xa37561b7), C32e(0x49face7d),
	C32e(0x8da47b52), C32e(0x42a13edd), C32e(0x93bc715e), C32e(0xa2269713),
	C32e(0x0457f5a6), C32e(0xb86968b9), C32e(0x00000000), C32e(0x74992cc1),
	C32e(0xa0806040), C32e(0x21dd1fe3), C32e(0x43f2c879), C32e(0x2c77edb6),
	C32e(0xd9b3bed4), C32e(0xca01468d), C32e(0x70ced967), C32e(0xdde44b72),
	C32e(0x7933de94), C32e(0x672bd498), C32e(0x237be8b0), C32e(0xde114a85),
	C32e(0xbd6d6bbb), C32e(0x7e912ac5), C32e(0x349ee54f), C32e(0x3ac116ed),
	C32e(0x5417c586), C32e(0x622fd79a), C32e(0xffcc5566), C32e(0xa7229411),
	C32e(0x4a0fcf8a), C32e(0x30c910e9), C32e(0x0a080604), C32e(0x98e781fe),
	C32e(0x0b5bf0a0), C32e(0xccf04478), C32e(0xd54aba25), C32e(0x3e96e34b),
	C32e(0x0e5ff3a2), C32e(0x19bafe5d), C32e(0x5b1bc080), C32e(0x850a8a05),
	C32e(0xec7ead3f), C32e(0xdf42bc21), C32e(0xd8e04870), C32e(0x0cf904f1),
	C32e(0x7ac6df63), C32e(0x58eec177), C32e(0x9f4575af), C32e(0xa5846342),
	C32e(0x50403020), C32e(0x2ed11ae5), C32e(0x12e10efd), C32e(0xb7656dbf),
	C32e(0xd4194c81), C32e(0x3c301418), C32e(0x5f4c3526), C32e(0x719d2fc3),
	C32e(0x3867e1be), C32e(0xfd6aa235), C32e(0x4f0bcc88), C32e(0x4b5c392e),
	C32e(0xf93d5793), C32e(0x0daaf255), C32e(0x9de382fc), C32e(0xc9f4477a),
	C32e(0xef8bacc8), C32e(0x326fe7ba), C32e(0x7d642b32), C32e(0xa4d795e6),
	C32e(0xfb9ba0c0), C32e(0xb3329819), C32e(0x6827d19e), C32e(0x815d7fa3),
	C32e(0xaa886644), C32e(0x82a87e54), C32e(0xe676ab3b), C32e(0x9e16830b),
	C32e(0x4503ca8c), C32e(0x7b9529c7), C32e(0x6ed6d36b), C32e(0x44503c28),
	C32e(0x8b5579a7), C32e(0x3d63e2bc), C32e(0x272c1d16), C32e(0x9a4176ad),
	C32e(0x4dad3bdb), C32e(0xfac85664), C32e(0xd2e84e74), C32e(0x22281e14),
	C32e(0x763fdb92), C32e(0x1e180a0c), C32e(0xb4906c48), C32e(0x376be4b8),
	C32e(0xe7255d9f), C32e(0xb2616ebd), C32e(0x2a86ef43), C32e(0xf193a6c4),
	C32e(0xe372a839), C32e(0xf762a431), C32e(0x59bd37d3), C32e(0x86ff8bf2),
	C32e(0x56b132d5), C32e(0xc50d438b), C32e(0xebdc596e), C32e(0xc2afb7da),
	C32e(0x8f028c01), C32e(0xac7964b1), C32e(0x6d23d29c), C32e(0x3b92e049),
	C32e(0xc7abb4d8), C32e(0x1543faac), C32e(0x09fd07f3), C32e(0x6f8525cf),
	C32e(0xea8fafca), C32e(0x89f38ef4), C32e(0x208ee947), C32e(0x28201810),
	C32e(0x64ded56f), C32e(0x83fb88f0), C32e(0xb1946f4a), C32e(0x96b8725c),
	C32e(0x6c702438), C32e(0x08aef157), C32e(0x52e6c773), C32e(0xf3355197),
	C32e(0x658d23cb), C32e(0x84597ca1), C32e(0xbfcb9ce8), C32e(0x637c213e),
	C32e(0x7c37dd96), C32e(0x7fc2dc61), C32e(0x911a860d), C32e(0x941e850f),
	C32e(0xabdb90e0), C32e(0xc6f8427c), C32e(0x57e2c471), C32e(0xe583aacc),
	C32e(0x733bd890), C32e(0x0f0c0506), C32e(0x03f501f7), C32e(0x3638121c),
	C32e(0xfe9fa3c2), C32e(0xe1d45f6a), C32e(0x1047f9ae), C32e(0x6bd2d069),
	C32e(0xa82e9117), C32e(0xe8295899), C32e(0x6974273a), C32e(0xd04eb927),
	C32e(0x48a938d9), C32e(0x35cd13eb), C32e(0xce56b32b), C32e(0x55443322),
	C32e(0xd6bfbbd2), C32e(0x904970a9), C32e(0x800e8907), C32e(0xf266a733),
	C32e(0xc15ab62d), C32e(0x6678223c), C32e(0xad2a9215), C32e(0x608920c9),
	C32e(0xdb154987), C32e(0x1a4fffaa), C32e(0x88a07850), C32e(0x8e517aa5),
	C32e(0x8a068f03), C32e(0x13b2f859), C32e(0x9b128009), C32e(0x3934171a),
	C32e(0x75cada65), C32e(0x53b531d7), C32e(0x5113c684), C32e(0xd3bbb8d0),
	C32e(0x5e1fc382), C32e(0xcb52b029), C32e(0x99b4775a), C32e(0x333c111e),
	C32e(0x46f6cb7b), C32e(0x1f4bfca8), C32e(0x61dad66d), C32e(0x4e583a2c)
};

uint32_t T1up_cpu[] = {
	C32e(0xc6c632f4), C32e(0xf8f86f97), C32e(0xeeee5eb0), C32e(0xf6f67a8c),
	C32e(0xffffe817), C32e(0xd6d60adc), C32e(0xdede16c8), C32e(0x91916dfc),
	C32e(0x606090f0), C32e(0x02020705), C32e(0xcece2ee0), C32e(0x5656d187),
	C32e(0xe7e7cc2b), C32e(0xb5b513a6), C32e(0x4d4d7c31), C32e(0xecec59b5),
	C32e(0x8f8f40cf), C32e(0x1f1fa3bc), C32e(0x898949c0), C32e(0xfafa6892),
	C32e(0xefefd03f), C32e(0xb2b29426), C32e(0x8e8ece40), C32e(0xfbfbe61d),
	C32e(0x41416e2f), C32e(0xb3b31aa9), C32e(0x5f5f431c), C32e(0x45456025),
	C32e(0x2323f9da), C32e(0x53535102), C32e(0xe4e445a1), C32e(0x9b9b76ed),
	C32e(0x7575285d), C32e(0xe1e1c524), C32e(0x3d3dd4e9), C32e(0x4c4cf2be),
	C32e(0x6c6c82ee), C32e(0x7e7ebdc3), C32e(0xf5f5f306), C32e(0x838352d1),
	C32e(0x68688ce4), C32e(0x51515607), C32e(0xd1d18d5c), C32e(0xf9f9e118),
	C32e(0xe2e24cae), C32e(0xabab3e95), C32e(0x626297f5), C32e(0x2a2a6b41),
	C32e(0x08081c14), C32e(0x959563f6), C32e(0x4646e9af), C32e(0x9d9d7fe2),
	C32e(0x30304878), C32e(0x3737cff8), C32e(0x0a0a1b11), C32e(0x2f2febc4),
	C32e(0x0e0e151b), C32e(0x24247e5a), C32e(0x1b1badb6), C32e(0xdfdf9847),
	C32e(0xcdcda76a), C32e(0x4e4ef5bb), C32e(0x7f7f334c), C32e(0xeaea50ba),
	C32e(0x12123f2d), C32e(0x1d1da4b9), C32e(0x5858c49c), C32e(0x34344672),
	C32e(0x36364177), C32e(0xdcdc11cd), C32e(0xb4b49d29), C32e(0x5b5b4d16),
	C32e(0xa4a4a501), C32e(0x7676a1d7), C32e(0xb7b714a3), C32e(0x7d7d3449),
	C32e(0x5252df8d), C32e(0xdddd9f42), C32e(0x5e5ecd93), C32e(0x1313b1a2),
	C32e(0xa6a6a204), C32e(0xb9b901b8), C32e(0x00000000), C32e(0xc1c1b574),
	C32e(0x4040e0a0), C32e(0xe3e3c221), C32e(0x79793a43), C32e(0xb6b69a2c),
	C32e(0xd4d40dd9), C32e(0x8d8d47ca), C32e(0x67671770), C32e(0x7272afdd),
	C32e(0x9494ed79), C32e(0x9898ff67), C32e(0xb0b09323), C32e(0x85855bde),
	C32e(0xbbbb06bd), C32e(0xc5c5bb7e), C32e(0x4f4f7b34), C32e(0xededd73a),
	C32e(0x8686d254), C32e(0x9a9af862), C32e(0x666699ff), C32e(0x1111b6a7),
	C32e(0x8a8ac04a), C32e(0xe9e9d930), C32e(0x04040e0a), C32e(0xfefe6698),
	C32e(0xa0a0ab0b), C32e(0x7878b4cc), C32e(0x2525f0d5), C32e(0x4b4b753e),
	C32e(0xa2a2ac0e), C32e(0x5d5d4419), C32e(0x8080db5b), C32e(0x05058085),
	C32e(0x3f3fd3ec), C32e(0x2121fedf), C32e(0x7070a8d8), C32e(0xf1f1fd0c),
	C32e(0x6363197a), C32e(0x77772f58), C32e(0xafaf309f), C32e(0x4242e7a5),
	C32e(0x20207050), C32e(0xe5e5cb2e), C32e(0xfdfdef12), C32e(0xbfbf08b7),
	C32e(0x818155d4), C32e(0x1818243c), C32e(0x2626795f), C32e(0xc3c3b271),
	C32e(0xbebe8638), C32e(0x3535c8fd), C32e(0x8888c74f), C32e(0x2e2e654b),
	C32e(0x93936af9), C32e(0x5555580d), C32e(0xfcfc619d), C32e(0x7a7ab3c9),
	C32e(0xc8c827ef), C32e(0xbaba8832), C32e(0x32324f7d), C32e(0xe6e642a4),
	C32e(0xc0c03bfb), C32e(0x1919aab3), C32e(0x9e9ef668), C32e(0xa3a32281),
	C32e(0x4444eeaa), C32e(0x5454d682), C32e(0x3b3bdde6), C32e(0x0b0b959e),
	C32e(0x8c8cc945), C32e(0xc7c7bc7b), C32e(0x6b6b056e), C32e(0x28286c44),
	C32e(0xa7a72c8b), C32e(0xbcbc813d), C32e(0x16163127), C32e(0xadad379a),
	C32e(0xdbdb964d), C32e(0x64649efa), C32e(0x7474a6d2), C32e(0x14143622),
	C32e(0x9292e476), C32e(0x0c0c121e), C32e(0x4848fcb4), C32e(0xb8b88f37),
	C32e(0x9f9f78e7), C32e(0xbdbd0fb2), C32e(0x4343692a), C32e(0xc4c435f1),
	C32e(0x3939dae3), C32e(0x3131c6f7), C32e(0xd3d38a59), C32e(0xf2f27486),
	C32e(0xd5d58356), C32e(0x8b8b4ec5), C32e(0x6e6e85eb), C32e(0xdada18c2),
	C32e(0x01018e8f), C32e(0xb1b11dac), C32e(0x9c9cf16d), C32e(0x4949723b),
	C32e(0xd8d81fc7), C32e(0xacacb915), C32e(0xf3f3fa09), C32e(0xcfcfa06f),
	C32e(0xcaca20ea), C32e(0xf4f47d89), C32e(0x47476720), C32e(0x10103828),
	C32e(0x6f6f0b64), C32e(0xf0f07383), C32e(0x4a4afbb1), C32e(0x5c5cca96),
	C32e(0x3838546c), C32e(0x57575f08), C32e(0x73732152), C32e(0x979764f3),
	C32e(0xcbcbae65), C32e(0xa1a12584), C32e(0xe8e857bf), C32e(0x3e3e5d63),
	C32e(0x9696ea7c), C32e(0x61611e7f), C32e(0x0d0d9c91), C32e(0x0f0f9b94),
	C32e(0xe0e04bab), C32e(0x7c7cbac6), C32e(0x71712657), C32e(0xcccc29e5),
	C32e(0x9090e373), C32e(0x0606090f), C32e(0xf7f7f403), C32e(0x1c1c2a36),
	C32e(0xc2c23cfe), C32e(0x6a6a8be1), C32e(0xaeaebe10), C32e(0x6969026b),
	C32e(0x1717bfa8), C32e(0x999971e8), C32e(0x3a3a5369), C32e(0x2727f7d0),
	C32e(0xd9d99148), C32e(0xebebde35), C32e(0x2b2be5ce), C32e(0x22227755),
	C32e(0xd2d204d6), C32e(0xa9a93990), C32e(0x07078780), C32e(0x3333c1f2),
	C32e(0x2d2decc1), C32e(0x3c3c5a66), C32e(0x1515b8ad), C32e(0xc9c9a960),
	C32e(0x87875cdb), C32e(0xaaaab01a), C32e(0x5050d888), C32e(0xa5a52b8e),
	C32e(0x0303898a), C32e(0x59594a13), C32e(0x0909929b), C32e(0x1a1a2339),
	C32e(0x65651075), C32e(0xd7d78453), C32e(0x8484d551), C32e(0xd0d003d3),
	C32e(0x8282dc5e), C32e(0x2929e2cb), C32e(0x5a5ac399), C32e(0x1e1e2d33),
	C32e(0x7b7b3d46), C32e(0xa8a8b71f), C32e(0x6d6d0c61), C32e(0x2c2c624e)
};

uint32_t T1dn_cpu[] = {
	C32e(0xa5f497a5), C32e(0x8497eb84), C32e(0x99b0c799), C32e(0x8d8cf78d),
	C32e(0x0d17e50d), C32e(0xbddcb7bd), C32e(0xb1c8a7b1), C32e(0x54fc3954),
	C32e(0x50f0c050), C32e(0x03050403), C32e(0xa9e087a9), C32e(0x7d87ac7d),
	C32e(0x192bd519), C32e(0x62a67162), C32e(0xe6319ae6), C32e(0x9ab5c39a),
	C32e(0x45cf0545), C32e(0x9dbc3e9d), C32e(0x40c00940), C32e(0x8792ef87),
	C32e(0x153fc515), C32e(0xeb267feb), C32e(0xc94007c9), C32e(0x0b1ded0b),
	C32e(0xec2f82ec), C32e(0x67a97d67), C32e(0xfd1cbefd), C32e(0xea258aea),
	C32e(0xbfda46bf), C32e(0xf702a6f7), C32e(0x96a1d396), C32e(0x5bed2d5b),
	C32e(0xc25deac2), C32e(0x1c24d91c), C32e(0xaee97aae), C32e(0x6abe986a),
	C32e(0x5aeed85a), C32e(0x41c3fc41), C32e(0x0206f102), C32e(0x4fd11d4f),
	C32e(0x5ce4d05c), C32e(0xf407a2f4), C32e(0x345cb934), C32e(0x0818e908),
	C32e(0x93aedf93), C32e(0x73954d73), C32e(0x53f5c453), C32e(0x3f41543f),
	C32e(0x0c14100c), C32e(0x52f63152), C32e(0x65af8c65), C32e(0x5ee2215e),
	C32e(0x28786028), C32e(0xa1f86ea1), C32e(0x0f11140f), C32e(0xb5c45eb5),
	C32e(0x091b1c09), C32e(0x365a4836), C32e(0x9bb6369b), C32e(0x3d47a53d),
	C32e(0x266a8126), C32e(0x69bb9c69), C32e(0xcd4cfecd), C32e(0x9fbacf9f),
	C32e(0x1b2d241b), C32e(0x9eb93a9e), C32e(0x749cb074), C32e(0x2e72682e),
	C32e(0x2d776c2d), C32e(0xb2cda3b2), C32e(0xee2973ee), C32e(0xfb16b6fb),
	C32e(0xf60153f6), C32e(0x4dd7ec4d), C32e(0x61a37561), C32e(0xce49face),
	C32e(0x7b8da47b), C32e(0x3e42a13e), C32e(0x7193bc71), C32e(0x97a22697),
	C32e(0xf50457f5), C32e(0x68b86968), C32e(0x00000000), C32e(0x2c74992c),
	C32e(0x60a08060), C32e(0x1f21dd1f), C32e(0xc843f2c8), C32e(0xed2c77ed),
	C32e(0xbed9b3be), C32e(0x46ca0146), C32e(0xd970ced9), C32e(0x4bdde44b),
	C32e(0xde7933de), C32e(0xd4672bd4), C32e(0xe8237be8), C32e(0x4ade114a),
	C32e(0x6bbd6d6b), C32e(0x2a7e912a), C32e(0xe5349ee5), C32e(0x163ac116),
	C32e(0xc55417c5), C32e(0xd7622fd7), C32e(0x55ffcc55), C32e(0x94a72294),
	C32e(0xcf4a0fcf), C32e(0x1030c910), C32e(0x060a0806), C32e(0x8198e781),
	C32e(0xf00b5bf0), C32e(0x44ccf044), C32e(0xbad54aba), C32e(0xe33e96e3),
	C32e(0xf30e5ff3), C32e(0xfe19bafe), C32e(0xc05b1bc0), C32e(0x8a850a8a),
	C32e(0xadec7ead), C32e(0xbcdf42bc), C32e(0x48d8e048), C32e(0x040cf904),
	C32e(0xdf7ac6df), C32e(0xc158eec1), C32e(0x759f4575), C32e(0x63a58463),
	C32e(0x30504030), C32e(0x1a2ed11a), C32e(0x0e12e10e), C32e(0x6db7656d),
	C32e(0x4cd4194c), C32e(0x143c3014), C32e(0x355f4c35), C32e(0x2f719d2f),
	C32e(0xe13867e1), C32e(0xa2fd6aa2), C32e(0xcc4f0bcc), C32e(0x394b5c39),
	C32e(0x57f93d57), C32e(0xf20daaf2), C32e(0x829de382), C32e(0x47c9f447),
	C32e(0xacef8bac), C32e(0xe7326fe7), C32e(0x2b7d642b), C32e(0x95a4d795),
	C32e(0xa0fb9ba0), C32e(0x98b33298), C32e(0xd16827d1), C32e(0x7f815d7f),
	C32e(0x66aa8866), C32e(0x7e82a87e), C32e(0xabe676ab), C32e(0x839e1683),
	C32e(0xca4503ca), C32e(0x297b9529), C32e(0xd36ed6d3), C32e(0x3c44503c),
	C32e(0x798b5579), C32e(0xe23d63e2), C32e(0x1d272c1d), C32e(0x769a4176),
	C32e(0x3b4dad3b), C32e(0x56fac856), C32e(0x4ed2e84e), C32e(0x1e22281e),
	C32e(0xdb763fdb), C32e(0x0a1e180a), C32e(0x6cb4906c), C32e(0xe4376be4),
	C32e(0x5de7255d), C32e(0x6eb2616e), C32e(0xef2a86ef), C32e(0xa6f193a6),
	C32e(0xa8e372a8), C32e(0xa4f762a4), C32e(0x3759bd37), C32e(0x8b86ff8b),
	C32e(0x3256b132), C32e(0x43c50d43), C32e(0x59ebdc59), C32e(0xb7c2afb7),
	C32e(0x8c8f028c), C32e(0x64ac7964), C32e(0xd26d23d2), C32e(0xe03b92e0),
	C32e(0xb4c7abb4), C32e(0xfa1543fa), C32e(0x0709fd07), C32e(0x256f8525),
	C32e(0xafea8faf), C32e(0x8e89f38e), C32e(0xe9208ee9), C32e(0x18282018),
	C32e(0xd564ded5), C32e(0x8883fb88), C32e(0x6fb1946f), C32e(0x7296b872),
	C32e(0x246c7024), C32e(0xf108aef1), C32e(0xc752e6c7), C32e(0x51f33551),
	C32e(0x23658d23), C32e(0x7c84597c), C32e(0x9cbfcb9c), C32e(0x21637c21),
	C32e(0xdd7c37dd), C32e(0xdc7fc2dc), C32e(0x86911a86), C32e(0x85941e85),
	C32e(0x90abdb90), C32e(0x42c6f842), C32e(0xc457e2c4), C32e(0xaae583aa),
	C32e(0xd8733bd8), C32e(0x050f0c05), C32e(0x0103f501), C32e(0x12363812),
	C32e(0xa3fe9fa3), C32e(0x5fe1d45f), C32e(0xf91047f9), C32e(0xd06bd2d0),
	C32e(0x91a82e91), C32e(0x58e82958), C32e(0x27697427), C32e(0xb9d04eb9),
	C32e(0x3848a938), C32e(0x1335cd13), C32e(0xb3ce56b3), C32e(0x33554433),
	C32e(0xbbd6bfbb), C32e(0x70904970), C32e(0x89800e89), C32e(0xa7f266a7),
	C32e(0xb6c15ab6), C32e(0x22667822), C32e(0x92ad2a92), C32e(0x20608920),
	C32e(0x49db1549), C32e(0xff1a4fff), C32e(0x7888a078), C32e(0x7a8e517a),
	C32e(0x8f8a068f), C32e(0xf813b2f8), C32e(0x809b1280), C32e(0x17393417),
	C32e(0xda75cada), C32e(0x3153b531), C32e(0xc65113c6), C32e(0xb8d3bbb8),
	C32e(0xc35e1fc3), C32e(0xb0cb52b0), C32e(0x7799b477), C32e(0x11333c11),
	C32e(0xcb46f6cb), C32e(0xfc1f4bfc), C32e(0xd661dad6), C32e(0x3a4e583a)
};

uint32_t T2up_cpu[] = {
	C32e(0xa5c6c632), C32e(0x84f8f86f), C32e(0x99eeee5e), C32e(0x8df6f67a),
	C32e(0x0dffffe8), C32e(0xbdd6d60a), C32e(0xb1dede16), C32e(0x5491916d),
	C32e(0x50606090), C32e(0x03020207), C32e(0xa9cece2e), C32e(0x7d5656d1),
	C32e(0x19e7e7cc), C32e(0x62b5b513), C32e(0xe64d4d7c), C32e(0x9aecec59),
	C32e(0x458f8f40), C32e(0x9d1f1fa3), C32e(0x40898949), C32e(0x87fafa68),
	C32e(0x15efefd0), C32e(0xebb2b294), C32e(0xc98e8ece), C32e(0x0bfbfbe6),
	C32e(0xec41416e), C32e(0x67b3b31a), C32e(0xfd5f5f43), C32e(0xea454560),
	C32e(0xbf2323f9), C32e(0xf7535351), C32e(0x96e4e445), C32e(0x5b9b9b76),
	C32e(0xc2757528), C32e(0x1ce1e1c5), C32e(0xae3d3dd4), C32e(0x6a4c4cf2),
	C32e(0x5a6c6c82), C32e(0x417e7ebd), C32e(0x02f5f5f3), C32e(0x4f838352),
	C32e(0x5c68688c), C32e(0xf4515156), C32e(0x34d1d18d), C32e(0x08f9f9e1),
	C32e(0x93e2e24c), C32e(0x73abab3e), C32e(0x53626297), C32e(0x3f2a2a6b),
	C32e(0x0c08081c), C32e(0x52959563), C32e(0x654646e9), C32e(0x5e9d9d7f),
	C32e(0x28303048), C32e(0xa13737cf), C32e(0x0f0a0a1b), C32e(0xb52f2feb),
	C32e(0x090e0e15), C32e(0x3624247e), C32e(0x9b1b1bad), C32e(0x3ddfdf98),
	C32e(0x26cdcda7), C32e(0x694e4ef5), C32e(0xcd7f7f33), C32e(0x9feaea50),
	C32e(0x1b12123f), C32e(0x9e1d1da4), C32e(0x745858c4), C32e(0x2e343446),
	C32e(0x2d363641), C32e(0xb2dcdc11), C32e(0xeeb4b49d), C32e(0xfb5b5b4d),
	C32e(0xf6a4a4a5), C32e(0x4d7676a1), C32e(0x61b7b714), C32e(0xce7d7d34),
	C32e(0x7b5252df), C32e(0x3edddd9f), C32e(0x715e5ecd), C32e(0x971313b1),
	C32e(0xf5a6a6a2), C32e(0x68b9b901), C32e(0x00000000), C32e(0x2cc1c1b5),
	C32e(0x604040e0), C32e(0x1fe3e3c2), C32e(0xc879793a), C32e(0xedb6b69a),
	C32e(0xbed4d40d), C32e(0x468d8d47), C32e(0xd9676717), C32e(0x4b7272af),
	C32e(0xde9494ed), C32e(0xd49898ff), C32e(0xe8b0b093), C32e(0x4a85855b),
	C32e(0x6bbbbb06), C32e(0x2ac5c5bb), C32e(0xe54f4f7b), C32e(0x16ededd7),
	C32e(0xc58686d2), C32e(0xd79a9af8), C32e(0x55666699), C32e(0x941111b6),
	C32e(0xcf8a8ac0), C32e(0x10e9e9d9), C32e(0x0604040e), C32e(0x81fefe66),
	C32e(0xf0a0a0ab), C32e(0x447878b4), C32e(0xba2525f0), C32e(0xe34b4b75),
	C32e(0xf3a2a2ac), C32e(0xfe5d5d44), C32e(0xc08080db), C32e(0x8a050580),
	C32e(0xad3f3fd3), C32e(0xbc2121fe), C32e(0x487070a8), C32e(0x04f1f1fd),
	C32e(0xdf636319), C32e(0xc177772f), C32e(0x75afaf30), C32e(0x634242e7),
	C32e(0x30202070), C32e(0x1ae5e5cb), C32e(0x0efdfdef), C32e(0x6dbfbf08),
	C32e(0x4c818155), C32e(0x14181824), C32e(0x35262679), C32e(0x2fc3c3b2),
	C32e(0xe1bebe86), C32e(0xa23535c8), C32e(0xcc8888c7), C32e(0x392e2e65),
	C32e(0x5793936a), C32e(0xf2555558), C32e(0x82fcfc61), C32e(0x477a7ab3),
	C32e(0xacc8c827), C32e(0xe7baba88), C32e(0x2b32324f), C32e(0x95e6e642),
	C32e(0xa0c0c03b), C32e(0x981919aa), C32e(0xd19e9ef6), C32e(0x7fa3a322),
	C32e(0x664444ee), C32e(0x7e5454d6), C32e(0xab3b3bdd), C32e(0x830b0b95),
	C32e(0xca8c8cc9), C32e(0x29c7c7bc), C32e(0xd36b6b05), C32e(0x3c28286c),
	C32e(0x79a7a72c), C32e(0xe2bcbc81), C32e(0x1d161631), C32e(0x76adad37),
	C32e(0x3bdbdb96), C32e(0x5664649e), C32e(0x4e7474a6), C32e(0x1e141436),
	C32e(0xdb9292e4), C32e(0x0a0c0c12), C32e(0x6c4848fc), C32e(0xe4b8b88f),
	C32e(0x5d9f9f78), C32e(0x6ebdbd0f), C32e(0xef434369), C32e(0xa6c4c435),
	C32e(0xa83939da), C32e(0xa43131c6), C32e(0x37d3d38a), C32e(0x8bf2f274),
	C32e(0x32d5d583), C32e(0x438b8b4e), C32e(0x596e6e85), C32e(0xb7dada18),
	C32e(0x8c01018e), C32e(0x64b1b11d), C32e(0xd29c9cf1), C32e(0xe0494972),
	C32e(0xb4d8d81f), C32e(0xfaacacb9), C32e(0x07f3f3fa), C32e(0x25cfcfa0),
	C32e(0xafcaca20), C32e(0x8ef4f47d), C32e(0xe9474767), C32e(0x18101038),
	C32e(0xd56f6f0b), C32e(0x88f0f073), C32e(0x6f4a4afb), C32e(0x725c5cca),
	C32e(0x24383854), C32e(0xf157575f), C32e(0xc7737321), C32e(0x51979764),
	C32e(0x23cbcbae), C32e(0x7ca1a125), C32e(0x9ce8e857), C32e(0x213e3e5d),
	C32e(0xdd9696ea), C32e(0xdc61611e), C32e(0x860d0d9c), C32e(0x850f0f9b),
	C32e(0x90e0e04b), C32e(0x427c7cba), C32e(0xc4717126), C32e(0xaacccc29),
	C32e(0xd89090e3), C32e(0x05060609), C32e(0x01f7f7f4), C32e(0x121c1c2a),
	C32e(0xa3c2c23c), C32e(0x5f6a6a8b), C32e(0xf9aeaebe), C32e(0xd0696902),
	C32e(0x911717bf), C32e(0x58999971), C32e(0x273a3a53), C32e(0xb92727f7),
	C32e(0x38d9d991), C32e(0x13ebebde), C32e(0xb32b2be5), C32e(0x33222277),
	C32e(0xbbd2d204), C32e(0x70a9a939), C32e(0x89070787), C32e(0xa73333c1),
	C32e(0xb62d2dec), C32e(0x223c3c5a), C32e(0x921515b8), C32e(0x20c9c9a9),
	C32e(0x4987875c), C32e(0xffaaaab0), C32e(0x785050d8), C32e(0x7aa5a52b),
	C32e(0x8f030389), C32e(0xf859594a), C32e(0x80090992), C32e(0x171a1a23),
	C32e(0xda656510), C32e(0x31d7d784), C32e(0xc68484d5), C32e(0xb8d0d003),
	C32e(0xc38282dc), C32e(0xb02929e2), C32e(0x775a5ac3), C32e(0x111e1e2d),
	C32e(0xcb7b7b3d), C32e(0xfca8a8b7), C32e(0xd66d6d0c), C32e(0x3a2c2c62)
};

uint32_t T2dn_cpu[] = {
	C32e(0xf4a5f497), C32e(0x978497eb), C32e(0xb099b0c7), C32e(0x8c8d8cf7),
	C32e(0x170d17e5), C32e(0xdcbddcb7), C32e(0xc8b1c8a7), C32e(0xfc54fc39),
	C32e(0xf050f0c0), C32e(0x05030504), C32e(0xe0a9e087), C32e(0x877d87ac),
	C32e(0x2b192bd5), C32e(0xa662a671), C32e(0x31e6319a), C32e(0xb59ab5c3),
	C32e(0xcf45cf05), C32e(0xbc9dbc3e), C32e(0xc040c009), C32e(0x928792ef),
	C32e(0x3f153fc5), C32e(0x26eb267f), C32e(0x40c94007), C32e(0x1d0b1ded),
	C32e(0x2fec2f82), C32e(0xa967a97d), C32e(0x1cfd1cbe), C32e(0x25ea258a),
	C32e(0xdabfda46), C32e(0x02f702a6), C32e(0xa196a1d3), C32e(0xed5bed2d),
	C32e(0x5dc25dea), C32e(0x241c24d9), C32e(0xe9aee97a), C32e(0xbe6abe98),
	C32e(0xee5aeed8), C32e(0xc341c3fc), C32e(0x060206f1), C32e(0xd14fd11d),
	C32e(0xe45ce4d0), C32e(0x07f407a2), C32e(0x5c345cb9), C32e(0x180818e9),
	C32e(0xae93aedf), C32e(0x9573954d), C32e(0xf553f5c4), C32e(0x413f4154),
	C32e(0x140c1410), C32e(0xf652f631), C32e(0xaf65af8c), C32e(0xe25ee221),
	C32e(0x78287860), C32e(0xf8a1f86e), C32e(0x110f1114), C32e(0xc4b5c45e),
	C32e(0x1b091b1c), C32e(0x5a365a48), C32e(0xb69bb636), C32e(0x473d47a5),
	C32e(0x6a266a81), C32e(0xbb69bb9c), C32e(0x4ccd4cfe), C32e(0xba9fbacf),
	C32e(0x2d1b2d24), C32e(0xb99eb93a), C32e(0x9c749cb0), C32e(0x722e7268),
	C32e(0x772d776c), C32e(0xcdb2cda3), C32e(0x29ee2973), C32e(0x16fb16b6),
	C32e(0x01f60153), C32e(0xd74dd7ec), C32e(0xa361a375), C32e(0x49ce49fa),
	C32e(0x8d7b8da4), C32e(0x423e42a1), C32e(0x937193bc), C32e(0xa297a226),
	C32e(0x04f50457), C32e(0xb868b869), C32e(0x00000000), C32e(0x742c7499),
	C32e(0xa060a080), C32e(0x211f21dd), C32e(0x43c843f2), C32e(0x2ced2c77),
	C32e(0xd9bed9b3), C32e(0xca46ca01), C32e(0x70d970ce), C32e(0xdd4bdde4),
	C32e(0x79de7933), C32e(0x67d4672b), C32e(0x23e8237b), C32e(0xde4ade11),
	C32e(0xbd6bbd6d), C32e(0x7e2a7e91), C32e(0x34e5349e), C32e(0x3a163ac1),
	C32e(0x54c55417), C32e(0x62d7622f), C32e(0xff55ffcc), C32e(0xa794a722),
	C32e(0x4acf4a0f), C32e(0x301030c9), C32e(0x0a060a08), C32e(0x988198e7),
	C32e(0x0bf00b5b), C32e(0xcc44ccf0), C32e(0xd5bad54a), C32e(0x3ee33e96),
	C32e(0x0ef30e5f), C32e(0x19fe19ba), C32e(0x5bc05b1b), C32e(0x858a850a),
	C32e(0xecadec7e), C32e(0xdfbcdf42), C32e(0xd848d8e0), C32e(0x0c040cf9),
	C32e(0x7adf7ac6), C32e(0x58c158ee), C32e(0x9f759f45), C32e(0xa563a584),
	C32e(0x50305040), C32e(0x2e1a2ed1), C32e(0x120e12e1), C32e(0xb76db765),
	C32e(0xd44cd419), C32e(0x3c143c30), C32e(0x5f355f4c), C32e(0x712f719d),
	C32e(0x38e13867), C32e(0xfda2fd6a), C32e(0x4fcc4f0b), C32e(0x4b394b5c),
	C32e(0xf957f93d), C32e(0x0df20daa), C32e(0x9d829de3), C32e(0xc947c9f4),
	C32e(0xefacef8b), C32e(0x32e7326f), C32e(0x7d2b7d64), C32e(0xa495a4d7),
	C32e(0xfba0fb9b), C32e(0xb398b332), C32e(0x68d16827), C32e(0x817f815d),
	C32e(0xaa66aa88), C32e(0x827e82a8), C32e(0xe6abe676), C32e(0x9e839e16),
	C32e(0x45ca4503), C32e(0x7b297b95), C32e(0x6ed36ed6), C32e(0x443c4450),
	C32e(0x8b798b55), C32e(0x3de23d63), C32e(0x271d272c), C32e(0x9a769a41),
	C32e(0x4d3b4dad), C32e(0xfa56fac8), C32e(0xd24ed2e8), C32e(0x221e2228),
	C32e(0x76db763f), C32e(0x1e0a1e18), C32e(0xb46cb490), C32e(0x37e4376b),
	C32e(0xe75de725), C32e(0xb26eb261), C32e(0x2aef2a86), C32e(0xf1a6f193),
	C32e(0xe3a8e372), C32e(0xf7a4f762), C32e(0x593759bd), C32e(0x868b86ff),
	C32e(0x563256b1), C32e(0xc543c50d), C32e(0xeb59ebdc), C32e(0xc2b7c2af),
	C32e(0x8f8c8f02), C32e(0xac64ac79), C32e(0x6dd26d23), C32e(0x3be03b92),
	C32e(0xc7b4c7ab), C32e(0x15fa1543), C32e(0x090709fd), C32e(0x6f256f85),
	C32e(0xeaafea8f), C32e(0x898e89f3), C32e(0x20e9208e), C32e(0x28182820),
	C32e(0x64d564de), C32e(0x838883fb), C32e(0xb16fb194), C32e(0x967296b8),
	C32e(0x6c246c70), C32e(0x08f108ae), C32e(0x52c752e6), C32e(0xf351f335),
	C32e(0x6523658d), C32e(0x847c8459), C32e(0xbf9cbfcb), C32e(0x6321637c),
	C32e(0x7cdd7c37), C32e(0x7fdc7fc2), C32e(0x9186911a), C32e(0x9485941e),
	C32e(0xab90abdb), C32e(0xc642c6f8), C32e(0x57c457e2), C32e(0xe5aae583),
	C32e(0x73d8733b), C32e(0x0f050f0c), C32e(0x030103f5), C32e(0x36123638),
	C32e(0xfea3fe9f), C32e(0xe15fe1d4), C32e(0x10f91047), C32e(0x6bd06bd2),
	C32e(0xa891a82e), C32e(0xe858e829), C32e(0x69276974), C32e(0xd0b9d04e),
	C32e(0x483848a9), C32e(0x351335cd), C32e(0xceb3ce56), C32e(0x55335544),
	C32e(0xd6bbd6bf), C32e(0x90709049), C32e(0x8089800e), C32e(0xf2a7f266),
	C32e(0xc1b6c15a), C32e(0x66226678), C32e(0xad92ad2a), C32e(0x60206089),
	C32e(0xdb49db15), C32e(0x1aff1a4f), C32e(0x887888a0), C32e(0x8e7a8e51),
	C32e(0x8a8f8a06), C32e(0x13f813b2), C32e(0x9b809b12), C32e(0x39173934),
	C32e(0x75da75ca), C32e(0x533153b5), C32e(0x51c65113), C32e(0xd3b8d3bb),
	C32e(0x5ec35e1f), C32e(0xcbb0cb52), C32e(0x997799b4), C32e(0x3311333c),
	C32e(0x46cb46f6), C32e(0x1ffc1f4b), C32e(0x61d661da), C32e(0x4e3a4e58)
};

uint32_t T3up_cpu[] = {
	C32e(0x97a5c6c6), C32e(0xeb84f8f8), C32e(0xc799eeee), C32e(0xf78df6f6),
	C32e(0xe50dffff), C32e(0xb7bdd6d6), C32e(0xa7b1dede), C32e(0x39549191),
	C32e(0xc0506060), C32e(0x04030202), C32e(0x87a9cece), C32e(0xac7d5656),
	C32e(0xd519e7e7), C32e(0x7162b5b5), C32e(0x9ae64d4d), C32e(0xc39aecec),
	C32e(0x05458f8f), C32e(0x3e9d1f1f), C32e(0x09408989), C32e(0xef87fafa),
	C32e(0xc515efef), C32e(0x7febb2b2), C32e(0x07c98e8e), C32e(0xed0bfbfb),
	C32e(0x82ec4141), C32e(0x7d67b3b3), C32e(0xbefd5f5f), C32e(0x8aea4545),
	C32e(0x46bf2323), C32e(0xa6f75353), C32e(0xd396e4e4), C32e(0x2d5b9b9b),
	C32e(0xeac27575), C32e(0xd91ce1e1), C32e(0x7aae3d3d), C32e(0x986a4c4c),
	C32e(0xd85a6c6c), C32e(0xfc417e7e), C32e(0xf102f5f5), C32e(0x1d4f8383),
	C32e(0xd05c6868), C32e(0xa2f45151), C32e(0xb934d1d1), C32e(0xe908f9f9),
	C32e(0xdf93e2e2), C32e(0x4d73abab), C32e(0xc4536262), C32e(0x543f2a2a),
	C32e(0x100c0808), C32e(0x31529595), C32e(0x8c654646), C32e(0x215e9d9d),
	C32e(0x60283030), C32e(0x6ea13737), C32e(0x140f0a0a), C32e(0x5eb52f2f),
	C32e(0x1c090e0e), C32e(0x48362424), C32e(0x369b1b1b), C32e(0xa53ddfdf),
	C32e(0x8126cdcd), C32e(0x9c694e4e), C32e(0xfecd7f7f), C32e(0xcf9feaea),
	C32e(0x241b1212), C32e(0x3a9e1d1d), C32e(0xb0745858), C32e(0x682e3434),
	C32e(0x6c2d3636), C32e(0xa3b2dcdc), C32e(0x73eeb4b4), C32e(0xb6fb5b5b),
	C32e(0x53f6a4a4), C32e(0xec4d7676), C32e(0x7561b7b7), C32e(0xface7d7d),
	C32e(0xa47b5252), C32e(0xa13edddd), C32e(0xbc715e5e), C32e(0x26971313),
	C32e(0x57f5a6a6), C32e(0x6968b9b9), C32e(0x00000000), C32e(0x992cc1c1),
	C32e(0x80604040), C32e(0xdd1fe3e3), C32e(0xf2c87979), C32e(0x77edb6b6),
	C32e(0xb3bed4d4), C32e(0x01468d8d), C32e(0xced96767), C32e(0xe44b7272),
	C32e(0x33de9494), C32e(0x2bd49898), C32e(0x7be8b0b0), C32e(0x114a8585),
	C32e(0x6d6bbbbb), C32e(0x912ac5c5), C32e(0x9ee54f4f), C32e(0xc116eded),
	C32e(0x17c58686), C32e(0x2fd79a9a), C32e(0xcc556666), C32e(0x22941111),
	C32e(0x0fcf8a8a), C32e(0xc910e9e9), C32e(0x08060404), C32e(0xe781fefe),
	C32e(0x5bf0a0a0), C32e(0xf0447878), C32e(0x4aba2525), C32e(0x96e34b4b),
	C32e(0x5ff3a2a2), C32e(0xbafe5d5d), C32e(0x1bc08080), C32e(0x0a8a0505),
	C32e(0x7ead3f3f), C32e(0x42bc2121), C32e(0xe0487070), C32e(0xf904f1f1),
	C32e(0xc6df6363), C32e(0xeec17777), C32e(0x4575afaf), C32e(0x84634242),
	C32e(0x40302020), C32e(0xd11ae5e5), C32e(0xe10efdfd), C32e(0x656dbfbf),
	C32e(0x194c8181), C32e(0x30141818), C32e(0x4c352626), C32e(0x9d2fc3c3),
	C32e(0x67e1bebe), C32e(0x6aa23535), C32e(0x0bcc8888), C32e(0x5c392e2e),
	C32e(0x3d579393), C32e(0xaaf25555), C32e(0xe382fcfc), C32e(0xf4477a7a),
	C32e(0x8bacc8c8), C32e(0x6fe7baba), C32e(0x642b3232), C32e(0xd795e6e6),
	C32e(0x9ba0c0c0), C32e(0x32981919), C32e(0x27d19e9e), C32e(0x5d7fa3a3),
	C32e(0x88664444), C32e(0xa87e5454), C32e(0x76ab3b3b), C32e(0x16830b0b),
	C32e(0x03ca8c8c), C32e(0x9529c7c7), C32e(0xd6d36b6b), C32e(0x503c2828),
	C32e(0x5579a7a7), C32e(0x63e2bcbc), C32e(0x2c1d1616), C32e(0x4176adad),
	C32e(0xad3bdbdb), C32e(0xc8566464), C32e(0xe84e7474), C32e(0x281e1414),
	C32e(0x3fdb9292), C32e(0x180a0c0c), C32e(0x906c4848), C32e(0x6be4b8b8),
	C32e(0x255d9f9f), C32e(0x616ebdbd), C32e(0x86ef4343), C32e(0x93a6c4c4),
	C32e(0x72a83939), C32e(0x62a43131), C32e(0xbd37d3d3), C32e(0xff8bf2f2),
	C32e(0xb132d5d5), C32e(0x0d438b8b), C32e(0xdc596e6e), C32e(0xafb7dada),
	C32e(0x028c0101), C32e(0x7964b1b1), C32e(0x23d29c9c), C32e(0x92e04949),
	C32e(0xabb4d8d8), C32e(0x43faacac), C32e(0xfd07f3f3), C32e(0x8525cfcf),
	C32e(0x8fafcaca), C32e(0xf38ef4f4), C32e(0x8ee94747), C32e(0x20181010),
	C32e(0xded56f6f), C32e(0xfb88f0f0), C32e(0x946f4a4a), C32e(0xb8725c5c),
	C32e(0x70243838), C32e(0xaef15757), C32e(0xe6c77373), C32e(0x35519797),
	C32e(0x8d23cbcb), C32e(0x597ca1a1), C32e(0xcb9ce8e8), C32e(0x7c213e3e),
	C32e(0x37dd9696), C32e(0xc2dc6161), C32e(0x1a860d0d), C32e(0x1e850f0f),
	C32e(0xdb90e0e0), C32e(0xf8427c7c), C32e(0xe2c47171), C32e(0x83aacccc),
	C32e(0x3bd89090), C32e(0x0c050606), C32e(0xf501f7f7), C32e(0x38121c1c),
	C32e(0x9fa3c2c2), C32e(0xd45f6a6a), C32e(0x47f9aeae), C32e(0xd2d06969),
	C32e(0x2e911717), C32e(0x29589999), C32e(0x74273a3a), C32e(0x4eb92727),
	C32e(0xa938d9d9), C32e(0xcd13ebeb), C32e(0x56b32b2b), C32e(0x44332222),
	C32e(0xbfbbd2d2), C32e(0x4970a9a9), C32e(0x0e890707), C32e(0x66a73333),
	C32e(0x5ab62d2d), C32e(0x78223c3c), C32e(0x2a921515), C32e(0x8920c9c9),
	C32e(0x15498787), C32e(0x4fffaaaa), C32e(0xa0785050), C32e(0x517aa5a5),
	C32e(0x068f0303), C32e(0xb2f85959), C32e(0x12800909), C32e(0x34171a1a),
	C32e(0xcada6565), C32e(0xb531d7d7), C32e(0x13c68484), C32e(0xbbb8d0d0),
	C32e(0x1fc38282), C32e(0x52b02929), C32e(0xb4775a5a), C32e(0x3c111e1e),
	C32e(0xf6cb7b7b), C32e(0x4bfca8a8), C32e(0xdad66d6d), C32e(0x583a2c2c)
};

uint32_t T3dn_cpu[] = {
	C32e(0x32f4a5f4), C32e(0x6f978497), C32e(0x5eb099b0), C32e(0x7a8c8d8c),
	C32e(0xe8170d17), C32e(0x0adcbddc), C32e(0x16c8b1c8), C32e(0x6dfc54fc),
	C32e(0x90f050f0), C32e(0x07050305), C32e(0x2ee0a9e0), C32e(0xd1877d87),
	C32e(0xcc2b192b), C32e(0x13a662a6), C32e(0x7c31e631), C32e(0x59b59ab5),
	C32e(0x40cf45cf), C32e(0xa3bc9dbc), C32e(0x49c040c0), C32e(0x68928792),
	C32e(0xd03f153f), C32e(0x9426eb26), C32e(0xce40c940), C32e(0xe61d0b1d),
	C32e(0x6e2fec2f), C32e(0x1aa967a9), C32e(0x431cfd1c), C32e(0x6025ea25),
	C32e(0xf9dabfda), C32e(0x5102f702), C32e(0x45a196a1), C32e(0x76ed5bed),
	C32e(0x285dc25d), C32e(0xc5241c24), C32e(0xd4e9aee9), C32e(0xf2be6abe),
	C32e(0x82ee5aee), C32e(0xbdc341c3), C32e(0xf3060206), C32e(0x52d14fd1),
	C32e(0x8ce45ce4), C32e(0x5607f407), C32e(0x8d5c345c), C32e(0xe1180818),
	C32e(0x4cae93ae), C32e(0x3e957395), C32e(0x97f553f5), C32e(0x6b413f41),
	C32e(0x1c140c14), C32e(0x63f652f6), C32e(0xe9af65af), C32e(0x7fe25ee2),
	C32e(0x48782878), C32e(0xcff8a1f8), C32e(0x1b110f11), C32e(0xebc4b5c4),
	C32e(0x151b091b), C32e(0x7e5a365a), C32e(0xadb69bb6), C32e(0x98473d47),
	C32e(0xa76a266a), C32e(0xf5bb69bb), C32e(0x334ccd4c), C32e(0x50ba9fba),
	C32e(0x3f2d1b2d), C32e(0xa4b99eb9), C32e(0xc49c749c), C32e(0x46722e72),
	C32e(0x41772d77), C32e(0x11cdb2cd), C32e(0x9d29ee29), C32e(0x4d16fb16),
	C32e(0xa501f601), C32e(0xa1d74dd7), C32e(0x14a361a3), C32e(0x3449ce49),
	C32e(0xdf8d7b8d), C32e(0x9f423e42), C32e(0xcd937193), C32e(0xb1a297a2),
	C32e(0xa204f504), C32e(0x01b868b8), C32e(0x00000000), C32e(0xb5742c74),
	C32e(0xe0a060a0), C32e(0xc2211f21), C32e(0x3a43c843), C32e(0x9a2ced2c),
	C32e(0x0dd9bed9), C32e(0x47ca46ca), C32e(0x1770d970), C32e(0xafdd4bdd),
	C32e(0xed79de79), C32e(0xff67d467), C32e(0x9323e823), C32e(0x5bde4ade),
	C32e(0x06bd6bbd), C32e(0xbb7e2a7e), C32e(0x7b34e534), C32e(0xd73a163a),
	C32e(0xd254c554), C32e(0xf862d762), C32e(0x99ff55ff), C32e(0xb6a794a7),
	C32e(0xc04acf4a), C32e(0xd9301030), C32e(0x0e0a060a), C32e(0x66988198),
	C32e(0xab0bf00b), C32e(0xb4cc44cc), C32e(0xf0d5bad5), C32e(0x753ee33e),
	C32e(0xac0ef30e), C32e(0x4419fe19), C32e(0xdb5bc05b), C32e(0x80858a85),
	C32e(0xd3ecadec), C32e(0xfedfbcdf), C32e(0xa8d848d8), C32e(0xfd0c040c),
	C32e(0x197adf7a), C32e(0x2f58c158), C32e(0x309f759f), C32e(0xe7a563a5),
	C32e(0x70503050), C32e(0xcb2e1a2e), C32e(0xef120e12), C32e(0x08b76db7),
	C32e(0x55d44cd4), C32e(0x243c143c), C32e(0x795f355f), C32e(0xb2712f71),
	C32e(0x8638e138), C32e(0xc8fda2fd), C32e(0xc74fcc4f), C32e(0x654b394b),
	C32e(0x6af957f9), C32e(0x580df20d), C32e(0x619d829d), C32e(0xb3c947c9),
	C32e(0x27efacef), C32e(0x8832e732), C32e(0x4f7d2b7d), C32e(0x42a495a4),
	C32e(0x3bfba0fb), C32e(0xaab398b3), C32e(0xf668d168), C32e(0x22817f81),
	C32e(0xeeaa66aa), C32e(0xd6827e82), C32e(0xdde6abe6), C32e(0x959e839e),
	C32e(0xc945ca45), C32e(0xbc7b297b), C32e(0x056ed36e), C32e(0x6c443c44),
	C32e(0x2c8b798b), C32e(0x813de23d), C32e(0x31271d27), C32e(0x379a769a),
	C32e(0x964d3b4d), C32e(0x9efa56fa), C32e(0xa6d24ed2), C32e(0x36221e22),
	C32e(0xe476db76), C32e(0x121e0a1e), C32e(0xfcb46cb4), C32e(0x8f37e437),
	C32e(0x78e75de7), C32e(0x0fb26eb2), C32e(0x692aef2a), C32e(0x35f1a6f1),
	C32e(0xdae3a8e3), C32e(0xc6f7a4f7), C32e(0x8a593759), C32e(0x74868b86),
	C32e(0x83563256), C32e(0x4ec543c5), C32e(0x85eb59eb), C32e(0x18c2b7c2),
	C32e(0x8e8f8c8f), C32e(0x1dac64ac), C32e(0xf16dd26d), C32e(0x723be03b),
	C32e(0x1fc7b4c7), C32e(0xb915fa15), C32e(0xfa090709), C32e(0xa06f256f),
	C32e(0x20eaafea), C32e(0x7d898e89), C32e(0x6720e920), C32e(0x38281828),
	C32e(0x0b64d564), C32e(0x73838883), C32e(0xfbb16fb1), C32e(0xca967296),
	C32e(0x546c246c), C32e(0x5f08f108), C32e(0x2152c752), C32e(0x64f351f3),
	C32e(0xae652365), C32e(0x25847c84), C32e(0x57bf9cbf), C32e(0x5d632163),
	C32e(0xea7cdd7c), C32e(0x1e7fdc7f), C32e(0x9c918691), C32e(0x9b948594),
	C32e(0x4bab90ab), C32e(0xbac642c6), C32e(0x2657c457), C32e(0x29e5aae5),
	C32e(0xe373d873), C32e(0x090f050f), C32e(0xf4030103), C32e(0x2a361236),
	C32e(0x3cfea3fe), C32e(0x8be15fe1), C32e(0xbe10f910), C32e(0x026bd06b),
	C32e(0xbfa891a8), C32e(0x71e858e8), C32e(0x53692769), C32e(0xf7d0b9d0),
	C32e(0x91483848), C32e(0xde351335), C32e(0xe5ceb3ce), C32e(0x77553355),
	C32e(0x04d6bbd6), C32e(0x39907090), C32e(0x87808980), C32e(0xc1f2a7f2),
	C32e(0xecc1b6c1), C32e(0x5a662266), C32e(0xb8ad92ad), C32e(0xa9602060),
	C32e(0x5cdb49db), C32e(0xb01aff1a), C32e(0xd8887888), C32e(0x2b8e7a8e),
	C32e(0x898a8f8a), C32e(0x4a13f813), C32e(0x929b809b), C32e(0x23391739),
	C32e(0x1075da75), C32e(0x84533153), C32e(0xd551c651), C32e(0x03d3b8d3),
	C32e(0xdc5ec35e), C32e(0xe2cbb0cb), C32e(0xc3997799), C32e(0x2d331133),
	C32e(0x3d46cb46), C32e(0xb71ffc1f), C32e(0x0c61d661), C32e(0x624e3a4e)
};

__device__ void groestl512_perm_P(uint32_t *a)
{
	uint32_t t[32];

//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
#pragma unroll 16
		for(int k=0;k<16;k++)
		{
			a[(k*2)+0] ^= PC32up(k * 0x10, r);
			//a[(k<<1)+1] ^= PC32dn(k * 0x10, r);
		}

		// RBTT
#pragma unroll 16
		for(int k=0;k<32;k+=2)
		{
			t[k + 0] =	T0up( B32_0(a[k & 0x1f]) ) ^
						T1up( B32_1(a[(k + 2) & 0x1f]) ) ^
						T2up( B32_2(a[(k + 4) & 0x1f]) ) ^
						T3up( B32_3(a[(k + 6) & 0x1f]) ) ^
						T0dn( B32_0(a[(k + 9) & 0x1f]) ) ^
						T1dn( B32_1(a[(k + 11) & 0x1f]) ) ^
						T2dn( B32_2(a[(k + 13) & 0x1f]) ) ^
						T3dn( B32_3(a[(k + 23) & 0x1f]) );

			t[k + 1] =	T0dn( B32_0(a[k & 0x1f]) ) ^
						T1dn( B32_1(a[(k + 2) & 0x1f]) ) ^
						T2dn( B32_2(a[(k + 4) & 0x1f]) ) ^
						T3dn( B32_3(a[(k + 6) & 0x1f]) ) ^
						T0up( B32_0(a[(k + 9) & 0x1f]) ) ^
						T1up( B32_1(a[(k + 11) & 0x1f]) ) ^
						T2up( B32_2(a[(k + 13) & 0x1f]) ) ^
						T3up( B32_3(a[(k + 23) & 0x1f]) );
		}
#pragma unroll 32
		for(int k=0;k<32;k++)
			a[k] = t[k];
	}
}

__device__ void groestl512_perm_Q(uint32_t *a)
{
//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		uint32_t t[32];

#pragma unroll 16
		for(int k=0;k<16;k++)
		{
			a[(k*2)+0] ^= QC32up(k * 0x10, r);
			a[(k*2)+1] ^= QC32dn(k * 0x10, r);
		}

		// RBTT
#pragma unroll 16
		for(int k=0;k<32;k+=2)
		{
			t[k + 0] =	T0up( B32_0(a[(k + 2) & 0x1f]) ) ^
						T1up( B32_1(a[(k + 6) & 0x1f]) ) ^
						T2up( B32_2(a[(k + 10) & 0x1f]) ) ^
						T3up( B32_3(a[(k + 22) & 0x1f]) ) ^
						T0dn( B32_0(a[(k + 1) & 0x1f]) ) ^
						T1dn( B32_1(a[(k + 5) & 0x1f]) ) ^
						T2dn( B32_2(a[(k + 9) & 0x1f]) ) ^
						T3dn( B32_3(a[(k + 13) & 0x1f]) );

			t[k + 1] =	T0dn( B32_0(a[(k + 2) & 0x1f]) ) ^
						T1dn( B32_1(a[(k + 6) & 0x1f]) ) ^
						T2dn( B32_2(a[(k + 10) & 0x1f]) ) ^
						T3dn( B32_3(a[(k + 22) & 0x1f]) ) ^
						T0up( B32_0(a[(k + 1) & 0x1f]) ) ^
						T1up( B32_1(a[(k + 5) & 0x1f]) ) ^
						T2up( B32_2(a[(k + 9) & 0x1f]) ) ^
						T3up( B32_3(a[(k + 13) & 0x1f]) );
		}
#pragma unroll 32
		for(int k=0;k<32;k++)
			a[k] = t[k];
	}
}

template <int BLOCKSIZE> __global__ void groestl512_gpu_hash(uint32_t threads, uint32_t startNounce, void *outputHash, uint32_t *heftyHashes, uint32_t *nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t message[32];
		uint32_t state[32];

		// lese message ein & verknüpfe diese mit dem hash1 von hefty1
		// lese den state ein

#pragma unroll 32
		for(int k=0;k<32;k++)
		{
			state[k] = groestl_gpu_state[k];
			message[k] = groestl_gpu_msg[k];
		}

		uint32_t nounce = nonceVector[thread];
		// nounce setzen
		//message[19] = startNounce + thread;
		message[19] = nounce;

		uint32_t hashPosition = nounce - startNounce;

		// den richtigen Hefty1 Hash holen
//			memcpy(&message[21], &heftyHashes[8 * hashPosition], sizeof(uint32_t) * 8);
		uint32_t *heftyHash = &heftyHashes[8 * hashPosition];
#pragma unroll 8
		for (int k=0; k<8; ++k)
			message[BLOCKSIZE/4+k] = heftyHash[k];

		uint32_t g[32];
#pragma unroll 32
		for(int u=0;u<32;u++)
			g[u] = message[u] ^ state[u];

		// Perm
		groestl512_perm_P(g);
		groestl512_perm_Q(message);

#pragma unroll 32
		for(int u=0;u<32;u++)
		{
			state[u] ^= g[u] ^ message[u];
			g[u] = state[u];
		}

		groestl512_perm_P(g);

#pragma unroll 32
		for(int u=0;u<32;u++)
			state[u] ^= g[u];

		// kopiere Ergebnis
#pragma unroll 16
		for(int k=0;k<16;k++)
			((uint32_t*)outputHash)[16*hashPosition+k] = state[k + 16];
	}
}

#define texDef(id, texname, texmem, texsource, texsize) { \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	d_textures[thr_id][id] = texmem; \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); \
	} \
}

// Setup Function
__host__
void groestl512_cpu_init(int thr_id, uint32_t threads)
{
	// Texturen mit obigem Makro initialisieren
	texDef(0, t0up, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
	texDef(1, t0dn, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
	texDef(2, t1up, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
	texDef(3, t1dn, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
	texDef(4, t2up, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
	texDef(5, t2dn, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
	texDef(6, t3up, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
	texDef(7, t3dn, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);

	// Speicher für alle Ergebnisse belegen
	hipMalloc(&d_hash4output[thr_id], (size_t) 64 * threads);
}

__host__
void groestl512_cpu_free(int thr_id)
{
	for (int i=0; i <8; i++)
		hipFree(d_textures[thr_id][i]);

	hipFree(d_hash4output[thr_id]);
}

static int BLOCKSIZE = 84;

__host__
void groestl512_cpu_setBlock(void *data, int len)
	// data muss 80/84-Byte haben!
	// heftyHash hat 32-Byte
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, len);

	// Erweitere die Nachricht auf den Nachrichtenblock (padding)
	// Unsere Nachricht hat 112/116 Byte
	if (len == 84) {
		msgBlock[29] = 0x80;
		msgBlock[31] = 0x01000000;
	} else if (len == 80) {
		msgBlock[28] = 0x80;
		msgBlock[31] = 0x01000000;
	}
	// groestl512 braucht hierfür keinen CPU-Code (die einzige Runde wird
	// auf der GPU ausgeführt)

	// setze register
	uint32_t groestl_state_init[32];
	memset(groestl_state_init, 0, sizeof(uint32_t) * 32);
	groestl_state_init[31] = 0x20000;

	// state speichern
	hipMemcpyToSymbol(HIP_SYMBOL(groestl_gpu_state), groestl_state_init, 128);

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(HIP_SYMBOL(groestl_gpu_msg), msgBlock, 128);
	BLOCKSIZE = len;
}

__host__ void groestl512_cpu_copyHeftyHash(int thr_id, uint32_t threads, void *heftyHashes, int copy)
{
	// Hefty1 Hashes kopieren (eigentlich nur zum debuggen)
	if (copy)
		CUDA_SAFE_CALL(hipMemcpy(heavy_heftyHashes[thr_id], heftyHashes, 8 * sizeof(uint32_t) * threads, hipMemcpyHostToDevice));
}

__host__ void groestl512_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce)
{
	const uint32_t threadsperblock = 128;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Größe des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	if (BLOCKSIZE == 84)
		groestl512_gpu_hash<84><<<grid, block, shared_size>>>(threads, startNounce, d_hash4output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
	else if (BLOCKSIZE == 80)
		groestl512_gpu_hash<80><<<grid, block, shared_size>>>(threads, startNounce, d_hash4output[thr_id], heavy_heftyHashes[thr_id], heavy_nonceVector[thr_id]);
}
