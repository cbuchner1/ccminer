#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "miner.h"

#include "cuda_helper.h"

#define USE_SHARED 1

// globaler Speicher für alle HeftyHashes aller Threads
uint32_t *heavy_heftyHashes[MAX_GPUS];

/* Hash-Tabellen */
__constant__ uint32_t hefty_gpu_constantTable[64];
#if USE_SHARED
#define heftyLookUp(x) (*((uint32_t*)heftytab + (x)))
#else
#define heftyLookUp(x) hefty_gpu_constantTable[x]
#endif

// muss expandiert werden
__constant__ uint32_t hefty_gpu_blockHeader[16]; // 2x512 Bit Message
__constant__ uint32_t hefty_gpu_register[8];
__constant__ uint32_t hefty_gpu_sponge[4];

uint32_t hefty_cpu_hashTable[] = {
    0x6a09e667UL,
    0xbb67ae85UL,
    0x3c6ef372UL,
    0xa54ff53aUL,
    0x510e527fUL,
    0x9b05688cUL,
    0x1f83d9abUL,
    0x5be0cd19UL
};

uint32_t hefty_cpu_constantTable[] = {
    0x428a2f98UL, 0x71374491UL, 0xb5c0fbcfUL, 0xe9b5dba5UL,
    0x3956c25bUL, 0x59f111f1UL, 0x923f82a4UL, 0xab1c5ed5UL,
    0xd807aa98UL, 0x12835b01UL, 0x243185beUL, 0x550c7dc3UL,
    0x72be5d74UL, 0x80deb1feUL, 0x9bdc06a7UL, 0xc19bf174UL,
    0xe49b69c1UL, 0xefbe4786UL, 0x0fc19dc6UL, 0x240ca1ccUL,
    0x2de92c6fUL, 0x4a7484aaUL, 0x5cb0a9dcUL, 0x76f988daUL,
    0x983e5152UL, 0xa831c66dUL, 0xb00327c8UL, 0xbf597fc7UL,
    0xc6e00bf3UL, 0xd5a79147UL, 0x06ca6351UL, 0x14292967UL,
    0x27b70a85UL, 0x2e1b2138UL, 0x4d2c6dfcUL, 0x53380d13UL,
    0x650a7354UL, 0x766a0abbUL, 0x81c2c92eUL, 0x92722c85UL,
    0xa2bfe8a1UL, 0xa81a664bUL, 0xc24b8b70UL, 0xc76c51a3UL,
    0xd192e819UL, 0xd6990624UL, 0xf40e3585UL, 0x106aa070UL,
    0x19a4c116UL, 0x1e376c08UL, 0x2748774cUL, 0x34b0bcb5UL,
    0x391c0cb3UL, 0x4ed8aa4aUL, 0x5b9cca4fUL, 0x682e6ff3UL,
    0x748f82eeUL, 0x78a5636fUL, 0x84c87814UL, 0x8cc70208UL,
    0x90befffaUL, 0xa4506cebUL, 0xbef9a3f7UL, 0xc67178f2UL
};

#if 0
#define S(x, n)          (((x) >> (n)) | ((x) << (32 - (n))))
#else
__host__ __device__
static uint32_t S(uint32_t x, int n)
{
    return (((x) >> (n)) | ((x) << (32 - (n))));
}
#endif

#define R(x, n)          ((x) >> (n))
#define Ch(x, y, z)      ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)     ((x & (y | z)) | (y & z))
#define S0(x)            (S(x, 2) ^ S(x, 13) ^ S(x, 22))
#define S1(x)            (S(x, 6) ^ S(x, 11) ^ S(x, 25))
#define s0(x)            (S(x, 7) ^ S(x, 18) ^ R(x, 3))
#define s1(x)            (S(x, 17) ^ S(x, 19) ^ R(x, 10))

#define SWAB32(x)        ( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

// uint8_t
#define smoosh4(x)       ( ((x)>>4) ^ ((x) & 0x0F) )

__host__ __forceinline__ __device__
uint8_t smoosh2(uint32_t x)
{
    uint16_t w = (x >> 16) ^ (x & 0xffff);
    uint8_t n = smoosh4( (uint8_t)( (w >> 8) ^ (w & 0xFF) ) );
    return 24 - (((n >> 2) ^ (n & 0x03)) << 3);
}
// 4 auf einmal
#define smoosh4Quad(x)   ( (((x)>>4) ^ (x)) & 0x0F0F0F0F )
#define getByte(x,y)     ( ((x) >> (y)) & 0xFF )

__host__ __forceinline__ __device__
void Mangle(uint32_t *inp)
{
    uint32_t r = smoosh4Quad(inp[0]);
    uint32_t inp0org;
    uint32_t tmp0Mask, tmp1Mask;
    uint32_t in1, in2, isAddition;
    int32_t tmp;
    uint8_t b;

    inp[1] = inp[1] ^ S(inp[0], getByte(r, 24));

    r += 0x01010101;
    tmp = smoosh2(inp[1]);
    b = getByte(r,tmp);
    inp0org = S(inp[0], b);
    tmp0Mask = (uint32_t) -((tmp >> 3) & 1); // Bit 3 an Position 0
    tmp1Mask = (uint32_t) -((tmp >> 4) & 1); // Bit 4 an Position 0

    in1 =    (inp[2] & ~inp0org) |
            (tmp1Mask & ~inp[2] & inp0org) |
            (~tmp0Mask & ~inp[2] & inp0org);
    in2 = inp[2] += ~inp0org;
    isAddition = ~tmp0Mask & tmp1Mask;
    inp[2] = isAddition ? in2 : in1;

    r += 0x01010101;
    tmp = smoosh2(inp[1] ^ inp[2]);
    b = getByte(r,tmp);
    inp0org = S(inp[0], b);
    tmp0Mask = (uint32_t) -((tmp >> 3) & 1); // Bit 3 an Position 0
    tmp1Mask = (uint32_t) -((tmp >> 4) & 1); // Bit 4 an Position 0

    in1 =    (inp[3] & ~inp0org) |
            (tmp1Mask & ~inp[3] & inp0org) |
            (~tmp0Mask & ~inp[3] & inp0org);
    in2 = inp[3] += ~inp0org;
    isAddition = ~tmp0Mask & tmp1Mask;
    inp[3] = isAddition ? in2 : in1;

    inp[0] ^= (inp[1] ^ inp[2]) + inp[3];
}

__host__ __forceinline__ __device__
void Absorb(uint32_t *inp, uint32_t x)
{
    inp[0] ^= x;
    Mangle(inp);
}

__host__ __forceinline__ __device__
uint32_t Squeeze(uint32_t *inp)
{
    uint32_t y = inp[0];
    Mangle(inp);
    return y;
}

__host__ __forceinline__ __device__
uint32_t Br(uint32_t *sponge, uint32_t x)
{
    uint32_t r = Squeeze(sponge);
    uint32_t t = ((r >> 8) & 0x1F);
    uint32_t y = 1 << t;

    uint32_t a = (((r>>1) & 0x01) << t) & y;
    uint32_t b = ((r & 0x01) << t) & y;
    uint32_t c = x & y;

    uint32_t retVal = (x & ~y) | (~b & c) | (a & ~c);
    return retVal;
}

__device__ __forceinline__
void hefty_gpu_round(uint32_t *regs, uint32_t W, uint32_t K, uint32_t *sponge)
{
    uint32_t tmpBr;

    uint32_t brG = Br(sponge, regs[6]);
    uint32_t brF = Br(sponge, regs[5]);
    uint32_t tmp1 = Ch(regs[4], brF, brG) + regs[7] + W + K;
    uint32_t brE = Br(sponge, regs[4]);
    uint32_t tmp2 = tmp1 + S1(brE);
    uint32_t brC = Br(sponge, regs[2]);
    uint32_t brB = Br(sponge, regs[1]);
    uint32_t brA = Br(sponge, regs[0]);
    uint32_t tmp3 = Maj(brA, brB, brC);
    tmpBr = Br(sponge, regs[0]);
    uint32_t tmp4 = tmp3 + S0(tmpBr);
    tmpBr = Br(sponge, tmp2);

    #pragma unroll 7
    for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
    regs[0] = tmp2 + tmp4;
    regs[4] += tmpBr;
}

__host__
void hefty_cpu_round(uint32_t *regs, uint32_t W, uint32_t K, uint32_t *sponge)
{
    uint32_t tmpBr;

    uint32_t brG = Br(sponge, regs[6]);
    uint32_t brF = Br(sponge, regs[5]);
    uint32_t tmp1 = Ch(regs[4], brF, brG) + regs[7] + W + K;
    uint32_t brE = Br(sponge, regs[4]);
    uint32_t tmp2 = tmp1 + S1(brE);
    uint32_t brC = Br(sponge, regs[2]);
    uint32_t brB = Br(sponge, regs[1]);
    uint32_t brA = Br(sponge, regs[0]);
    uint32_t tmp3 = Maj(brA, brB, brC);
    tmpBr = Br(sponge, regs[0]);
    uint32_t tmp4 = tmp3 + S0(tmpBr);
    tmpBr = Br(sponge, tmp2);

    for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
    regs[0] = tmp2 + tmp4;
    regs[4] += tmpBr;
}

__global__
void hefty_gpu_hash(uint32_t threads, uint32_t startNounce, uint32_t *outputHash)
{
#if USE_SHARED
    extern __shared__ unsigned char heftytab[];
    if(threadIdx.x < 64)
    {
        *((uint32_t*)heftytab + threadIdx.x) = hefty_gpu_constantTable[threadIdx.x];
    }

    __syncthreads();
#endif

    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        // bestimme den aktuellen Zähler
        uint32_t nounce = startNounce + thread;

        // jeder thread in diesem  Block bekommt sein eigenes W Array im Shared memory
        // reduktion von 256 byte auf 128 byte
        uint32_t W1[16];
        uint32_t W2[16];

        // Initialisiere die register a bis h mit der Hash-Tabelle
        uint32_t regs[8];
        uint32_t hash[8];
        uint32_t sponge[4];

#pragma unroll 4
        for(int k=0; k < 4; k++)
            sponge[k] = hefty_gpu_sponge[k];

        // pre
#pragma unroll 8
        for (int k=0; k < 8; k++)
        {
            regs[k] = hefty_gpu_register[k];
            hash[k] = regs[k];
        }

        //memcpy(W, &hefty_gpu_blockHeader[0], sizeof(uint32_t) * 16); // verbleibende 20 bytes aus Block 2 plus padding
#pragma unroll 16
        for(int k=0;k<16;k++)
            W1[k] = hefty_gpu_blockHeader[k];
        W1[3] = SWAB32(nounce);

        // 2. Runde
#pragma unroll 16
        for(int j=0;j<16;j++)
            Absorb(sponge, W1[j] ^ heftyLookUp(j));

// Progress W1 (Bytes 0...63)
#pragma unroll 16
        for(int j=0;j<16;j++)
        {
            Absorb(sponge, regs[3] ^ regs[7]);
            hefty_gpu_round(regs, W1[j], heftyLookUp(j), sponge);
        }

// Progress W2 (Bytes 64...127) then W3 (Bytes 128...191) ...


        for(int k=0;k<3;k++)
        {

            for(int j=0;j<2;j++)
                W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];

            for(int j=2;j<7;j++)
                W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];


            for(int j=7;j<15;j++)
                W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

            W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];


            for(int j=0;j<16;j++)
            {
                Absorb(sponge, regs[3] + regs[7]);
                hefty_gpu_round(regs, W2[j], heftyLookUp(j + ((k+1)<<4)), sponge);
            }

            for(int j=0;j<16;j++)
                W1[j] = W2[j];
        }

#pragma unroll 8
        for(int k=0;k<8;k++)
            hash[k] += regs[k];

#pragma unroll 8
        for(int k=0;k<8;k++)
            ((uint32_t*)outputHash)[(thread<<3)+k] = SWAB32(hash[k]);
    }
}

__host__
void hefty_cpu_init(int thr_id, uint32_t threads)
{
    hipSetDevice(device_map[thr_id]);

    // Kopiere die Hash-Tabellen in den GPU-Speicher
    hipMemcpyToSymbol(HIP_SYMBOL( hefty_gpu_constantTable),
                        hefty_cpu_constantTable,
                        sizeof(uint32_t) * 64 );

    // Speicher für alle Hefty1 hashes belegen
    CUDA_SAFE_CALL(hipMalloc(&heavy_heftyHashes[thr_id], (size_t) 32 * threads));
}

__host__
void hefty_cpu_free(int thr_id)
{
    hipFree(heavy_heftyHashes[thr_id]);
}

__host__
void hefty_cpu_setBlock(int thr_id, uint32_t threads, void *data, int len)
// data muss 80/84-Byte haben!
{
    // Nachricht expandieren und setzen
    uint32_t msgBlock[32];

    memset(msgBlock, 0, sizeof(msgBlock));
    memcpy(&msgBlock[0], data, len);
    if (len == 84) {
        msgBlock[21] |= 0x80;
        msgBlock[31] = 672; // bitlen
    } else if (len == 80) {
        msgBlock[20] |= 0x80;
        msgBlock[31] = 640; // bitlen
    }

    for(int i=0;i<31;i++) // Byteorder drehen
        msgBlock[i] = SWAB32(msgBlock[i]);

    // die erste Runde wird auf der CPU durchgeführt, da diese für
    // alle Threads gleich ist. Der Hash wird dann an die Threads
    // übergeben

    // Erstelle expandierten Block W
    uint32_t W[64];
    memcpy(W, &msgBlock[0], sizeof(uint32_t) * 16);
    for(int j=16;j<64;j++)
        W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];

    // Initialisiere die register a bis h mit der Hash-Tabelle
    uint32_t regs[8];
    uint32_t hash[8];
    uint32_t sponge[4];

    // pre
    memset(sponge, 0, sizeof(uint32_t) * 4);
    for (int k=0; k < 8; k++)
    {
        regs[k] = hefty_cpu_hashTable[k];
        hash[k] = regs[k];
    }

    // 1. Runde
    for(int j=0;j<16;j++)
        Absorb(sponge, W[j] ^ hefty_cpu_constantTable[j]);

    for(int j=0;j<16;j++)
    {
        Absorb(sponge, regs[3] ^ regs[7]);
        hefty_cpu_round(regs, W[j], hefty_cpu_constantTable[j], sponge);
    }

    for(int j=16;j<64;j++)
    {
        Absorb(sponge, regs[3] + regs[7]);
        hefty_cpu_round(regs, W[j], hefty_cpu_constantTable[j], sponge);
    }

    for(int k=0;k<8;k++)
        hash[k] += regs[k];

    // sponge speichern
    hipMemcpyToSymbol(HIP_SYMBOL(hefty_gpu_sponge), sponge, 16);
    // hash speichern
    hipMemcpyToSymbol(HIP_SYMBOL(hefty_gpu_register), hash, 32);
    // Blockheader setzen (korrekte Nonce fehlt da drin noch)
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(hefty_gpu_blockHeader), &msgBlock[16], 64));
}

__host__
void hefty_cpu_hash(int thr_id, uint32_t threads, int startNounce)
{
    uint32_t threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Größe des dynamischen Shared Memory Bereichs
#if USE_SHARED
    int shared_size = 8 * 64 * sizeof(uint32_t);
#else
    int shared_size = 0;
#endif

    hefty_gpu_hash <<< grid, block, shared_size >>> (threads, startNounce, heavy_heftyHashes[thr_id]);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, 0, thr_id);
}

__global__
__launch_bounds__(128, 8)
void hefty_gpu_copy(const uint32_t threads, uint32_t* d_heftyhash, uint64_t* d_hash)
{
    const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        const uint32_t offset = thread * 8U; // 32 / sizeof(uint32_t);
        uint4 *psrc = (uint4*) (&d_heftyhash[offset]);
        uint4 *pdst = (uint4*) (&d_hash[offset]);
        pdst[0] = psrc[0];
        pdst[1] = psrc[1];
        pdst[2] = make_uint4(0,0,0,0);
        pdst[3] = make_uint4(0,0,0,0);
    }
}

__host__
void hefty_copy_hashes(int thr_id, uint32_t threads, uint32_t* d_outputhash)
{
    const uint32_t threadsperblock = 128;
    dim3 grid((threads + threadsperblock - 1) / threadsperblock);
    dim3 block(threadsperblock);
    hefty_gpu_copy <<< grid, block >>> (threads, heavy_heftyHashes[thr_id], (uint64_t*) d_outputhash);
    hipStreamSynchronize(NULL);
}

