#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#define __byte_perm(x,y,n) x
#endif

#include "miner.h"

__constant__ uint32_t pTarget[8]; // Single GPU
__constant__ uint32_t groestlcoin_gpu_msg[32];

static uint32_t *d_resultNonce[MAX_GPUS];

#if __CUDA_ARCH__ >= 300
// 64 Registers Variant for Compute 3.0+
#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"
#endif

#define SWAB32(x) cuda_swab32(x)

__global__ __launch_bounds__(256, 4)
void groestlcoin_gpu_hash_quad(uint32_t threads, uint32_t startNounce, uint32_t *resNounce)
{
#if __CUDA_ARCH__ >= 300
	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) / 4;
	if (thread < threads)
	{
		// GROESTL
		uint32_t paddedInput[8];

		#pragma unroll 8
		for(int k=0;k<8;k++) paddedInput[k] = groestlcoin_gpu_msg[4*k+threadIdx.x%4];

		uint32_t nounce = startNounce + thread;
		if ((threadIdx.x % 4) == 3)
			paddedInput[4] = SWAB32(nounce);  // 4*4+3 = 19

		uint32_t msgBitsliced[8];
		to_bitslice_quad(paddedInput, msgBitsliced);

		uint32_t state[8];
		for (int round=0; round<2; round++)
		{
			groestl512_progressMessage_quad(state, msgBitsliced);

			if (round < 1)
			{
				// Verkettung zweier Runden inclusive Padding.
				msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + ((threadIdx.x%4)==3)*0x2000);
				msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
				msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
				msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
				msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
				msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
				msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
				msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + ((threadIdx.x%4)==0)*0x0010);
			}
		}

		// Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
		uint32_t out_state[16];
		from_bitslice_quad(state, out_state);

		if (threadIdx.x % 4 == 0)
		{
			int i, position = -1;
			bool rc = true;

			#pragma unroll 8
			for (i = 7; i >= 0; i--) {
				if (out_state[i] > pTarget[i]) {
					if(position < i) {
						position = i;
						rc = false;
					}
				 }
				 if (out_state[i] < pTarget[i]) {
					if(position < i) {
						position = i;
						rc = true;
					}
				 }
			}

			if(rc && resNounce[0] > nounce)
				resNounce[0] = nounce;
		}
	}
#endif
}

__host__
void groestlcoin_cpu_init(int thr_id, uint32_t threads)
{
	// to check if the binary supports SM3+
	cuda_get_arch(thr_id);

	CUDA_SAFE_CALL(hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)));
}

__host__
void groestlcoin_cpu_free(int thr_id)
{
	hipFree(d_resultNonce[thr_id]);
}

__host__
void groestlcoin_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
	uint32_t msgBlock[32] = { 0 };

	memcpy(&msgBlock[0], data, 80);

	// Erweitere die Nachricht auf den Nachrichtenblock (padding)
	// Unsere Nachricht hat 80 Byte
	msgBlock[20] = 0x80;
	msgBlock[31] = 0x01000000;

	// groestl512 braucht hierfür keinen CPU-Code (die einzige Runde wird
	// auf der GPU ausgeführt)

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(HIP_SYMBOL(groestlcoin_gpu_msg), msgBlock, 128);

	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32);
}

__host__
void groestlcoin_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonce)
{
	uint32_t threadsperblock = 256;

	// Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
	// mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
	int factor = 4;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
	dim3 block(threadsperblock);

	int dev_id = device_map[thr_id];
	if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
		gpulog(LOG_ERR, thr_id, "Sorry, This algo is not supported by this GPU arch (SM 3.0 required)");
		proper_exit(EXIT_CODE_CUDA_ERROR);
	}

	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	groestlcoin_gpu_hash_quad <<<grid, block>>> (threads, startNounce, d_resultNonce[thr_id]);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	// MyStreamSynchronize(NULL, 0, thr_id);

	hipMemcpy(resNonce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
