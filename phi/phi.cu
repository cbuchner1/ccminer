#include "hip/hip_runtime.h"
//
//
//  PHI1612 algo
//  Skein + JH + CubeHash + Fugue + Gost + Echo
//
//  Implemented by anorganix @ bitcointalk on 01.10.2017
//  Feel free to send some satoshis to 1Bitcoin8tfbtGAQNFxDRUVUfFgFWKoWi9
//
//

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_fugue.h"
#include "sph/sph_streebog.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_hash, int swap);
extern void streebog_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void streebog_hash_64_maxwell(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_free(int thr_id);

extern void tribus_echo512_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target);

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

extern "C" void phi_hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_skein512_context ctx_skein;
	sph_jh512_context ctx_jh;
	sph_cubehash512_context ctx_cubehash;
	sph_fugue512_context ctx_fugue;
	sph_gost512_context ctx_gost;
	sph_echo512_context ctx_echo;

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*)hash, 64);
	sph_gost512_close(&ctx_gost, (void*)hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	memcpy(output, hash, 32);
}

#define _DEBUG_PREFIX "phi"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_phi(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];

	int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 19 : 18; // 2^18 = 262144 cuda threads
	if (device_sm[dev_id] >= 600) intensity = 20;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xf;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		quark_skein512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		if (use_compat_kernels[thr_id])
			x11_echo512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t)64 * throughput), -1);
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];

	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80((void*)endiandata);
	if (use_compat_kernels[thr_id])
		cuda_check_cpu_setTarget(ptarget);
	else
		hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));

	do {
		int order = 0;

		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (use_compat_kernels[thr_id]) {
			streebog_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
			x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		} else {
			streebog_hash_64_maxwell(thr_id, throughput, d_hash[thr_id]);
			tribus_echo512_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id], AS_U64(&ptarget[6]));
			hipMemcpy(&work->nonces[0], d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
		}

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNonce = pdata[19];
			uint32_t _ALIGN(64) vhash[8];
			if (!use_compat_kernels[thr_id]) work->nonces[0] += startNonce;
			be32enc(&endiandata[19], work->nonces[0]);
			phi_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				*hashes_done = pdata[19] - first_nonce + throughput;
				//work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				//if (work->nonces[1] != 0) {
				if (work->nonces[1] != UINT32_MAX) {
					work->nonces[1] += startNonce;
					be32enc(&endiandata[19], work->nonces[1]);
					phi_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				}
				else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_phi(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();
	hipFree(d_hash[thr_id]);
	hipFree(d_resNonce[thr_id]);
	x13_fugue512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
