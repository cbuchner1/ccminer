#include "hip/hip_runtime.h"
//
//  PHI2 algo (with smart contracts header)
//  CubeHash + Lyra2 x2 + JH + Gost or Echo + Skein
//
//  Implemented by tpruvot in May 2018
//

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_streebog.h"
#include "sph/sph_echo.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

#include <stdio.h>
#include <memory.h>

extern void cubehash512_setBlock_80(int thr_id, uint32_t* endiandata);
extern void cubehash512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

extern void cubehash512_setBlock_144(int thr_id, uint32_t* endiandata);
extern void cubehash512_cuda_hash_144(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2_cuda_hash_64(int thr_id, const uint32_t threads, uint64_t* d_hash_256, uint32_t* d_hash_512, bool gtx750ti);

extern void streebog_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void phi_streebog_hash_64_filtered(int thr_id, const uint32_t threads, uint32_t *g_hash, uint32_t *d_filter);
extern void phi_echo512_cpu_hash_64_filtered(int thr_id, const uint32_t threads, uint32_t* g_hash, uint32_t* d_filter);

extern uint32_t phi_filter_cuda(const int thr_id, const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_br2, uint32_t* d_nonces);
extern void phi_merge_cuda(const int thr_id, const uint32_t threads, uint32_t *outpHashes, uint32_t* d_br2, uint32_t* d_nonces);
extern void phi_final_compress_cuda(const int thr_id, const uint32_t threads, uint32_t *d_hashes);

static uint64_t* d_matrix[MAX_GPUS];
static uint32_t* d_hash_512[MAX_GPUS];
static uint64_t* d_hash_256[MAX_GPUS];
static uint32_t* d_hash_br2[MAX_GPUS];
static uint32_t* d_nonce_br[MAX_GPUS];

static bool has_roots;

extern "C" void phi2_hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[64];
	unsigned char _ALIGN(128) hashA[64];
	unsigned char _ALIGN(128) hashB[64];

	sph_cubehash512_context ctx_cubehash;
	sph_jh512_context ctx_jh;
	sph_gost512_context ctx_gost;
	sph_echo512_context ctx_echo;
	sph_skein512_context ctx_skein;

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, input, has_roots ? 144 : 80);
	sph_cubehash512_close(&ctx_cubehash, (void*)hashB);

	LYRA2(&hashA[ 0], 32, &hashB[ 0], 32, &hashB[ 0], 32, 1, 8, 8);
	LYRA2(&hashA[32], 32, &hashB[32], 32, &hashB[32], 32, 1, 8, 8);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*)hashA, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	if (hash[0] & 1) {
		sph_gost512_init(&ctx_gost);
		sph_gost512(&ctx_gost, (const void*)hash, 64);
		sph_gost512_close(&ctx_gost, (void*)hash);
	} else {
		sph_echo512_init(&ctx_echo);
		sph_echo512(&ctx_echo, (const void*)hash, 64);
		sph_echo512_close(&ctx_echo, (void*)hash);

		sph_echo512_init(&ctx_echo);
		sph_echo512(&ctx_echo, (const void*)hash, 64);
		sph_echo512_close(&ctx_echo, (void*)hash);
	}

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	for (int i=0; i<32; i++)
		hash[i] ^= hash[i+32];

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "phi-"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };
static __thread bool gtx750ti = false;

extern "C" int scanhash_phi2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];

	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 17 : 16;
	if (device_sm[dev_id] == 500) intensity = 15;
	if (device_sm[dev_id] == 600) intensity = 17;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);
	if (init[thr_id]) throughput = max(throughput & 0xffffff80, 128); // for shared mem

	if (opt_benchmark)
		ptarget[7] = 0xff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);
		gtx750ti = (strstr(device_name[dev_id], "GTX 750 Ti") != NULL);

		size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 16 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
		CUDA_CALL_OR_RET_X(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash_256[thr_id], (size_t)32 * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash_512[thr_id], (size_t)64 * throughput), -1);
		CUDA_CALL_OR_RET_X(hipMalloc(&d_nonce_br[thr_id], sizeof(uint32_t) * throughput), -1);
		if (use_compat_kernels[thr_id]) {
			CUDA_CALL_OR_RET_X(hipMalloc(&d_hash_br2[thr_id], (size_t)64 * throughput), -1);
		}

		lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		if (use_compat_kernels[thr_id]) x11_echo512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	has_roots = false;
	uint32_t endiandata[36];
	for (int k = 0; k < 36; k++) {
		be32enc(&endiandata[k], pdata[k]);
		if (k >= 20 && pdata[k]) has_roots = true;
	}

	cuda_check_cpu_setTarget(ptarget);
	if (has_roots)
		cubehash512_setBlock_144(thr_id, endiandata);
	else
		cubehash512_setBlock_80(thr_id, endiandata);

	do {
		int order = 0;
		if (has_roots)
			cubehash512_cuda_hash_144(thr_id, throughput, pdata[19], d_hash_512[thr_id]);
		else
			cubehash512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash_512[thr_id]);
		order++;
		TRACE("cube   ");

		lyra2_cuda_hash_64(thr_id, throughput, d_hash_256[thr_id], d_hash_512[thr_id], gtx750ti);
		order++;
		TRACE("lyra   ");

		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_512[thr_id], order++);
		TRACE("jh     ");

		order++;
		if (!use_compat_kernels[thr_id]) {
			phi_filter_cuda(thr_id, throughput, d_hash_512[thr_id], NULL, d_nonce_br[thr_id]);
			phi_streebog_hash_64_filtered(thr_id, throughput, d_hash_512[thr_id], d_nonce_br[thr_id]);
			phi_echo512_cpu_hash_64_filtered(thr_id, throughput, d_hash_512[thr_id], d_nonce_br[thr_id]);
			phi_echo512_cpu_hash_64_filtered(thr_id, throughput, d_hash_512[thr_id], d_nonce_br[thr_id]);
		} else {
			// todo: nonces vector to reduce amount of hashes to compute
			phi_filter_cuda(thr_id, throughput, d_hash_512[thr_id], d_hash_br2[thr_id], d_nonce_br[thr_id]);
			streebog_cpu_hash_64(thr_id, throughput, d_hash_512[thr_id]);
			x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order);
			x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order);
			phi_merge_cuda(thr_id, throughput, d_hash_512[thr_id], d_hash_br2[thr_id], d_nonce_br[thr_id]);
		}
		TRACE("mix    ");

		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_512[thr_id], order++);
		TRACE("skein  ");

		phi_final_compress_cuda(thr_id, throughput, d_hash_512[thr_id]);
		TRACE("xor  ");

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash_512[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			phi2_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				*hashes_done = pdata[19] - first_nonce + throughput;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash_512[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					phi2_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				if (pdata[19] > max_nonce) pdata[19] = max_nonce;
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! thr=%x", work->nonces[0], throughput);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_phi2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();
	hipFree(d_matrix[thr_id]);
	hipFree(d_hash_512[thr_id]);
	hipFree(d_hash_256[thr_id]);
	hipFree(d_nonce_br[thr_id]);
	if (use_compat_kernels[thr_id]) hipFree(d_hash_br2[thr_id]);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
