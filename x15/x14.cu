#include "hip/hip_runtime.h"
/*
 * X14 algorithm
 * Added in ccminer by Tanguy Pruvot - 2014
 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "x11/cuda_x11.h"

// Memory for the hash functions
static uint32_t *d_hash[MAX_GPUS] = { 0 };

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_free(int thr_id);

extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);


// X14 CPU Hash function
extern "C" void x14hash(void *output, const void *input)
{
	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, 64);
	sph_bmw512_close(&ctx_bmw, hashB);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hashB, 64);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, 64);
	sph_skein512_close(&ctx_skein, hashB);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, hashB, 64);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hash, 64);
	sph_keccak512_close(&ctx_keccak, hashB);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hashB, 64);
	sph_luffa512_close(&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hash, 64);
	sph_cubehash512_close(&ctx_cubehash, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hash, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hash, 64);
	sph_hamsi512_close(&ctx_hamsi, hashB);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hashB, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, 64);
	sph_shabal512_close(&ctx_shabal, hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_x14(int thr_id,  struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput);

		CUDA_LOG_ERROR();

		init[thr_id] = true;
	}

	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		CUDA_LOG_ERROR();

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			/* check now with the CPU to confirm */
			be32enc(&endiandata[19], work->nonces[0]);
			x14hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					x14hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	CUDA_LOG_ERROR();

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_x14(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x13_fugue512_cpu_free(thr_id);

	hipFree(d_hash[thr_id]);
	d_hash[thr_id] = NULL;

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
