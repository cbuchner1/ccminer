#include "hip/hip_runtime.h"
/*
 * X14 algorithm
 * Added in ccminer by Tanguy Pruvot - 2014
 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
}

#include "miner.h"

#include "cuda_helper.h"

// Memory for the hash functions
static uint32_t *d_hash[MAX_GPUS];

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_luffaCubehash512_cpu_init(int thr_id, uint32_t threads);
extern void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash, int order);

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
											uint32_t *d_noncesTrue, size_t *nrmTrue, uint32_t *d_noncesFalse, size_t *nrmFalse, int order);

// X14 CPU Hash function
extern "C" void x14hash(void *output, const void *input)
{
	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, 64);
	sph_bmw512_close(&ctx_bmw, hashB);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hashB, 64);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, 64);
	sph_skein512_close(&ctx_skein, hashB);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, hashB, 64);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hash, 64);
	sph_keccak512_close(&ctx_keccak, hashB);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hashB, 64);
	sph_luffa512_close(&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hash, 64);
	sph_cubehash512_close(&ctx_cubehash, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hash, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hash, 64);
	sph_hamsi512_close(&ctx_hamsi, hashB);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hashB, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, 64);
	sph_shabal512_close(&ctx_shabal, hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_x14(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	uint32_t throughput =  device_intensity(thr_id, __func__, 1U << 19); // 19=256*256*8;
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			/* check now with the CPU to confirm */
			be32enc(&endiandata[19], foundNonce);
			x14hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}
		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
