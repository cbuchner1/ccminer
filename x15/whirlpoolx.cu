#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis, tpruvot)
 */
extern "C" {
#include "sph/sph_whirlpool.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS] = { 0 };

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_cpu_free(int thr_id);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce);
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	// compress the 48 first bytes of the hash to 32
	for (int i = 0; i < 32; i++) {
		hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirlx(int thr_id,  struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int intensity = is_windows() ? 20 : 22;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), -1);

		whirlpoolx_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, ptarget);
	whirlpoolx_precompute(thr_id);

	do {
		uint32_t foundNonce = whirlpoolx_cpu_hash(thr_id, throughput, pdata[19]);

		*(hashes_done) = pdata[19] - first_nonce + throughput;

		if (foundNonce != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			whirlxHash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				work_set_target_ratio(work, vhash64);
				pdata[19] = foundNonce;
				return 1;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*(hashes_done) = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_whirlx(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	whirlpoolx_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
