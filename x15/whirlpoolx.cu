#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis, tpruvot)
 */
extern "C" {
#include "sph/sph_whirlpool.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce);
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	// compress the 48 first bytes of the hash to 32
	for (int i = 0; i < 32; i++) {
		hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirlpoolx(int thr_id, uint32_t *pdata, const uint32_t *ptarget,
	uint32_t max_nonce, unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int intensity = is_windows() ? 20 : 22;
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << intensity);
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 64 * throughput), 0);

		whirlpoolx_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, ptarget);
	whirlpoolx_precompute(thr_id);
	do {
		uint32_t foundNonce = whirlpoolx_cpu_hash(thr_id, throughput, pdata[19]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			whirlxHash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = foundNonce;
				return 1;
			} else {
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

		if (((uint64_t)pdata[19]+throughput) >= max_nonce) {
			break;
		}

	} while (!work_restart[thr_id].restart);

	*(hashes_done) = pdata[19] - first_nonce + 1;

	return 0;
}
