#include "hip/hip_runtime.h"
/*
 * whirlpool routine
 */
extern "C" {
#include <sph/sph_whirlpool.h>
#include <miner.h>
}

#include <cuda_helper.h>

//#define SM3_VARIANT

#ifdef SM3_VARIANT
static uint32_t *d_hash[MAX_GPUS];
extern void whirlpool512_init_sm3(int thr_id, uint32_t threads, int mode);
extern void whirlpool512_free_sm3(int thr_id);
extern void whirlpool512_setBlock_80_sm3(void *pdata, const void *ptarget);
extern void whirlpool512_hash_64_sm3(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void whirlpool512_hash_80_sm3(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern uint32_t whirlpool512_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
//#define _DEBUG
#define _DEBUG_PREFIX "whirl"
#include <cuda_debug.cuh>
#else
extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_free(int thr_id);
extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, const uint64_t target);
#endif


// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

void whirl_midstate(void *state, const void *input)
{
	sph_whirlpool_context ctx;

	sph_whirlpool1_init(&ctx);
	sph_whirlpool1(&ctx, input, 64);

	memcpy(state, ctx.state, 64);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirl(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(128) endiandata[20];
	uint32_t* pdata = work->data;
	uint32_t* ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 19=256*256*8;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);
	if (init[thr_id]) throughput = max(throughput, 256); // shared mem requirement

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
#ifdef SM3_VARIANT
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));
		whirlpool512_init_sm3(thr_id, throughput, 1 /* old whirlpool */);
#else
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);
#endif
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

#ifdef SM3_VARIANT
	whirlpool512_setBlock_80_sm3((void*)endiandata, ptarget);
#else
	whirlpool512_setBlock_80((void*)endiandata, ptarget);
#endif

	do {
#ifdef SM3_VARIANT
		int order = 1;
		whirlpool512_hash_80_sm3(thr_id, throughput, pdata[19], d_hash[thr_id]);
		TRACE64(" 80 :", d_hash);
		whirlpool512_hash_64_sm3(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);
		whirlpool512_hash_64_sm3(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);
		work->nonces[0] = whirlpool512_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
#else
		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], work->nonces, *(uint64_t*)&ptarget[6]);
#endif
		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			wcoinhash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1; // cursor
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}
		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_whirl(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

#ifdef SM3_VARIANT
	hipFree(d_hash[thr_id]);
	whirlpool512_free_sm3(thr_id);
#else
	x15_whirlpool_cpu_free(thr_id);
#endif
	init[thr_id] = false;

	hipDeviceSynchronize();
}

