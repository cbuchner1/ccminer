#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x15_whirlpool_cpu_free(int thr_id);

extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern uint32_t whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

//#define _DEBUG
#define _DEBUG_PREFIX "whirl"
#include "cuda_debug.cuh"

// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

void whirl_midstate(void *state, const void *input)
{
	sph_whirlpool_context ctx;

	sph_whirlpool1_init(&ctx);
	sph_whirlpool1(&ctx, input, 64);

	memcpy(state, ctx.state, 64);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirl(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(128) endiandata[20];
	uint32_t* pdata = work->data;
	uint32_t* ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 19=256*256*8;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);
	if (init[thr_id]) throughput = max(throughput, 256); // shared mem requirement

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		uint32_t foundNonce;
		int order = 0;

		*hashes_done = pdata[19] - first_nonce + throughput;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE64(" 80 :", d_hash);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);

		foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash[8];
			be32enc(&endiandata[19], foundNonce);
			wcoinhash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				int res = 1;
				work_set_target_ratio(work, vhash);
				#if 0
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				#endif
				pdata[19] = foundNonce;
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}
		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_whirl(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	x15_whirlpool_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}

