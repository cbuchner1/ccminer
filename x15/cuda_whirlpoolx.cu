#include "hip/hip_runtime.h"
/*
 * Built on cbuchner1's implementation, actual hashing code
 * based on sphlib 3.0
 */
#include <stdio.h>
#include <memory.h>

#define threadsPerBlock 1024

#include "cuda_helper.h"

__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)
__constant__ uint64_t c_xtra[8];
__constant__ uint64_t c_tmp[72];
__constant__ uint64_t pTarget[4];

static uint32_t *h_wxnounce[MAX_GPUS] = { 0 };
static uint32_t *d_WXNonce[MAX_GPUS] = { 0 };

/**
 * Whirlpool CUDA kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014 djm34 & tpruvot & SP & Provos Alexis
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 * @author djm34
 * @author tpruvot
 * @author SP
 * @author Provos Alexis
 */

__constant__ __align__(64) uint64_t mixTob0Tox[256];

const uint64_t plain_T0[256]= {
	0xD83078C018601818,0x2646AF05238C2323,0xB891F97EC63FC6C6,0xFBCD6F13E887E8E8,0xCB13A14C87268787,0x116D62A9B8DAB8B8,0x0902050801040101,0x0D9E6E424F214F4F,0x9B6CEEAD36D83636,
	0xFF510459A6A2A6A6,0x0CB9BDDED26FD2D2,0x0EF706FBF5F3F5F5,0x96F280EF79F97979,0x30DECE5F6FA16F6F,0x6D3FEFFC917E9191,0xF8A407AA52555252,0x47C0FD27609D6060,0x35657689BCCABCBC,
	0x372BCDAC9B569B9B,0x8A018C048E028E8E,0xD25B1571A3B6A3A3,0x6C183C600C300C0C,0x84F68AFF7BF17B7B,0x806AE1B535D43535,0xF53A69E81D741D1D,0xB3DD4753E0A7E0E0,0x21B3ACF6D77BD7D7,
	0x9C99ED5EC22FC2C2,0x435C966D2EB82E2E,0x29967A624B314B4B,0x5DE121A3FEDFFEFE,0xD5AE168257415757,0xBD2A41A815541515,0xE8EEB69F77C17777,0x926EEBA537DC3737,0x9ED7567BE5B3E5E5,
	0x1323D98C9F469F9F,0x23FD17D3F0E7F0F0,0x20947F6A4A354A4A,0x44A9959EDA4FDADA,0xA2B025FA587D5858,0xCF8FCA06C903C9C9,0x7C528D5529A42929,0x5A1422500A280A0A,0x507F4FE1B1FEB1B1,
	0xC95D1A69A0BAA0A0,0x14D6DA7F6BB16B6B,0xD917AB5C852E8585,0x3C677381BDCEBDBD,0x8FBA34D25D695D5D,0x9020508010401010,0x07F503F3F4F7F4F4,0xDD8BC016CB0BCBCB,0xD37CC6ED3EF83E3E,
	0x2D0A112805140505,0x78CEE61F67816767,0x97D55373E4B7E4E4,0x024EBB25279C2727,0x7382583241194141,0xA70B9D2C8B168B8B,0xF6530151A7A6A7A7,0xB2FA94CF7DE97D7D,0x4937FBDC956E9595,
	0x56AD9F8ED847D8D8,0x70EB308BFBCBFBFB,0xCDC17123EE9FEEEE,0xBBF891C77CED7C7C,0x71CCE31766856666,0x7BA78EA6DD53DDDD,0xAF2E4BB8175C1717,0x458E460247014747,0x1A21DC849E429E9E,
	0xD489C51ECA0FCACA,0x585A99752DB42D2D,0x2E637991BFC6BFBF,0x3F0E1B38071C0707,0xAC472301AD8EADAD,0xB0B42FEA5A755A5A,0xEF1BB56C83368383,0xB666FF8533CC3333,0x5CC6F23F63916363,
	0x12040A1002080202,0x93493839AA92AAAA,0xDEE2A8AF71D97171,0xC68DCF0EC807C8C8,0xD1327DC819641919,0x3B92707249394949,0x5FAF9A86D943D9D9,0x31F91DC3F2EFF2F2,0xA8DB484BE3ABE3E3,
	0xB9B62AE25B715B5B,0xBC0D9234881A8888,0x3E29C8A49A529A9A,0x0B4CBE2D26982626,0xBF64FA8D32C83232,0x597D4AE9B0FAB0B0,0xF2CF6A1BE983E9E9,0x771E33780F3C0F0F,0x33B7A6E6D573D5D5,
	0xF41DBA74803A8080,0x27617C99BEC2BEBE,0xEB87DE26CD13CDCD,0x8968E4BD34D03434,0x3290757A483D4848,0x54E324ABFFDBFFFF,0x8DF48FF77AF57A7A,0x643DEAF4907A9090,0x9DBE3EC25F615F5F,
	0x3D40A01D20802020,0x0FD0D56768BD6868,0xCA3472D01A681A1A,0xB7412C19AE82AEAE,0x7D755EC9B4EAB4B4,0xCEA8199A544D5454,0x7F3BE5EC93769393,0x2F44AA0D22882222,0x63C8E907648D6464,
	0x2AFF12DBF1E3F1F1,0xCCE6A2BF73D17373,0x82245A9012481212,0x7A805D3A401D4040,0x4810284008200808,0x959BE856C32BC3C3,0xDFC57B33EC97ECEC,0x4DAB9096DB4BDBDB,0xC05F1F61A1BEA1A1,
	0x9107831C8D0E8D8D,0xC87AC9F53DF43D3D,0x5B33F1CC97669797,0x0000000000000000,0xF983D436CF1BCFCF,0x6E5687452BAC2B2B,0xE1ECB39776C57676,0xE619B06482328282,0x28B1A9FED67FD6D6,
	0xC33677D81B6C1B1B,0x74775BC1B5EEB5B5,0xBE432911AF86AFAF,0x1DD4DF776AB56A6A,0xEAA00DBA505D5050,0x578A4C1245094545,0x38FB18CBF3EBF3F3,0xAD60F09D30C03030,0xC4C3742BEF9BEFEF,
	0xDA7EC3E53FFC3F3F,0xC7AA1C9255495555,0xDB591079A2B2A2A2,0xE9C96503EA8FEAEA,0x6ACAEC0F65896565,0x036968B9BAD2BABA,0x4A5E93652FBC2F2F,0x8E9DE74EC027C0C0,0x60A181BEDE5FDEDE,
	0xFC386CE01C701C1C,0x46E72EBBFDD3FDFD,0x1F9A64524D294D4D,0x7639E0E492729292,0xFAEABC8F75C97575,0x360C1E3006180606,0xAE0998248A128A8A,0x4B7940F9B2F2B2B2,0x85D15963E6BFE6E6,
	0x7E1C36700E380E0E,0xE73E63F81F7C1F1F,0x55C4F73762956262,0x3AB5A3EED477D4D4,0x814D3229A89AA8A8,0x5231F4C496629696,0x62EF3A9BF9C3F9F9,0xA397F666C533C5C5,0x104AB13525942525,
	0xABB220F259795959,0xD015AE54842A8484,0xC5E4A7B772D57272,0xEC72DDD539E43939,0x1698615A4C2D4C4C,0x94BC3BCA5E655E5E,0x9FF085E778FD7878,0xE570D8DD38E03838,0x980586148C0A8C8C,
	0x17BFB2C6D163D1D1,0xE4570B41A5AEA5A5,0xA1D94D43E2AFE2E2,0x4EC2F82F61996161,0x427B45F1B3F6B3B3,0x3442A51521842121,0x0825D6949C4A9C9C,0xEE3C66F01E781E1E,0x6186522243114343,
	0xB193FC76C73BC7C7,0x4FE52BB3FCD7FCFC,0x2408142004100404,0xE3A208B251595151,0x252FC7BC995E9999,0x22DAC44F6DA96D6D,0x651A39680D340D0D,0x79E93583FACFFAFA,0x69A384B6DF5BDFDF,
	0xA9FC9BD77EE57E7E,0x1948B43D24902424,0xFE76D7C53BEC3B3B,0x9A4B3D31AB96ABAB,0xF081D13ECE1FCECE,0x9922558811441111,0x8303890C8F068F8F,0x049C6B4A4E254E4E,0x667351D1B7E6B7B7,
	0xE0CB600BEB8BEBEB,0xC178CCFD3CF03C3C,0xFD1FBF7C813E8181,0x4035FED4946A9494,0x1CF30CEBF7FBF7F7,0x186F67A1B9DEB9B9,0x8B265F98134C1313,0x51589C7D2CB02C2C,0x05BBB8D6D36BD3D3,
	0x8CD35C6BE7BBE7E7,0x39DCCB576EA56E6E,0xAA95F36EC437C4C4,0x1B060F18030C0303,0xDCAC138A56455656,0x5E88491A440D4444,0xA0FE9EDF7FE17F7F,0x884F3721A99EA9A9,0x6754824D2AA82A2A,
	0x0A6B6DB1BBD6BBBB,0x879FE246C123C1C1,0xF1A602A253515353,0x72A58BAEDC57DCDC,0x531627580B2C0B0B,0x0127D39C9D4E9D9D,0x2BD8C1476CAD6C6C,0xA462F59531C43131,0xF3E8B98774CD7474,
	0x15F109E3F6FFF6F6,0x4C8C430A46054646,0xA5452609AC8AACAC,0xB50F973C891E8989,0xB42844A014501414,0xBADF425BE1A3E1E1,0xA62C4EB016581616,0xF774D2CD3AE83A3A,0x06D2D06F69B96969,
	0x41122D4809240909,0xD7E0ADA770DD7070,0x6F7154D9B6E2B6B6,0x1EBDB7CED067D0D0,0xD6C77E3BED93EDED,0xE285DB2ECC17CCCC,0x6884572A42154242,0x2C2DC2B4985A9898,0xED550E49A4AAA4A4,
	0x7550885D28A02828,0x86B831DA5C6D5C5C,0x6BED3F93F8C7F8F8,0xC211A44486228686
};

/**
 * Round constants.
 */
__constant__ uint64_t InitVector_RC[10];

const uint64_t plain_RC[10] = {
	0x4F01B887E8C62318,0x52916F79F5D2A636,0x357B0CA38E9BBC60,0x57FE4B2EC2D7E01D,0xDA4AF09FE5377715,
	0x856BA0B10A29C958,0x67053ECBF4105DBD,0xD8957DA78B4127E4,0x9E4717DD667CEEFB,0x33835AAD07BF2DCA
};

/* ====================================================================== */

__device__ __forceinline__
static uint64_t ROUND_ELT(const uint64_t* sharedMemory, const uint64_t* __restrict__ in, const int i0, const int i1, const int i2, const int i3, const int i4, const int i5, const int i6, const int i7)
{
	uint32_t* in32 = (uint32_t*)in;
	return xor8(	sharedMemory[__byte_perm(in32[(i0 << 1)], 0, 0x4440)],
			sharedMemory[__byte_perm(in32[(i1 << 1)], 0, 0x4441) + 256],
			sharedMemory[__byte_perm(in32[(i2 << 1)], 0, 0x4442) + 512],
			sharedMemory[__byte_perm(in32[(i3 << 1)], 0, 0x4443) + 768],
			sharedMemory[__byte_perm(in32[(i4 << 1) + 1], 0, 0x4440) + 1024],
			sharedMemory[__byte_perm(in32[(i5 << 1) + 1], 0, 0x4441) + 1280],
			sharedMemory[__byte_perm(in32[(i6 << 1) + 1], 0, 0x4442) + 1536],
			sharedMemory[__byte_perm(in32[(i7 << 1) + 1], 0, 0x4443) + 1792]);
}

#define TRANSFER(dst, src) { \
	dst[0] = src ## 0; \
	dst[1] = src ## 1; \
	dst[2] = src ## 2; \
	dst[3] = src ## 3; \
	dst[4] = src ## 4; \
	dst[5] = src ## 5; \
	dst[6] = src ## 6; \
	dst[7] = src ## 7; \
}

#define ROUND(table, in, out, c0, c1, c2, c3, c4, c5, c6, c7) { \
	out ## 0 = xor1(ROUND_ELT(table, in, 0, 7, 6, 5, 4, 3, 2, 1), c0); \
	out ## 1 = xor1(ROUND_ELT(table, in, 1, 0, 7, 6, 5, 4, 3, 2), c1); \
	out ## 2 = xor1(ROUND_ELT(table, in, 2, 1, 0, 7, 6, 5, 4, 3), c2); \
	out ## 3 = xor1(ROUND_ELT(table, in, 3, 2, 1, 0, 7, 6, 5, 4), c3); \
	out ## 4 = xor1(ROUND_ELT(table, in, 4, 3, 2, 1, 0, 7, 6, 5), c4); \
	out ## 5 = xor1(ROUND_ELT(table, in, 5, 4, 3, 2, 1, 0, 7, 6), c5); \
	out ## 6 = xor1(ROUND_ELT(table, in, 6, 5, 4, 3, 2, 1, 0, 7), c6); \
	out ## 7 = xor1(ROUND_ELT(table, in, 7, 6, 5, 4, 3, 2, 1, 0), c7); \
}

#define ROUND1(table, in, out, c) { \
	out ## 0 = xor1(ROUND_ELT(table, in, 0, 7, 6, 5, 4, 3, 2, 1), c); \
	out ## 1 = ROUND_ELT(table, in, 1, 0, 7, 6, 5, 4, 3, 2); \
	out ## 2 = ROUND_ELT(table, in, 2, 1, 0, 7, 6, 5, 4, 3); \
	out ## 3 = ROUND_ELT(table, in, 3, 2, 1, 0, 7, 6, 5, 4); \
	out ## 4 = ROUND_ELT(table, in, 4, 3, 2, 1, 0, 7, 6, 5); \
	out ## 5 = ROUND_ELT(table, in, 5, 4, 3, 2, 1, 0, 7, 6); \
	out ## 6 = ROUND_ELT(table, in, 6, 5, 4, 3, 2, 1, 0, 7); \
	out ## 7 = ROUND_ELT(table, in, 7, 6, 5, 4, 3, 2, 1, 0); \
}

#define ROUND_KSCHED(table, in, out, c) \
	ROUND1(table, in, out, c) \
	TRANSFER(in, out)

#define ROUND_WENC(table, in, key, out) \
	ROUND(table, in, out, key[0], key[1], key[2],key[3], key[4], key[5], key[6], key[7]) \
	TRANSFER(in, out)

static uint64_t* d_xtra[MAX_GPUS] = { 0 };
static uint64_t* d_tmp[MAX_GPUS] = { 0 };

__device__ __forceinline__
static void whirlpoolx_getShared(uint64_t* sharedMemory)
{
	if (threadIdx.x < 256) {
		sharedMemory[threadIdx.x] = mixTob0Tox[threadIdx.x];
		sharedMemory[threadIdx.x+256]  = ROTL64(sharedMemory[threadIdx.x], 8);
		sharedMemory[threadIdx.x+512]  = ROTL64(sharedMemory[threadIdx.x],16);
		sharedMemory[threadIdx.x+768]  = ROTL64(sharedMemory[threadIdx.x],24);
		sharedMemory[threadIdx.x+1024] = ROTL64(sharedMemory[threadIdx.x],32);
		sharedMemory[threadIdx.x+1280] = ROTR64(sharedMemory[threadIdx.x],24);
		sharedMemory[threadIdx.x+1536] = ROTR64(sharedMemory[threadIdx.x],16);
		sharedMemory[threadIdx.x+1792] = ROTR64(sharedMemory[threadIdx.x], 8);
	}
	__syncthreads();
}


__global__
void whirlpoolx_gpu_precompute(uint32_t threads, uint64_t* d_xtra, uint64_t* d_tmp)
{
	__shared__ uint64_t sharedMemory[2048];

	whirlpoolx_getShared(sharedMemory);
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t n[8];
		uint64_t h[8] = { 0 };

		#pragma unroll 8
		for (int i=0; i<8; i++) {
			n[i] = c_PaddedMessage80[i];  // read data
		}
		//#pragma unroll 10
		for (unsigned int r=0; r < 10; r++) {
			uint64_t tmp0, tmp1, tmp2, tmp3, tmp4, tmp5, tmp6, tmp7;
			ROUND_KSCHED(sharedMemory, h, tmp, InitVector_RC[r]);
			ROUND_WENC(sharedMemory, n, h, tmp);
		}
		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			h[i] = xor1(n[i],c_PaddedMessage80[i]);
		}

		if(threadIdx.x==0)d_xtra[threadIdx.x]=h[1];
		uint64_t atLastCalc=xor1(h[3],h[5]);

		//////////////////////////////////
		n[0] = c_PaddedMessage80[8];    //read data
		n[1] = c_PaddedMessage80[9]; //whirlpool
		n[2] = 0x0000000000000080; //whirlpool
		n[3] = 0;
		n[4] = 0;
		n[5] = 0;
		n[6] = 0;
		n[7] = 0x8002000000000000;

		n[0] = xor1(n[0],h[0]);
		n[2] = xor1(n[2],h[2]);	n[3] = h[3];
		n[4] = h[4];	n[5] = h[5];
		n[6] = h[6];	n[7] = xor1(n[7],h[7]);
		uint64_t tmp[8];
		tmp[0] = xor1(ROUND_ELT(sharedMemory, h, 0, 7, 6, 5, 4, 3, 2, 1),InitVector_RC[0]);
		tmp[1] = ROUND_ELT(sharedMemory, h, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp[2] = ROUND_ELT(sharedMemory, h, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp[3] = ROUND_ELT(sharedMemory, h, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp[4] = ROUND_ELT(sharedMemory, h, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp[5] = ROUND_ELT(sharedMemory, h, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp[6] = ROUND_ELT(sharedMemory, h, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp[7] = ROUND_ELT(sharedMemory, h, 7, 6, 5, 4, 3, 2, 1, 0);

		uint64_t tmp2[8];
		uint32_t* n32 = (uint32_t*)n;
		tmp2[0]=xor8(	sharedMemory[__byte_perm(n32[ 0], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[14], 0, 0x4441) + 256],
				sharedMemory[__byte_perm(n32[12], 0, 0x4442) + 512]	,sharedMemory[__byte_perm(n32[10], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[ 9], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[ 7], 0, 0x4441) + 1280],
				sharedMemory[__byte_perm(n32[ 5], 0, 0x4442) + 1536]	,tmp[0]);

		tmp2[1]=xor8(	tmp[1]							,sharedMemory[__byte_perm(n32[ 0], 0, 0x4441) + 256],
				sharedMemory[__byte_perm(n32[14], 0, 0x4442) +  512]	,sharedMemory[__byte_perm(n32[12], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[11], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[ 9], 0, 0x4441) + 1280],
				sharedMemory[__byte_perm(n32[ 7], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[ 5], 0, 0x4443) + 1792]);

		tmp2[2]=xor8(	sharedMemory[__byte_perm(n32[ 4], 0, 0x4440)]  		,tmp[2]						    ,
				sharedMemory[__byte_perm(n32[ 0], 0, 0x4442) +  512]	,sharedMemory[__byte_perm(n32[14], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[13], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[11], 0, 0x4441) + 1280],
				sharedMemory[__byte_perm(n32[ 9], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[ 7], 0, 0x4443) + 1792]);

		tmp2[3]=xor8(	sharedMemory[__byte_perm(n32[ 6], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[ 4], 0, 0x4441) + 256],
				tmp[3]							,sharedMemory[__byte_perm(n32[ 0], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[15], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[13], 0, 0x4441) + 1280],
				sharedMemory[__byte_perm(n32[11], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[ 9], 0, 0x4443) + 1792]);

		tmp2[4]=xor8(	sharedMemory[__byte_perm(n32[ 8], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[ 6], 0, 0x4441) + 256]  ,
				sharedMemory[__byte_perm(n32[ 4], 0, 0x4442) +  512]	,tmp[4]						      ,
				sharedMemory[__byte_perm(n32[ 1], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[15], 0, 0x4441) + 1280] ,
				sharedMemory[__byte_perm(n32[13], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[11], 0, 0x4443) + 1792]);

		tmp2[5]=xor8(	sharedMemory[__byte_perm(n32[10], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[ 8], 0, 0x4441) + 256],
				sharedMemory[__byte_perm(n32[ 6], 0, 0x4442) +  512]	,sharedMemory[__byte_perm(n32[ 4], 0, 0x4443) + 768],
				tmp[5]							,sharedMemory[__byte_perm(n32[ 1], 0, 0x4441) + 1280],
				sharedMemory[__byte_perm(n32[15], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[13], 0, 0x4443) + 1792]);

		tmp2[6]=xor8(	sharedMemory[__byte_perm(n32[12], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[10], 0, 0x4441) + 256],
				sharedMemory[__byte_perm(n32[ 8], 0, 0x4442) +  512]	,sharedMemory[__byte_perm(n32[ 6], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[ 5], 0, 0x4440) + 1024]	,tmp[6],
				sharedMemory[__byte_perm(n32[ 1], 0, 0x4442) + 1536]	,sharedMemory[__byte_perm(n32[15], 0, 0x4443) + 1792]);

		tmp2[7]=xor8(	sharedMemory[__byte_perm(n32[14], 0, 0x4440)]  		,sharedMemory[__byte_perm(n32[12], 0, 0x4441) + 256],
				sharedMemory[__byte_perm(n32[10], 0, 0x4442) +  512]	,sharedMemory[__byte_perm(n32[ 8], 0, 0x4443) + 768],
				sharedMemory[__byte_perm(n32[ 7], 0, 0x4440) + 1024]	,sharedMemory[__byte_perm(n32[ 5], 0, 0x4441) + 1280],
				tmp[7]							,sharedMemory[__byte_perm(n32[ 1], 0, 0x4443) + 1792]);

		n[1] ^= h[1];
		tmp2[1]^=sharedMemory[__byte_perm(n32[2], 0, 0x4440)];
		tmp2[2]^=sharedMemory[__byte_perm(n32[2], 0, 0x4441) + 256];
		tmp2[3]^=sharedMemory[__byte_perm(n32[2], 0, 0x4442) + 512];
		tmp2[4]^=sharedMemory[__byte_perm(n32[2], 0, 0x4443) + 768];

		d_tmp[threadIdx.x]=tmp2[threadIdx.x];

		uint64_t tmp3[8];
		tmp3[0] = xor1(ROUND_ELT(sharedMemory, tmp, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[1]);
		tmp3[1] = ROUND_ELT(sharedMemory, tmp, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp3[2] = ROUND_ELT(sharedMemory, tmp, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp3[3] = ROUND_ELT(sharedMemory, tmp, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp3[4] = ROUND_ELT(sharedMemory, tmp, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp3[5] = ROUND_ELT(sharedMemory, tmp, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp3[6] = ROUND_ELT(sharedMemory, tmp, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp3[7] = ROUND_ELT(sharedMemory, tmp, 7, 6, 5, 4, 3, 2, 1, 0);

		n32 = (uint32_t*)tmp2;
		uint64_t tmp4[8];
		tmp4[0]=(	sharedMemory[__byte_perm(n32[ 9], 0, 0x4440) + 1024]	^sharedMemory[__byte_perm(n32[ 7], 0, 0x4441) + 1280]^
			sharedMemory[__byte_perm(n32[ 5], 0, 0x4442) + 1536]	^sharedMemory[__byte_perm(n32[ 3], 0, 0x4443) + 1792]) ^tmp3[0];

		tmp4[1]=(sharedMemory[__byte_perm(n32[ 2], 0, 0x4440)]		^sharedMemory[__byte_perm(n32[ 9], 0, 0x4441) + 1280]^
			sharedMemory[__byte_perm(n32[ 7], 0, 0x4442) + 1536]	^sharedMemory[__byte_perm(n32[ 5], 0, 0x4443) + 1792]) ^tmp3[1];

		tmp4[2]=(sharedMemory[__byte_perm(n32[ 4], 0, 0x4440)]  	^sharedMemory[__byte_perm(n32[ 2], 0, 0x4441) + 256]^
			sharedMemory[__byte_perm(n32[ 9], 0, 0x4442) + 1536]	^sharedMemory[__byte_perm(n32[ 7], 0, 0x4443) + 1792]) ^tmp3[2];

		tmp4[3]=(sharedMemory[__byte_perm(n32[ 6], 0, 0x4440)]  	^sharedMemory[__byte_perm(n32[ 4], 0, 0x4441) + 256]^
			sharedMemory[__byte_perm(n32[ 2], 0, 0x4442) +  512]	^sharedMemory[__byte_perm(n32[ 9], 0, 0x4443) + 1792]) ^tmp3[3];

		tmp4[4]=(sharedMemory[__byte_perm(n32[ 8], 0, 0x4440)]  	^sharedMemory[__byte_perm(n32[ 6], 0, 0x4441) + 256]^
			sharedMemory[__byte_perm(n32[ 4], 0, 0x4442) +  512]	^sharedMemory[__byte_perm(n32[ 2], 0, 0x4443) + 768]) ^tmp3[4];

		tmp4[5]=(sharedMemory[__byte_perm(n32[ 8], 0, 0x4441) + 256]	^sharedMemory[__byte_perm(n32[ 6], 0, 0x4442) +  512]^
			sharedMemory[__byte_perm(n32[ 4], 0, 0x4443) + 768]	^sharedMemory[__byte_perm(n32[ 3], 0, 0x4440) + 1024]) ^tmp3[5];

		tmp4[6]=(sharedMemory[__byte_perm(n32[ 8], 0, 0x4442) + 512]	^sharedMemory[__byte_perm(n32[ 6], 0, 0x4443) + 768]^
			sharedMemory[__byte_perm(n32[ 5], 0, 0x4440) + 1024]	^sharedMemory[__byte_perm(n32[ 3], 0, 0x4441) + 1280]) ^tmp3[6];

		tmp4[7]=(sharedMemory[__byte_perm(n32[ 8], 0, 0x4443) + 768]	^sharedMemory[__byte_perm(n32[ 7], 0, 0x4440) + 1024]^
			sharedMemory[__byte_perm(n32[ 5], 0, 0x4441) + 1280]	^sharedMemory[__byte_perm(n32[ 3], 0, 0x4442) + 1536]) ^tmp3[7];

		d_tmp[threadIdx.x+16]=tmp4[threadIdx.x];

		uint64_t tmp5[8];
		tmp5[0] = xor1(ROUND_ELT(sharedMemory, tmp3, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[2]);
		tmp5[1] = ROUND_ELT(sharedMemory, tmp3, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp5[2] = ROUND_ELT(sharedMemory, tmp3, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp5[3] = ROUND_ELT(sharedMemory, tmp3, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp5[4] = ROUND_ELT(sharedMemory, tmp3, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp5[5] = ROUND_ELT(sharedMemory, tmp3, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp5[6] = ROUND_ELT(sharedMemory, tmp3, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp5[7] = ROUND_ELT(sharedMemory, tmp3, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+8]=tmp5[threadIdx.x];

		uint64_t tmp6[8];
		tmp6[0] = xor1(ROUND_ELT(sharedMemory, tmp5, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[3]);
		tmp6[1] = ROUND_ELT(sharedMemory, tmp5, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp6[2] = ROUND_ELT(sharedMemory, tmp5, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp6[3] = ROUND_ELT(sharedMemory, tmp5, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp6[4] = ROUND_ELT(sharedMemory, tmp5, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp6[5] = ROUND_ELT(sharedMemory, tmp5, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp6[6] = ROUND_ELT(sharedMemory, tmp5, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp6[7] = ROUND_ELT(sharedMemory, tmp5, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+24]=tmp6[threadIdx.x];

		uint64_t tmp7[8];
		tmp7[0] = xor1(ROUND_ELT(sharedMemory, tmp6, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[4]);
		tmp7[1] = ROUND_ELT(sharedMemory, tmp6, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp7[2] = ROUND_ELT(sharedMemory, tmp6, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp7[3] = ROUND_ELT(sharedMemory, tmp6, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp7[4] = ROUND_ELT(sharedMemory, tmp6, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp7[5] = ROUND_ELT(sharedMemory, tmp6, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp7[6] = ROUND_ELT(sharedMemory, tmp6, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp7[7] = ROUND_ELT(sharedMemory, tmp6, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+32]=tmp7[threadIdx.x];

		uint64_t tmp8[8];
		tmp8[0] = xor1(ROUND_ELT(sharedMemory, tmp7, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[5]);
		tmp8[1] = ROUND_ELT(sharedMemory, tmp7, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp8[2] = ROUND_ELT(sharedMemory, tmp7, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp8[3] = ROUND_ELT(sharedMemory, tmp7, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp8[4] = ROUND_ELT(sharedMemory, tmp7, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp8[5] = ROUND_ELT(sharedMemory, tmp7, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp8[6] = ROUND_ELT(sharedMemory, tmp7, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp8[7] = ROUND_ELT(sharedMemory, tmp7, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+40]=tmp8[threadIdx.x];

		uint64_t tmp9[8];
		tmp9[0] = xor1(ROUND_ELT(sharedMemory, tmp8, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[6]);
		tmp9[1] = ROUND_ELT(sharedMemory, tmp8, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp9[2] = ROUND_ELT(sharedMemory, tmp8, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp9[3] = ROUND_ELT(sharedMemory, tmp8, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp9[4] = ROUND_ELT(sharedMemory, tmp8, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp9[5] = ROUND_ELT(sharedMemory, tmp8, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp9[6] = ROUND_ELT(sharedMemory, tmp8, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp9[7] = ROUND_ELT(sharedMemory, tmp8, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+48]=tmp9[threadIdx.x];

		uint64_t tmp10[8];
		tmp10[0] = xor1(ROUND_ELT(sharedMemory, tmp9, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[7]);
		tmp10[1] = ROUND_ELT(sharedMemory, tmp9, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp10[2] = ROUND_ELT(sharedMemory, tmp9, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp10[3] = ROUND_ELT(sharedMemory, tmp9, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp10[4] = ROUND_ELT(sharedMemory, tmp9, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp10[5] = ROUND_ELT(sharedMemory, tmp9, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp10[6] = ROUND_ELT(sharedMemory, tmp9, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp10[7] = ROUND_ELT(sharedMemory, tmp9, 7, 6, 5, 4, 3, 2, 1, 0);


		d_tmp[threadIdx.x+56]=tmp10[threadIdx.x];

		uint64_t tmp11[8];
		tmp11[0] = xor1(ROUND_ELT(sharedMemory, tmp10, 0, 7, 6, 5, 4, 3, 2, 1), InitVector_RC[8]);
		tmp11[1] = ROUND_ELT(sharedMemory, tmp10, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp11[2] = ROUND_ELT(sharedMemory, tmp10, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp11[3] = ROUND_ELT(sharedMemory, tmp10, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp11[4] = ROUND_ELT(sharedMemory, tmp10, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp11[5] = ROUND_ELT(sharedMemory, tmp10, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp11[6] = ROUND_ELT(sharedMemory, tmp10, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp11[7] = ROUND_ELT(sharedMemory, tmp10, 7, 6, 5, 4, 3, 2, 1, 0);

		d_tmp[threadIdx.x+64]=tmp11[threadIdx.x];

		if(threadIdx.x==1){
			tmp[0]=ROUND_ELT(sharedMemory,tmp11, 3, 2, 1, 0, 7, 6, 5, 4);
			tmp[1]=ROUND_ELT(sharedMemory,tmp11, 5, 4, 3, 2, 1, 0, 7, 6);
			tmp[4] = xor3(tmp[0],tmp[1],atLastCalc);
			d_xtra[threadIdx.x]=tmp[4];
		}
	}
}

__global__ __launch_bounds__(threadsPerBlock,2)
void whirlpoolx_gpu_hash(uint32_t threads, uint32_t startNounce, uint32_t *resNounce)
{
	__shared__ uint64_t sharedMemory[2048];

	whirlpoolx_getShared(sharedMemory);

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t n[8];
		uint64_t tmp[8];
		uint32_t nounce = startNounce + thread;

		n[1] = xor1(REPLACE_HIDWORD(c_PaddedMessage80[9], cuda_swab32(nounce)),c_xtra[0]);

		uint32_t* n32 = (uint32_t*)&n[0];
		n[0]=sharedMemory[__byte_perm(n32[3], 0, 0x4443) + 1792];
		n[5]=sharedMemory[__byte_perm(n32[3], 0, 0x4440) + 1024];
		n[6]=sharedMemory[__byte_perm(n32[3], 0, 0x4441) + 1280];
		n[7]=sharedMemory[__byte_perm(n32[3], 0, 0x4442) + 1536];
		n[0]=xor1(c_tmp[0],n[0]);
		n[1]=c_tmp[1];
		n[2]=c_tmp[2];
		n[3]=c_tmp[3];
		n[4]=c_tmp[4];
		n[5]=xor1(c_tmp[5],n[5]);
		n[6]=xor1(c_tmp[6],n[6]);
		n[7]=xor1(c_tmp[7],n[7]);

		tmp[0]=xor3(sharedMemory[__byte_perm(n32[10],0,0x4443)+768],sharedMemory[__byte_perm(n32[12],0,0x4442)+512],sharedMemory[__byte_perm(n32[14],0,0x4441)+256]);
		tmp[1]=xor3(sharedMemory[__byte_perm(n32[11],0,0x4440)+1024],sharedMemory[__byte_perm(n32[12],0,0x4443)+768],sharedMemory[__byte_perm(n32[14],0,0x4442)+512]);
		tmp[2]=xor3(sharedMemory[__byte_perm(n32[11],0,0x4441)+1280],sharedMemory[__byte_perm(n32[13],0,0x4440)+1024],sharedMemory[__byte_perm(n32[14],0,0x4443)+768]);
		tmp[3]=xor3(sharedMemory[__byte_perm(n32[11],0,0x4442)+1536],sharedMemory[__byte_perm(n32[13],0,0x4441)+1280],sharedMemory[__byte_perm(n32[15],0,0x4440)+1024]);
		tmp[4]=xor3(sharedMemory[__byte_perm(n32[11],0,0x4443)+1792],sharedMemory[__byte_perm(n32[13],0,0x4442)+1536],sharedMemory[__byte_perm(n32[15],0,0x4441)+1280]);
		tmp[5]=xor3(sharedMemory[__byte_perm(n32[10],0,0x4440)],sharedMemory[__byte_perm(n32[13],0,0x4443)+1792],sharedMemory[__byte_perm(n32[15],0,0x4442)+1536]);
		tmp[6]=xor3(sharedMemory[__byte_perm(n32[12],0,0x4440)],sharedMemory[__byte_perm(n32[10],0,0x4441)+256],sharedMemory[__byte_perm(n32[15],0,0x4443)+1792]);
		tmp[7]=xor3(sharedMemory[__byte_perm(n32[14],0,0x4440)],sharedMemory[__byte_perm(n32[12],0,0x4441)+256],sharedMemory[__byte_perm(n32[10],0,0x4442)+ 512]);

		tmp[0]=xor3(sharedMemory[__byte_perm(n32[ 0], 0, 0x4440)],tmp[0],c_tmp[0+16]);
		tmp[1]=xor3(sharedMemory[__byte_perm(n32[ 0], 0, 0x4441) + 256],tmp[1],c_tmp[1+16]);
		tmp[2]=xor3(sharedMemory[__byte_perm(n32[ 0], 0, 0x4442) +  512],tmp[2],c_tmp[2+16]);
		tmp[3]=xor3(sharedMemory[__byte_perm(n32[ 0], 0, 0x4443) + 768],tmp[3],c_tmp[3+16]);
		tmp[4]=xor3(sharedMemory[__byte_perm(n32[ 1], 0, 0x4440) + 1024],tmp[4],c_tmp[4+16]);
		tmp[5]=xor3(sharedMemory[__byte_perm(n32[ 1], 0, 0x4441) + 1280],tmp[5],c_tmp[5+16]);
		tmp[6]=xor3(sharedMemory[__byte_perm(n32[ 1], 0, 0x4442) + 1536],tmp[6],c_tmp[6+16]);
		tmp[7]=xor3(sharedMemory[__byte_perm(n32[ 1], 0, 0x4443) + 1792],tmp[7],c_tmp[7+16]);

		n[0]=tmp[0];
		n[1]=tmp[1];
		n[2]=tmp[2];
		n[3]=tmp[3];
		n[4]=tmp[4];
		n[5]=tmp[5];
		n[6]=tmp[6];
		n[7]=tmp[7];

		tmp[0] = xor1(ROUND_ELT(sharedMemory, n, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+8]);
		tmp[1] = xor1(ROUND_ELT(sharedMemory, n, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+8]);
		tmp[2] = xor1(ROUND_ELT(sharedMemory, n, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+8]);
		tmp[3] = xor1(ROUND_ELT(sharedMemory, n, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+8]);
		tmp[4] = xor1(ROUND_ELT(sharedMemory, n, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+8]);
		tmp[5] = xor1(ROUND_ELT(sharedMemory, n, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+8]);
		tmp[6] = xor1(ROUND_ELT(sharedMemory, n, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+8]);
		tmp[7] = xor1(ROUND_ELT(sharedMemory, n, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+8]);

		n[0] = xor1(ROUND_ELT(sharedMemory, tmp, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+24]);
		n[1] = xor1(ROUND_ELT(sharedMemory, tmp, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+24]);
		n[2] = xor1(ROUND_ELT(sharedMemory, tmp, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+24]);
		n[3] = xor1(ROUND_ELT(sharedMemory, tmp, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+24]);
		n[4] = xor1(ROUND_ELT(sharedMemory, tmp, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+24]);
		n[5] = xor1(ROUND_ELT(sharedMemory, tmp, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+24]);
		n[6] = xor1(ROUND_ELT(sharedMemory, tmp, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+24]);
		n[7] = xor1(ROUND_ELT(sharedMemory, tmp, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+24]);

		tmp[0] = xor1(ROUND_ELT(sharedMemory, n, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+32]);
		tmp[1] = xor1(ROUND_ELT(sharedMemory, n, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+32]);
		tmp[2] = xor1(ROUND_ELT(sharedMemory, n, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+32]);
		tmp[3] = xor1(ROUND_ELT(sharedMemory, n, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+32]);
		tmp[4] = xor1(ROUND_ELT(sharedMemory, n, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+32]);
		tmp[5] = xor1(ROUND_ELT(sharedMemory, n, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+32]);
		tmp[6] = xor1(ROUND_ELT(sharedMemory, n, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+32]);
		tmp[7] = xor1(ROUND_ELT(sharedMemory, n, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+32]);

		n[0] = xor1(ROUND_ELT(sharedMemory, tmp, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+40]);
		n[1] = xor1(ROUND_ELT(sharedMemory, tmp, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+40]);
		n[2] = xor1(ROUND_ELT(sharedMemory, tmp, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+40]);
		n[3] = xor1(ROUND_ELT(sharedMemory, tmp, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+40]);
		n[4] = xor1(ROUND_ELT(sharedMemory, tmp, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+40]);
		n[5] = xor1(ROUND_ELT(sharedMemory, tmp, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+40]);
		n[6] = xor1(ROUND_ELT(sharedMemory, tmp, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+40]);
		n[7] = xor1(ROUND_ELT(sharedMemory, tmp, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+40]);

		tmp[0] = xor1(ROUND_ELT(sharedMemory, n, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+48]);
		tmp[1] = xor1(ROUND_ELT(sharedMemory, n, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+48]);
		tmp[2] = xor1(ROUND_ELT(sharedMemory, n, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+48]);
		tmp[3] = xor1(ROUND_ELT(sharedMemory, n, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+48]);
		tmp[4] = xor1(ROUND_ELT(sharedMemory, n, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+48]);
		tmp[5] = xor1(ROUND_ELT(sharedMemory, n, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+48]);
		tmp[6] = xor1(ROUND_ELT(sharedMemory, n, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+48]);
		tmp[7] = xor1(ROUND_ELT(sharedMemory, n, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+48]);

		n[0] = xor1(ROUND_ELT(sharedMemory, tmp, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+56]);
		n[1] = xor1(ROUND_ELT(sharedMemory, tmp, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+56]);
		n[2] = xor1(ROUND_ELT(sharedMemory, tmp, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+56]);
		n[3] = xor1(ROUND_ELT(sharedMemory, tmp, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+56]);
		n[4] = xor1(ROUND_ELT(sharedMemory, tmp, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+56]);
		n[5] = xor1(ROUND_ELT(sharedMemory, tmp, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+56]);
		n[6] = xor1(ROUND_ELT(sharedMemory, tmp, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+56]);
		n[7] = xor1(ROUND_ELT(sharedMemory, tmp, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+56]);

		tmp[0] = xor1(ROUND_ELT(sharedMemory, n, 0, 7, 6, 5, 4, 3, 2, 1), c_tmp[0+64]);
		tmp[1] = xor1(ROUND_ELT(sharedMemory, n, 1, 0, 7, 6, 5, 4, 3, 2), c_tmp[1+64]);
		tmp[2] = xor1(ROUND_ELT(sharedMemory, n, 2, 1, 0, 7, 6, 5, 4, 3), c_tmp[2+64]);
		tmp[3] = xor1(ROUND_ELT(sharedMemory, n, 3, 2, 1, 0, 7, 6, 5, 4), c_tmp[3+64]);
		tmp[4] = xor1(ROUND_ELT(sharedMemory, n, 4, 3, 2, 1, 0, 7, 6, 5), c_tmp[4+64]);
		tmp[5] = xor1(ROUND_ELT(sharedMemory, n, 5, 4, 3, 2, 1, 0, 7, 6), c_tmp[5+64]);
		tmp[6] = xor1(ROUND_ELT(sharedMemory, n, 6, 5, 4, 3, 2, 1, 0, 7), c_tmp[6+64]);
		tmp[7] = xor1(ROUND_ELT(sharedMemory, n, 7, 6, 5, 4, 3, 2, 1, 0), c_tmp[7+64]);

		if (xor3(c_xtra[1], ROUND_ELT(sharedMemory, tmp, 3, 2, 1, 0, 7, 6, 5, 4), ROUND_ELT(sharedMemory, tmp, 5, 4, 3, 2, 1, 0, 7, 6)) <= pTarget[3]) {
			atomicMin(&resNounce[0], nounce);
		}
	}
}

__host__
extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyToSymbol(HIP_SYMBOL(InitVector_RC), plain_RC, sizeof(plain_RC), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(mixTob0Tox), plain_T0, sizeof(plain_T0), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_WXNonce[thr_id], sizeof(uint32_t));
	hipHostMalloc(&h_wxnounce[thr_id], sizeof(uint32_t));
	hipMalloc(&d_xtra[thr_id], 8 * sizeof(uint64_t));
	CUDA_SAFE_CALL(hipMalloc(&d_tmp[thr_id], 8 * 9 * sizeof(uint64_t))); // d_tmp[threadIdx.x+64] (7+64)
}

__host__
extern void whirlpoolx_cpu_free(int thr_id)
{
	hipFree(d_WXNonce[thr_id]);
	hipHostFree(h_wxnounce[thr_id]);
	hipFree(d_xtra[thr_id]);
	hipFree(d_tmp[thr_id]);
}

__host__
void whirlpoolx_setBlock_80(void *pdata, const void *ptarget)
{
	uint64_t PaddedMessage[16];
	memcpy(PaddedMessage, pdata, 80);
	memset((uint8_t*)&PaddedMessage+80, 0, 48);
	((uint8_t*)PaddedMessage)[80] = 0x80; /* ending */
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 4*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void whirlpoolx_precompute(int thr_id)
{
	dim3 grid(1);
	dim3 block(256);

	whirlpoolx_gpu_precompute <<<grid, block>>>(8, d_xtra[thr_id], d_tmp[thr_id]);
	hipDeviceSynchronize();

	hipMemcpyToSymbol(HIP_SYMBOL(c_xtra), d_xtra[thr_id], 8 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_tmp), d_tmp[thr_id], 8 * 9 * sizeof(uint64_t), 0, hipMemcpyDeviceToDevice));
}

__host__
uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce)
{
	dim3 grid((threads + threadsPerBlock-1) / threadsPerBlock);
	dim3 block(threadsPerBlock);

	hipMemset(d_WXNonce[thr_id], 0xff, sizeof(uint32_t));

	whirlpoolx_gpu_hash<<<grid, block>>>(threads, startNounce, d_WXNonce[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_wxnounce[thr_id], d_WXNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	return *(h_wxnounce[thr_id]);
}
