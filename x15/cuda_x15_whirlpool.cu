#include "hip/hip_runtime.h"
/**
 * Whirlpool-512 CUDA implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014-2016 djm34, tpruvot, SP, Provos Alexis
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 * @author djm34 (initial draft)
 * @author tpruvot (dual old/whirlpool modes, midstate)
 * @author SP ("final" function opt and tuning)
 * @author Provos Alexis (Applied partial shared memory utilization, precomputations, merging & tuning for 970/750ti under CUDA7.5 -> +93% increased throughput of whirlpool)
 */


// Change with caution, used by shared mem fetch
#define TPB80 384
#define TPB64 384

extern "C" {
#include <sph/sph_whirlpool.h>
#include <miner.h>
}

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>
#include <cuda_vectors.h>

#define xor3x(a,b,c) (a^b^c)

#include "cuda_whirlpool_tables.cuh"

__device__ static uint64_t b0[256];
__device__ static uint64_t b7[256];

__constant__ static uint2 precomputed_round_key_64[72];
__constant__ static uint2 precomputed_round_key_80[80];

__device__ static uint2 c_PaddedMessage80[16];

/**
 * Round constants.
 */
__device__ uint2 InitVector_RC[10];

static uint32_t *d_resNonce[MAX_GPUS] = { 0 };

//--------START OF WHIRLPOOL DEVICE MACROS---------------------------------------------------------------------------
__device__ __forceinline__
void static TRANSFER(uint2 *const __restrict__ dst,const uint2 *const __restrict__ src){
	dst[0] = src[ 0];
	dst[1] = src[ 1];
	dst[2] = src[ 2];
	dst[3] = src[ 3];
	dst[4] = src[ 4];
	dst[5] = src[ 5];
	dst[6] = src[ 6];
	dst[7] = src[ 7];
}

__device__ __forceinline__
static uint2 d_ROUND_ELT_LDG(const uint2 sharedMemory[7][256],const uint2 *const __restrict__ in,const int i0, const int i1, const int i2, const int i3, const int i4, const int i5, const int i6, const int i7){
	uint2 ret = __ldg((uint2*)&b0[__byte_perm(in[i0].x, 0, 0x4440)]);
	ret ^= sharedMemory[1][__byte_perm(in[i1].x, 0, 0x4441)];
	ret ^= sharedMemory[2][__byte_perm(in[i2].x, 0, 0x4442)];
	ret ^= sharedMemory[3][__byte_perm(in[i3].x, 0, 0x4443)];
	ret ^= sharedMemory[4][__byte_perm(in[i4].y, 0, 0x4440)];
	ret ^= ROR24(__ldg((uint2*)&b0[__byte_perm(in[i5].y, 0, 0x4441)]));
	ret ^= ROR8(__ldg((uint2*)&b7[__byte_perm(in[i6].y, 0, 0x4442)]));
	ret ^= __ldg((uint2*)&b7[__byte_perm(in[i7].y, 0, 0x4443)]);
	return ret;
}

__device__ __forceinline__
static uint2 d_ROUND_ELT(const uint2 sharedMemory[7][256],const uint2 *const __restrict__ in,const int i0, const int i1, const int i2, const int i3, const int i4, const int i5, const int i6, const int i7){

	uint2 ret = __ldg((uint2*)&b0[__byte_perm(in[i0].x, 0, 0x4440)]);
	ret ^= sharedMemory[1][__byte_perm(in[i1].x, 0, 0x4441)];
	ret ^= sharedMemory[2][__byte_perm(in[i2].x, 0, 0x4442)];
	ret ^= sharedMemory[3][__byte_perm(in[i3].x, 0, 0x4443)];
	ret ^= sharedMemory[4][__byte_perm(in[i4].y, 0, 0x4440)];
	ret ^= sharedMemory[5][__byte_perm(in[i5].y, 0, 0x4441)];
	ret ^= ROR8(__ldg((uint2*)&b7[__byte_perm(in[i6].y, 0, 0x4442)]));
	ret ^= __ldg((uint2*)&b7[__byte_perm(in[i7].y, 0, 0x4443)]);
	return ret;
}

__device__ __forceinline__
static uint2 d_ROUND_ELT1_LDG(const uint2 sharedMemory[7][256],const uint2 *const __restrict__ in,const int i0, const int i1, const int i2, const int i3, const int i4, const int i5, const int i6, const int i7, const uint2 c0){

	uint2 ret = __ldg((uint2*)&b0[__byte_perm(in[i0].x, 0, 0x4440)]);
	ret ^= sharedMemory[1][__byte_perm(in[i1].x, 0, 0x4441)];
	ret ^= sharedMemory[2][__byte_perm(in[i2].x, 0, 0x4442)];
	ret ^= sharedMemory[3][__byte_perm(in[i3].x, 0, 0x4443)];
	ret ^= sharedMemory[4][__byte_perm(in[i4].y, 0, 0x4440)];
	ret ^= ROR24(__ldg((uint2*)&b0[__byte_perm(in[i5].y, 0, 0x4441)]));
	ret ^= ROR8(__ldg((uint2*)&b7[__byte_perm(in[i6].y, 0, 0x4442)]));
	ret ^= __ldg((uint2*)&b7[__byte_perm(in[i7].y, 0, 0x4443)]);
	ret ^= c0;
	return ret;
}

__device__ __forceinline__
static uint2 d_ROUND_ELT1(const uint2 sharedMemory[7][256],const uint2 *const __restrict__ in,const int i0, const int i1, const int i2, const int i3, const int i4, const int i5, const int i6, const int i7, const uint2 c0){
	uint2 ret = __ldg((uint2*)&b0[__byte_perm(in[i0].x, 0, 0x4440)]);
	ret ^= sharedMemory[1][__byte_perm(in[i1].x, 0, 0x4441)];
	ret ^= sharedMemory[2][__byte_perm(in[i2].x, 0, 0x4442)];
	ret ^= sharedMemory[3][__byte_perm(in[i3].x, 0, 0x4443)];
	ret ^= sharedMemory[4][__byte_perm(in[i4].y, 0, 0x4440)];
	ret ^= sharedMemory[5][__byte_perm(in[i5].y, 0, 0x4441)];
	ret ^= ROR8(__ldg((uint2*)&b7[__byte_perm(in[i6].y, 0, 0x4442)]));//sharedMemory[6][__byte_perm(in[i6].y, 0, 0x4442)]
	ret ^= __ldg((uint2*)&b7[__byte_perm(in[i7].y, 0, 0x4443)]);//sharedMemory[7][__byte_perm(in[i7].y, 0, 0x4443)]
	ret ^= c0;
	return ret;
}

//--------END OF WHIRLPOOL DEVICE MACROS-----------------------------------------------------------------------------

//--------START OF WHIRLPOOL HOST MACROS-----------------------------------------------------------------------------

#define table_skew(val,num) SPH_ROTL64(val,8*num)
#define BYTE(x, n)     ((unsigned)((x) >> (8 * (n))) & 0xFF)

#define ROUND_ELT(table, in, i0, i1, i2, i3, i4, i5, i6, i7) \
	(table[BYTE(in[i0], 0)] \
	^ table_skew(table[BYTE(in[i1], 1)], 1) \
	^ table_skew(table[BYTE(in[i2], 2)], 2) \
	^ table_skew(table[BYTE(in[i3], 3)], 3) \
	^ table_skew(table[BYTE(in[i4], 4)], 4) \
	^ table_skew(table[BYTE(in[i5], 5)], 5) \
	^ table_skew(table[BYTE(in[i6], 6)], 6) \
	^ table_skew(table[BYTE(in[i7], 7)], 7))

#define ROUND(table, in, out, c0, c1, c2, c3, c4, c5, c6, c7)   do { \
		out[0] = ROUND_ELT(table, in, 0, 7, 6, 5, 4, 3, 2, 1) ^ c0; \
		out[1] = ROUND_ELT(table, in, 1, 0, 7, 6, 5, 4, 3, 2) ^ c1; \
		out[2] = ROUND_ELT(table, in, 2, 1, 0, 7, 6, 5, 4, 3) ^ c2; \
		out[3] = ROUND_ELT(table, in, 3, 2, 1, 0, 7, 6, 5, 4) ^ c3; \
		out[4] = ROUND_ELT(table, in, 4, 3, 2, 1, 0, 7, 6, 5) ^ c4; \
		out[5] = ROUND_ELT(table, in, 5, 4, 3, 2, 1, 0, 7, 6) ^ c5; \
		out[6] = ROUND_ELT(table, in, 6, 5, 4, 3, 2, 1, 0, 7) ^ c6; \
		out[7] = ROUND_ELT(table, in, 7, 6, 5, 4, 3, 2, 1, 0) ^ c7; \
	} while (0)

__host__
static void ROUND_KSCHED(const uint64_t *in,uint64_t *out,const uint64_t c){
	const uint64_t *a = in;
	uint64_t *b = out;
	ROUND(old1_T0, a, b, c, 0, 0, 0, 0, 0, 0, 0);
}


//--------END OF WHIRLPOOL HOST MACROS-------------------------------------------------------------------------------

__host__
void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode)
{
	uint64_t* table0 = NULL;

	switch (mode) {
	case 0: /* x15 with rotated T1-T7 (based on T0) */
		table0 = (uint64_t*)plain_T0;
		hipMemcpyToSymbol(HIP_SYMBOL(InitVector_RC), plain_RC, 10*sizeof(uint64_t),0, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(precomputed_round_key_64), plain_precomputed_round_key_64, 72*sizeof(uint64_t),0, hipMemcpyHostToDevice);
		break;
	case 1: /* old whirlpool */
		table0 = (uint64_t*)old1_T0;
		hipMemcpyToSymbol(HIP_SYMBOL(InitVector_RC), old1_RC, 10*sizeof(uint64_t),0,hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(precomputed_round_key_64), old1_precomputed_round_key_64, 72*sizeof(uint64_t),0, hipMemcpyHostToDevice);
		break;
	default:
		applog(LOG_ERR,"Bad whirlpool mode");
		exit(0);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(b0), table0, 256*sizeof(uint64_t),0, hipMemcpyHostToDevice);
	uint64_t table7[256];
	for(int i=0;i<256;i++){
		table7[i] = ROTR64(table0[i],8);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(b7), table7, 256*sizeof(uint64_t),0, hipMemcpyHostToDevice);

	CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));

	cuda_get_arch(thr_id);
}

__host__
static void whirl_midstate(void *state, const void *input)
{
	sph_whirlpool_context ctx;

	sph_whirlpool1_init(&ctx);
	sph_whirlpool1(&ctx, input, 64);

	memcpy(state, ctx.state, 64);
}

__host__
void whirlpool512_setBlock_80(void *pdata, const void *ptarget)
{
	uint64_t PaddedMessage[16];

	memcpy(PaddedMessage, pdata, 80);
	memset(((uint8_t*)&PaddedMessage)+80, 0, 48);
	((uint8_t*)&PaddedMessage)[80] = 0x80; /* ending */

	// compute constant first block
	uint64_t midstate[16] = { 0 };
	whirl_midstate(midstate, pdata);
	memcpy(PaddedMessage, midstate, 64);

	uint64_t round_constants[80];
	uint64_t n[8];

	n[0] = PaddedMessage[0] ^ PaddedMessage[8];    //read data
	n[1] = PaddedMessage[1] ^ PaddedMessage[9];
	n[2] = PaddedMessage[2] ^ 0x0000000000000080; //whirlpool
	n[3] = PaddedMessage[3];
	n[4] = PaddedMessage[4];
	n[5] = PaddedMessage[5];
	n[6] = PaddedMessage[6];
	n[7] = PaddedMessage[7] ^ 0x8002000000000000;

	ROUND_KSCHED(PaddedMessage,round_constants,old1_RC[0]);

	for(int i=1;i<10;i++){
		ROUND_KSCHED(&round_constants[8*(i-1)],&round_constants[8*i],old1_RC[i]);
	}

	//USE the same memory place to store keys and state
	round_constants[ 0]^= old1_T0[BYTE(n[0], 0)]
	 ^ table_skew(old1_T0[BYTE(n[7], 1)], 1) ^ table_skew(old1_T0[BYTE(n[6], 2)], 2) ^ table_skew(old1_T0[BYTE(n[5], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[4], 4)], 4) ^ table_skew(old1_T0[BYTE(n[3], 5)], 5) ^ table_skew(old1_T0[BYTE(n[2], 6)], 6);

	round_constants[ 1]^= old1_T0[BYTE(n[1], 0)]
	 ^ table_skew(old1_T0[BYTE(n[0], 1)], 1) ^ table_skew(old1_T0[BYTE(n[7], 2)], 2) ^ table_skew(old1_T0[BYTE(n[6], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[5], 4)], 4) ^ table_skew(old1_T0[BYTE(n[4], 5)], 5) ^ table_skew(old1_T0[BYTE(n[3], 6)], 6)
	 ^ table_skew(old1_T0[BYTE(n[2], 7)], 7);

	round_constants[ 2]^= old1_T0[BYTE(n[2], 0)]
	 ^ table_skew(old1_T0[BYTE(n[1], 1)], 1) ^ table_skew(old1_T0[BYTE(n[0], 2)], 2) ^ table_skew(old1_T0[BYTE(n[7], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[6], 4)], 4) ^ table_skew(old1_T0[BYTE(n[5], 5)], 5) ^ table_skew(old1_T0[BYTE(n[4], 6)], 6)
	 ^ table_skew(old1_T0[BYTE(n[3], 7)], 7);

	round_constants[ 3]^= old1_T0[BYTE(n[3], 0)]
	 ^ table_skew(old1_T0[BYTE(n[2], 1)], 1) ^ table_skew(old1_T0[BYTE(n[1], 2)], 2) ^ table_skew(old1_T0[BYTE(n[0], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[7], 4)], 4) ^ table_skew(old1_T0[BYTE(n[6], 5)], 5) ^ table_skew(old1_T0[BYTE(n[5], 6)], 6)
	 ^ table_skew(old1_T0[BYTE(n[4], 7)], 7);

	round_constants[ 4]^= old1_T0[BYTE(n[4], 0)]
	 ^ table_skew(old1_T0[BYTE(n[3], 1)], 1) ^ table_skew(old1_T0[BYTE(n[2], 2)], 2) ^ table_skew(old1_T0[BYTE(n[1], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[0], 4)], 4) ^ table_skew(old1_T0[BYTE(n[7], 5)], 5) ^ table_skew(old1_T0[BYTE(n[6], 6)], 6)
	 ^ table_skew(old1_T0[BYTE(n[5], 7)], 7);

	round_constants[ 5]^= old1_T0[BYTE(n[5], 0)]
	 ^ table_skew(old1_T0[BYTE(n[4], 1)], 1) ^ table_skew(old1_T0[BYTE(n[3], 2)], 2) ^ table_skew(old1_T0[BYTE(n[2], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[0], 5)], 5) ^ table_skew(old1_T0[BYTE(n[7], 6)], 6) ^ table_skew(old1_T0[BYTE(n[6], 7)], 7);

	round_constants[ 6]^= old1_T0[BYTE(n[6], 0)]
	 ^ table_skew(old1_T0[BYTE(n[5], 1)], 1) ^ table_skew(old1_T0[BYTE(n[4], 2)], 2) ^ table_skew(old1_T0[BYTE(n[3], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[2], 4)], 4) ^ table_skew(old1_T0[BYTE(n[0], 6)], 6) ^ table_skew(old1_T0[BYTE(n[7], 7)], 7);

	round_constants[ 7]^= old1_T0[BYTE(n[7], 0)]
	 ^ table_skew(old1_T0[BYTE(n[6], 1)], 1) ^ table_skew(old1_T0[BYTE(n[5], 2)], 2) ^ table_skew(old1_T0[BYTE(n[4], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[3], 4)], 4) ^ table_skew(old1_T0[BYTE(n[2], 5)], 5) ^ table_skew(old1_T0[BYTE(n[0], 7)], 7);

	for(int i=1;i<5;i++)
		n[i] = round_constants[i];

	round_constants[ 8]^= table_skew(old1_T0[BYTE(n[4], 4)], 4)
	 ^ table_skew(old1_T0[BYTE(n[3], 5)], 5) ^ table_skew(old1_T0[BYTE(n[2], 6)], 6) ^ table_skew(old1_T0[BYTE(n[1], 7)], 7);

	round_constants[ 9]^= old1_T0[BYTE(n[1], 0)]
	 ^ table_skew(old1_T0[BYTE(n[4], 5)], 5) ^ table_skew(old1_T0[BYTE(n[3], 6)], 6) ^ table_skew(old1_T0[BYTE(n[2], 7)], 7);

	round_constants[10]^= old1_T0[BYTE(n[2], 0)]
	 ^ table_skew(old1_T0[BYTE(n[1], 1)], 1) ^ table_skew(old1_T0[BYTE(n[4], 6)], 6) ^ table_skew(old1_T0[BYTE(n[3], 7)], 7);

	round_constants[11]^= old1_T0[BYTE(n[3], 0)]
	 ^ table_skew(old1_T0[BYTE(n[2], 1)], 1) ^ table_skew(old1_T0[BYTE(n[1], 2)], 2) ^ table_skew(old1_T0[BYTE(n[4], 7)], 7);

	round_constants[12]^= old1_T0[BYTE(n[4], 0)]
	 ^ table_skew(old1_T0[BYTE(n[3], 1)], 1) ^ table_skew(old1_T0[BYTE(n[2], 2)], 2) ^ table_skew(old1_T0[BYTE(n[1], 3)], 3);

	round_constants[13]^= table_skew(old1_T0[BYTE(n[4], 1)], 1) ^ table_skew(old1_T0[BYTE(n[3], 2)], 2)
	 ^ table_skew(old1_T0[BYTE(n[2], 3)], 3) ^ table_skew(old1_T0[BYTE(n[1], 4)], 4);

	round_constants[14]^= table_skew(old1_T0[BYTE(n[4], 2)], 2) ^ table_skew(old1_T0[BYTE(n[3], 3)], 3)
	 ^ table_skew(old1_T0[BYTE(n[2], 4)], 4) ^ table_skew(old1_T0[BYTE(n[1], 5)], 5);

	round_constants[15]^= table_skew(old1_T0[BYTE(n[4], 3)], 3) ^  table_skew(old1_T0[BYTE(n[3], 4)], 4)
	 ^ table_skew(old1_T0[BYTE(n[2], 5)], 5) ^ table_skew(old1_T0[BYTE(n[1], 6)], 6);

	PaddedMessage[0] ^= PaddedMessage[8];

	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 128, 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(precomputed_round_key_80), round_constants, 80*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__
extern void x15_whirlpool_cpu_free(int thr_id)
{
	if (d_resNonce[thr_id])
		hipFree(d_resNonce[thr_id]);
}

__global__
__launch_bounds__(TPB80,2)
void oldwhirlpool_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t* resNonce, const uint64_t target)
{
	__shared__ uint2 sharedMemory[7][256];

	if (threadIdx.x < 256) {
		const uint2 tmp = __ldg((uint2*)&b0[threadIdx.x]);
		sharedMemory[0][threadIdx.x] = tmp;
		sharedMemory[1][threadIdx.x] = ROL8(tmp);
		sharedMemory[2][threadIdx.x] = ROL16(tmp);
		sharedMemory[3][threadIdx.x] = ROL24(tmp);
		sharedMemory[4][threadIdx.x] = SWAPUINT2(tmp);
		sharedMemory[5][threadIdx.x] = ROR24(tmp);
		sharedMemory[6][threadIdx.x] = ROR16(tmp);
	}

	__syncthreads();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads){

		uint2 hash[8], state[8],n[8], tmp[8];
		uint32_t nonce = cuda_swab32(startNounce + thread);
		uint2 temp = c_PaddedMessage80[9];
		temp.y = nonce;

		/// round 2 ///////
		//////////////////////////////////
		temp = temp ^ c_PaddedMessage80[1];

		*(uint2x4*)&n[ 0]   = *(uint2x4*)&precomputed_round_key_80[ 0];
		*(uint2x4*)&n[ 4]   = *(uint2x4*)&precomputed_round_key_80[ 4];
		*(uint2x4*)&tmp[ 0] = *(uint2x4*)&precomputed_round_key_80[ 8];
		*(uint2x4*)&tmp[ 4] = *(uint2x4*)&precomputed_round_key_80[12];

		n[ 0]^= __ldg((uint2*)&b7[__byte_perm(temp.y, 0, 0x4443)]);
		n[ 5]^= sharedMemory[4][__byte_perm(temp.y, 0, 0x4440)];
		n[ 6]^= sharedMemory[5][__byte_perm(temp.y, 0, 0x4441)];
		n[ 7]^= sharedMemory[6][__byte_perm(temp.y, 0, 0x4442)];

		tmp[ 0]^= __ldg((uint2*)&b0[__byte_perm(n[0].x, 0, 0x4440)]);
		tmp[ 0]^= sharedMemory[1][__byte_perm(n[7].x, 0, 0x4441)];
		tmp[ 0]^= sharedMemory[2][__byte_perm(n[6].x, 0, 0x4442)];
		tmp[ 0]^= sharedMemory[3][__byte_perm(n[5].x, 0, 0x4443)];

		tmp[ 1]^= sharedMemory[1][__byte_perm(n[0].x, 0, 0x4441)];
		tmp[ 1]^= sharedMemory[2][__byte_perm(n[7].x, 0, 0x4442)];
		tmp[ 1]^= sharedMemory[3][__byte_perm(n[6].x, 0, 0x4443)];
		tmp[ 1]^= sharedMemory[4][__byte_perm(n[5].y, 0, 0x4440)];

		tmp[ 2]^= sharedMemory[2][__byte_perm(n[0].x, 0, 0x4442)];
		tmp[ 2]^= sharedMemory[3][__byte_perm(n[7].x, 0, 0x4443)];
		tmp[ 2]^= sharedMemory[4][__byte_perm(n[6].y, 0, 0x4440)];
		tmp[ 2]^= sharedMemory[5][__byte_perm(n[5].y, 0, 0x4441)];

		tmp[ 3]^= sharedMemory[3][__byte_perm(n[0].x, 0, 0x4443)];
		tmp[ 3]^= sharedMemory[4][__byte_perm(n[7].y, 0, 0x4440)];
		tmp[ 3]^= ROR24(__ldg((uint2*)&b0[__byte_perm(n[6].y, 0, 0x4441)]));
		tmp[ 3]^= ROR8(__ldg((uint2*)&b7[__byte_perm(n[5].y, 0, 0x4442)]));

		tmp[ 4]^= sharedMemory[4][__byte_perm(n[0].y, 0, 0x4440)];
		tmp[ 4]^= sharedMemory[5][__byte_perm(n[7].y, 0, 0x4441)];
		tmp[ 4]^= ROR8(__ldg((uint2*)&b7[__byte_perm(n[6].y, 0, 0x4442)]));
		tmp[ 4]^= __ldg((uint2*)&b7[__byte_perm(n[5].y, 0, 0x4443)]);

		tmp[ 5]^= __ldg((uint2*)&b0[__byte_perm(n[5].x, 0, 0x4440)]);
		tmp[ 5]^= sharedMemory[5][__byte_perm(n[0].y, 0, 0x4441)];
		tmp[ 5]^= sharedMemory[6][__byte_perm(n[7].y, 0, 0x4442)];
		tmp[ 5]^= __ldg((uint2*)&b7[__byte_perm(n[6].y, 0, 0x4443)]);

		tmp[ 6]^= __ldg((uint2*)&b0[__byte_perm(n[6].x, 0, 0x4440)]);
		tmp[ 6]^= sharedMemory[1][__byte_perm(n[5].x, 0, 0x4441)];
		tmp[ 6]^= sharedMemory[6][__byte_perm(n[0].y, 0, 0x4442)];
		tmp[ 6]^= __ldg((uint2*)&b7[__byte_perm(n[7].y, 0, 0x4443)]);

		tmp[ 7]^= __ldg((uint2*)&b0[__byte_perm(n[7].x, 0, 0x4440)]);
		tmp[ 7]^= sharedMemory[1][__byte_perm(n[6].x, 0, 0x4441)];
		tmp[ 7]^= sharedMemory[2][__byte_perm(n[5].x, 0, 0x4442)];
		tmp[ 7]^= __ldg((uint2*)&b7[__byte_perm(n[0].y, 0, 0x4443)]);

		TRANSFER(n, tmp);

		for (int i=2; i<10; i++) {
			tmp[ 0] = d_ROUND_ELT1_LDG(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, precomputed_round_key_80[i*8+0]);
			tmp[ 1] = d_ROUND_ELT1(    sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, precomputed_round_key_80[i*8+1]);
			tmp[ 2] = d_ROUND_ELT1(    sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, precomputed_round_key_80[i*8+2]);
			tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, precomputed_round_key_80[i*8+3]);
			tmp[ 4] = d_ROUND_ELT1_LDG(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, precomputed_round_key_80[i*8+4]);
			tmp[ 5] = d_ROUND_ELT1(    sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, precomputed_round_key_80[i*8+5]);
			tmp[ 6] = d_ROUND_ELT1(    sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, precomputed_round_key_80[i*8+6]);
			tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, precomputed_round_key_80[i*8+7]);
			TRANSFER(n, tmp);
		}

		state[0] = c_PaddedMessage80[0] ^ n[0];
		state[1] = c_PaddedMessage80[1] ^ n[1] ^ vectorize(REPLACE_HIDWORD(devectorize(c_PaddedMessage80[9]),nonce));
		state[2] = c_PaddedMessage80[2] ^ n[2] ^ vectorize(0x0000000000000080);
		state[3] = c_PaddedMessage80[3] ^ n[3];
		state[4] = c_PaddedMessage80[4] ^ n[4];
		state[5] = c_PaddedMessage80[5] ^ n[5];
		state[6] = c_PaddedMessage80[6] ^ n[6];
		state[7] = c_PaddedMessage80[7] ^ n[7] ^ vectorize(0x8002000000000000);

		#pragma unroll 2
		for(int r=0;r<2;r++){
			#pragma unroll 8
			for(int i=0;i<8;i++)
				hash[ i] = n[ i] = state[ i];

			uint2 h[8] = {
				{0xC0EE0B30,0x672990AF},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},
				{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828}
			};

			tmp[ 0] = d_ROUND_ELT1_LDG(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, h[0]);
			tmp[ 1] = d_ROUND_ELT1(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, h[1]);
			tmp[ 2] = d_ROUND_ELT1(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, h[2]);
			tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, h[3]);
			tmp[ 4] = d_ROUND_ELT1(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, h[4]);
			tmp[ 5] = d_ROUND_ELT1_LDG(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, h[5]);
			tmp[ 6] = d_ROUND_ELT1(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, h[6]);
			tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, h[7]);
			TRANSFER(n, tmp);
	//		#pragma unroll 10
			for (int i=1; i <10; i++){
				tmp[ 0] = d_ROUND_ELT1_LDG(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, precomputed_round_key_64[(i-1)*8+0]);
				tmp[ 1] = d_ROUND_ELT1(    sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, precomputed_round_key_64[(i-1)*8+1]);
				tmp[ 2] = d_ROUND_ELT1(    sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, precomputed_round_key_64[(i-1)*8+2]);
				tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, precomputed_round_key_64[(i-1)*8+3]);
				tmp[ 4] = d_ROUND_ELT1(    sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, precomputed_round_key_64[(i-1)*8+4]);
				tmp[ 5] = d_ROUND_ELT1(    sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, precomputed_round_key_64[(i-1)*8+5]);
				tmp[ 6] = d_ROUND_ELT1(    sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, precomputed_round_key_64[(i-1)*8+6]);
				tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, precomputed_round_key_64[(i-1)*8+7]);
				TRANSFER(n, tmp);
			}
			#pragma unroll 8
			for (int i=0; i<8; i++)
				state[i] = n[i] ^ hash[i];

			#pragma unroll 6
			for (int i=1; i<7; i++)
				n[i]=vectorize(0);

			n[0] = vectorize(0x80);
			n[7] = vectorize(0x2000000000000);

			#pragma unroll 8
			for (int i=0; i < 8; i++) {
				h[i] = state[i];
				n[i] = n[i] ^ h[i];
			}

	//		#pragma unroll 10
			for (int i=0; i < 10; i++) {
				tmp[ 0] = d_ROUND_ELT1(sharedMemory, h, 0, 7, 6, 5, 4, 3, 2, 1, InitVector_RC[i]);
				tmp[ 1] = d_ROUND_ELT(sharedMemory, h, 1, 0, 7, 6, 5, 4, 3, 2);
				tmp[ 2] = d_ROUND_ELT_LDG(sharedMemory, h, 2, 1, 0, 7, 6, 5, 4, 3);
				tmp[ 3] = d_ROUND_ELT(sharedMemory, h, 3, 2, 1, 0, 7, 6, 5, 4);
				tmp[ 4] = d_ROUND_ELT_LDG(sharedMemory, h, 4, 3, 2, 1, 0, 7, 6, 5);
				tmp[ 5] = d_ROUND_ELT(sharedMemory, h, 5, 4, 3, 2, 1, 0, 7, 6);
				tmp[ 6] = d_ROUND_ELT_LDG(sharedMemory, h, 6, 5, 4, 3, 2, 1, 0, 7);
				tmp[ 7] = d_ROUND_ELT(sharedMemory, h, 7, 6, 5, 4, 3, 2, 1, 0);
				TRANSFER(h, tmp);
				tmp[ 0] = d_ROUND_ELT1(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, tmp[0]);
				tmp[ 1] = d_ROUND_ELT1(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, tmp[1]);
				tmp[ 2] = d_ROUND_ELT1_LDG(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, tmp[2]);
				tmp[ 3] = d_ROUND_ELT1(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, tmp[3]);
				tmp[ 4] = d_ROUND_ELT1(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, tmp[4]);
				tmp[ 5] = d_ROUND_ELT1(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, tmp[5]);
				tmp[ 6] = d_ROUND_ELT1(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, tmp[6]);
				tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, tmp[7]);
				TRANSFER(n, tmp);
			}

			state[0] = xor3x(state[0], n[0], vectorize(0x80));
			state[1] = state[1]^ n[1];
			state[2] = state[2]^ n[2];
			state[3] = state[3]^ n[3];
			state[4] = state[4]^ n[4];
			state[5] = state[5]^ n[5];
			state[6] = state[6]^ n[6];
			state[7] = xor3x(state[7], n[7], vectorize(0x2000000000000));
		}

		uint2 h[8] = {
			{0xC0EE0B30,0x672990AF},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},
			{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828}
		};

		#pragma unroll 8
		for(int i=0;i<8;i++)
			n[i]=hash[i] = state[ i];

		tmp[ 0] = d_ROUND_ELT1(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, h[0]);
		tmp[ 1] = d_ROUND_ELT1_LDG(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, h[1]);
		tmp[ 2] = d_ROUND_ELT1(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, h[2]);
		tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, h[3]);
		tmp[ 4] = d_ROUND_ELT1(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, h[4]);
		tmp[ 5] = d_ROUND_ELT1_LDG(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, h[5]);
		tmp[ 6] = d_ROUND_ELT1(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, h[6]);
		tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, h[7]);
		TRANSFER(n, tmp);
//		#pragma unroll 10
		for (int i=1; i <10; i++){
			tmp[ 0] = d_ROUND_ELT1_LDG(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, precomputed_round_key_64[(i-1)*8+0]);
			tmp[ 1] = d_ROUND_ELT1(    sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, precomputed_round_key_64[(i-1)*8+1]);
			tmp[ 2] = d_ROUND_ELT1(    sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, precomputed_round_key_64[(i-1)*8+2]);
			tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, precomputed_round_key_64[(i-1)*8+3]);
			tmp[ 4] = d_ROUND_ELT1(    sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, precomputed_round_key_64[(i-1)*8+4]);
			tmp[ 5] = d_ROUND_ELT1(    sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, precomputed_round_key_64[(i-1)*8+5]);
			tmp[ 6] = d_ROUND_ELT1(    sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, precomputed_round_key_64[(i-1)*8+6]);
			tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, precomputed_round_key_64[(i-1)*8+7]);
			TRANSFER(n, tmp);
		}

		#pragma unroll 8
		for (int i=0; i<8; i++)
			n[ i] = h[i] = n[i] ^ hash[i];

		uint2 backup = h[ 3];

		n[0]^= vectorize(0x80);
		n[7]^= vectorize(0x2000000000000);

//		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			tmp[ 0] = d_ROUND_ELT1(sharedMemory, h, 0, 7, 6, 5, 4, 3, 2, 1, InitVector_RC[i]);
			tmp[ 1] = d_ROUND_ELT(sharedMemory, h, 1, 0, 7, 6, 5, 4, 3, 2);
			tmp[ 2] = d_ROUND_ELT_LDG(sharedMemory, h, 2, 1, 0, 7, 6, 5, 4, 3);
			tmp[ 3] = d_ROUND_ELT(sharedMemory, h, 3, 2, 1, 0, 7, 6, 5, 4);
			tmp[ 4] = d_ROUND_ELT_LDG(sharedMemory, h, 4, 3, 2, 1, 0, 7, 6, 5);
			tmp[ 5] = d_ROUND_ELT(sharedMemory, h, 5, 4, 3, 2, 1, 0, 7, 6);
			tmp[ 6] = d_ROUND_ELT_LDG(sharedMemory, h, 6, 5, 4, 3, 2, 1, 0, 7);
			tmp[ 7] = d_ROUND_ELT(sharedMemory, h, 7, 6, 5, 4, 3, 2, 1, 0);
			TRANSFER(h, tmp);
			tmp[ 0] = d_ROUND_ELT1(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, tmp[0]);
			tmp[ 1] = d_ROUND_ELT1(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, tmp[1]);
			tmp[ 2] = d_ROUND_ELT1_LDG(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, tmp[2]);
			tmp[ 3] = d_ROUND_ELT1(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, tmp[3]);
			tmp[ 4] = d_ROUND_ELT1(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, tmp[4]);
			tmp[ 5] = d_ROUND_ELT1(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, tmp[5]);
			tmp[ 6] = d_ROUND_ELT1(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, tmp[6]);
			tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, tmp[7]);
			TRANSFER(n, tmp);
		}
		tmp[ 0] = d_ROUND_ELT1(sharedMemory, h, 0, 7, 6, 5, 4, 3, 2, 1, InitVector_RC[8]);
		tmp[ 1] = d_ROUND_ELT(sharedMemory, h, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp[ 2] = d_ROUND_ELT_LDG(sharedMemory, h, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp[ 3] = d_ROUND_ELT(sharedMemory, h, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp[ 4] = d_ROUND_ELT_LDG(sharedMemory, h, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp[ 5] = d_ROUND_ELT(sharedMemory, h, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp[ 6] = d_ROUND_ELT(sharedMemory, h, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp[ 7] = d_ROUND_ELT(sharedMemory, h, 7, 6, 5, 4, 3, 2, 1, 0);
		TRANSFER(h, tmp);
		tmp[ 0] = d_ROUND_ELT1(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, tmp[0]);
		tmp[ 1] = d_ROUND_ELT1(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, tmp[1]);
		tmp[ 2] = d_ROUND_ELT1(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, tmp[2]);
		tmp[ 3] = d_ROUND_ELT1(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, tmp[3]);
		tmp[ 4] = d_ROUND_ELT1(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, tmp[4]);
		tmp[ 5] = d_ROUND_ELT1(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, tmp[5]);
		tmp[ 6] = d_ROUND_ELT1_LDG(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, tmp[6]);
		tmp[ 7] = d_ROUND_ELT1(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, tmp[7]);

		n[ 3] = backup ^ d_ROUND_ELT(sharedMemory,  h, 3, 2, 1, 0, 7, 6, 5, 4)
			^ d_ROUND_ELT(sharedMemory,tmp, 3, 2, 1, 0, 7, 6, 5, 4);

		if(devectorize(n[3]) <= target) {
			uint32_t tmp = atomicExch(&resNonce[0], thread);
			if (tmp != UINT32_MAX)
				resNonce[1] = tmp;
		}

	} // thread < threads
}

/* only for whirlpool algo, no data out!! */
__host__
void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_resNonces, const uint64_t target)
{
	dim3 grid((threads + TPB80-1) / TPB80);
	dim3 block(TPB80);

	hipMemset(d_resNonce[thr_id], 0xff, 2*sizeof(uint32_t));

	oldwhirlpool_gpu_hash_80<<<grid, block>>>(threads, startNounce, d_resNonce[thr_id], target);

	hipMemcpy(h_resNonces, d_resNonce[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (h_resNonces[0] != UINT32_MAX) h_resNonces[0] += startNounce;
	if (h_resNonces[1] != UINT32_MAX) h_resNonces[1] += startNounce;
}

__global__
__launch_bounds__(TPB64,2)
void x15_whirlpool_gpu_hash_64(uint32_t threads, uint64_t *g_hash)
{
	__shared__ uint2 sharedMemory[7][256];

	if (threadIdx.x < 256) {
		const uint2 tmp = __ldg((uint2*)&b0[threadIdx.x]);
		sharedMemory[0][threadIdx.x] = tmp;
		sharedMemory[1][threadIdx.x] = ROL8(tmp);
		sharedMemory[2][threadIdx.x] = ROL16(tmp);
		sharedMemory[3][threadIdx.x] = ROL24(tmp);
		sharedMemory[4][threadIdx.x] = SWAPUINT2(tmp);
		sharedMemory[5][threadIdx.x] = ROR24(tmp);
		sharedMemory[6][threadIdx.x] = ROR16(tmp);
	}

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads){

		uint2 hash[8], n[8], h[ 8];
		uint2 tmp[8] = {
			{0xC0EE0B30,0x672990AF},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},
			{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828},{0x28282828,0x28282828}
		};

		*(uint2x4*)&hash[ 0] = __ldg4((uint2x4*)&g_hash[(thread<<3) + 0]);
		*(uint2x4*)&hash[ 4] = __ldg4((uint2x4*)&g_hash[(thread<<3) + 4]);

		__syncthreads();

		#pragma unroll 8
		for(int i=0;i<8;i++)
			n[i]=hash[i];

		tmp[ 0]^= d_ROUND_ELT(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1);
		tmp[ 1]^= d_ROUND_ELT_LDG(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2);
		tmp[ 2]^= d_ROUND_ELT(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3);
		tmp[ 3]^= d_ROUND_ELT_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4);
		tmp[ 4]^= d_ROUND_ELT(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5);
		tmp[ 5]^= d_ROUND_ELT_LDG(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6);
		tmp[ 6]^= d_ROUND_ELT(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7);
		tmp[ 7]^= d_ROUND_ELT_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0);
		for (int i=1; i <10; i++){
			TRANSFER(n, tmp);
			tmp[ 0] = d_ROUND_ELT1_LDG(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, precomputed_round_key_64[(i-1)*8+0]);
			tmp[ 1] = d_ROUND_ELT1(    sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, precomputed_round_key_64[(i-1)*8+1]);
			tmp[ 2] = d_ROUND_ELT1(    sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, precomputed_round_key_64[(i-1)*8+2]);
			tmp[ 3] = d_ROUND_ELT1_LDG(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, precomputed_round_key_64[(i-1)*8+3]);
			tmp[ 4] = d_ROUND_ELT1(    sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, precomputed_round_key_64[(i-1)*8+4]);
			tmp[ 5] = d_ROUND_ELT1(    sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, precomputed_round_key_64[(i-1)*8+5]);
			tmp[ 6] = d_ROUND_ELT1(    sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, precomputed_round_key_64[(i-1)*8+6]);
			tmp[ 7] = d_ROUND_ELT1_LDG(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, precomputed_round_key_64[(i-1)*8+7]);
		}

		TRANSFER(h, tmp);
		#pragma unroll 8
		for (int i=0; i<8; i++)
			hash[ i] = h[i] = h[i] ^ hash[i];

		#pragma unroll 6
		for (int i=1; i<7; i++)
			n[i]=vectorize(0);

		n[0] = vectorize(0x80);
		n[7] = vectorize(0x2000000000000);

		#pragma unroll 8
		for (int i=0; i < 8; i++) {
			n[i] = n[i] ^ h[i];
		}

//		#pragma unroll 10
		for (int i=0; i < 10; i++) {
			tmp[ 0] = InitVector_RC[i];
			tmp[ 0]^= d_ROUND_ELT(sharedMemory, h, 0, 7, 6, 5, 4, 3, 2, 1);
			tmp[ 1] = d_ROUND_ELT(sharedMemory, h, 1, 0, 7, 6, 5, 4, 3, 2);
			tmp[ 2] = d_ROUND_ELT_LDG(sharedMemory, h, 2, 1, 0, 7, 6, 5, 4, 3);
			tmp[ 3] = d_ROUND_ELT(sharedMemory, h, 3, 2, 1, 0, 7, 6, 5, 4);
			tmp[ 4] = d_ROUND_ELT_LDG(sharedMemory, h, 4, 3, 2, 1, 0, 7, 6, 5);
			tmp[ 5] = d_ROUND_ELT(sharedMemory, h, 5, 4, 3, 2, 1, 0, 7, 6);
			tmp[ 6] = d_ROUND_ELT(sharedMemory, h, 6, 5, 4, 3, 2, 1, 0, 7);
			tmp[ 7] = d_ROUND_ELT(sharedMemory, h, 7, 6, 5, 4, 3, 2, 1, 0);
			TRANSFER(h, tmp);
			tmp[ 0] = d_ROUND_ELT1(sharedMemory,n, 0, 7, 6, 5, 4, 3, 2, 1, tmp[0]);
			tmp[ 1] = d_ROUND_ELT1_LDG(sharedMemory,n, 1, 0, 7, 6, 5, 4, 3, 2, tmp[1]);
			tmp[ 2] = d_ROUND_ELT1(sharedMemory,n, 2, 1, 0, 7, 6, 5, 4, 3, tmp[2]);
			tmp[ 3] = d_ROUND_ELT1(sharedMemory,n, 3, 2, 1, 0, 7, 6, 5, 4, tmp[3]);
			tmp[ 4] = d_ROUND_ELT1_LDG(sharedMemory,n, 4, 3, 2, 1, 0, 7, 6, 5, tmp[4]);
			tmp[ 5] = d_ROUND_ELT1(sharedMemory,n, 5, 4, 3, 2, 1, 0, 7, 6, tmp[5]);
			tmp[ 6] = d_ROUND_ELT1_LDG(sharedMemory,n, 6, 5, 4, 3, 2, 1, 0, 7, tmp[6]);
			tmp[ 7] = d_ROUND_ELT1(sharedMemory,n, 7, 6, 5, 4, 3, 2, 1, 0, tmp[7]);
			TRANSFER(n, tmp);
		}

		hash[0] = xor3x(hash[0], n[0], vectorize(0x80));
		hash[1] = hash[1]^ n[1];
		hash[2] = hash[2]^ n[2];
		hash[3] = hash[3]^ n[3];
		hash[4] = hash[4]^ n[4];
		hash[5] = hash[5]^ n[5];
		hash[6] = hash[6]^ n[6];
		hash[7] = xor3x(hash[7], n[7], vectorize(0x2000000000000));

		*(uint2x4*)&g_hash[(thread<<3)+ 0] = *(uint2x4*)&hash[ 0];
		*(uint2x4*)&g_hash[(thread<<3)+ 4] = *(uint2x4*)&hash[ 4];
	}
}

__host__
static void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	dim3 grid((threads + TPB64-1) / TPB64);
	dim3 block(TPB64);

	x15_whirlpool_gpu_hash_64 <<<grid, block>>> (threads, (uint64_t*)d_hash);
}

__host__
void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	x15_whirlpool_cpu_hash_64(thr_id, threads, d_hash);
}

