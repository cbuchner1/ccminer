#include "hip/hip_runtime.h"
/**
 * SKEIN512 80 + SHA256 64
 * by tpruvot@github - 2015
 */

#include "sph/sph_skein.h"

#include "miner.h"
#include "cuda_helper.h"

#include <openssl/sha.h>

static uint32_t *d_hash[MAX_GPUS];
static __thread bool sm5 = true;

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int swap);

extern void skeincoin_init(int thr_id);
extern void skeincoin_free(int thr_id);
extern void skeincoin_setBlock_80(int thr_id, void *pdata);
extern uint32_t skeincoin_hash_sm5(int thr_id, uint32_t threads, uint32_t startNounce, int swap, uint64_t target64, uint32_t *secNonce);

static __device__ uint32_t sha256_hashTable[] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

static __device__ __constant__ uint32_t sha256_constantTable[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

static __device__ __constant__ uint32_t sha256_endingTable[] = {
	0x80000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000,
	0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000200,
	0x80000000, 0x01400000, 0x00205000, 0x00005088, 0x22000800, 0x22550014, 0x05089742, 0xa0000020,
	0x5a880000, 0x005c9400, 0x0016d49d, 0xfa801f00, 0xd33225d0, 0x11675959, 0xf6e6bfda, 0xb30c1549,
	0x08b2b050, 0x9d7c4c27, 0x0ce2a393, 0x88e6e1ea, 0xa52b4335, 0x67a16f49, 0xd732016f, 0x4eeb2e91,
	0x5dbf55e5, 0x8eee2335, 0xe2bc5ec2, 0xa83f4394, 0x45ad78f7, 0x36f3d0cd, 0xd99c05e8, 0xb0511dc7,
	0x69bc7ac4, 0xbd11375b, 0xe3ba71e5, 0x3b209ff2, 0x18feee17, 0xe25ad9e7, 0x13375046, 0x0515089d,
	0x4f0d0f04, 0x2627484e, 0x310128d2, 0xc668b434, 0x420841cc, 0x62d311b8, 0xe59ba771, 0x85a7a484
};

/* Elementary functions used by SHA256 */
#define SWAB32(x)     cuda_swab32(x)
//#define ROTR32(x,n)   SPH_ROTR32(x,n)

#define R(x, n)       ((x) >> (n))
#define Ch(x, y, z)   ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)  ((x & (y | z)) | (y & z))
#define S0(x)         (ROTR32(x, 2) ^ ROTR32(x, 13) ^ ROTR32(x, 22))
#define S1(x)         (ROTR32(x, 6) ^ ROTR32(x, 11) ^ ROTR32(x, 25))
#define s0(x)         (ROTR32(x, 7) ^ ROTR32(x, 18) ^ R(x, 3))
#define s1(x)         (ROTR32(x, 17) ^ ROTR32(x, 19) ^ R(x, 10))

#define ADVANCED_SHA2

#ifndef ADVANCED_SHA2

/* SHA256 round function */
#define RND(a, b, c, d, e, f, g, h, k) \
	do { \
		t0 = h + S1(e) + Ch(e, f, g) + k; \
		t1 = S0(a) + Maj(a, b, c); \
		d += t0; \
		h  = t0 + t1; \
	} while (0)

/* Adjusted round function for rotating state */
#define RNDr(S, W, i) \
	RND(S[(64 - i) & 7], S[(65 - i) & 7], \
	    S[(66 - i) & 7], S[(67 - i) & 7], \
	    S[(68 - i) & 7], S[(69 - i) & 7], \
	    S[(70 - i) & 7], S[(71 - i) & 7], \
	    W[i] + sha256_constantTable[i])

static __constant__ uint32_t sha256_ending[16] = {
	0x80000000UL, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x200UL
};
__device__
void sha256_transform_gpu(uint32_t *state, uint32_t *message)
{
	uint32_t S[8];
	uint32_t W[64];
	uint32_t t0, t1;

	/* Initialize work variables. */
	for (int i = 0; i < 8; i++) {
		S[i] = state[i];
	}

	for (int i = 0; i < 16; i++) {
		W[i] = message[i];
	}

	for (int i = 16; i < 64; i += 2) {
		W[i] = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i + 1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15];
	}

	/* 3. Mix. */
	#pragma unroll
	for (int i = 0; i < 64; i++) {
		RNDr(S, W, i);
	}

	for (int i = 0; i < 8; i++)
		state[i] += S[i];
}
#endif

#ifdef ADVANCED_SHA2
__device__
void skeincoin_gpu_sha256(uint32_t *message)
{
	uint32_t W1[16];
	uint32_t W2[16];

	uint32_t regs[8];
	uint32_t hash[8];

	// Init with Hash-Table
	#pragma unroll 8
	for (int k=0; k < 8; k++) {
		hash[k] = regs[k] = sha256_hashTable[k];
	}

	#pragma unroll 16
	for (int k = 0; k<16; k++)
		W1[k] = SWAB32(message[k]);

	// Progress W1
	#pragma unroll 16
	for (int j = 0; j<16; j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_constantTable[j] + W1[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int k = 6; k >= 0; k--) regs[k + 1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	// Progress W2...W3

	////// PART 1
	#pragma unroll 2
	for (int j = 0; j<2; j++)
		W2[j] = s1(W1[14 + j]) + W1[9 + j] + s0(W1[1 + j]) + W1[j];
	#pragma unroll 5
	for (int j = 2; j<7; j++)
		W2[j] = s1(W2[j - 2]) + W1[9 + j] + s0(W1[1 + j]) + W1[j];

	#pragma unroll 8
	for (int j = 7; j<15; j++)
		W2[j] = s1(W2[j - 2]) + W2[j - 7] + s0(W1[1 + j]) + W1[j];

	W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

	// Round function
	#pragma unroll 16
	for (int j = 0; j<16; j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_constantTable[j + 16] + W2[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l = 6; l >= 0; l--) regs[l + 1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	////// PART 2
	#pragma unroll 2
	for (int j = 0; j<2; j++)
		W1[j] = s1(W2[14 + j]) + W2[9 + j] + s0(W2[1 + j]) + W2[j];

	#pragma unroll 5
	for (int j = 2; j<7; j++)
		W1[j] = s1(W1[j - 2]) + W2[9 + j] + s0(W2[1 + j]) + W2[j];

	#pragma unroll 8
	for (int j = 7; j<15; j++)
		W1[j] = s1(W1[j - 2]) + W1[j - 7] + s0(W2[1 + j]) + W2[j];

	W1[15] = s1(W1[13]) + W1[8] + s0(W1[0]) + W2[15];

	// Round function
	#pragma unroll 16
	for (int j = 0; j<16; j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_constantTable[j + 32] + W1[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l = 6; l >= 0; l--) regs[l + 1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	////// PART 3
	#pragma unroll 2
	for (int j = 0; j<2; j++)
		W2[j] = s1(W1[14 + j]) + W1[9 + j] + s0(W1[1 + j]) + W1[j];

	#pragma unroll 5
	for (int j = 2; j<7; j++)
		W2[j] = s1(W2[j - 2]) + W1[9 + j] + s0(W1[1 + j]) + W1[j];

	#pragma unroll 8
	for (int j = 7; j<15; j++)
		W2[j] = s1(W2[j - 2]) + W2[j - 7] + s0(W1[1 + j]) + W1[j];

	W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

	// Round function
	#pragma unroll 16
	for (int j = 0; j<16; j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_constantTable[j + 48] + W2[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int l = 6; l >= 0; l--) regs[l + 1] = regs[l];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	#pragma unroll 8
	for (int k = 0; k<8; k++)
		hash[k] += regs[k];

#if 1
	/////
	///// Second Pass (ending)
	/////
	#pragma unroll 8
	for (int k = 0; k<8; k++)
		regs[k] = hash[k];

	// Progress W1
	#pragma unroll 64
	for (int j = 0; j<64; j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_constantTable[j] + sha256_endingTable[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);

		#pragma unroll 7
		for (int k = 6; k >= 0; k--) regs[k + 1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	#pragma unroll 8
	for (int k = 0; k<8; k++)
		hash[k] += regs[k];

	// Final Hash
	#pragma unroll 8
	for (int k = 0; k<8; k++)
		message[k] = SWAB32(hash[k]);
#else
	// sha256_transform only, require an additional sha256_transform_gpu() call
	#pragma unroll 8
	for (int k = 0; k<8; k++)
		message[k] = hash[k];
#endif
}
#endif

__global__
void sha2_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *hashBuffer)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *hash = &hashBuffer[thread << 4];
#ifdef ADVANCED_SHA2
		skeincoin_gpu_sha256(hash);
#else
		uint32_t state[16];
		uint32_t msg[16];
		#pragma unroll
		for (int i = 0; i < 8; i++)
			state[i] = sha256_hashTable[i];

		#pragma unroll
		for (int i = 0; i < 16; i++)
			msg[i] = SWAB32(hash[i]);

		sha256_transform_gpu(state, msg);
		sha256_transform_gpu(state, sha256_ending);

		#pragma unroll
		for (int i = 0; i < 8; i++)
			hash[i] = SWAB32(state[i]);
#endif
	}
}

__host__
void sha2_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHashes)
{
	uint32_t threadsperblock = 128;
	dim3 block(threadsperblock);
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);

	sha2_gpu_hash_64 <<< grid, block >>>(threads, startNounce, d_outputHashes);

	// required once per scan loop to prevent cpu 100% usage (linux)
	MyStreamSynchronize(NULL, 0, thr_id);
}

extern "C" void skeincoinhash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;
	SHA256_CTX sha256;

	uint32_t hash[16];

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, hash);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char *)hash, 64);
	SHA256_Final((unsigned char *)hash, &sha256);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_skeincoin(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];

	sm5 = (device_sm[device_map[thr_id]] >= 500);
	bool checkSecnonce = (have_stratum || have_longpoll) && !sm5;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 20);
	if (init[thr_id]) throughput = min(throughput, (max_nonce - first_nonce));

	uint64_t target64 = 0;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x03;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);

		if (sm5) {
			skeincoin_init(thr_id);
		} else {
			hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput);
			quark_skein512_cpu_init(thr_id, throughput);
			cuda_check_cpu_init(thr_id, throughput);
			CUDA_SAFE_CALL(hipDeviceSynchronize());
		}

		init[thr_id] = true;
	}

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	if (sm5) {
		skeincoin_setBlock_80(thr_id, (void*)endiandata);
		target64 = ((uint64_t*)ptarget)[3];
	} else {
		skein512_cpu_setBlock_80((void*)endiandata);
		cuda_check_cpu_setTarget(ptarget);
	}

	do {
		// Hash with CUDA
		*hashes_done = pdata[19] - first_nonce + throughput;

		if (sm5) {
			/* cuda_skeincoin.cu */
			work->nonces[0] = skeincoin_hash_sm5(thr_id, throughput, pdata[19], 1, target64, &work->nonces[1]);
		} else {
			/* quark/cuda_skein512.cu */
			skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
			sha2_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		}

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			endiandata[19] = swab32(work->nonces[0]);
			skeincoinhash(vhash, endiandata);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (checkSecnonce) {
					work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], work->valid_nonces);
					if (work->nonces[1] != 0) {
						endiandata[19] = swab32(work->nonces[1]);
						skeincoinhash(vhash, endiandata);
						if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
							work->valid_nonces++;
							bn_set_target_ratio(work, vhash, 1);
						}
						pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
					} else {
						pdata[19] = work->nonces[0] + 1;
					}
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor for next scan
				}
				return work->valid_nonces;
			}
			 else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_skeincoin(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	if (sm5)
		skeincoin_free(thr_id);
	else {
		hipFree(d_hash[thr_id]);
		cuda_check_cpu_free(thr_id);
	}

	init[thr_id] = false;

	hipDeviceSynchronize();
}
