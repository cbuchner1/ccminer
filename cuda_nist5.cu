#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "quark/cuda_quark.h"

static uint32_t *d_hash[MAX_GPUS];

// Original nist5hash Funktion aus einem miner Quelltext
extern "C" void nist5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    uint8_t hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_nist5(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int res = 0;

	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << 20); // 256*256*16
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00FF;

	if (!init[thr_id])
	{
		hipDeviceSynchronize();
		hipSetDevice(device_map[thr_id]);

		// Constants copy/init (no device alloc in these algos)
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		// char[64] work space for hashes results
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)64 * throughput));

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

#ifdef USE_STREAMS
	hipStream_t stream[5];
	for (int i = 0; i < 5; i++)
		hipStreamCreate(&stream[i]);
#endif

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			nist5hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash64);
				if (secNonce != 0) {
					be32enc(&endiandata[19], secNonce);
					nist5hash(vhash64, endiandata);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio)
						work_set_target_ratio(work, vhash64);
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				goto out;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

out:
//	*hashes_done = pdata[19] - first_nonce;
#ifdef USE_STREAMS
	for (int i = 0; i < 5; i++)
		hipStreamDestroy(stream[i]);
#endif

	return res;
}

// ressources cleanup
extern "C" void free_nist5(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
