#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void quark_blake512_cpu_init(int thr_id, uint32_t threads);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint32_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_jh512_cpu_init(int thr_id, uint32_t threads);
extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);


// Original nist5hash Funktion aus einem miner Quelltext
extern "C" void nist5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    uint8_t hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_nist5(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  device_intensity(thr_id, __func__, 1 << 20); // 256*256*16
	throughput = min(throughput,  (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00FF;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			nist5hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			}
			else {
				applog(LOG_WARNING, "GPU #%d: result for nonce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
