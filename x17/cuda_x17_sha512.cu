#include "hip/hip_runtime.h"
/*
 * sha-512 cuda kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014 djm34
 *               2016 tpruvot
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 */
#include <stdio.h>

#define NEED_HASH_512

#include "cuda_helper.h"

#define SWAP64(u64) cuda_swab64(u64)

static __constant__ uint64_t c_WB[80];

static const uint64_t WB[80] = {
	0x428A2F98D728AE22, 0x7137449123EF65CD, 0xB5C0FBCFEC4D3B2F, 0xE9B5DBA58189DBBC,
	0x3956C25BF348B538, 0x59F111F1B605D019, 0x923F82A4AF194F9B, 0xAB1C5ED5DA6D8118,
	0xD807AA98A3030242, 0x12835B0145706FBE, 0x243185BE4EE4B28C, 0x550C7DC3D5FFB4E2,
	0x72BE5D74F27B896F, 0x80DEB1FE3B1696B1, 0x9BDC06A725C71235, 0xC19BF174CF692694,
	0xE49B69C19EF14AD2, 0xEFBE4786384F25E3, 0x0FC19DC68B8CD5B5, 0x240CA1CC77AC9C65,
	0x2DE92C6F592B0275, 0x4A7484AA6EA6E483, 0x5CB0A9DCBD41FBD4, 0x76F988DA831153B5,
	0x983E5152EE66DFAB, 0xA831C66D2DB43210, 0xB00327C898FB213F, 0xBF597FC7BEEF0EE4,
	0xC6E00BF33DA88FC2, 0xD5A79147930AA725, 0x06CA6351E003826F, 0x142929670A0E6E70,
	0x27B70A8546D22FFC, 0x2E1B21385C26C926, 0x4D2C6DFC5AC42AED, 0x53380D139D95B3DF,
	0x650A73548BAF63DE, 0x766A0ABB3C77B2A8, 0x81C2C92E47EDAEE6, 0x92722C851482353B,
	0xA2BFE8A14CF10364, 0xA81A664BBC423001, 0xC24B8B70D0F89791, 0xC76C51A30654BE30,
	0xD192E819D6EF5218, 0xD69906245565A910, 0xF40E35855771202A, 0x106AA07032BBD1B8,
	0x19A4C116B8D2D0C8, 0x1E376C085141AB53, 0x2748774CDF8EEB99, 0x34B0BCB5E19B48A8,
	0x391C0CB3C5C95A63, 0x4ED8AA4AE3418ACB, 0x5B9CCA4F7763E373, 0x682E6FF3D6B2B8A3,
	0x748F82EE5DEFB2FC, 0x78A5636F43172F60, 0x84C87814A1F0AB72, 0x8CC702081A6439EC,
	0x90BEFFFA23631E28, 0xA4506CEBDE82BDE9, 0xBEF9A3F7B2C67915, 0xC67178F2E372532B,
	0xCA273ECEEA26619C, 0xD186B8C721C0C207, 0xEADA7DD6CDE0EB1E, 0xF57D4F7FEE6ED178,
	0x06F067AA72176FBA, 0x0A637DC5A2C898A6, 0x113F9804BEF90DAE, 0x1B710B35131C471B,
	0x28DB77F523047D84, 0x32CAAB7B40C72493, 0x3C9EBE0A15C9BEBC, 0x431D67C49C100D4C,
	0x4CC5D4BECB3E42B6, 0x597F299CFC657E2A, 0x5FCB6FAB3AD6FAEC, 0x6C44198C4A475817
};

#define BSG5_0(x) xor3(ROTR64(x,28), ROTR64(x,34), ROTR64(x,39))
#define SSG5_0(x) xor3(ROTR64(x, 1), ROTR64(x ,8), shr_t64(x,7))
#define SSG5_1(x) xor3(ROTR64(x,19), ROTR64(x,61), shr_t64(x,6))

//#define MAJ(X, Y, Z)   (((X) & (Y)) | (((X) | (Y)) & (Z)))
#define MAJ(x, y, z)   andor(x,y,z)

__device__ __forceinline__
uint64_t Tone(uint64_t* K, uint64_t* r, uint64_t* W, const int a, const int i)
{
	//asm("// TONE \n");
	const uint64_t e = r[(a+4) & 7];
	uint64_t BSG51 = xor3(ROTR64(e, 14), ROTR64(e, 18), ROTR64(e, 41));
	const uint64_t f = r[(a+5) & 7];
	const uint64_t g = r[(a+6) & 7];
	uint64_t CHl = ((f ^ g) & e) ^ g; // xandx(e, f, g);
	return (r[(a+7) & 7] + BSG51 + CHl + K[i] + W[i]);
}

#define SHA3_STEP(K, r, W, ord, i) { \
	const int a = (8 - ord) & 7; \
	uint64_t T1 = Tone(K, r, W, a, i); \
	r[(a+3) & 7] += T1; \
	uint64_t T2 = (BSG5_0(r[a]) + MAJ(r[a], r[(a+1) & 7], r[(a+2) & 7])); \
	r[(a+7) & 7] = T1 + T2; \
}

__global__
/*__launch_bounds__(256, 4)*/
void x17_sha512_gpu_hash_64(const uint32_t threads, uint64_t *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint64_t hashPosition = thread;
		uint64_t *pHash = &g_hash[hashPosition*8U];

		uint64_t W[80];
		#pragma unroll
		for (int i = 0; i < 8; i ++) {
			W[i] = SWAP64(pHash[i]);
		}
		W[8] = 0x8000000000000000;

		#pragma unroll 69
		for (int i = 9; i<78; i++) {
			W[i] = 0U;
		}
		W[15] = 0x0000000000000200;

		#pragma unroll 64
		for (int i = 16; i < 80; i ++) {
			W[i] = SSG5_1(W[i-2]) + W[i-7];
			W[i] += SSG5_0(W[i-15]) + W[i-16];
		}

		const uint64_t IV512[8] = {
			0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
			0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
			0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
			0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
		};

		uint64_t r[8];
		#pragma unroll
		for (int i = 0; i < 8; i ++) {
			r[i] = IV512[i];
		}

#if CUDART_VERSION >= 7050
		#pragma unroll 10
#endif
		for (int i = 0; i < 80; i += 8) {
			#pragma unroll
			for (int ord = 0; ord < 8; ord++) {
				SHA3_STEP(c_WB, r, W, ord, i+ord);
			}
		}

		#pragma unroll
		for (int u = 0; u < 4; u ++) {
			pHash[u] = SWAP64(r[u] + IV512[u]);
		}

#ifdef NEED_HASH_512
		#pragma unroll
		for (int u = 4; u < 8; u ++) {
			pHash[u] = SWAP64(r[u] + IV512[u]);
		}
#endif
	}
}

__host__
void x17_sha512_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_WB), WB, 80*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__
void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x17_sha512_gpu_hash_64 <<<grid, block>>> (threads, (uint64_t*)d_hash);
}
