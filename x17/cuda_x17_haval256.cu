#include "hip/hip_runtime.h"
/*
 * haval-256 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014 djm34
 *               2016 tpruvot
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 */
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

#define F1(x6, x5, x4, x3, x2, x1, x0) \
	(((x1) & ((x0) ^ (x4))) ^ ((x2) & (x5)) ^ ((x3) & (x6)) ^ (x0))

#define F2(x6, x5, x4, x3, x2, x1, x0) \
	(((x2) & (((x1) & ~(x3)) ^ ((x4) & (x5)) ^ (x6) ^ (x0))) \
	^ ((x4) & ((x1) ^ (x5))) ^ ((x3 & (x5)) ^ (x0)))

#define F3(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ (x6) ^ (x0))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ (x0))

#define F4(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ ((x4) | (x6)) ^ (x5))) \
	^ ((x4) & ((~(x2) & (x5)) ^ (x1) ^ (x6) ^ (x0))) \
	^ ((x2) & (x6)) ^ (x0))

#define F5(x6, x5, x4, x3, x2, x1, x0) \
	(((x0) & ~(((x1) & (x2) & (x3)) ^ (x5))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ ((x3) & (x6)))

#define FP5_1(x6, x5, x4, x3, x2, x1, x0) \
	F1(x3, x4, x1, x0, x5, x2, x6)
#define FP5_2(x6, x5, x4, x3, x2, x1, x0) \
	F2(x6, x2, x1, x0, x3, x4, x5)
#define FP5_3(x6, x5, x4, x3, x2, x1, x0) \
	F3(x2, x6, x0, x4, x3, x1, x5)
#define FP5_4(x6, x5, x4, x3, x2, x1, x0) \
	F4(x1, x5, x3, x2, x0, x4, x6)
#define FP5_5(x6, x5, x4, x3, x2, x1, x0) \
	F5(x2, x5, x0, x6, x4, x3, x1)

#define STEP(n, p, x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
	uint32_t t = FP ## n ## _ ## p(x6, x5, x4, x3, x2, x1, x0); \
	(x7) = (uint32_t)(ROTR32(t, 7) + ROTR32((x7), 11) + (w) + (c)); \
}

#define PASS1(n, in) { \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 1], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[ 2], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[ 5], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[ 6], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[ 7], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[ 8], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[10], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[11], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[12], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[13], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[14], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[15], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[16], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[17], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[18], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[19], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[20], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[21], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[22], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[23], 0U); \
 \
	STEP(n, 1, s7, s6, s5, s4, s3, s2, s1, s0, in[24], 0U); \
	STEP(n, 1, s6, s5, s4, s3, s2, s1, s0, s7, in[25], 0U); \
	STEP(n, 1, s5, s4, s3, s2, s1, s0, s7, s6, in[26], 0U); \
	STEP(n, 1, s4, s3, s2, s1, s0, s7, s6, s5, in[27], 0U); \
	STEP(n, 1, s3, s2, s1, s0, s7, s6, s5, s4, in[28], 0U); \
	STEP(n, 1, s2, s1, s0, s7, s6, s5, s4, s3, in[29], 0U); \
	STEP(n, 1, s1, s0, s7, s6, s5, s4, s3, s2, in[30], 0U); \
	STEP(n, 1, s0, s7, s6, s5, s4, s3, s2, s1, in[31], 0U); \
}

#define PASS2(n, in) { \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], 0x452821E6); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[14], 0x38D01377); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[26], 0xBE5466CF); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[18], 0x34E90C6C); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[11], 0xC0AC29B7); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[28], 0xC97C50DD); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 7], 0x3F84D5B5); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[16], 0xB5470917); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[ 0], 0x9216D5D9); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[23], 0x8979FB1B); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[20], 0xD1310BA6); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[22], 0x98DFB5AC); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], 0x2FFD72DB); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[10], 0xD01ADFB7); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[ 4], 0xB8E1AFED); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 8], 0x6A267E96); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[30], 0xBA7C9045); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], 0xF12C7F99); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0x24A19947); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[ 9], 0xB3916CF7); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x0801F2E2); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[24], 0x858EFC16); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[29], 0x636920D8); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[ 6], 0x71574E69); \
 \
	STEP(n, 2, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0xA458FEA3); \
	STEP(n, 2, s6, s5, s4, s3, s2, s1, s0, s7, in[12], 0xF4933D7E); \
	STEP(n, 2, s5, s4, s3, s2, s1, s0, s7, s6, in[15], 0x0D95748F); \
	STEP(n, 2, s4, s3, s2, s1, s0, s7, s6, s5, in[13], 0x728EB658); \
	STEP(n, 2, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], 0x718BCD58); \
	STEP(n, 2, s2, s1, s0, s7, s6, s5, s4, s3, in[25], 0x82154AEE); \
	STEP(n, 2, s1, s0, s7, s6, s5, s4, s3, s2, in[31], 0x7B54A41D); \
	STEP(n, 2, s0, s7, s6, s5, s4, s3, s2, s1, in[27], 0xC25A59B5); \
}

#define PASS3(n, in) { \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0x9C30D539); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0x2AF26013); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 4], 0xC5D1B023); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[20], 0x286085F0); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[28], 0xCA417918); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[17], 0xB8DB38EF); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 8], 0x8E79DCB0); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[22], 0x603A180E); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[29], 0x6C9E0E8B); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[14], 0xB01E8A3E); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[25], 0xD71577C1); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[12], 0xBD314B27); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[24], 0x78AF2FDA); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[30], 0x55605C60); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[16], 0xE65525F3); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[26], 0xAA55AB94); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[31], 0x57489862); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[15], 0x63E81440); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[ 7], 0x55CA396A); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[ 3], 0x2AAB10B6); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[ 1], 0xB4CC5C34); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[ 0], 0x1141E8CE); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[18], 0xA15486AF); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[27], 0x7C72E993); \
 \
	STEP(n, 3, s7, s6, s5, s4, s3, s2, s1, s0, in[13], 0xB3EE1411); \
	STEP(n, 3, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], 0x636FBC2A); \
	STEP(n, 3, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0x2BA9C55D); \
	STEP(n, 3, s4, s3, s2, s1, s0, s7, s6, s5, in[10], 0x741831F6); \
	STEP(n, 3, s3, s2, s1, s0, s7, s6, s5, s4, in[23], 0xCE5C3E16); \
	STEP(n, 3, s2, s1, s0, s7, s6, s5, s4, s3, in[11], 0x9B87931E); \
	STEP(n, 3, s1, s0, s7, s6, s5, s4, s3, s2, in[ 5], 0xAFD6BA33); \
	STEP(n, 3, s0, s7, s6, s5, s4, s3, s2, s1, in[ 2], 0x6C24CF5C); \
}

#define PASS4(n, in) { \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[24], 0x7A325381); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 4], 0x28958677); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 0], 0x3B8F4898); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[14], 0x6B4BB9AF); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 2], 0xC4BFE81B); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[ 7], 0x66282193); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[28], 0x61D809CC); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[23], 0xFB21A991); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[26], 0x487CAC60); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[ 6], 0x5DEC8032); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[30], 0xEF845D5D); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[20], 0xE98575B1); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[18], 0xDC262302); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[25], 0xEB651B88); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[19], 0x23893E81); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 3], 0xD396ACC5); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[22], 0x0F6D6FF3); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[11], 0x83F44239); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[31], 0x2E0B4482); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[21], 0xA4842004); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[ 8], 0x69C8F04A); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[27], 0x9E1F9B5E); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[12], 0x21C66842); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[ 9], 0xF6E96C9A); \
 \
	STEP(n, 4, s7, s6, s5, s4, s3, s2, s1, s0, in[ 1], 0x670C9C61); \
	STEP(n, 4, s6, s5, s4, s3, s2, s1, s0, s7, in[29], 0xABD388F0); \
	STEP(n, 4, s5, s4, s3, s2, s1, s0, s7, s6, in[ 5], 0x6A51A0D2); \
	STEP(n, 4, s4, s3, s2, s1, s0, s7, s6, s5, in[15], 0xD8542F68); \
	STEP(n, 4, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x960FA728); \
	STEP(n, 4, s2, s1, s0, s7, s6, s5, s4, s3, in[10], 0xAB5133A3); \
	STEP(n, 4, s1, s0, s7, s6, s5, s4, s3, s2, in[16], 0x6EEF0B6C); \
	STEP(n, 4, s0, s7, s6, s5, s4, s3, s2, s1, in[13], 0x137A3BE4); \
}

#define PASS5(n, in) { \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[27], 0xBA3BF050); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 3], 0x7EFB2A98); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[21], 0xA1F1651D); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[26], 0x39AF0176); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[17], 0x66CA593E); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[11], 0x82430E88); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[20], 0x8CEE8619); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[29], 0x456F9FB4); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[19], 0x7D84A5C3); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 0], 0x3B8B5EBE); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[12], 0xE06F75D8); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[ 7], 0x85C12073); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[13], 0x401A449F); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 8], 0x56C16AA6); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[31], 0x4ED3AA62); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[10], 0x363F7706); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 5], 0x1BFEDF72); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[ 9], 0x429B023D); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[14], 0x37D0D724); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[30], 0xD00A1248); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[18], 0xDB0FEAD3); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 6], 0x49F1C09B); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[28], 0x075372C9); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[24], 0x80991B7B); \
 \
	STEP(n, 5, s7, s6, s5, s4, s3, s2, s1, s0, in[ 2], 0x25D479D8); \
	STEP(n, 5, s6, s5, s4, s3, s2, s1, s0, s7, in[23], 0xF6E8DEF7); \
	STEP(n, 5, s5, s4, s3, s2, s1, s0, s7, s6, in[16], 0xE3FE501A); \
	STEP(n, 5, s4, s3, s2, s1, s0, s7, s6, s5, in[22], 0xB6794C3B); \
	STEP(n, 5, s3, s2, s1, s0, s7, s6, s5, s4, in[ 4], 0x976CE0BD); \
	STEP(n, 5, s2, s1, s0, s7, s6, s5, s4, s3, in[ 1], 0x04C006BA); \
	STEP(n, 5, s1, s0, s7, s6, s5, s4, s3, s2, in[25], 0xC1A94FB6); \
	STEP(n, 5, s0, s7, s6, s5, s4, s3, s2, s1, in[15], 0x409F60C4); \
}

__global__ /* __launch_bounds__(256, 6) */
void x17_haval256_gpu_hash_64(const uint32_t threads, uint64_t *g_hash, const int outlen)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint64_t hashPosition = thread*8U;
		uint64_t *pHash = &g_hash[hashPosition];

		uint32_t s0, s1, s2, s3, s4, s5, s6, s7;
		const uint32_t u0 = s0 = 0x243F6A88;
		const uint32_t u1 = s1 = 0x85A308D3;
		const uint32_t u2 = s2 = 0x13198A2E;
		const uint32_t u3 = s3 = 0x03707344;
		const uint32_t u4 = s4 = 0xA4093822;
		const uint32_t u5 = s5 = 0x299F31D0;
		const uint32_t u6 = s6 = 0x082EFA98;
		const uint32_t u7 = s7 = 0xEC4E6C89;

		union {
			uint32_t h4[16];
			uint64_t h8[8];
		} hash;

		#pragma unroll
		for (int i=0; i<8; i++) {
			hash.h8[i] = pHash[i];
		}

		///////// input big /////////////////////

		uint32_t buf[32];

		#pragma unroll
		for (int i=0; i<16; i++)
			buf[i] = hash.h4[i];

		buf[16] = 0x00000001;

		#pragma unroll
		for (int i=17; i<29; i++)
			buf[i] = 0;

		buf[29] = 0x40290000;
		buf[30] = 0x00000200;
		buf[31] = 0;

		PASS1(5, buf);
		PASS2(5, buf);
		PASS3(5, buf);
		PASS4(5, buf);
		PASS5(5, buf);

		hash.h4[0] = s0 + u0;
		hash.h4[1] = s1 + u1;
		hash.h4[2] = s2 + u2;
		hash.h4[3] = s3 + u3;
		hash.h4[4] = s4 + u4;
		hash.h4[5] = s5 + u5;
		hash.h4[6] = s6 + u6;
		hash.h4[7] = s7 + u7;

		pHash[0] = hash.h8[0];
		pHash[1] = hash.h8[1];
		pHash[2] = hash.h8[2];
		pHash[3] = hash.h8[3];

		if (outlen == 512) {
			pHash[4] = 0; //hash.h8[4];
			pHash[5] = 0; //hash.h8[5];
			pHash[6] = 0; //hash.h8[6];
			pHash[7] = 0; //hash.h8[7];
		}
	}
}

__host__
void x17_haval256_cpu_init(int thr_id, uint32_t threads)
{
}

__host__
void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const int outlen)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x17_haval256_gpu_hash_64 <<<grid, block>>> (threads, (uint64_t*)d_hash, outlen);
}
