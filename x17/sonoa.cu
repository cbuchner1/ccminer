#include "hip/hip_runtime.h"
/**
 * x97 SONO
 **/

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

#define NBN 2

static uint32_t *d_hash[MAX_GPUS];

extern void x16_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x13_fugue512_cpu_free(int thr_id);

extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int flag);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x15_whirlpool_cpu_free(int thr_id);

extern void x17_sha512_cpu_init(int thr_id, uint32_t threads);
extern void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x17_haval256_cpu_init(int thr_id, uint32_t threads);
extern void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const int outlen);

// CPU Hash Validation
extern "C" void sonoa_hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[64];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;


	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*)hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, (const void*)hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_shabal512(&ctx_shabal, (const void*)hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_shabal512(&ctx_shabal, (const void*)hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*)hash);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, (const void*)hash, 64);
	sph_whirlpool_close(&ctx_whirlpool, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_shabal512(&ctx_shabal, (const void*)hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*)hash);

	sph_whirlpool(&ctx_whirlpool, (const void*)hash, 64);
	sph_whirlpool_close(&ctx_whirlpool, (void*)hash);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512, (const void*)hash, 64);
	sph_sha512_close(&ctx_sha512, (void*)hash);

	sph_whirlpool(&ctx_whirlpool, (const void*)hash, 64);
	sph_whirlpool_close(&ctx_whirlpool, (void*)hash);


	sph_bmw512(&ctx_bmw, (const void*)hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*)hash);

	sph_groestl512(&ctx_groestl, (const void*)hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*)hash);

	sph_skein512(&ctx_skein, (const void*)hash, 64);
	sph_skein512_close(&ctx_skein, (void*)hash);

	sph_jh512(&ctx_jh, (const void*)hash, 64);
	sph_jh512_close(&ctx_jh, (void*)hash);

	sph_keccak512(&ctx_keccak, (const void*)hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*)hash);

	sph_luffa512(&ctx_luffa, (const void*)hash, 64);
	sph_luffa512_close(&ctx_luffa, (void*)hash);

	sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*)hash);

	sph_shavite512(&ctx_shavite, (const void*)hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*)hash);

	sph_simd512(&ctx_simd, (const void*)hash, 64);
	sph_simd512_close(&ctx_simd, (void*)hash);

	sph_echo512(&ctx_echo, (const void*)hash, 64);
	sph_echo512_close(&ctx_echo, (void*)hash);

	sph_hamsi512(&ctx_hamsi, (const void*)hash, 64);
	sph_hamsi512_close(&ctx_hamsi, (void*)hash);

	sph_fugue512(&ctx_fugue, (const void*)hash, 64);
	sph_fugue512_close(&ctx_fugue, (void*)hash);

	sph_shabal512(&ctx_shabal, (const void*)hash, 64);
	sph_shabal512_close(&ctx_shabal, (void*)hash);

	sph_whirlpool(&ctx_whirlpool, (const void*)hash, 64);
	sph_whirlpool_close(&ctx_whirlpool, (void*)hash);

	sph_sha512(&ctx_sha512, (const void*)hash, 64);
	sph_sha512_close(&ctx_sha512, (void*)hash);

	sph_haval256_5_init(&ctx_haval);
	sph_haval256_5(&ctx_haval, (const void*)hash, 64);
	sph_haval256_5_close(&ctx_haval, (void*)hash);

	memcpy(output, hash, 32);
}

#define x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash) \
  x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash, order++); \
  if (use_compat_kernels[thr_id]) x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash, order++); \
  else x16_echo512_cpu_hash_64(thr_id, throughput, d_hash)


static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_sonoa(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];

	uint32_t default_throughput = 1 << 18;
	if (device_sm[dev_id] <= 500) default_throughput = 1 << 18;
	else if (device_sm[dev_id] <= 520) default_throughput = 1 << 18;
	else if (device_sm[dev_id]  > 520) default_throughput = (1 << 19) + (1 << 18);

	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	throughput &= 0xFFFFFF00;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);
		if (use_compat_kernels[thr_id])
			x11_echo512_cpu_init(thr_id, throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x17_sha512_cpu_init(thr_id, throughput);
		x17_haval256_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint64_t) * throughput));

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	int warn = 0;
	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x16_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 256); order++;

		*hashes_done = pdata[19] - first_nonce + throughput;

                work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
                if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			sonoa_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					sonoa_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!warn) {
					warn++;
					pdata[19] = work->nonces[0] + 1;
					continue;
				} else {
					if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
					warn = 0;
				}
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

extern "C" void free_sonoa(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x13_fugue512_cpu_free(thr_id);
	x15_whirlpool_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
