#include "hip/hip_runtime.h"
/**
 * Blake2-B CUDA Implementation
 *
 * tpruvot@github July 2016
 *
 */

#include <miner.h>

#include <string.h>
#include <stdint.h>

#include <sph/blake2b.h>

#include <cuda_helper.h>
#include <cuda_vector_uint2x4.h>

#define TPB 512
#define NBN 2

static uint32_t *d_resNonces[MAX_GPUS];

__device__ uint64_t d_data[10];

static __constant__ const int8_t blake2b_sigma[12][16] = {
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
	{ 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  } ,
	{ 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  } ,
	{ 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  } ,
	{ 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 } ,
	{ 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  } ,
	{ 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 } ,
	{ 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 } ,
	{ 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  } ,
	{ 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  } ,
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
	{ 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  }
};

// host mem align
#define A 64

extern "C" void blake2b_hash(void *output, const void *input)
{
	uint8_t _ALIGN(A) hash[32];
	blake2b_ctx ctx;

	blake2b_init(&ctx, 32, NULL, 0);
	blake2b_update(&ctx, input, 80);
	blake2b_final(&ctx, hash);

	memcpy(output, hash, 32);
}

// ----------------------------------------------------------------

__device__ __forceinline__
static void G(const int r, const int i, uint64_t &a, uint64_t &b, uint64_t &c, uint64_t &d, uint64_t const m[16])
{
	a = a + b + m[ blake2b_sigma[r][2*i] ];
	((uint2*)&d)[0] = SWAPUINT2( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
	c = c + d;
	((uint2*)&b)[0] = ROR24( ((uint2*)&b)[0] ^ ((uint2*)&c)[0] );
	a = a + b + m[ blake2b_sigma[r][2*i+1] ];
	((uint2*)&d)[0] = ROR16( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
	c = c + d;
	((uint2*)&b)[0] = ROR2( ((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}

#define ROUND(r) \
	G(r, 0, v[0], v[4], v[ 8], v[12], m); \
	G(r, 1, v[1], v[5], v[ 9], v[13], m); \
	G(r, 2, v[2], v[6], v[10], v[14], m); \
	G(r, 3, v[3], v[7], v[11], v[15], m); \
	G(r, 4, v[0], v[5], v[10], v[15], m); \
	G(r, 5, v[1], v[6], v[11], v[12], m); \
	G(r, 6, v[2], v[7], v[ 8], v[13], m); \
	G(r, 7, v[3], v[4], v[ 9], v[14], m);

// simplified for the last round
__device__ __forceinline__
static void H(const int r, const int i, uint64_t &a, uint64_t &b, uint64_t &c, uint64_t &d, uint64_t const m[16])
{
	a = a + b + m[ blake2b_sigma[r][2*i] ];
	((uint2*)&d)[0] = SWAPUINT2( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
	c = c + d;
	((uint2*)&b)[0] = ROR24( ((uint2*)&b)[0] ^ ((uint2*)&c)[0] );
	a = a + b + m[ blake2b_sigma[r][2*i+1] ];
	((uint2*)&d)[0] = ROR16( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
	c = c + d;
}

// we only check v[0] and v[8]
#define ROUND_F(r) \
	G(r, 0, v[0], v[4], v[ 8], v[12], m); \
	G(r, 1, v[1], v[5], v[ 9], v[13], m); \
	G(r, 2, v[2], v[6], v[10], v[14], m); \
	G(r, 3, v[3], v[7], v[11], v[15], m); \
	G(r, 4, v[0], v[5], v[10], v[15], m); \
	G(r, 5, v[1], v[6], v[11], v[12], m); \
	H(r, 6, v[2], v[7], v[ 8], v[13], m);

__global__
//__launch_bounds__(128, 8) /* to force 64 regs */
void blake2b_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint2 target2)
{
	const uint32_t nonce = (blockDim.x * blockIdx.x + threadIdx.x) + startNonce;
	__shared__ uint64_t s_target;
	if (!threadIdx.x) s_target = devectorize(target2);

	uint64_t m[16];

	m[0] = d_data[0];
	m[1] = d_data[1];
	m[2] = d_data[2];
	m[3] = d_data[3];
	m[4] = d_data[4] | nonce;
	m[5] = d_data[5];
	m[6] = d_data[6];
	m[7] = d_data[7];
	m[8] = d_data[8];
	m[9] = d_data[9];

	m[10] = m[11] = 0;
	m[12] = m[13] = m[14] = m[15] = 0;

	uint64_t v[16] = {
		0x6a09e667f2bdc928, 0xbb67ae8584caa73b, 0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
		0x510e527fade682d1, 0x9b05688c2b3e6c1f, 0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
		0x6a09e667f3bcc908, 0xbb67ae8584caa73b, 0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
		0x510e527fade68281, 0x9b05688c2b3e6c1f, 0xe07c265404be4294, 0x5be0cd19137e2179
	};

	ROUND( 0 );
	ROUND( 1 );
	ROUND( 2 );
	ROUND( 3 );
	ROUND( 4 );
	ROUND( 5 );
	ROUND( 6 );
	ROUND( 7 );
	ROUND( 8 );
	ROUND( 9 );
	ROUND( 10 );
	ROUND_F( 11 );

	uint64_t h64 = cuda_swab64(0x6a09e667f2bdc928 ^ v[0] ^ v[8]);
	if (h64 <= s_target) {
		resNonce[1] = resNonce[0];
		resNonce[0] = nonce;
		s_target = h64;
	}
	// if (!nonce) printf("%016lx ", s_target);
}

__host__
uint32_t blake2b_hash_cuda(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint2 target2, uint32_t &secNonce)
{
	uint32_t resNonces[NBN] = { UINT32_MAX, UINT32_MAX };
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonces[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake2b_gpu_hash <<<grid, block, 8>>> (threads, startNonce, d_resNonces[thr_id], target2);
	hipDeviceSynchronize();

	if (hipSuccess == hipMemcpy(resNonces, d_resNonces[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = resNonces[0];
		secNonce = resNonces[1];
		if (secNonce == result) secNonce = UINT32_MAX;
	}
	return result;
}

__host__
void blake2b_setBlock(uint32_t *data)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 80, 0, hipMemcpyHostToDevice));
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_sia(int thr_id, struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(A) hash[8];
	uint32_t _ALIGN(A) vhashcpu[8];
	uint32_t _ALIGN(A) inputdata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t Htarg = ptarget[7];
	const uint32_t first_nonce = pdata[8];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 28 : 25;
	if (device_sm[dev_id] >= 520 && is_windows()) intensity = 26;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonces[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	memcpy(inputdata, pdata, 80);
	inputdata[11] = 0; // nbits

	const uint2 target = make_uint2(ptarget[6], ptarget[7]);

	blake2b_setBlock(inputdata);

	do {
		work->nonces[0] = blake2b_hash_cuda(thr_id, throughput, pdata[8], target, work->nonces[1]);

		*hashes_done = pdata[8] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			work->valid_nonces = 0;
			inputdata[8] = work->nonces[0];
			blake2b_hash(hash, inputdata);
			if (swab32(hash[0]) <= Htarg) {
				// sia hash target is reversed (start of hash)
				swab256(vhashcpu, hash);
				if (fulltest(vhashcpu, ptarget)) {
					work_set_target_ratio(work, vhashcpu);
					work->valid_nonces++;
					pdata[8] = work->nonces[0] + 1;
				}
			} else {
				gpu_increment_reject(thr_id);
			}

			if (work->nonces[1] != UINT32_MAX) {
				inputdata[8] = work->nonces[1];
				blake2b_hash(hash, inputdata);
				if (swab32(hash[0]) <= Htarg) {
					swab256(vhashcpu, hash);
					if (fulltest(vhashcpu, ptarget)) {
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio[0]) {
							work->sharediff[1] = work->sharediff[0];
							work->shareratio[1] = work->shareratio[0];
							xchg(work->nonces[1], work->nonces[0]);
							work_set_target_ratio(work, vhashcpu);
						} else {
							bn_set_target_ratio(work, vhashcpu, 1);
						}
						work->valid_nonces++;
						pdata[8] = work->nonces[1] + 1;
					}
				} else {
					gpu_increment_reject(thr_id);
				}
			}
			if (work->valid_nonces) {
				return work->valid_nonces;
			}
		}

		if ((uint64_t) throughput + pdata[8] >= max_nonce) {
			pdata[8] = max_nonce;
			break;
		}

		pdata[8] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[8] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_sia(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_resNonces[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
